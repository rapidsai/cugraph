/*
 * Copyright (c) 2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 */

// Pagerank solver
// Author: Alex Fender afender@nvidia.com

#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
#include <string>
 #include <sstream>
#include <iostream>
#include <iomanip>
#include "hipcub/hipcub.hpp"
#include <algorithm>
#include <iomanip>

#include <rmm_utils.h>

#include "utilities/graph_utils.cuh"
#include "utilities/error_utils.h"
#include <cugraph.h>

namespace cugraph
{

#ifdef DEBUG
  #define PR_VERBOSE
#endif
template <typename IndexType, typename ValueType>
bool pagerankIteration(IndexType n, IndexType e, IndexType *cscPtr, IndexType *cscInd,ValueType *cscVal,
                       ValueType alpha, ValueType *a, ValueType *b, float tolerance, int iter, int max_iter,
                       ValueType * &tmp,  void* cub_d_temp_storage, size_t  cub_temp_storage_bytes,
                       ValueType * &pr, ValueType *residual) {
    ValueType  dot_res;
    CUDA_TRY(hipcub::DeviceSpmv::CsrMV(cub_d_temp_storage, cub_temp_storage_bytes, cscVal,
                                    cscPtr, cscInd, tmp, pr, n, n, e));

    scal(n, alpha, pr);
    dot_res = dot( n, a, tmp);
    axpy(n, dot_res,  b,  pr);
    scal(n, (ValueType)1.0/nrm2(n, pr) , pr);
    axpy(n, (ValueType)-1.0,  pr,  tmp);
    *residual = nrm2(n, tmp);
    if (*residual < tolerance)
    {
        scal(n, (ValueType)1.0/nrm1(n,pr), pr);
        return true;
    }
    else
    {
        if (iter< max_iter)
        {
            std::swap(pr, tmp);
        }
        else
        {
           scal(n, (ValueType)1.0/nrm1(n,pr), pr);
        }
        return false;
    }
}

template <typename IndexType, typename ValueType>
int pagerank(IndexType n, IndexType e, IndexType *cscPtr, IndexType *cscInd, ValueType *cscVal,
             IndexType *prsVtx, ValueType *prsVal, IndexType prsLen, bool has_personalization,
             ValueType alpha, ValueType *a, bool has_guess, float tolerance, int max_iter,
             ValueType * &pagerank_vector, ValueType * &residual) {
  int max_it, i = 0 ;
  float tol;
  bool converged = false;
  ValueType randomProbability = static_cast<ValueType>( 1.0/n);
  ValueType *b=0, *tmp=0;
  void* cub_d_temp_storage = NULL;
  size_t cub_temp_storage_bytes = 0;

  if (max_iter > 0)
      max_it = max_iter;
  else
      max_it =  500;

  if (tolerance == 0.0f)
      tol =  1.0E-6f;
  else if (tolerance < 1.0f && tolerance > 0.0f)
      tol = tolerance;
  else
      return -1;

  if (alpha <= 0.0f || alpha >= 1.0f)
      return -1;

  hipStream_t stream{nullptr};

  ALLOC_TRY((void**)&b, sizeof(ValueType) * n, stream);
#if 1/* temporary solution till https://github.com/NVlabs/cub/issues/162 is resolved */
  CUDA_TRY(hipMalloc((void**)&tmp, sizeof(ValueType) * n));
#else
  ALLOC_TRY((void**)&tmp, sizeof(ValueType) * n, stream);
#endif
  cudaCheckError();

  if (!has_guess) {
       fill(n, pagerank_vector, randomProbability);
       fill(n, tmp, randomProbability);
  }
  else {
    copy(n, pagerank_vector, tmp);
  }

  if (has_personalization) {
    ValueType sum = nrm1(prsLen, prsVal);
    if (static_cast<ValueType>(0) == sum) {
      fill(n, b, randomProbability);
    } else {
      scal(n, static_cast<ValueType>(1.0/sum), prsVal);
      fill(n, b, static_cast<ValueType>(0));
      scatter(prsLen, prsVal, b, prsVtx);
    }
  } else {
    fill(n, b, randomProbability);
  }
  update_dangling_nodes(n, a, alpha);

  CUDA_TRY(hipcub::DeviceSpmv::CsrMV(cub_d_temp_storage, cub_temp_storage_bytes, cscVal,
                                  cscPtr, cscInd, tmp, pagerank_vector, n, n, e));
   // Allocate temporary storage
  ALLOC_TRY ((void**)&cub_d_temp_storage, cub_temp_storage_bytes, stream);
  cudaCheckError()
#ifdef PR_VERBOSE
  std::stringstream ss;
  ss.str(std::string());
  ss <<" ------------------PageRank------------------"<< std::endl;
  ss <<" --------------------------------------------"<< std::endl;
  ss << std::setw(10) << "Iteration" << std::setw(15) << "Residual" << std::endl;
  ss <<" --------------------------------------------"<< std::endl;
  std::cout<<ss.str();
#endif

  while (!converged && i < max_it)
  {
      i++;
      converged = pagerankIteration(n, e, cscPtr, cscInd, cscVal,
                                    alpha, a, b, tol, i, max_it, tmp,
                                    cub_d_temp_storage, cub_temp_storage_bytes,
                                    pagerank_vector, residual);
#ifdef PR_VERBOSE
      ss.str(std::string());
      ss << std::setw(10) << i ;
      ss.precision(3);
      ss << std::setw(15) << std::scientific << *residual  << std::endl;
      std::cout<<ss.str();
#endif
  }
  #ifdef PR_VERBOSE
  std::cout <<" --------------------------------------------"<< std::endl;
  #endif
  //printv(n,pagerank_vector,0);

  ALLOC_FREE_TRY(b, stream);
#if 1/* temporary solution till https://github.com/NVlabs/cub/issues/162 is resolved */
  CUDA_TRY(hipFree(tmp));
#else
  ALLOC_FREE_TRY(tmp, stream);
#endif
  ALLOC_FREE_TRY(cub_d_temp_storage, stream);

  return converged ? 0 : 1;
}

//template int pagerank<int, half> (  int n, int e, int *cscPtr, int *cscInd,half *cscVal, half alpha, half *a, bool has_guess, float tolerance, int max_iter, half * &pagerank_vector, half * &residual);
template int pagerank<int, float> (  int n, int e, int *cscPtr, int *cscInd,float *cscVal,
        int *prsVtx, float *prsVal, int prsLen, bool has_personalization,
        float alpha, float *a, bool has_guess, float tolerance, int max_iter, float * &pagerank_vector, float * &residual);
template int pagerank<int, double> (  int n, int e, int *cscPtr, int *cscInd,double *cscVal,
        int *prsVtx,  double *prsVal, int prsLen, bool has_personalization,
        double alpha, double *a, bool has_guess, float tolerance, int max_iter, double * &pagerank_vector, double * &residual);

} //namespace cugraph

template <typename WT>
gdf_error gdf_pagerank_impl (gdf_graph *graph,
                      gdf_column *pagerank,
                      gdf_column *personalization_subset, gdf_column *personalization_values,
                      float alpha = 0.85,
                      float tolerance = 1e-4, int max_iter = 200,
                      bool has_guess = false) {
  bool has_personalization = false;
  int *prsVtx = nullptr;
  WT  *prsVal = nullptr;
  int prsLen = 0;
  GDF_REQUIRE((personalization_subset == nullptr) == (personalization_values == nullptr), GDF_INVALID_API_CALL);
  if (personalization_subset != nullptr) {
    has_personalization = true;
    prsVtx = reinterpret_cast<int*>(personalization_subset->data);
    prsVal = reinterpret_cast<WT* >(personalization_values->data);
    prsLen = reinterpret_cast<int >(personalization_subset->size);
    GDF_REQUIRE(pagerank->dtype == personalization_values->dtype, GDF_DTYPE_MISMATCH);
    GDF_REQUIRE(personalization_subset->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);
    GDF_REQUIRE(personalization_subset->size == personalization_values->size, GDF_COLUMN_SIZE_MISMATCH);
    GDF_REQUIRE(personalization_subset->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
    GDF_REQUIRE(personalization_values->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  }
  GDF_REQUIRE( graph->edgeList != nullptr, GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( graph->edgeList->src_indices->size == graph->edgeList->dest_indices->size, GDF_COLUMN_SIZE_MISMATCH );
  GDF_REQUIRE( graph->edgeList->src_indices->dtype == graph->edgeList->dest_indices->dtype, GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( graph->edgeList->src_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( graph->edgeList->dest_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( pagerank != nullptr , GDF_INVALID_API_CALL );
  GDF_REQUIRE( pagerank->data != nullptr , GDF_INVALID_API_CALL );
  GDF_REQUIRE( pagerank->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( pagerank->size > 0 , GDF_INVALID_API_CALL );

  int m=pagerank->size, nnz = graph->edgeList->src_indices->size, status = 0;
  WT *d_pr, *d_val = nullptr, *d_leaf_vector = nullptr;
  WT res = 1.0;
  WT *residual = &res;

  if (graph->transposedAdjList == nullptr) {
    gdf_add_transposed_adj_list(graph);
  }
  hipStream_t stream{nullptr};
  ALLOC_TRY((void**)&d_leaf_vector, sizeof(WT) * m, stream);
  ALLOC_TRY((void**)&d_val, sizeof(WT) * nnz , stream);
#if 1/* temporary solution till https://github.com/NVlabs/cub/issues/162 is resolved */
  CUDA_TRY(hipMalloc((void**)&d_pr, sizeof(WT) * m));
#else
  ALLOC_TRY((void**)&d_pr, sizeof(WT) * m, stream);
#endif

  //  The templating for HT_matrix_csc_coo assumes that m, nnz and data are all the same type
  cugraph::HT_matrix_csc_coo(m, nnz, (int *)graph->transposedAdjList->offsets->data, (int *)graph->transposedAdjList->indices->data, d_val, d_leaf_vector);

  if (has_guess)
  {
    GDF_REQUIRE( pagerank->data != nullptr, GDF_VALIDITY_UNSUPPORTED );
    cugraph::copy<WT>(m, (WT*)pagerank->data, d_pr);
  }

  status = cugraph::pagerank<int32_t,WT>( m,nnz, (int*)graph->transposedAdjList->offsets->data, (int*)graph->transposedAdjList->indices->data, d_val,
          prsVtx, prsVal, prsLen, has_personalization,
    alpha, d_leaf_vector, has_guess, tolerance, max_iter, d_pr, residual);

  if (status !=0)
    switch ( status ) {
      case -1: std::cerr<< "Error : bad parameters in Pagerank"<<std::endl; return GDF_CUDA_ERROR;
      case 1: std::cerr<< "Warning : Pagerank did not reached the desired tolerance"<<std::endl;  return GDF_CUDA_ERROR;
      default:  std::cerr<< "Pagerank failed"<<std::endl;  return GDF_CUDA_ERROR;
    }

  cugraph::copy<WT>(m, d_pr, (WT*)pagerank->data);

  ALLOC_FREE_TRY(d_val, stream);
#if 1/* temporary solution till https://github.com/NVlabs/cub/issues/162 is resolved */
  CUDA_TRY(hipFree(d_pr));
#else
  ALLOC_FREE_TRY(d_pr, stream);
#endif
  ALLOC_FREE_TRY(d_leaf_vector, stream);

  return GDF_SUCCESS;
}

gdf_error gdf_pagerank(gdf_graph *graph, gdf_column *pagerank,
        gdf_column *personalization_subset, gdf_column *personalization_values,
        float alpha, float tolerance, int max_iter, bool has_guess) {
  //
  //  page rank operates on CSR and can't currently support 64-bit integers.
  //
  //  If csr doesn't exist, create it.  Then check type to make sure it is 32-bit.
  //
  GDF_REQUIRE(graph->adjList != nullptr || graph->edgeList != nullptr, GDF_INVALID_API_CALL);
  gdf_error err = gdf_add_adj_list(graph);
  if (err != GDF_SUCCESS)
    return err;

  GDF_REQUIRE(graph->adjList->offsets->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);
  GDF_REQUIRE(graph->adjList->indices->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);

  switch (pagerank->dtype) {
    case GDF_FLOAT32:   return gdf_pagerank_impl<float>(graph, pagerank,
                                personalization_subset, personalization_values,
                                alpha, tolerance, max_iter, has_guess);
    case GDF_FLOAT64:   return gdf_pagerank_impl<double>(graph, pagerank,
                                personalization_subset, personalization_values,
                                alpha, tolerance, max_iter, has_guess);
    default: return GDF_UNSUPPORTED_DTYPE;
  }
}
