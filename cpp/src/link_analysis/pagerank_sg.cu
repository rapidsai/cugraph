/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <link_analysis/pagerank_impl.cuh>

namespace cugraph {

// SG instantiation
template void pagerank(raft::handle_t const& handle,
                       graph_view_t<int32_t, int32_t, float, true, false> const& graph_view,
                       std::optional<float const*> precomputed_vertex_out_weight_sums,
                       std::optional<int32_t const*> personalization_vertices,
                       std::optional<float const*> personalization_values,
                       std::optional<int32_t> personalization_vector_size,
                       float* pageranks,
                       float alpha,
                       float epsilon,
                       size_t max_iterations,
                       bool has_initial_guess,
                       bool do_expensive_check);

template void pagerank(raft::handle_t const& handle,
                       graph_view_t<int32_t, int32_t, double, true, false> const& graph_view,
                       std::optional<double const*> precomputed_vertex_out_weight_sums,
                       std::optional<int32_t const*> personalization_vertices,
                       std::optional<double const*> personalization_values,
                       std::optional<int32_t> personalization_vector_size,
                       double* pageranks,
                       double alpha,
                       double epsilon,
                       size_t max_iterations,
                       bool has_initial_guess,
                       bool do_expensive_check);

template void pagerank(raft::handle_t const& handle,
                       graph_view_t<int32_t, int64_t, float, true, false> const& graph_view,
                       std::optional<float const*> precomputed_vertex_out_weight_sums,
                       std::optional<int32_t const*> personalization_vertices,
                       std::optional<float const*> personalization_values,
                       std::optional<int32_t> personalization_vector_size,
                       float* pageranks,
                       float alpha,
                       float epsilon,
                       size_t max_iterations,
                       bool has_initial_guess,
                       bool do_expensive_check);

template void pagerank(raft::handle_t const& handle,
                       graph_view_t<int32_t, int64_t, double, true, false> const& graph_view,
                       std::optional<double const*> precomputed_vertex_out_weight_sums,
                       std::optional<int32_t const*> personalization_vertices,
                       std::optional<double const*> personalization_values,
                       std::optional<int32_t> personalization_vector_size,
                       double* pageranks,
                       double alpha,
                       double epsilon,
                       size_t max_iterations,
                       bool has_initial_guess,
                       bool do_expensive_check);

template void pagerank(raft::handle_t const& handle,
                       graph_view_t<int64_t, int64_t, float, true, false> const& graph_view,
                       std::optional<float const*> precomputed_vertex_out_weight_sums,
                       std::optional<int64_t const*> personalization_vertices,
                       std::optional<float const*> personalization_values,
                       std::optional<int64_t> personalization_vector_size,
                       float* pageranks,
                       float alpha,
                       float epsilon,
                       size_t max_iterations,
                       bool has_initial_guess,
                       bool do_expensive_check);

template void pagerank(raft::handle_t const& handle,
                       graph_view_t<int64_t, int64_t, double, true, false> const& graph_view,
                       std::optional<double const*> precomputed_vertex_out_weight_sums,
                       std::optional<int64_t const*> personalization_vertices,
                       std::optional<double const*> personalization_values,
                       std::optional<int64_t> personalization_vector_size,
                       double* pageranks,
                       double alpha,
                       double epsilon,
                       size_t max_iterations,
                       bool has_initial_guess,
                       bool do_expensive_check);

}  // namespace cugraph
