/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cugraph/detail/shuffle_wrappers.hpp>
#include <cugraph/experimental/detail/graph_utils.cuh>
#include <cugraph/partition_manager.hpp>
#include <cugraph/utilities/shuffle_comm.cuh>

#include <raft/random/rng.cuh>

#include <rmm/exec_policy.hpp>

#include <tuple>

namespace cugraph {
namespace detail {

template <typename vertex_t, typename weight_t>
std::tuple<rmm::device_uvector<vertex_t>,
           rmm::device_uvector<vertex_t>,
           std::optional<rmm::device_uvector<weight_t>>>
shuffle_edgelist_by_edge(raft::handle_t const& handle,
                         rmm::device_uvector<vertex_t>& d_edgelist_rows,
                         rmm::device_uvector<vertex_t>& d_edgelist_cols,
                         std::optional<rmm::device_uvector<weight_t>>& d_edgelist_weights,
                         bool store_transposed)
{
  auto& comm               = handle.get_comms();
  auto const comm_size     = comm.get_size();
  auto& row_comm           = handle.get_subcomm(cugraph::partition_2d::key_naming_t().row_name());
  auto const row_comm_size = row_comm.get_size();
  auto& col_comm           = handle.get_subcomm(cugraph::partition_2d::key_naming_t().col_name());
  auto const col_comm_size = col_comm.get_size();

  // TODO:  Make a shuffle_edges and shuffle_vertices out of these...
  rmm::device_uvector<vertex_t> d_rx_edgelist_rows(0, handle.get_stream());
  rmm::device_uvector<vertex_t> d_rx_edgelist_cols(0, handle.get_stream());
  std::optional<rmm::device_uvector<weight_t>> d_rx_edgelist_weights{std::nullopt};
  if (d_edgelist_weights) {
    auto edge_first = thrust::make_zip_iterator(
      thrust::make_tuple(store_transposed ? d_edgelist_cols.begin() : d_edgelist_rows.begin(),
                         store_transposed ? d_edgelist_rows.begin() : d_edgelist_cols.begin(),
                         (*d_edgelist_weights).begin()));

    std::forward_as_tuple(std::tie(store_transposed ? d_rx_edgelist_cols : d_rx_edgelist_rows,
                                   store_transposed ? d_rx_edgelist_rows : d_rx_edgelist_cols,
                                   d_rx_edgelist_weights),
                          std::ignore) =
      cugraph::experimental::groupby_gpuid_and_shuffle_values(
        comm,  // handle.get_comms(),
        edge_first,
        edge_first + d_edgelist_rows.size(),
        [key_func =
           cugraph::experimental::detail::compute_gpu_id_from_edge_t<vertex_t>{
             comm_size, row_comm_size, col_comm_size}] __device__(auto val) {
          return key_func(thrust::get<0>(val), thrust::get<1>(val));
        },
        handle.get_stream());
  } else {
    auto edge_first = thrust::make_zip_iterator(
      thrust::make_tuple(store_transposed ? d_edgelist_cols.begin() : d_edgelist_rows.begin(),
                         store_transposed ? d_edgelist_rows.begin() : d_edgelist_cols.begin()));

    std::forward_as_tuple(std::tie(store_transposed ? d_rx_edgelist_cols : d_rx_edgelist_rows,
                                   store_transposed ? d_rx_edgelist_rows : d_rx_edgelist_cols),
                          std::ignore) =
      cugraph::experimental::groupby_gpuid_and_shuffle_values(
        comm,  // handle.get_comms(),
        edge_first,
        edge_first + d_edgelist_rows.size(),
        [key_func =
           cugraph::experimental::detail::compute_gpu_id_from_edge_t<vertex_t>{
             comm_size, row_comm_size, col_comm_size}] __device__(auto val) {
          return key_func(thrust::get<0>(val), thrust::get<1>(val));
        },
        handle.get_stream());
  }

  return std::make_tuple(
    std::move(d_rx_edgelist_rows), std::move(d_rx_edgelist_cols), std::move(d_rx_edgelist_weights));
}

template std::tuple<rmm::device_uvector<int32_t>,
                    rmm::device_uvector<int32_t>,
                    std::optional<rmm::device_uvector<float>>>
shuffle_edgelist_by_edge(raft::handle_t const& handle,
                         rmm::device_uvector<int32_t>& d_edgelist_rows,
                         rmm::device_uvector<int32_t>& d_edgelist_cols,
                         std::optional<rmm::device_uvector<float>>& d_edgelist_weights,
                         bool store_transposed);

template std::tuple<rmm::device_uvector<int32_t>,
                    rmm::device_uvector<int32_t>,
                    std::optional<rmm::device_uvector<double>>>
shuffle_edgelist_by_edge(raft::handle_t const& handle,
                         rmm::device_uvector<int32_t>& d_edgelist_rows,
                         rmm::device_uvector<int32_t>& d_edgelist_cols,
                         std::optional<rmm::device_uvector<double>>& d_edgelist_weights,
                         bool store_transposed);

template std::tuple<rmm::device_uvector<int64_t>,
                    rmm::device_uvector<int64_t>,
                    std::optional<rmm::device_uvector<float>>>
shuffle_edgelist_by_edge(raft::handle_t const& handle,
                         rmm::device_uvector<int64_t>& d_edgelist_rows,
                         rmm::device_uvector<int64_t>& d_edgelist_cols,
                         std::optional<rmm::device_uvector<float>>& d_edgelist_weights,
                         bool store_transposed);

template std::tuple<rmm::device_uvector<int64_t>,
                    rmm::device_uvector<int64_t>,
                    std::optional<rmm::device_uvector<double>>>
shuffle_edgelist_by_edge(raft::handle_t const& handle,
                         rmm::device_uvector<int64_t>& d_edgelist_rows,
                         rmm::device_uvector<int64_t>& d_edgelist_cols,
                         std::optional<rmm::device_uvector<double>>& d_edgelist_weights,
                         bool store_transposed);

template <typename vertex_t>
rmm::device_uvector<vertex_t> shuffle_vertices(raft::handle_t const& handle,
                                               rmm::device_uvector<vertex_t>& d_vertices)
{
  auto& comm           = handle.get_comms();
  auto const comm_size = comm.get_size();

  rmm::device_uvector<vertex_t> d_rx_vertices(0, handle.get_stream());
  std::tie(d_rx_vertices, std::ignore) = cugraph::experimental::groupby_gpuid_and_shuffle_values(
    comm,  // handle.get_comms(),
    d_vertices.begin(),
    d_vertices.end(),
    [key_func =
       cugraph::experimental::detail::compute_gpu_id_from_vertex_t<vertex_t>{
         comm_size}] __device__(auto val) { return key_func(val); },
    handle.get_stream());

  return d_rx_vertices;
}

template rmm::device_uvector<int32_t> shuffle_vertices(raft::handle_t const& handle,
                                                       rmm::device_uvector<int32_t>& d_vertices);

template rmm::device_uvector<int64_t> shuffle_vertices(raft::handle_t const& handle,
                                                       rmm::device_uvector<int64_t>& d_vertices);

template <typename vertex_t, typename weight_t>
rmm::device_uvector<size_t> groupby_and_count_by_edge(
  raft::handle_t const& handle,
  rmm::device_uvector<vertex_t>& d_edgelist_rows,
  rmm::device_uvector<vertex_t>& d_edgelist_cols,
  std::optional<rmm::device_uvector<weight_t>>& d_edgelist_weights,
  size_t number_of_local_adj_matrix_partitions)
{
  auto& comm               = handle.get_comms();
  auto const comm_size     = comm.get_size();
  auto const comm_rank     = comm.get_rank();
  auto& row_comm           = handle.get_subcomm(cugraph::partition_2d::key_naming_t().row_name());
  auto const row_comm_size = row_comm.get_size();
  auto const row_comm_rank = row_comm.get_rank();
  auto& col_comm           = handle.get_subcomm(cugraph::partition_2d::key_naming_t().col_name());
  auto const col_comm_size = col_comm.get_size();
  auto const col_comm_rank = col_comm.get_rank();

  auto local_partition_id_op =
    [comm_size,
     key_func = cugraph::experimental::detail::compute_partition_id_from_edge_t<vertex_t>{
       comm_size, row_comm_size, col_comm_size}] __device__(auto pair) {
      return key_func(thrust::get<0>(pair), thrust::get<1>(pair)) /
             comm_size;  // global partition id to local partition id
    };

  auto pair_first =
    thrust::make_zip_iterator(thrust::make_tuple(d_edgelist_rows.begin(), d_edgelist_cols.begin()));

  return d_edgelist_weights
           ? cugraph::experimental::groupby_and_count(pair_first,
                                                      pair_first + d_edgelist_rows.size(),
                                                      d_edgelist_weights->begin(),
                                                      local_partition_id_op,
                                                      number_of_local_adj_matrix_partitions,
                                                      handle.get_stream())
           : cugraph::experimental::groupby_and_count(pair_first,
                                                      pair_first + d_edgelist_rows.size(),
                                                      local_partition_id_op,
                                                      number_of_local_adj_matrix_partitions,
                                                      handle.get_stream());
}

template rmm::device_uvector<size_t> groupby_and_count_by_edge(
  raft::handle_t const& handle,
  rmm::device_uvector<int32_t>& d_edgelist_rows,
  rmm::device_uvector<int32_t>& d_edgelist_cols,
  std::optional<rmm::device_uvector<float>>& d_edgelist_weights,
  size_t number_of_local_adj_matrix_partitions);

template rmm::device_uvector<size_t> groupby_and_count_by_edge(
  raft::handle_t const& handle,
  rmm::device_uvector<int32_t>& d_edgelist_rows,
  rmm::device_uvector<int32_t>& d_edgelist_cols,
  std::optional<rmm::device_uvector<double>>& d_edgelist_weights,
  size_t number_of_local_adj_matrix_partitions);

template rmm::device_uvector<size_t> groupby_and_count_by_edge(
  raft::handle_t const& handle,
  rmm::device_uvector<int64_t>& d_edgelist_rows,
  rmm::device_uvector<int64_t>& d_edgelist_cols,
  std::optional<rmm::device_uvector<float>>& d_edgelist_weights,
  size_t number_of_local_adj_matrix_partitions);

template rmm::device_uvector<size_t> groupby_and_count_by_edge(
  raft::handle_t const& handle,
  rmm::device_uvector<int64_t>& d_edgelist_rows,
  rmm::device_uvector<int64_t>& d_edgelist_cols,
  std::optional<rmm::device_uvector<double>>& d_edgelist_weights,
  size_t number_of_local_adj_matrix_partitions);

}  // namespace detail
}  // namespace cugraph
