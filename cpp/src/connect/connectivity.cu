#include "hip/hip_runtime.h"
#include "weak_cc.cuh"

#include "utilities/graph_utils.cuh"
#include "utilities/error_utils.h"
#include <cugraph.h>
#include <algo_types.h>

#include <iostream>
#include <type_traits>
#include <cstdint>

//
/**
 * @brief Compute connected components. 
 * The weak version was imported from cuML.
 * This implementation comes from [1] and solves component labeling problem in
 * parallel on CSR-indexes based upon the vertex degree and adjacency graph.
 *
 * [1] Hawick, K.A et al, 2010. "Parallel graph component labelling with GPUs and CUDA"
 *
 * @tparam Type the numeric type of non-floating point elements
 * @tparam TPB_X the threads to use per block when configuring the kernel
 * @param graph input graph; assumed undirected for weakly CC [in]
 * @param labels gdf_column for the output labels [out]
 * @param connectivity_type 0=WEAK; 1=STRONG
 * @param stream the cuda stream
 */
template<typename IndexT,
         int TPB_X = 32>
std::enable_if_t<std::is_signed<IndexT>::value,gdf_error>
gdf_connected_components_impl(gdf_graph *graph,
                              gdf_column *labels,
                              cugraph_connect_t connectivity_type,
                              hipStream_t stream)
{
  static auto row_offsets_ = [](const gdf_graph* G){
    return static_cast<const IndexT*>(G->adjList->offsets->data);
  };

  static auto col_indices_ = [](const gdf_graph* G){
    return static_cast<const IndexT*>(G->adjList->indices->data);
  };

  static auto nrows_ = [](const gdf_graph* G){
    return G->adjList->offsets->size - 1;
  };

  static auto nnz_ = [](const gdf_graph* G){
    return G->adjList->indices->size;
  };


  GDF_REQUIRE(graph != nullptr, GDF_INVALID_API_CALL);
  std::cout<<"Passed 1.\n";

  if( !graph->adjList )
    std::cerr<<"########### graph->adjlist nullptr.\n";
  
  GDF_REQUIRE(graph->adjList != nullptr, GDF_INVALID_API_CALL);
  std::cout<<"Passed 2.\n";
  
  GDF_REQUIRE(row_offsets_(graph) != nullptr, GDF_INVALID_API_CALL);
  std::cout<<"Passed 3.\n";

  GDF_REQUIRE(col_indices_(graph) != nullptr, GDF_INVALID_API_CALL);
  std::cout<<"Passed 4.\n";
  
  GDF_REQUIRE(labels != nullptr, GDF_INVALID_API_CALL);
  std::cout<<"Passed 5.\n";
  
  GDF_REQUIRE(labels->data != nullptr, GDF_INVALID_API_CALL);
  std::cout<<"Passed 6.\n";
  
  auto type_id = graph->adjList->offsets->dtype;
  GDF_REQUIRE( type_id == GDF_INT32 || type_id == GDF_INT64, GDF_UNSUPPORTED_DTYPE);

  std::cout<<"Passed 7.\n";
  
  GDF_REQUIRE( type_id == graph->adjList->indices->dtype, GDF_UNSUPPORTED_DTYPE);

  std::cout<<"Passed 8.\n";
  
  //TODO: relax this requirement:
  //
  GDF_REQUIRE( type_id == labels->dtype, GDF_UNSUPPORTED_DTYPE);

  std::cout<<"Passed 9.\n";

  //bool flag_dir = graph->prop->directed;//useless, for the time being...
  //TODO: direction_checker() to se this flag correctly; prop is not even allocated!

  std::cout<<"Passed 10.\n";
  
  if( connectivity_type == CUGRAPH_WEAK )
    {
      //check if graph is undirected; return w/ error, if not?
      //Yes, for now; in the future we may remove this constraint; 
      //
      //GDF_REQUIRE(flag_dir == false, GDF_INVALID_API_CALL);//useless check
      
      IndexT* p_d_labels = static_cast<IndexT*>(labels->data);
      const IndexT* p_d_row_offsets = row_offsets_(graph);
      const IndexT* p_d_col_ind = col_indices_(graph);

      IndexT nnz = nnz_(graph);
      IndexT nrows = nrows_(graph);

      std::cout<<"############## "
               <<"nrows = "<<nrows
               <<"; nnz = "<<nnz
               <<"p_d_labels valid: "<<(p_d_labels != nullptr)
               <<"p_d_row_offsets valid: "<<(p_d_row_offsets != nullptr)
               <<"p_d_col_ind valid: " << (p_d_col_ind != nullptr) <<"\n";
      MLCommon::Sparse::weak_cc_entry<IndexT, TPB_X>(p_d_labels,
                                                     p_d_row_offsets,
                                                     p_d_col_ind,
                                                     nnz,
                                                     nrows,
                                                     stream);

    }
  else
    {
      //dump error message and return unsupported, for now:
      //
      std::cerr<<"ERROR: Feature not supported, yet;"
               <<" at: " << __FILE__ << ":" << __LINE__ << std::endl;
      
      return GDF_INVALID_API_CALL;//for now...
    }
  return GDF_SUCCESS;
}

/**
 * @brief Compute connected components. 
 * The weak version was imported from cuML.
 * This implementation comes from [1] and solves component labeling problem in
 * parallel on CSR-indexes based upon the vertex degree and adjacency graph.
 *
 * [1] Hawick, K.A et al, 2010. "Parallel graph component labelling with GPUs and CUDA"
 *
 
 * @param graph input graph; assumed undirected for weakly CC [in]
 * @param labels gdf_column for the output labels [out]
 * @param connectivity_type 0=WEAK; 1=STRONG
 */
gdf_error gdf_connected_components(gdf_graph *graph,
                                   gdf_column *labels,
                                   cugraph_connect_t connectivity_type)
{
  hipStream_t stream{nullptr};

  std::cout<<"############# reached here...; type is int32: "<<(labels->dtype==GDF_INT32) << "\n";
  
  switch( labels->dtype )//currently graph's row offsets, col_indices and labels are same type; that may change in the future
    {
    case GDF_INT32:
      return gdf_connected_components_impl<int32_t>(graph, labels, connectivity_type, stream);
      //    case GDF_INT64:
      //return gdf_connected_components_impl<int64_t>(graph, labels, connectivity_type, stream);
      // PROBLEM: relies on atomicMin(), which won't work w/ int64_t
      // should work with `unsigned long long` but using signed `Type`'s
      //(initialized to `-1`)
    default:
      break;//warning eater
    }
  return GDF_UNSUPPORTED_DTYPE;
}
