#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
/** ---------------------------------------------------------------------------*
 * @brief Functions for computing the two hop neighbor pairs of a graph
 *
 * @file two_hop_neighbors.cu
 * ---------------------------------------------------------------------------**/

#include <rmm/thrust_rmm_allocator.h>
#include <rmm_utils.h>
#include <algorithms.hpp>
#include <graph.hpp>
#include "two_hop_neighbors.cuh"
#include "utilities/error_utils.h"

#include <thrust/execution_policy.h>
#include <thrust/scan.h>
#include <thrust/transform.h>

namespace cugraph {

template <typename VT, typename ET, typename WT>
ET get_two_hop_neighbors(experimental::GraphCSR<VT, ET, WT> const &graph, VT **first, VT **second)
{
  hipStream_t stream{nullptr};

  rmm::device_vector<ET> exsum_degree(graph.number_of_edges + 1);
  ET *d_exsum_degree = exsum_degree.data().get();

  // Find the degree of the out vertex of each edge
  degree_iterator<ET> deg_it(graph.offsets);
  deref_functor<degree_iterator<ET>, ET> deref(deg_it);
  exsum_degree[0] = ET{0};
  thrust::transform(rmm::exec_policy(stream)->on(stream),
                    graph.indices,
                    graph.indices + graph.number_of_edges,
                    d_exsum_degree + 1,
                    deref);

  // Take the inclusive sum of the degrees
  thrust::inclusive_scan(rmm::exec_policy(stream)->on(stream),
                         d_exsum_degree + 1,
                         d_exsum_degree + graph.number_of_edges + 1,
                         d_exsum_degree + 1);

  // Copy out the last value to get the size of scattered output
  ET output_size = exsum_degree[graph.number_of_edges];

  // Allocate memory for the scattered output
  rmm::device_vector<VT> first_pair(output_size);
  rmm::device_vector<VT> second_pair(output_size);

  VT *d_first_pair  = first_pair.data().get();
  VT *d_second_pair = second_pair.data().get();

  // Figure out number of blocks and allocate memory for block bucket offsets
  ET num_blocks = (output_size + TWO_HOP_BLOCK_SIZE - 1) / TWO_HOP_BLOCK_SIZE;
  rmm::device_vector<ET> block_bucket_offsets(num_blocks + 1);

  ET *d_block_bucket_offsets = block_bucket_offsets.data().get();

  // Compute the block bucket offsets
  dim3 grid, block;
  block.x = 512;
  grid.x  = min((ET)MAXBLOCKS, (num_blocks / 512) + 1);
  compute_bucket_offsets_kernel<<<grid, block, 0, nullptr>>>(
    d_exsum_degree, d_block_bucket_offsets, graph.number_of_edges, output_size);

  block_bucket_offsets[num_blocks] = graph.number_of_edges;

  // Scatter the expanded edge lists into temp space
  grid.x = min((ET)MAXBLOCKS, num_blocks);
  scatter_expand_kernel<<<grid, block, 0, nullptr>>>(d_exsum_degree,
                                                     graph.indices,
                                                     graph.offsets,
                                                     d_block_bucket_offsets,
                                                     graph.number_of_vertices,
                                                     output_size,
                                                     num_blocks,
                                                     d_first_pair,
                                                     d_second_pair);

  // TODO:  This would be faster in a hash table (no sorting), unless there's
  //        some reason that the result has to be sorted
  // Remove duplicates and self pairings
  auto tuple_start = thrust::make_zip_iterator(thrust::make_tuple(d_first_pair, d_second_pair));
  auto tuple_end   = tuple_start + output_size;
  thrust::sort(rmm::exec_policy(stream)->on(stream), tuple_start, tuple_end);
  tuple_end = thrust::copy_if(rmm::exec_policy(stream)->on(stream),
                              tuple_start,
                              tuple_end,
                              tuple_start,
                              self_loop_flagger<VT>());
  tuple_end = thrust::unique(rmm::exec_policy(stream)->on(stream), tuple_start, tuple_end);

  // Get things ready to return
  ET outputSize = tuple_end - tuple_start;

  ALLOC_TRY(first, sizeof(VT) * outputSize, nullptr);
  ALLOC_TRY(second, sizeof(VT) * outputSize, nullptr);
  hipMemcpy(*first, d_first_pair, sizeof(VT) * outputSize, hipMemcpyDefault);
  hipMemcpy(*second, d_second_pair, sizeof(VT) * outputSize, hipMemcpyDefault);

  return outputSize;
}

template int get_two_hop_neighbors(experimental::GraphCSR<int, int, float> const &, int **, int **);

template int64_t get_two_hop_neighbors(experimental::GraphCSR<int32_t, int64_t, float> const &,
                                       int32_t **,
                                       int32_t **);

}  // namespace cugraph
