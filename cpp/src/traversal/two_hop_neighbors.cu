#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
/** ---------------------------------------------------------------------------*
 * @brief Functions for computing the two hop neighbor pairs of a graph
 *
 * @file two_hop_neighbors.cu
 * ---------------------------------------------------------------------------**/

#include "two_hop_neighbors.cuh"
#include <cugraph/algorithms.hpp>
#include <cugraph/legacy/graph.hpp>
#include <cugraph/utilities/error.hpp>
#include <rmm/device_vector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/scan.h>
#include <thrust/transform.h>

namespace cugraph {

template <typename VT, typename ET, typename WT>
std::unique_ptr<legacy::GraphCOO<VT, ET, WT>> get_two_hop_neighbors(
  legacy::GraphCSRView<VT, ET, WT> const& graph)
{
  hipStream_t stream{nullptr};

  rmm::device_vector<ET> exsum_degree(graph.number_of_edges + 1);
  ET* d_exsum_degree = exsum_degree.data().get();

  // Find the degree of the out vertex of each edge
  degree_iterator<ET> deg_it(graph.offsets);
  deref_functor<degree_iterator<ET>, ET> deref(deg_it);
  exsum_degree[0] = ET{0};
  thrust::transform(rmm::exec_policy(stream),
                    graph.indices,
                    graph.indices + graph.number_of_edges,
                    d_exsum_degree + 1,
                    deref);

  // Take the inclusive sum of the degrees
  thrust::inclusive_scan(rmm::exec_policy(stream),
                         d_exsum_degree + 1,
                         d_exsum_degree + graph.number_of_edges + 1,
                         d_exsum_degree + 1);

  // Copy out the last value to get the size of scattered output
  ET output_size = exsum_degree[graph.number_of_edges];

  // Allocate memory for the scattered output
  rmm::device_vector<VT> first_pair(output_size);
  rmm::device_vector<VT> second_pair(output_size);

  VT* d_first_pair  = first_pair.data().get();
  VT* d_second_pair = second_pair.data().get();

  // Figure out number of blocks and allocate memory for block bucket offsets
  ET num_blocks = (output_size + TWO_HOP_BLOCK_SIZE - 1) / TWO_HOP_BLOCK_SIZE;
  rmm::device_vector<ET> block_bucket_offsets(num_blocks + 1);

  ET* d_block_bucket_offsets = block_bucket_offsets.data().get();

  // Compute the block bucket offsets
  dim3 grid, block;
  block.x = 512;
  grid.x  = min((ET)MAXBLOCKS, (num_blocks / 512) + 1);
  compute_bucket_offsets_kernel<<<grid, block, 0, nullptr>>>(
    d_exsum_degree, d_block_bucket_offsets, graph.number_of_edges, output_size);

  block_bucket_offsets[num_blocks] = graph.number_of_edges;

  // Scatter the expanded edge lists into temp space
  grid.x = min((ET)MAXBLOCKS, num_blocks);
  scatter_expand_kernel<<<grid, block, 0, nullptr>>>(d_exsum_degree,
                                                     graph.indices,
                                                     graph.offsets,
                                                     d_block_bucket_offsets,
                                                     graph.number_of_vertices,
                                                     output_size,
                                                     num_blocks,
                                                     d_first_pair,
                                                     d_second_pair);

  // TODO:  This would be faster in a hash table (no sorting), unless there's
  //        some reason that the result has to be sorted
  // Remove duplicates and self pairings
  auto tuple_start = thrust::make_zip_iterator(thrust::make_tuple(d_first_pair, d_second_pair));
  auto tuple_end   = tuple_start + output_size;
  thrust::sort(rmm::exec_policy(stream), tuple_start, tuple_end);
  tuple_end = thrust::copy_if(
    rmm::exec_policy(stream), tuple_start, tuple_end, tuple_start, self_loop_flagger<VT>());
  tuple_end = thrust::unique(rmm::exec_policy(stream), tuple_start, tuple_end);

  // Get things ready to return
  ET outputSize = tuple_end - tuple_start;

  auto result =
    std::make_unique<legacy::GraphCOO<VT, ET, WT>>(graph.number_of_vertices, outputSize, false);

  hipMemcpy(result->src_indices(), d_first_pair, sizeof(VT) * outputSize, hipMemcpyDefault);
  hipMemcpy(result->dst_indices(), d_second_pair, sizeof(VT) * outputSize, hipMemcpyDefault);

  return result;
}

template std::unique_ptr<legacy::GraphCOO<int, int, float>> get_two_hop_neighbors(
  legacy::GraphCSRView<int, int, float> const&);

template std::unique_ptr<legacy::GraphCOO<int, int, double>> get_two_hop_neighbors(
  legacy::GraphCSRView<int, int, double> const&);

}  // namespace cugraph
