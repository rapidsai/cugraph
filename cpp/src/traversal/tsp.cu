#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <converters/COOtoCSR.cuh>
#include "utilities/graph_utils.cuh"

#include "tsp.hpp"
#include "tsp_kernels.hpp"
#include "tsp_knn.hpp"
#include "tsp_utils.hpp"

namespace cugraph {
namespace detail {

TSP::TSP(const raft::handle_t &handle,
         int *route,
         const float *x_pos,
         const float *y_pos,
         const int nodes,
         const int restarts,
         const int k,
         const bool verbose)
  : handle_(handle), route_(route), x_pos_(x_pos), y_pos_(y_pos), nodes_(nodes),
  restarts_(restarts), k_(k), verbose_(verbose)
{
  stream_      = handle_.get_stream();
  max_blocks_  = handle_.get_device_properties().maxGridSize[0];
  max_threads_ = handle_.get_device_properties().maxThreadsPerBlock;
  sm_count_    = handle_.get_device_properties().multiProcessorCount;
  // how large a grid we want to run, this is fixed
  restart_batch_ = 4096;
}

void TSP::allocate()
{
  // Scalars
  mylock_scalar_.set_value(1, stream_);
  n_climbs_scalar_.set_value(1, stream_);
  best_tour_scalar_.set_value(1, stream_);

  mylock_    = mylock_scalar_.data();
  n_climbs_  = n_climbs_scalar_.data();
  best_tour_ = best_tour_scalar_.data();

  // Vectors
  neighbors_vec_.resize(k_ * nodes_);
  work_vec_.resize(4 * restart_batch_ * ((3 * nodes_ + 2 + 31) / 32 * 32));

  neighbors_ = neighbors_vec_.data().get();
  work_      = work_vec_.data().get();
}

float TSP::compute()
{
  int num_graphs       = 1;
  float valid_coo_dist = 0.f;

  int num_restart_batch_es = (restarts_ + restart_batch_ - 1) / restart_batch_;
  int restart_resid        = restarts_ - (num_restart_batch_es - 1) * restart_batch_;
  if (verbose_) {
    printf(" doing %d batches of size %d, with %d tail \n",
        num_restart_batch_es - 1,
        restart_batch_,
        restart_resid);
    printf("configuration: %d nodes, %d restart\n", nodes_, restarts_);
  }
  // Tell the cache how we want it to behave
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(simulOpt), hipFuncCachePreferEqual);

  int threads = best_thread_count(nodes_);
  if (verbose_)
    printf(" calculated best thread number = %d\n", threads);
  // pre-allocate workspace for climbs, each block needs a separate permutation space and search
  // buffer

  float *pos = (float *)malloc(sizeof(float) * (nodes_ + 1) * 2);
  if (pos == NULL) {
    fprintf(stderr, "cannot allocate pos\n");
    exit(-1);
  }

  int *offsets = (int *)malloc((sizeof(int) * (2)));
  offsets[0]   = 0;
  offsets[1]   = nodes_;

  for (int g = 0; g < num_graphs; g++) {
    int global_best = INT_MAX;
    float *soln     = NULL;
    int best        = 0;

    if (verbose_)
      printf("optimizing graph %d kswap = %d \n", g, kswaps);
    for (int b = 0; b < num_restart_batch_es; b++) {
      Init<<<1, 1, 0, stream_>>>(mylock_, n_climbs_, best_tour_);
      CHECK_CUDA(stream_);

      if (b == num_restart_batch_es - 1) restart_batch_ = restart_resid;

      simulOpt<<<restart_batch_, threads, sizeof(int) * threads, stream_>>>(mylock_,
                                                                            n_climbs_,
                                                                            best_tour_,
                                                                            k_,
                                                                            nodes_,
                                                                            neighbors_,
                                                                            x_pos_ + offsets[g],
                                                                            y_pos_ + offsets[g],
                                                                            work_);
      CHECK_CUDA(stream_);
      hipDeviceSynchronize();

      CUDA_TRY(hipMemcpy(&best, best_tour_, sizeof(int), hipMemcpyDeviceToHost));
      hipDeviceSynchronize();
      if (verbose_)
        printf("best reported by kernel = %d\n", best);

      if (best < global_best) {
        global_best = best;
        CUDA_TRY(hipMemcpyFromSymbol(&soln, HIP_SYMBOL(best_soln), sizeof(void *)));
        hipDeviceSynchronize();

        CUDA_TRY(hipMemcpy(pos, soln, sizeof(float) * (nodes_ + 1) * 2, hipMemcpyDeviceToHost));
        hipDeviceSynchronize();
      }
    }

    if (verbose_)
      printf("Optimized tour length = %d\n", global_best);

      for (int i = 0; i < nodes_; i++) {
        if (verbose_)
          printf("%.1f %.1f\n", pos[i], pos[i + nodes_ + 1]);
        valid_coo_dist += cpudist(i, i + 1);
      }
  }
  return valid_coo_dist;
}

void TSP::knn()
{
  int numpackages  = nodes_;
  int *neighbors_h = (int *)malloc(k_ * nodes_ * sizeof(int));
  float *input_x_h = (float *)malloc(nodes_ * sizeof(float));
  float *input_y_h = (float *)malloc(nodes_ * sizeof(float));
  CUDA_TRY(hipMemcpy(input_x_h, x_pos_, sizeof(float) * nodes_, hipMemcpyDeviceToHost));
  CUDA_TRY(hipMemcpy(input_y_h, y_pos_, sizeof(float) * nodes_, hipMemcpyDeviceToHost));

  // re-scale arbitrary inputs to fit inside (0,1024)x(0,1024) box
  float xmin = 1e6;
  float xmax = -1e6;
  float ymin = 1e6;
  float ymax = -1e6;
  for (int np = 0; np < numpackages; np++) {
    float xc = input_x_h[np];
    if (xc < xmin) xmin = xc;
    if (xc > xmax) xmax = xc;
    float yc = input_y_h[np];
    if (yc < ymin) ymin = yc;
    if (yc > ymax) ymax = yc;
  }

  // Calculate affine transform A*x + b so that all (x,y) pairs lie in (0,1024)x(0,1024)
  // also calculate inverse so we can recover the original coords
  // We need to use the same scaling for x and y so the Euclidean distance is just scaled
  // otherwise we can get bad neighbors as a result of the scaling
  float forward_b = max(-xmin, -ymin);
  float forward_A = 1024. / max((xmax + forward_b), (ymax + forward_b));
  float back_A    = 1. / forward_A;
  float back_b    = -forward_b;
  affineTrans(numpackages, 1, input_x_h, forward_A, forward_b);
  affineTrans(numpackages, 1, input_y_h, forward_A, forward_b);

  findKneighbors(numpackages, k_, &input_x_h, &input_y_h, &neighbors_h, 0);

  // Reverse the transform
  affineTrans(numpackages, 0, input_x_h, back_A, back_b);
  affineTrans(numpackages, 0, input_y_h, back_A, back_b);

  for (int np = 0; np < numpackages; np++) {
    float xc = input_x_h[np];
    if (xc < xmin) xmin = xc;
    if (xc > xmax) xmax = xc;
    float yc = input_y_h[np];
    if (yc < ymin) ymin = yc;
    if (yc > ymax) ymax = yc;
  }
  CUDA_TRY(hipMemcpy(neighbors_, neighbors_h, sizeof(int) * k_ * nodes_, hipMemcpyHostToDevice));
}

}  // namespace detail

float traveling_salesman(const raft::handle_t &handle,
                         int *route,
                         const float *x_pos,
                         const float *y_pos,
                         const int nodes,
                         const int restarts,
                         const int k,
                         const bool verbose)
{
  RAFT_EXPECTS(route != nullptr, "route should be of size V");
  RAFT_EXPECTS(nodes > 0, "0 vertices");
  RAFT_EXPECTS(restarts > 0, "0 restarts");
  RAFT_EXPECTS(k > 0, "0 neighbors");

  cugraph::detail::TSP tsp(handle, route, x_pos, y_pos,
                           nodes, restarts, k, verbose);
  tsp.allocate();
  tsp.knn();
  return tsp.compute();
}

}  // namespace cugraph
