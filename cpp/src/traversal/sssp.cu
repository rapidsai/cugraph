#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// Author: Prasun Gera pgera@nvidia.com

#include <algorithm>
#include <utilities/error.hpp>

#include "graph.hpp"

#include "sssp.cuh"
#include "sssp_kernels.cuh"
#include "traversal_common.cuh"

namespace cugraph {
namespace detail {

template <typename IndexType, typename DistType>
void SSSP<IndexType, DistType>::setup()
{
  // Working data
  // Each vertex can be in the frontier at most once
  frontier.resize(n);
  new_frontier.resize(n);

  // size of bitmaps for vertices
  vertices_bmap_size = (n / (8 * sizeof(int)) + 1);

  // size of bitmaps for edges
  edges_bmap_size = (nnz / (8 * sizeof(int)) + 1);

  // ith bit of isolated_bmap is set <=> degree of ith vertex = 0
  isolated_bmap.resize(vertices_bmap_size);

  // Allocate buffer for data that need to be reset every iteration
  iter_buffer_size = sizeof(int) * (edges_bmap_size + vertices_bmap_size) + sizeof(IndexType);
  iter_buffer.resize(iter_buffer_size);
  // ith bit of relaxed_edges_bmap <=> ith edge was relaxed
  relaxed_edges_bmap = static_cast<int *>(iter_buffer.data());
  // ith bit of next_frontier_bmap <=> vertex is active in the next frontier
  next_frontier_bmap = static_cast<int *>(iter_buffer.data()) + edges_bmap_size;
  // num vertices in the next frontier
  d_new_frontier_cnt = next_frontier_bmap + vertices_bmap_size;

  // vertices_degree[i] = degree of vertex i
  vertex_degree.resize(n);

  // frontier_vertex_degree[i] is the degree of vertex frontier[i]
  frontier_vertex_degree.resize(n);

  // exclusive sum of frontier_vertex_degree
  exclusive_sum_frontier_vertex_degree.resize(n + 1);

  // We use buckets of edges (32 edges per bucket for now, see exact macro in
  // sssp_kernels). frontier_vertex_degree_buckets_offsets[i] is the index k
  // such as frontier[k] is the source of the first edge of the bucket
  // See top down kernels for more details
  size_t bucket_off_size =
    ((nnz / TOP_DOWN_EXPAND_DIMX + 1) * NBUCKETS_PER_BLOCK + 2) * sizeof(IndexType);
  exclusive_sum_frontier_vertex_buckets_offsets.resize(bucket_off_size);

  // Repurpose d_new_frontier_cnt temporarily
  IndexType *d_nisolated = d_new_frontier_cnt;
  hipMemsetAsync(d_nisolated, 0, sizeof(IndexType), stream);

  // Computing isolated_bmap
  // Only dependent on graph - not source vertex - done once
  traversal::flag_isolated_vertices(
    n, isolated_bmap.data().get(), row_offsets, vertex_degree.data().get(), d_nisolated, stream);

  hipMemcpyAsync(&nisolated, d_nisolated, sizeof(IndexType), hipMemcpyDeviceToHost, stream);

  // We need nisolated to be ready to use
  // nisolated is the number of isolated (zero out-degree) vertices
  hipStreamSynchronize(stream);
}

template <typename IndexType, typename DistType>
void SSSP<IndexType, DistType>::configure(DistType *_distances,
                                          IndexType *_predecessors,
                                          int *_edge_mask)
{
  distances    = _distances;
  predecessors = _predecessors;
  edge_mask    = _edge_mask;

  useEdgeMask         = (edge_mask != NULL);
  computeDistances    = (distances != NULL);
  computePredecessors = (predecessors != NULL);

  // We need distances for SSSP even if the caller doesn't need them
  if (!computeDistances) {
    distances_vals.resize(n);
    distances = distances_vals.data().get();
  }
  // Need next_distances in either case
  next_distances_vals.resize(n);
  next_distances = next_distances_vals.data().get();
}

template <typename IndexType, typename DistType>
void SSSP<IndexType, DistType>::traverse(IndexType source_vertex)
{
  // Init distances to infinities
  traversal::fill_vec(distances, n, traversal::vec_t<DistType>::max, stream);
  traversal::fill_vec(next_distances, n, traversal::vec_t<DistType>::max, stream);

  // If needed, set all predecessors to non-existent (-1)
  if (computePredecessors) { hipMemsetAsync(predecessors, -1, n * sizeof(IndexType), stream); }

  //
  // Initial frontier
  //

  hipMemsetAsync(&distances[source_vertex], 0, sizeof(DistType), stream);
  hipMemsetAsync(&next_distances[source_vertex], 0, sizeof(DistType), stream);

  int current_isolated_bmap_source_vert = 0;

  hipMemcpyAsync(&current_isolated_bmap_source_vert,
                  isolated_bmap.data().get() + (source_vertex / INT_SIZE),
                  sizeof(int),
                  hipMemcpyDeviceToHost);

  // We need current_isolated_bmap_source_vert
  hipStreamSynchronize(stream);

  int m = (1 << (source_vertex % INT_SIZE));

  // If source is isolated (zero outdegree), we are done
  if ((m & current_isolated_bmap_source_vert)) {
    // Init distances and predecessors are done; stream is synchronized
  }

  // Adding source_vertex to init frontier
  hipMemcpyAsync(
    frontier.data().get(), &source_vertex, sizeof(IndexType), hipMemcpyHostToDevice, stream);

  // Number of vertices in the frontier and number of out-edges from the
  // frontier
  IndexType mf, nf;
  nf        = 1;
  int iters = 0;

  while (nf > 0) {
    // Typical pre-top down workflow. set_frontier_degree + exclusive-scan
    traversal::set_frontier_degree(frontier_vertex_degree.data().get(),
                                   frontier.data().get(),
                                   vertex_degree.data().get(),
                                   nf,
                                   stream);

    traversal::exclusive_sum(frontier_vertex_degree.data().get(),
                             exclusive_sum_frontier_vertex_degree.data().get(),
                             nf + 1,
                             stream);

    hipMemcpyAsync(&mf,
                    exclusive_sum_frontier_vertex_degree.data().get() + nf,
                    sizeof(IndexType),
                    hipMemcpyDeviceToHost,
                    stream);

    // We need mf to know the next kernel's launch dims
    hipStreamSynchronize(stream);

    traversal::compute_bucket_offsets(exclusive_sum_frontier_vertex_degree.data().get(),
                                      exclusive_sum_frontier_vertex_buckets_offsets.data().get(),
                                      nf,
                                      mf,
                                      stream);

    // Reset the transient structures to 0
    hipMemsetAsync(iter_buffer.data(), 0, iter_buffer_size, stream);

    sssp_kernels::frontier_expand(row_offsets,
                                  col_indices,
                                  edge_weights,
                                  frontier.data().get(),
                                  nf,
                                  mf,
                                  new_frontier.data().get(),
                                  d_new_frontier_cnt,
                                  exclusive_sum_frontier_vertex_degree.data().get(),
                                  exclusive_sum_frontier_vertex_buckets_offsets.data().get(),
                                  distances,
                                  next_distances,
                                  predecessors,
                                  edge_mask,
                                  next_frontier_bmap,
                                  relaxed_edges_bmap,
                                  isolated_bmap.data().get(),
                                  stream);

    hipMemcpyAsync(&nf, d_new_frontier_cnt, sizeof(IndexType), hipMemcpyDeviceToHost, stream);

    // Copy next_distances to distances
    hipMemcpyAsync(
      distances, next_distances, n * sizeof(DistType), hipMemcpyDeviceToDevice, stream);

    // We need nf for the loop
    CUDA_TRY(hipStreamSynchronize(stream));

    // Swap frontiers
    // IndexType *tmp = frontier;
    // frontier       = new_frontier;
    // new_frontier   = tmp;
    new_frontier.swap(frontier);
    iters++;

    if (iters > n) {
      // Bail out. Got a graph with a negative cycle
      CUGRAPH_FAIL("ERROR: Max iterations exceeded. Check the graph for negative weight cycles");
    }
  }
}

template <typename IndexType, typename DistType>
void SSSP<IndexType, DistType>::clean()
{
}

}  // namespace detail

/**
 * ---------------------------------------------------------------------------*
 * @brief Native sssp with predecessors
 *
 * @file sssp.cu
 * --------------------------------------------------------------------------*/
template <typename VT, typename ET, typename WT>
void sssp(experimental::GraphCSRView<VT, ET, WT> const &graph,
          WT *distances,
          VT *predecessors,
          const VT source_vertex)
{
  CUGRAPH_EXPECTS(distances || predecessors, "Invalid API parameter, both outputs are nullptr");

  if (typeid(VT) != typeid(int)) CUGRAPH_FAIL("Unsupported vertex id data type, please use int");
  if (typeid(ET) != typeid(int)) CUGRAPH_FAIL("Unsupported edge id data type, please use int");
  if (typeid(WT) != typeid(float) && typeid(WT) != typeid(double))
    CUGRAPH_FAIL("Unsupported weight data type, please use float or double");

  int num_vertices = graph.number_of_vertices;
  int num_edges    = graph.number_of_edges;

  const ET *offsets_ptr      = graph.offsets;
  const VT *indices_ptr      = graph.indices;
  const WT *edge_weights_ptr = nullptr;

  // Both if / else branch operate own calls due to
  // thrust::device_vector lifetime
  if (!graph.edge_data) {
    // Generate unit weights

    // FIXME: This should fallback to BFS, but for now it'll go through the
    // SSSP path since BFS needs the directed flag, which should not be
    // necessary for the SSSP API. We can pass directed to the BFS call, but
    // BFS also does only integer distances right now whereas we need float or
    // double

    rmm::device_vector<WT> d_edge_weights(num_edges, static_cast<WT>(1));
    edge_weights_ptr = thrust::raw_pointer_cast(&d_edge_weights.front());
    cugraph::detail::SSSP<VT, WT> sssp(
      num_vertices, num_edges, offsets_ptr, indices_ptr, edge_weights_ptr);
    sssp.configure(distances, predecessors, nullptr);
    sssp.traverse(source_vertex);
  } else {
    // SSSP is not defined for graphs with negative weight cycles
    // Warn user about any negative edges
    if (graph.prop.has_negative_edges == experimental::PropType::PROP_TRUE)
      std::cerr << "WARN: The graph has negative weight edges. SSSP will not "
                   "converge if the graph has negative weight cycles\n";
    edge_weights_ptr = graph.edge_data;
    cugraph::detail::SSSP<VT, WT> sssp(
      num_vertices, num_edges, offsets_ptr, indices_ptr, edge_weights_ptr);
    sssp.configure(distances, predecessors, nullptr);
    sssp.traverse(source_vertex);
  }
}

// explicit instantiation
template void sssp<int, int, float>(experimental::GraphCSRView<int, int, float> const &graph,
                                    float *distances,
                                    int *predecessors,
                                    const int source_vertex);
template void sssp<int, int, double>(experimental::GraphCSRView<int, int, double> const &graph,
                                     double *distances,
                                     int *predecessors,
                                     const int source_vertex);

}  // namespace cugraph
