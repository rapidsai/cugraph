#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// Author: Prasun Gera pgera@nvidia.com

#include <cugraph.h>
#include <rmm_utils.h>
#include <algorithm>

#include "traversal_common.cuh"
#include "sssp.cuh"
#include "sssp_kernels.cuh"
#include "utilities/error_utils.h"

namespace cugraph {

template <typename IndexType, typename DistType>
void SSSP<IndexType, DistType>::setup() {
  // Working data
  // Each vertex can be in the frontier at most once
  ALLOC_TRY(&frontier, n * sizeof(IndexType), nullptr);
  ALLOC_TRY(&new_frontier, n * sizeof(IndexType), nullptr);

  // size of bitmaps for vertices
  vertices_bmap_size = (n / (8 * sizeof(int)) + 1);

  // size of bitmaps for edges
  edges_bmap_size = (nnz / (8 * sizeof(int)) + 1);

  // ith bit of isolated_bmap is set <=> degree of ith vertex = 0
  ALLOC_TRY(&isolated_bmap, sizeof(int) * vertices_bmap_size, nullptr);

  // Allocate buffer for data that need to be reset every iteration
  iter_buffer_size =
      sizeof(int) * (edges_bmap_size + vertices_bmap_size) + sizeof(IndexType);
  ALLOC_TRY(&iter_buffer, iter_buffer_size, nullptr);
  // ith bit of relaxed_edges_bmap <=> ith edge was relaxed
  relaxed_edges_bmap = (int*)iter_buffer;
  // ith bit of next_frontier_bmap <=> vertex is active in the next frontier
  next_frontier_bmap = (int*)iter_buffer + edges_bmap_size;
  // num vertices in the next frontier
  d_new_frontier_cnt = next_frontier_bmap + vertices_bmap_size;

  // vertices_degree[i] = degree of vertex i
  ALLOC_TRY(&vertex_degree, sizeof(IndexType) * n, nullptr);

  // Cub working data
  traversal::cub_exclusive_sum_alloc(
      n + 1, d_cub_exclusive_sum_storage, cub_exclusive_sum_storage_bytes);

  // frontier_vertex_degree[i] is the degree of vertex frontier[i]
  ALLOC_TRY(&frontier_vertex_degree, n * sizeof(IndexType), nullptr);

  // exclusive sum of frontier_vertex_degree
  ALLOC_TRY(&exclusive_sum_frontier_vertex_degree,
            (n + 1) * sizeof(IndexType),
            nullptr);

  // We use buckets of edges (32 edges per bucket for now, see exact macro in
  // sssp_kernels). frontier_vertex_degree_buckets_offsets[i] is the index k
  // such as frontier[k] is the source of the first edge of the bucket
  // See top down kernels for more details
  size_t bucket_off_size =
      ((nnz / TOP_DOWN_EXPAND_DIMX + 1) * NBUCKETS_PER_BLOCK + 2) *
      sizeof(IndexType);
  ALLOC_TRY(&exclusive_sum_frontier_vertex_buckets_offsets,
            bucket_off_size,
            nullptr);

  // Repurpose d_new_frontier_cnt temporarily
  IndexType* d_nisolated = d_new_frontier_cnt;
  hipMemsetAsync(d_nisolated, 0, sizeof(IndexType), stream);

  // Computing isolated_bmap
  // Only dependent on graph - not source vertex - done once
  traversal::flag_isolated_vertices(
      n, isolated_bmap, row_offsets, vertex_degree, d_nisolated, stream);

  hipMemcpyAsync(&nisolated,
                  d_nisolated,
                  sizeof(IndexType),
                  hipMemcpyDeviceToHost,
                  stream);

  // We need nisolated to be ready to use
  // nisolated is the number of isolated (zero out-degree) vertices
  hipStreamSynchronize(stream);
}

template <typename IndexType, typename DistType>
void SSSP<IndexType, DistType>::configure(DistType* _distances,
                                          IndexType* _predecessors,
                                          int* _edge_mask) {
  distances = _distances;
  predecessors = _predecessors;
  edge_mask = _edge_mask;

  useEdgeMask = (edge_mask != NULL);
  computeDistances = (distances != NULL);
  computePredecessors = (predecessors != NULL);

  // We need distances for SSSP even if the caller doesn't need them
  if (!computeDistances)
    ALLOC_TRY(&distances, n * sizeof(DistType), nullptr);
  // Need next_distances in either case
  ALLOC_TRY(&next_distances, n * sizeof(DistType), nullptr);
}

template <typename IndexType, typename DistType>
gdf_error SSSP<IndexType, DistType>::traverse(IndexType source_vertex) {
  // Init distances to infinities
  traversal::fill_vec(distances, n, traversal::vec_t<DistType>::max, stream);
  traversal::fill_vec(
      next_distances, n, traversal::vec_t<DistType>::max, stream);

  // If needed, set all predecessors to non-existent (-1)
  if (computePredecessors) {
    hipMemsetAsync(predecessors, -1, n * sizeof(IndexType), stream);
  }

  //
  // Initial frontier
  //

  hipMemsetAsync(&distances[source_vertex], 0, sizeof(DistType), stream);
  hipMemsetAsync(&next_distances[source_vertex], 0, sizeof(DistType), stream);

  int current_isolated_bmap_source_vert = 0;

  hipMemcpyAsync(&current_isolated_bmap_source_vert,
                  &isolated_bmap[source_vertex / INT_SIZE],
                  sizeof(int),
                  hipMemcpyDeviceToHost);

  // We need current_isolated_bmap_source_vert
  hipStreamSynchronize(stream);

  int m = (1 << (source_vertex % INT_SIZE));

  // If source is isolated (zero outdegree), we are done
  if ((m & current_isolated_bmap_source_vert)) {
    // Init distances and predecessors are done; stream is synchronized
    return GDF_SUCCESS;
  }

  // Adding source_vertex to init frontier
  hipMemcpyAsync(&frontier[0],
                  &source_vertex,
                  sizeof(IndexType),
                  hipMemcpyHostToDevice,
                  stream);

  // Number of vertices in the frontier and number of out-edges from the
  // frontier
  IndexType mf, nf;
  nf = 1;
  int iters = 0;

  while (nf > 0) {
    // Typical pre-top down workflow. set_frontier_degree + exclusive-scan
    traversal::set_frontier_degree(
        frontier_vertex_degree, frontier, vertex_degree, nf, stream);

    traversal::exclusive_sum(d_cub_exclusive_sum_storage,
                             cub_exclusive_sum_storage_bytes,
                             frontier_vertex_degree,
                             exclusive_sum_frontier_vertex_degree,
                             nf + 1,
                             stream);

    hipMemcpyAsync(&mf,
                    &exclusive_sum_frontier_vertex_degree[nf],
                    sizeof(IndexType),
                    hipMemcpyDeviceToHost,
                    stream);

    // We need mf to know the next kernel's launch dims
    hipStreamSynchronize(stream);

    traversal::compute_bucket_offsets(
        exclusive_sum_frontier_vertex_degree,
        exclusive_sum_frontier_vertex_buckets_offsets,
        nf,
        mf,
        stream);

    // Reset the transient structures to 0
    hipMemsetAsync(iter_buffer, 0, iter_buffer_size, stream);

    sssp_kernels::frontier_expand(
        row_offsets,
        col_indices,
        edge_weights,
        frontier,
        nf,
        mf,
        new_frontier,
        d_new_frontier_cnt,
        exclusive_sum_frontier_vertex_degree,
        exclusive_sum_frontier_vertex_buckets_offsets,
        distances,
        next_distances,
        predecessors,
        edge_mask,
        next_frontier_bmap,
        relaxed_edges_bmap,
        isolated_bmap,
        stream);

    hipMemcpyAsync(&nf,
                    d_new_frontier_cnt,
                    sizeof(IndexType),
                    hipMemcpyDeviceToHost,
                    stream);

    // Copy next_distances to distances
    hipMemcpyAsync(distances,
                    next_distances,
                    n * sizeof(DistType),
                    hipMemcpyDeviceToDevice,
                    stream);

    cudaCheckError();

    // We need nf for the loop
    hipStreamSynchronize(stream);

    // Swap frontiers
    IndexType* tmp = frontier;
    frontier = new_frontier;
    new_frontier = tmp;
    iters++;

    if (iters > n) {
      // Bail out. Got a graph with a negative cycle
      std::cerr << "ERROR: Max iterations exceeded. Check the graph for "
                   "negative weight cycles\n";
      return GDF_INVALID_API_CALL;
    }
  }
  return GDF_SUCCESS;
}

template <typename IndexType, typename DistType>
void SSSP<IndexType, DistType>::clean() {
  // the vectors have a destructor that takes care of cleaning
  ALLOC_FREE_TRY(frontier, nullptr);
  ALLOC_FREE_TRY(new_frontier, nullptr);
  ALLOC_FREE_TRY(isolated_bmap, nullptr);
  ALLOC_FREE_TRY(vertex_degree, nullptr);
  ALLOC_FREE_TRY(d_cub_exclusive_sum_storage, nullptr);
  ALLOC_FREE_TRY(frontier_vertex_degree, nullptr);
  ALLOC_FREE_TRY(exclusive_sum_frontier_vertex_degree, nullptr);
  ALLOC_FREE_TRY(exclusive_sum_frontier_vertex_buckets_offsets, nullptr);
  ALLOC_FREE_TRY(iter_buffer, nullptr);

  // Distances were working data
  if (!computeDistances)
    ALLOC_FREE_TRY(distances, nullptr);

  // next_distances were working data
  ALLOC_FREE_TRY(next_distances, nullptr);
}

}  // end namespace cugraph

/**
 * ---------------------------------------------------------------------------*
 * @brief Native sssp with predecessors
 *
 * @file sssp.cu
 * --------------------------------------------------------------------------*/

gdf_error gdf_sssp(gdf_graph* gdf_G,
                   gdf_column* sssp_distances,
                   gdf_column* predecessors,
                   const int source_vert) {

  GDF_REQUIRE(gdf_G->adjList != nullptr, GDF_INVALID_API_CALL);

  void *sssp_dist_ptr, *pred_ptr;
  // NOTE: gdf_column struct doesn't have a default constructor. So we can get
  // garbage values for member fields. Right now, it's the caller's
  // responsibility to ensure that the fields are initialised if the gdf_column
  // ptr is not null
  sssp_dist_ptr = (sssp_distances && sssp_distances->size)
      ? sssp_distances->data
      : nullptr;
  pred_ptr =
      (predecessors && predecessors->size) ? predecessors->data : nullptr;

  GDF_REQUIRE(sssp_dist_ptr || pred_ptr, GDF_INVALID_API_CALL);

  if (sssp_dist_ptr) {
    GDF_REQUIRE(!sssp_distances->valid, GDF_VALIDITY_UNSUPPORTED);
    // Integral types are possible, but we don't want to deal with overflow
    // conditions right now
    GDF_REQUIRE(sssp_distances->dtype == GDF_FLOAT32 ||
                    sssp_distances->dtype == GDF_FLOAT64,
                GDF_INVALID_API_CALL);
  }
  GDF_REQUIRE(gdf_G->adjList->offsets->dtype == GDF_INT32,
              GDF_UNSUPPORTED_DTYPE);
  GDF_REQUIRE(gdf_G->adjList->indices->dtype == GDF_INT32,
              GDF_UNSUPPORTED_DTYPE);

  if (pred_ptr)
    GDF_REQUIRE(predecessors->dtype == gdf_G->adjList->indices->dtype,
                GDF_UNSUPPORTED_DTYPE);

  if (sssp_dist_ptr)
    GDF_REQUIRE(gdf_G->adjList->offsets->size - 1 <= sssp_distances->size,
                GDF_INVALID_API_CALL);

  if (!gdf_G->adjList->edge_data) {
    // Generate unit weights

    // TODO: This should fallback to BFS, but for now it'll go through the
    // SSSP path since BFS needs the directed flag, which should not be
    // necessary for the SSSP API. We can pass directed to the BFS call, but
    // BFS also does only integer distances right now whereas we need float or
    // double

    void* d_edge_data;
    gdf_G->adjList->edge_data = new gdf_column;
    hipStream_t stream{nullptr};

    // If distances array is given and is double, generate the weights in
    // double
    if (sssp_dist_ptr && sssp_distances->dtype == GDF_FLOAT64) {
      std::vector<double> h_edge_data(gdf_G->adjList->indices->size, 1.0);
      size_t edge_data_size = sizeof(double) * h_edge_data.size();
      ALLOC_TRY((void**)&d_edge_data, edge_data_size, stream);
      CUDA_TRY(hipMemcpy(d_edge_data,
                          &h_edge_data[0],
                          edge_data_size,
                          hipMemcpyHostToDevice));
      gdf_column_view(gdf_G->adjList->edge_data,
                      d_edge_data,
                      nullptr,
                      gdf_G->adjList->indices->size,
                      GDF_FLOAT64);

    } else {
      // Else generate float
      std::vector<float> h_edge_data(gdf_G->adjList->indices->size, 1.0);
      size_t edge_data_size = sizeof(float) * h_edge_data.size();
      ALLOC_TRY((void**)&d_edge_data, edge_data_size, stream);
      CUDA_TRY(hipMemcpy(d_edge_data,
                          &h_edge_data[0],
                          edge_data_size,
                          hipMemcpyHostToDevice));
      gdf_column_view(gdf_G->adjList->edge_data,
                      d_edge_data,
                      nullptr,
                      gdf_G->adjList->indices->size,
                      GDF_FLOAT32);
    }
  } else {
    // Got weighted graph
    GDF_REQUIRE(
        gdf_G->adjList->edge_data->size == gdf_G->adjList->indices->size,
        GDF_INVALID_API_CALL);

    GDF_REQUIRE(gdf_G->adjList->edge_data->dtype == GDF_FLOAT32 ||
                    gdf_G->adjList->edge_data->dtype == GDF_FLOAT64,
                GDF_INVALID_API_CALL);

    if (sssp_dist_ptr)
      GDF_REQUIRE(gdf_G->adjList->edge_data->dtype == sssp_distances->dtype,
                  GDF_UNSUPPORTED_DTYPE);

    // SSSP is not defined for graphs with negative weight cycles
    // Warn user about any negative edges
    if (gdf_G->prop && gdf_G->prop->has_negative_edges == GDF_PROP_TRUE)
      std::cerr << "WARN: The graph has negative weight edges. SSSP will not "
                   "converge if the graph has negative weight cycles\n";
  }

  int n = gdf_G->adjList->offsets->size - 1;
  int e = gdf_G->adjList->indices->size;
  int* offsets_ptr = (int*)gdf_G->adjList->offsets->data;
  int* indices_ptr = (int*)gdf_G->adjList->indices->data;

  void* edge_weights_ptr = static_cast<void*>(gdf_G->adjList->edge_data->data);
  gdf_error ret;

  if (gdf_G->adjList->edge_data->dtype == GDF_FLOAT32) {
    cugraph::SSSP<int, float> sssp(
        n, e, offsets_ptr, indices_ptr, static_cast<float*>(edge_weights_ptr));

    sssp.configure(static_cast<float*>(sssp_dist_ptr),
                   static_cast<int*>(pred_ptr),
                   nullptr);

    ret = sssp.traverse(source_vert);
  } else if (gdf_G->adjList->edge_data->dtype == GDF_FLOAT64) {
    cugraph::SSSP<int, double> sssp(n,
                                    e,
                                    offsets_ptr,
                                    indices_ptr,
                                    static_cast<double*>(edge_weights_ptr));

    sssp.configure(static_cast<double*>(sssp_dist_ptr),
                   static_cast<int*>(pred_ptr),
                   nullptr);

    ret = sssp.traverse(source_vert);
  } else {
    GDF_REQUIRE(gdf_G->adjList->edge_data->dtype == GDF_FLOAT32 ||
                    gdf_G->adjList->edge_data->dtype == GDF_FLOAT64,
                GDF_INVALID_API_CALL);
  }

  return ret;
}
