#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// Author: Prasun Gera pgera@nvidia.com

#include <cugraph.h>
#include <rmm_utils.h>
#include <algorithm>

#include "graph.hpp"

#include "sssp.cuh"
#include "sssp_kernels.cuh"
#include "traversal_common.cuh"
#include "utilities/error_utils.h"

namespace cugraph {
namespace detail {

template <typename IndexType, typename DistType>
void SSSP<IndexType, DistType>::setup()
{
  // Working data
  // Each vertex can be in the frontier at most once
  ALLOC_TRY(&frontier, n * sizeof(IndexType), nullptr);
  ALLOC_TRY(&new_frontier, n * sizeof(IndexType), nullptr);

  // size of bitmaps for vertices
  vertices_bmap_size = (n / (8 * sizeof(int)) + 1);

  // size of bitmaps for edges
  edges_bmap_size = (nnz / (8 * sizeof(int)) + 1);

  // ith bit of isolated_bmap is set <=> degree of ith vertex = 0
  ALLOC_TRY(&isolated_bmap, sizeof(int) * vertices_bmap_size, nullptr);

  // Allocate buffer for data that need to be reset every iteration
  iter_buffer_size = sizeof(int) * (edges_bmap_size + vertices_bmap_size) + sizeof(IndexType);
  ALLOC_TRY(&iter_buffer, iter_buffer_size, nullptr);
  // ith bit of relaxed_edges_bmap <=> ith edge was relaxed
  relaxed_edges_bmap = (int *)iter_buffer;
  // ith bit of next_frontier_bmap <=> vertex is active in the next frontier
  next_frontier_bmap = (int *)iter_buffer + edges_bmap_size;
  // num vertices in the next frontier
  d_new_frontier_cnt = next_frontier_bmap + vertices_bmap_size;

  // vertices_degree[i] = degree of vertex i
  ALLOC_TRY(&vertex_degree, sizeof(IndexType) * n, nullptr);

  // Cub working data
  traversal::cub_exclusive_sum_alloc(
    n + 1, d_cub_exclusive_sum_storage, cub_exclusive_sum_storage_bytes);

  // frontier_vertex_degree[i] is the degree of vertex frontier[i]
  ALLOC_TRY(&frontier_vertex_degree, n * sizeof(IndexType), nullptr);

  // exclusive sum of frontier_vertex_degree
  ALLOC_TRY(&exclusive_sum_frontier_vertex_degree, (n + 1) * sizeof(IndexType), nullptr);

  // We use buckets of edges (32 edges per bucket for now, see exact macro in
  // sssp_kernels). frontier_vertex_degree_buckets_offsets[i] is the index k
  // such as frontier[k] is the source of the first edge of the bucket
  // See top down kernels for more details
  size_t bucket_off_size =
    ((nnz / TOP_DOWN_EXPAND_DIMX + 1) * NBUCKETS_PER_BLOCK + 2) * sizeof(IndexType);
  ALLOC_TRY(&exclusive_sum_frontier_vertex_buckets_offsets, bucket_off_size, nullptr);

  // Repurpose d_new_frontier_cnt temporarily
  IndexType *d_nisolated = d_new_frontier_cnt;
  hipMemsetAsync(d_nisolated, 0, sizeof(IndexType), stream);

  // Computing isolated_bmap
  // Only dependent on graph - not source vertex - done once
  traversal::flag_isolated_vertices(
    n, isolated_bmap, row_offsets, vertex_degree, d_nisolated, stream);

  hipMemcpyAsync(&nisolated, d_nisolated, sizeof(IndexType), hipMemcpyDeviceToHost, stream);

  // We need nisolated to be ready to use
  // nisolated is the number of isolated (zero out-degree) vertices
  hipStreamSynchronize(stream);
}

template <typename IndexType, typename DistType>
void SSSP<IndexType, DistType>::configure(DistType *_distances,
                                          IndexType *_predecessors,
                                          int *_edge_mask)
{
  distances    = _distances;
  predecessors = _predecessors;
  edge_mask    = _edge_mask;

  useEdgeMask         = (edge_mask != NULL);
  computeDistances    = (distances != NULL);
  computePredecessors = (predecessors != NULL);

  // We need distances for SSSP even if the caller doesn't need them
  if (!computeDistances) ALLOC_TRY(&distances, n * sizeof(DistType), nullptr);
  // Need next_distances in either case
  ALLOC_TRY(&next_distances, n * sizeof(DistType), nullptr);
}

template <typename IndexType, typename DistType>
void SSSP<IndexType, DistType>::traverse(IndexType source_vertex)
{
  // Init distances to infinities
  traversal::fill_vec(distances, n, traversal::vec_t<DistType>::max, stream);
  traversal::fill_vec(next_distances, n, traversal::vec_t<DistType>::max, stream);

  // If needed, set all predecessors to non-existent (-1)
  if (computePredecessors) { hipMemsetAsync(predecessors, -1, n * sizeof(IndexType), stream); }

  //
  // Initial frontier
  //

  hipMemsetAsync(&distances[source_vertex], 0, sizeof(DistType), stream);
  hipMemsetAsync(&next_distances[source_vertex], 0, sizeof(DistType), stream);

  int current_isolated_bmap_source_vert = 0;

  hipMemcpyAsync(&current_isolated_bmap_source_vert,
                  &isolated_bmap[source_vertex / INT_SIZE],
                  sizeof(int),
                  hipMemcpyDeviceToHost);

  // We need current_isolated_bmap_source_vert
  hipStreamSynchronize(stream);

  int m = (1 << (source_vertex % INT_SIZE));

  // If source is isolated (zero outdegree), we are done
  if ((m & current_isolated_bmap_source_vert)) {
    // Init distances and predecessors are done; stream is synchronized
  }

  // Adding source_vertex to init frontier
  hipMemcpyAsync(&frontier[0], &source_vertex, sizeof(IndexType), hipMemcpyHostToDevice, stream);

  // Number of vertices in the frontier and number of out-edges from the
  // frontier
  IndexType mf, nf;
  nf        = 1;
  int iters = 0;

  while (nf > 0) {
    // Typical pre-top down workflow. set_frontier_degree + exclusive-scan
    traversal::set_frontier_degree(frontier_vertex_degree, frontier, vertex_degree, nf, stream);

    traversal::exclusive_sum(d_cub_exclusive_sum_storage,
                             cub_exclusive_sum_storage_bytes,
                             frontier_vertex_degree,
                             exclusive_sum_frontier_vertex_degree,
                             nf + 1,
                             stream);

    hipMemcpyAsync(&mf,
                    &exclusive_sum_frontier_vertex_degree[nf],
                    sizeof(IndexType),
                    hipMemcpyDeviceToHost,
                    stream);

    // We need mf to know the next kernel's launch dims
    hipStreamSynchronize(stream);

    traversal::compute_bucket_offsets(exclusive_sum_frontier_vertex_degree,
                                      exclusive_sum_frontier_vertex_buckets_offsets,
                                      nf,
                                      mf,
                                      stream);

    // Reset the transient structures to 0
    hipMemsetAsync(iter_buffer, 0, iter_buffer_size, stream);

    sssp_kernels::frontier_expand(row_offsets,
                                  col_indices,
                                  edge_weights,
                                  frontier,
                                  nf,
                                  mf,
                                  new_frontier,
                                  d_new_frontier_cnt,
                                  exclusive_sum_frontier_vertex_degree,
                                  exclusive_sum_frontier_vertex_buckets_offsets,
                                  distances,
                                  next_distances,
                                  predecessors,
                                  edge_mask,
                                  next_frontier_bmap,
                                  relaxed_edges_bmap,
                                  isolated_bmap,
                                  stream);

    hipMemcpyAsync(&nf, d_new_frontier_cnt, sizeof(IndexType), hipMemcpyDeviceToHost, stream);

    // Copy next_distances to distances
    hipMemcpyAsync(
      distances, next_distances, n * sizeof(DistType), hipMemcpyDeviceToDevice, stream);

    CUDA_CHECK_LAST();

    // We need nf for the loop
    hipStreamSynchronize(stream);

    // Swap frontiers
    IndexType *tmp = frontier;
    frontier       = new_frontier;
    new_frontier   = tmp;
    iters++;

    if (iters > n) {
      // Bail out. Got a graph with a negative cycle
      CUGRAPH_FAIL("ERROR: Max iterations exceeded. Check the graph for negative weight cycles");
    }
  }
}

template <typename IndexType, typename DistType>
void SSSP<IndexType, DistType>::clean()
{
  // the vectors have a destructor that takes care of cleaning
  ALLOC_FREE_TRY(frontier, nullptr);
  ALLOC_FREE_TRY(new_frontier, nullptr);
  ALLOC_FREE_TRY(isolated_bmap, nullptr);
  ALLOC_FREE_TRY(vertex_degree, nullptr);
  ALLOC_FREE_TRY(d_cub_exclusive_sum_storage, nullptr);
  ALLOC_FREE_TRY(frontier_vertex_degree, nullptr);
  ALLOC_FREE_TRY(exclusive_sum_frontier_vertex_degree, nullptr);
  ALLOC_FREE_TRY(exclusive_sum_frontier_vertex_buckets_offsets, nullptr);
  ALLOC_FREE_TRY(iter_buffer, nullptr);

  // Distances were working data
  if (!computeDistances) ALLOC_FREE_TRY(distances, nullptr);

  // next_distances were working data
  ALLOC_FREE_TRY(next_distances, nullptr);
}

}  // namespace detail

/**
 * ---------------------------------------------------------------------------*
 * @brief Native sssp with predecessors
 *
 * @file sssp.cu
 * --------------------------------------------------------------------------*/
template <typename VT, typename ET, typename WT>
void sssp(experimental::GraphCSR<VT, ET, WT> const &graph,
          WT *distances,
          VT *predecessors,
          const VT source_vertex)
{
  CUGRAPH_EXPECTS(distances || predecessors, "Invalid API parameter, both outputs are nullptr");

  if (typeid(VT) != typeid(int)) CUGRAPH_FAIL("Unsupported vertex id data type, please use int");
  if (typeid(ET) != typeid(int)) CUGRAPH_FAIL("Unsupported edge id data type, please use int");
  if (typeid(WT) != typeid(float) && typeid(WT) != typeid(double))
    CUGRAPH_FAIL("Unsupported weight data type, please use float or double");

  int num_vertices = graph.number_of_vertices;
  int num_edges    = graph.number_of_edges;

  const ET *offsets_ptr      = graph.offsets;
  const VT *indices_ptr      = graph.indices;
  const WT *edge_weights_ptr = nullptr;

  // Both if / else branch operate own calls due to
  // thrust::device_vector lifetime
  if (!graph.edge_data) {
    // Generate unit weights

    // TODO: This should fallback to BFS, but for now it'll go through the
    // SSSP path since BFS needs the directed flag, which should not be
    // necessary for the SSSP API. We can pass directed to the BFS call, but
    // BFS also does only integer distances right now whereas we need float or
    // double

    thrust::device_vector<WT> d_edge_weights(num_edges, static_cast<WT>(1));
    edge_weights_ptr = thrust::raw_pointer_cast(&d_edge_weights.front());
    cugraph::detail::SSSP<VT, WT> sssp(
      num_vertices, num_edges, offsets_ptr, indices_ptr, edge_weights_ptr);
    sssp.configure(distances, predecessors, nullptr);
    sssp.traverse(source_vertex);
  } else {
    // SSSP is not defined for graphs with negative weight cycles
    // Warn user about any negative edges
    if (graph.prop.has_negative_edges == experimental::PropType::PROP_TRUE)
      std::cerr << "WARN: The graph has negative weight edges. SSSP will not "
                   "converge if the graph has negative weight cycles\n";
    edge_weights_ptr = graph.edge_data;
    cugraph::detail::SSSP<VT, WT> sssp(
      num_vertices, num_edges, offsets_ptr, indices_ptr, edge_weights_ptr);
    sssp.configure(distances, predecessors, nullptr);
    sssp.traverse(source_vertex);
  }
}

// explicit instantiation
template void sssp<int, int, float>(experimental::GraphCSR<int, int, float> const &graph,
                                    float *distances,
                                    int *predecessors,
                                    const int source_vertex);
template void sssp<int, int, double>(experimental::GraphCSR<int, int, double> const &graph,
                                     double *distances,
                                     int *predecessors,
                                     const int source_vertex);

}  // namespace cugraph
