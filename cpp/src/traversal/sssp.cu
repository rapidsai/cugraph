#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// Author: Prasun Gera pgera@nvidia.com

#include <cugraph.h>
#include <rmm_utils.h>
#include <algorithm>

#include "traversal_common.cuh"
#include "sssp.cuh"
#include "sssp_kernels.cuh"
#include "utilities/error_utils.h"

namespace cugraph { 
namespace detail {

template <typename IndexType, typename DistType>
void SSSP<IndexType, DistType>::setup() {
  // Working data
  // Each vertex can be in the frontier at most once
  ALLOC_TRY(&frontier, n * sizeof(IndexType), nullptr);
  ALLOC_TRY(&new_frontier, n * sizeof(IndexType), nullptr);

  // size of bitmaps for vertices
  vertices_bmap_size = (n / (8 * sizeof(int)) + 1);

  // size of bitmaps for edges
  edges_bmap_size = (nnz / (8 * sizeof(int)) + 1);

  // ith bit of isolated_bmap is set <=> degree of ith vertex = 0
  ALLOC_TRY(&isolated_bmap, sizeof(int) * vertices_bmap_size, nullptr);

  // Allocate buffer for data that need to be reset every iteration
  iter_buffer_size =
      sizeof(int) * (edges_bmap_size + vertices_bmap_size) + sizeof(IndexType);
  ALLOC_TRY(&iter_buffer, iter_buffer_size, nullptr);
  // ith bit of relaxed_edges_bmap <=> ith edge was relaxed
  relaxed_edges_bmap = (int*)iter_buffer;
  // ith bit of next_frontier_bmap <=> vertex is active in the next frontier
  next_frontier_bmap = (int*)iter_buffer + edges_bmap_size;
  // num vertices in the next frontier
  d_new_frontier_cnt = next_frontier_bmap + vertices_bmap_size;

  // vertices_degree[i] = degree of vertex i
  ALLOC_TRY(&vertex_degree, sizeof(IndexType) * n, nullptr);

  // Cub working data
  traversal::cub_exclusive_sum_alloc(
      n + 1, d_cub_exclusive_sum_storage, cub_exclusive_sum_storage_bytes);

  // frontier_vertex_degree[i] is the degree of vertex frontier[i]
  ALLOC_TRY(&frontier_vertex_degree, n * sizeof(IndexType), nullptr);

  // exclusive sum of frontier_vertex_degree
  ALLOC_TRY(&exclusive_sum_frontier_vertex_degree,
            (n + 1) * sizeof(IndexType),
            nullptr);

  // We use buckets of edges (32 edges per bucket for now, see exact macro in
  // sssp_kernels). frontier_vertex_degree_buckets_offsets[i] is the index k
  // such as frontier[k] is the source of the first edge of the bucket
  // See top down kernels for more details
  size_t bucket_off_size =
      ((nnz / TOP_DOWN_EXPAND_DIMX + 1) * NBUCKETS_PER_BLOCK + 2) *
      sizeof(IndexType);
  ALLOC_TRY(&exclusive_sum_frontier_vertex_buckets_offsets,
            bucket_off_size,
            nullptr);

  // Repurpose d_new_frontier_cnt temporarily
  IndexType* d_nisolated = d_new_frontier_cnt;
  hipMemsetAsync(d_nisolated, 0, sizeof(IndexType), stream);

  // Computing isolated_bmap
  // Only dependent on graph - not source vertex - done once
  traversal::flag_isolated_vertices(
      n, isolated_bmap, row_offsets, vertex_degree, d_nisolated, stream);

  hipMemcpyAsync(&nisolated,
                  d_nisolated,
                  sizeof(IndexType),
                  hipMemcpyDeviceToHost,
                  stream);

  // We need nisolated to be ready to use
  // nisolated is the number of isolated (zero out-degree) vertices
  hipStreamSynchronize(stream);
}

template <typename IndexType, typename DistType>
void SSSP<IndexType, DistType>::configure(DistType* _distances,
                                          IndexType* _predecessors,
                                          int* _edge_mask) {
  distances = _distances;
  predecessors = _predecessors;
  edge_mask = _edge_mask;

  useEdgeMask = (edge_mask != NULL);
  computeDistances = (distances != NULL);
  computePredecessors = (predecessors != NULL);

  // We need distances for SSSP even if the caller doesn't need them
  if (!computeDistances)
    ALLOC_TRY(&distances, n * sizeof(DistType), nullptr);
  // Need next_distances in either case
  ALLOC_TRY(&next_distances, n * sizeof(DistType), nullptr);
}

template <typename IndexType, typename DistType>
void SSSP<IndexType, DistType>::traverse(IndexType source_vertex) {
  // Init distances to infinities
  traversal::fill_vec(distances, n, traversal::vec_t<DistType>::max, stream);
  traversal::fill_vec(
      next_distances, n, traversal::vec_t<DistType>::max, stream);

  // If needed, set all predecessors to non-existent (-1)
  if (computePredecessors) {
    hipMemsetAsync(predecessors, -1, n * sizeof(IndexType), stream);
  }

  //
  // Initial frontier
  //

  hipMemsetAsync(&distances[source_vertex], 0, sizeof(DistType), stream);
  hipMemsetAsync(&next_distances[source_vertex], 0, sizeof(DistType), stream);

  int current_isolated_bmap_source_vert = 0;

  hipMemcpyAsync(&current_isolated_bmap_source_vert,
                  &isolated_bmap[source_vertex / INT_SIZE],
                  sizeof(int),
                  hipMemcpyDeviceToHost);

  // We need current_isolated_bmap_source_vert
  hipStreamSynchronize(stream);

  int m = (1 << (source_vertex % INT_SIZE));

  // If source is isolated (zero outdegree), we are done
  if ((m & current_isolated_bmap_source_vert)) {
    // Init distances and predecessors are done; stream is synchronized
    
  }

  // Adding source_vertex to init frontier
  hipMemcpyAsync(&frontier[0],
                  &source_vertex,
                  sizeof(IndexType),
                  hipMemcpyHostToDevice,
                  stream);

  // Number of vertices in the frontier and number of out-edges from the
  // frontier
  IndexType mf, nf;
  nf = 1;
  int iters = 0;

  while (nf > 0) {
    // Typical pre-top down workflow. set_frontier_degree + exclusive-scan
    traversal::set_frontier_degree(
        frontier_vertex_degree, frontier, vertex_degree, nf, stream);

    traversal::exclusive_sum(d_cub_exclusive_sum_storage,
                             cub_exclusive_sum_storage_bytes,
                             frontier_vertex_degree,
                             exclusive_sum_frontier_vertex_degree,
                             nf + 1,
                             stream);

    hipMemcpyAsync(&mf,
                    &exclusive_sum_frontier_vertex_degree[nf],
                    sizeof(IndexType),
                    hipMemcpyDeviceToHost,
                    stream);

    // We need mf to know the next kernel's launch dims
    hipStreamSynchronize(stream);

    traversal::compute_bucket_offsets(
        exclusive_sum_frontier_vertex_degree,
        exclusive_sum_frontier_vertex_buckets_offsets,
        nf,
        mf,
        stream);

    // Reset the transient structures to 0
    hipMemsetAsync(iter_buffer, 0, iter_buffer_size, stream);

    sssp_kernels::frontier_expand(
        row_offsets,
        col_indices,
        edge_weights,
        frontier,
        nf,
        mf,
        new_frontier,
        d_new_frontier_cnt,
        exclusive_sum_frontier_vertex_degree,
        exclusive_sum_frontier_vertex_buckets_offsets,
        distances,
        next_distances,
        predecessors,
        edge_mask,
        next_frontier_bmap,
        relaxed_edges_bmap,
        isolated_bmap,
        stream);

    hipMemcpyAsync(&nf,
                    d_new_frontier_cnt,
                    sizeof(IndexType),
                    hipMemcpyDeviceToHost,
                    stream);

    // Copy next_distances to distances
    hipMemcpyAsync(distances,
                    next_distances,
                    n * sizeof(DistType),
                    hipMemcpyDeviceToDevice,
                    stream);

    CUDA_CHECK_LAST();

    // We need nf for the loop
    hipStreamSynchronize(stream);

    // Swap frontiers
    IndexType* tmp = frontier;
    frontier = new_frontier;
    new_frontier = tmp;
    iters++;

    if (iters > n) {
      // Bail out. Got a graph with a negative cycle
      CUGRAPH_FAIL("ERROR: Max iterations exceeded. Check the graph for negative weight cycles");
    }
  }
  
}

template <typename IndexType, typename DistType>
void SSSP<IndexType, DistType>::clean() {
  // the vectors have a destructor that takes care of cleaning
  ALLOC_FREE_TRY(frontier, nullptr);
  ALLOC_FREE_TRY(new_frontier, nullptr);
  ALLOC_FREE_TRY(isolated_bmap, nullptr);
  ALLOC_FREE_TRY(vertex_degree, nullptr);
  ALLOC_FREE_TRY(d_cub_exclusive_sum_storage, nullptr);
  ALLOC_FREE_TRY(frontier_vertex_degree, nullptr);
  ALLOC_FREE_TRY(exclusive_sum_frontier_vertex_degree, nullptr);
  ALLOC_FREE_TRY(exclusive_sum_frontier_vertex_buckets_offsets, nullptr);
  ALLOC_FREE_TRY(iter_buffer, nullptr);

  // Distances were working data
  if (!computeDistances)
    ALLOC_FREE_TRY(distances, nullptr);

  // next_distances were working data
  ALLOC_FREE_TRY(next_distances, nullptr);
}

} //namespace

/**
 * ---------------------------------------------------------------------------*
 * @brief Native sssp with predecessors
 *
 * @file sssp.cu
 * --------------------------------------------------------------------------*/
template <typename VT, typename WT>
void sssp(Graph* graph,                                            
          WT *distances,                                                  
          VT *predecessors,                                               
          const VT source_vertex) {

  CUGRAPH_EXPECTS(graph->adjList != nullptr, "Invalid API parameter: Graph is empty");

  CUGRAPH_EXPECTS(distances || predecessors, "Invalid API parameter, both outputs are nullptr");

  if (typeid(WT) != typeid(float) && typeid(WT) != typeid(double))
    CUGRAPH_FAIL("Unsupported weight data type, please use float or double");
  if (typeid(VT) != typeid(int) )
    CUGRAPH_FAIL("Unsupported vertex data type, please use int");

  // TODO fix me after gdf_column is removed from Graph
  CUGRAPH_EXPECTS(graph->adjList->offsets->dtype == GDF_INT32,
              "Unsupported data type");
  CUGRAPH_EXPECTS(graph->adjList->indices->dtype == GDF_INT32,
              "Unsupported data type");
  // TODO fix me after gdf_column is removed from Graph
  // if (predecessors) CUGRAPH_EXPECTS(typeid(predecessors) == typeid(graph->adjList->indices), "predecessors and ID type mismatch");

  if (!graph->adjList->edge_data) {
    // Generate unit weights

    // TODO: This should fallback to BFS, but for now it'll go through the
    // SSSP path since BFS needs the directed flag, which should not be
    // necessary for the SSSP API. We can pass directed to the BFS call, but
    // BFS also does only integer distances right now whereas we need float or
    // double

    void* d_edge_data;
    graph->adjList->edge_data = new gdf_column;
    hipStream_t stream{nullptr};

    // If distances array is given and is double, generate the weights in
    // double
    if (distances && typeid(WT) == typeid(double)) {
      std::vector<double> h_edge_data(graph->adjList->indices->size, 1.0);
      size_t edge_data_size = sizeof(double) * h_edge_data.size();
      ALLOC_TRY((void**)&d_edge_data, edge_data_size, stream);
      CUDA_TRY(hipMemcpy(d_edge_data,
                          &h_edge_data[0],
                          edge_data_size,
                          hipMemcpyHostToDevice));
      gdf_column_view(graph->adjList->edge_data,
                      d_edge_data,
                      nullptr,
                      graph->adjList->indices->size,
                      GDF_FLOAT64);

    } else {
      // Else generate float
      std::vector<float> h_edge_data(graph->adjList->indices->size, 1.0);
      size_t edge_data_size = sizeof(float) * h_edge_data.size();
      ALLOC_TRY((void**)&d_edge_data, edge_data_size, stream);
      CUDA_TRY(hipMemcpy(d_edge_data,
                          &h_edge_data[0],
                          edge_data_size,
                          hipMemcpyHostToDevice));
      gdf_column_view(graph->adjList->edge_data,
                      d_edge_data,
                      nullptr,
                      graph->adjList->indices->size,
                      GDF_FLOAT32);
    }
  } else {
    // Got weighted graph
    CUGRAPH_EXPECTS(
        graph->adjList->edge_data->size == graph->adjList->indices->size,
        "Graph sizes mismatch");
    // TODO fix me after gdf_column is removed from Graph
    CUGRAPH_EXPECTS(graph->adjList->edge_data->dtype == GDF_FLOAT32 ||
                    graph->adjList->edge_data->dtype == GDF_FLOAT64,
                "Invalid API parameter");
    // TODO fix me after gdf_column is removed from Graph
    // if (distances) CUGRAPH_EXPECTS(typeid(distances) == typeid(graph->adjList->edge_data), "distances and weights type mismatch");

    // SSSP is not defined for graphs with negative weight cycles
    // Warn user about any negative edges
    if (graph->prop && graph->prop->has_negative_edges == GDF_PROP_TRUE)
      std::cerr << "WARN: The graph has negative weight edges. SSSP will not "
                   "converge if the graph has negative weight cycles\n";
  }

  int n = graph->adjList->offsets->size - 1;
  int e = graph->adjList->indices->size;
  int* offsets_ptr = (int*)graph->adjList->offsets->data;
  int* indices_ptr = (int*)graph->adjList->indices->data;

  void* edge_weights_ptr = static_cast<void*>(graph->adjList->edge_data->data);

  if (typeid(WT) == typeid(float)) {
    cugraph::detail::SSSP<VT, WT> sssp(
        n, e, offsets_ptr, indices_ptr, static_cast<WT*>(edge_weights_ptr));

    sssp.configure(distances, predecessors, nullptr);
    sssp.traverse(source_vertex);
  } else if (typeid(WT) == typeid(double)) {
    cugraph::detail::SSSP<VT, WT> sssp(n,
                                    e,
                                    offsets_ptr,
                                    indices_ptr,
                                    static_cast<WT*>(edge_weights_ptr));

    sssp.configure(distances, predecessors, nullptr);
    sssp.traverse(source_vertex);
  } else {
    CUGRAPH_EXPECTS(graph->adjList->edge_data->dtype == GDF_FLOAT32 ||
                    graph->adjList->edge_data->dtype == GDF_FLOAT64,
                "Invalid API parameter");
  }
}

// explicit instantiation
template void sssp<int, float>(Graph* graph, float *distances, int *predecessors, const int source_vertex);
template void sssp<int, double>(Graph* graph, double *distances, int *predecessors, const int source_vertex);

} //namespace
