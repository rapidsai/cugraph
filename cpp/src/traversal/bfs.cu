#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 */

#include <algorithm>
#include <iomanip>
#include <limits>
#include "bfs.cuh"

#include "graph.hpp"

#include <utilities/error.hpp>
#include "bfs_kernels.cuh"
#include "traversal_common.cuh"
#include "utilities/graph_utils.cuh"

namespace cugraph {
namespace detail {
enum BFS_ALGO_STATE { TOPDOWN, BOTTOMUP };

template <typename IndexType>
void BFS<IndexType>::setup()
{
  // Determinism flag, false by default
  deterministic = false;

  // Working data
  // Each vertex can be in the frontier at most once
  // We will update frontier during the execution
  // We need the orig to reset frontier, or ALLOC_FREE_TRY
  original_frontier.resize(number_of_vertices);
  frontier = original_frontier.data().get();

  // size of bitmaps for vertices
  vertices_bmap_size = (number_of_vertices / (8 * sizeof(int)) + 1);
  // ith bit of visited_bmap is set <=> ith vertex is visited

  visited_bmap.resize(vertices_bmap_size);

  // ith bit of isolated_bmap is set <=> degree of ith vertex = 0
  isolated_bmap.resize(vertices_bmap_size);

  // vertices_degree[i] = degree of vertex i
  vertex_degree.resize(number_of_vertices);

  // We will need (n+1) ints buffer for two differents things (bottom up or top down) - sharing it
  // since those uses are mutually exclusive
  buffer_np1_1.resize(number_of_vertices + 1);
  buffer_np1_2.resize(number_of_vertices + 1);

  // Using buffers : top down

  // frontier_vertex_degree[i] is the degree of vertex frontier[i]
  frontier_vertex_degree = buffer_np1_1.data().get();
  // exclusive sum of frontier_vertex_degree
  exclusive_sum_frontier_vertex_degree = buffer_np1_2.data().get();

  // Using buffers : bottom up
  // contains list of unvisited vertices
  unvisited_queue = buffer_np1_1.data().get();
  // size of the "last" unvisited queue : size_last_unvisited_queue
  // refers to the size of unvisited_queue
  // which may not be up to date (the queue may contains vertices that are now
  // visited)

  // We may leave vertices unvisited after bottom up main kernels - storing them
  // here
  left_unvisited_queue = buffer_np1_2.data().get();

  // We use buckets of edges (32 edges per bucket for now, see exact macro in bfs_kernels).
  // frontier_vertex_degree_buckets_offsets[i] is the index k such as frontier[k] is the source of
  // the first edge of the bucket See top down kernels for more details
  exclusive_sum_frontier_vertex_buckets_offsets.resize(
    ((number_of_edges / TOP_DOWN_EXPAND_DIMX + 1) * NBUCKETS_PER_BLOCK + 2));

  // Init device-side counters
  // Those counters must be/can be reset at each bfs iteration
  // Keeping them adjacent in memory allow use call only one hipMemset - launch latency is the
  // current bottleneck
  d_counters_pad.resize(4);

  d_new_frontier_cnt   = d_counters_pad.data().get();
  d_mu                 = d_counters_pad.data().get() + 1;
  d_unvisited_cnt      = d_counters_pad.data().get() + 2;
  d_left_unvisited_cnt = d_counters_pad.data().get() + 3;

  // Lets use this int* for the next 3 lines
  // Its dereferenced value is not initialized - so we dont care about what we
  // put in it
  IndexType *d_nisolated = d_new_frontier_cnt;
  hipMemsetAsync(d_nisolated, 0, sizeof(IndexType), stream);

  // Computing isolated_bmap
  // Only dependent on graph - not source vertex - done once
  traversal::flag_isolated_vertices(number_of_vertices,
                                    isolated_bmap.data().get(),
                                    row_offsets,
                                    vertex_degree.data().get(),
                                    d_nisolated,
                                    stream);
  hipMemcpyAsync(&nisolated, d_nisolated, sizeof(IndexType), hipMemcpyDeviceToHost, stream);

  // We need nisolated to be ready to use
  hipStreamSynchronize(stream);
}

template <typename IndexType>
void BFS<IndexType>::configure(IndexType *_distances,
                               IndexType *_predecessors,
                               double *_sp_counters,
                               int *_edge_mask)
{
  distances    = _distances;
  predecessors = _predecessors;
  edge_mask    = _edge_mask;
  sp_counters  = _sp_counters;

  useEdgeMask         = (edge_mask != NULL);
  computeDistances    = (distances != NULL);
  computePredecessors = (predecessors != NULL);

  // We need distances to use bottom up
  if (directed && !computeDistances) {
    distances_vals.resize(number_of_vertices);
    distances = distances_vals.data().get();
  }

  // In case the shortest path counters is required, previous_bmap has to be allocated
  if (sp_counters) { previous_visited_bmap.resize(vertices_bmap_size); }
}

template <typename IndexType>
void BFS<IndexType>::traverse(IndexType source_vertex)
{
  // Init visited_bmap
  // If the graph is undirected, we not that
  // we will never discover isolated vertices (in degree = out degree = 0)
  // we avoid a lot of work by flagging them now
  // in g500 graphs they represent ~25% of total vertices
  // more than that for wiki and twitter graphs

  if (directed) {
    hipMemsetAsync(visited_bmap.data().get(), 0, vertices_bmap_size * sizeof(int), stream);
  } else {
    hipMemcpyAsync(visited_bmap.data().get(),
                    isolated_bmap.data().get(),
                    vertices_bmap_size * sizeof(int),
                    hipMemcpyDeviceToDevice,
                    stream);
  }

  // If needed, setting all vertices as undiscovered (inf distance)
  // We dont use computeDistances here
  // if the graph is undirected, we may need distances even if
  // computeDistances is false
  if (distances)
    traversal::fill_vec(distances, number_of_vertices, traversal::vec_t<IndexType>::max, stream);

  // If needed, setting all predecessors to non-existent (-1)
  if (computePredecessors) {
    hipMemsetAsync(predecessors, -1, number_of_vertices * sizeof(IndexType), stream);
  }

  if (sp_counters) {
    hipMemsetAsync(sp_counters, 0, number_of_vertices * sizeof(double), stream);
    double value = 1;
    hipMemcpyAsync(sp_counters + source_vertex, &value, sizeof(double), hipMemcpyHostToDevice);
  }

  //
  // Initial frontier
  //

  frontier = original_frontier.data().get();

  if (distances) { hipMemsetAsync(&distances[source_vertex], 0, sizeof(IndexType), stream); }

  // Setting source_vertex as visited
  // There may be bit already set on that bmap (isolated vertices) - if the
  // graph is undirected
  int current_visited_bmap_source_vert = 0;

  if (!directed) {
    hipMemcpyAsync(&current_visited_bmap_source_vert,
                    visited_bmap.data().get() + (source_vertex / INT_SIZE),
                    sizeof(int),
                    hipMemcpyDeviceToHost);
    // We need current_visited_bmap_source_vert
    hipStreamSynchronize(stream);
  }

  int m = (1 << (source_vertex % INT_SIZE));

  // In that case, source is isolated, done now
  if (!directed && (m & current_visited_bmap_source_vert)) {
    // Init distances and predecessors are done, (cf Streamsync in previous if)
    return;
  }

  m |= current_visited_bmap_source_vert;

  hipMemcpyAsync(visited_bmap.data().get() + (source_vertex / INT_SIZE),
                  &m,
                  sizeof(int),
                  hipMemcpyHostToDevice,
                  stream);

  // Adding source_vertex to init frontier
  hipMemcpyAsync(&frontier[0], &source_vertex, sizeof(IndexType), hipMemcpyHostToDevice, stream);

  // mf : edges in frontier
  // nf : vertices in frontier
  // mu : edges undiscovered
  // nu : nodes undiscovered
  // lvl : current frontier's depth
  IndexType mf, nf, mu, nu;
  bool growing;
  IndexType lvl = 1;

  // Frontier has one vertex
  nf = 1;

  // all edges are undiscovered (by def isolated vertices have 0 edges)
  mu = number_of_edges;

  // all non isolated vertices are undiscovered (excepted source vertex, which is in frontier)
  // That number is wrong if source_vertex is also isolated - but it's not important
  nu = number_of_vertices - nisolated - nf;

  // Last frontier was 0, now it is 1
  growing = true;

  IndexType size_last_left_unvisited_queue = number_of_vertices;  // we just need value > 0
  IndexType size_last_unvisited_queue      = 0;                   // queue empty

  // Typical pre-top down workflow. set_frontier_degree + exclusive-scan
  traversal::set_frontier_degree(
    frontier_vertex_degree, frontier, vertex_degree.data().get(), nf, stream);
  traversal::exclusive_sum(
    frontier_vertex_degree, exclusive_sum_frontier_vertex_degree, nf + 1, stream);

  hipMemcpyAsync(&mf,
                  &exclusive_sum_frontier_vertex_degree[nf],
                  sizeof(IndexType),
                  hipMemcpyDeviceToHost,
                  stream);

  // We need mf
  hipStreamSynchronize(stream);

  // At first we know we have to use top down
  BFS_ALGO_STATE algo_state = TOPDOWN;

  // useDistances : we check if a vertex is a parent using distances in bottom up - distances become
  // working data undirected g : need parents to be in children's neighbors

  // In case the shortest path counters need to be computeed, the bottom_up approach cannot be used
  bool can_use_bottom_up = (!sp_counters && !directed && distances);

  while (nf > 0) {
    // Each vertices can appear only once in the frontierer array - we know it will fit
    new_frontier     = frontier + nf;
    IndexType old_nf = nf;
    resetDevicePointers();

    if (can_use_bottom_up) {
      // Choosing algo
      // Finite machine described in http://parlab.eecs.berkeley.edu/sites/all/parlab/files/main.pdf

      switch (algo_state) {
        case TOPDOWN:
          if (mf > mu / alpha) algo_state = BOTTOMUP;
          break;
        case BOTTOMUP:
          if (!growing && nf < number_of_vertices / beta) {
            // We need to prepare the switch back to top down
            // We couldnt keep track of mu during bottom up - because we dont know what mf is.
            // Computing mu here
            bfs_kernels::count_unvisited_edges(unvisited_queue,
                                               size_last_unvisited_queue,
                                               visited_bmap.data().get(),
                                               vertex_degree.data().get(),
                                               d_mu,
                                               stream);

            // Typical pre-top down workflow. set_frontier_degree + exclusive-scan
            traversal::set_frontier_degree(
              frontier_vertex_degree, frontier, vertex_degree.data().get(), nf, stream);
            traversal::exclusive_sum(
              frontier_vertex_degree, exclusive_sum_frontier_vertex_degree, nf + 1, stream);

            hipMemcpyAsync(&mf,
                            &exclusive_sum_frontier_vertex_degree[nf],
                            sizeof(IndexType),
                            hipMemcpyDeviceToHost,
                            stream);

            hipMemcpyAsync(&mu, d_mu, sizeof(IndexType), hipMemcpyDeviceToHost, stream);

            // We will need mf and mu
            hipStreamSynchronize(stream);
            algo_state = TOPDOWN;
          }
          break;
      }
    }

    // Executing algo

    switch (algo_state) {
      case TOPDOWN:
        // This step is only required if sp_counters is not nullptr
        if (sp_counters) {
          hipMemcpyAsync(previous_visited_bmap.data().get(),
                          visited_bmap.data().get(),
                          vertices_bmap_size * sizeof(int),
                          hipMemcpyDeviceToDevice,
                          stream);
          // We need to copy the visited_bmap before doing the traversal
          hipStreamSynchronize(stream);
        }
        traversal::compute_bucket_offsets(
          exclusive_sum_frontier_vertex_degree,
          exclusive_sum_frontier_vertex_buckets_offsets.data().get(),
          nf,
          mf,
          stream);
        bfs_kernels::frontier_expand(row_offsets,
                                     col_indices,
                                     frontier,
                                     nf,
                                     mf,
                                     lvl,
                                     new_frontier,
                                     d_new_frontier_cnt,
                                     exclusive_sum_frontier_vertex_degree,
                                     exclusive_sum_frontier_vertex_buckets_offsets.data().get(),
                                     previous_visited_bmap.data().get(),
                                     visited_bmap.data().get(),
                                     distances,
                                     predecessors,
                                     sp_counters,
                                     edge_mask,
                                     isolated_bmap.data().get(),
                                     directed,
                                     stream,
                                     deterministic);

        mu -= mf;

        hipMemcpyAsync(&nf, d_new_frontier_cnt, sizeof(IndexType), hipMemcpyDeviceToHost, stream);
        CHECK_CUDA(stream);

        // We need nf
        hipStreamSynchronize(stream);

        if (nf) {
          // Typical pre-top down workflow. set_frontier_degree + exclusive-scan
          traversal::set_frontier_degree(
            frontier_vertex_degree, new_frontier, vertex_degree.data().get(), nf, stream);
          traversal::exclusive_sum(
            frontier_vertex_degree, exclusive_sum_frontier_vertex_degree, nf + 1, stream);
          hipMemcpyAsync(&mf,
                          &exclusive_sum_frontier_vertex_degree[nf],
                          sizeof(IndexType),
                          hipMemcpyDeviceToHost,
                          stream);

          // We need mf
          hipStreamSynchronize(stream);
        }
        break;

      case BOTTOMUP:
        bfs_kernels::fill_unvisited_queue(visited_bmap.data().get(),
                                          vertices_bmap_size,
                                          number_of_vertices,
                                          unvisited_queue,
                                          d_unvisited_cnt,
                                          stream,
                                          deterministic);

        size_last_unvisited_queue = nu;

        bfs_kernels::bottom_up_main(unvisited_queue,
                                    size_last_unvisited_queue,
                                    left_unvisited_queue,
                                    d_left_unvisited_cnt,
                                    visited_bmap.data().get(),
                                    row_offsets,
                                    col_indices,
                                    lvl,
                                    new_frontier,
                                    d_new_frontier_cnt,
                                    distances,
                                    predecessors,
                                    edge_mask,
                                    stream,
                                    deterministic);

        // The number of vertices left unvisited decreases
        // If it wasnt necessary last time, it wont be this time
        if (size_last_left_unvisited_queue) {
          hipMemcpyAsync(&size_last_left_unvisited_queue,
                          d_left_unvisited_cnt,
                          sizeof(IndexType),
                          hipMemcpyDeviceToHost,
                          stream);
          CHECK_CUDA(stream);
          // We need last_left_unvisited_size
          hipStreamSynchronize(stream);
          bfs_kernels::bottom_up_large(left_unvisited_queue,
                                       size_last_left_unvisited_queue,
                                       visited_bmap.data().get(),
                                       row_offsets,
                                       col_indices,
                                       lvl,
                                       new_frontier,
                                       d_new_frontier_cnt,
                                       distances,
                                       predecessors,
                                       edge_mask,
                                       stream,
                                       deterministic);
        }
        hipMemcpyAsync(&nf, d_new_frontier_cnt, sizeof(IndexType), hipMemcpyDeviceToHost, stream);
        CHECK_CUDA(stream);

        // We will need nf
        hipStreamSynchronize(stream);
        break;
    }

    // Updating undiscovered edges count
    nu -= nf;

    // Using new frontier
    frontier = new_frontier;
    growing  = (nf > old_nf);

    ++lvl;
  }
}

template <typename IndexType>
void BFS<IndexType>::resetDevicePointers()
{
  hipMemsetAsync(d_counters_pad.data().get(), 0, 4 * sizeof(IndexType), stream);
}

template <typename IndexType>
void BFS<IndexType>::clean()
{
  // the vectors have a destructor that takes care of cleaning
}

template class BFS<int>;
}  // namespace detail

// NOTE: SP counter increase extremely fast on large graph
//       It can easily reach 1e40~1e70 on GAP-road.mtx
template <typename VT, typename ET, typename WT>
void bfs(experimental::GraphCSRView<VT, ET, WT> const &graph,
         VT *distances,
         VT *predecessors,
         double *sp_counters,
         const VT start_vertex,
         bool directed)
{
  CUGRAPH_EXPECTS(typeid(VT) == typeid(int), "Unsupported vertex id data type, please use int");
  CUGRAPH_EXPECTS(typeid(ET) == typeid(int), "Unsupported edge id data type, please use int");
  CUGRAPH_EXPECTS((typeid(WT) == typeid(float)) || (typeid(WT) == typeid(double)),
                  "Unsupported weight data type, please use float or double");

  VT number_of_vertices = graph.number_of_vertices;
  ET number_of_edges    = graph.number_of_edges;

  const VT *indices_ptr = graph.indices;
  const ET *offsets_ptr = graph.offsets;

  int alpha = 15;
  int beta  = 18;
  // FIXME: Use VT and ET in the BFS detail
  cugraph::detail::BFS<VT> bfs(
    number_of_vertices, number_of_edges, offsets_ptr, indices_ptr, directed, alpha, beta);
  bfs.configure(distances, predecessors, sp_counters, nullptr);
  bfs.traverse(start_vertex);
}

template void bfs<int, int, float>(experimental::GraphCSRView<int, int, float> const &graph,
                                   int *distances,
                                   int *predecessors,
                                   double *sp_counters,
                                   const int source_vertex,
                                   bool directed);
template void bfs<int, int, double>(experimental::GraphCSRView<int, int, double> const &graph,
                                    int *distances,
                                    int *predecessors,
                                    double *sp_counters,
                                    const int source_vertex,
                                    bool directed);

}  // namespace cugraph
