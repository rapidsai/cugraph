/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "barnes_hut.hpp"
#include "exact_fa2.hpp"

namespace cugraph {

template <typename vertex_t, typename edge_t, typename weight_t>
void force_atlas2(raft::handle_t const &handle,
                  GraphCOOView<vertex_t, edge_t, weight_t> &graph,
                  float *pos,
                  const int max_iter,
                  float *x_start,
                  float *y_start,
                  bool outbound_attraction_distribution,
                  bool lin_log_mode,
                  bool prevent_overlapping,
                  const float edge_weight_influence,
                  const float jitter_tolerance,
                  bool barnes_hut_optimize,
                  const float barnes_hut_theta,
                  const float scaling_ratio,
                  bool strong_gravity_mode,
                  const float gravity,
                  bool verbose,
                  internals::GraphBasedDimRedCallback *callback)
{
  CUGRAPH_EXPECTS(pos != nullptr, "Invalid input argument: pos array should be of size 2 * V");
  CUGRAPH_EXPECTS(graph.number_of_vertices != 0, "Invalid input: Graph is empty");

  if (!barnes_hut_optimize) {
    cugraph::detail::exact_fa2<vertex_t, edge_t, weight_t>(handle,
                                                           graph,
                                                           pos,
                                                           max_iter,
                                                           x_start,
                                                           y_start,
                                                           outbound_attraction_distribution,
                                                           lin_log_mode,
                                                           prevent_overlapping,
                                                           edge_weight_influence,
                                                           jitter_tolerance,
                                                           scaling_ratio,
                                                           strong_gravity_mode,
                                                           gravity,
                                                           verbose,
                                                           callback);
  } else {
    cugraph::detail::barnes_hut<vertex_t, edge_t, weight_t>(handle,
                                                            graph,
                                                            pos,
                                                            max_iter,
                                                            x_start,
                                                            y_start,
                                                            outbound_attraction_distribution,
                                                            lin_log_mode,
                                                            prevent_overlapping,
                                                            edge_weight_influence,
                                                            jitter_tolerance,
                                                            barnes_hut_theta,
                                                            scaling_ratio,
                                                            strong_gravity_mode,
                                                            gravity,
                                                            verbose,
                                                            callback);
  }
}

template void force_atlas2<int, int, float>(raft::handle_t const &handle,
                                            GraphCOOView<int, int, float> &graph,
                                            float *pos,
                                            const int max_iter,
                                            float *x_start,
                                            float *y_start,
                                            bool outbound_attraction_distribution,
                                            bool lin_log_mode,
                                            bool prevent_overlapping,
                                            const float edge_weight_influence,
                                            const float jitter_tolerance,
                                            bool barnes_hut_optimize,
                                            const float barnes_hut_theta,
                                            const float scaling_ratio,
                                            bool strong_gravity_mode,
                                            const float gravity,
                                            bool verbose,
                                            internals::GraphBasedDimRedCallback *callback);

template void force_atlas2<int, int, double>(raft::handle_t const &handle,
                                             GraphCOOView<int, int, double> &graph,
                                             float *pos,
                                             const int max_iter,
                                             float *x_start,
                                             float *y_start,
                                             bool outbound_attraction_distribution,
                                             bool lin_log_mode,
                                             bool prevent_overlapping,
                                             const float edge_weight_influence,
                                             const float jitter_tolerance,
                                             bool barnes_hut_optimize,
                                             const float barnes_hut_theta,
                                             const float scaling_ratio,
                                             bool strong_gravity_mode,
                                             const float gravity,
                                             bool verbose,
                                             internals::GraphBasedDimRedCallback *callback);

}  // namespace cugraph
