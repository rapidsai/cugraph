#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithm>
#include <iomanip>
#include "include/bfs.hxx"
#include <limits>

#include <rmm/rmm.h>

#include "include/nvgraph_error.hxx"
#include "bfs_kernels.cu"

using namespace bfs_kernels;

namespace nvgraph {
  enum BFS_ALGO_STATE {
    TOPDOWN, BOTTOMUP
  };

  template<typename IndexType>
  NVGRAPH_ERROR Bfs<IndexType>::setup() {

    // Determinism flag, false by default
    deterministic = false;

    auto rmm_result = RMM_SUCCESS;

    //Working data
    //Each vertex can be in the frontier at most once
    rmm_result = RMM_ALLOC(&frontier, n * sizeof(IndexType), stream);
    rmmCheckError(rmm_result);

    //We will update frontier during the execution
    //We need the orig to reset frontier, or hipFree
    original_frontier = frontier;

    //size of bitmaps for vertices
    vertices_bmap_size = (n / (8 * sizeof(int)) + 1);
    //ith bit of visited_bmap is set <=> ith vertex is visited
    rmm_result = RMM_ALLOC(&visited_bmap, sizeof(int) * vertices_bmap_size, stream);
    rmmCheckError(rmm_result);

    //ith bit of isolated_bmap is set <=> degree of ith vertex = 0
    rmm_result = RMM_ALLOC(&isolated_bmap, sizeof(int) * vertices_bmap_size, stream);
    rmmCheckError(rmm_result);

    //vertices_degree[i] = degree of vertex i
    rmm_result = RMM_ALLOC(&vertex_degree, sizeof(IndexType) * n, stream);
    rmmCheckError(rmm_result);

    //Cub working data
    cub_exclusive_sum_alloc(n + 1, d_cub_exclusive_sum_storage, cub_exclusive_sum_storage_bytes);

    //We will need (n+1) ints buffer for two differents things (bottom up or top down) - sharing it since those uses are mutually exclusive
    rmm_result = RMM_ALLOC(&buffer_np1_1, (n + 1) * sizeof(IndexType), stream);
    rmmCheckError(rmm_result);

    rmm_result = RMM_ALLOC(&buffer_np1_2, (n + 1) * sizeof(IndexType), stream);
    rmmCheckError(rmm_result);

    //Using buffers : top down

    //frontier_vertex_degree[i] is the degree of vertex frontier[i]
    frontier_vertex_degree = buffer_np1_1;
    //exclusive sum of frontier_vertex_degree
    exclusive_sum_frontier_vertex_degree = buffer_np1_2;

    //Using buffers : bottom up

    //contains list of unvisited vertices
    unvisited_queue = buffer_np1_1;
    //size of the "last" unvisited queue : size_last_unvisited_queue
    //refers to the size of unvisited_queue
    //which may not be up to date (the queue may contains vertices that are now visited)

    //We may leave vertices unvisited after bottom up main kernels - storing them here
    left_unvisited_queue = buffer_np1_2;

    //We use buckets of edges (32 edges per bucket for now, see exact macro in bfs_kernels). frontier_vertex_degree_buckets_offsets[i] is the index k such as frontier[k] is the source of the first edge of the bucket
    //See top down kernels for more details
    rmm_result = RMM_ALLOC(&exclusive_sum_frontier_vertex_buckets_offsets,
                           ((nnz / TOP_DOWN_EXPAND_DIMX + 1) * NBUCKETS_PER_BLOCK + 2) * sizeof(IndexType),
                           stream);
    rmmCheckError(rmm_result);

    //Init device-side counters
    //Those counters must be/can be reset at each bfs iteration
    //Keeping them adjacent in memory allow use call only one hipMemset - launch latency is the current bottleneck
    rmm_result = RMM_ALLOC(&d_counters_pad, 4 * sizeof(IndexType), stream);
    rmmCheckError(rmm_result);

    d_new_frontier_cnt = &d_counters_pad[0];
    d_mu = &d_counters_pad[1];
    d_unvisited_cnt = &d_counters_pad[2];
    d_left_unvisited_cnt = &d_counters_pad[3];

    //Lets use this int* for the next 3 lines
    //Its dereferenced value is not initialized - so we dont care about what we put in it
    IndexType * d_nisolated = d_new_frontier_cnt;
    hipMemsetAsync(d_nisolated, 0, sizeof(IndexType), stream);
    cudaCheckError()
    ;

    //Computing isolated_bmap
    //Only dependent on graph - not source vertex - done once
    flag_isolated_vertices(n, isolated_bmap, row_offsets, vertex_degree, d_nisolated, stream);
    hipMemcpyAsync(&nisolated, d_nisolated, sizeof(IndexType), hipMemcpyDeviceToHost, stream);
    cudaCheckError()
    ;

    //We need nisolated to be ready to use
    hipStreamSynchronize(stream);
    cudaCheckError()
    ;

    return NVGRAPH_OK;
  }

  template<typename IndexType>
  NVGRAPH_ERROR Bfs<IndexType>::configure(  IndexType *_distances,
                              IndexType *_predecessors,
                              int *_edge_mask)
                              {
    distances = _distances;
    predecessors = _predecessors;
    edge_mask = _edge_mask;

    useEdgeMask = (edge_mask != NULL);
    computeDistances = (distances != NULL);
    computePredecessors = (predecessors != NULL);

    //We need distances to use bottom up
    if (directed && !computeDistances) {
      auto rmm_result = RMM_ALLOC(&distances, n * sizeof(IndexType), stream);
      rmmCheckError(rmm_result);
    }

    return NVGRAPH_OK;
  }

  template<typename IndexType>
  NVGRAPH_ERROR Bfs<IndexType>::traverse(IndexType source_vertex) {

    //Init visited_bmap
    //If the graph is undirected, we not that
    //we will never discover isolated vertices (in degree = out degree = 0)
    //we avoid a lot of work by flagging them now
    //in g500 graphs they represent ~25% of total vertices
    //more than that for wiki and twitter graphs

    if (directed) {
      hipMemsetAsync(visited_bmap, 0, vertices_bmap_size * sizeof(int), stream);
    } else {
      hipMemcpyAsync(  visited_bmap,
                  isolated_bmap,
                  vertices_bmap_size * sizeof(int),
                  hipMemcpyDeviceToDevice,
                  stream);
    }
    cudaCheckError()
    ;

    //If needed, setting all vertices as undiscovered (inf distance)
    //We dont use computeDistances here
    //if the graph is undirected, we may need distances even if
    //computeDistances is false
    if (distances)
      fill_vec(distances, n, vec_t<IndexType>::max, stream);

    //If needed, setting all predecessors to non-existent (-1)
    if (computePredecessors)
    {
      hipMemsetAsync(predecessors, -1, n * sizeof(IndexType), stream);
      cudaCheckError()
      ;
    }

    //
    //Initial frontier
    //

    frontier = original_frontier;

    if (distances)
    {
      hipMemsetAsync(&distances[source_vertex], 0, sizeof(IndexType), stream);
      cudaCheckError()
      ;
    }

    //Setting source_vertex as visited
    //There may be bit already set on that bmap (isolated vertices) - if the graph is undirected
    int current_visited_bmap_source_vert = 0;

    if (!directed) {
      hipMemcpyAsync(&current_visited_bmap_source_vert,
                  &visited_bmap[source_vertex / INT_SIZE],
                  sizeof(int),
                  hipMemcpyDeviceToHost);
      cudaCheckError()
      ;
      //We need current_visited_bmap_source_vert
      hipStreamSynchronize(stream);
      cudaCheckError()
      ;
      //We could detect that source is isolated here
    }

    int m = (1 << (source_vertex % INT_SIZE));

    //In that case, source is isolated, done now
    if (!directed && (m & current_visited_bmap_source_vert)) {
      //Init distances and predecessors are done, (cf Streamsync in previous if)
      cudaCheckError()
      ;
      return NVGRAPH_OK;
    }

    m |= current_visited_bmap_source_vert;

    hipMemcpyAsync(  &visited_bmap[source_vertex / INT_SIZE],
                &m,
                sizeof(int),
                hipMemcpyHostToDevice,
                stream);
    cudaCheckError()
    ;

    //Adding source_vertex to init frontier
    hipMemcpyAsync(  &frontier[0],
                &source_vertex,
                sizeof(IndexType),
                hipMemcpyHostToDevice,
                stream);
    cudaCheckError()
    ;

    //mf : edges in frontier
    //nf : vertices in frontier
    //mu : edges undiscovered
    //nu : nodes undiscovered
    //lvl : current frontier's depth
    IndexType mf, nf, mu, nu;
    bool growing;
    IndexType lvl = 1;

    //Frontier has one vertex
    nf = 1;

    //all edges are undiscovered (by def isolated vertices have 0 edges)
    mu = nnz;

    //all non isolated vertices are undiscovered (excepted source vertex, which is in frontier)
    //That number is wrong if source_vertex is also isolated - but it's not important
    nu = n - nisolated - nf;

    //Last frontier was 0, now it is 1
    growing = true;

    IndexType size_last_left_unvisited_queue = n; //we just need value > 0
    IndexType size_last_unvisited_queue = 0; //queue empty

    //Typical pre-top down workflow. set_frontier_degree + exclusive-scan
    set_frontier_degree(frontier_vertex_degree, frontier, vertex_degree, nf, stream);
    exclusive_sum(  d_cub_exclusive_sum_storage,
              cub_exclusive_sum_storage_bytes,
              frontier_vertex_degree,
              exclusive_sum_frontier_vertex_degree,
              nf + 1,
              stream);

    hipMemcpyAsync(  &mf,
                &exclusive_sum_frontier_vertex_degree[nf],
                sizeof(IndexType),
                hipMemcpyDeviceToHost,
                stream);
    cudaCheckError()
    ;

    //We need mf
    hipStreamSynchronize(stream);
    cudaCheckError()
    ;

    //At first we know we have to use top down
    BFS_ALGO_STATE algo_state = TOPDOWN;

    //useDistances : we check if a vertex is a parent using distances in bottom up - distances become working data
    //undirected g : need parents to be in children's neighbors
    bool can_use_bottom_up = !directed && distances;

    while (nf > 0) {
      //Each vertices can appear only once in the frontierer array - we know it will fit
      new_frontier = frontier + nf;
      IndexType old_nf = nf;
      resetDevicePointers();

      if (can_use_bottom_up) {
        //Choosing algo
        //Finite machine described in http://parlab.eecs.berkeley.edu/sites/all/parlab/files/main.pdf

        switch (algo_state) {
        case TOPDOWN:
          if (mf > mu / alpha)
            algo_state = BOTTOMUP;
          break;
        case BOTTOMUP:
          if (!growing && nf < n / beta) {

            //We need to prepare the switch back to top down
            //We couldnt keep track of mu during bottom up - because we dont know what mf is. Computing mu here
            count_unvisited_edges(  unvisited_queue,
                            size_last_unvisited_queue,
                            visited_bmap,
                            vertex_degree,
                            d_mu,
                            stream);

            //Typical pre-top down workflow. set_frontier_degree + exclusive-scan
            set_frontier_degree(frontier_vertex_degree, frontier, vertex_degree, nf, stream);
            exclusive_sum(  d_cub_exclusive_sum_storage,
                      cub_exclusive_sum_storage_bytes,
                      frontier_vertex_degree,
                      exclusive_sum_frontier_vertex_degree,
                      nf + 1,
                      stream);

            hipMemcpyAsync(  &mf,
                        &exclusive_sum_frontier_vertex_degree[nf],
                        sizeof(IndexType),
                        hipMemcpyDeviceToHost,
                        stream);
            cudaCheckError()
            ;

            hipMemcpyAsync(&mu, d_mu, sizeof(IndexType), hipMemcpyDeviceToHost, stream);
            cudaCheckError()
            ;

            //We will need mf and mu
            hipStreamSynchronize(stream);
            cudaCheckError()
            ;

            algo_state = TOPDOWN;
          }
          break;
        }
      }

      //Executing algo

      switch (algo_state) {
      case TOPDOWN:
        compute_bucket_offsets(  exclusive_sum_frontier_vertex_degree,
                        exclusive_sum_frontier_vertex_buckets_offsets,
                        nf,
                        mf,
                        stream);
        frontier_expand(  row_offsets,
                    col_indices,
                    frontier,
                    nf,
                    mf,
                    lvl,
                    new_frontier,
                    d_new_frontier_cnt,
                    exclusive_sum_frontier_vertex_degree,
                    exclusive_sum_frontier_vertex_buckets_offsets,
                    visited_bmap,
                    distances,
                    predecessors,
                    edge_mask,
                    isolated_bmap,
                    directed,
                    stream,
                    deterministic);

        mu -= mf;

        hipMemcpyAsync(  &nf,
                    d_new_frontier_cnt,
                    sizeof(IndexType),
                    hipMemcpyDeviceToHost,
                    stream);
        cudaCheckError();

        //We need nf
        hipStreamSynchronize(stream);
        cudaCheckError();

        if (nf) {

          //Typical pre-top down workflow. set_frontier_degree + exclusive-scan
          set_frontier_degree(frontier_vertex_degree, new_frontier, vertex_degree, nf, stream);
          exclusive_sum(  d_cub_exclusive_sum_storage,
                    cub_exclusive_sum_storage_bytes,
                    frontier_vertex_degree,
                    exclusive_sum_frontier_vertex_degree,
                    nf + 1,
                    stream);
          hipMemcpyAsync(  &mf,
                      &exclusive_sum_frontier_vertex_degree[nf],
                      sizeof(IndexType),
                      hipMemcpyDeviceToHost,
                      stream);
          cudaCheckError()
          ;

          //We need mf
          hipStreamSynchronize(stream);
          cudaCheckError()
          ;
        }
        break;

      case BOTTOMUP:
        fill_unvisited_queue(visited_bmap,
                      vertices_bmap_size,
                      n,
                      unvisited_queue,
                      d_unvisited_cnt,
                      stream,
                      deterministic);

        size_last_unvisited_queue = nu;

        bottom_up_main(unvisited_queue,
                  size_last_unvisited_queue,
                  left_unvisited_queue,
                  d_left_unvisited_cnt,
                  visited_bmap,
                  row_offsets,
                  col_indices,
                  lvl,
                  new_frontier,
                  d_new_frontier_cnt,
                  distances,
                  predecessors,
                  edge_mask,
                  stream,
                  deterministic);

        //The number of vertices left unvisited decreases
        //If it wasnt necessary last time, it wont be this time
        if (size_last_left_unvisited_queue) {
          hipMemcpyAsync(  &size_last_left_unvisited_queue,
                      d_left_unvisited_cnt,
                      sizeof(IndexType),
                      hipMemcpyDeviceToHost,
                      stream);
          cudaCheckError()
          ;
          //We need last_left_unvisited_size
          hipStreamSynchronize(stream);
          cudaCheckError()
          ;
          bottom_up_large(  left_unvisited_queue,
                      size_last_left_unvisited_queue,
                      visited_bmap,
                      row_offsets,
                      col_indices,
                      lvl,
                      new_frontier,
                      d_new_frontier_cnt,
                      distances,
                      predecessors,
                      edge_mask,
                      stream,
                      deterministic);
        }
        hipMemcpyAsync(  &nf,
                    d_new_frontier_cnt,
                    sizeof(IndexType),
                    hipMemcpyDeviceToHost,
                    stream);
        cudaCheckError()
        ;

        //We will need nf
        hipStreamSynchronize(stream);
        cudaCheckError()
        ;

        break;
      }

      //Updating undiscovered edges count
      nu -= nf;

      //Using new frontier
      frontier = new_frontier;
      growing = (nf > old_nf);

      ++lvl;
    }

    cudaCheckError()
    ;
    return NVGRAPH_OK;
  }

  //Just used for benchmarks now
  template<typename IndexType>
  NVGRAPH_ERROR Bfs<IndexType>::traverse(IndexType *source_vertices, IndexType nsources) {
    for (IndexType i = 0; i < nsources; ++i)
      traverse(source_vertices[i]);

    return NVGRAPH_OK;
  }

  template<typename IndexType>
  void Bfs<IndexType>::resetDevicePointers() {
    hipMemsetAsync(d_counters_pad, 0, 4 * sizeof(IndexType), stream);
    cudaCheckError()
    ;
  }

  template<typename IndexType>
  void Bfs<IndexType>::clean() {
    cudaCheckError()
    ;

    //the vectors have a destructor that takes care of cleaning
    RMM_FREE(original_frontier, stream);//Better to be error checked, but we do not have a policy for error checking yet (in particular for void functions), so I defer error check as future work.
    RMM_FREE(visited_bmap, stream);//Better to be error checked, but we do not have a policy for error checking yet (in particular for void functions), so I defer error check as future work.
    RMM_FREE(isolated_bmap, stream);//Better to be error checked, but we do not have a policy for error checking yet (in particular for void functions), so I defer error check as future work.
    RMM_FREE(vertex_degree, stream);//Better to be error checked, but we do not have a policy for error checking yet (in particular for void functions), so I defer error check as future work.
    RMM_FREE(d_cub_exclusive_sum_storage, stream);//Better to be error checked, but we do not have a policy for error checking yet (in particular for void functions), so I defer error check as future work.
    RMM_FREE(buffer_np1_1, stream);//Better to be error checked, but we do not have a policy for error checking yet (in particular for void functions), so I defer error check as future work.
    RMM_FREE(buffer_np1_2, stream);//Better to be error checked, but we do not have a policy for error checking yet (in particular for void functions), so I defer error check as future work.
    RMM_FREE(exclusive_sum_frontier_vertex_buckets_offsets, stream);//Better to be error checked, but we do not have a policy for error checking yet (in particular for void functions), so I defer error check as future work.
    RMM_FREE(d_counters_pad, stream);//Better to be error checked, but we do not have a policy for error checking yet (in particular for void functions), so I defer error check as future work.

    //In that case, distances is a working data
    if (directed && !computeDistances)
      RMM_FREE(distances, stream);//Better to be error checked, but we do not have a policy for error checking yet (in particular for void functions), so I defer error check as future work.

    cudaCheckError()
    ;
  }

  template class Bfs<int> ;
} // end namespace nvgraph
