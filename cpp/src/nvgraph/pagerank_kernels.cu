#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <thrust/device_vector.h>
#include <algorithm>

#include "include/nvgraph_error.hxx"
#include "include/nvgraph_vector_kernels.hxx"
#include "include/pagerank_kernels.hxx"

namespace nvgraph {

template <typename ValueType_>
__global__ void update_dn_kernel(int num_vertices, ValueType_* aa, ValueType_ beta)
{
  int tidx = blockDim.x * blockIdx.x + threadIdx.x;
  for (int r = tidx; r < num_vertices; r += blockDim.x * gridDim.x) {
    // NOTE 1 : a = alpha*a + (1-alpha)e
    if (aa[r] == 0.0) aa[r] = beta;  // NOTE 2 : alpha*0 + (1-alpha)*1 = (1-alpha)
  }
}

template <typename ValueType_>
void update_dangling_nodes(int num_vertices,
                           ValueType_* dangling_nodes,
                           ValueType_ damping_factor,
                           hipStream_t stream)
{
  int num_threads   = 256;
  int max_grid_size = 4096;
  int num_blocks    = std::min(max_grid_size, (num_vertices / num_threads) + 1);
  ValueType_ beta   = 1.0 - damping_factor;
  update_dn_kernel<<<num_blocks, num_threads, 0, stream>>>(num_vertices, dangling_nodes, beta);
  cudaCheckError();
}

// Explicit

template void update_dangling_nodes<double>(int num_vertices,
                                            double* dangling_nodes,
                                            double damping_factor,
                                            hipStream_t stream);
template void update_dangling_nodes<float>(int num_vertices,
                                           float* dangling_nodes,
                                           float damping_factor,
                                           hipStream_t stream);
}  // end namespace nvgraph
