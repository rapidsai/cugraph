/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithms.hpp>
#include <experimental/graph_view.hpp>
#include <graph.hpp>
#include <partition_manager.hpp>
#include <raft/handle.hpp>
#include <utilities/cython.hpp>
#include <utilities/error.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <thrust/copy.h>
#include <thrust/iterator/counting_iterator.h>

namespace cugraph {
namespace cython {

namespace detail {

// FIXME: Add description of this function
template <typename vertex_t,
          typename edge_t,
          typename weight_t,
          bool transposed,
          bool multi_gpu,
          std::enable_if_t<multi_gpu>* = nullptr>
std::unique_ptr<experimental::graph_t<vertex_t, edge_t, weight_t, transposed, multi_gpu>>
create_graph(raft::handle_t const& handle, graph_container_t const& graph_container)
{
  std::vector<experimental::edgelist_t<vertex_t, edge_t, weight_t>> edgelist(
    {{reinterpret_cast<vertex_t*>(graph_container.src_vertices),
      reinterpret_cast<vertex_t*>(graph_container.dst_vertices),
      reinterpret_cast<weight_t*>(graph_container.weights),
      static_cast<edge_t>(graph_container.num_partition_edges)}});

  std::vector<vertex_t> partition_offsets_vector(
    reinterpret_cast<vertex_t*>(graph_container.vertex_partition_offsets),
    reinterpret_cast<vertex_t*>(graph_container.vertex_partition_offsets) +
      (graph_container.row_comm_size * graph_container.col_comm_size) + 1);

  experimental::partition_t<vertex_t> partition(partition_offsets_vector,
                                                graph_container.hypergraph_partitioned,
                                                graph_container.row_comm_size,
                                                graph_container.col_comm_size,
                                                graph_container.row_comm_rank,
                                                graph_container.col_comm_rank);

  return std::make_unique<experimental::graph_t<vertex_t, edge_t, weight_t, transposed, multi_gpu>>(
    handle,
    edgelist,
    partition,
    static_cast<vertex_t>(graph_container.num_global_vertices),
    static_cast<edge_t>(graph_container.num_global_edges),
    graph_container.graph_props,
    // FIXME:  This currently fails if sorted_by_degree is true...
    // graph_container.sorted_by_degree,
    false,
    graph_container.do_expensive_check);
}

template <typename vertex_t,
          typename edge_t,
          typename weight_t,
          bool transposed,
          bool multi_gpu,
          std::enable_if_t<!multi_gpu>* = nullptr>
std::unique_ptr<experimental::graph_t<vertex_t, edge_t, weight_t, transposed, multi_gpu>>
create_graph(raft::handle_t const& handle, graph_container_t const& graph_container)
{
  experimental::edgelist_t<vertex_t, edge_t, weight_t> edgelist{
    reinterpret_cast<vertex_t*>(graph_container.src_vertices),
    reinterpret_cast<vertex_t*>(graph_container.dst_vertices),
    reinterpret_cast<weight_t*>(graph_container.weights),
    static_cast<edge_t>(graph_container.num_partition_edges)};

  return std::make_unique<experimental::graph_t<vertex_t, edge_t, weight_t, transposed, multi_gpu>>(
    handle,
    edgelist,
    static_cast<vertex_t>(graph_container.num_global_vertices),
    graph_container.graph_props,
    graph_container.sorted_by_degree,
    graph_container.do_expensive_check);
}

}  // namespace detail

// Populates a graph_container_t with a pointer to a new graph object and sets
// the meta-data accordingly.  The graph container owns the pointer and it is
// assumed it will delete it on destruction.
void populate_graph_container(graph_container_t& graph_container,
                              raft::handle_t& handle,
                              void* src_vertices,
                              void* dst_vertices,
                              void* weights,
                              void* vertex_partition_offsets,
                              numberTypeEnum vertexType,
                              numberTypeEnum edgeType,
                              numberTypeEnum weightType,
                              size_t num_partition_edges,
                              size_t num_global_vertices,
                              size_t num_global_edges,
                              bool sorted_by_degree,
                              bool transposed,
                              bool multi_gpu)
{
  CUGRAPH_EXPECTS(graph_container.graph_type == graphTypeEnum::null,
                  "populate_graph_container() can only be called on an empty container.");

  bool do_expensive_check{false};
  bool hypergraph_partitioned{false};

  auto& row_comm           = handle.get_subcomm(cugraph::partition_2d::key_naming_t().row_name());
  auto const row_comm_rank = row_comm.get_rank();
  auto const row_comm_size = row_comm.get_size();  // pcols
  auto& col_comm           = handle.get_subcomm(cugraph::partition_2d::key_naming_t().col_name());
  auto const col_comm_rank = col_comm.get_rank();
  auto const col_comm_size = col_comm.get_size();  // prows

  graph_container.vertex_partition_offsets = vertex_partition_offsets;
  graph_container.src_vertices             = src_vertices;
  graph_container.dst_vertices             = dst_vertices;
  graph_container.weights                  = weights;
  graph_container.num_partition_edges      = num_partition_edges;
  graph_container.num_global_vertices      = num_global_vertices;
  graph_container.num_global_edges         = num_global_edges;
  graph_container.vertexType               = vertexType;
  graph_container.edgeType                 = edgeType;
  graph_container.weightType               = weightType;
  graph_container.transposed               = transposed;
  graph_container.is_multi_gpu             = multi_gpu;
  graph_container.hypergraph_partitioned   = hypergraph_partitioned;
  graph_container.row_comm_size            = row_comm_size;
  graph_container.col_comm_size            = col_comm_size;
  graph_container.row_comm_rank            = row_comm_rank;
  graph_container.col_comm_rank            = col_comm_rank;
  graph_container.sorted_by_degree         = sorted_by_degree;
  graph_container.do_expensive_check       = do_expensive_check;

  experimental::graph_properties_t graph_props{.is_symmetric = false, .is_multigraph = false};
  graph_container.graph_props = graph_props;

  graph_container.graph_type = graphTypeEnum::graph_t;
}

void populate_graph_container_legacy(graph_container_t& graph_container,
                                     graphTypeEnum legacyType,
                                     raft::handle_t const& handle,
                                     void* offsets,
                                     void* indices,
                                     void* weights,
                                     numberTypeEnum offsetType,
                                     numberTypeEnum indexType,
                                     numberTypeEnum weightType,
                                     size_t num_global_vertices,
                                     size_t num_global_edges,
                                     int* local_vertices,
                                     int* local_edges,
                                     int* local_offsets)
{
  CUGRAPH_EXPECTS(graph_container.graph_type == graphTypeEnum::null,
                  "populate_graph_container() can only be called on an empty container.");

  // FIXME: This is soon-to-be legacy code left in place until the new graph_t
  // class is supported everywhere else. Remove everything down to the comment
  // line after the return stmnt.
  // Keep new code below return stmnt enabled to ensure it builds.
  if (weightType == numberTypeEnum::floatType) {
    switch (legacyType) {
      case graphTypeEnum::LegacyCSR: {
        graph_container.graph_ptr_union.GraphCSRViewFloatPtr =
          std::make_unique<GraphCSRView<int, int, float>>(reinterpret_cast<int*>(offsets),
                                                          reinterpret_cast<int*>(indices),
                                                          reinterpret_cast<float*>(weights),
                                                          num_global_vertices,
                                                          num_global_edges);
        graph_container.graph_type = graphTypeEnum::GraphCSRViewFloat;
        (graph_container.graph_ptr_union.GraphCSRViewFloatPtr)
          ->set_local_data(local_vertices, local_edges, local_offsets);
        (graph_container.graph_ptr_union.GraphCSRViewFloatPtr)
          ->set_handle(const_cast<raft::handle_t*>(&handle));
      } break;
      case graphTypeEnum::LegacyCSC: {
        graph_container.graph_ptr_union.GraphCSCViewFloatPtr =
          std::make_unique<GraphCSCView<int, int, float>>(reinterpret_cast<int*>(offsets),
                                                          reinterpret_cast<int*>(indices),
                                                          reinterpret_cast<float*>(weights),
                                                          num_global_vertices,
                                                          num_global_edges);
        graph_container.graph_type = graphTypeEnum::GraphCSCViewFloat;
        (graph_container.graph_ptr_union.GraphCSCViewFloatPtr)
          ->set_local_data(local_vertices, local_edges, local_offsets);
        (graph_container.graph_ptr_union.GraphCSCViewFloatPtr)
          ->set_handle(const_cast<raft::handle_t*>(&handle));
      } break;
      case graphTypeEnum::LegacyCOO: {
        graph_container.graph_ptr_union.GraphCOOViewFloatPtr =
          std::make_unique<GraphCOOView<int, int, float>>(reinterpret_cast<int*>(offsets),
                                                          reinterpret_cast<int*>(indices),
                                                          reinterpret_cast<float*>(weights),
                                                          num_global_vertices,
                                                          num_global_edges);
        graph_container.graph_type = graphTypeEnum::GraphCOOViewFloat;
        (graph_container.graph_ptr_union.GraphCOOViewFloatPtr)
          ->set_local_data(local_vertices, local_edges, local_offsets);
        (graph_container.graph_ptr_union.GraphCOOViewFloatPtr)
          ->set_handle(const_cast<raft::handle_t*>(&handle));
      } break;
      default: CUGRAPH_FAIL("unsupported graphTypeEnum value"); break;
    }

  } else {
    switch (legacyType) {
      case graphTypeEnum::LegacyCSR: {
        graph_container.graph_ptr_union.GraphCSRViewDoublePtr =
          std::make_unique<GraphCSRView<int, int, double>>(reinterpret_cast<int*>(offsets),
                                                           reinterpret_cast<int*>(indices),
                                                           reinterpret_cast<double*>(weights),
                                                           num_global_vertices,
                                                           num_global_edges);
        graph_container.graph_type = graphTypeEnum::GraphCSRViewDouble;
        (graph_container.graph_ptr_union.GraphCSRViewDoublePtr)
          ->set_local_data(local_vertices, local_edges, local_offsets);
        (graph_container.graph_ptr_union.GraphCSRViewDoublePtr)
          ->set_handle(const_cast<raft::handle_t*>(&handle));
      } break;
      case graphTypeEnum::LegacyCSC: {
        graph_container.graph_ptr_union.GraphCSCViewDoublePtr =
          std::make_unique<GraphCSCView<int, int, double>>(reinterpret_cast<int*>(offsets),
                                                           reinterpret_cast<int*>(indices),
                                                           reinterpret_cast<double*>(weights),
                                                           num_global_vertices,
                                                           num_global_edges);
        graph_container.graph_type = graphTypeEnum::GraphCSCViewDouble;
        (graph_container.graph_ptr_union.GraphCSCViewDoublePtr)
          ->set_local_data(local_vertices, local_edges, local_offsets);
        (graph_container.graph_ptr_union.GraphCSCViewDoublePtr)
          ->set_handle(const_cast<raft::handle_t*>(&handle));
      } break;
      case graphTypeEnum::LegacyCOO: {
        graph_container.graph_ptr_union.GraphCOOViewDoublePtr =
          std::make_unique<GraphCOOView<int, int, double>>(reinterpret_cast<int*>(offsets),
                                                           reinterpret_cast<int*>(indices),
                                                           reinterpret_cast<double*>(weights),
                                                           num_global_vertices,
                                                           num_global_edges);
        graph_container.graph_type = graphTypeEnum::GraphCOOViewDouble;
        (graph_container.graph_ptr_union.GraphCOOViewDoublePtr)
          ->set_local_data(local_vertices, local_edges, local_offsets);
        (graph_container.graph_ptr_union.GraphCOOViewDoublePtr)
          ->set_handle(const_cast<raft::handle_t*>(&handle));
      } break;
      default: CUGRAPH_FAIL("unsupported graphTypeEnum value"); break;
    }
  }
  return;
}

////////////////////////////////////////////////////////////////////////////////

namespace detail {

// Final, fully-templatized call.
template <bool transposed,
          typename return_t,
          typename function_t,
          typename vertex_t,
          typename edge_t,
          typename weight_t,
          bool is_multi_gpu>
return_t call_function(raft::handle_t const& handle,
                       graph_container_t const& graph_container,
                       function_t function)
{
  auto graph =
    create_graph<vertex_t, edge_t, weight_t, transposed, is_multi_gpu>(handle, graph_container);

  return function(handle, graph->view());
}

// Makes another call based on vertex_t and edge_t
template <bool transposed,
          typename return_t,
          typename function_t,
          typename weight_t,
          bool is_multi_gpu>
return_t call_function(raft::handle_t const& handle,
                       graph_container_t const& graph_container,
                       function_t function)
{
  // Since only vertex/edge types (int32,int32), (int32,int64), and
  // (int64,int64) are being supported, explicitely check for those types and
  // ensure (int64,int32) is rejected as unsupported.
  if ((graph_container.vertexType == numberTypeEnum::int32Type) &&
      (graph_container.edgeType == numberTypeEnum::int32Type)) {
    return call_function<transposed,
                         return_t,
                         function_t,
                         int32_t,
                         int32_t,
                         weight_t,
                         is_multi_gpu>(handle, graph_container, function);
  } else if ((graph_container.vertexType == numberTypeEnum::int32Type) &&
             (graph_container.edgeType == numberTypeEnum::int64Type)) {
    return call_function<transposed,
                         return_t,
                         function_t,
                         int32_t,
                         int64_t,
                         weight_t,
                         is_multi_gpu>(handle, graph_container, function);
  } else if ((graph_container.vertexType == numberTypeEnum::int64Type) &&
             (graph_container.edgeType == numberTypeEnum::int64Type)) {
    return call_function<transposed,
                         return_t,
                         function_t,
                         int64_t,
                         int64_t,
                         weight_t,
                         is_multi_gpu>(handle, graph_container, function);
  } else {
    CUGRAPH_FAIL("vertexType/edgeType combination unsupported");
  }
}

// Makes another call based on weight_t
template <bool transposed, typename return_t, typename function_t, bool is_multi_gpu>
return_t call_function(raft::handle_t const& handle,
                       graph_container_t const& graph_container,
                       function_t function)
{
  if (graph_container.weightType == numberTypeEnum::floatType) {
    return call_function<transposed, return_t, function_t, float, is_multi_gpu>(
      handle, graph_container, function);
  } else if (graph_container.weightType == numberTypeEnum::doubleType) {
    return call_function<transposed, return_t, function_t, double, is_multi_gpu>(
      handle, graph_container, function);
  } else {
    CUGRAPH_FAIL("weightType unsupported");
  }
}

// Makes another call based on multi_gpu
template <bool transposed, typename return_t, typename function_t>
return_t call_function(raft::handle_t const& handle,
                       graph_container_t const& graph_container,
                       function_t function)
{
  if (graph_container.is_multi_gpu) {
    return call_function<transposed, return_t, function_t, true>(handle, graph_container, function);
  } else {
    return call_function<transposed, return_t, function_t, false>(
      handle, graph_container, function);
  }
}

// Initial call_function() call starts here.
// This makes another call based on transposed
template <typename return_t, typename function_t>
return_t call_function(raft::handle_t const& handle,
                       graph_container_t const& graph_container,
                       function_t function)
{
  if (graph_container.transposed) {
    return call_function<true, return_t, function_t>(handle, graph_container, function);
  } else {
    return call_function<false, return_t, function_t>(handle, graph_container, function);
  }
}

template <typename weight_t>
class louvain_functor {
 public:
  louvain_functor(void* identifiers, void* parts, size_t max_level, weight_t resolution)
    : identifiers_(identifiers), parts_(parts), max_level_(max_level), resolution_(resolution)
  {
  }

  template <typename graph_view_t>
  std::pair<size_t, weight_t> operator()(raft::handle_t const& handle,
                                         graph_view_t const& graph_view)
  {
    thrust::copy(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                 thrust::make_counting_iterator(graph_view.get_local_vertex_first()),
                 thrust::make_counting_iterator(graph_view.get_local_vertex_last()),
                 reinterpret_cast<typename graph_view_t::vertex_type*>(identifiers_));

    return cugraph::louvain(handle,
                            graph_view,
                            reinterpret_cast<typename graph_view_t::vertex_type*>(parts_),
                            max_level_,
                            resolution_);
  }

 private:
  void* identifiers_;  // FIXME: this will be used in a future PR
  void* parts_;
  size_t max_level_;
  weight_t resolution_;
};

}  // namespace detail

// Wrapper for calling Louvain using a graph container
template <typename weight_t>
std::pair<size_t, weight_t> call_louvain(raft::handle_t const& handle,
                                         graph_container_t const& graph_container,
                                         void* identifiers,
                                         void* parts,
                                         size_t max_level,
                                         weight_t resolution)
{
  // LEGACY PATH - remove when migration to graph_t types complete
  if (graph_container.graph_type == graphTypeEnum::GraphCSRViewFloat) {
    graph_container.graph_ptr_union.GraphCSRViewFloatPtr->get_vertex_identifiers(
      static_cast<int32_t*>(identifiers));
    return louvain(handle,
                   *(graph_container.graph_ptr_union.GraphCSRViewFloatPtr),
                   reinterpret_cast<int32_t*>(parts),
                   max_level,
                   static_cast<float>(resolution));
  } else if (graph_container.graph_type == graphTypeEnum::GraphCSRViewDouble) {
    graph_container.graph_ptr_union.GraphCSRViewDoublePtr->get_vertex_identifiers(
      static_cast<int32_t*>(identifiers));
    return louvain(handle,
                   *(graph_container.graph_ptr_union.GraphCSRViewDoublePtr),
                   reinterpret_cast<int32_t*>(parts),
                   max_level,
                   static_cast<double>(resolution));
  }

  // NON-LEGACY PATH
  detail::louvain_functor<weight_t> functor{identifiers, parts, max_level, resolution};

  return detail::call_function<false, std::pair<size_t, weight_t>>(
    handle, graph_container, functor);
}

// Wrapper for calling Pagerank through a graph container
template <typename vertex_t, typename weight_t>
void call_pagerank(raft::handle_t const& handle,
                   graph_container_t const& graph_container,
                   vertex_t* identifiers,
                   weight_t* p_pagerank,
                   vertex_t personalization_subset_size,
                   vertex_t* personalization_subset,
                   weight_t* personalization_values,
                   double alpha,
                   double tolerance,
                   int64_t max_iter,
                   bool has_guess)
{
  if (graph_container.graph_type == graphTypeEnum::GraphCSCViewFloat) {
    pagerank(handle,
             *(graph_container.graph_ptr_union.GraphCSCViewFloatPtr),
             reinterpret_cast<float*>(p_pagerank),
             static_cast<int32_t>(personalization_subset_size),
             reinterpret_cast<int32_t*>(personalization_subset),
             reinterpret_cast<float*>(personalization_values),
             alpha,
             tolerance,
             max_iter,
             has_guess);
    graph_container.graph_ptr_union.GraphCSCViewFloatPtr->get_vertex_identifiers(
      reinterpret_cast<int32_t*>(identifiers));
  } else if (graph_container.graph_type == graphTypeEnum::GraphCSCViewDouble) {
    pagerank(handle,
             *(graph_container.graph_ptr_union.GraphCSCViewDoublePtr),
             reinterpret_cast<double*>(p_pagerank),
             static_cast<int32_t>(personalization_subset_size),
             reinterpret_cast<int32_t*>(personalization_subset),
             reinterpret_cast<double*>(personalization_values),
             alpha,
             tolerance,
             max_iter,
             has_guess);
    graph_container.graph_ptr_union.GraphCSCViewDoublePtr->get_vertex_identifiers(
      reinterpret_cast<int32_t*>(identifiers));
  } else if (graph_container.graph_type == graphTypeEnum::graph_t) {
    if (graph_container.edgeType == numberTypeEnum::int32Type) {
      auto graph =
        detail::create_graph<int32_t, int32_t, weight_t, true, true>(handle, graph_container);
      cugraph::experimental::pagerank(handle,
                                      graph->view(),
                                      static_cast<weight_t*>(nullptr),
                                      reinterpret_cast<int32_t*>(personalization_subset),
                                      reinterpret_cast<weight_t*>(personalization_values),
                                      static_cast<int32_t>(personalization_subset_size),
                                      reinterpret_cast<weight_t*>(p_pagerank),
                                      static_cast<weight_t>(alpha),
                                      static_cast<weight_t>(tolerance),
                                      max_iter,
                                      has_guess,
                                      false);
    } else if (graph_container.edgeType == numberTypeEnum::int64Type) {
      auto graph =
        detail::create_graph<vertex_t, int64_t, weight_t, true, true>(handle, graph_container);
      cugraph::experimental::pagerank(handle,
                                      graph->view(),
                                      static_cast<weight_t*>(nullptr),
                                      reinterpret_cast<vertex_t*>(personalization_subset),
                                      reinterpret_cast<weight_t*>(personalization_values),
                                      static_cast<vertex_t>(personalization_subset_size),
                                      reinterpret_cast<weight_t*>(p_pagerank),
                                      static_cast<weight_t>(alpha),
                                      static_cast<weight_t>(tolerance),
                                      max_iter,
                                      has_guess,
                                      false);
    } else {
      CUGRAPH_FAIL("vertexType/edgeType combination unsupported");
    }
  }
}

// Wrapper for calling BFS through a graph container
template <typename vertex_t, typename weight_t>
void call_bfs(raft::handle_t const& handle,
              graph_container_t const& graph_container,
              vertex_t* identifiers,
              vertex_t* distances,
              vertex_t* predecessors,
              double* sp_counters,
              const vertex_t start_vertex,
              bool directed)
{
  if (graph_container.graph_type == graphTypeEnum::GraphCSRViewFloat) {
    graph_container.graph_ptr_union.GraphCSRViewFloatPtr->get_vertex_identifiers(
      reinterpret_cast<int32_t*>(identifiers));
    bfs(handle,
        *(graph_container.graph_ptr_union.GraphCSRViewFloatPtr),
        reinterpret_cast<int32_t*>(distances),
        reinterpret_cast<int32_t*>(predecessors),
        sp_counters,
        static_cast<int32_t>(start_vertex),
        directed);
  } else if (graph_container.graph_type == graphTypeEnum::GraphCSRViewDouble) {
    graph_container.graph_ptr_union.GraphCSRViewDoublePtr->get_vertex_identifiers(
      reinterpret_cast<int32_t*>(identifiers));
    bfs(handle,
        *(graph_container.graph_ptr_union.GraphCSRViewDoublePtr),
        reinterpret_cast<int32_t*>(distances),
        reinterpret_cast<int32_t*>(predecessors),
        sp_counters,
        static_cast<int32_t>(start_vertex),
        directed);
  } else if (graph_container.graph_type == graphTypeEnum::graph_t) {
    if (graph_container.edgeType == numberTypeEnum::int32Type) {
      auto graph =
        detail::create_graph<int32_t, int32_t, weight_t, false, true>(handle, graph_container);
      cugraph::experimental::bfs(handle,
                                 graph->view(),
                                 reinterpret_cast<int32_t*>(distances),
                                 reinterpret_cast<int32_t*>(predecessors),
                                 static_cast<int32_t>(start_vertex));
    } else if (graph_container.edgeType == numberTypeEnum::int64Type) {
      auto graph =
        detail::create_graph<vertex_t, int64_t, weight_t, false, true>(handle, graph_container);
      cugraph::experimental::bfs(handle,
                                 graph->view(),
                                 reinterpret_cast<vertex_t*>(distances),
                                 reinterpret_cast<vertex_t*>(predecessors),
                                 static_cast<vertex_t>(start_vertex));
    } else {
      CUGRAPH_FAIL("vertexType/edgeType combination unsupported");
    }
  }
}

// Wrapper for calling SSSP through a graph container
template <typename vertex_t, typename weight_t>
void call_sssp(raft::handle_t const& handle,
               graph_container_t const& graph_container,
               vertex_t* identifiers,
               weight_t* distances,
               vertex_t* predecessors,
               const vertex_t source_vertex)
{
  if (graph_container.graph_type == graphTypeEnum::GraphCSRViewFloat) {
    graph_container.graph_ptr_union.GraphCSRViewFloatPtr->get_vertex_identifiers(
      reinterpret_cast<int32_t*>(identifiers));
    sssp(  // handle, TODO: clarify: no raft_handle_t? why?
      *(graph_container.graph_ptr_union.GraphCSRViewFloatPtr),
      reinterpret_cast<float*>(distances),
      reinterpret_cast<int32_t*>(predecessors),
      static_cast<int32_t>(source_vertex));
  } else if (graph_container.graph_type == graphTypeEnum::GraphCSRViewDouble) {
    graph_container.graph_ptr_union.GraphCSRViewDoublePtr->get_vertex_identifiers(
      reinterpret_cast<int32_t*>(identifiers));
    sssp(  // handle, TODO: clarify: no raft_handle_t? why?
      *(graph_container.graph_ptr_union.GraphCSRViewDoublePtr),
      reinterpret_cast<double*>(distances),
      reinterpret_cast<int32_t*>(predecessors),
      static_cast<int32_t>(source_vertex));
  } else if (graph_container.graph_type == graphTypeEnum::graph_t) {
    if (graph_container.edgeType == numberTypeEnum::int32Type) {
      auto graph =
        detail::create_graph<int32_t, int32_t, weight_t, false, true>(handle, graph_container);
      cugraph::experimental::sssp(handle,
                                  graph->view(),
                                  reinterpret_cast<weight_t*>(distances),
                                  reinterpret_cast<int32_t*>(predecessors),
                                  static_cast<int32_t>(source_vertex));
    } else if (graph_container.edgeType == numberTypeEnum::int64Type) {
      auto graph =
        detail::create_graph<vertex_t, int64_t, weight_t, false, true>(handle, graph_container);
      cugraph::experimental::sssp(handle,
                                  graph->view(),
                                  reinterpret_cast<weight_t*>(distances),
                                  reinterpret_cast<vertex_t*>(predecessors),
                                  static_cast<vertex_t>(source_vertex));
    } else {
      CUGRAPH_FAIL("vertexType/edgeType combination unsupported");
    }
  }
}

// Helper for setting up subcommunicators
void init_subcomms(raft::handle_t& handle, size_t row_comm_size)
{
  partition_2d::subcomm_factory_t<partition_2d::key_naming_t, int> subcomm_factory(handle,
                                                                                   row_comm_size);
}

// Explicit instantiations

template std::pair<size_t, float> call_louvain(raft::handle_t const& handle,
                                               graph_container_t const& graph_container,
                                               void* identifiers,
                                               void* parts,
                                               size_t max_level,
                                               float resolution);

template std::pair<size_t, double> call_louvain(raft::handle_t const& handle,
                                                graph_container_t const& graph_container,
                                                void* identifiers,
                                                void* parts,
                                                size_t max_level,
                                                double resolution);

template void call_pagerank(raft::handle_t const& handle,
                            graph_container_t const& graph_container,
                            int* identifiers,
                            float* p_pagerank,
                            int32_t personalization_subset_size,
                            int32_t* personalization_subset,
                            float* personalization_values,
                            double alpha,
                            double tolerance,
                            int64_t max_iter,
                            bool has_guess);

template void call_pagerank(raft::handle_t const& handle,
                            graph_container_t const& graph_container,
                            int* identifiers,
                            double* p_pagerank,
                            int32_t personalization_subset_size,
                            int32_t* personalization_subset,
                            double* personalization_values,
                            double alpha,
                            double tolerance,
                            int64_t max_iter,
                            bool has_guess);

template void call_pagerank(raft::handle_t const& handle,
                            graph_container_t const& graph_container,
                            int64_t* identifiers,
                            float* p_pagerank,
                            int64_t personalization_subset_size,
                            int64_t* personalization_subset,
                            float* personalization_values,
                            double alpha,
                            double tolerance,
                            int64_t max_iter,
                            bool has_guess);

template void call_pagerank(raft::handle_t const& handle,
                            graph_container_t const& graph_container,
                            int64_t* identifiers,
                            double* p_pagerank,
                            int64_t personalization_subset_size,
                            int64_t* personalization_subset,
                            double* personalization_values,
                            double alpha,
                            double tolerance,
                            int64_t max_iter,
                            bool has_guess);

template void call_bfs<int32_t, float>(raft::handle_t const& handle,
                                       graph_container_t const& graph_container,
                                       int32_t* identifiers,
                                       int32_t* distances,
                                       int32_t* predecessors,
                                       double* sp_counters,
                                       const int32_t start_vertex,
                                       bool directed);

template void call_bfs<int32_t, double>(raft::handle_t const& handle,
                                        graph_container_t const& graph_container,
                                        int32_t* identifiers,
                                        int32_t* distances,
                                        int32_t* predecessors,
                                        double* sp_counters,
                                        const int32_t start_vertex,
                                        bool directed);

template void call_bfs<int64_t, float>(raft::handle_t const& handle,
                                       graph_container_t const& graph_container,
                                       int64_t* identifiers,
                                       int64_t* distances,
                                       int64_t* predecessors,
                                       double* sp_counters,
                                       const int64_t start_vertex,
                                       bool directed);

template void call_bfs<int64_t, double>(raft::handle_t const& handle,
                                        graph_container_t const& graph_container,
                                        int64_t* identifiers,
                                        int64_t* distances,
                                        int64_t* predecessors,
                                        double* sp_counters,
                                        const int64_t start_vertex,
                                        bool directed);

template void call_sssp(raft::handle_t const& handle,
                        graph_container_t const& graph_container,
                        int32_t* identifiers,
                        float* distances,
                        int32_t* predecessors,
                        const int32_t source_vertex);

template void call_sssp(raft::handle_t const& handle,
                        graph_container_t const& graph_container,
                        int32_t* identifiers,
                        double* distances,
                        int32_t* predecessors,
                        const int32_t source_vertex);

template void call_sssp(raft::handle_t const& handle,
                        graph_container_t const& graph_container,
                        int64_t* identifiers,
                        float* distances,
                        int64_t* predecessors,
                        const int64_t source_vertex);

template void call_sssp(raft::handle_t const& handle,
                        graph_container_t const& graph_container,
                        int64_t* identifiers,
                        double* distances,
                        int64_t* predecessors,
                        const int64_t source_vertex);

}  // namespace cython
}  // namespace cugraph
