/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "spmv_1D.cuh"

namespace cugraph {
namespace opg {
template <typename VT, typename ET, typename WT>
OPGcsrmv<VT, ET, WT>::OPGcsrmv(const raft::comms::comms_t &comm_,
                               VT *local_vertices_,
                               VT *part_off_,
                               ET *off_,
                               VT *ind_,
                               WT *val_,
                               WT *x)
  : comm(comm_),
    local_vertices(local_vertices_),
    part_off(part_off_),
    off(off_),
    ind(ind_),
    val(val_)
{
  stream = nullptr;
  i      = comm.get_rank();
  p      = comm.get_size();
  v_glob = part_off[p - 1] + local_vertices[p - 1];
  v_loc  = local_vertices[i];
  VT tmp;
  CUDA_TRY(hipMemcpy(&tmp, &off[v_loc], sizeof(VT), hipMemcpyDeviceToHost));
  e_loc = tmp;
  y_loc.resize(v_loc);
  WT h_one  = 1.0;
  WT h_zero = 0.0;

  spmv.setup(v_loc, v_glob, e_loc, &h_one, val, off, ind, x, &h_zero, y_loc.data().get());
}

template <typename VT, typename ET, typename WT>
OPGcsrmv<VT, ET, WT>::~OPGcsrmv()
{
}

template <typename VT, typename ET, typename WT>
void OPGcsrmv<VT, ET, WT>::run(WT *x)
{
  WT h_one  = 1.0;
  WT h_zero = 0.0;
  spmv.run(v_loc, v_glob, e_loc, &h_one, val, off, ind, x, &h_zero, y_loc.data().get());
  // FIXME https://github.com/rapidsai/raft/issues/21
  size_t recvbuf[comm.get_size()];
  for (int i = 0; i < comm.get_size(); i++) recvbuf[i] = local_vertices[i];
  comm.allgatherv(y_loc.data().get(), x, recvbuf, part_off, stream);
}

template class OPGcsrmv<int, int, double>;
template class OPGcsrmv<int, int, float>;

}  // namespace opg
}  // namespace cugraph
