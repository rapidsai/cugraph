/*
 * Copyright (c) 2021-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "shuffle_vertices.cuh"

namespace cugraph {

namespace detail {

template rmm::device_uvector<int64_t> shuffle_int_vertices_to_local_gpu_by_vertex_partitioning(
  raft::handle_t const& handle,
  rmm::device_uvector<int64_t>&& vertices,
  raft::host_span<int64_t const> vertex_partition_range_lasts,
  std::optional<large_buffer_type_t> large_buffer_type);

template std::tuple<rmm::device_uvector<int64_t>, dataframe_buffer_type_t<int32_t>>
shuffle_int_vertex_value_pairs_to_local_gpu_by_vertex_partitioning<int64_t, int32_t>(
  raft::handle_t const& handle,
  rmm::device_uvector<int64_t>&& d_vertices,
  dataframe_buffer_type_t<int32_t>&& d_values,
  raft::host_span<int64_t const> vertex_partition_range_lasts,
  std::optional<large_buffer_type_t> large_buffer_type);

template std::tuple<rmm::device_uvector<int64_t>, dataframe_buffer_type_t<int64_t>>
shuffle_int_vertex_value_pairs_to_local_gpu_by_vertex_partitioning<int64_t, int64_t>(
  raft::handle_t const& handle,
  rmm::device_uvector<int64_t>&& d_vertices,
  dataframe_buffer_type_t<int64_t>&& d_values,
  raft::host_span<int64_t const> vertex_partition_range_lasts,
  std::optional<large_buffer_type_t> large_buffer_type);

template std::tuple<rmm::device_uvector<int64_t>, dataframe_buffer_type_t<size_t>>
shuffle_int_vertex_value_pairs_to_local_gpu_by_vertex_partitioning<int64_t, size_t>(
  raft::handle_t const& handle,
  rmm::device_uvector<int64_t>&& d_vertices,
  dataframe_buffer_type_t<size_t>&& d_values,
  raft::host_span<int64_t const> vertex_partition_range_lasts,
  std::optional<large_buffer_type_t> large_buffer_type);

template std::tuple<rmm::device_uvector<int64_t>,
                    dataframe_buffer_type_t<cuda::std::tuple<int32_t, int32_t>>>
shuffle_int_vertex_value_pairs_to_local_gpu_by_vertex_partitioning<
  int64_t,
  cuda::std::tuple<int32_t, int32_t>>(
  raft::handle_t const& handle,
  rmm::device_uvector<int64_t>&& d_vertices,
  dataframe_buffer_type_t<cuda::std::tuple<int32_t, int32_t>>&& d_values,
  raft::host_span<int64_t const> vertex_partition_range_lasts,
  std::optional<large_buffer_type_t> large_buffer_type);

template std::tuple<rmm::device_uvector<int64_t>,
                    dataframe_buffer_type_t<cuda::std::tuple<int32_t, int64_t>>>
shuffle_int_vertex_value_pairs_to_local_gpu_by_vertex_partitioning<
  int64_t,
  cuda::std::tuple<int32_t, int64_t>>(
  raft::handle_t const& handle,
  rmm::device_uvector<int64_t>&& d_vertices,
  dataframe_buffer_type_t<cuda::std::tuple<int32_t, int64_t>>&& d_values,
  raft::host_span<int64_t const> vertex_partition_range_lasts,
  std::optional<large_buffer_type_t> large_buffer_type);

template rmm::device_uvector<int64_t> shuffle_ext_vertices_to_local_gpu_by_vertex_partitioning(
  raft::handle_t const& handle,
  rmm::device_uvector<int64_t>&& d_vertices,
  std::optional<large_buffer_type_t> large_buffer_type);

template std::tuple<rmm::device_uvector<int64_t>, rmm::device_uvector<int64_t>>
shuffle_ext_vertex_value_pairs_to_local_gpu_by_vertex_partitioning(
  raft::handle_t const& handle,
  rmm::device_uvector<int64_t>&& vertices,
  rmm::device_uvector<int64_t>&& values,
  std::optional<large_buffer_type_t> large_buffer_type);

}  // namespace detail

template rmm::device_uvector<int64_t> shuffle_ext_vertices(
  raft::handle_t const& handle,
  rmm::device_uvector<int64_t>&& d_vertices,
  std::optional<large_buffer_type_t> large_buffer_type);

template std::tuple<rmm::device_uvector<int64_t>, rmm::device_uvector<int32_t>>
shuffle_ext_vertex_value_pairs(raft::handle_t const& handle,
                               rmm::device_uvector<int64_t>&& vertices,
                               rmm::device_uvector<int32_t>&& values,
                               std::optional<large_buffer_type_t> large_buffer_type);

template std::tuple<rmm::device_uvector<int64_t>, rmm::device_uvector<int64_t>>
shuffle_ext_vertex_value_pairs(raft::handle_t const& handle,
                               rmm::device_uvector<int64_t>&& vertices,
                               rmm::device_uvector<int64_t>&& values,
                               std::optional<large_buffer_type_t> large_buffer_type);

template std::tuple<rmm::device_uvector<int64_t>, rmm::device_uvector<size_t>>
shuffle_ext_vertex_value_pairs(raft::handle_t const& handle,
                               rmm::device_uvector<int64_t>&& vertices,
                               rmm::device_uvector<size_t>&& values,
                               std::optional<large_buffer_type_t> large_buffer_type);

}  // namespace cugraph
