#include <gunrock/app/sm/sm_app.cuh>

#include <thrust/sequence.h>

#include <cugraph.h>
#include "utilities/error_utils.h"
#include "utilities/graph_utils.cuh"

#include <array>
#include <cstdint>
#include <iostream>
#include <type_traits>

//#define _DEBUG_SM_

namespace cugraph {
namespace detail {

//
/**
 * @brief Subgraph matching.
 * API for gunrock implementation.
 *
 * @tparam VertexT the indexing type for vertices
 * @tparam SizeT the type for sizes/dimensions
 * @tparam GValueT the type for edge weights
 * @param  graph_src input source graph (to search into); assumed undirected [in]
 * @param  graph_query input query graph (to search for); assumed undirected [in]
 * @param  subgraphs   Return number of subgraphs [out]
 * @param  stream the cuda stream [in / optional]
 */
template <typename VertexT, typename SizeT, typename GValueT>
void subgraph_matching_impl(Graph* graph_src,
                            Graph* graph_query,
                            VertexT* subgraphs,
                            hipStream_t stream = nullptr)
{
  static auto row_offsets_ = [](const Graph* G) {
    return static_cast<const SizeT*>(G->adjList->offsets->data);
  };

  static auto col_indices_ = [](const Graph* G) {
    return static_cast<const VertexT*>(G->adjList->indices->data);
  };

  static auto values_ = [](const Graph* G) {
    return static_cast<const GValueT*>(G->adjList->edge_data->data);
  };

  static auto nrows_ = [](const Graph* G) {
    return static_cast<SizeT>(G->adjList->offsets->size - 1);
  };

  static auto nnz_ = [](const Graph* G) { return static_cast<SizeT>(G->adjList->indices->size); };
  std::array<Graph*, 2> arr_graph = {graph_src, graph_query};

  // check consistency of both graphs:
  //
  for (auto&& graph : arr_graph) {
    CUGRAPH_EXPECTS(graph != nullptr, "Invalid API parameter");

    CUGRAPH_EXPECTS(graph->adjList != nullptr, "Invalid API parameter");

    CUGRAPH_EXPECTS(row_offsets_(graph) != nullptr, "Invalid API parameter");

    CUGRAPH_EXPECTS(col_indices_(graph) != nullptr, "Invalid API parameter");

    auto type_id = graph->adjList->offsets->dtype;
    CUGRAPH_EXPECTS(type_id == GDF_INT32 || type_id == GDF_INT64, "Unsupported data type");

    CUGRAPH_EXPECTS(type_id == graph->adjList->indices->dtype, "Unsupported data type");

    const SizeT* p_d_row_offsets = row_offsets_(graph);
    const VertexT* p_d_col_ind   = col_indices_(graph);
    const GValueT* p_d_values    = values_(graph);

    assert(p_d_values);

    SizeT nnz   = nnz_(graph);
    SizeT nrows = nrows_(graph);
  }

  // TODO: call into proper Gunrock API (non-existent, yet)
  //
  // below is the wrong API to call;
  // Gunrock has yet to properly expose one...
  //
  // auto t_elapsed = sm(nrows,
  //                     nnz,
  //                     p_d_row_offsets,
  //                     p_d_col_ind,
  //                     p_d_values,
  //                     1,
  //                     subgraphs);
}

}  // namespace detail

/**
 * @brief Subgraph matching.
 * API for gunrock implementation.
 *
 * @param  graph_src input source graph (to search into); assumed undirected [in]
 * @param  graph_query input query graph (to search for); assumed undirected [in]
 * @param  subgraphs   Return number of matched subgraphs [out]
 */
void subgraph_matching(Graph* graph_src, Graph* graph_query, gdf_column* subgraphs)

{
  static auto row_offsets_t_ = [](const Graph* G) { return G->adjList->offsets->dtype; };

  static auto col_indices_t_ = [](const Graph* G) { return G->adjList->indices->dtype; };

  static auto values_t_ = [](const Graph* G) { return G->adjList->edge_data->dtype; };

  auto subg_dtype = subgraphs->dtype;
  // auto ro_dtype   = row_offsets_t_(graph_src);//not yet necessary...possibly later, when smoke
  // clears out
  auto ci_src_dtype = col_indices_t_(graph_src);
  auto ci_qry_dtype = col_indices_t_(graph_query);
  // auto v_dtype    = values_t_(graph_src);//not yet necessary...possibly later, when smoke clears
  // out

  // currently Gunrock's API requires that graph's col indices and subgraphs must be same type:
  //
  CUGRAPH_EXPECTS(subg_dtype == ci_src_dtype, "Invalid API parameter");
  CUGRAPH_EXPECTS(subg_dtype == ci_qry_dtype, "Invalid API parameter");

  // TODO: hopefully multi-type-dispatch on various combos of types:
  //
  int* p_d_subg = static_cast<int*>(subgraphs->data);
  return detail::subgraph_matching_impl<int, int, unsigned long>(graph_src, graph_query, p_d_subg);
}

}  // namespace cugraph