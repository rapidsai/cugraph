#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
/** ---------------------------------------------------------------------------*
 * @brief Functions for computing the two hop neighbor pairs of a graph
 *
 * @file two_hop_neighbors.cu
 * ---------------------------------------------------------------------------**/

#include "two_hop_neighbors.cuh"
#include "utilities/error_utils.h"
#include <rmm_utils.h>

#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/execution_policy.h>

template<typename IndexType>
gdf_error gdf_get_two_hop_neighbors_impl(IndexType num_verts,
                                         IndexType* offsets,
                                         IndexType* indices,
                                         IndexType** first,
                                         IndexType** second,
                                         IndexType& outputSize) {
    // Get the number of edges from the adjacency representation
    IndexType num_edges;
    hipMemcpy(&num_edges, &offsets[num_verts], sizeof(IndexType), hipMemcpyDefault);

    // Allocate memory for temporary stuff
    IndexType *exsum_degree = nullptr;
    IndexType *first_pair = nullptr;
    IndexType *second_pair = nullptr;
    IndexType *block_bucket_offsets = nullptr;

    ALLOC_MANAGED_TRY(&exsum_degree, sizeof(IndexType) * (num_edges + 1), nullptr);

    // Find the degree of the out vertex of each edge
    degree_iterator<IndexType> deg_it(offsets);
    deref_functor<degree_iterator<IndexType>, IndexType> deref(deg_it);
    rmm_temp_allocator allocator(nullptr);
    thrust::fill(thrust::cuda::par(allocator).on(nullptr), exsum_degree, exsum_degree + 1, 0);
    thrust::transform(thrust::cuda::par(allocator).on(nullptr),
                                        indices,
                                        indices + num_edges,
                                        exsum_degree + 1,
                                        deref);

    // Take the inclusive sum of the degrees
    thrust::inclusive_scan(thrust::cuda::par(allocator).on(nullptr),
                                                    exsum_degree + 1,
                                                    exsum_degree + num_edges + 1,
                                                    exsum_degree + 1);

    // Copy out the last value to get the size of scattered output
    IndexType output_size;
    hipMemcpy(&output_size, &exsum_degree[num_edges], sizeof(IndexType), hipMemcpyDefault);

    // Allocate memory for the scattered output
    ALLOC_MANAGED_TRY(&second_pair, sizeof(IndexType) * output_size, nullptr);
    ALLOC_MANAGED_TRY(&first_pair, sizeof(IndexType) * output_size, nullptr);

    // Figure out number of blocks and allocate memory for block bucket offsets
    IndexType num_blocks = (output_size + TWO_HOP_BLOCK_SIZE - 1) / TWO_HOP_BLOCK_SIZE;
    ALLOC_MANAGED_TRY(&block_bucket_offsets, sizeof(IndexType) * (num_blocks + 1), nullptr);

    // Compute the block bucket offsets
    dim3 grid, block;
    block.x = 512;
    grid.x = min((IndexType) MAXBLOCKS, (num_blocks / 512) + 1);
    compute_bucket_offsets_kernel<<<grid, block, 0, nullptr>>>(exsum_degree,
                                                               block_bucket_offsets,
                                                               num_edges,
                                                               output_size);
    hipMemcpy(&block_bucket_offsets[num_blocks], &num_edges, sizeof(IndexType), hipMemcpyDefault);

    // Scatter the expanded edge lists into temp space
    grid.x = min((IndexType) MAXBLOCKS, num_blocks);
    scatter_expand_kernel<<<grid, block, 0, nullptr>>>(exsum_degree,
                                                       indices,
                                                       offsets,
                                                       block_bucket_offsets,
                                                       num_verts,
                                                       output_size,
                                                       num_blocks,
                                                       first_pair,
                                                       second_pair);

    // Remove duplicates and self pairings
    auto tuple_start = thrust::make_zip_iterator(thrust::make_tuple(first_pair, second_pair));
    auto tuple_end = tuple_start + output_size;
    thrust::sort(thrust::cuda::par(allocator).on(nullptr), tuple_start, tuple_end);
    tuple_end = thrust::copy_if(thrust::cuda::par(allocator).on(nullptr),
                                                            tuple_start,
                                                            tuple_end,
                                                            tuple_start,
                                                            self_loop_flagger<IndexType>());
    tuple_end = thrust::unique(thrust::cuda::par(allocator).on(nullptr), tuple_start, tuple_end);

    // Get things ready to return
    outputSize = tuple_end - tuple_start;
    ALLOC_MANAGED_TRY(first, sizeof(IndexType) * outputSize, nullptr);
    ALLOC_MANAGED_TRY(second, sizeof(IndexType) * outputSize, nullptr);
    hipMemcpy(*first, first_pair, sizeof(IndexType) * outputSize, hipMemcpyDefault);
    hipMemcpy(*second, second_pair, sizeof(IndexType) * outputSize, hipMemcpyDefault);

    // Free up temporary stuff
    ALLOC_FREE_TRY(exsum_degree, nullptr);
    ALLOC_FREE_TRY(first_pair, nullptr);
    ALLOC_FREE_TRY(second_pair, nullptr);
    ALLOC_FREE_TRY(block_bucket_offsets, nullptr);

    return GDF_SUCCESS;
}

gdf_error gdf_get_two_hop_neighbors(gdf_graph* graph, gdf_column* first, gdf_column* second) {
    GDF_REQUIRE(graph != nullptr, GDF_INVALID_API_CALL);
    GDF_REQUIRE(first != nullptr, GDF_INVALID_API_CALL);
    GDF_REQUIRE(second != nullptr, GDF_INVALID_API_CALL);
    GDF_TRY(gdf_add_adj_list(graph));

    size_t num_verts = graph->adjList->offsets->size - 1;
    switch (graph->adjList->offsets->dtype) {
        case GDF_INT32: {
            int32_t* first_ptr;
            int32_t* second_ptr;
            int32_t outputSize;
            gdf_get_two_hop_neighbors_impl((int32_t) num_verts,
                                           (int32_t*) graph->adjList->offsets->data,
                                           (int32_t*) graph->adjList->indices->data,
                                           &first_ptr,
                                           &second_ptr,
                                           outputSize);
            first->data = first_ptr;
            first->dtype = GDF_INT32;
            first->size = outputSize;
            second->data = second_ptr;
            second->dtype = GDF_INT32;
            second->size = outputSize;
            break;
        }
        case GDF_INT64: {
            int64_t* first_ptr;
            int64_t* second_ptr;
            int64_t outputSize;
            gdf_get_two_hop_neighbors_impl((int64_t) num_verts,
                                           (int64_t*) graph->adjList->offsets->data,
                                           (int64_t*) graph->adjList->indices->data,
                                           &first_ptr,
                                           &second_ptr,
                                           outputSize);
            first->data = first_ptr;
            first->dtype = GDF_INT64;
            first->size = outputSize;
            second->data = second_ptr;
            second->dtype = GDF_INT64;
            second->size = outputSize;
            break;
        }
        default:
            return GDF_UNSUPPORTED_DTYPE;
    }

    return GDF_SUCCESS;
}
