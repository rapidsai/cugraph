#include "hip/hip_runtime.h"
// -*-c++-*-

 /*
 * Copyright (c) 2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 */

// Graph analytics features
// Author: Alex Fender afender@nvidia.com

#include <cugraph.h>
#include "graph_utils.cuh"
#include "pagerank.cuh"
#include "COOtoCSR.cuh"
#include "utilities/error_utils.h"
#include "bfs.cuh"
#include "renumber.cuh"
#include "snmg/spmv.cuh"
#include <hip/library_types.h>
#include <nvgraph/nvgraph.h>
#include <thrust/device_vector.h>

#include <rmm_utils.h>

template<typename T>
using Vector = thrust::device_vector<T, rmm_allocator<T>>;

/*
 * cudf has gdf_column_free and using this is, in general, better design than
 * creating our own, but we will keep this as cudf is planning to remove the
 * function. cudf plans to redesign cudf::column to fundamentally solve this
 * problem, so once they finished the redesign, we need to update this code to
 * use their new features. Until that time, we may rely on this as a temporary
 * solution.
 */
void gdf_col_delete(gdf_column* col) {
  if (col != nullptr) {
    auto stream = hipStream_t{nullptr};
    if (col->data != nullptr) {
      ALLOC_FREE_TRY(col->data, stream);
    }
    if (col->valid != nullptr) {
      ALLOC_FREE_TRY(col->valid, stream);
    }
#if 0/* Currently, gdf_column_view does not set col_name, and col_name can have
        an arbitrary value, so freeing col_name can lead to freeing a ranodom
        address. This problem should be cleaned up once cudf finishes
        redesigning cudf::column. */
    if (col->col_name != nullptr) {
      free(col->col_name);
    }
#endif
    delete col;
  }
}

void gdf_col_release(gdf_column* col) {
  delete col;
}

void cpy_column_view(const gdf_column *in, gdf_column *out) {
  if (in != nullptr && out !=nullptr) {
    gdf_column_view(out, in->data, in->valid, in->size, in->dtype);
  }
}

gdf_error gdf_adj_list_view(gdf_graph *graph, const gdf_column *offsets,
                                 const gdf_column *indices, const gdf_column *edge_data) {
  //This function returns an error if this graph object has at least one graph
  //representation to prevent a single object storing two different graphs.
  GDF_REQUIRE( ((graph->edgeList == nullptr) && (graph->adjList == nullptr) &&
    (graph->transposedAdjList == nullptr)), GDF_INVALID_API_CALL);
  GDF_REQUIRE( offsets->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( (offsets->dtype == indices->dtype), GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( ((offsets->dtype == GDF_INT32) || (offsets->dtype == GDF_INT64)), GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( (offsets->size > 0), GDF_DATASET_EMPTY );

  graph->adjList = new gdf_adj_list;
  graph->adjList->offsets = new gdf_column;
  graph->adjList->indices = new gdf_column;
  graph->adjList->ownership = 0;

  cpy_column_view(offsets, graph->adjList->offsets);
  cpy_column_view(indices, graph->adjList->indices);
  if (edge_data) {
      GDF_REQUIRE( indices->size == edge_data->size, GDF_COLUMN_SIZE_MISMATCH );
      graph->adjList->edge_data = new gdf_column;
      cpy_column_view(edge_data, graph->adjList->edge_data);
  }
  else {
    graph->adjList->edge_data = nullptr;
  }
  return GDF_SUCCESS;
}

gdf_error gdf_adj_list::get_vertex_identifiers(gdf_column *identifiers) {
  GDF_REQUIRE( offsets != nullptr , GDF_INVALID_API_CALL);
  GDF_REQUIRE( offsets->data != nullptr , GDF_INVALID_API_CALL);
  cugraph::sequence<int>((int)offsets->size-1, (int*)identifiers->data);
  return GDF_SUCCESS;
}

gdf_error gdf_adj_list::get_source_indices (gdf_column *src_indices) {
  GDF_REQUIRE( offsets != nullptr , GDF_INVALID_API_CALL);
  GDF_REQUIRE( offsets->data != nullptr , GDF_INVALID_API_CALL);
  GDF_REQUIRE( src_indices->size == indices->size, GDF_COLUMN_SIZE_MISMATCH );
  GDF_REQUIRE( src_indices->dtype == indices->dtype, GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( src_indices->size > 0, GDF_DATASET_EMPTY );
  cugraph::offsets_to_indices<int>((int*)offsets->data, offsets->size-1, (int*)src_indices->data);

  return GDF_SUCCESS;
}

gdf_error gdf_renumber_vertices(const gdf_column *src, const gdf_column *dst,
				gdf_column *src_renumbered, gdf_column *dst_renumbered,
				gdf_column *numbering_map) {

  GDF_REQUIRE( src->size == dst->size, GDF_COLUMN_SIZE_MISMATCH );
  GDF_REQUIRE( src->dtype == dst->dtype, GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( ((src->dtype == GDF_INT32) || (src->dtype == GDF_INT64)), GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( src->size > 0, GDF_DATASET_EMPTY ); 

  //
  //  TODO: we're currently renumbering without using valid.  We need to
  //        worry about that at some point, but for now we'll just
  //        copy the valid pointers to the new columns and go from there.
  //
  hipStream_t stream{nullptr};

  size_t src_size = src->size;
  size_t new_size;

  //
  // TODO:  I assume int64_t for output.  A few thoughts:
  //
  //    * I could match src->dtype - since if the raw values fit in an int32_t,
  //      then the renumbered values must fit within an int32_t
  //    * If new_size < (2^31 - 1) then I could allocate 32-bit integers
  //      and copy them in order to make the final footprint smaller.
  //
  //
  //  NOTE:  Forcing match right now - it appears that cugraph is artficially
  //         forcing the type to be 32
  if (src->dtype == GDF_INT32) {
    int32_t *tmp;

    ALLOC_MANAGED_TRY((void**) &tmp, sizeof(int32_t) * src->size, stream);
    gdf_column_view(src_renumbered, tmp, src->valid, src->size, src->dtype);
  
    ALLOC_MANAGED_TRY((void**) &tmp, sizeof(int32_t) * src->size, stream);
    gdf_column_view(dst_renumbered, tmp, dst->valid, dst->size, dst->dtype);

    gdf_error err = cugraph::renumber_vertices(src_size,
					       (const int32_t *) src->data,
					       (const int32_t *) dst->data,
					       (int32_t *) src_renumbered->data,
					       (int32_t *) dst_renumbered->data,
					       &new_size, &tmp);
    if (err != GDF_SUCCESS)
      return err;

    gdf_column_view(numbering_map, tmp, nullptr, new_size, src->dtype);
  } else if (src->dtype == GDF_INT64) {

    //
    //  NOTE: At the moment, we force the renumbered graph to use
    //        32-bit integer ids.  Since renumbering is going to make
    //        the vertex range dense, this limits us to 2 billion
    //        vertices.
    //
    //        The renumbering code supports 64-bit integer generation
    //        so we can run this with int64_t output if desired...
    //        but none of the algorithms support that.
    //
    int64_t *tmp;
    ALLOC_MANAGED_TRY((void**) &tmp, sizeof(int32_t) * src->size, stream);
    gdf_column_view(src_renumbered, tmp, src->valid, src->size, GDF_INT32);
  
    ALLOC_MANAGED_TRY((void**) &tmp, sizeof(int32_t) * src->size, stream);
    gdf_column_view(dst_renumbered, tmp, dst->valid, dst->size, GDF_INT32);

    gdf_error err = cugraph::renumber_vertices(src_size,
					       (const int64_t *) src->data,
					       (const int64_t *) dst->data,
					       (int32_t *) src_renumbered->data,
					       (int32_t *) dst_renumbered->data,
					       &new_size, &tmp);
    if (err != GDF_SUCCESS)
      return err;

    //
    //  If there are too many vertices then the renumbering overflows so we'll
    //  return an error.
    //
    if (new_size > 0x7fffffff) {
      ALLOC_FREE_TRY(src_renumbered, stream);
      ALLOC_FREE_TRY(dst_renumbered, stream);
      return GDF_COLUMN_SIZE_TOO_BIG;
    }

    gdf_column_view(numbering_map, tmp, nullptr, new_size, src->dtype);
  } else {
    return GDF_UNSUPPORTED_DTYPE;
  }

  return GDF_SUCCESS;
}

gdf_error gdf_edge_list_view(gdf_graph *graph, const gdf_column *src_indices,
                                 const gdf_column *dest_indices, const gdf_column *edge_data) {
  //This function returns an error if this graph object has at least one graph
  //representation to prevent a single object storing two different graphs.
  GDF_REQUIRE( ((graph->edgeList == nullptr) && (graph->adjList == nullptr) &&
    (graph->transposedAdjList == nullptr)), GDF_INVALID_API_CALL);
  GDF_REQUIRE( src_indices->size == dest_indices->size, GDF_COLUMN_SIZE_MISMATCH );
  GDF_REQUIRE( src_indices->dtype == dest_indices->dtype, GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( ((src_indices->dtype == GDF_INT32) || (src_indices->dtype == GDF_INT64)), GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( src_indices->size > 0, GDF_DATASET_EMPTY );
  GDF_REQUIRE( src_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( dest_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );

  graph->edgeList = new gdf_edge_list;
  graph->edgeList->src_indices = new gdf_column;
  graph->edgeList->dest_indices = new gdf_column;
  graph->edgeList->ownership = 0;

  cpy_column_view(src_indices, graph->edgeList->src_indices);
  cpy_column_view(dest_indices, graph->edgeList->dest_indices);
  if (edge_data) {
      GDF_REQUIRE( src_indices->size == edge_data->size, GDF_COLUMN_SIZE_MISMATCH );
      graph->edgeList->edge_data = new gdf_column;
      cpy_column_view(edge_data, graph->edgeList->edge_data);
  }
  else {
    graph->edgeList->edge_data = nullptr;
  }

  return GDF_SUCCESS;
}

template <typename T, typename WT>
gdf_error gdf_add_adj_list_impl (gdf_graph *graph) {
    if (graph->adjList == nullptr) {
      GDF_REQUIRE( graph->edgeList != nullptr , GDF_INVALID_API_CALL);
      int nnz = graph->edgeList->src_indices->size, status = 0;
      graph->adjList = new gdf_adj_list;
      graph->adjList->offsets = new gdf_column;
      graph->adjList->indices = new gdf_column;
      graph->adjList->ownership = 1;

    if (graph->edgeList->edge_data!= nullptr) {
      graph->adjList->edge_data = new gdf_column;

      CSR_Result_Weighted<int32_t,WT> adj_list;
      status = ConvertCOOtoCSR_weighted((int*)graph->edgeList->src_indices->data, (int*)graph->edgeList->dest_indices->data, (WT*)graph->edgeList->edge_data->data, nnz, adj_list);

      gdf_column_view(graph->adjList->offsets, adj_list.rowOffsets,
                            nullptr, adj_list.size+1, graph->edgeList->src_indices->dtype);
      gdf_column_view(graph->adjList->indices, adj_list.colIndices,
                            nullptr, adj_list.nnz, graph->edgeList->src_indices->dtype);
      gdf_column_view(graph->adjList->edge_data, adj_list.edgeWeights,
                          nullptr, adj_list.nnz, graph->edgeList->edge_data->dtype);
    }
    else {
      CSR_Result<int> adj_list;
      status = ConvertCOOtoCSR((int*)graph->edgeList->src_indices->data,(int*)graph->edgeList->dest_indices->data, nnz, adj_list);
      gdf_column_view(graph->adjList->offsets, adj_list.rowOffsets,
                            nullptr, adj_list.size+1, graph->edgeList->src_indices->dtype);
      gdf_column_view(graph->adjList->indices, adj_list.colIndices,
                            nullptr, adj_list.nnz, graph->edgeList->src_indices->dtype);
    }
    if (status !=0) {
      std::cerr << "Could not generate the adj_list" << std::endl;
      return GDF_CUDA_ERROR;
    }
  }
  return GDF_SUCCESS;
}

gdf_error gdf_add_edge_list (gdf_graph *graph) {
    if (graph->edgeList == nullptr) {
      GDF_REQUIRE( graph->adjList != nullptr , GDF_INVALID_API_CALL);
      int *d_src;
      graph->edgeList = new gdf_edge_list;
      graph->edgeList->src_indices = new gdf_column;
      graph->edgeList->dest_indices = new gdf_column;
      graph->edgeList->ownership = 2;

      CUDA_TRY(hipMallocManaged ((void**)&d_src, sizeof(int) * graph->adjList->indices->size));

      cugraph::offsets_to_indices<int>((int*)graph->adjList->offsets->data,
                                  graph->adjList->offsets->size-1,
                                  (int*)d_src);

      gdf_column_view(graph->edgeList->src_indices, d_src,
                      nullptr, graph->adjList->indices->size, graph->adjList->indices->dtype);
      cpy_column_view(graph->adjList->indices, graph->edgeList->dest_indices);

      if (graph->adjList->edge_data != nullptr) {
        graph->edgeList->edge_data = new gdf_column;
        cpy_column_view(graph->adjList->edge_data, graph->edgeList->edge_data);
      }
  }
  return GDF_SUCCESS;
}


template <typename WT>
gdf_error gdf_add_transposed_adj_list_impl (gdf_graph *graph) {
    if (graph->transposedAdjList == nullptr ) {
      GDF_REQUIRE( graph->edgeList != nullptr , GDF_INVALID_API_CALL);
      int nnz = graph->edgeList->src_indices->size, status = 0;
      graph->transposedAdjList = new gdf_adj_list;
      graph->transposedAdjList->offsets = new gdf_column;
      graph->transposedAdjList->indices = new gdf_column;
      graph->transposedAdjList->ownership = 1;

      if (graph->edgeList->edge_data) {
        graph->transposedAdjList->edge_data = new gdf_column;
        CSR_Result_Weighted<int32_t,WT> adj_list;
        status = ConvertCOOtoCSR_weighted( (int*)graph->edgeList->dest_indices->data, (int*)graph->edgeList->src_indices->data, (WT*)graph->edgeList->edge_data->data, nnz, adj_list);
        gdf_column_view(graph->transposedAdjList->offsets, adj_list.rowOffsets,
                              nullptr, adj_list.size+1, graph->edgeList->src_indices->dtype);
        gdf_column_view(graph->transposedAdjList->indices, adj_list.colIndices,
                              nullptr, adj_list.nnz, graph->edgeList->src_indices->dtype);
        gdf_column_view(graph->transposedAdjList->edge_data, adj_list.edgeWeights,
                            nullptr, adj_list.nnz, graph->edgeList->edge_data->dtype);
      }
      else {

        CSR_Result<int> adj_list;
        status = ConvertCOOtoCSR((int*)graph->edgeList->dest_indices->data, (int*)graph->edgeList->src_indices->data, nnz, adj_list);
        gdf_column_view(graph->transposedAdjList->offsets, adj_list.rowOffsets,
                              nullptr, adj_list.size+1, graph->edgeList->src_indices->dtype);
        gdf_column_view(graph->transposedAdjList->indices, adj_list.colIndices,
                              nullptr, adj_list.nnz, graph->edgeList->src_indices->dtype);
      }
      if (status !=0) {
        std::cerr << "Could not generate the adj_list" << std::endl;
        return GDF_CUDA_ERROR;
      }
    }
    return GDF_SUCCESS;
}

gdf_error gdf_degree_impl(int n, int e, gdf_column* col_ptr, gdf_column* degree, bool offsets) {
  if(offsets == true) {
    dim3 nthreads, nblocks;
    nthreads.x = min(n, CUDA_MAX_KERNEL_THREADS);
    nthreads.y = 1;
    nthreads.z = 1;
    nblocks.x = min((n + nthreads.x - 1) / nthreads.x, CUDA_MAX_BLOCKS);
    nblocks.y = 1;
    nblocks.z = 1;

    switch (col_ptr->dtype) {
      case GDF_INT32:   cugraph::degree_offsets<int32_t, float> <<<nblocks, nthreads>>>(n, e, static_cast<int*>(col_ptr->data), static_cast<int*>(degree->data));break;
      default: return GDF_UNSUPPORTED_DTYPE;
    }
  }
  else {
    dim3 nthreads, nblocks;
    nthreads.x = min(e, CUDA_MAX_KERNEL_THREADS);
    nthreads.y = 1;
    nthreads.z = 1;
    nblocks.x = min((e + nthreads.x - 1) / nthreads.x, CUDA_MAX_BLOCKS);
    nblocks.y = 1;
    nblocks.z = 1;

    switch (col_ptr->dtype) {
      case GDF_INT32:   cugraph::degree_coo<int32_t, float> <<<nblocks, nthreads>>>(n, e, static_cast<int*>(col_ptr->data), static_cast<int*>(degree->data));break;
      default: return GDF_UNSUPPORTED_DTYPE;
    }
  }
  return GDF_SUCCESS;
}


gdf_error gdf_degree(gdf_graph *graph, gdf_column *degree, int x) {
  // Calculates the degree of all vertices of the graph
  // x = 0: in+out degree
  // x = 1: in-degree
  // x = 2: out-degree
  GDF_REQUIRE(graph->adjList != nullptr || graph->transposedAdjList != nullptr, GDF_INVALID_API_CALL);
  int n;
  int e;
  if(graph->adjList != nullptr) {
    n = graph->adjList->offsets->size -1;
    e = graph->adjList->indices->size;
  }
  else {
    n = graph->transposedAdjList->offsets->size - 1;
    e = graph->transposedAdjList->indices->size;
  }

  if(x!=1) {
    // Computes out-degree for x=0 and x=2
    if(graph->adjList)
      gdf_degree_impl(n, e, graph->adjList->offsets, degree, true);
    else
      gdf_degree_impl(n, e, graph->transposedAdjList->indices, degree, false);
  }

  if(x!=2) {
    // Computes in-degree for x=0 and x=1
    if(graph->adjList)
      gdf_degree_impl(n, e, graph->adjList->indices, degree, false);
    else
      gdf_degree_impl(n, e, graph->transposedAdjList->offsets, degree, true);
  }
  return GDF_SUCCESS;
}


template <typename WT>
gdf_error gdf_pagerank_impl (gdf_graph *graph,
                      gdf_column *pagerank, float alpha = 0.85,
                      float tolerance = 1e-4, int max_iter = 200,
                      bool has_guess = false) {
  GDF_REQUIRE( graph->edgeList != nullptr, GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( graph->edgeList->src_indices->size == graph->edgeList->dest_indices->size, GDF_COLUMN_SIZE_MISMATCH );
  GDF_REQUIRE( graph->edgeList->src_indices->dtype == graph->edgeList->dest_indices->dtype, GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( graph->edgeList->src_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( graph->edgeList->dest_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( pagerank != nullptr , GDF_INVALID_API_CALL );
  GDF_REQUIRE( pagerank->data != nullptr , GDF_INVALID_API_CALL );
  GDF_REQUIRE( pagerank->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( pagerank->size > 0 , GDF_INVALID_API_CALL );

  int m=pagerank->size, nnz = graph->edgeList->src_indices->size, status = 0;
  WT *d_pr, *d_val = nullptr, *d_leaf_vector = nullptr;
  WT res = 1.0;
  WT *residual = &res;

  if (graph->transposedAdjList == nullptr) {
    gdf_add_transposed_adj_list(graph);
  }
  hipStream_t stream{nullptr};
  ALLOC_MANAGED_TRY((void**)&d_leaf_vector, sizeof(WT) * m, stream);
  ALLOC_MANAGED_TRY((void**)&d_val, sizeof(WT) * nnz , stream);
  ALLOC_MANAGED_TRY((void**)&d_pr,    sizeof(WT) * m, stream);

  //  The templating for HT_matrix_csc_coo assumes that m, nnz and data are all the same type
  cugraph::HT_matrix_csc_coo(m, nnz, (int *)graph->transposedAdjList->offsets->data, (int *)graph->transposedAdjList->indices->data, d_val, d_leaf_vector);

  if (has_guess)
  {
    GDF_REQUIRE( pagerank->data != nullptr, GDF_VALIDITY_UNSUPPORTED );
    cugraph::copy<WT>(m, (WT*)pagerank->data, d_pr);
  }

  status = cugraph::pagerank<int32_t,WT>( m,nnz, (int*)graph->transposedAdjList->offsets->data, (int*)graph->transposedAdjList->indices->data,
    d_val, alpha, d_leaf_vector, false, tolerance, max_iter, d_pr, residual);

  if (status !=0)
    switch ( status ) {
      case -1: std::cerr<< "Error : bad parameters in Pagerank"<<std::endl; return GDF_CUDA_ERROR;
      case 1: std::cerr<< "Warning : Pagerank did not reached the desired tolerance"<<std::endl;  return GDF_CUDA_ERROR;
      default:  std::cerr<< "Pagerank failed"<<std::endl;  return GDF_CUDA_ERROR;
    }

  cugraph::copy<WT>(m, d_pr, (WT*)pagerank->data);

  ALLOC_FREE_TRY(d_val, stream);
  ALLOC_FREE_TRY(d_pr, stream);
  ALLOC_FREE_TRY(d_leaf_vector, stream);

  return GDF_SUCCESS;
}

gdf_error gdf_add_adj_list(gdf_graph *graph) {
  if (graph->adjList != nullptr)
    return GDF_SUCCESS;

  GDF_REQUIRE( graph->edgeList != nullptr , GDF_INVALID_API_CALL);
  GDF_REQUIRE( graph->edgeList->src_indices->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE );

  if (graph->edgeList->edge_data != nullptr) {
    switch (graph->edgeList->edge_data->dtype) {
      case GDF_FLOAT32:   return gdf_add_adj_list_impl<int32_t, float>(graph);
      case GDF_FLOAT64:   return gdf_add_adj_list_impl<int32_t, double>(graph);
      default: return GDF_UNSUPPORTED_DTYPE;
    }
  }
  else {
    return gdf_add_adj_list_impl<int32_t, float>(graph);
  }
}

gdf_error gdf_add_transposed_adj_list(gdf_graph *graph) {
  if (graph->edgeList == nullptr)
    gdf_add_edge_list(graph);

  GDF_REQUIRE(graph->edgeList->src_indices->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);
  GDF_REQUIRE(graph->edgeList->dest_indices->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);

  if (graph->edgeList->edge_data != nullptr) {
    switch (graph->edgeList->edge_data->dtype) {
      case GDF_FLOAT32:   return gdf_add_transposed_adj_list_impl<float>(graph);
      case GDF_FLOAT64:   return gdf_add_transposed_adj_list_impl<double>(graph);
      default: return GDF_UNSUPPORTED_DTYPE;
    }
  }
  else {
    return gdf_add_transposed_adj_list_impl<float>(graph);
  }
}

gdf_error gdf_delete_adj_list(gdf_graph *graph) {
  if (graph->adjList) {
    delete graph->adjList;
  }
  graph->adjList = nullptr;
  return GDF_SUCCESS;
}

gdf_error gdf_delete_edge_list(gdf_graph *graph) {
  if (graph->edgeList) {
    delete graph->edgeList;
  }
  graph->edgeList = nullptr;
  return GDF_SUCCESS;
}

gdf_error gdf_delete_transposed_adj_list(gdf_graph *graph) {
  if (graph->transposedAdjList) {
    delete graph->transposedAdjList;
  }
  graph->transposedAdjList = nullptr;
  return GDF_SUCCESS;
}

gdf_error gdf_pagerank(gdf_graph *graph, gdf_column *pagerank, float alpha, float tolerance, int max_iter, bool has_guess) {
  //
  //  page rank operates on CSR and can't currently support 64-bit integers.
  //
  //  If csr doesn't exist, create it.  Then check type to make sure it is 32-bit.
  //
  GDF_REQUIRE(graph->adjList != nullptr || graph->edgeList != nullptr, GDF_INVALID_API_CALL);
  gdf_error err = gdf_add_adj_list(graph);
  if (err != GDF_SUCCESS)
    return err;

  GDF_REQUIRE(graph->adjList->offsets->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);
  GDF_REQUIRE(graph->adjList->indices->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);

  switch (pagerank->dtype) {
    case GDF_FLOAT32:   return gdf_pagerank_impl<float>(graph, pagerank, alpha, tolerance, max_iter, has_guess);
    case GDF_FLOAT64:   return gdf_pagerank_impl<double>(graph, pagerank, alpha, tolerance, max_iter, has_guess);
    default: return GDF_UNSUPPORTED_DTYPE;
  }
}

gdf_error gdf_bfs(gdf_graph *graph, gdf_column *distances, gdf_column *predecessors, int start_vertex, bool directed) {
  GDF_REQUIRE(graph->adjList != nullptr || graph->edgeList != nullptr, GDF_INVALID_API_CALL);
  gdf_error err = gdf_add_adj_list(graph);
  if (err != GDF_SUCCESS)
    return err;
  GDF_REQUIRE(graph->adjList->offsets->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);
  GDF_REQUIRE(graph->adjList->indices->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);
  GDF_REQUIRE(distances->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);
  GDF_REQUIRE(predecessors->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);

  int n = graph->adjList->offsets->size - 1;
  int e = graph->adjList->indices->size;
  int* offsets_ptr = (int*)graph->adjList->offsets->data;
  int* indices_ptr = (int*)graph->adjList->indices->data;
  int* distances_ptr = (int*)distances->data;
  int* predecessors_ptr = (int*)predecessors->data;
  int alpha = 15;
  int beta = 18;

  cugraph::Bfs<int> bfs(n, e, offsets_ptr, indices_ptr, directed, alpha, beta);
  bfs.configure(distances_ptr, predecessors_ptr, nullptr);
  bfs.traverse(start_vertex);
  return GDF_SUCCESS;
}

gdf_error gdf_louvain(gdf_graph *graph, void *final_modularity, void *num_level, gdf_column *louvain_parts) {
  GDF_REQUIRE(graph->adjList != nullptr || graph->edgeList != nullptr, GDF_INVALID_API_CALL);
  gdf_error err = gdf_add_adj_list(graph);
  if (err != GDF_SUCCESS)
    return err;

  size_t n = graph->adjList->offsets->size - 1;
  size_t e = graph->adjList->indices->size;

  void* offsets_ptr = graph->adjList->offsets->data;
  void* indices_ptr = graph->adjList->indices->data;

  void* value_ptr;
  Vector<float> d_values;
  if(graph->adjList->edge_data) {
      value_ptr = graph->adjList->edge_data->data;
  }
  else {
      hipStream_t stream { nullptr };
      rmm_temp_allocator allocator(stream);
      d_values.resize(graph->adjList->indices->size);
      thrust::fill(thrust::cuda::par(allocator).on(stream), d_values.begin(), d_values.end(), 1.0);
      value_ptr = (void * ) thrust::raw_pointer_cast(d_values.data());
  }

  void* louvain_parts_ptr = louvain_parts->data;

  auto gdf_to_cudadtype= [](gdf_column *col){
    hipDataType cuda_dtype;
    switch(col->dtype){
      case GDF_INT8: cuda_dtype = HIP_R_8I; break;
      case GDF_INT32: cuda_dtype = HIP_R_32I; break;
      case GDF_FLOAT32: cuda_dtype = HIP_R_32F; break;
      case GDF_FLOAT64: cuda_dtype = HIP_R_64F; break;
      default: throw new std::invalid_argument("Cannot convert data type");
      }return cuda_dtype;
  };

  hipDataType index_type = gdf_to_cudadtype(graph->adjList->indices);
  hipDataType val_type = graph->adjList->edge_data? gdf_to_cudadtype(graph->adjList->edge_data): HIP_R_32F;

  nvgraphLouvain(index_type, val_type, n, e, offsets_ptr, indices_ptr, value_ptr, 1, 0, NULL,
                 final_modularity, louvain_parts_ptr, num_level);
  return GDF_SUCCESS;
}

template <typename idx_t,typename val_t>
gdf_error gdf_snmg_csrmv_impl (size_t * part_offsets, gdf_column * off, gdf_column * ind, gdf_column * val, gdf_column ** x_cols){
  
  GDF_REQUIRE( part_offsets != nullptr, GDF_INVALID_API_CALL );
  GDF_REQUIRE( off != nullptr, GDF_INVALID_API_CALL );
  GDF_REQUIRE( ind != nullptr, GDF_INVALID_API_CALL );
  GDF_REQUIRE( val != nullptr, GDF_INVALID_API_CALL );
  GDF_REQUIRE( x_cols != nullptr, GDF_INVALID_API_CALL );
  GDF_REQUIRE( off->size > 0, GDF_INVALID_API_CALL );
  GDF_REQUIRE( ind->size > 0, GDF_INVALID_API_CALL );
  GDF_REQUIRE( val->size > 0, GDF_INVALID_API_CALL );
  GDF_REQUIRE( ind->size == val->size, GDF_COLUMN_SIZE_MISMATCH ); 
  GDF_REQUIRE( off->dtype == ind->dtype, GDF_UNSUPPORTED_DTYPE );  
  GDF_REQUIRE( off->null_count + ind->null_count + val->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );                 

  gdf_error status;
  auto p = omp_get_num_threads();

  val_t* x[p];
  for (auto i = 0; i < p; ++i)
  {
    GDF_REQUIRE( x_cols[i] != nullptr, GDF_INVALID_API_CALL );
    GDF_REQUIRE( x_cols[i]->size > 0, GDF_INVALID_API_CALL );
    x[i]= static_cast<val_t*>(x_cols[i]->data);
  }
  status = cugraph::snmg_csrmv<idx_t,val_t>(part_offsets,
                                      static_cast<idx_t*>(off->data), 
                                      static_cast<idx_t*>(ind->data), 
                                      static_cast<val_t*>(val->data), 
                                      x);
  return status;
}

gdf_error gdf_snmg_csrmv (size_t * part_offsets, gdf_column * off, gdf_column * ind, gdf_column * val, gdf_column ** x_cols){
    switch (val->dtype) {
      case GDF_FLOAT32:   return gdf_snmg_csrmv_impl<int32_t,float>(part_offsets, off, ind, val, x_cols);
      case GDF_FLOAT64:   return gdf_snmg_csrmv_impl<int32_t,double>(part_offsets, off, ind, val, x_cols);
      default: return GDF_UNSUPPORTED_DTYPE;
    }
}
