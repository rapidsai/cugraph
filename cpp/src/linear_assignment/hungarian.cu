#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cugraph/graph.hpp>
#include <cugraph/utilities/error.hpp>

#include <raft/lap/lap.cuh>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/for_each.h>
#include <thrust/random.h>

#include <iostream>
#include <limits>

//#define TIMING

#ifdef TIMING
#include <utilities/high_res_timer.hpp>
#endif

namespace cugraph {
namespace detail {

template <typename weight_t>
weight_t default_precision()
{
  return 0;
}

template <>
float default_precision()
{
  return float{1e-6};
}

template <>
double default_precision()
{
  return double{1e-6};
}

template <typename index_t, typename weight_t>
weight_t hungarian(raft::handle_t const &handle,
                   index_t num_rows,
                   index_t num_cols,
                   weight_t const *d_original_cost,
                   index_t *d_assignment,
                   weight_t precision,
                   rmm::cuda_stream_view stream_view)
{
  // FIXME: if num_cols != num_rows we can copy it and fill with zeros to make it square //
  //  TODO:  Can Date/Nagi implementation in raft handle rectangular matrices?
  //
  CUGRAPH_EXPECTS(num_rows == num_cols, "Current implementation only supports square matrices");

  rmm::device_uvector<index_t> col_assignments_v(num_rows, stream_view);

  // Create an instance of LinearAssignmentProblem using problem size, number of subproblems
  raft::lap::LinearAssignmentProblem<index_t, weight_t> lpx(handle, num_rows, 1, precision);

  // Solve LAP(s) for given cost matrix
  lpx.solve(d_original_cost, d_assignment, col_assignments_v.data());

  return lpx.getPrimalObjectiveValue(0);
}

template <typename vertex_t, typename edge_t, typename weight_t>
weight_t hungarian_sparse(raft::handle_t const &handle,
                          GraphCOOView<vertex_t, edge_t, weight_t> const &graph,
                          vertex_t num_workers,
                          vertex_t const *workers,
                          vertex_t *assignment,
                          weight_t precision,
                          rmm::cuda_stream_view stream_view)
{
  CUGRAPH_EXPECTS(assignment != nullptr, "Invalid input argument: assignment pointer is NULL");
  CUGRAPH_EXPECTS(graph.edge_data != nullptr,
                  "Invalid input argument: graph must have edge data (costs)");

#ifdef TIMING
  HighResTimer hr_timer;

  hr_timer.start("prep");
#endif

  //
  //  Translate sparse matrix into dense bipartite matrix.
  //    rows are the workers, columns are the tasks
  //
  vertex_t num_rows = num_workers;
  vertex_t num_cols = graph.number_of_vertices - num_rows;

  vertex_t matrix_dimension = std::max(num_rows, num_cols);

  rmm::device_uvector<weight_t> cost_v(matrix_dimension * matrix_dimension, stream_view);
  rmm::device_uvector<vertex_t> tasks_v(num_cols, stream_view);
  rmm::device_uvector<vertex_t> temp_tasks_v(graph.number_of_vertices, stream_view);
  rmm::device_uvector<vertex_t> temp_workers_v(graph.number_of_vertices, stream_view);

  weight_t *d_cost         = cost_v.data();
  vertex_t *d_tasks        = tasks_v.data();
  vertex_t *d_temp_tasks   = temp_tasks_v.data();
  vertex_t *d_temp_workers = temp_workers_v.data();
  vertex_t *d_src_indices  = graph.src_indices;
  vertex_t *d_dst_indices  = graph.dst_indices;
  weight_t *d_edge_data    = graph.edge_data;

  //
  //  Renumber vertices internally.  Workers will become
  //  rows, tasks will become columns
  //
  thrust::sequence(rmm::exec_policy(stream_view), temp_tasks_v.begin(), temp_tasks_v.end());

  thrust::for_each(rmm::exec_policy(stream_view),
                   workers,
                   workers + num_workers,
                   [d_temp_tasks] __device__(vertex_t v) { d_temp_tasks[v] = -1; });

  auto temp_end = thrust::copy_if(rmm::exec_policy(stream_view),
                                  temp_tasks_v.begin(),
                                  temp_tasks_v.end(),
                                  d_tasks,
                                  [] __device__(vertex_t v) { return v >= 0; });

  vertex_t size = thrust::distance(d_tasks, temp_end);
  tasks_v.resize(size, stream_view);

  //
  // Now we'll assign costs into the dense array
  //
  thrust::fill(
    rmm::exec_policy(stream_view), temp_workers_v.begin(), temp_workers_v.end(), vertex_t{-1});
  thrust::fill(
    rmm::exec_policy(stream_view), temp_tasks_v.begin(), temp_tasks_v.end(), vertex_t{-1});
  thrust::fill(rmm::exec_policy(stream_view), cost_v.begin(), cost_v.end(), weight_t{0});

  thrust::for_each(
    rmm::exec_policy(stream_view),
    thrust::make_counting_iterator<vertex_t>(0),
    thrust::make_counting_iterator<vertex_t>(num_rows),
    [d_temp_workers, workers] __device__(vertex_t v) { d_temp_workers[workers[v]] = v; });

  thrust::for_each(
    rmm::exec_policy(stream_view),
    thrust::make_counting_iterator<vertex_t>(0),
    thrust::make_counting_iterator<vertex_t>(num_cols),
    [d_temp_tasks, d_tasks] __device__(vertex_t v) { d_temp_tasks[d_tasks[v]] = v; });

  thrust::for_each(rmm::exec_policy(stream_view),
                   thrust::make_counting_iterator<edge_t>(0),
                   thrust::make_counting_iterator<edge_t>(graph.number_of_edges),
                   [d_temp_workers,
                    d_temp_tasks,
                    d_cost,
                    matrix_dimension,
                    d_src_indices,
                    d_dst_indices,
                    d_edge_data] __device__(edge_t loc) {
                     vertex_t src = d_temp_workers[d_src_indices[loc]];
                     vertex_t dst = d_temp_tasks[d_dst_indices[loc]];

                     if ((src >= 0) && (dst >= 0)) {
                       d_cost[src * matrix_dimension + dst] = d_edge_data[loc];
                     }
                   });

#ifdef TIMING
  hr_timer.stop();

  hr_timer.start("hungarian");
#endif

  //
  //  temp_assignment_v will hold the assignment in the dense
  //  bipartite matrix numbering
  //
  rmm::device_uvector<vertex_t> temp_assignment_v(matrix_dimension, stream_view);
  vertex_t *d_temp_assignment = temp_assignment_v.data();

  weight_t min_cost = detail::hungarian(
    handle, matrix_dimension, matrix_dimension, d_cost, d_temp_assignment, precision, stream_view);

#ifdef TIMING
  hr_timer.stop();

  hr_timer.start("translate");
#endif

  //
  //  Translate the assignment back to the original vertex ids
  //
  thrust::for_each(rmm::exec_policy(stream_view),
                   thrust::make_counting_iterator<vertex_t>(0),
                   thrust::make_counting_iterator<vertex_t>(num_rows),
                   [d_tasks, d_temp_assignment, assignment] __device__(vertex_t id) {
                     assignment[id] = d_tasks[d_temp_assignment[id]];
                   });

#ifdef TIMING
  hr_timer.stop();

  hr_timer.display(std::cout);
#endif

  return min_cost;
}

}  // namespace detail

template <typename vertex_t, typename edge_t, typename weight_t>
weight_t hungarian(raft::handle_t const &handle,
                   GraphCOOView<vertex_t, edge_t, weight_t> const &graph,
                   vertex_t num_workers,
                   vertex_t const *workers,
                   vertex_t *assignment)
{
  rmm::cuda_stream_view stream_view{};

  return detail::hungarian_sparse(handle,
                                  graph,
                                  num_workers,
                                  workers,
                                  assignment,
                                  detail::default_precision<weight_t>(),
                                  stream_view);
}

template <typename vertex_t, typename edge_t, typename weight_t>
weight_t hungarian(raft::handle_t const &handle,
                   GraphCOOView<vertex_t, edge_t, weight_t> const &graph,
                   vertex_t num_workers,
                   vertex_t const *workers,
                   vertex_t *assignment,
                   weight_t precision)
{
  rmm::cuda_stream_view stream_view{};

  return detail::hungarian_sparse(
    handle, graph, num_workers, workers, assignment, precision, stream_view);
}

template int32_t hungarian<int32_t, int32_t, int32_t>(
  raft::handle_t const &,
  GraphCOOView<int32_t, int32_t, int32_t> const &,
  int32_t,
  int32_t const *,
  int32_t *,
  int32_t);

template float hungarian<int32_t, int32_t, float>(raft::handle_t const &,
                                                  GraphCOOView<int32_t, int32_t, float> const &,
                                                  int32_t,
                                                  int32_t const *,
                                                  int32_t *,
                                                  float);
template double hungarian<int32_t, int32_t, double>(raft::handle_t const &,
                                                    GraphCOOView<int32_t, int32_t, double> const &,
                                                    int32_t,
                                                    int32_t const *,
                                                    int32_t *,
                                                    double);

template int32_t hungarian<int32_t, int32_t, int32_t>(
  raft::handle_t const &,
  GraphCOOView<int32_t, int32_t, int32_t> const &,
  int32_t,
  int32_t const *,
  int32_t *);

template float hungarian<int32_t, int32_t, float>(raft::handle_t const &,
                                                  GraphCOOView<int32_t, int32_t, float> const &,
                                                  int32_t,
                                                  int32_t const *,
                                                  int32_t *);
template double hungarian<int32_t, int32_t, double>(raft::handle_t const &,
                                                    GraphCOOView<int32_t, int32_t, double> const &,
                                                    int32_t,
                                                    int32_t const *,
                                                    int32_t *);

namespace dense {

template <typename index_t, typename weight_t>
weight_t hungarian(raft::handle_t const &handle,
                   weight_t const *costs,
                   index_t num_rows,
                   index_t num_cols,
                   index_t *assignment)
{
  rmm::cuda_stream_view stream_view{};

  return detail::hungarian(handle,
                           num_rows,
                           num_cols,
                           costs,
                           assignment,
                           detail::default_precision<weight_t>(),
                           stream_view);
}

template <typename index_t, typename weight_t>
weight_t hungarian(raft::handle_t const &handle,
                   weight_t const *costs,
                   index_t num_rows,
                   index_t num_cols,
                   index_t *assignment,
                   weight_t precision)
{
  rmm::cuda_stream_view stream_view{};

  return detail::hungarian(handle, num_rows, num_cols, costs, assignment, precision, stream_view);
}

template int32_t hungarian<int32_t, int32_t>(
  raft::handle_t const &, int32_t const *, int32_t, int32_t, int32_t *);
template float hungarian<int32_t, float>(
  raft::handle_t const &, float const *, int32_t, int32_t, int32_t *);
template double hungarian<int32_t, double>(
  raft::handle_t const &, double const *, int32_t, int32_t, int32_t *);
template int32_t hungarian<int32_t, int32_t>(
  raft::handle_t const &, int32_t const *, int32_t, int32_t, int32_t *, int32_t);
template float hungarian<int32_t, float>(
  raft::handle_t const &, float const *, int32_t, int32_t, int32_t *, float);
template double hungarian<int32_t, double>(
  raft::handle_t const &, double const *, int32_t, int32_t, int32_t *, double);

}  // namespace dense

}  // namespace cugraph
