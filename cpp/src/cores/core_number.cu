/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * ---------------------------------------------------------------------------*
 * @brief Core Number implementation
 *
 * @file core_number.cu
 * --------------------------------------------------------------------------*/

#include <cugraph.h>
#include "utilities/error_utils.h"
#include <Hornet.hpp>
#include <Static/CoreNumber/CoreNumber.cuh>
#include <rmm_utils.h>
#include <nvgraph_gdf.h>

gdf_error core_number_impl(gdf_graph *graph,
                          int *core_number) {
  using HornetGraph = hornet::gpu::HornetStatic<int>;
  using HornetInit  = hornet::HornetInit<int>;
  using CoreNumber  = hornets_nest::CoreNumberStatic;
  HornetInit init(graph->numberOfVertices, graph->adjList->indices->size,
      static_cast<int*>(graph->adjList->offsets->data),
      static_cast<int*>(graph->adjList->indices->data));
  HornetGraph hnt(init, hornet::DeviceType::DEVICE);
  CoreNumber cn(hnt, core_number);
  cn.run();
  return GDF_SUCCESS;
}

gdf_error gdf_core_number(gdf_graph *graph,
                          gdf_column *core_number) {
  GDF_REQUIRE(graph->adjList != nullptr, GDF_INVALID_API_CALL);
  GDF_REQUIRE(graph->adjList->offsets->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);
  GDF_REQUIRE(graph->adjList->indices->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);
  GDF_REQUIRE(core_number->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);
  GDF_REQUIRE(core_number->size == graph->numberOfVertices, GDF_COLUMN_SIZE_MISMATCH);

  return core_number_impl(graph, static_cast<int*>(core_number->data));
}

struct FilterEdges {
  int k;
  int* core_number;

  FilterEdges(int _k, thrust::device_ptr<int> core_num) :
    k(_k), core_number(core_num.get()) {}

  template <typename T>
  __host__ __device__
    bool operator()(T t) {
      int src = thrust::get<0>(t);
      int dst = thrust::get<1>(t);
      return (core_number[src] >= k) && (core_number[dst] >= k);
    }
};

template <typename WT>
gdf_error extract_edges(
    gdf_graph *i_graph,
    gdf_graph *o_graph,
    thrust::device_ptr<int> c_ptr,
    int k,
    int filteredEdgeCount) {
  hipStream_t stream{nullptr};

  //Allocate output columns
  o_graph->edgeList = new gdf_edge_list;
  o_graph->edgeList->src_indices = new gdf_column;
  o_graph->edgeList->dest_indices = new gdf_column;

  bool hasData = (i_graph->edgeList->edge_data != nullptr);

  //Allocate underlying memory for output columns
  int *o_src, *o_dst, *o_wgt;
  ALLOC_TRY((void**)&o_src, sizeof(int) * filteredEdgeCount, stream);
  ALLOC_TRY((void**)&o_dst, sizeof(int) * filteredEdgeCount, stream);

  int *i_src = static_cast<int*>(i_graph->edgeList->src_indices->data);
  int *i_dst = static_cast<int*>(i_graph->edgeList->dest_indices->data);
  WT  *i_wgt = nullptr;

  gdf_column_view(o_graph->edgeList->src_indices, o_src,
      nullptr, filteredEdgeCount, GDF_INT32);
  gdf_column_view(o_graph->edgeList->dest_indices, o_dst,
      nullptr, filteredEdgeCount, GDF_INT32);

  //Set pointers and allocate memory/columns in case input graph has edge_data
  if (hasData) {
    o_graph->edgeList->edge_data   = new gdf_column;
    ALLOC_TRY((void**)&o_wgt, sizeof(WT)  * filteredEdgeCount, stream);
    i_wgt = static_cast<WT*>(i_graph->edgeList->edge_data->data);
    gdf_column_view(o_graph->edgeList->edge_data,   o_wgt,
        nullptr, filteredEdgeCount, i_graph->edgeList->edge_data->dtype);
  }

  gdf_size_type nE = i_graph->edgeList->src_indices->size;

  //If an edge satisfies k-core conditions i.e. core_num[src] and core_num[dst]
  //are both greater than or equal to k, copy it to the output graph
  if (hasData) {
    auto inEdge = thrust::make_zip_iterator(thrust::make_tuple(
          thrust::device_pointer_cast(i_src),
          thrust::device_pointer_cast(i_dst),
          thrust::device_pointer_cast(i_wgt)));
    auto outEdge = thrust::make_zip_iterator(thrust::make_tuple(
          thrust::device_pointer_cast(o_src),
          thrust::device_pointer_cast(o_dst),
          thrust::device_pointer_cast(o_wgt)));
    auto ptr = thrust::copy_if(rmm::exec_policy(stream)->on(stream),
        inEdge, inEdge + nE,
        outEdge,
        FilterEdges(k, c_ptr));
    if ((ptr - outEdge) != filteredEdgeCount) { return GDF_CUDA_ERROR; }
  } else {
    auto inEdge = thrust::make_zip_iterator(thrust::make_tuple(
          thrust::device_pointer_cast(i_src),
          thrust::device_pointer_cast(i_dst)));
    auto outEdge = thrust::make_zip_iterator(thrust::make_tuple(
          thrust::device_pointer_cast(o_src),
          thrust::device_pointer_cast(o_dst)));
    auto ptr = thrust::copy_if(rmm::exec_policy(stream)->on(stream),
        inEdge, inEdge + nE,
        outEdge,
        FilterEdges(k, c_ptr));
    if ((ptr - outEdge) != filteredEdgeCount) { return GDF_CUDA_ERROR; }
  }

  return GDF_SUCCESS;
}

//Extract a subgraph from in_graph (with or without weights)
//to out_graph based on whether edges in in_graph satisfy kcore
//conditions.
//i.e. All edges (s,d,w) in in_graph are copied over to out_graph
//if core_num[s] and core_num[d] are greater than or equal to k.
gdf_error extract_subgraph(gdf_graph *in_graph,
                           gdf_graph *out_graph,
                           int * vid,
                           int * core_num,
                           int k,
                           gdf_size_type len,
                           gdf_size_type nV) {
  hipStream_t stream{nullptr};

  rmm::device_vector<int> c;
  thrust::device_ptr<int> c_ptr = thrust::device_pointer_cast(core_num);
  //We cannot assume that the user provided core numbers per vertex will be in
  //order. Therefore, they need to be reordered by the vertex ids in a temporary
  //array.
  c.resize(nV, 0);
  thrust::device_ptr<int> v_ptr = thrust::device_pointer_cast(vid);
  thrust::scatter(rmm::exec_policy(stream)->on(stream),
      c_ptr, c_ptr + len,
      v_ptr, c.begin());
  c_ptr = thrust::device_pointer_cast(c.data().get());

  gdf_error err = gdf_add_edge_list(in_graph);
  thrust::device_ptr<int> src =
    thrust::device_pointer_cast(static_cast<int*>(in_graph->edgeList->src_indices->data));
  thrust::device_ptr<int> dst =
    thrust::device_pointer_cast(static_cast<int*>(in_graph->edgeList->dest_indices->data));

  //Count number of edges in the input graph that satisfy kcore conditions
  //i.e. core_num[src] and core_num[dst] are both greater than or equal to k
  gdf_size_type nE = in_graph->edgeList->src_indices->size;
  auto edge = thrust::make_zip_iterator(thrust::make_tuple(src, dst));
  int filteredEdgeCount = thrust::count_if(rmm::exec_policy(stream)->on(stream),
      edge, edge + nE, FilterEdges(k, c_ptr));

  //Extract the relevant edges that have satisfied k-core conditions and put them in the output graph
  if (in_graph->edgeList->edge_data != nullptr) {
    switch (in_graph->edgeList->edge_data->dtype) {
      case GDF_FLOAT32:   return extract_edges<float> (in_graph, out_graph, c_ptr, k, filteredEdgeCount);
      case GDF_FLOAT64:   return extract_edges<double>(in_graph, out_graph, c_ptr, k, filteredEdgeCount);
      default: return GDF_UNSUPPORTED_DTYPE;
    }
  }
  else {
    return extract_edges<float> (in_graph, out_graph, c_ptr, k, filteredEdgeCount);
  }
}

gdf_error gdf_k_core(gdf_graph *in_graph,
                     int k,
                     gdf_column *vertex_id,
                     gdf_column *core_number,
                     gdf_graph *out_graph) {
  GDF_REQUIRE(out_graph != nullptr && in_graph != nullptr, GDF_INVALID_API_CALL);
  gdf_size_type nV = in_graph->numberOfVertices;

  GDF_REQUIRE(in_graph->adjList->offsets->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);
  GDF_REQUIRE(in_graph->adjList->indices->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);
  GDF_REQUIRE((vertex_id != nullptr) && (core_number != nullptr), GDF_INVALID_API_CALL);
  GDF_REQUIRE(vertex_id->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);
  GDF_REQUIRE(core_number->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);
  GDF_REQUIRE(core_number->size == vertex_id->size, GDF_INVALID_API_CALL);
  GDF_REQUIRE(core_number->size == nV, GDF_INVALID_API_CALL);
  GDF_REQUIRE(k >= 0, GDF_INVALID_API_CALL);

  int * vertex_identifier_ptr = static_cast<int*>(vertex_id->data);
  int * core_number_ptr = static_cast<int*>(core_number->data);
  gdf_size_type vLen = vertex_id->size;

  gdf_error err = extract_subgraph(in_graph, out_graph,
      vertex_identifier_ptr, core_number_ptr,
      k, vLen, nV);
  GDF_REQUIRE(err, GDF_SUCCESS);

  return GDF_SUCCESS;
}
