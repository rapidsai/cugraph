/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * ---------------------------------------------------------------------------*
 * @brief Core Number implementation
 *
 * @file core_number.cu
 * --------------------------------------------------------------------------*/

#include <cugraph.h>
#include "utilities/error_utils.h"
#include <Hornet.hpp>
#include <Static/CoreNumber/CoreNumber.cuh>
#include <rmm_utils.h>
#include <nvgraph_gdf.h>

namespace cugraph {
namespace detail {

void core_number_impl(Graph *graph,
                          int *core_number) {
  using HornetGraph = hornet::gpu::HornetStatic<int>;
  using HornetInit  = hornet::HornetInit<int>;
  using CoreNumber  = hornets_nest::CoreNumberStatic;
  HornetInit init(graph->numberOfVertices, graph->adjList->indices->size,
      static_cast<int*>(graph->adjList->offsets->data),
      static_cast<int*>(graph->adjList->indices->data));
  HornetGraph hnt(init, hornet::DeviceType::DEVICE);
  CoreNumber cn(hnt, core_number);
  cn.run();
  
}

struct FilterEdges {
  int k;
  int* core_number;

  FilterEdges(int _k, thrust::device_ptr<int> core_num) :
    k(_k), core_number(core_num.get()) {}

  template <typename T>
  __host__ __device__
    bool operator()(T t) {
      int src = thrust::get<0>(t);
      int dst = thrust::get<1>(t);
      return (core_number[src] >= k) && (core_number[dst] >= k);
    }
};

template <typename WT>
void extract_edges(
    Graph *i_graph,
    Graph *o_graph,
    thrust::device_ptr<int> c_ptr,
    int k,
    int filteredEdgeCount) {
  hipStream_t stream{nullptr};

  //Allocate output columns
  o_graph->edgeList = new gdf_edge_list;
  o_graph->edgeList->src_indices = new gdf_column;
  o_graph->edgeList->dest_indices = new gdf_column;
  o_graph->edgeList->ownership = 2;

  bool hasData = (i_graph->edgeList->edge_data != nullptr);

  //Allocate underlying memory for output columns
  int *o_src, *o_dst, *o_wgt;
  ALLOC_TRY((void**)&o_src, sizeof(int) * filteredEdgeCount, stream);
  ALLOC_TRY((void**)&o_dst, sizeof(int) * filteredEdgeCount, stream);

  int *i_src = static_cast<int*>(i_graph->edgeList->src_indices->data);
  int *i_dst = static_cast<int*>(i_graph->edgeList->dest_indices->data);
  WT  *i_wgt = nullptr;

  gdf_column_view(o_graph->edgeList->src_indices, o_src,
      nullptr, filteredEdgeCount, GDF_INT32);
  gdf_column_view(o_graph->edgeList->dest_indices, o_dst,
      nullptr, filteredEdgeCount, GDF_INT32);

  //Set pointers and allocate memory/columns in case input graph has edge_data
  if (hasData) {
    o_graph->edgeList->edge_data   = new gdf_column;
    ALLOC_TRY((void**)&o_wgt, sizeof(WT)  * filteredEdgeCount, stream);
    i_wgt = static_cast<WT*>(i_graph->edgeList->edge_data->data);
    gdf_column_view(o_graph->edgeList->edge_data,   o_wgt,
        nullptr, filteredEdgeCount, i_graph->edgeList->edge_data->dtype);
  }

  gdf_size_type nE = i_graph->edgeList->src_indices->size;

  //If an edge satisfies k-core conditions i.e. core_num[src] and core_num[dst]
  //are both greater than or equal to k, copy it to the output graph
  if (hasData) {
    auto inEdge = thrust::make_zip_iterator(thrust::make_tuple(
          thrust::device_pointer_cast(i_src),
          thrust::device_pointer_cast(i_dst),
          thrust::device_pointer_cast(i_wgt)));
    auto outEdge = thrust::make_zip_iterator(thrust::make_tuple(
          thrust::device_pointer_cast(o_src),
          thrust::device_pointer_cast(o_dst),
          thrust::device_pointer_cast(o_wgt)));
    auto ptr = thrust::copy_if(rmm::exec_policy(stream)->on(stream),
        inEdge, inEdge + nE,
        outEdge,
        FilterEdges(k, c_ptr));
    if ((ptr - outEdge) != filteredEdgeCount) { CUGRAPH_FAIL("Edge extraction failed"); }
  } else {
    auto inEdge = thrust::make_zip_iterator(thrust::make_tuple(
          thrust::device_pointer_cast(i_src),
          thrust::device_pointer_cast(i_dst)));
    auto outEdge = thrust::make_zip_iterator(thrust::make_tuple(
          thrust::device_pointer_cast(o_src),
          thrust::device_pointer_cast(o_dst)));
    auto ptr = thrust::copy_if(rmm::exec_policy(stream)->on(stream),
        inEdge, inEdge + nE,
        outEdge,
        FilterEdges(k, c_ptr));
    if ((ptr - outEdge) != filteredEdgeCount) { CUGRAPH_FAIL("Edge extraction failed"); }
  }
  
}

} //namespace

//Extract a subgraph from in_graph (with or without weights)
//to out_graph based on whether edges in in_graph satisfy kcore
//conditions.
//i.e. All edges (s,d,w) in in_graph are copied over to out_graph
//if core_num[s] and core_num[d] are greater than or equal to k.
void extract_subgraph(Graph *in_graph,
                           Graph *out_graph,
                           int * vid,
                           int * core_num,
                           int k,
                           gdf_size_type len,
                           gdf_size_type nV) {
  hipStream_t stream{nullptr};

  rmm::device_vector<int> c;
  thrust::device_ptr<int> c_ptr = thrust::device_pointer_cast(core_num);
  //We cannot assume that the user provided core numbers per vertex will be in
  //order. Therefore, they need to be reordered by the vertex ids in a temporary
  //array.
  c.resize(nV, 0);
  thrust::device_ptr<int> v_ptr = thrust::device_pointer_cast(vid);
  thrust::scatter(rmm::exec_policy(stream)->on(stream),
      c_ptr, c_ptr + len,
      v_ptr, c.begin());
  c_ptr = thrust::device_pointer_cast(c.data().get());

  cugraph::add_edge_list(in_graph);
  thrust::device_ptr<int> src =
    thrust::device_pointer_cast(static_cast<int*>(in_graph->edgeList->src_indices->data));
  thrust::device_ptr<int> dst =
    thrust::device_pointer_cast(static_cast<int*>(in_graph->edgeList->dest_indices->data));

  //Count number of edges in the input graph that satisfy kcore conditions
  //i.e. core_num[src] and core_num[dst] are both greater than or equal to k
  gdf_size_type nE = in_graph->edgeList->src_indices->size;
  auto edge = thrust::make_zip_iterator(thrust::make_tuple(src, dst));
  int filteredEdgeCount = thrust::count_if(rmm::exec_policy(stream)->on(stream),
      edge, edge + nE, detail::FilterEdges(k, c_ptr));

  //Extract the relevant edges that have satisfied k-core conditions and put them in the output graph
  if (in_graph->edgeList->edge_data != nullptr) {
    switch (in_graph->edgeList->edge_data->dtype) {
      case GDF_FLOAT32:   return detail::extract_edges<float> (in_graph, out_graph, c_ptr, k, filteredEdgeCount);
      case GDF_FLOAT64:   return detail::extract_edges<double>(in_graph, out_graph, c_ptr, k, filteredEdgeCount);
      default: CUGRAPH_FAIL("Unsupported data type: edge data needs to be float32 or float64");
    }
  }
  else {
    return detail::extract_edges<float> (in_graph, out_graph, c_ptr, k, filteredEdgeCount);
  }
}

void core_number(Graph *graph,
                gdf_column *core_number) {

  CHECK_GRAPH(graph)
  CUGRAPH_EXPECTS(graph->adjList->offsets->dtype == GDF_INT32, "Unsupported data type: graph needs to be int32");
  CUGRAPH_EXPECTS(graph->adjList->indices->dtype == GDF_INT32, "Unsupported data type: graph needs to be int32");
  CUGRAPH_EXPECTS(core_number->dtype == GDF_INT32, "Unsupported data type: core number needs to be int32");
  CUGRAPH_EXPECTS(core_number->size == graph->numberOfVertices, "Column size mismatch");

  return detail::core_number_impl(graph, static_cast<int*>(core_number->data));
}

void k_core(Graph *in_graph,
                     int k,
                     gdf_column *vertex_id,
                     gdf_column *core_number,
                     Graph *out_graph) {

  CUGRAPH_EXPECTS(out_graph != nullptr, "Invalid API parameter: out_graph is NULL");
  CUGRAPH_EXPECTS(in_graph != nullptr, "Invalid API parameter: in_graph is NULL");

  gdf_size_type nV = in_graph->numberOfVertices;
  CUGRAPH_EXPECTS(in_graph->adjList->offsets->dtype == GDF_INT32, "Unsupported data type: graph needs to be int32");
  CUGRAPH_EXPECTS(in_graph->adjList->indices->dtype == GDF_INT32, "Unsupported data type: graph needs to be int32");
  CUGRAPH_EXPECTS((vertex_id != nullptr) && (core_number != nullptr), "Invalid API parameter");
  CUGRAPH_EXPECTS(vertex_id->dtype == GDF_INT32, "Unsupported data type");
  CUGRAPH_EXPECTS(core_number->dtype == GDF_INT32, "Unsupported data type");
  CUGRAPH_EXPECTS(core_number->size == vertex_id->size, "Invalid API parameter");
  CUGRAPH_EXPECTS(core_number->size == nV, "Invalid API parameter");
  CUGRAPH_EXPECTS(k >= 0, "Invalid API parameter");

  int * vertex_identifier_ptr = static_cast<int*>(vertex_id->data);
  int * core_number_ptr = static_cast<int*>(core_number->data);
  gdf_size_type vLen = vertex_id->size;

  extract_subgraph(in_graph, out_graph,
      vertex_identifier_ptr, core_number_ptr,
      k, vLen, nV);
}

} //namespace cugraph