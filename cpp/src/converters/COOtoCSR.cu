/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <functions.hpp>
#include "COOtoCSR.cuh"

namespace cugraph {

template <typename vertex_t, typename edge_t>
vertex_t coo2csr(
  edge_t num_edges, vertex_t const *src, vertex_t const *dst, edge_t **offsets, vertex_t **indices)
{
  CSR_Result<vertex_t> result;
  ConvertCOOtoCSR(src, dst, num_edges, result);

  *offsets = result.rowOffsets;
  *indices = result.colIndices;
  return result.size;
}

template <typename vertex_t, typename edge_t, typename weight_t>
vertex_t coo2csr_weighted(edge_t num_edges,
                          vertex_t const *src,
                          vertex_t const *dst,
                          weight_t const *weights,
                          edge_t **offsets,
                          vertex_t **indices,
                          weight_t **csr_weights)
{
  CSR_Result_Weighted<vertex_t, weight_t> result;
  ConvertCOOtoCSR_weighted(src, dst, weights, num_edges, result);

  *offsets     = result.rowOffsets;
  *indices     = result.colIndices;
  *csr_weights = result.edgeWeights;

  return result.size;
}

template int32_t coo2csr<int32_t, int32_t>(
  int32_t, int32_t const *, int32_t const *, int32_t **, int32_t **);
template int32_t coo2csr_weighted<int32_t, int32_t, float>(
  int32_t, int32_t const *, int32_t const *, float const *, int32_t **, int32_t **, float **);
template int32_t coo2csr_weighted<int32_t, int32_t, double>(
  int32_t, int32_t const *, int32_t const *, double const *, int32_t **, int32_t **, double **);

}  // namespace cugraph
