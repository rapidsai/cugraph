/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <community/flatten_dendrogram.cuh>
#include <community/legacy/leiden.cuh>

#include <rmm/device_uvector.hpp>

namespace cugraph {

template <typename vertex_t, typename edge_t, typename weight_t>
std::pair<size_t, weight_t> leiden(raft::handle_t const& handle,
                                   legacy::GraphCSRView<vertex_t, edge_t, weight_t> const& graph,
                                   vertex_t* clustering,
                                   size_t max_level,
                                   weight_t resolution)
{
  CUGRAPH_EXPECTS(graph.edge_data != nullptr,
                  "Invalid input argument: leiden expects a weighted graph");
  CUGRAPH_EXPECTS(clustering != nullptr,
                  "Invalid input argument: clustering is null, should be a device pointer to "
                  "memory for storing the result");

  legacy::Leiden<legacy::GraphCSRView<vertex_t, edge_t, weight_t>> runner(handle, graph);
  weight_t wt = runner(max_level, resolution);

  rmm::device_uvector<vertex_t> vertex_ids_v(graph.number_of_vertices, handle.get_stream());

  thrust::copy(handle.get_thrust_policy(),
               thrust::make_counting_iterator<vertex_t>(0),  // MNMG - base vertex id
               thrust::make_counting_iterator<vertex_t>(
                 graph.number_of_vertices),  // MNMG - base vertex id + number_of_vertices
               vertex_ids_v.begin());

  partition_at_level<vertex_t, false>(handle,
                                      runner.get_dendrogram(),
                                      vertex_ids_v.data(),
                                      clustering,
                                      runner.get_dendrogram().num_levels());

  // FIXME: Consider returning the Dendrogram at some point
  return std::make_pair(runner.get_dendrogram().num_levels(), wt);
}

// Explicit template instantations
template std::pair<size_t, float> leiden(raft::handle_t const&,
                                         legacy::GraphCSRView<int32_t, int32_t, float> const&,
                                         int32_t*,
                                         size_t,
                                         float);

template std::pair<size_t, double> leiden(raft::handle_t const&,
                                          legacy::GraphCSRView<int32_t, int32_t, double> const&,
                                          int32_t*,
                                          size_t,
                                          double);

}  // namespace cugraph
