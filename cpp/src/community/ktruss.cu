#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/**
 * ---------------------------------------------------------------------------*
 * @brief KTruss implementation
 *
 * @file ktruss.cu
 * --------------------------------------------------------------------------*/

#include <utilities/error.hpp>

#include <Hornet.hpp>
#include <StandardAPI.hpp>
#include <algorithms.hpp>
#include "Static/KTruss/KTruss.cuh"

using namespace hornets_nest;

namespace cugraph {

namespace detail {

template <typename VT, typename ET, typename WT>
std::unique_ptr<experimental::GraphCOO<VT, ET, WT>> ktruss_subgraph_impl(
  experimental::GraphCOOView<VT, ET, WT> const &graph, int k, rmm::mr::device_memory_resource *mr)
{
  using HornetGraph = hornet::gpu::Hornet<VT>;
  using UpdatePtr   = hornet::BatchUpdatePtr<VT, hornet::EMPTY, hornet::DeviceType::DEVICE>;
  using Update      = hornet::gpu::BatchUpdate<VT>;
  hipStream_t stream{nullptr};
  UpdatePtr ptr(graph.number_of_edges, graph.src_indices, graph.dst_indices);
  Update batch(ptr);

  HornetGraph hnt(graph.number_of_vertices + 1);
  hnt.insert(batch);
  CUGRAPH_EXPECTS(hipPeekAtLastError() == hipSuccess, "KTruss : Failed to initialize graph");

  KTruss kt(hnt);

  kt.init();
  kt.reset();
  kt.createOffSetArray();
  // NOTE : These parameters will become obsolete once we move to the updated
  // algorithm (https://ieeexplore.ieee.org/document/8547581)
  kt.setInitParameters(4,      // Number of threads per block per list intersection
                       8,      // Number of intersections per block
                       2,      // log2(Number of threads)
                       64000,  // Total number of blocks launched
                       32);    // Thread block dimension
  kt.reset();
  kt.sortHornet();

  kt.runForK(k);
  CUGRAPH_EXPECTS(hipPeekAtLastError() == hipSuccess, "KTruss : Failed to run");

  auto out_graph = std::make_unique<experimental::GraphCOO<VT, ET, WT>>(
    graph.number_of_vertices, kt.getGraphEdgeCount(), graph.has_data(), stream, mr);

  kt.copyGraph(out_graph->src_indices(), out_graph->dst_indices());

  kt.release();
  CUGRAPH_EXPECTS(hipPeekAtLastError() == hipSuccess, "KTruss : Failed to release");

  return out_graph;
}
template <typename VT, typename ET, typename WT>
std::unique_ptr<experimental::GraphCOO<VT, ET, WT>> weighted_ktruss_subgraph_impl(
  experimental::GraphCOOView<VT, ET, WT> const &graph, int k, rmm::mr::device_memory_resource *mr)
{
  using HornetGraph = hornet::gpu::Hornet<VT, hornet::EMPTY, hornet::TypeList<WT>>;
  using UpdatePtr   = hornet::BatchUpdatePtr<VT, hornet::TypeList<WT>, hornet::DeviceType::DEVICE>;
  using Update      = hornet::gpu::BatchUpdate<VT, hornet::TypeList<WT>>;
  hipStream_t stream{nullptr};
  UpdatePtr ptr(graph.number_of_edges, graph.src_indices, graph.dst_indices, graph.edge_data);
  Update batch(ptr);

  HornetGraph hnt(graph.number_of_vertices + 1);
  hnt.insert(batch);
  CUGRAPH_EXPECTS(hipPeekAtLastError() == hipSuccess, "KTruss : Failed to initialize graph");

  KTrussWeighted<WT> kt(hnt);

  kt.init();
  kt.reset();
  kt.createOffSetArray();
  // NOTE : These parameters will become obsolete once we move to the updated
  // algorithm (https://ieeexplore.ieee.org/document/8547581)
  kt.setInitParameters(4,      // Number of threads per block per list intersection
                       8,      // Number of intersections per block
                       2,      // log2(Number of threads)
                       64000,  // Total number of blocks launched
                       32);    // Thread block dimension
  kt.reset();
  kt.sortHornet();

  kt.runForK(k);
  CUGRAPH_EXPECTS(hipPeekAtLastError() == hipSuccess, "KTruss : Failed to run");

  auto out_graph = std::make_unique<experimental::GraphCOO<VT, ET, WT>>(
    graph.number_of_vertices, kt.getGraphEdgeCount(), graph.has_data(), stream, mr);

  kt.copyGraph(out_graph->src_indices(), out_graph->dst_indices(), out_graph->edge_data());

  kt.release();
  CUGRAPH_EXPECTS(hipPeekAtLastError() == hipSuccess, "KTruss : Failed to release");

  return out_graph;
}

}  // namespace detail

template <typename VT, typename ET, typename WT>
std::unique_ptr<experimental::GraphCOO<VT, ET, WT>> k_truss_subgraph(
  experimental::GraphCOOView<VT, ET, WT> const &graph, int k, rmm::mr::device_memory_resource *mr)
{
  CUGRAPH_EXPECTS(graph.src_indices != nullptr, "Graph source indices cannot be a nullptr");
  CUGRAPH_EXPECTS(graph.dst_indices != nullptr, "Graph destination indices cannot be a nullptr");

  if (graph.edge_data == nullptr) {
    return detail::ktruss_subgraph_impl(graph, k, mr);
  } else {
    return detail::weighted_ktruss_subgraph_impl(graph, k, mr);
  }
}

template std::unique_ptr<experimental::GraphCOO<int32_t, int32_t, float>>
k_truss_subgraph<int, int, float>(experimental::GraphCOOView<int, int, float> const &,
                                  int,
                                  rmm::mr::device_memory_resource *);

template std::unique_ptr<experimental::GraphCOO<int32_t, int32_t, double>>
k_truss_subgraph<int, int, double>(experimental::GraphCOOView<int, int, double> const &,
                                   int,
                                   rmm::mr::device_memory_resource *);

}  // namespace cugraph
