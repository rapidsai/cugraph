// -*-c++-*-

/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
/** ---------------------------------------------------------------------------*
 * @brief Wrapper functions for Nvgraph
 *
 * @file nvgraph_gdf.cu
 * ---------------------------------------------------------------------------**/

#include <cugraph.h>
#include <nvgraph/nvgraph.h>
#include <rmm_utils.h>
#include <thrust/device_vector.h>
#include <ctime>
#include "converters/nvgraph.cuh"
#include "utilities/error_utils.h"

namespace cugraph {

void balancedCutClustering_nvgraph(Graph* gdf_G,
                                   const int num_clusters,
                                   const int num_eigen_vects,
                                   const float evs_tolerance,
                                   const int evs_max_iter,
                                   const float kmean_tolerance,
                                   const int kmean_max_iter,
                                   gdf_column* clustering)
{
  CHECK_GRAPH(gdf_G);
  CUGRAPH_EXPECTS(clustering != nullptr, "Invalid API parameter: clustering is NULL");
  CUGRAPH_EXPECTS(clustering->data != nullptr, "Invalid API parameter: clustering data is NULL");
  CUGRAPH_EXPECTS(!clustering->valid, "Column must be valid");

  // Initialize Nvgraph and wrap the graph
  nvgraphHandle_t nvg_handle    = nullptr;
  nvgraphGraphDescr_t nvgraph_G = nullptr;
  hipDataType settype;
  rmm::device_vector<double> d_val;

  NVG_TRY(nvgraphCreate(&nvg_handle));
  createGraph_nvgraph(nvg_handle, gdf_G, &nvgraph_G, false);
  int weight_index = 0;

  hipStream_t stream{nullptr};

  if (gdf_G->adjList->edge_data == nullptr) {
    // use a fp64 vector  [1,...,1]
    settype = HIP_R_64F;
    d_val.resize(gdf_G->adjList->indices->size);
    thrust::fill(rmm::exec_policy(stream)->on(stream), d_val.begin(), d_val.end(), 1.0);
    NVG_TRY(nvgraphAttachEdgeData(
      nvg_handle, nvgraph_G, weight_index, settype, (void*)thrust::raw_pointer_cast(d_val.data())));
  } else {
    switch (gdf_G->adjList->edge_data->dtype) {
      case GDF_FLOAT32: settype = HIP_R_32F; break;
      case GDF_FLOAT64: settype = HIP_R_64F; break;
      default:
        CUGRAPH_FAIL("Unsupported data type: Graph Edge Data Type Needs to be float32 or float64");
    }
  }

  // Pack parameters for call to Nvgraph
  SpectralClusteringParameter param;
  param.n_clusters      = num_clusters;
  param.n_eig_vects     = num_eigen_vects;
  param.algorithm       = NVGRAPH_BALANCED_CUT_LANCZOS;
  param.evs_tolerance   = evs_tolerance;
  param.evs_max_iter    = evs_max_iter;
  param.kmean_tolerance = kmean_tolerance;
  param.kmean_max_iter  = kmean_max_iter;

  // Make call to Nvgraph balancedCutClustering
  void* eig_vals      = malloc(num_eigen_vects * sizeof(double));
  void* eig_vects     = malloc(num_eigen_vects * clustering->size * sizeof(double));
  nvgraphStatus_t err = nvgraphSpectralClustering(
    nvg_handle, nvgraph_G, weight_index, &param, (int*)clustering->data, eig_vals, eig_vects);
  free(eig_vals);
  free(eig_vects);
  NVG_TRY(err);
  NVG_TRY(nvgraphDestroyGraphDescr(nvg_handle, nvgraph_G));
  NVG_TRY(nvgraphDestroy(nvg_handle));
}

void spectralModularityMaximization_nvgraph(Graph* gdf_G,
                                            const int n_clusters,
                                            const int n_eig_vects,
                                            const float evs_tolerance,
                                            const int evs_max_iter,
                                            const float kmean_tolerance,
                                            const int kmean_max_iter,
                                            gdf_column* clustering)
{
  CHECK_GRAPH(gdf_G);
  CUGRAPH_EXPECTS(clustering != nullptr, "Invalid API parameter: clustering is NULL");
  CUGRAPH_EXPECTS(clustering->data != nullptr, "Invalid API parameter: clustering data is NULL");
  CUGRAPH_EXPECTS(!clustering->valid, "Column must be valid");

  // Ensure that the input graph has values
  CUGRAPH_EXPECTS(gdf_G->adjList->edge_data != nullptr, "Invalid API parameter: edge data is NULL");

  // Initialize Nvgraph and wrap the graph
  nvgraphHandle_t nvg_handle    = nullptr;
  nvgraphGraphDescr_t nvgraph_G = nullptr;
  NVG_TRY(nvgraphCreate(&nvg_handle));
  createGraph_nvgraph(nvg_handle, gdf_G, &nvgraph_G, false);
  int weight_index = 0;

  // Pack parameters for call to Nvgraph
  SpectralClusteringParameter param;
  param.n_clusters      = n_clusters;
  param.n_eig_vects     = n_eig_vects;
  param.algorithm       = NVGRAPH_MODULARITY_MAXIMIZATION;
  param.evs_tolerance   = evs_tolerance;
  param.evs_max_iter    = evs_max_iter;
  param.kmean_tolerance = kmean_tolerance;
  param.kmean_max_iter  = kmean_max_iter;

  // Make call to Nvgraph balancedCutClustering
  void* eig_vals      = malloc(n_eig_vects * sizeof(double));
  void* eig_vects     = malloc(n_eig_vects * clustering->size * sizeof(double));
  nvgraphStatus_t err = nvgraphSpectralClustering(
    nvg_handle, nvgraph_G, weight_index, &param, (int*)clustering->data, eig_vals, eig_vects);
  free(eig_vals);
  free(eig_vects);
  NVG_TRY(err);
  NVG_TRY(nvgraphDestroyGraphDescr(nvg_handle, nvgraph_G));
  NVG_TRY(nvgraphDestroy(nvg_handle));
}

void analyzeClustering_modularity_nvgraph(Graph* gdf_G,
                                          const int n_clusters,
                                          gdf_column* clustering,
                                          float* score)
{
  CHECK_GRAPH(gdf_G);
  CUGRAPH_EXPECTS(gdf_G->adjList->edge_data != nullptr, "Invalid API parameter: edge data is NULL");
  CUGRAPH_EXPECTS(clustering != nullptr, "Invalid API parameter: clustering is NULL");
  CUGRAPH_EXPECTS(clustering->data != nullptr, "Invalid API parameter: clustering data is NULL");
  CUGRAPH_EXPECTS(!clustering->valid, "Column must be valid");

  // Initialize Nvgraph and wrap the graph
  nvgraphHandle_t nvg_handle    = nullptr;
  nvgraphGraphDescr_t nvgraph_G = nullptr;
  NVG_TRY(nvgraphCreate(&nvg_handle));
  createGraph_nvgraph(nvg_handle, gdf_G, &nvgraph_G, false);
  int weight_index = 0;

  // Make Nvgraph call

  NVG_TRY(nvgraphAnalyzeClustering(nvg_handle,
                                   nvgraph_G,
                                   weight_index,
                                   n_clusters,
                                   (const int*)clustering->data,
                                   NVGRAPH_MODULARITY,
                                   score));
}

void analyzeClustering_edge_cut_nvgraph(Graph* gdf_G,
                                        const int n_clusters,
                                        gdf_column* clustering,
                                        float* score)
{
  CHECK_GRAPH(gdf_G);
  CUGRAPH_EXPECTS(clustering != nullptr, "Invalid API parameter: clustering is NULL");
  CUGRAPH_EXPECTS(clustering->data != nullptr, "Invalid API parameter: clustering data is NULL");
  CUGRAPH_EXPECTS(!clustering->valid, "Column must be valid");

  // Initialize Nvgraph and wrap the graph
  nvgraphHandle_t nvg_handle    = nullptr;
  nvgraphGraphDescr_t nvgraph_G = nullptr;
  hipDataType settype;
  rmm::device_vector<double> d_val;

  NVG_TRY(nvgraphCreate(&nvg_handle));
  createGraph_nvgraph(nvg_handle, gdf_G, &nvgraph_G, false);
  int weight_index = 0;

  hipStream_t stream{nullptr};

  if (gdf_G->adjList->edge_data == nullptr) {
    // use a fp64 vector  [1,...,1]
    settype = HIP_R_64F;
    d_val.resize(gdf_G->adjList->indices->size);
    thrust::fill(rmm::exec_policy(stream)->on(stream), d_val.begin(), d_val.end(), 1.0);
    NVG_TRY(nvgraphAttachEdgeData(
      nvg_handle, nvgraph_G, weight_index, settype, (void*)thrust::raw_pointer_cast(d_val.data())));
  } else {
    switch (gdf_G->adjList->edge_data->dtype) {
      case GDF_FLOAT32: settype = HIP_R_32F; break;
      case GDF_FLOAT64: settype = HIP_R_64F; break;
      default:
        CUGRAPH_FAIL("Unsupported data type: Graph Edge Data Type Needs to be float32 or float64");
    }
  }

  // Make Nvgraph call

  NVG_TRY(nvgraphAnalyzeClustering(nvg_handle,
                                   nvgraph_G,
                                   weight_index,
                                   n_clusters,
                                   (const int*)clustering->data,
                                   NVGRAPH_EDGE_CUT,
                                   score));
}

void analyzeClustering_ratio_cut_nvgraph(Graph* gdf_G,
                                         const int n_clusters,
                                         gdf_column* clustering,
                                         float* score)
{
  CHECK_GRAPH(gdf_G);
  CUGRAPH_EXPECTS(gdf_G->adjList->edge_data != nullptr,
                  "Invalid API parameter: graph edge data is NULL");
  CUGRAPH_EXPECTS(clustering != nullptr, "Invalid API parameter: clustering is NULL");
  CUGRAPH_EXPECTS(clustering->data != nullptr, "Invalid API parameter: clustering data is NULL");
  CUGRAPH_EXPECTS(!clustering->valid, "Column must be valid");

  // Initialize Nvgraph and wrap the graph
  nvgraphHandle_t nvg_handle    = nullptr;
  nvgraphGraphDescr_t nvgraph_G = nullptr;
  NVG_TRY(nvgraphCreate(&nvg_handle));
  createGraph_nvgraph(nvg_handle, gdf_G, &nvgraph_G, false);
  int weight_index = 0;

  // Make Nvgraph call

  NVG_TRY(nvgraphAnalyzeClustering(nvg_handle,
                                   nvgraph_G,
                                   weight_index,
                                   n_clusters,
                                   (const int*)clustering->data,
                                   NVGRAPH_RATIO_CUT,
                                   score));
}

void extract_subgraph_vertex_nvgraph(Graph* gdf_G, gdf_column* vertices, Graph* result)
{
  CHECK_GRAPH(gdf_G);
  CUGRAPH_EXPECTS(vertices != nullptr, "Invalid API parameter: vertices is NULL");
  CUGRAPH_EXPECTS(vertices->data != nullptr, "Invalid API parameter: vertice data is NULL");
  CUGRAPH_EXPECTS(!vertices->valid, "vertices must be valid");

  // Initialize Nvgraph and wrap the graph
  nvgraphHandle_t nvg_handle = nullptr;
  nvgraphGraphDescr_t nvg_G  = nullptr;
  NVG_TRY(nvgraphCreate(&nvg_handle));
  createGraph_nvgraph(nvg_handle, gdf_G, &nvg_G, false);

  // Create an Nvgraph graph descriptor for the result and initialize
  nvgraphGraphDescr_t nvg_result = nullptr;
  NVG_TRY(nvgraphCreateGraphDescr(nvg_handle, &nvg_result));

  // Call Nvgraph function to get subgraph (into nv_result descriptor)
  NVG_TRY(nvgraphExtractSubgraphByVertex(
    nvg_handle, nvg_G, nvg_result, (int*)vertices->data, vertices->size));

  // Get the vertices and edges of the created subgraph to allocate memory:
  nvgraphCSRTopology32I_st topo;
  topo.source_offsets      = nullptr;
  topo.destination_indices = nullptr;
  nvgraphTopologyType_t TT = NVGRAPH_CSR_32;
  NVG_TRY(nvgraphGetGraphStructure(nvg_handle, nvg_result, (void*)&topo, &TT));
  if (TT != NVGRAPH_CSR_32) CUGRAPH_FAIL("Unsupported nvgraph topology: Only CSR 32 is supported");
  int num_verts              = topo.nvertices;
  int num_edges              = topo.nedges;
  result->adjList            = new gdf_adj_list;
  result->adjList->offsets   = new gdf_column;
  result->adjList->indices   = new gdf_column;
  result->adjList->ownership = 0;
  int *offsets, *indices;

  hipStream_t stream{nullptr};

  ALLOC_TRY((void**)&offsets, sizeof(int32_t) * (num_verts + 1), stream);
  ALLOC_TRY((void**)&indices, sizeof(int32_t) * num_edges, stream);

  gdf_column_view(result->adjList->offsets, offsets, nullptr, num_verts + 1, GDF_INT32);
  gdf_column_view(result->adjList->indices, indices, nullptr, num_edges, GDF_INT32);

  // Call nvgraphGetGraphStructure again to copy out the data
  topo.source_offsets      = (int*)result->adjList->offsets->data;
  topo.destination_indices = (int*)result->adjList->indices->data;
  NVG_TRY(nvgraphGetGraphStructure(nvg_handle, nvg_result, (void*)&topo, &TT));
}

void triangle_count_nvgraph(Graph* G, uint64_t* result)
{
  CHECK_GRAPH(G);

  // Initialize Nvgraph and wrap the graph
  nvgraphHandle_t nvg_handle = nullptr;
  nvgraphGraphDescr_t nvg_G  = nullptr;
  NVG_TRY(nvgraphCreate(&nvg_handle));
  createGraph_nvgraph(nvg_handle, G, &nvg_G, false);

  // Make Nvgraph call
  NVG_TRY(nvgraphTriangleCount(nvg_handle, nvg_G, result));
}

void louvain(
  Graph* graph, void* final_modularity, void* num_level, void* louvain_parts_ptr, int max_iter)
{
  CHECK_GRAPH(graph);

  size_t n = graph->adjList->offsets->size - 1;
  size_t e = graph->adjList->indices->size;

  void* offsets_ptr = graph->adjList->offsets->data;
  void* indices_ptr = graph->adjList->indices->data;

  void* value_ptr;
  rmm::device_vector<float> d_values;
  if (graph->adjList->edge_data) {
    value_ptr = graph->adjList->edge_data->data;
  } else {
    hipStream_t stream{nullptr};
    d_values.resize(graph->adjList->indices->size);
    thrust::fill(rmm::exec_policy(stream)->on(stream), d_values.begin(), d_values.end(), 1.0);
    value_ptr = (void*)thrust::raw_pointer_cast(d_values.data());
  }

  auto gdf_to_cudadtype = [](gdf_column* col) {
    hipDataType cuda_dtype;
    switch (col->dtype) {
      case GDF_INT8: cuda_dtype = HIP_R_8I; break;
      case GDF_INT32: cuda_dtype = HIP_R_32I; break;
      case GDF_FLOAT32: cuda_dtype = HIP_R_32F; break;
      case GDF_FLOAT64: cuda_dtype = HIP_R_64F; break;
      default: throw new std::invalid_argument("Cannot convert data type");
    }
    return cuda_dtype;
  };

  hipDataType index_type = gdf_to_cudadtype(graph->adjList->indices);
  hipDataType val_type =
    graph->adjList->edge_data ? gdf_to_cudadtype(graph->adjList->edge_data) : HIP_R_32F;

  nvgraphLouvain(index_type,
                 val_type,
                 n,
                 e,
                 offsets_ptr,
                 indices_ptr,
                 value_ptr,
                 1,
                 0,
                 NULL,
                 final_modularity,
                 louvain_parts_ptr,
                 num_level,
                 max_iter);
}

}  // namespace cugraph
