// -*-c++-*-

/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
/** ---------------------------------------------------------------------------*
 * @brief Wrapper functions for Nvgraph
 *
 * @file nvgraph_gdf.cu
 * ---------------------------------------------------------------------------**/

#include <cugraph.h>
#include <nvgraph/nvgraph.h>
#include <thrust/device_vector.h>
#include <ctime>
#include "utilities/error_utils.h"
#include "converters/nvgraph.cuh"
#include <rmm_utils.h>

namespace cugraph {

void balancedCutClustering_nvgraph(Graph* cugraph_G,
                                            const int num_clusters,
                                            const int num_eigen_vects,
                                            const float evs_tolerance,
                                            const int evs_max_iter,
                                            const float kmean_tolerance,
                                            const int kmean_max_iter,
                                            gdf_column* clustering) {

  CUGRAPH_EXPECTS(cugraph_G != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(cugraph_G->adjList != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(clustering != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(clustering->data != nullptr, "Invalid API parameter");

  // Initialize Nvgraph and wrap the graph
  nvgraphHandle_t nvg_handle = nullptr;
  nvgraphGraphDescr_t nvgraph_G = nullptr;
        hipDataType settype;
        rmm::device_vector<double> d_val;

  NVG_TRY(nvgraphCreate(&nvg_handle));
  createGraph_nvgraph(nvg_handle, cugraph_G, &nvgraph_G, false);
  int weight_index = 0;

  hipStream_t stream{nullptr};

  if (cugraph_G->adjList->edge_data == nullptr) {
    // use a fp64 vector  [1,...,1]
    settype = HIP_R_64F;
    d_val.resize(cugraph_G->e);
    thrust::fill(rmm::exec_policy(stream)->on(stream), d_val.begin(), d_val.end(), 1.0);
    NVG_TRY(nvgraphAttachEdgeData(nvg_handle,
                                  nvgraph_G,
                                  weight_index,
                                  settype,
                                  (void * ) thrust::raw_pointer_cast(d_val.data())));
  }
  else {
    switch (typeid(cugraph_G->adjList->edge_data)) {
      case GDF_FLOAT32:
        settype = HIP_R_32F;
        break;
      case GDF_FLOAT64:
        settype = HIP_R_64F;
        break;
      default:
        CUGRAPH_FAIL("Unsupported data type");
    }
  }


  // Pack parameters for call to Nvgraph
  SpectralClusteringParameter param;
  param.n_clusters = num_clusters;
  param.n_eig_vects = num_eigen_vects;
  param.algorithm = NVGRAPH_BALANCED_CUT_LANCZOS;
  param.evs_tolerance = evs_tolerance;
  param.evs_max_iter = evs_max_iter;
  param.kmean_tolerance = kmean_tolerance;
  param.kmean_max_iter = kmean_max_iter;

  // Make call to Nvgraph balancedCutClustering
  void* eig_vals = malloc(num_eigen_vects * sizeof(double));
  void* eig_vects = malloc(num_eigen_vects * clustering->size * sizeof(double));
  nvgraphStatus_t err = nvgraphSpectralClustering(nvg_handle,
                                                  nvgraph_G,
                                                  weight_index,
                                                  &param,
                                                  (int*) clustering->data,
                                                  eig_vals,
                                                  eig_vects);
  free(eig_vals);
  free(eig_vects);
  NVG_TRY(err);
  NVG_TRY(nvgraphDestroyGraphDescr(nvg_handle, nvgraph_G));
  NVG_TRY(nvgraphDestroy(nvg_handle));
  
}

void spectralModularityMaximization_nvgraph(Graph* cugraph_G,
                                                      const int n_clusters,
                                                      const int n_eig_vects,
                                                      const float evs_tolerance,
                                                      const int evs_max_iter,
                                                      const float kmean_tolerance,
                                                      const int kmean_max_iter,
                                                      gdf_column* clustering) {

  CUGRAPH_EXPECTS(cugraph_G != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(cugraph_G->adjList != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(clustering != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(clustering->data != nullptr, "Invalid API parameter");

  // Ensure that the input graph has values
  CUGRAPH_EXPECTS(cugraph_G->adjList->edge_data != nullptr, "Invalid API parameter");

  // Initialize Nvgraph and wrap the graph
  nvgraphHandle_t nvg_handle = nullptr;
  nvgraphGraphDescr_t nvgraph_G = nullptr;
  NVG_TRY(nvgraphCreate(&nvg_handle));
  createGraph_nvgraph(nvg_handle, cugraph_G, &nvgraph_G, false);
  int weight_index = 0;

  // Pack parameters for call to Nvgraph
  SpectralClusteringParameter param;
  param.n_clusters = n_clusters;
  param.n_eig_vects = n_eig_vects;
  param.algorithm = NVGRAPH_MODULARITY_MAXIMIZATION;
  param.evs_tolerance = evs_tolerance;
  param.evs_max_iter = evs_max_iter;
  param.kmean_tolerance = kmean_tolerance;
  param.kmean_max_iter = kmean_max_iter;

  // Make call to Nvgraph balancedCutClustering
  void* eig_vals = malloc(n_eig_vects * sizeof(double));
  void* eig_vects = malloc(n_eig_vects * clustering->size * sizeof(double));
  nvgraphStatus_t err = nvgraphSpectralClustering(nvg_handle,
                                                  nvgraph_G,
                                                  weight_index,
                                                  &param,
                                                  (int*) clustering->data,
                                                  eig_vals,
                                                  eig_vects);
  free(eig_vals);
  free(eig_vects);
  NVG_TRY(err);
  NVG_TRY(nvgraphDestroyGraphDescr(nvg_handle, nvgraph_G));
  NVG_TRY(nvgraphDestroy(nvg_handle));
  
}

void analyzeClustering_modularity_nvgraph(Graph* cugraph_G,
                                                    const int n_clusters,
                                                    gdf_column* clustering,
                                                    float* score) {

  CUGRAPH_EXPECTS(cugraph_G != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(cugraph_G->adjList != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(cugraph_G->adjList->edge_data != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(clustering != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(clustering->data != nullptr, "Invalid API parameter");

  // Initialize Nvgraph and wrap the graph
  nvgraphHandle_t nvg_handle = nullptr;
  nvgraphGraphDescr_t nvgraph_G = nullptr;
  NVG_TRY(nvgraphCreate(&nvg_handle));
  createGraph_nvgraph(nvg_handle, cugraph_G, &nvgraph_G, false);
  int weight_index = 0;

  // Make Nvgraph call

  NVG_TRY(nvgraphAnalyzeClustering(nvg_handle,
                                    nvgraph_G,
                                    weight_index,
                                    n_clusters,
                                    (const int* )clustering->data,
                                    NVGRAPH_MODULARITY,
                                    score));
  
}

void analyzeClustering_edge_cut_nvgraph(Graph* cugraph_G,
                                                  const int n_clusters,
                                                  gdf_column* clustering,
                                                  float* score) {

  CUGRAPH_EXPECTS(cugraph_G != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(cugraph_G->adjList != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(clustering != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(clustering->data != nullptr, "Invalid API parameter");

  // Initialize Nvgraph and wrap the graph
  nvgraphHandle_t nvg_handle = nullptr;
  nvgraphGraphDescr_t nvgraph_G = nullptr;
        hipDataType settype;
        rmm::device_vector<double> d_val;

  NVG_TRY(nvgraphCreate(&nvg_handle));
  createGraph_nvgraph(nvg_handle, cugraph_G, &nvgraph_G, false);
  int weight_index = 0;

  hipStream_t stream{nullptr};

  if (cugraph_G->adjList->edge_data == nullptr) {
    // use a fp64 vector  [1,...,1]
    settype = HIP_R_64F;
    d_val.resize(cugraph_G->e);
    thrust::fill(rmm::exec_policy(stream)->on(stream), d_val.begin(), d_val.end(), 1.0);
    NVG_TRY(nvgraphAttachEdgeData(nvg_handle,
                                  nvgraph_G,
                                  weight_index,
                                  settype,
                                  (void * ) thrust::raw_pointer_cast(d_val.data())));
  }
  else {
    switch (typeid(cugraph_G->adjList->edge_data)) {
      case GDF_FLOAT32:
        settype = HIP_R_32F;
        break;
      case GDF_FLOAT64:
        settype = HIP_R_64F;
        break;
      default:
        CUGRAPH_FAIL("Unsupported data type");
      }
  }

  // Make Nvgraph call

  NVG_TRY(nvgraphAnalyzeClustering(nvg_handle,
                                    nvgraph_G,
                                    weight_index,
                                    n_clusters,
                                    (const int* )clustering->data,
                                    NVGRAPH_EDGE_CUT,
                                    score));
  
}

void analyzeClustering_ratio_cut_nvgraph(Graph* cugraph_G,
                                                  const int n_clusters,
                                                  gdf_column* clustering,
                                                  float* score) {

  CUGRAPH_EXPECTS(cugraph_G != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(cugraph_G->adjList != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(cugraph_G->adjList->edge_data != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(clustering != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(clustering->data != nullptr, "Invalid API parameter");

  // Initialize Nvgraph and wrap the graph
  nvgraphHandle_t nvg_handle = nullptr;
  nvgraphGraphDescr_t nvgraph_G = nullptr;
  NVG_TRY(nvgraphCreate(&nvg_handle));
  createGraph_nvgraph(nvg_handle, cugraph_G, &nvgraph_G, false);
  int weight_index = 0;

  // Make Nvgraph call

  NVG_TRY(nvgraphAnalyzeClustering(nvg_handle,
                                    nvgraph_G,
                                    weight_index,
                                    n_clusters,
                                    (const int* )clustering->data,
                                    NVGRAPH_RATIO_CUT,
                                    score));
  
}


void extract_subgraph_vertex_nvgraph(Graph* cugraph_G,
                                              gdf_column* vertices,
                                              Graph* result) {

  CUGRAPH_EXPECTS(cugraph_G != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(cugraph_G->adjList != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(vertices != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(vertices->data != nullptr, "Invalid API parameter");

  // Initialize Nvgraph and wrap the graph
  nvgraphHandle_t nvg_handle = nullptr;
  nvgraphGraphDescr_t nvg_G = nullptr;
  NVG_TRY(nvgraphCreate(&nvg_handle));
  createGraph_nvgraph(nvg_handle, cugraph_G, &nvg_G, false);

  // Create an Nvgraph graph descriptor for the result and initialize
  nvgraphGraphDescr_t nvg_result = nullptr;
  NVG_TRY(nvgraphCreateGraphDescr(nvg_handle, &nvg_result));

  // Call Nvgraph function to get subgraph (into nv_result descriptor)
  NVG_TRY(nvgraphExtractSubgraphByVertex(nvg_handle,
					 nvg_G,
					 nvg_result,
					 (int*)vertices->data,
					 vertices->size));

  // Get the vertices and edges of the created subgraph to allocate memory:
  nvgraphCSRTopology32I_st topo;
  topo.source_offsets = nullptr;
  topo.destination_indices = nullptr;
  nvgraphTopologyType_t TT = NVGRAPH_CSR_32;
  NVG_TRY(nvgraphGetGraphStructure(nvg_handle, nvg_result, (void*)&topo, &TT));
  if (TT != NVGRAPH_CSR_32)
    CUGRAPH_FAIL("Unsupported nvgraph topology");
  int num_verts = topo.nvertices;
  int num_edges = topo.nedges;
  result->adjList = new adj_list;
  result->adjList->offsets = new gdf_column;
  result->adjList->indices = new gdf_column;
  result->adjList->ownership = 0;
  int *offsets, *indices;

  hipStream_t stream { nullptr };

  ALLOC_TRY((void**) &offsets, sizeof(int32_t) * (num_verts + 1), stream);
  ALLOC_TRY((void**) &indices, sizeof(int32_t) * num_edges, stream);

  gdf_column_view(result->adjList->offsets,
                  offsets,
                  nullptr,
                  num_verts + 1,
                  GDF_INT32);
  gdf_column_view(result->adjList->indices,
                  indices,
                  nullptr,
                  num_edges,
                  GDF_INT32);

  // Call nvgraphGetGraphStructure again to copy out the data
  topo.source_offsets = (int*)result->adjList->offsets;
  topo.destination_indices = (int*)result->adjList->indices;
  NVG_TRY(nvgraphGetGraphStructure(nvg_handle, nvg_result, (void*)&topo, &TT));

  
}

void triangle_count_nvgraph(Graph* G, uint64_t* result) {
  
  CUGRAPH_EXPECTS(G != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(G->adjList != nullptr, "Invalid API parameter");

  // Initialize Nvgraph and wrap the graph
  nvgraphHandle_t nvg_handle = nullptr;
  nvgraphGraphDescr_t nvg_G = nullptr;
  NVG_TRY(nvgraphCreate(&nvg_handle));
  createGraph_nvgraph(nvg_handle, G, &nvg_G, false);

  // Make Nvgraph call
  NVG_TRY(nvgraphTriangleCount(nvg_handle, nvg_G, result));
  
}


void louvain(Graph *graph, void *final_modularity, void *num_level, void *louvain_parts_ptr, int max_iter) {

  CUGRAPH_EXPECTS(graph->adjList != nullptr, "Invalid API parameter");

  size_t n = graph->v;
  size_t e = graph->e;

  void* offsets_ptr = graph->adjList->offsets;
  void* indices_ptr = graph->adjList->indices;

  void* value_ptr;
  rmm::device_vector<float> d_values;
  if(graph->adjList->edge_data) {
      value_ptr = graph->adjList->edge_data;
  }
  else {
      hipStream_t stream {nullptr};
      d_values.resize(graph->e);
      thrust::fill(rmm::exec_policy(stream)->on(stream), d_values.begin(), d_values.end(), 1.0);
      value_ptr = (void * ) thrust::raw_pointer_cast(d_values.data());
  }

  void* louvain_parts_ptr = louvain_parts->data;

  hipDataType index_type = HIP_R_32I;
  hipDataType val_type = HIP_R_32F;
  if (graph->adjList->edge_data)
    if (typeid(graph->adjList->edge_data) == HIP_R_32F;

  nvgraphLouvain(index_type, val_type, n, e, offsets_ptr, indices_ptr, value_ptr, 1, 0, NULL,
                 final_modularity, louvain_parts_ptr, num_level, max_iter);
  
}

} //namespace cugraph
