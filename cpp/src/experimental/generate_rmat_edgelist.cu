#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <experimental/scramble.cuh>

#include <experimental/graph_generator.hpp>
#include <utilities/error.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <raft/handle.hpp>
#include <raft/random/rng.cuh>
#include <rmm/device_uvector.hpp>

#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>

#include <tuple>

namespace cugraph {
namespace experimental {

template <typename vertex_t>
std::tuple<rmm::device_uvector<vertex_t>, rmm::device_uvector<vertex_t>> generate_rmat_edgelist(
  raft::handle_t const& handle,
  size_t scale,
  size_t num_edges,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool clip_and_flip,
  bool scramble_vertex_ids)
{
  CUGRAPH_EXPECTS(size_t{1} << scale <= std::numeric_limits<vertex_t>::max(),
                  "Invalid input argument: scale too large for vertex_t.");
  CUGRAPH_EXPECTS((a >= 0.0) && (b >= 0.0) && (c >= 0.0) && (a + b + c <= 1.0),
                  "Invalid input argument: a, b, c should be non-negative and a + b + c should not "
                  "be larger than 1.0.");

  raft::random::Rng rng(seed + 10);
  // to limit memory footprint (1024 is a tuning parameter)
  auto max_edges_to_generate_per_iteration =
    static_cast<size_t>(handle.get_device_properties().multiProcessorCount) * 1024;
  rmm::device_uvector<float> rands(
    std::min(num_edges, max_edges_to_generate_per_iteration) * 2 * scale, handle.get_stream());

  rmm::device_uvector<vertex_t> srcs(num_edges, handle.get_stream());
  rmm::device_uvector<vertex_t> dsts(num_edges, handle.get_stream());

  size_t num_edges_generated{0};
  while (num_edges_generated < num_edges) {
    auto num_edges_to_generate =
      std::min(num_edges - num_edges_generated, max_edges_to_generate_per_iteration);
    auto pair_first = thrust::make_zip_iterator(thrust::make_tuple(srcs.begin(), dsts.begin())) +
                      num_edges_generated;
    rng.uniform<float, size_t>(
      rands.data(), num_edges_to_generate * 2 * scale, 0.0f, 1.0f, handle.get_stream());
    thrust::transform(
      rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
      thrust::make_counting_iterator(size_t{0}),
      thrust::make_counting_iterator(num_edges_to_generate),
      pair_first,
      // if a + b == 0.0, a_norm is irrelevant, if (1.0 - (a+b)) == 0.0, c_norm is irrelevant
      [scale,
       clip_and_flip,
       rands    = rands.data(),
       a_plus_b = a + b,
       a_norm   = (a + b) > 0.0 ? a / (a + b) : 0.0,
       c_norm   = (1.0 - (a + b)) > 0.0 ? c / (1.0 - (a + b)) : 0.0] __device__(auto i) {
        vertex_t src{0};
        vertex_t dst{0};
        for (size_t bit = scale - 1; bit != 0; --bit) {
          auto r0          = rands[i * 2 * scale + 2 * bit];
          auto r1          = rands[i * 2 * scale + 2 * bit + 1];
          auto src_bit_set = r0 > a_plus_b;
          auto dst_bit_set = r1 > (src_bit_set ? c_norm : a_norm);
          if (clip_and_flip) {
            if (src == dst) {
              if (!src_bit_set && dst_bit_set) {
                src_bit_set = !src_bit_set;
                dst_bit_set = !dst_bit_set;
              }
            }
          }
          src += src_bit_set ? static_cast<vertex_t>(1 << bit) : 0;
          dst += dst_bit_set ? static_cast<vertex_t>(1 << bit) : 0;
        }
        return thrust::make_tuple(src, dst);
      });
    num_edges_generated += num_edges_to_generate;
  }

  if (scramble_vertex_ids) {
    rands.resize(0, handle.get_stream());
    rands.shrink_to_fit(handle.get_stream());

    auto pair_first = thrust::make_zip_iterator(thrust::make_tuple(srcs.begin(), dsts.begin()));
    thrust::transform(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                      pair_first,
                      pair_first + srcs.size(),
                      pair_first,
                      [scale] __device__(auto pair) {
                        return thrust::make_tuple(detail::scramble(thrust::get<0>(pair), scale),
                                                  detail::scramble(thrust::get<1>(pair), scale));
                      });
  }

  return std::make_tuple(std::move(srcs), std::move(dsts));
}

// explicit instantiation

template std::tuple<rmm::device_uvector<int32_t>, rmm::device_uvector<int32_t>>
generate_rmat_edgelist<int32_t>(raft::handle_t const& handle,
                                size_t scale,
                                size_t num_edges,
                                double a,
                                double b,
                                double c,
                                uint64_t seed,
                                bool clip_and_flip,
                                bool scramble_vertex_ids);

template std::tuple<rmm::device_uvector<int64_t>, rmm::device_uvector<int64_t>>
generate_rmat_edgelist<int64_t>(raft::handle_t const& handle,
                                size_t scale,
                                size_t num_edges,
                                double a,
                                double b,
                                double c,
                                uint64_t seed,
                                bool clip_and_flip,
                                bool scramble_vertex_ids);

}  // namespace experimental
}  // namespace cugraph
