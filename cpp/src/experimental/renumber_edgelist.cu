#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cugraph/experimental/detail/graph_utils.cuh>
#include <cugraph/experimental/graph_functions.hpp>
#include <cugraph/experimental/graph_view.hpp>
#include <cugraph/utilities/device_comm.cuh>
#include <cugraph/utilities/error.hpp>
#include <cugraph/utilities/host_barrier.hpp>
#include <cugraph/utilities/host_scalar_comm.cuh>
#include <cugraph/utilities/shuffle_comm.cuh>

#include <cuco/static_map.cuh>
#include <raft/handle.hpp>
#include <rmm/device_uvector.hpp>
#include <rmm/exec_policy.hpp>
#include <rmm/mr/device/per_device_resource.hpp>
#include <rmm/mr/device/polymorphic_allocator.hpp>

#include <thrust/binary_search.h>
#include <thrust/copy.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/sort.h>
#include <thrust/tuple.h>

#include <algorithm>
#include <iterator>
#include <numeric>
#include <tuple>
#include <utility>

namespace cugraph {
namespace experimental {
namespace detail {

template <typename vertex_t, typename edge_t, bool multi_gpu>
rmm::device_uvector<vertex_t> compute_renumber_map(
  raft::handle_t const& handle,
  std::optional<std::tuple<vertex_t const*, vertex_t>> optional_vertex_span,
  std::vector<vertex_t const*> const& edgelist_major_vertices,
  std::vector<vertex_t const*> const& edgelist_minor_vertices,
  std::vector<edge_t> const& edgelist_edge_counts)
{
  // FIXME: compare this sort based approach with hash based approach in both speed and memory
  // footprint

  // 1. acquire (unique major label, count) pairs

  if (multi_gpu) {
    auto& comm = handle.get_comms();

    // barrier is necessary here to avoid potential overlap (which can leads to deadlock) between
    // two different communicators (beginning of col_comm)
#if 1
    // FIXME: temporary hack till UCC is integrated into RAFT (so we can use UCC barrier with DASK
    // and MPI barrier with MPI)
    host_barrier(comm, handle.get_stream_view());
#else
    handle.get_stream_view().synchronize();
    ;
    comm.barrier();  // currently, this is ncclAllReduce
#endif
  }

  rmm::device_uvector<vertex_t> major_labels(0, handle.get_stream());
  rmm::device_uvector<edge_t> major_counts(0, handle.get_stream());
  for (size_t i = 0; i < edgelist_major_vertices.size(); ++i) {
    rmm::device_uvector<vertex_t> tmp_major_labels(0, handle.get_stream());
    rmm::device_uvector<edge_t> tmp_major_counts(0, handle.get_stream());
    {
      rmm::device_uvector<vertex_t> sorted_major_labels(edgelist_edge_counts[i],
                                                        handle.get_stream());
      thrust::copy(rmm::exec_policy(handle.get_stream_view()),
                   edgelist_major_vertices[i],
                   edgelist_major_vertices[i] + edgelist_edge_counts[i],
                   sorted_major_labels.begin());
      // FIXME: better refactor this sort-count_if-reduce_by_key routine for reuse
      thrust::sort(rmm::exec_policy(handle.get_stream_view()),
                   sorted_major_labels.begin(),
                   sorted_major_labels.end());
      auto num_unique_labels =
        thrust::count_if(rmm::exec_policy(handle.get_stream_view()),
                         thrust::make_counting_iterator(size_t{0}),
                         thrust::make_counting_iterator(sorted_major_labels.size()),
                         [labels = sorted_major_labels.data()] __device__(auto i) {
                           return (i == 0) || (labels[i - 1] != labels[i]);
                         });
      tmp_major_labels.resize(num_unique_labels, handle.get_stream());
      tmp_major_counts.resize(tmp_major_labels.size(), handle.get_stream());
      thrust::reduce_by_key(rmm::exec_policy(handle.get_stream_view()),
                            sorted_major_labels.begin(),
                            sorted_major_labels.end(),
                            thrust::make_constant_iterator(edge_t{1}),
                            tmp_major_labels.begin(),
                            tmp_major_counts.begin());
    }

    if (multi_gpu) {
      auto& col_comm = handle.get_subcomm(cugraph::partition_2d::key_naming_t().col_name());
      auto const col_comm_rank = col_comm.get_rank();
      auto const col_comm_size = col_comm.get_size();

      rmm::device_uvector<vertex_t> rx_major_labels(0, handle.get_stream());
      rmm::device_uvector<edge_t> rx_major_counts(0, handle.get_stream());
      auto rx_sizes = host_scalar_gather(
        col_comm, tmp_major_labels.size(), static_cast<int>(i), handle.get_stream());
      std::vector<size_t> rx_displs{};
      if (static_cast<int>(i) == col_comm_rank) {
        rx_displs.assign(col_comm_size, size_t{0});
        std::partial_sum(rx_sizes.begin(), rx_sizes.end() - 1, rx_displs.begin() + 1);
        rx_major_labels.resize(rx_displs.back() + rx_sizes.back(), handle.get_stream());
        rx_major_counts.resize(rx_major_labels.size(), handle.get_stream());
      }
      device_gatherv(col_comm,
                     thrust::make_zip_iterator(
                       thrust::make_tuple(tmp_major_labels.begin(), tmp_major_counts.begin())),
                     thrust::make_zip_iterator(
                       thrust::make_tuple(rx_major_labels.begin(), rx_major_counts.begin())),
                     tmp_major_labels.size(),
                     rx_sizes,
                     rx_displs,
                     static_cast<int>(i),
                     handle.get_stream());
      if (static_cast<int>(i) == col_comm_rank) {
        major_labels = std::move(rx_major_labels);
        major_counts = std::move(rx_major_counts);
      }
    } else {
      assert(i == 0);
      major_labels = std::move(tmp_major_labels);
      major_counts = std::move(tmp_major_counts);
    }
  }
  if (multi_gpu) {
    // FIXME: better refactor this sort-count_if-reduce_by_key routine for reuse
    thrust::sort_by_key(rmm::exec_policy(handle.get_stream_view()),
                        major_labels.begin(),
                        major_labels.end(),
                        major_counts.begin());
    auto num_unique_labels = thrust::count_if(rmm::exec_policy(handle.get_stream_view()),
                                              thrust::make_counting_iterator(size_t{0}),
                                              thrust::make_counting_iterator(major_labels.size()),
                                              [labels = major_labels.data()] __device__(auto i) {
                                                return (i == 0) || (labels[i - 1] != labels[i]);
                                              });
    rmm::device_uvector<vertex_t> tmp_major_labels(num_unique_labels, handle.get_stream());
    rmm::device_uvector<edge_t> tmp_major_counts(tmp_major_labels.size(), handle.get_stream());
    thrust::reduce_by_key(rmm::exec_policy(handle.get_stream_view()),
                          major_labels.begin(),
                          major_labels.end(),
                          major_counts.begin(),
                          tmp_major_labels.begin(),
                          tmp_major_counts.begin());
    major_labels = std::move(tmp_major_labels);
    major_counts = std::move(tmp_major_counts);
  }

  // 2. acquire unique minor labels

  std::vector<edge_t> minor_displs(edgelist_minor_vertices.size(), edge_t{0});
  std::partial_sum(
    edgelist_edge_counts.begin(), edgelist_edge_counts.end() - 1, minor_displs.begin() + 1);
  rmm::device_uvector<vertex_t> minor_labels(minor_displs.back() + edgelist_edge_counts.back(),
                                             handle.get_stream());
  for (size_t i = 0; i < edgelist_minor_vertices.size(); ++i) {
    thrust::copy(rmm::exec_policy(handle.get_stream_view()),
                 edgelist_minor_vertices[i],
                 edgelist_minor_vertices[i] + edgelist_edge_counts[i],
                 minor_labels.begin() + minor_displs[i]);
  }
  thrust::sort(
    rmm::exec_policy(handle.get_stream_view()), minor_labels.begin(), minor_labels.end());
  minor_labels.resize(thrust::distance(minor_labels.begin(),
                                       thrust::unique(rmm::exec_policy(handle.get_stream_view()),
                                                      minor_labels.begin(),
                                                      minor_labels.end())),
                      handle.get_stream());
  if (multi_gpu) {
    auto& comm               = handle.get_comms();
    auto& row_comm           = handle.get_subcomm(cugraph::partition_2d::key_naming_t().row_name());
    auto const row_comm_size = row_comm.get_size();

    // barrier is necessary here to avoid potential overlap (which can leads to deadlock) between
    // two different communicators (beginning of row_comm)
#if 1
    // FIXME: temporary hack till UCC is integrated into RAFT (so we can use UCC barrier with DASK
    // and MPI barrier with MPI)
    host_barrier(comm, handle.get_stream_view());
#else
    handle.get_stream_view().synchronize();
    comm.barrier();  // currently, this is ncclAllReduce
#endif

    if (row_comm_size > 1) {
      rmm::device_uvector<vertex_t> rx_minor_labels(0, handle.get_stream());
      std::tie(rx_minor_labels, std::ignore) = groupby_gpuid_and_shuffle_values(
        row_comm,
        minor_labels.begin(),
        minor_labels.end(),
        [key_func = detail::compute_gpu_id_from_vertex_t<vertex_t>{row_comm_size}] __device__(
          auto val) { return key_func(val); },
        handle.get_stream());
      thrust::sort(
        rmm::exec_policy(handle.get_stream_view()), rx_minor_labels.begin(), rx_minor_labels.end());
      rx_minor_labels.resize(
        thrust::distance(rx_minor_labels.begin(),
                         thrust::unique(rmm::exec_policy(handle.get_stream_view()),
                                        rx_minor_labels.begin(),
                                        rx_minor_labels.end())),
        handle.get_stream());
      minor_labels = std::move(rx_minor_labels);
    }

    // barrier is necessary here to avoid potential overlap (which can leads to deadlock) between
    // two different communicators (end of row_comm)
#if 1
    // FIXME: temporary hack till UCC is integrated into RAFT (so we can use UCC barrier with DASK
    // and MPI barrier with MPI)
    //
    host_barrier(comm, handle.get_stream_view());
#else
    handle.get_stream_view().synchronize();
    comm.barrier();  // currently, this is ncclAllReduce
#endif
  }
  minor_labels.shrink_to_fit(handle.get_stream_view());

  // 3. merge major and minor labels and vertex labels

  rmm::device_uvector<vertex_t> merged_labels(major_labels.size() + minor_labels.size(),
                                              handle.get_stream_view());
  rmm::device_uvector<edge_t> merged_counts(merged_labels.size(), handle.get_stream_view());
  thrust::merge_by_key(rmm::exec_policy(handle.get_stream_view()),
                       major_labels.begin(),
                       major_labels.end(),
                       minor_labels.begin(),
                       minor_labels.end(),
                       major_counts.begin(),
                       thrust::make_constant_iterator(edge_t{0}),
                       merged_labels.begin(),
                       merged_counts.begin());

  major_labels.resize(0, handle.get_stream());
  major_counts.resize(0, handle.get_stream());
  minor_labels.resize(0, handle.get_stream());
  major_labels.shrink_to_fit(handle.get_stream());
  major_counts.shrink_to_fit(handle.get_stream());
  minor_labels.shrink_to_fit(handle.get_stream());

  rmm::device_uvector<vertex_t> labels(merged_labels.size(), handle.get_stream());
  rmm::device_uvector<edge_t> counts(labels.size(), handle.get_stream());
  auto pair_it = thrust::reduce_by_key(rmm::exec_policy(handle.get_stream_view()),
                                       merged_labels.begin(),
                                       merged_labels.end(),
                                       merged_counts.begin(),
                                       labels.begin(),
                                       counts.begin());
  merged_labels.resize(0, handle.get_stream());
  merged_counts.resize(0, handle.get_stream());
  merged_labels.shrink_to_fit(handle.get_stream());
  merged_counts.shrink_to_fit(handle.get_stream());
  labels.resize(thrust::distance(labels.begin(), thrust::get<0>(pair_it)), handle.get_stream());
  counts.resize(labels.size(), handle.get_stream());
  labels.shrink_to_fit(handle.get_stream());
  counts.shrink_to_fit(handle.get_stream());

  // 4. if vertices != nullptr, add isolated vertices

  rmm::device_uvector<vertex_t> isolated_vertices(0, handle.get_stream());
  if (optional_vertex_span) {
    auto [vertices, num_vertices] = *optional_vertex_span;
    auto num_isolated_vertices    = thrust::count_if(
      rmm::exec_policy(handle.get_stream_view()),
      vertices,
      vertices + num_vertices,
      [label_first = labels.begin(), label_last = labels.end()] __device__(auto v) {
        return !thrust::binary_search(thrust::seq, label_first, label_last, v);
      });
    isolated_vertices.resize(num_isolated_vertices, handle.get_stream());
    thrust::copy_if(rmm::exec_policy(handle.get_stream_view()),
                    vertices,
                    vertices + num_vertices,
                    isolated_vertices.begin(),
                    [label_first = labels.begin(), label_last = labels.end()] __device__(auto v) {
                      return !thrust::binary_search(thrust::seq, label_first, label_last, v);
                    });
  }

  if (isolated_vertices.size() > 0) {
    labels.resize(labels.size() + isolated_vertices.size(), handle.get_stream());
    counts.resize(labels.size(), handle.get_stream());
    thrust::copy(rmm::exec_policy(handle.get_stream_view()),
                 isolated_vertices.begin(),
                 isolated_vertices.end(),
                 labels.end() - isolated_vertices.size());
    thrust::fill(rmm::exec_policy(handle.get_stream_view()),
                 counts.end() - isolated_vertices.size(),
                 counts.end(),
                 edge_t{0});
  }

  // 6. sort by degree

  thrust::sort_by_key(rmm::exec_policy(handle.get_stream_view()),
                      counts.begin(),
                      counts.end(),
                      labels.begin(),
                      thrust::greater<edge_t>());

  return labels;
}

template <typename vertex_t, typename edge_t, bool multi_gpu>
void expensive_check_edgelist(
  raft::handle_t const& handle,
  std::optional<std::tuple<vertex_t const*, vertex_t>> optional_vertex_span,
  std::vector<vertex_t const*> const& edgelist_major_vertices,
  std::vector<vertex_t const*> const& edgelist_minor_vertices,
  std::vector<edge_t> const& edgelist_edge_counts)
{
  rmm::device_uvector<vertex_t> sorted_local_vertices(size_t{0}, handle.get_stream());
  if (optional_vertex_span) {
    auto [vertices, num_vertices] = *optional_vertex_span;
    sorted_local_vertices.resize(num_vertices, handle.get_stream());
    thrust::copy(rmm::exec_policy(handle.get_stream_view()),
                 vertices,
                 vertices + num_vertices,
                 sorted_local_vertices.begin());
    thrust::sort(rmm::exec_policy(handle.get_stream_view()),
                 sorted_local_vertices.begin(),
                 sorted_local_vertices.end());
    CUGRAPH_EXPECTS(static_cast<size_t>(thrust::distance(
                      sorted_local_vertices.begin(),
                      thrust::unique(rmm::exec_policy(handle.get_stream_view()),
                                     sorted_local_vertices.begin(),
                                     sorted_local_vertices.end()))) == sorted_local_vertices.size(),
                    "Invalid input argument: local_vertices should not have duplicates.");
  }

  if (multi_gpu) {
    auto& comm               = handle.get_comms();
    auto const comm_size     = comm.get_size();
    auto const comm_rank     = comm.get_rank();
    auto& row_comm           = handle.get_subcomm(cugraph::partition_2d::key_naming_t().row_name());
    auto const row_comm_size = row_comm.get_size();
    auto const row_comm_rank = row_comm.get_rank();
    auto& col_comm           = handle.get_subcomm(cugraph::partition_2d::key_naming_t().col_name());
    auto const col_comm_size = col_comm.get_size();
    auto const col_comm_rank = col_comm.get_rank();

    CUGRAPH_EXPECTS((edgelist_major_vertices.size() == edgelist_minor_vertices.size()) &&
                      (edgelist_major_vertices.size() == static_cast<size_t>(col_comm_size)),
                    "Invalid input argument: both edgelist_major_vertices.size() & "
                    "edgelist_minor_vertices.size() should coincide with col_comm_size.");

    auto [local_vertices, num_local_vertices] = *optional_vertex_span;
    CUGRAPH_EXPECTS(
      thrust::count_if(
        rmm::exec_policy(handle.get_stream_view()),
        local_vertices,
        local_vertices + num_local_vertices,
        [comm_rank,
         key_func =
           detail::compute_gpu_id_from_vertex_t<vertex_t>{comm_size}] __device__(auto val) {
          return key_func(val) != comm_rank;
        }) == 0,
      "Invalid input argument: local_vertices should be pre-shuffled.");

    for (size_t i = 0; i < edgelist_major_vertices.size(); ++i) {
      auto edge_first = thrust::make_zip_iterator(
        thrust::make_tuple(edgelist_major_vertices[i], edgelist_minor_vertices[i]));
      CUGRAPH_EXPECTS(
        thrust::count_if(
          rmm::exec_policy(handle.get_stream_view()),
          edge_first,
          edge_first + edgelist_edge_counts[i],
          [comm_size,
           comm_rank,
           row_comm_rank,
           col_comm_size,
           col_comm_rank,
           i,
           gpu_id_key_func =
             detail::compute_gpu_id_from_edge_t<vertex_t>{comm_size, row_comm_size, col_comm_size},
           partition_id_key_func =
             detail::compute_partition_id_from_edge_t<vertex_t>{
               comm_size, row_comm_size, col_comm_size}] __device__(auto edge) {
            return (gpu_id_key_func(thrust::get<0>(edge), thrust::get<1>(edge)) != comm_rank) ||
                   (partition_id_key_func(thrust::get<0>(edge), thrust::get<1>(edge)) !=
                    row_comm_rank * col_comm_size + col_comm_rank + i * comm_size);
          }) == 0,
        "Invalid input argument: edgelist_major_vertices & edgelist_minor_vertices should be "
        "pre-shuffled.");

      if (optional_vertex_span) {
        auto& row_comm = handle.get_subcomm(cugraph::partition_2d::key_naming_t().row_name());
        auto& col_comm = handle.get_subcomm(cugraph::partition_2d::key_naming_t().col_name());

        // FIXME: this barrier is unnecessary if the above host_scalar_allreduce is a true host
        // operation (as it serves as a barrier) barrier is necessary here to avoid potential
        // overlap (which can leads to deadlock) between two different communicators (beginning of
        // col_comm)
#if 1
        // FIXME: temporary hack till UCC is integrated into RAFT (so we can use UCC barrier with
        // DASK and MPI barrier with MPI)
        host_barrier(comm, handle.get_stream_view());
#else
        handle.get_stream_view().synchronize();
        comm.barrier();  // currently, this is ncclAllReduce
#endif

        rmm::device_uvector<vertex_t> sorted_major_vertices(0, handle.get_stream());
        {
          auto recvcounts =
            host_scalar_allgather(col_comm, sorted_local_vertices.size(), handle.get_stream());
          std::vector<size_t> displacements(recvcounts.size(), size_t{0});
          std::partial_sum(recvcounts.begin(), recvcounts.end() - 1, displacements.begin() + 1);
          sorted_major_vertices.resize(displacements.back() + recvcounts.back(),
                                       handle.get_stream());
          device_allgatherv(col_comm,
                            sorted_local_vertices.data(),
                            sorted_major_vertices.data(),
                            recvcounts,
                            displacements,
                            handle.get_stream());
          thrust::sort(rmm::exec_policy(handle.get_stream_view()),
                       sorted_major_vertices.begin(),
                       sorted_major_vertices.end());
        }

        // barrier is necessary here to avoid potential overlap (which can leads to deadlock)
        // between two different communicators (beginning of row_comm)
#if 1
        // FIXME: temporary hack till UCC is integrated into RAFT (so we can use UCC barrier with
        // DASK and MPI barrier with MPI)
        host_barrier(comm, handle.get_stream_view());
#else
        handle.get_stream_view().synchronize();
        comm.barrier();  // currently, this is ncclAllReduce
#endif

        rmm::device_uvector<vertex_t> sorted_minor_vertices(0, handle.get_stream());
        {
          auto recvcounts =
            host_scalar_allgather(row_comm, sorted_local_vertices.size(), handle.get_stream());
          std::vector<size_t> displacements(recvcounts.size(), size_t{0});
          std::partial_sum(recvcounts.begin(), recvcounts.end() - 1, displacements.begin() + 1);
          sorted_minor_vertices.resize(displacements.back() + recvcounts.back(),
                                       handle.get_stream());
          device_allgatherv(row_comm,
                            sorted_local_vertices.data(),
                            sorted_minor_vertices.data(),
                            recvcounts,
                            displacements,
                            handle.get_stream());
          thrust::sort(rmm::exec_policy(handle.get_stream_view()),
                       sorted_minor_vertices.begin(),
                       sorted_minor_vertices.end());
        }

        // barrier is necessary here to avoid potential overlap (which can leads to deadlock)
        // between two different communicators (end of row_comm)
#if 1
        // FIXME: temporary hack till UCC is integrated into RAFT (so we can use UCC barrier with
        // DASK and MPI barrier with MPI)
        host_barrier(comm, handle.get_stream_view());
#else
        handle.get_stream_view().synchronize();
        comm.barrier();  // currently, this is ncclAllReduce
#endif

        auto edge_first = thrust::make_zip_iterator(
          thrust::make_tuple(edgelist_major_vertices[i], edgelist_minor_vertices[i]));
        CUGRAPH_EXPECTS(
          thrust::count_if(
            rmm::exec_policy(handle.get_stream_view()),
            edge_first,
            edge_first + edgelist_edge_counts[i],
            [num_major_vertices    = static_cast<vertex_t>(sorted_major_vertices.size()),
             sorted_major_vertices = sorted_major_vertices.data(),
             num_minor_vertices    = static_cast<vertex_t>(sorted_minor_vertices.size()),
             sorted_minor_vertices = sorted_minor_vertices.data()] __device__(auto e) {
              return !thrust::binary_search(thrust::seq,
                                            sorted_major_vertices,
                                            sorted_major_vertices + num_major_vertices,
                                            thrust::get<0>(e)) ||
                     !thrust::binary_search(thrust::seq,
                                            sorted_minor_vertices,
                                            sorted_minor_vertices + num_minor_vertices,
                                            thrust::get<1>(e));
            }) == 0,
          "Invalid input argument: edgelist_major_vertices and/or edgelist_mior_vertices have "
          "invalid vertex ID(s).");
      }
    }
  } else {
    assert(edgelist_major_vertices.size() == 1);
    assert(edgelist_minor_vertices.size() == 1);

    if (optional_vertex_span) {
      auto edge_first = thrust::make_zip_iterator(
        thrust::make_tuple(edgelist_major_vertices[0], edgelist_minor_vertices[0]));
      CUGRAPH_EXPECTS(
        thrust::count_if(
          rmm::exec_policy(handle.get_stream_view()),
          edge_first,
          edge_first + edgelist_edge_counts[0],
          [sorted_local_vertices = sorted_local_vertices.data(),
           num_sorted_local_vertices =
             static_cast<vertex_t>(sorted_local_vertices.size())] __device__(auto e) {
            return !thrust::binary_search(thrust::seq,
                                          sorted_local_vertices,
                                          sorted_local_vertices + num_sorted_local_vertices,
                                          thrust::get<0>(e)) ||
                   !thrust::binary_search(thrust::seq,
                                          sorted_local_vertices,
                                          sorted_local_vertices + num_sorted_local_vertices,
                                          thrust::get<1>(e));
          }) == 0,
        "Invalid input argument: edgelist_major_vertices and/or edgelist_minor_vertices have "
        "invalid vertex ID(s).");
    }
  }
}

template <typename vertex_t, typename edge_t, bool multi_gpu>
std::enable_if_t<multi_gpu,
                 std::tuple<rmm::device_uvector<vertex_t>, partition_t<vertex_t>, vertex_t, edge_t>>
renumber_edgelist(raft::handle_t const& handle,
                  std::optional<std::tuple<vertex_t const*, vertex_t>> optional_local_vertex_span,
                  std::vector<vertex_t*> const& edgelist_major_vertices /* [INOUT] */,
                  std::vector<vertex_t*> const& edgelist_minor_vertices /* [INOUT] */,
                  std::vector<edge_t> const& edgelist_edge_counts,
                  bool do_expensive_check)
{
  auto& comm               = handle.get_comms();
  auto const comm_size     = comm.get_size();
  auto const comm_rank     = comm.get_rank();
  auto& row_comm           = handle.get_subcomm(cugraph::partition_2d::key_naming_t().row_name());
  auto const row_comm_size = row_comm.get_size();
  auto const row_comm_rank = row_comm.get_rank();
  auto& col_comm           = handle.get_subcomm(cugraph::partition_2d::key_naming_t().col_name());
  auto const col_comm_size = col_comm.get_size();
  auto const col_comm_rank = col_comm.get_rank();

  std::vector<vertex_t const*> edgelist_const_major_vertices(edgelist_major_vertices.size());
  std::vector<vertex_t const*> edgelist_const_minor_vertices(edgelist_const_major_vertices.size());
  for (size_t i = 0; i < edgelist_const_major_vertices.size(); ++i) {
    edgelist_const_major_vertices[i] = edgelist_major_vertices[i];
    edgelist_const_minor_vertices[i] = edgelist_minor_vertices[i];
  }

  if (do_expensive_check) {
    expensive_check_edgelist<vertex_t, edge_t, multi_gpu>(handle,
                                                          optional_local_vertex_span,
                                                          edgelist_const_major_vertices,
                                                          edgelist_const_minor_vertices,
                                                          edgelist_edge_counts);
  }

  // 1. compute renumber map

  auto renumber_map_labels =
    detail::compute_renumber_map<vertex_t, edge_t, multi_gpu>(handle,
                                                              optional_local_vertex_span,
                                                              edgelist_const_major_vertices,
                                                              edgelist_const_minor_vertices,
                                                              edgelist_edge_counts);
  // 2. initialize partition_t object, number_of_vertices, and number_of_edges for the coarsened
  // graph

  auto vertex_counts = host_scalar_allgather(
    comm, static_cast<vertex_t>(renumber_map_labels.size()), handle.get_stream());
  std::vector<vertex_t> vertex_partition_offsets(comm_size + 1, 0);
  std::partial_sum(
    vertex_counts.begin(), vertex_counts.end(), vertex_partition_offsets.begin() + 1);

  partition_t<vertex_t> partition(
    vertex_partition_offsets, row_comm_size, col_comm_size, row_comm_rank, col_comm_rank);

  auto number_of_vertices = vertex_partition_offsets.back();
  auto number_of_edges    = host_scalar_allreduce(
    comm,
    std::accumulate(edgelist_edge_counts.begin(), edgelist_edge_counts.end(), edge_t{0}),
    handle.get_stream());

  // 3. renumber edges

  double constexpr load_factor = 0.7;

  // FIXME: compare this hash based approach with a binary search based approach in both memory
  // footprint and execution time

  // FIXME: this barrier is unnecessary if the above host_scalar_allgather is a true host operation
  // (as it serves as a barrier) barrier is necessary here to avoid potential overlap (which can
  // leads to deadlock) between two different communicators (beginning of col_comm)
#if 1
  // FIXME: temporary hack till UCC is integrated into RAFT (so we can use UCC barrier with DASK and
  // MPI barrier with MPI)
  host_barrier(comm, handle.get_stream_view());
#else
  handle.get_stream_view().synchronize();
  comm.barrier();  // currently, this is ncclAllReduce
#endif

  for (size_t i = 0; i < edgelist_major_vertices.size(); ++i) {
    rmm::device_uvector<vertex_t> renumber_map_major_labels(
      col_comm_rank == static_cast<int>(i) ? vertex_t{0}
                                           : partition.get_matrix_partition_major_size(i),
      handle.get_stream());
    device_bcast(col_comm,
                 renumber_map_labels.data(),
                 renumber_map_major_labels.data(),
                 partition.get_matrix_partition_major_size(i),
                 i,
                 handle.get_stream());

    CUDA_TRY(hipStreamSynchronize(
      handle.get_stream()));  // cuco::static_map currently does not take stream

    auto poly_alloc = rmm::mr::polymorphic_allocator<char>(rmm::mr::get_current_device_resource());
    auto stream_adapter = rmm::mr::make_stream_allocator_adaptor(poly_alloc, hipStream_t{nullptr});
    cuco::static_map<vertex_t, vertex_t, cuda::thread_scope_device, decltype(stream_adapter)>
      renumber_map{
        // cuco::static_map requires at least one empty slot
        std::max(static_cast<size_t>(
                   static_cast<double>(partition.get_matrix_partition_major_size(i)) / load_factor),
                 static_cast<size_t>(partition.get_matrix_partition_major_size(i)) + 1),
        invalid_vertex_id<vertex_t>::value,
        invalid_vertex_id<vertex_t>::value,
        stream_adapter};
    auto pair_first = thrust::make_zip_iterator(thrust::make_tuple(
      col_comm_rank == static_cast<int>(i) ? renumber_map_labels.begin()
                                           : renumber_map_major_labels.begin(),
      thrust::make_counting_iterator(partition.get_matrix_partition_major_first(i))));
    renumber_map.insert(pair_first, pair_first + partition.get_matrix_partition_major_size(i));
    renumber_map.find(edgelist_major_vertices[i],
                      edgelist_major_vertices[i] + edgelist_edge_counts[i],
                      edgelist_major_vertices[i]);
  }

  // barrier is necessary here to avoid potential overlap (which can leads to deadlock) between two
  // different communicators (beginning of row_comm)
#if 1
  // FIXME: temporary hack till UCC is integrated into RAFT (so we can use UCC barrier with DASK and
  // MPI barrier with MPI)
  host_barrier(comm, handle.get_stream_view());
#else
  handle.get_stream_view().synchronize();
  comm.barrier();  // currently, this is ncclAllReduce
#endif
  {
    rmm::device_uvector<vertex_t> renumber_map_minor_labels(
      partition.get_matrix_partition_minor_size(), handle.get_stream());
    std::vector<size_t> recvcounts(row_comm_size);
    for (int i = 0; i < row_comm_size; ++i) {
      recvcounts[i] = partition.get_vertex_partition_size(col_comm_rank * row_comm_size + i);
    }
    std::vector<size_t> displacements(recvcounts.size(), 0);
    std::partial_sum(recvcounts.begin(), recvcounts.end() - 1, displacements.begin() + 1);
    device_allgatherv(row_comm,
                      renumber_map_labels.begin(),
                      renumber_map_minor_labels.begin(),
                      recvcounts,
                      displacements,
                      handle.get_stream());

    CUDA_TRY(hipStreamSynchronize(
      handle.get_stream()));  // cuco::static_map currently does not take stream

    auto poly_alloc = rmm::mr::polymorphic_allocator<char>(rmm::mr::get_current_device_resource());
    auto stream_adapter = rmm::mr::make_stream_allocator_adaptor(poly_alloc, hipStream_t{nullptr});
    cuco::static_map<vertex_t, vertex_t, cuda::thread_scope_device, decltype(stream_adapter)>
      renumber_map{// cuco::static_map requires at least one empty slot
                   std::max(static_cast<size_t>(
                              static_cast<double>(renumber_map_minor_labels.size()) / load_factor),
                            renumber_map_minor_labels.size() + 1),
                   invalid_vertex_id<vertex_t>::value,
                   invalid_vertex_id<vertex_t>::value,
                   stream_adapter};
    auto pair_first = thrust::make_zip_iterator(thrust::make_tuple(
      renumber_map_minor_labels.begin(),
      thrust::make_counting_iterator(partition.get_matrix_partition_minor_first())));
    renumber_map.insert(pair_first, pair_first + renumber_map_minor_labels.size());
    for (size_t i = 0; i < edgelist_major_vertices.size(); ++i) {
      renumber_map.find(edgelist_minor_vertices[i],
                        edgelist_minor_vertices[i] + edgelist_edge_counts[i],
                        edgelist_minor_vertices[i]);
    }
  }
  // barrier is necessary here to avoid potential overlap (which can leads to deadlock) between two
  // different communicators (end of row_comm)
#if 1
  // FIXME: temporary hack till UCC is integrated into RAFT (so we can use UCC barrier with DASK and
  // MPI barrier with MPI)
  host_barrier(comm, handle.get_stream_view());
#else
  handle.get_stream_view().synchronize();
  comm.barrier();  // currently, this is ncclAllReduce
#endif

  return std::make_tuple(
    std::move(renumber_map_labels), partition, number_of_vertices, number_of_edges);
}

template <typename vertex_t, typename edge_t, bool multi_gpu>
std::enable_if_t<!multi_gpu, rmm::device_uvector<vertex_t>> renumber_edgelist(
  raft::handle_t const& handle,
  std::optional<std::tuple<vertex_t const*, vertex_t>> optional_vertex_span,
  vertex_t* edgelist_major_vertices /* [INOUT] */,
  vertex_t* edgelist_minor_vertices /* [INOUT] */,
  edge_t num_edgelist_edges,
  bool do_expensive_check)
{
  if (do_expensive_check) {
    expensive_check_edgelist<vertex_t, edge_t, multi_gpu>(
      handle,
      optional_vertex_span,
      std::vector<vertex_t const*>{edgelist_major_vertices},
      std::vector<vertex_t const*>{edgelist_minor_vertices},
      std::vector<edge_t>{num_edgelist_edges});
  }

  auto renumber_map_labels = detail::compute_renumber_map<vertex_t, edge_t, multi_gpu>(
    handle,
    optional_vertex_span,
    std::vector<vertex_t const*>{edgelist_major_vertices},
    std::vector<vertex_t const*>{edgelist_minor_vertices},
    std::vector<edge_t>{num_edgelist_edges});

  double constexpr load_factor = 0.7;

  // FIXME: compare this hash based approach with a binary search based approach in both memory
  // footprint and execution time

  auto poly_alloc = rmm::mr::polymorphic_allocator<char>(rmm::mr::get_current_device_resource());
  auto stream_adapter = rmm::mr::make_stream_allocator_adaptor(poly_alloc, hipStream_t{nullptr});
  cuco::static_map<vertex_t, vertex_t, cuda::thread_scope_device, decltype(stream_adapter)>
    renumber_map{
      // cuco::static_map requires at least one empty slot
      std::max(static_cast<size_t>(static_cast<double>(renumber_map_labels.size()) / load_factor),
               renumber_map_labels.size() + 1),
      invalid_vertex_id<vertex_t>::value,
      invalid_vertex_id<vertex_t>::value,
      stream_adapter};
  auto pair_first = thrust::make_zip_iterator(
    thrust::make_tuple(renumber_map_labels.begin(), thrust::make_counting_iterator(vertex_t{0})));
  renumber_map.insert(pair_first, pair_first + renumber_map_labels.size());
  renumber_map.find(
    edgelist_major_vertices, edgelist_major_vertices + num_edgelist_edges, edgelist_major_vertices);
  renumber_map.find(
    edgelist_minor_vertices, edgelist_minor_vertices + num_edgelist_edges, edgelist_minor_vertices);

  return renumber_map_labels;
}

}  // namespace detail

template <typename vertex_t, typename edge_t, bool multi_gpu>
std::enable_if_t<multi_gpu,
                 std::tuple<rmm::device_uvector<vertex_t>, partition_t<vertex_t>, vertex_t, edge_t>>
renumber_edgelist(raft::handle_t const& handle,
                  std::optional<std::tuple<vertex_t const*, vertex_t>> optional_local_vertex_span,
                  std::vector<vertex_t*> const& edgelist_major_vertices /* [INOUT] */,
                  std::vector<vertex_t*> const& edgelist_minor_vertices /* [INOUT] */,
                  std::vector<edge_t> const& edgelist_edge_counts,
                  bool do_expensive_check)
{
  return detail::renumber_edgelist<vertex_t, edge_t, multi_gpu>(handle,
                                                                optional_local_vertex_span,
                                                                edgelist_major_vertices,
                                                                edgelist_minor_vertices,
                                                                edgelist_edge_counts,
                                                                do_expensive_check);
}

template <typename vertex_t, typename edge_t, bool multi_gpu>
std::enable_if_t<!multi_gpu, rmm::device_uvector<vertex_t>> renumber_edgelist(
  raft::handle_t const& handle,
  std::optional<std::tuple<vertex_t const*, vertex_t>> optional_vertex_span,
  vertex_t* edgelist_major_vertices /* [INOUT] */,
  vertex_t* edgelist_minor_vertices /* [INOUT] */,
  edge_t num_edgelist_edges,
  bool do_expensive_check)
{
  return detail::renumber_edgelist<vertex_t, edge_t, multi_gpu>(handle,
                                                                optional_vertex_span,
                                                                edgelist_major_vertices,
                                                                edgelist_minor_vertices,
                                                                num_edgelist_edges,
                                                                do_expensive_check);
}

// explicit instantiation directives (EIDir's):
//

// instantiations for <vertex_t == int32_t, edge_t == int32_t>
//
template std::tuple<rmm::device_uvector<int32_t>, partition_t<int32_t>, int32_t, int32_t>
renumber_edgelist<int32_t, int32_t, true>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int32_t const*, int32_t>> optional_local_vertex_span,
  std::vector<int32_t*> const& edgelist_major_vertices /* [INOUT] */,
  std::vector<int32_t*> const& edgelist_minor_vertices /* [INOUT] */,
  std::vector<int32_t> const& edgelist_edge_counts,
  bool do_expensive_check);

template rmm::device_uvector<int32_t> renumber_edgelist<int32_t, int32_t, false>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int32_t const*, int32_t>> optional_vertex_span,
  int32_t* edgelist_major_vertices /* [INOUT] */,
  int32_t* edgelist_minor_vertices /* [INOUT] */,
  int32_t num_edgelist_edges,
  bool do_expensive_check);

// instantiations for <vertex_t == int32_t, edge_t == int64_t>
//
template std::tuple<rmm::device_uvector<int32_t>, partition_t<int32_t>, int32_t, int64_t>
renumber_edgelist<int32_t, int64_t, true>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int32_t const*, int32_t>> optional_local_vertex_span,
  std::vector<int32_t*> const& edgelist_major_vertices /* [INOUT] */,
  std::vector<int32_t*> const& edgelist_minor_vertices /* [INOUT] */,
  std::vector<int64_t> const& edgelist_edge_counts,
  bool do_expensive_check);

template rmm::device_uvector<int32_t> renumber_edgelist<int32_t, int64_t, false>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int32_t const*, int32_t>> optional_vertex_span,
  int32_t* edgelist_major_vertices /* [INOUT] */,
  int32_t* edgelist_minor_vertices /* [INOUT] */,
  int64_t num_edgelist_edges,
  bool do_expensive_check);

// instantiations for <vertex_t == int64_t, edge_t == int64_t>
//
template std::tuple<rmm::device_uvector<int64_t>, partition_t<int64_t>, int64_t, int64_t>
renumber_edgelist<int64_t, int64_t, true>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int64_t const*, int64_t>> optional_local_vertex_span,
  std::vector<int64_t*> const& edgelist_major_vertices /* [INOUT] */,
  std::vector<int64_t*> const& edgelist_minor_vertices /* [INOUT] */,
  std::vector<int64_t> const& edgelist_edge_counts,
  bool do_expensive_check);

template rmm::device_uvector<int64_t> renumber_edgelist<int64_t, int64_t, false>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int64_t const*, int64_t>> optional_vertex_span,
  int64_t* edgelist_major_vertices /* [INOUT] */,
  int64_t* edgelist_minor_vertices /* [INOUT] */,
  int64_t num_edgelist_edges,
  bool do_expensive_check);

}  // namespace experimental
}  // namespace cugraph
