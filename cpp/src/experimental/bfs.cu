#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cugraph/algorithms.hpp>
#include <cugraph/experimental/graph_view.hpp>
#include <cugraph/prims/reduce_op.cuh>
#include <cugraph/prims/update_frontier_v_push_if_out_nbr.cuh>
#include <cugraph/prims/vertex_frontier.cuh>
#include <cugraph/utilities/error.hpp>
#include <cugraph/vertex_partition_device.cuh>

#include <rmm/thrust_rmm_allocator.h>
#include <raft/handle.hpp>

#include <thrust/fill.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/optional.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>

#include <limits>
#include <type_traits>

namespace cugraph {
namespace experimental {
namespace detail {

template <typename GraphViewType, typename PredecessorIterator>
void bfs(raft::handle_t const &handle,
         GraphViewType const &push_graph_view,
         typename GraphViewType::vertex_type *distances,
         PredecessorIterator predecessor_first,
         typename GraphViewType::vertex_type source_vertex,
         bool direction_optimizing,
         typename GraphViewType::vertex_type depth_limit,
         bool do_expensive_check)
{
  using vertex_t = typename GraphViewType::vertex_type;

  static_assert(std::is_integral<vertex_t>::value,
                "GraphViewType::vertex_type should be integral.");
  static_assert(!GraphViewType::is_adj_matrix_transposed,
                "GraphViewType should support the push model.");

  auto const num_vertices = push_graph_view.get_number_of_vertices();
  if (num_vertices == 0) { return; }

  // 1. check input arguments

  CUGRAPH_EXPECTS(
    push_graph_view.is_symmetric() || !direction_optimizing,
    "Invalid input argument: input graph should be symmetric for direction optimizing BFS.");
  CUGRAPH_EXPECTS(push_graph_view.is_valid_vertex(source_vertex),
                  "Invalid input argument: source vertex out-of-range.");

  if (do_expensive_check) {
    // nothing to do
  }

  // 2. initialize distances and predecessors

  auto constexpr invalid_distance = std::numeric_limits<vertex_t>::max();
  auto constexpr invalid_vertex   = invalid_vertex_id<vertex_t>::value;

  auto val_first = thrust::make_zip_iterator(thrust::make_tuple(distances, predecessor_first));
  thrust::transform(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                    thrust::make_counting_iterator(push_graph_view.get_local_vertex_first()),
                    thrust::make_counting_iterator(push_graph_view.get_local_vertex_last()),
                    val_first,
                    [source_vertex] __device__(auto val) {
                      auto distance = invalid_distance;
                      if (val == source_vertex) { distance = vertex_t{0}; }
                      return thrust::make_tuple(distance, invalid_vertex);
                    });

  // 3. initialize BFS frontier

  enum class Bucket { cur, next, num_buckets };
  VertexFrontier<vertex_t,
                 void,
                 GraphViewType::is_multi_gpu,
                 static_cast<size_t>(Bucket::num_buckets)>
    vertex_frontier(handle);

  if (push_graph_view.is_local_vertex_nocheck(source_vertex)) {
    vertex_frontier.get_bucket(static_cast<size_t>(Bucket::cur)).insert(source_vertex);
  }

  // 4. BFS iteration

  vertex_t depth{0};
  while (true) {
    if (direction_optimizing) {
      CUGRAPH_FAIL("unimplemented.");
    } else {
      vertex_partition_device_t<GraphViewType> vertex_partition(push_graph_view);

      update_frontier_v_push_if_out_nbr(
        handle,
        push_graph_view,
        vertex_frontier,
        static_cast<size_t>(Bucket::cur),
        std::vector<size_t>{static_cast<size_t>(Bucket::next)},
        thrust::make_constant_iterator(0) /* dummy */,
        thrust::make_constant_iterator(0) /* dummy */,
        [vertex_partition, distances] __device__(
          vertex_t src, vertex_t dst, auto src_val, auto dst_val) {
          auto push = true;
          if (vertex_partition.is_local_vertex_nocheck(dst)) {
            auto distance =
              *(distances + vertex_partition.get_local_vertex_offset_from_vertex_nocheck(dst));
            if (distance != invalid_distance) { push = false; }
          }
          return push ? thrust::optional<vertex_t>{src} : thrust::nullopt;
        },
        reduce_op::any<vertex_t>(),
        distances,
        thrust::make_zip_iterator(thrust::make_tuple(distances, predecessor_first)),
        [depth] __device__(auto v, auto v_val, auto pushed_val) {
          return (v_val == invalid_distance)
                   ? thrust::optional<
                       thrust::tuple<size_t, thrust::tuple<vertex_t, vertex_t>>>{thrust::make_tuple(
                       static_cast<size_t>(Bucket::next),
                       thrust::make_tuple(depth + 1, pushed_val))}
                   : thrust::nullopt;
        });

      vertex_frontier.get_bucket(static_cast<size_t>(Bucket::cur)).clear();
      vertex_frontier.get_bucket(static_cast<size_t>(Bucket::cur)).shrink_to_fit();
      vertex_frontier.swap_buckets(static_cast<size_t>(Bucket::cur),
                                   static_cast<size_t>(Bucket::next));
      if (vertex_frontier.get_bucket(static_cast<size_t>(Bucket::cur)).aggregate_size() == 0) {
        break;
      }
    }

    depth++;
    if (depth >= depth_limit) { break; }
  }

  CUDA_TRY(hipStreamSynchronize(
    handle.get_stream()));  // this is as necessary vertex_frontier will become out-of-scope once
                            // this function returns (FIXME: should I stream sync in VertexFrontier
                            // destructor?)
}

}  // namespace detail

template <typename vertex_t, typename edge_t, typename weight_t, bool multi_gpu>
void bfs(raft::handle_t const &handle,
         graph_view_t<vertex_t, edge_t, weight_t, false, multi_gpu> const &graph_view,
         vertex_t *distances,
         vertex_t *predecessors,
         vertex_t source_vertex,
         bool direction_optimizing,
         vertex_t depth_limit,
         bool do_expensive_check)
{
  if (predecessors != nullptr) {
    detail::bfs(handle,
                graph_view,
                distances,
                predecessors,
                source_vertex,
                direction_optimizing,
                depth_limit,
                do_expensive_check);
  } else {
    detail::bfs(handle,
                graph_view,
                distances,
                thrust::make_discard_iterator(),
                source_vertex,
                direction_optimizing,
                depth_limit,
                do_expensive_check);
  }
}

// explicit instantiation

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int32_t, float, false, true> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t source_vertex,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int32_t, double, false, true> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t source_vertex,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int64_t, float, false, true> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t source_vertex,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int64_t, double, false, true> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t source_vertex,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int64_t, int64_t, float, false, true> const &graph_view,
                  int64_t *distances,
                  int64_t *predecessors,
                  int64_t source_vertex,
                  bool direction_optimizing,
                  int64_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int64_t, int64_t, double, false, true> const &graph_view,
                  int64_t *distances,
                  int64_t *predecessors,
                  int64_t source_vertex,
                  bool direction_optimizing,
                  int64_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int32_t, float, false, false> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t source_vertex,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int32_t, double, false, false> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t source_vertex,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int64_t, float, false, false> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t source_vertex,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int64_t, double, false, false> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t source_vertex,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int64_t, int64_t, float, false, false> const &graph_view,
                  int64_t *distances,
                  int64_t *predecessors,
                  int64_t source_vertex,
                  bool direction_optimizing,
                  int64_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int64_t, int64_t, double, false, false> const &graph_view,
                  int64_t *distances,
                  int64_t *predecessors,
                  int64_t source_vertex,
                  bool direction_optimizing,
                  int64_t depth_limit,
                  bool do_expensive_check);

}  // namespace experimental
}  // namespace cugraph
