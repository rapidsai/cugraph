#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <algorithms.hpp>
#include <experimental/graph_view.hpp>
#include <patterns/reduce_op.cuh>
#include <patterns/update_frontier_v_push_if_out_nbr.cuh>
#include <patterns/vertex_frontier.cuh>
#include <utilities/error.hpp>
#include <vertex_partition_device.cuh>

#include <rmm/thrust_rmm_allocator.h>
#include <raft/handle.hpp>

#include <thrust/fill.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>

#include <limits>
#include <type_traits>

namespace cugraph {
namespace experimental {
namespace detail {

template <typename GraphViewType, typename PredecessorIterator>
void bfs(raft::handle_t const &handle,
         GraphViewType const &push_graph_view,
         typename GraphViewType::vertex_type *distances,
         PredecessorIterator predecessor_first,
         typename GraphViewType::vertex_type source_vertex,
         bool direction_optimizing,
         typename GraphViewType::vertex_type depth_limit,
         bool do_expensive_check)
{
  using vertex_t = typename GraphViewType::vertex_type;

  static_assert(std::is_integral<vertex_t>::value,
                "GraphViewType::vertex_type should be integral.");
  static_assert(!GraphViewType::is_adj_matrix_transposed,
                "GraphViewType should support the push model.");

  auto const num_vertices = push_graph_view.get_number_of_vertices();
  if (num_vertices == 0) { return; }

  // 1. check input arguments

  CUGRAPH_EXPECTS(
    push_graph_view.is_symmetric() || !direction_optimizing,
    "Invalid input argument: input graph should be symmetric for direction optimizing BFS.");
  CUGRAPH_EXPECTS(push_graph_view.is_valid_vertex(source_vertex),
                  "Invalid input argument: source vertex out-of-range.");

  if (do_expensive_check) {
    // nothing to do
  }

  // 2. initialize distances and predecessors

  auto constexpr invalid_distance = std::numeric_limits<vertex_t>::max();
  auto constexpr invalid_vertex   = invalid_vertex_id<vertex_t>::value;

  auto val_first = thrust::make_zip_iterator(thrust::make_tuple(distances, predecessor_first));
  thrust::transform(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                    thrust::make_counting_iterator(push_graph_view.get_local_vertex_first()),
                    thrust::make_counting_iterator(push_graph_view.get_local_vertex_last()),
                    val_first,
                    [source_vertex] __device__(auto val) {
                      auto distance = invalid_distance;
                      if (val == source_vertex) { distance = vertex_t{0}; }
                      return thrust::make_tuple(distance, invalid_vertex);
                    });

  // 3. initialize BFS frontier

  enum class Bucket { cur, num_buckets };
  std::vector<size_t> bucket_sizes(static_cast<size_t>(Bucket::num_buckets),
                                   push_graph_view.get_number_of_local_vertices());
  VertexFrontier<thrust::tuple<vertex_t>,
                 vertex_t,
                 GraphViewType::is_multi_gpu,
                 static_cast<size_t>(Bucket::num_buckets)>
    vertex_frontier(handle, bucket_sizes);

  if (push_graph_view.is_local_vertex_nocheck(source_vertex)) {
    vertex_frontier.get_bucket(static_cast<size_t>(Bucket::cur)).insert(source_vertex);
  }

  // 4. BFS iteration

  vertex_t depth{0};
  auto cur_local_vertex_frontier_first =
    vertex_frontier.get_bucket(static_cast<size_t>(Bucket::cur)).begin();
  auto cur_vertex_frontier_aggregate_size =
    vertex_frontier.get_bucket(static_cast<size_t>(Bucket::cur)).aggregate_size();
  while (true) {
    if (direction_optimizing) {
      CUGRAPH_FAIL("unimplemented.");
    } else {
      vertex_partition_device_t<GraphViewType> vertex_partition(push_graph_view);

      auto cur_local_vertex_frontier_last =
        vertex_frontier.get_bucket(static_cast<size_t>(Bucket::cur)).end();
      update_frontier_v_push_if_out_nbr(
        handle,
        push_graph_view,
        cur_local_vertex_frontier_first,
        cur_local_vertex_frontier_last,
        thrust::make_constant_iterator(0) /* dummy */,
        thrust::make_constant_iterator(0) /* dummy */,
        [vertex_partition, distances] __device__(
          vertex_t src, vertex_t dst, auto src_val, auto dst_val) {
          auto push = true;
          if (vertex_partition.is_local_vertex_nocheck(dst)) {
            auto distance =
              *(distances + vertex_partition.get_local_vertex_offset_from_vertex_nocheck(dst));
            if (distance != invalid_distance) { push = false; }
          }
          // FIXME: need to test this works properly if payload size is 0 (returns a tuple of size
          // 1)
          return thrust::make_tuple(push, src);
        },
        reduce_op::any<thrust::tuple<vertex_t>>(),
        distances,
        thrust::make_zip_iterator(thrust::make_tuple(distances, predecessor_first)),
        vertex_frontier,
        [depth] __device__(auto v_val, auto pushed_val) {
          auto idx = (v_val == invalid_distance)
                       ? static_cast<size_t>(Bucket::cur)
                       : VertexFrontier<thrust::tuple<vertex_t>, vertex_t>::kInvalidBucketIdx;
          return thrust::make_tuple(idx, depth + 1, thrust::get<0>(pushed_val));
        });

      auto new_vertex_frontier_aggregate_size =
        vertex_frontier.get_bucket(static_cast<size_t>(Bucket::cur)).aggregate_size() -
        cur_vertex_frontier_aggregate_size;
      if (new_vertex_frontier_aggregate_size == 0) { break; }

      cur_local_vertex_frontier_first = cur_local_vertex_frontier_last;
      cur_vertex_frontier_aggregate_size += new_vertex_frontier_aggregate_size;
    }

    depth++;
    if (depth >= depth_limit) { break; }
  }

  CUDA_TRY(hipStreamSynchronize(
    handle.get_stream()));  // this is as necessary vertex_frontier will become out-of-scope once
                            // this function returns (FIXME: should I stream sync in VertexFrontier
                            // destructor?)

  return;
}

}  // namespace detail

template <typename vertex_t, typename edge_t, typename weight_t, bool multi_gpu>
void bfs(raft::handle_t const &handle,
         graph_view_t<vertex_t, edge_t, weight_t, false, multi_gpu> const &graph_view,
         vertex_t *distances,
         vertex_t *predecessors,
         vertex_t source_vertex,
         bool direction_optimizing,
         vertex_t depth_limit,
         bool do_expensive_check)
{
  if (predecessors != nullptr) {
    detail::bfs(handle,
                graph_view,
                distances,
                predecessors,
                source_vertex,
                direction_optimizing,
                depth_limit,
                do_expensive_check);
  } else {
    detail::bfs(handle,
                graph_view,
                distances,
                thrust::make_discard_iterator(),
                source_vertex,
                direction_optimizing,
                depth_limit,
                do_expensive_check);
  }
}

// explicit instantiation

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int32_t, float, false, true> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t source_vertex,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int32_t, double, false, true> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t source_vertex,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int64_t, float, false, true> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t source_vertex,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int64_t, double, false, true> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t source_vertex,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int64_t, int64_t, float, false, true> const &graph_view,
                  int64_t *distances,
                  int64_t *predecessors,
                  int64_t source_vertex,
                  bool direction_optimizing,
                  int64_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int64_t, int64_t, double, false, true> const &graph_view,
                  int64_t *distances,
                  int64_t *predecessors,
                  int64_t source_vertex,
                  bool direction_optimizing,
                  int64_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int32_t, float, false, false> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t source_vertex,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int32_t, double, false, false> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t source_vertex,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int64_t, float, false, false> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t source_vertex,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int32_t, int64_t, double, false, false> const &graph_view,
                  int32_t *distances,
                  int32_t *predecessors,
                  int32_t source_vertex,
                  bool direction_optimizing,
                  int32_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int64_t, int64_t, float, false, false> const &graph_view,
                  int64_t *distances,
                  int64_t *predecessors,
                  int64_t source_vertex,
                  bool direction_optimizing,
                  int64_t depth_limit,
                  bool do_expensive_check);

template void bfs(raft::handle_t const &handle,
                  graph_view_t<int64_t, int64_t, double, false, false> const &graph_view,
                  int64_t *distances,
                  int64_t *predecessors,
                  int64_t source_vertex,
                  bool direction_optimizing,
                  int64_t depth_limit,
                  bool do_expensive_check);

}  // namespace experimental
}  // namespace cugraph
