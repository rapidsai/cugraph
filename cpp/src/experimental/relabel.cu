/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <experimental/include_cuco_static_map.cuh>

#include <experimental/detail/graph_utils.cuh>
#include <experimental/graph.hpp>
#include <experimental/graph_functions.hpp>
#include <experimental/graph_view.hpp>
#include <patterns/copy_to_adj_matrix_row_col.cuh>
#include <utilities/error.hpp>
#include <utilities/shuffle_comm.cuh>

#include <rmm/thrust_rmm_allocator.h>
#include <raft/handle.hpp>
#include <rmm/device_uvector.hpp>

#include <thrust/copy.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/sort.h>
#include <thrust/tuple.h>

#include <algorithm>
#include <iterator>
#include <numeric>
#include <tuple>
#include <utility>

namespace cugraph {
namespace experimental {

template <typename vertex_t, bool multi_gpu>
void relabel(raft::handle_t const& handle,
             std::tuple<vertex_t const*, vertex_t const*> old_new_label_pairs,
             vertex_t num_label_pairs,
             vertex_t* labels /* [INOUT] */,
             vertex_t num_labels,
             bool do_expensive_check)
{
  double constexpr load_factor = 0.7;

  // FIXME: remove this check once we drop Pascal support
  CUGRAPH_EXPECTS(handle.get_device_properties().major >= 7,
                  "Relabel not supported on Pascal and older architectures.");

#ifdef CUCO_STATIC_MAP_DEFINED
  if (multi_gpu) {
    auto& comm           = handle.get_comms();
    auto const comm_size = comm.get_size();

    auto key_func = detail::compute_gpu_id_from_vertex_t<vertex_t>{comm_size};

    // find unique old labels (to be relabeled)

    rmm::device_uvector<vertex_t> unique_old_labels(num_labels, handle.get_stream());
    thrust::copy(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                 labels,
                 labels + num_labels,
                 unique_old_labels.data());
    thrust::sort(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                 unique_old_labels.begin(),
                 unique_old_labels.end());
    unique_old_labels.resize(
      thrust::distance(
        unique_old_labels.begin(),
        thrust::unique(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                       unique_old_labels.begin(),
                       unique_old_labels.end())),
      handle.get_stream());
    unique_old_labels.shrink_to_fit(handle.get_stream());

    // collect new labels for the unique old labels

    rmm::device_uvector<vertex_t> new_labels_for_unique_old_labels(0, handle.get_stream());
    {
      // shuffle the old_new_label_pairs based on applying the compute_gpu_id_from_vertex_t functor
      // to the old labels

      rmm::device_uvector<vertex_t> rx_label_pair_old_labels(0, handle.get_stream());
      rmm::device_uvector<vertex_t> rx_label_pair_new_labels(0, handle.get_stream());
      {
        rmm::device_uvector<vertex_t> label_pair_old_labels(num_label_pairs, handle.get_stream());
        rmm::device_uvector<vertex_t> label_pair_new_labels(num_label_pairs, handle.get_stream());
        thrust::copy(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                     std::get<0>(old_new_label_pairs),
                     std::get<0>(old_new_label_pairs) + num_label_pairs,
                     label_pair_old_labels.begin());
        thrust::copy(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                     std::get<1>(old_new_label_pairs),
                     std::get<1>(old_new_label_pairs) + num_label_pairs,
                     label_pair_new_labels.begin());
        auto pair_first = thrust::make_zip_iterator(
          thrust::make_tuple(label_pair_old_labels.begin(), label_pair_new_labels.begin()));
        std::forward_as_tuple(std::tie(rx_label_pair_old_labels, rx_label_pair_new_labels),
                              std::ignore) =
          sort_and_shuffle_values(
            handle.get_comms(),
            pair_first,
            pair_first + num_label_pairs,
            [key_func] __device__(auto val) { return key_func(thrust::get<0>(val)); },
            handle.get_stream());
      }

      // update intermediate relabel map

      cuco::static_map<vertex_t, vertex_t> relabel_map{
        static_cast<size_t>(static_cast<double>(rx_label_pair_old_labels.size()) / load_factor),
        invalid_vertex_id<vertex_t>::value,
        invalid_vertex_id<vertex_t>::value};

      auto pair_first = thrust::make_transform_iterator(
        thrust::make_zip_iterator(
          thrust::make_tuple(rx_label_pair_old_labels.begin(), rx_label_pair_new_labels.begin())),
        [] __device__(auto val) {
          return thrust::make_pair(thrust::get<0>(val), thrust::get<1>(val));
        });
      relabel_map.insert(pair_first, pair_first + rx_label_pair_old_labels.size());

      rx_label_pair_old_labels.resize(0, handle.get_stream());
      rx_label_pair_new_labels.resize(0, handle.get_stream());
      rx_label_pair_old_labels.shrink_to_fit(handle.get_stream());
      rx_label_pair_new_labels.shrink_to_fit(handle.get_stream());

      // shuffle unique_old_labels, relabel using the intermediate relabel map, and shuffle back

      {
        rmm::device_uvector<vertex_t> rx_unique_old_labels(0, handle.get_stream());
        std::vector<size_t> rx_value_counts{};
        std::tie(rx_unique_old_labels, rx_value_counts) = sort_and_shuffle_values(
          handle.get_comms(),
          unique_old_labels.begin(),
          unique_old_labels.end(),
          [key_func] __device__(auto val) { return key_func(val); },
          handle.get_stream());

        CUDA_TRY(hipStreamSynchronize(
          handle.get_stream()));  // cuco::static_map currently does not take stream

        relabel_map.find(
          rx_unique_old_labels.begin(),
          rx_unique_old_labels.end(),
          rx_unique_old_labels
            .begin());  // now rx_unique_old_lables hold new labels for the corresponding old labels

        std::tie(new_labels_for_unique_old_labels, std::ignore) = shuffle_values(
          handle.get_comms(), rx_unique_old_labels.begin(), rx_value_counts, handle.get_stream());
      }
    }

    cuco::static_map<vertex_t, vertex_t> relabel_map(
      static_cast<size_t>(static_cast<double>(unique_old_labels.size()) / load_factor),
      invalid_vertex_id<vertex_t>::value,
      invalid_vertex_id<vertex_t>::value);

    auto pair_first = thrust::make_transform_iterator(
      thrust::make_zip_iterator(
        thrust::make_tuple(unique_old_labels.begin(), new_labels_for_unique_old_labels.begin())),
      [] __device__(auto val) {
        return thrust::make_pair(thrust::get<0>(val), thrust::get<1>(val));
      });

    relabel_map.insert(pair_first, pair_first + unique_old_labels.size());
    relabel_map.find(labels, labels + num_labels, labels);
  } else {
    cuco::static_map<vertex_t, vertex_t> relabel_map(
      static_cast<size_t>(static_cast<double>(num_label_pairs) / load_factor),
      invalid_vertex_id<vertex_t>::value,
      invalid_vertex_id<vertex_t>::value);

    auto pair_first = thrust::make_transform_iterator(
      thrust::make_zip_iterator(
        thrust::make_tuple(std::get<0>(old_new_label_pairs), std::get<1>(old_new_label_pairs))),
      [] __device__(auto val) {
        return thrust::make_pair(thrust::get<0>(val), thrust::get<1>(val));
      });

    relabel_map.insert(pair_first, pair_first + num_label_pairs);
    relabel_map.find(labels, labels + num_labels, labels);
  }

  if (do_expensive_check) {
    CUGRAPH_EXPECTS(
      thrust::count(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                    labels,
                    labels + num_labels,
                    invalid_vertex_id<vertex_t>::value) == 0,
      "Invalid input argument: labels include old label values missing in old_new_label_pairs.");
  }
#endif

  return;
}

// explicit instantiation

template void relabel<int32_t, true>(raft::handle_t const& handle,
                                     std::tuple<int32_t const*, int32_t const*> old_new_label_pairs,
                                     int32_t num_label_pairs,
                                     int32_t* labels,
                                     int32_t num_labels,
                                     bool do_expensive_check);

template void relabel<int32_t, false>(
  raft::handle_t const& handle,
  std::tuple<int32_t const*, int32_t const*> old_new_label_pairs,
  int32_t num_label_pairs,
  int32_t* labels,
  int32_t num_labels,
  bool do_expensive_check);

}  // namespace experimental
}  // namespace cugraph
