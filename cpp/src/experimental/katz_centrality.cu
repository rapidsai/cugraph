#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cugraph/algorithms.hpp>
#include <cugraph/experimental/graph_view.hpp>
#include <cugraph/patterns/copy_to_adj_matrix_row_col.cuh>
#include <cugraph/patterns/copy_v_transform_reduce_in_out_nbr.cuh>
#include <cugraph/patterns/count_if_v.cuh>
#include <cugraph/patterns/transform_reduce_v.cuh>
#include <cugraph/utilities/error.hpp>

#include <rmm/thrust_rmm_allocator.h>
#include <raft/handle.hpp>

#include <thrust/fill.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>

namespace cugraph {
namespace experimental {
namespace detail {

template <typename GraphViewType, typename result_t>
void katz_centrality(raft::handle_t const &handle,
                     GraphViewType const &pull_graph_view,
                     result_t const *betas,
                     result_t *katz_centralities,
                     result_t alpha,
                     result_t beta,  // relevant only if betas == nullptr
                     result_t epsilon,
                     size_t max_iterations,
                     bool has_initial_guess,
                     bool normalize,
                     bool do_expensive_check)
{
  using vertex_t = typename GraphViewType::vertex_type;
  using weight_t = typename GraphViewType::weight_type;

  static_assert(std::is_integral<vertex_t>::value,
                "GraphViewType::vertex_type should be integral.");
  static_assert(std::is_floating_point<result_t>::value,
                "result_t should be a floating-point type.");
  static_assert(GraphViewType::is_adj_matrix_transposed,
                "GraphViewType should support the pull model.");

  auto const num_vertices = pull_graph_view.get_number_of_vertices();
  if (num_vertices == 0) { return; }

  // 1. check input arguments

  CUGRAPH_EXPECTS((alpha >= 0.0) && (alpha <= 1.0),
                  "Invalid input argument: alpha should be in [0.0, 1.0].");
  CUGRAPH_EXPECTS(epsilon >= 0.0, "Invalid input argument: epsilon should be non-negative.");

  if (do_expensive_check) {
    // FIXME: should I check for betas?

    if (has_initial_guess) {
      auto num_negative_values = count_if_v(
        handle, pull_graph_view, katz_centralities, [] __device__(auto val) { return val < 0.0; });
      CUGRAPH_EXPECTS(num_negative_values == 0,
                      "Invalid input argument: initial guess values should be non-negative.");
    }
  }

  // 2. initialize katz centrality values

  if (!has_initial_guess) {
    thrust::fill(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                 katz_centralities,
                 katz_centralities + pull_graph_view.get_number_of_local_vertices(),
                 result_t{0.0});
  }

  // 3. katz centrality iteration

  // old katz centrality values
  rmm::device_uvector<result_t> tmp_katz_centralities(
    pull_graph_view.get_number_of_local_vertices(), handle.get_stream());
  rmm::device_uvector<result_t> adj_matrix_row_katz_centralities(
    pull_graph_view.get_number_of_local_adj_matrix_partition_rows(), handle.get_stream());
  auto new_katz_centralities = katz_centralities;
  auto old_katz_centralities = tmp_katz_centralities.data();
  size_t iter{0};
  while (true) {
    std::swap(new_katz_centralities, old_katz_centralities);

    copy_to_adj_matrix_row(
      handle, pull_graph_view, old_katz_centralities, adj_matrix_row_katz_centralities.begin());

    copy_v_transform_reduce_in_nbr(
      handle,
      pull_graph_view,
      adj_matrix_row_katz_centralities.begin(),
      thrust::make_constant_iterator(0) /* dummy */,
      [alpha] __device__(vertex_t src, vertex_t dst, weight_t w, auto src_val, auto dst_val) {
        return static_cast<result_t>(alpha * src_val * w);
      },
      betas != nullptr ? result_t{0.0} : beta,
      new_katz_centralities);

    if (betas != nullptr) {
      auto val_first = thrust::make_zip_iterator(thrust::make_tuple(new_katz_centralities, betas));
      thrust::transform(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                        val_first,
                        val_first + pull_graph_view.get_number_of_local_vertices(),
                        new_katz_centralities,
                        [] __device__(auto val) {
                          auto const katz_centrality = thrust::get<0>(val);
                          auto const beta            = thrust::get<1>(val);
                          return katz_centrality + beta;
                        });
    }

    auto diff_sum = transform_reduce_v(
      handle,
      pull_graph_view,
      thrust::make_zip_iterator(thrust::make_tuple(new_katz_centralities, old_katz_centralities)),
      [] __device__(auto val) { return std::abs(thrust::get<0>(val) - thrust::get<1>(val)); },
      result_t{0.0});

    iter++;

    if (diff_sum < epsilon) {
      break;
    } else if (iter >= max_iterations) {
      CUGRAPH_FAIL("Katz Centrality failed to converge.");
    }
  }

  if (new_katz_centralities != katz_centralities) {
    thrust::copy(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                 new_katz_centralities,
                 new_katz_centralities + pull_graph_view.get_number_of_local_vertices(),
                 katz_centralities);
  }

  if (normalize) {
    auto l2_norm = transform_reduce_v(
      handle,
      pull_graph_view,
      katz_centralities,
      [] __device__(auto val) { return val * val; },
      result_t{0.0});
    l2_norm = std::sqrt(l2_norm);
    CUGRAPH_EXPECTS(l2_norm > 0.0,
                    "L2 norm of the computed Katz Centrality values should be positive.");
    thrust::transform(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                      katz_centralities,
                      katz_centralities + pull_graph_view.get_number_of_local_vertices(),
                      katz_centralities,
                      [l2_norm] __device__(auto val) { return val / l2_norm; });
  }
}

}  // namespace detail

template <typename vertex_t, typename edge_t, typename weight_t, typename result_t, bool multi_gpu>
void katz_centrality(raft::handle_t const &handle,
                     graph_view_t<vertex_t, edge_t, weight_t, true, multi_gpu> const &graph_view,
                     result_t const *betas,
                     result_t *katz_centralities,
                     result_t alpha,
                     result_t beta,  // relevant only if beta == nullptr
                     result_t epsilon,
                     size_t max_iterations,
                     bool has_initial_guess,
                     bool normalize,
                     bool do_expensive_check)
{
  detail::katz_centrality(handle,
                          graph_view,
                          betas,
                          katz_centralities,
                          alpha,
                          beta,
                          epsilon,
                          max_iterations,
                          has_initial_guess,
                          normalize,
                          do_expensive_check);
}

// explicit instantiation

template void katz_centrality(raft::handle_t const &handle,
                              graph_view_t<int32_t, int32_t, float, true, true> const &graph_view,
                              float const *betas,
                              float *katz_centralities,
                              float alpha,
                              float beta,
                              float epsilon,
                              size_t max_iterations,
                              bool has_initial_guess,
                              bool normalize,
                              bool do_expensive_check);

template void katz_centrality(raft::handle_t const &handle,
                              graph_view_t<int32_t, int32_t, double, true, true> const &graph_view,
                              double const *betas,
                              double *katz_centralities,
                              double alpha,
                              double beta,
                              double epsilon,
                              size_t max_iterations,
                              bool has_initial_guess,
                              bool normalize,
                              bool do_expensive_check);

template void katz_centrality(raft::handle_t const &handle,
                              graph_view_t<int32_t, int64_t, float, true, true> const &graph_view,
                              float const *betas,
                              float *katz_centralities,
                              float alpha,
                              float beta,
                              float epsilon,
                              size_t max_iterations,
                              bool has_initial_guess,
                              bool normalize,
                              bool do_expensive_check);

template void katz_centrality(raft::handle_t const &handle,
                              graph_view_t<int32_t, int64_t, double, true, true> const &graph_view,
                              double const *betas,
                              double *katz_centralities,
                              double alpha,
                              double beta,
                              double epsilon,
                              size_t max_iterations,
                              bool has_initial_guess,
                              bool normalize,
                              bool do_expensive_check);

template void katz_centrality(raft::handle_t const &handle,
                              graph_view_t<int64_t, int64_t, float, true, true> const &graph_view,
                              float const *betas,
                              float *katz_centralities,
                              float alpha,
                              float beta,
                              float epsilon,
                              size_t max_iterations,
                              bool has_initial_guess,
                              bool normalize,
                              bool do_expensive_check);

template void katz_centrality(raft::handle_t const &handle,
                              graph_view_t<int64_t, int64_t, double, true, true> const &graph_view,
                              double const *betas,
                              double *katz_centralities,
                              double alpha,
                              double beta,
                              double epsilon,
                              size_t max_iterations,
                              bool has_initial_guess,
                              bool normalize,
                              bool do_expensive_check);

template void katz_centrality(raft::handle_t const &handle,
                              graph_view_t<int32_t, int32_t, float, true, false> const &graph_view,
                              float const *betas,
                              float *katz_centralities,
                              float alpha,
                              float beta,
                              float epsilon,
                              size_t max_iterations,
                              bool has_initial_guess,
                              bool normalize,
                              bool do_expensive_check);

template void katz_centrality(raft::handle_t const &handle,
                              graph_view_t<int32_t, int32_t, double, true, false> const &graph_view,
                              double const *betas,
                              double *katz_centralities,
                              double alpha,
                              double beta,
                              double epsilon,
                              size_t max_iterations,
                              bool has_initial_guess,
                              bool normalize,
                              bool do_expensive_check);

template void katz_centrality(raft::handle_t const &handle,
                              graph_view_t<int32_t, int64_t, float, true, false> const &graph_view,
                              float const *betas,
                              float *katz_centralities,
                              float alpha,
                              float beta,
                              float epsilon,
                              size_t max_iterations,
                              bool has_initial_guess,
                              bool normalize,
                              bool do_expensive_check);

template void katz_centrality(raft::handle_t const &handle,
                              graph_view_t<int32_t, int64_t, double, true, false> const &graph_view,
                              double const *betas,
                              double *katz_centralities,
                              double alpha,
                              double beta,
                              double epsilon,
                              size_t max_iterations,
                              bool has_initial_guess,
                              bool normalize,
                              bool do_expensive_check);

template void katz_centrality(raft::handle_t const &handle,
                              graph_view_t<int64_t, int64_t, float, true, false> const &graph_view,
                              float const *betas,
                              float *katz_centralities,
                              float alpha,
                              float beta,
                              float epsilon,
                              size_t max_iterations,
                              bool has_initial_guess,
                              bool normalize,
                              bool do_expensive_check);

template void katz_centrality(raft::handle_t const &handle,
                              graph_view_t<int64_t, int64_t, double, true, false> const &graph_view,
                              double const *betas,
                              double *katz_centralities,
                              double alpha,
                              double beta,
                              double epsilon,
                              size_t max_iterations,
                              bool has_initial_guess,
                              bool normalize,
                              bool do_expensive_check);

}  // namespace experimental
}  // namespace cugraph
