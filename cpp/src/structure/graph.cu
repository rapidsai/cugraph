#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 */

#include <graph.hpp>
#include "utilities/cuda_utils.cuh"
#include "utilities/error_utils.h"
#include "utilities/graph_utils.cuh"

namespace {

template <typename vertex_t, typename edge_t>
void degree_from_offsets(vertex_t number_of_vertices,
                         edge_t const *offsets,
                         edge_t *degree,
                         hipStream_t stream)
{
  // Computes out-degree for x = 0 and x = 2
  thrust::for_each(
    rmm::exec_policy(stream)->on(stream),
    thrust::make_counting_iterator<vertex_t>(0),
    thrust::make_counting_iterator<vertex_t>(number_of_vertices),
    [offsets, degree] __device__(vertex_t v) { degree[v] = offsets[v + 1] - offsets[v]; });
}

template <typename vertex_t, typename edge_t>
void degree_from_vertex_ids(edge_t number_of_edges,
                            vertex_t const *indices,
                            edge_t *degree,
                            hipStream_t stream)
{
  thrust::for_each(
    rmm::exec_policy(stream)->on(stream),
    thrust::make_counting_iterator<edge_t>(0),
    thrust::make_counting_iterator<edge_t>(number_of_edges),
    [indices, degree] __device__(edge_t e) { cugraph::atomicAdd(degree + indices[e], 1); });
}

}  // namespace

namespace cugraph {
namespace experimental {

template <typename VT, typename ET, typename WT>
void GraphBase<VT, ET, WT>::get_vertex_identifiers(VT *identifiers) const
{
  cugraph::detail::sequence<VT>(number_of_vertices, identifiers);
}

template <typename VT, typename ET, typename WT>
void GraphCompressedSparseBase<VT, ET, WT>::get_source_indices(VT *src_indices) const
{
  CUGRAPH_EXPECTS(offsets != nullptr, "No graph specified");
  cugraph::detail::offsets_to_indices<VT>(
    offsets, GraphBase<VT, ET, WT>::number_of_vertices, src_indices);
}

template <typename VT, typename ET, typename WT>
void GraphCOO<VT, ET, WT>::degree(ET *degree, DegreeDirection direction) const
{
  //
  // NOTE:  We assume offsets/indices are a CSR.  If a CSC is passed
  //        in then x should be modified to reflect the expected direction.
  //        (e.g. if you have a CSC and you want in-degree (x=1) then pass
  //        the offsets/indices and request an out-degree (x=2))
  //
  hipStream_t stream{nullptr};

  if (direction != DegreeDirection::IN) {
    degree_from_vertex_ids(GraphBase<VT, ET, WT>::number_of_edges, src_indices, degree, stream);
  }

  if (direction != DegreeDirection::OUT) {
    degree_from_vertex_ids(GraphBase<VT, ET, WT>::number_of_edges, dst_indices, degree, stream);
  }
}

template <typename VT, typename ET, typename WT>
void GraphCompressedSparseBase<VT, ET, WT>::degree(ET *degree, DegreeDirection direction) const
{
  //
  // NOTE:  We assume offsets/indices are a CSR.  If a CSC is passed
  //        in then x should be modified to reflect the expected direction.
  //        (e.g. if you have a CSC and you want in-degree (x=1) then pass
  //        the offsets/indices and request an out-degree (x=2))
  //
  hipStream_t stream{nullptr};

  if (direction != DegreeDirection::IN) {
    degree_from_offsets(GraphBase<VT, ET, WT>::number_of_vertices, offsets, degree, stream);
  }

  if (direction != DegreeDirection::OUT) {
    degree_from_vertex_ids(GraphBase<VT, ET, WT>::number_of_edges, indices, degree, stream);
  }
}

// explicit instantiation
template class GraphBase<int32_t, int32_t, float>;
template class GraphBase<int32_t, int32_t, double>;
template class GraphCOO<int32_t, int32_t, float>;
template class GraphCOO<int32_t, int32_t, double>;
template class GraphCompressedSparseBase<int32_t, int32_t, float>;
template class GraphCompressedSparseBase<int32_t, int32_t, double>;
}  // namespace experimental
}  // namespace cugraph
