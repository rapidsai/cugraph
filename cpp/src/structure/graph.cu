#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <graph.hpp>
#include "utilities/cuda_utils.cuh"
#include "utilities/error.hpp"
#include "utilities/graph_utils.cuh"

namespace {

template <typename vertex_t, typename edge_t>
void degree_from_offsets(vertex_t number_of_vertices,
                         edge_t const *offsets,
                         edge_t *degree,
                         hipStream_t stream)
{
  // Computes out-degree for x = 0 and x = 2
  thrust::for_each(
    rmm::exec_policy(stream)->on(stream),
    thrust::make_counting_iterator<vertex_t>(0),
    thrust::make_counting_iterator<vertex_t>(number_of_vertices),
    [offsets, degree] __device__(vertex_t v) { degree[v] = offsets[v + 1] - offsets[v]; });
}

template <typename vertex_t, typename edge_t>
void degree_from_vertex_ids(const raft::handle_t &handle,
                            vertex_t number_of_vertices,
                            edge_t number_of_edges,
                            vertex_t const *indices,
                            edge_t *degree,
                            hipStream_t stream)
{
  thrust::for_each(
    rmm::exec_policy(stream)->on(stream),
    thrust::make_counting_iterator<edge_t>(0),
    thrust::make_counting_iterator<edge_t>(number_of_edges),
    [indices, degree] __device__(edge_t e) { cugraph::atomicAdd(degree + indices[e], 1); });
  if (handle.comms_initialized()) {
    auto &comm = handle.get_comms();
    comm.allreduce(degree, degree, number_of_vertices, raft::comms::op_t::SUM, stream);
  }
}

}  // namespace

namespace cugraph {
namespace experimental {

template <typename VT, typename ET, typename WT>
void GraphViewBase<VT, ET, WT>::get_vertex_identifiers(VT *identifiers) const
{
  cugraph::detail::sequence<VT>(number_of_vertices, identifiers);
}

template <typename VT, typename ET, typename WT>
void GraphCompressedSparseBaseView<VT, ET, WT>::get_source_indices(VT *src_indices) const
{
  CUGRAPH_EXPECTS(offsets != nullptr, "No graph specified");
  cugraph::detail::offsets_to_indices<VT>(
    offsets, GraphViewBase<VT, ET, WT>::number_of_vertices, src_indices);
}

template <typename VT, typename ET, typename WT>
void GraphCOOView<VT, ET, WT>::degree(ET *degree, DegreeDirection direction) const
{
  //
  // NOTE:  We assume offsets/indices are a CSR.  If a CSC is passed
  //        in then x should be modified to reflect the expected direction.
  //        (e.g. if you have a CSC and you want in-degree (x=1) then pass
  //        the offsets/indices and request an out-degree (x=2))
  //
  hipStream_t stream{nullptr};

  if (direction != DegreeDirection::IN) {
    if (GraphViewBase<VT, ET, WT>::handle->comms_initialized())  // FIXME retrieve global source
                                                                 // indexing for the allreduce work
      CUGRAPH_FAIL("OPG degree not implemented for OUT degree");
    degree_from_vertex_ids(GraphViewBase<VT, ET, WT>::handle[0],
                           GraphViewBase<VT, ET, WT>::number_of_vertices,
                           GraphViewBase<VT, ET, WT>::number_of_edges,
                           src_indices,
                           degree,
                           stream);
  }

  if (direction != DegreeDirection::OUT) {
    degree_from_vertex_ids(GraphViewBase<VT, ET, WT>::handle[0],
                           GraphViewBase<VT, ET, WT>::number_of_vertices,
                           GraphViewBase<VT, ET, WT>::number_of_edges,
                           dst_indices,
                           degree,
                           stream);
  }
}

template <typename VT, typename ET, typename WT>
void GraphCompressedSparseBaseView<VT, ET, WT>::degree(ET *degree, DegreeDirection direction) const
{
  //
  // NOTE:  We assume offsets/indices are a CSR.  If a CSC is passed
  //        in then x should be modified to reflect the expected direction.
  //        (e.g. if you have a CSC and you want in-degree (x=1) then pass
  //        the offsets/indices and request an out-degree (x=2))
  //
  hipStream_t stream{nullptr};

  if (direction != DegreeDirection::IN) {
    if (GraphViewBase<VT, ET, WT>::handle->comms_initialized())
      CUGRAPH_FAIL("OPG degree not implemented for OUT degree");  // FIXME retrieve global
                                                                  // source indexing for
                                                                  // the allreduce to work
    degree_from_offsets(GraphViewBase<VT, ET, WT>::number_of_vertices, offsets, degree, stream);
  }

  if (direction != DegreeDirection::OUT) {
    degree_from_vertex_ids(GraphViewBase<VT, ET, WT>::handle[0],
                           GraphViewBase<VT, ET, WT>::number_of_vertices,
                           GraphViewBase<VT, ET, WT>::number_of_edges,
                           indices,
                           degree,
                           stream);
  }
}

// explicit instantiation
template class GraphViewBase<int32_t, int32_t, float>;
template class GraphViewBase<int32_t, int32_t, double>;
template class GraphCOOView<int32_t, int32_t, float>;
template class GraphCOOView<int32_t, int32_t, double>;
template class GraphCompressedSparseBaseView<int32_t, int32_t, float>;
template class GraphCompressedSparseBaseView<int32_t, int32_t, double>;
}  // namespace experimental
}  // namespace cugraph
