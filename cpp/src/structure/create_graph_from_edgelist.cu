#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cugraph/experimental/detail/graph_utils.cuh>
#include <cugraph/experimental/graph_functions.hpp>
#include <cugraph/utilities/error.hpp>
#include <cugraph/utilities/shuffle_comm.cuh>

#include <rmm/thrust_rmm_allocator.h>

#include <thrust/functional.h>
#include <thrust/transform_reduce.h>

#include <cstdint>

namespace cugraph {
namespace experimental {

namespace {

template <typename vertex_t,
          typename edge_t,
          typename weight_t,
          bool store_transposed,
          bool multi_gpu>
std::enable_if_t<
  multi_gpu,
  std::tuple<
    cugraph::experimental::graph_t<vertex_t, edge_t, weight_t, store_transposed, multi_gpu>,
    rmm::device_uvector<vertex_t>>>
create_graph_from_edgelist_impl(
  raft::handle_t const& handle,
  std::optional<std::tuple<vertex_t const*, vertex_t>> optional_local_vertex_span,
  rmm::device_uvector<vertex_t>&& edgelist_rows,
  rmm::device_uvector<vertex_t>&& edgelist_cols,
  rmm::device_uvector<weight_t>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber)
{
  CUGRAPH_EXPECTS(renumber, "renumber should be true if multi_gpu is true.");

  auto& comm               = handle.get_comms();
  auto const comm_size     = comm.get_size();
  auto const comm_rank     = comm.get_rank();
  auto& row_comm           = handle.get_subcomm(cugraph::partition_2d::key_naming_t().row_name());
  auto const row_comm_size = row_comm.get_size();
  auto& col_comm           = handle.get_subcomm(cugraph::partition_2d::key_naming_t().col_name());
  auto const col_comm_size = col_comm.get_size();

  auto local_partition_id_op =
    [comm_size,
     key_func = cugraph::experimental::detail::compute_partition_id_from_edge_t<vertex_t>{
       comm_size, row_comm_size, col_comm_size}] __device__(auto pair) {
      return key_func(thrust::get<0>(pair), thrust::get<1>(pair)) /
             comm_size;  // global partition id to local partition id
    };
  auto pair_first =
    store_transposed
      ? thrust::make_zip_iterator(thrust::make_tuple(edgelist_cols.begin(), edgelist_rows.begin()))
      : thrust::make_zip_iterator(thrust::make_tuple(edgelist_rows.begin(), edgelist_cols.begin()));
  auto edge_counts = graph_properties.is_weighted
                       ? cugraph::experimental::groupby_and_count(pair_first,
                                                                  pair_first + edgelist_rows.size(),
                                                                  edgelist_weights.begin(),
                                                                  local_partition_id_op,
                                                                  col_comm_size,
                                                                  handle.get_stream())
                       : cugraph::experimental::groupby_and_count(pair_first,
                                                                  pair_first + edgelist_rows.size(),
                                                                  local_partition_id_op,
                                                                  col_comm_size,
                                                                  handle.get_stream());

  std::vector<size_t> h_edge_counts(edge_counts.size());
  raft::update_host(
    h_edge_counts.data(), edge_counts.data(), edge_counts.size(), handle.get_stream());
  handle.get_stream_view().synchronize();

  std::vector<size_t> h_displacements(h_edge_counts.size(), size_t{0});
  std::partial_sum(h_edge_counts.begin(), h_edge_counts.end() - 1, h_displacements.begin() + 1);

  // 3. renumber

  rmm::device_uvector<vertex_t> renumber_map_labels(0, handle.get_stream());
  cugraph::experimental::partition_t<vertex_t> partition{};
  vertex_t number_of_vertices{};
  edge_t number_of_edges{};
  {
    std::vector<vertex_t*> major_ptrs(h_edge_counts.size());
    std::vector<vertex_t*> minor_ptrs(major_ptrs.size());
    std::vector<edge_t> counts(major_ptrs.size());
    for (size_t i = 0; i < h_edge_counts.size(); ++i) {
      major_ptrs[i] =
        (store_transposed ? edgelist_cols.begin() : edgelist_rows.begin()) + h_displacements[i];
      minor_ptrs[i] =
        (store_transposed ? edgelist_rows.begin() : edgelist_cols.begin()) + h_displacements[i];
      counts[i] = static_cast<edge_t>(h_edge_counts[i]);
    }
    std::tie(renumber_map_labels, partition, number_of_vertices, number_of_edges) =
      cugraph::experimental::renumber_edgelist<vertex_t, edge_t, multi_gpu>(
        handle, optional_local_vertex_span, major_ptrs, minor_ptrs, counts);
  }

  // 4. create a graph

  std::vector<cugraph::experimental::edgelist_t<vertex_t, edge_t, weight_t>> edgelists(
    h_edge_counts.size());
  for (size_t i = 0; i < h_edge_counts.size(); ++i) {
    edgelists[i] = cugraph::experimental::edgelist_t<vertex_t, edge_t, weight_t>{
      edgelist_rows.data() + h_displacements[i],
      edgelist_cols.data() + h_displacements[i],
      graph_properties.is_weighted ? edgelist_weights.data() + h_displacements[i]
                                   : static_cast<weight_t*>(nullptr),
      static_cast<edge_t>(h_edge_counts[i])};
  }

  return std::make_tuple(
    cugraph::experimental::graph_t<vertex_t, edge_t, weight_t, store_transposed, multi_gpu>(
      handle, edgelists, partition, number_of_vertices, number_of_edges, graph_properties, true),
    std::move(renumber_map_labels));
}

template <typename vertex_t,
          typename edge_t,
          typename weight_t,
          bool store_transposed,
          bool multi_gpu>
std::enable_if_t<
  !multi_gpu,
  std::tuple<
    cugraph::experimental::graph_t<vertex_t, edge_t, weight_t, store_transposed, multi_gpu>,
    rmm::device_uvector<vertex_t>>>
create_graph_from_edgelist_impl(
  raft::handle_t const& handle,
  std::optional<std::tuple<vertex_t const*, vertex_t>> optional_vertex_span,
  rmm::device_uvector<vertex_t>&& edgelist_rows,
  rmm::device_uvector<vertex_t>&& edgelist_cols,
  rmm::device_uvector<weight_t>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber)
{
  auto renumber_map_labels =
    renumber ? cugraph::experimental::renumber_edgelist<vertex_t, edge_t, multi_gpu>(
                 handle,
                 optional_vertex_span,
                 store_transposed ? edgelist_cols.data() : edgelist_rows.data(),
                 store_transposed ? edgelist_rows.data() : edgelist_cols.data(),
                 static_cast<edge_t>(edgelist_rows.size()))
             : rmm::device_uvector<vertex_t>(0, handle.get_stream());
  vertex_t num_vertices{};
  if (renumber) {
    num_vertices = static_cast<vertex_t>(renumber_map_labels.size());
  } else {
    if (optional_vertex_span) {
      num_vertices = std::get<1>(*optional_vertex_span);
    } else {
      auto edge_first =
        thrust::make_zip_iterator(thrust::make_tuple(edgelist_rows.begin(), edgelist_cols.begin()));
      num_vertices =
        thrust::transform_reduce(
          rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
          edge_first,
          edge_first + edgelist_rows.size(),
          [] __device__(auto e) { return std::max(thrust::get<0>(e), thrust::get<1>(e)); },
          vertex_t{0},
          thrust::maximum<vertex_t>()) +
        1;
    }
  }

  return std::make_tuple(
    cugraph::experimental::graph_t<vertex_t, edge_t, weight_t, store_transposed, multi_gpu>(
      handle,
      cugraph::experimental::edgelist_t<vertex_t, edge_t, weight_t>{
        edgelist_rows.data(),
        edgelist_cols.data(),
        graph_properties.is_weighted ? edgelist_weights.data() : static_cast<weight_t*>(nullptr),
        static_cast<edge_t>(edgelist_rows.size())},
      num_vertices,
      graph_properties,
      renumber ? true : false),
    std::move(renumber_map_labels));
}

}  // namespace

template <typename vertex_t,
          typename edge_t,
          typename weight_t,
          bool store_transposed,
          bool multi_gpu>
std::tuple<cugraph::experimental::graph_t<vertex_t, edge_t, weight_t, store_transposed, multi_gpu>,
           rmm::device_uvector<vertex_t>>
create_graph_from_edgelist(
  raft::handle_t const& handle,
  std::optional<std::tuple<vertex_t const*, vertex_t>> optional_vertex_span,
  rmm::device_uvector<vertex_t>&& edgelist_rows,
  rmm::device_uvector<vertex_t>&& edgelist_cols,
  rmm::device_uvector<weight_t>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber)
{
  return create_graph_from_edgelist_impl<vertex_t, edge_t, weight_t, store_transposed, multi_gpu>(
    handle,
    optional_vertex_span,
    std::move(edgelist_rows),
    std::move(edgelist_cols),
    std::move(edgelist_weights),
    graph_properties,
    renumber);
}

// explicit instantiations

template std::tuple<cugraph::experimental::graph_t<int32_t, int32_t, float, false, false>,
                    rmm::device_uvector<int32_t>>
create_graph_from_edgelist<int32_t, int32_t, float, false, false>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int32_t const*, int32_t>> optional_vertex_span,
  rmm::device_uvector<int32_t>&& edgelist_rows,
  rmm::device_uvector<int32_t>&& edgelist_cols,
  rmm::device_uvector<float>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int32_t, int32_t, float, false, true>,
                    rmm::device_uvector<int32_t>>
create_graph_from_edgelist<int32_t, int32_t, float, false, true>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int32_t const*, int32_t>> optional_vertex_span,
  rmm::device_uvector<int32_t>&& edgelist_rows,
  rmm::device_uvector<int32_t>&& edgelist_cols,
  rmm::device_uvector<float>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int32_t, int32_t, float, true, false>,
                    rmm::device_uvector<int32_t>>
create_graph_from_edgelist<int32_t, int32_t, float, true, false>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int32_t const*, int32_t>> optional_vertex_span,
  rmm::device_uvector<int32_t>&& edgelist_rows,
  rmm::device_uvector<int32_t>&& edgelist_cols,
  rmm::device_uvector<float>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int32_t, int32_t, float, true, true>,
                    rmm::device_uvector<int32_t>>
create_graph_from_edgelist<int32_t, int32_t, float, true, true>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int32_t const*, int32_t>> optional_vertex_span,
  rmm::device_uvector<int32_t>&& edgelist_rows,
  rmm::device_uvector<int32_t>&& edgelist_cols,
  rmm::device_uvector<float>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int32_t, int32_t, double, false, false>,
                    rmm::device_uvector<int32_t>>
create_graph_from_edgelist<int32_t, int32_t, double, false, false>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int32_t const*, int32_t>> optional_vertex_span,
  rmm::device_uvector<int32_t>&& edgelist_rows,
  rmm::device_uvector<int32_t>&& edgelist_cols,
  rmm::device_uvector<double>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int32_t, int32_t, double, false, true>,
                    rmm::device_uvector<int32_t>>
create_graph_from_edgelist<int32_t, int32_t, double, false, true>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int32_t const*, int32_t>> optional_vertex_span,
  rmm::device_uvector<int32_t>&& edgelist_rows,
  rmm::device_uvector<int32_t>&& edgelist_cols,
  rmm::device_uvector<double>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int32_t, int32_t, double, true, false>,
                    rmm::device_uvector<int32_t>>
create_graph_from_edgelist<int32_t, int32_t, double, true, false>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int32_t const*, int32_t>> optional_vertex_span,
  rmm::device_uvector<int32_t>&& edgelist_rows,
  rmm::device_uvector<int32_t>&& edgelist_cols,
  rmm::device_uvector<double>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int32_t, int32_t, double, true, true>,
                    rmm::device_uvector<int32_t>>
create_graph_from_edgelist<int32_t, int32_t, double, true, true>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int32_t const*, int32_t>> optional_vertex_span,
  rmm::device_uvector<int32_t>&& edgelist_rows,
  rmm::device_uvector<int32_t>&& edgelist_cols,
  rmm::device_uvector<double>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int32_t, int64_t, float, false, false>,
                    rmm::device_uvector<int32_t>>
create_graph_from_edgelist<int32_t, int64_t, float, false, false>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int32_t const*, int32_t>> optional_vertex_span,
  rmm::device_uvector<int32_t>&& edgelist_rows,
  rmm::device_uvector<int32_t>&& edgelist_cols,
  rmm::device_uvector<float>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int32_t, int64_t, float, false, true>,
                    rmm::device_uvector<int32_t>>
create_graph_from_edgelist<int32_t, int64_t, float, false, true>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int32_t const*, int32_t>> optional_vertex_span,
  rmm::device_uvector<int32_t>&& edgelist_rows,
  rmm::device_uvector<int32_t>&& edgelist_cols,
  rmm::device_uvector<float>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int32_t, int64_t, float, true, false>,
                    rmm::device_uvector<int32_t>>
create_graph_from_edgelist<int32_t, int64_t, float, true, false>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int32_t const*, int32_t>> optional_vertex_span,
  rmm::device_uvector<int32_t>&& edgelist_rows,
  rmm::device_uvector<int32_t>&& edgelist_cols,
  rmm::device_uvector<float>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int32_t, int64_t, float, true, true>,
                    rmm::device_uvector<int32_t>>
create_graph_from_edgelist<int32_t, int64_t, float, true, true>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int32_t const*, int32_t>> optional_vertex_span,
  rmm::device_uvector<int32_t>&& edgelist_rows,
  rmm::device_uvector<int32_t>&& edgelist_cols,
  rmm::device_uvector<float>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int32_t, int64_t, double, false, false>,
                    rmm::device_uvector<int32_t>>
create_graph_from_edgelist<int32_t, int64_t, double, false, false>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int32_t const*, int32_t>> optional_vertex_span,
  rmm::device_uvector<int32_t>&& edgelist_rows,
  rmm::device_uvector<int32_t>&& edgelist_cols,
  rmm::device_uvector<double>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int32_t, int64_t, double, false, true>,
                    rmm::device_uvector<int32_t>>
create_graph_from_edgelist<int32_t, int64_t, double, false, true>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int32_t const*, int32_t>> optional_vertex_span,
  rmm::device_uvector<int32_t>&& edgelist_rows,
  rmm::device_uvector<int32_t>&& edgelist_cols,
  rmm::device_uvector<double>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int32_t, int64_t, double, true, false>,
                    rmm::device_uvector<int32_t>>
create_graph_from_edgelist<int32_t, int64_t, double, true, false>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int32_t const*, int32_t>> optional_vertex_span,
  rmm::device_uvector<int32_t>&& edgelist_rows,
  rmm::device_uvector<int32_t>&& edgelist_cols,
  rmm::device_uvector<double>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int32_t, int64_t, double, true, true>,
                    rmm::device_uvector<int32_t>>
create_graph_from_edgelist<int32_t, int64_t, double, true, true>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int32_t const*, int32_t>> optional_vertex_span,
  rmm::device_uvector<int32_t>&& edgelist_rows,
  rmm::device_uvector<int32_t>&& edgelist_cols,
  rmm::device_uvector<double>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int64_t, int64_t, float, false, false>,
                    rmm::device_uvector<int64_t>>
create_graph_from_edgelist<int64_t, int64_t, float, false, false>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int64_t const*, int64_t>> optional_vertex_span,
  rmm::device_uvector<int64_t>&& edgelist_rows,
  rmm::device_uvector<int64_t>&& edgelist_cols,
  rmm::device_uvector<float>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int64_t, int64_t, float, false, true>,
                    rmm::device_uvector<int64_t>>
create_graph_from_edgelist<int64_t, int64_t, float, false, true>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int64_t const*, int64_t>> optional_vertex_span,
  rmm::device_uvector<int64_t>&& edgelist_rows,
  rmm::device_uvector<int64_t>&& edgelist_cols,
  rmm::device_uvector<float>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int64_t, int64_t, float, true, false>,
                    rmm::device_uvector<int64_t>>
create_graph_from_edgelist<int64_t, int64_t, float, true, false>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int64_t const*, int64_t>> optional_vertex_span,
  rmm::device_uvector<int64_t>&& edgelist_rows,
  rmm::device_uvector<int64_t>&& edgelist_cols,
  rmm::device_uvector<float>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int64_t, int64_t, float, true, true>,
                    rmm::device_uvector<int64_t>>
create_graph_from_edgelist<int64_t, int64_t, float, true, true>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int64_t const*, int64_t>> optional_vertex_span,
  rmm::device_uvector<int64_t>&& edgelist_rows,
  rmm::device_uvector<int64_t>&& edgelist_cols,
  rmm::device_uvector<float>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int64_t, int64_t, double, false, false>,
                    rmm::device_uvector<int64_t>>
create_graph_from_edgelist<int64_t, int64_t, double, false, false>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int64_t const*, int64_t>> optional_vertex_span,
  rmm::device_uvector<int64_t>&& edgelist_rows,
  rmm::device_uvector<int64_t>&& edgelist_cols,
  rmm::device_uvector<double>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int64_t, int64_t, double, false, true>,
                    rmm::device_uvector<int64_t>>
create_graph_from_edgelist<int64_t, int64_t, double, false, true>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int64_t const*, int64_t>> optional_vertex_span,
  rmm::device_uvector<int64_t>&& edgelist_rows,
  rmm::device_uvector<int64_t>&& edgelist_cols,
  rmm::device_uvector<double>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int64_t, int64_t, double, true, false>,
                    rmm::device_uvector<int64_t>>
create_graph_from_edgelist<int64_t, int64_t, double, true, false>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int64_t const*, int64_t>> optional_vertex_span,
  rmm::device_uvector<int64_t>&& edgelist_rows,
  rmm::device_uvector<int64_t>&& edgelist_cols,
  rmm::device_uvector<double>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

template std::tuple<cugraph::experimental::graph_t<int64_t, int64_t, double, true, true>,
                    rmm::device_uvector<int64_t>>
create_graph_from_edgelist<int64_t, int64_t, double, true, true>(
  raft::handle_t const& handle,
  std::optional<std::tuple<int64_t const*, int64_t>> optional_vertex_span,
  rmm::device_uvector<int64_t>&& edgelist_rows,
  rmm::device_uvector<int64_t>&& edgelist_cols,
  rmm::device_uvector<double>&& edgelist_weights,
  graph_properties_t graph_properties,
  bool renumber);

}  // namespace experimental
}  // namespace cugraph
