#include "hip/hip_runtime.h"
// -*-c++-*-

/*
 * Copyright (c) 2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 */

// Graph analytics features

#include <cugraph.h>
#include <hip/library_types.h>
#include <nvgraph/nvgraph.h>
#include <rmm_utils.h>
#include <thrust/device_vector.h>
#include <utilities/validation.cuh>
#include "converters/COOtoCSR.cuh"
#include "converters/renumber.cuh"
#include "utilities/cusparse_helper.h"
#include "utilities/error_utils.h"
#include "utilities/graph_utils.cuh"
/*
 * cudf has gdf_column_free and using this is, in general, better design than
 * creating our own, but we will keep this as cudf is planning to remove the
 * function. cudf plans to redesign cudf::column to fundamentally solve this
 * problem, so once they finished the redesign, we need to update this code to
 * use their new features. Until that time, we may rely on this as a temporary
 * solution.
 */

namespace cugraph {
int get_device(const void *ptr)
{
  hipPointerAttribute_t att;
  hipPointerGetAttributes(&att, ptr);
  return att.device;
}

void gdf_col_delete(gdf_column *col)
{
  if (col != nullptr) {
    hipStream_t stream{nullptr};
    if (col->data != nullptr) { ALLOC_FREE_TRY(col->data, stream); }
    if (col->valid != nullptr) { ALLOC_FREE_TRY(col->valid, stream); }
#if 0
    /* Currently, gdf_column_view does not set col_name, and col_name can have
        an arbitrary value, so freeing col_name can lead to freeing a ranodom
        address. This problem should be cleaned up once cudf finishes
        redesigning cudf::column. */
    if (col->col_name != nullptr) {
      free(col->col_name);
    }
#endif
    delete col;
  }
}

void gdf_col_release(gdf_column *col) { delete col; }

void cpy_column_view(const gdf_column *in, gdf_column *out)
{
  if (in != nullptr && out != nullptr) {
    gdf_column_view(out, in->data, in->valid, in->size, in->dtype);
  }
}

void transposed_adj_list_view(Graph *graph,
                              const gdf_column *offsets,
                              const gdf_column *indices,
                              const gdf_column *edge_data)
{
  // This function returns an error if this graph object has at least one graph
  // representation to prevent a single object storing two different graphs.
  CUGRAPH_EXPECTS(((graph->edgeList == nullptr) && (graph->adjList == nullptr) &&
                   (graph->transposedAdjList == nullptr)),
                  "Invalid API parameter: Graph data is NULL");

  CUGRAPH_EXPECTS(offsets->null_count == 0,
                  "Input column has non-zero null count: offsets->null_count is 0");
  CUGRAPH_EXPECTS(indices->null_count == 0,
                  "Input column has non-zero null count: indices->null_count is 0");
  CUGRAPH_EXPECTS((offsets->dtype == indices->dtype),
                  "Unsupported data type: graph data type mismatch");
  CUGRAPH_EXPECTS(((offsets->dtype == GDF_INT32)),
                  "Unsupported data type: graph is of wrong data type");
  CUGRAPH_EXPECTS((offsets->size > 0), "Column is empty");

  graph->transposedAdjList            = new gdf_adj_list;
  graph->transposedAdjList->offsets   = new gdf_column;
  graph->transposedAdjList->indices   = new gdf_column;
  graph->transposedAdjList->ownership = 0;

  cpy_column_view(offsets, graph->transposedAdjList->offsets);
  cpy_column_view(indices, graph->transposedAdjList->indices);

  if (!graph->prop) graph->prop = new Graph_properties();

  if (edge_data) {
    CUGRAPH_EXPECTS(indices->size == edge_data->size, "Column size mismatch");
    graph->transposedAdjList->edge_data = new gdf_column;
    cpy_column_view(edge_data, graph->transposedAdjList->edge_data);

    bool has_neg_val;

    switch (graph->adjList->edge_data->dtype) {
      case GDF_INT8:
        has_neg_val = cugraph::detail::has_negative_val(
          static_cast<int8_t *>(graph->transposedAdjList->edge_data->data),
          graph->transposedAdjList->edge_data->size);
        break;
      case GDF_INT16:
        has_neg_val = cugraph::detail::has_negative_val(
          static_cast<int16_t *>(graph->transposedAdjList->edge_data->data),
          graph->transposedAdjList->edge_data->size);
        break;
      case GDF_INT32:
        has_neg_val = cugraph::detail::has_negative_val(
          static_cast<int32_t *>(graph->transposedAdjList->edge_data->data),
          graph->transposedAdjList->edge_data->size);
        break;
      case GDF_INT64:
        has_neg_val = cugraph::detail::has_negative_val(
          static_cast<int64_t *>(graph->transposedAdjList->edge_data->data),
          graph->transposedAdjList->edge_data->size);
        break;
      case GDF_FLOAT32:
        has_neg_val = cugraph::detail::has_negative_val(
          static_cast<float *>(graph->transposedAdjList->edge_data->data),
          graph->transposedAdjList->edge_data->size);
        break;
      case GDF_FLOAT64:
        has_neg_val = cugraph::detail::has_negative_val(
          static_cast<double *>(graph->transposedAdjList->edge_data->data),
          graph->transposedAdjList->edge_data->size);
        break;
      default: has_neg_val = false;
    }
    graph->prop->has_negative_edges = (has_neg_val) ? GDF_PROP_TRUE : GDF_PROP_FALSE;
  } else {
    graph->transposedAdjList->edge_data = nullptr;
    graph->prop->has_negative_edges     = GDF_PROP_FALSE;
  }

  graph->numberOfVertices = graph->transposedAdjList->offsets->size - 1;
}

void adj_list_view(Graph *graph,
                   const gdf_column *offsets,
                   const gdf_column *indices,
                   const gdf_column *edge_data)
{
  // This function returns an error if this graph object has at least one graph
  // representation to prevent a single object storing two different graphs.
  CUGRAPH_EXPECTS(((graph->edgeList == nullptr) && (graph->adjList == nullptr) &&
                   (graph->transposedAdjList == nullptr)),
                  "Invalid API parameter: graph data is NULL");
  CUGRAPH_EXPECTS(offsets->null_count == 0, "Input column has non-zero null count");
  CUGRAPH_EXPECTS(indices->null_count == 0, "Input column has non-zero null count");
  CUGRAPH_EXPECTS((offsets->dtype == indices->dtype), "Unsupported data type");
  CUGRAPH_EXPECTS(((offsets->dtype == GDF_INT32)), "Unsupported data type");
  CUGRAPH_EXPECTS((offsets->size > 0), "Column is empty");

  graph->adjList            = new gdf_adj_list;
  graph->adjList->offsets   = new gdf_column;
  graph->adjList->indices   = new gdf_column;
  graph->adjList->ownership = 0;

  cpy_column_view(offsets, graph->adjList->offsets);
  cpy_column_view(indices, graph->adjList->indices);

  if (!graph->prop) graph->prop = new Graph_properties();

  if (edge_data) {
    CUGRAPH_EXPECTS(indices->size == edge_data->size, "Column size mismatch");
    graph->adjList->edge_data = new gdf_column;
    cpy_column_view(edge_data, graph->adjList->edge_data);

    bool has_neg_val;

    switch (graph->adjList->edge_data->dtype) {
      case GDF_INT8:
        has_neg_val = cugraph::detail::has_negative_val(
          static_cast<int8_t *>(graph->adjList->edge_data->data), graph->adjList->edge_data->size);
        break;
      case GDF_INT16:
        has_neg_val = cugraph::detail::has_negative_val(
          static_cast<int16_t *>(graph->adjList->edge_data->data), graph->adjList->edge_data->size);
        break;
      case GDF_INT32:
        has_neg_val = cugraph::detail::has_negative_val(
          static_cast<int32_t *>(graph->adjList->edge_data->data), graph->adjList->edge_data->size);
        break;
      case GDF_INT64:
        has_neg_val = cugraph::detail::has_negative_val(
          static_cast<int64_t *>(graph->adjList->edge_data->data), graph->adjList->edge_data->size);
        break;
      case GDF_FLOAT32:
        has_neg_val = cugraph::detail::has_negative_val(
          static_cast<float *>(graph->adjList->edge_data->data), graph->adjList->edge_data->size);
        break;
      case GDF_FLOAT64:
        has_neg_val = cugraph::detail::has_negative_val(
          static_cast<double *>(graph->adjList->edge_data->data), graph->adjList->edge_data->size);
        break;
      default: has_neg_val = false;
    }
    graph->prop->has_negative_edges = (has_neg_val) ? GDF_PROP_TRUE : GDF_PROP_FALSE;
  } else {
    graph->adjList->edge_data       = nullptr;
    graph->prop->has_negative_edges = GDF_PROP_FALSE;
  }

  graph->numberOfVertices = graph->adjList->offsets->size - 1;
}

void gdf_adj_list::get_vertex_identifiers(gdf_column *identifiers)
{
  CUGRAPH_EXPECTS(offsets != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(offsets->data != nullptr, "Invalid API parameter");
  cugraph::detail::sequence<int>((int)offsets->size - 1, (int *)identifiers->data);
}

void gdf_adj_list::get_source_indices(gdf_column *src_indices)
{
  CUGRAPH_EXPECTS(offsets != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(offsets->data != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(src_indices->size == indices->size, "Column size mismatch");
  CUGRAPH_EXPECTS(src_indices->dtype == indices->dtype, "Unsupported data type");
  CUGRAPH_EXPECTS(src_indices->size > 0, "Column is empty");

  cugraph::detail::offsets_to_indices<int>(
    (int *)offsets->data, offsets->size - 1, (int *)src_indices->data);
}

void edge_list_view(Graph *graph,
                    const gdf_column *src_indices,
                    const gdf_column *dest_indices,
                    const gdf_column *edge_data)
{
  // This function returns an error if this graph object has at least one graph
  // representation to prevent a single object storing two different graphs.

  CUGRAPH_EXPECTS(((graph->edgeList == nullptr) && (graph->adjList == nullptr) &&
                   (graph->transposedAdjList == nullptr)),
                  "Invalid API parameter");
  CUGRAPH_EXPECTS(src_indices->size == dest_indices->size, "Column size mismatch");
  CUGRAPH_EXPECTS(src_indices->dtype == dest_indices->dtype, "Unsupported data type");
  CUGRAPH_EXPECTS(src_indices->dtype == GDF_INT32, "Unsupported data type");
  CUGRAPH_EXPECTS(src_indices->size > 0, "Column is empty");
  CUGRAPH_EXPECTS(src_indices->null_count == 0, "Input column has non-zero null count");
  CUGRAPH_EXPECTS(dest_indices->null_count == 0, "Input column has non-zero null count");

  graph->edgeList               = new gdf_edge_list;
  graph->edgeList->src_indices  = new gdf_column;
  graph->edgeList->dest_indices = new gdf_column;
  graph->edgeList->ownership    = 0;

  cpy_column_view(src_indices, graph->edgeList->src_indices);
  cpy_column_view(dest_indices, graph->edgeList->dest_indices);

  if (!graph->prop) graph->prop = new Graph_properties();

  if (edge_data) {
    CUGRAPH_EXPECTS(src_indices->size == edge_data->size, "Column size mismatch");
    graph->edgeList->edge_data = new gdf_column;
    cpy_column_view(edge_data, graph->edgeList->edge_data);

    bool has_neg_val;

    switch (graph->edgeList->edge_data->dtype) {
      case GDF_INT8:
        has_neg_val =
          cugraph::detail::has_negative_val(static_cast<int8_t *>(graph->edgeList->edge_data->data),
                                            graph->edgeList->edge_data->size);
        break;
      case GDF_INT16:
        has_neg_val = cugraph::detail::has_negative_val(
          static_cast<int16_t *>(graph->edgeList->edge_data->data),
          graph->edgeList->edge_data->size);
        break;
      case GDF_INT32:
        has_neg_val = cugraph::detail::has_negative_val(
          static_cast<int32_t *>(graph->edgeList->edge_data->data),
          graph->edgeList->edge_data->size);
        break;
      case GDF_INT64:
        has_neg_val = cugraph::detail::has_negative_val(
          static_cast<int64_t *>(graph->edgeList->edge_data->data),
          graph->edgeList->edge_data->size);
        break;
      case GDF_FLOAT32:
        has_neg_val = cugraph::detail::has_negative_val(
          static_cast<float *>(graph->edgeList->edge_data->data), graph->edgeList->edge_data->size);
        break;
      case GDF_FLOAT64:
        has_neg_val =
          cugraph::detail::has_negative_val(static_cast<double *>(graph->edgeList->edge_data->data),
                                            graph->edgeList->edge_data->size);
        break;
      default: has_neg_val = false;
    }
    graph->prop->has_negative_edges = (has_neg_val) ? GDF_PROP_TRUE : GDF_PROP_FALSE;

  } else {
    graph->edgeList->edge_data      = nullptr;
    graph->prop->has_negative_edges = GDF_PROP_FALSE;
  }

  cugraph::detail::indexing_check<int>(static_cast<int *>(graph->edgeList->src_indices->data),
                                       static_cast<int *>(graph->edgeList->dest_indices->data),
                                       graph->edgeList->dest_indices->size);
}

template <typename T, typename WT>
void add_adj_list_impl(Graph *graph)
{
  if (graph->adjList == nullptr) {
    CUGRAPH_EXPECTS(graph->edgeList != nullptr, "Invalid API parameter");
    int nnz                   = graph->edgeList->src_indices->size;
    graph->adjList            = new gdf_adj_list;
    graph->adjList->offsets   = new gdf_column;
    graph->adjList->indices   = new gdf_column;
    graph->adjList->ownership = 1;

    if (graph->edgeList->edge_data != nullptr) {
      graph->adjList->edge_data = new gdf_column;

      CSR_Result_Weighted<int32_t, WT> adj_list;
      ConvertCOOtoCSR_weighted((int *)graph->edgeList->src_indices->data,
                               (int *)graph->edgeList->dest_indices->data,
                               (WT *)graph->edgeList->edge_data->data,
                               nnz,
                               adj_list);

      gdf_column_view(graph->adjList->offsets,
                      adj_list.rowOffsets,
                      nullptr,
                      adj_list.size + 1,
                      graph->edgeList->src_indices->dtype);
      gdf_column_view(graph->adjList->indices,
                      adj_list.colIndices,
                      nullptr,
                      adj_list.nnz,
                      graph->edgeList->src_indices->dtype);
      gdf_column_view(graph->adjList->edge_data,
                      adj_list.edgeWeights,
                      nullptr,
                      adj_list.nnz,
                      graph->edgeList->edge_data->dtype);
    } else {
      CSR_Result<int> adj_list;
      ConvertCOOtoCSR((int *)graph->edgeList->src_indices->data,
                      (int *)graph->edgeList->dest_indices->data,
                      nnz,
                      adj_list);
      gdf_column_view(graph->adjList->offsets,
                      adj_list.rowOffsets,
                      nullptr,
                      adj_list.size + 1,
                      graph->edgeList->src_indices->dtype);
      gdf_column_view(graph->adjList->indices,
                      adj_list.colIndices,
                      nullptr,
                      adj_list.nnz,
                      graph->edgeList->src_indices->dtype);
    }
    graph->numberOfVertices = graph->adjList->offsets->size - 1;
  }
}

void add_edge_list(Graph *graph)
{
  if (graph->edgeList == nullptr) {
    CUGRAPH_EXPECTS(graph->adjList != nullptr, "Invalid API parameter");
    int *d_src;
    graph->edgeList               = new gdf_edge_list;
    graph->edgeList->src_indices  = new gdf_column;
    graph->edgeList->dest_indices = new gdf_column;
    graph->edgeList->ownership    = 2;

    hipStream_t stream{nullptr};
    ALLOC_TRY((void **)&d_src, sizeof(int) * graph->adjList->indices->size, stream);

    cugraph::detail::offsets_to_indices<int>(
      (int *)graph->adjList->offsets->data, graph->adjList->offsets->size - 1, (int *)d_src);

    gdf_column_view(graph->edgeList->src_indices,
                    d_src,
                    nullptr,
                    graph->adjList->indices->size,
                    graph->adjList->indices->dtype);
    cpy_column_view(graph->adjList->indices, graph->edgeList->dest_indices);

    if (graph->adjList->edge_data != nullptr) {
      graph->edgeList->edge_data = new gdf_column;
      cpy_column_view(graph->adjList->edge_data, graph->edgeList->edge_data);
    }
  }
}

template <typename WT>
void add_transposed_adj_list_impl(Graph *graph)
{
  if (graph->transposedAdjList == nullptr) {
    CUGRAPH_EXPECTS(graph->edgeList != nullptr, "Invalid API parameter");
    int nnz                             = graph->edgeList->src_indices->size;
    graph->transposedAdjList            = new gdf_adj_list;
    graph->transposedAdjList->offsets   = new gdf_column;
    graph->transposedAdjList->indices   = new gdf_column;
    graph->transposedAdjList->ownership = 1;

    if (graph->edgeList->edge_data) {
      graph->transposedAdjList->edge_data = new gdf_column;
      CSR_Result_Weighted<int32_t, WT> adj_list;
      ConvertCOOtoCSR_weighted((int *)graph->edgeList->dest_indices->data,
                               (int *)graph->edgeList->src_indices->data,
                               (WT *)graph->edgeList->edge_data->data,
                               nnz,
                               adj_list);
      gdf_column_view(graph->transposedAdjList->offsets,
                      adj_list.rowOffsets,
                      nullptr,
                      adj_list.size + 1,
                      graph->edgeList->src_indices->dtype);
      gdf_column_view(graph->transposedAdjList->indices,
                      adj_list.colIndices,
                      nullptr,
                      adj_list.nnz,
                      graph->edgeList->src_indices->dtype);
      gdf_column_view(graph->transposedAdjList->edge_data,
                      adj_list.edgeWeights,
                      nullptr,
                      adj_list.nnz,
                      graph->edgeList->edge_data->dtype);
    } else {
      CSR_Result<int> adj_list;
      ConvertCOOtoCSR((int *)graph->edgeList->dest_indices->data,
                      (int *)graph->edgeList->src_indices->data,
                      nnz,
                      adj_list);
      gdf_column_view(graph->transposedAdjList->offsets,
                      adj_list.rowOffsets,
                      nullptr,
                      adj_list.size + 1,
                      graph->edgeList->src_indices->dtype);
      gdf_column_view(graph->transposedAdjList->indices,
                      adj_list.colIndices,
                      nullptr,
                      adj_list.nnz,
                      graph->edgeList->src_indices->dtype);
    }
    graph->numberOfVertices = graph->transposedAdjList->offsets->size - 1;
  }
}

void add_adj_list(Graph *graph)
{
  if (graph->adjList == nullptr) {
    CUGRAPH_EXPECTS(graph->edgeList != nullptr, "Invalid API parameter");
    CUGRAPH_EXPECTS(graph->edgeList->src_indices->dtype == GDF_INT32, "Unsupported data type");

    if (graph->edgeList->edge_data != nullptr) {
      switch (graph->edgeList->edge_data->dtype) {
        case GDF_FLOAT32: return cugraph::add_adj_list_impl<int32_t, float>(graph);
        case GDF_FLOAT64: return cugraph::add_adj_list_impl<int32_t, double>(graph);
        default: CUGRAPH_FAIL("Unsupported data type");
      }
    } else {
      return cugraph::add_adj_list_impl<int32_t, float>(graph);
    }
  }
}

void add_transposed_adj_list(Graph *graph)
{
  if (graph->transposedAdjList == nullptr) {
    if (graph->edgeList == nullptr) cugraph::add_edge_list(graph);

    CUGRAPH_EXPECTS(graph->edgeList->src_indices->dtype == GDF_INT32, "Unsupported data type");
    CUGRAPH_EXPECTS(graph->edgeList->dest_indices->dtype == GDF_INT32, "Unsupported data type");

    if (graph->edgeList->edge_data != nullptr) {
      switch (graph->edgeList->edge_data->dtype) {
        case GDF_FLOAT32: return cugraph::add_transposed_adj_list_impl<float>(graph);
        case GDF_FLOAT64: return cugraph::add_transposed_adj_list_impl<double>(graph);
        default: CUGRAPH_FAIL("Unsupported data type");
      }
    } else {
      return cugraph::add_transposed_adj_list_impl<float>(graph);
    }
  }
}

void delete_adj_list(Graph *graph)
{
  if (graph->adjList) { delete graph->adjList; }
  graph->adjList = nullptr;
}

void delete_edge_list(Graph *graph)
{
  if (graph->edgeList) { delete graph->edgeList; }
  graph->edgeList = nullptr;
}

void delete_transposed_adj_list(Graph *graph)
{
  if (graph->transposedAdjList) { delete graph->transposedAdjList; }
  graph->transposedAdjList = nullptr;
}

void number_of_vertices(Graph *graph)
{
  if (graph->numberOfVertices != 0)

    //
    //  int32_t implementation for now, since that's all that
    //  is supported elsewhere.
    //
    CUGRAPH_EXPECTS((graph->edgeList != nullptr), "Invalid API parameter");
  CUGRAPH_EXPECTS((graph->edgeList->src_indices->dtype == GDF_INT32), "Unsupported data type");

  int32_t h_max[2];
  int32_t *d_max;
  void *d_temp_storage      = nullptr;
  size_t temp_storage_bytes = 0;

  ALLOC_TRY(&d_max, sizeof(int32_t), nullptr);

  //
  //  Compute size of temp storage
  //
  int32_t *tmp = static_cast<int32_t *>(graph->edgeList->src_indices->data);

  hipcub::DeviceReduce::Max(
    d_temp_storage, temp_storage_bytes, tmp, d_max, graph->edgeList->src_indices->size);

  //
  //  Compute max of src indices and copy to host
  //
  ALLOC_TRY(&d_temp_storage, temp_storage_bytes, nullptr);
  hipcub::DeviceReduce::Max(
    d_temp_storage, temp_storage_bytes, tmp, d_max, graph->edgeList->src_indices->size);

  CUDA_TRY(hipMemcpy(h_max, d_max, sizeof(int32_t), hipMemcpyDeviceToHost));

  //
  //  Compute max of dest indices and copy to host
  //
  tmp = static_cast<int32_t *>(graph->edgeList->dest_indices->data);
  hipcub::DeviceReduce::Max(
    d_temp_storage, temp_storage_bytes, tmp, d_max, graph->edgeList->src_indices->size);
  CUDA_TRY(hipMemcpy(h_max + 1, d_max, sizeof(int32_t), hipMemcpyDeviceToHost));

  ALLOC_FREE_TRY(d_temp_storage, nullptr);
  ALLOC_FREE_TRY(d_max, nullptr);

  graph->numberOfVertices = 1 + std::max(h_max[0], h_max[1]);
}

}  // namespace cugraph
