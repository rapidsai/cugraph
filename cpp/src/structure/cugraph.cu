#include "hip/hip_runtime.h"
// -*-c++-*-

 /*
 * Copyright (c) 2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 */

// Graph analytics features

#include <cugraph.h>
#include "utilities/graph_utils.cuh"
#include "converters/COOtoCSR.cuh"
#include "utilities/error_utils.h"
#include "converters/renumber.cuh"
#include <hip/library_types.h>
#include <nvgraph/nvgraph.h>
#include <thrust/device_vector.h>
#include "utilities/cusparse_helper.h"
#include <rmm_utils.h>
#include <utilities/validation.cuh>
/*
 * cudf has gdf_column_free and using this is, in general, better design than
 * creating our own, but we will keep this as cudf is planning to remove the
 * function. cudf plans to redesign cudf::column to fundamentally solve this
 * problem, so once they finished the redesign, we need to update this code to
 * use their new features. Until that time, we may rely on this as a temporary
 * solution.
 */

int get_device(const void *ptr) {
    hipPointerAttribute_t att;
    hipPointerGetAttributes(&att, ptr);
    return att.device;
}

void gdf_col_delete(gdf_column* col) {
  if (col != nullptr) {
    hipStream_t stream {nullptr};
    if (col->data != nullptr) {
      ALLOC_FREE_TRY(col->data, stream);
    }
    if (col->valid != nullptr) {
      ALLOC_FREE_TRY(col->valid, stream);
    }
#if 0
    /* Currently, gdf_column_view does not set col_name, and col_name can have
        an arbitrary value, so freeing col_name can lead to freeing a ranodom
        address. This problem should be cleaned up once cudf finishes
        redesigning cudf::column. */
    if (col->col_name != nullptr) {
      free(col->col_name);
    }
#endif
    delete col;
  }
}

void gdf_col_release(gdf_column* col) {
  delete col;
}

void cpy_column_view(const gdf_column *in, gdf_column *out) {
  if (in != nullptr && out !=nullptr) {
    gdf_column_view(out, in->data, in->valid, in->size, in->dtype);
  }
}

gdf_error gdf_adj_list_view(gdf_graph *graph, const gdf_column *offsets,
                            const gdf_column *indices,
                            const gdf_column *edge_data) {
  //This function returns an error if this graph object has at least one graph
  //representation to prevent a single object storing two different graphs.
  GDF_REQUIRE( ((graph->edgeList == nullptr) && (graph->adjList == nullptr) &&
    (graph->transposedAdjList == nullptr)), GDF_INVALID_API_CALL);
  /*GDF_REQUIRE( offsets->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( (offsets->dtype == indices->dtype), GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( ((offsets->dtype == GDF_INT32)), GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( (offsets->size > 0), GDF_DATASET_EMPTY );
  */

  graph->adjList = new gdf_adj_list;
  graph->adjList->offsets = new gdf_column;
  graph->adjList->indices = new gdf_column;

  cpy_column_view(offsets, graph->adjList->offsets);
  cpy_column_view(indices, graph->adjList->indices);
  
  if (!graph->prop)
      graph->prop = new gdf_graph_properties();

  if (edge_data) {
    GDF_REQUIRE(indices->size == edge_data->size, GDF_COLUMN_SIZE_MISMATCH);
    graph->adjList->edge_data = new gdf_column;
    cpy_column_view(edge_data, graph->adjList->edge_data);
    
    bool has_neg_val;
    
    switch (graph->adjList->edge_data->dtype) {
    case GDF_INT8:
      has_neg_val = cugraph::has_negative_val(
          static_cast<int8_t *>(graph->adjList->edge_data->data),
          graph->adjList->edge_data->size);
      break;
    case GDF_INT16:
      has_neg_val = cugraph::has_negative_val(
          static_cast<int16_t *>(graph->adjList->edge_data->data),
          graph->adjList->edge_data->size);
      break;
    case GDF_INT32:
      has_neg_val = cugraph::has_negative_val(
          static_cast<int32_t *>(graph->adjList->edge_data->data),
          graph->adjList->edge_data->size);
      break;
    case GDF_INT64:
      has_neg_val = cugraph::has_negative_val(
          static_cast<int64_t *>(graph->adjList->edge_data->data),
          graph->adjList->edge_data->size);
      break;
    case GDF_FLOAT32:
      has_neg_val = cugraph::has_negative_val(
          static_cast<float *>(graph->adjList->edge_data->data),
          graph->adjList->edge_data->size);
      break;
    case GDF_FLOAT64:
      has_neg_val = cugraph::has_negative_val(
          static_cast<double *>(graph->adjList->edge_data->data),
          graph->adjList->edge_data->size);
      break;
    default:
      has_neg_val = false;
    }
    graph->prop->has_negative_edges =
        (has_neg_val) ? GDF_PROP_TRUE : GDF_PROP_FALSE;
  } else {
    graph->adjList->edge_data = nullptr;
    graph->prop->has_negative_edges = GDF_PROP_FALSE;
  }

  graph->numberOfVertices = graph->adjList->offsets->size - 1;
  return GDF_SUCCESS;
}

gdf_error gdf_adj_list::get_vertex_identifiers(gdf_column *identifiers) {
  GDF_REQUIRE( offsets != nullptr , GDF_INVALID_API_CALL);
  GDF_REQUIRE( offsets->data != nullptr , GDF_INVALID_API_CALL);
  cugraph::sequence<int>((int)offsets->size-1, (int*)identifiers->data);
  return GDF_SUCCESS;
}

gdf_error gdf_adj_list::get_source_indices (gdf_column *src_indices) {
  GDF_REQUIRE( offsets != nullptr , GDF_INVALID_API_CALL);
  GDF_REQUIRE( offsets->data != nullptr , GDF_INVALID_API_CALL);
  GDF_REQUIRE( src_indices->size == indices->size, GDF_COLUMN_SIZE_MISMATCH );
  GDF_REQUIRE( src_indices->dtype == indices->dtype, GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( src_indices->size > 0, GDF_DATASET_EMPTY );
  cugraph::offsets_to_indices<int>((int*)offsets->data, offsets->size-1, (int*)src_indices->data);

  return GDF_SUCCESS;
}

gdf_error gdf_edge_list_view(gdf_graph *graph, const gdf_column *src_indices,
                             const gdf_column *dest_indices, 
                             const gdf_column *edge_data) {
  //This function returns an error if this graph object has at least one graph
  //representation to prevent a single object storing two different graphs.
  GDF_REQUIRE( ((graph->edgeList == nullptr) && (graph->adjList == nullptr) &&
    (graph->transposedAdjList == nullptr)), GDF_INVALID_API_CALL);
  /*GDF_REQUIRE( src_indices->size == dest_indices->size, GDF_COLUMN_SIZE_MISMATCH );
  GDF_REQUIRE( src_indices->dtype == dest_indices->dtype, GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( ((src_indices->dtype == GDF_INT32)), GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( src_indices->size > 0, GDF_DATASET_EMPTY );
  GDF_REQUIRE( src_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( dest_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );*/

  graph->edgeList = new gdf_edge_list;
  graph->edgeList->src_indices = new gdf_column;
  graph->edgeList->dest_indices = new gdf_column;

  cpy_column_view(src_indices, graph->edgeList->src_indices);
  cpy_column_view(dest_indices, graph->edgeList->dest_indices);

  if (!graph->prop)
    graph->prop = new gdf_graph_properties();

  if (edge_data) {
    GDF_REQUIRE(src_indices->size == edge_data->size, GDF_COLUMN_SIZE_MISMATCH);
    graph->edgeList->edge_data = new gdf_column;
    cpy_column_view(edge_data, graph->edgeList->edge_data);

    bool has_neg_val;

    switch (graph->edgeList->edge_data->dtype) {
    case GDF_INT8:
      has_neg_val = cugraph::has_negative_val(
          static_cast<int8_t *>(graph->edgeList->edge_data->data),
          graph->edgeList->edge_data->size);
      break;
    case GDF_INT16:
      has_neg_val = cugraph::has_negative_val(
          static_cast<int16_t *>(graph->edgeList->edge_data->data),
          graph->edgeList->edge_data->size);
      break;
    case GDF_INT32:
      has_neg_val = cugraph::has_negative_val(
          static_cast<int32_t *>(graph->edgeList->edge_data->data),
          graph->edgeList->edge_data->size);
      break;
    case GDF_INT64:
      has_neg_val = cugraph::has_negative_val(
          static_cast<int64_t *>(graph->edgeList->edge_data->data),
          graph->edgeList->edge_data->size);
      break;
    case GDF_FLOAT32:
      has_neg_val = cugraph::has_negative_val(
          static_cast<float *>(graph->edgeList->edge_data->data),
          graph->edgeList->edge_data->size);
      break;
    case GDF_FLOAT64:
      has_neg_val = cugraph::has_negative_val(
          static_cast<double *>(graph->edgeList->edge_data->data),
          graph->edgeList->edge_data->size);
      break;
    default:
      has_neg_val = false;
    }
    graph->prop->has_negative_edges =
        (has_neg_val) ? GDF_PROP_TRUE : GDF_PROP_FALSE;

  } else {
    graph->edgeList->edge_data = nullptr;
    graph->prop->has_negative_edges = GDF_PROP_FALSE;
  }

  gdf_error status;
  status = cugraph::indexing_check<int> (
                                static_cast<int*>(graph->edgeList->src_indices->data), 
                                static_cast<int*>(graph->edgeList->dest_indices->data), 
                                graph->edgeList->dest_indices->size);

  return status;
}

template <typename T, typename WT>
gdf_error gdf_add_adj_list_impl (gdf_graph *graph) {
    if (graph->adjList == nullptr) {
      GDF_REQUIRE( graph->edgeList != nullptr , GDF_INVALID_API_CALL);
      int nnz = graph->edgeList->src_indices->size, status = 0;
      graph->adjList = new gdf_adj_list;
      graph->adjList->offsets = new gdf_column;
      graph->adjList->indices = new gdf_column;
      graph->adjList->ownership = 1;

    if (graph->edgeList->edge_data!= nullptr) {
      graph->adjList->edge_data = new gdf_column;

      CSR_Result_Weighted<int32_t,WT> adj_list;
      status = ConvertCOOtoCSR_weighted((int*)graph->edgeList->src_indices->data, (int*)graph->edgeList->dest_indices->data, (WT*)graph->edgeList->edge_data->data, nnz, adj_list);

      gdf_column_view(graph->adjList->offsets, adj_list.rowOffsets,
                            nullptr, adj_list.size+1, graph->edgeList->src_indices->dtype);
      gdf_column_view(graph->adjList->indices, adj_list.colIndices,
                            nullptr, adj_list.nnz, graph->edgeList->src_indices->dtype);
      gdf_column_view(graph->adjList->edge_data, adj_list.edgeWeights,
                          nullptr, adj_list.nnz, graph->edgeList->edge_data->dtype);
    }
    else {
      CSR_Result<int> adj_list;
      status = ConvertCOOtoCSR((int*)graph->edgeList->src_indices->data,(int*)graph->edgeList->dest_indices->data, nnz, adj_list);
      gdf_column_view(graph->adjList->offsets, adj_list.rowOffsets,
                            nullptr, adj_list.size+1, graph->edgeList->src_indices->dtype);
      gdf_column_view(graph->adjList->indices, adj_list.colIndices,
                            nullptr, adj_list.nnz, graph->edgeList->src_indices->dtype);
    }
    if (status !=0) {
      std::cerr << "Could not generate the adj_list" << std::endl;
      return GDF_CUDA_ERROR;
    }

    graph->numberOfVertices = graph->adjList->offsets->size - 1;
  }
  return GDF_SUCCESS;
}

gdf_error gdf_add_edge_list (gdf_graph *graph) {
    if (graph->edgeList == nullptr) {
      GDF_REQUIRE( graph->adjList != nullptr , GDF_INVALID_API_CALL);
      int *d_src;
      graph->edgeList = new gdf_edge_list;
      graph->edgeList->src_indices = new gdf_column;
      graph->edgeList->dest_indices = new gdf_column;
      graph->edgeList->ownership = 2;

      hipStream_t stream{nullptr};
      ALLOC_TRY((void**)&d_src, sizeof(int) * graph->adjList->indices->size, stream);

      cugraph::offsets_to_indices<int>((int*)graph->adjList->offsets->data,
                                  graph->adjList->offsets->size-1,
                                  (int*)d_src);

      gdf_column_view(graph->edgeList->src_indices, d_src,
                      nullptr, graph->adjList->indices->size, graph->adjList->indices->dtype);
      cpy_column_view(graph->adjList->indices, graph->edgeList->dest_indices);

      if (graph->adjList->edge_data != nullptr) {
        graph->edgeList->edge_data = new gdf_column;
        cpy_column_view(graph->adjList->edge_data, graph->edgeList->edge_data);
      }
  }
  return GDF_SUCCESS;
}


template <typename WT>
gdf_error gdf_add_transposed_adj_list_impl (gdf_graph *graph) {
    if (graph->transposedAdjList == nullptr ) {
      GDF_REQUIRE( graph->edgeList != nullptr , GDF_INVALID_API_CALL);
      int nnz = graph->edgeList->src_indices->size, status = 0;
      graph->transposedAdjList = new gdf_adj_list;
      graph->transposedAdjList->offsets = new gdf_column;
      graph->transposedAdjList->indices = new gdf_column;
      graph->transposedAdjList->ownership = 1;

      if (graph->edgeList->edge_data) {
        graph->transposedAdjList->edge_data = new gdf_column;
        CSR_Result_Weighted<int32_t,WT> adj_list;
        status = ConvertCOOtoCSR_weighted( (int*)graph->edgeList->dest_indices->data, (int*)graph->edgeList->src_indices->data, (WT*)graph->edgeList->edge_data->data, nnz, adj_list);
        gdf_column_view(graph->transposedAdjList->offsets, adj_list.rowOffsets,
                              nullptr, adj_list.size+1, graph->edgeList->src_indices->dtype);
        gdf_column_view(graph->transposedAdjList->indices, adj_list.colIndices,
                              nullptr, adj_list.nnz, graph->edgeList->src_indices->dtype);
        gdf_column_view(graph->transposedAdjList->edge_data, adj_list.edgeWeights,
                            nullptr, adj_list.nnz, graph->edgeList->edge_data->dtype);
      }
      else {

        CSR_Result<int> adj_list;
        status = ConvertCOOtoCSR((int*)graph->edgeList->dest_indices->data, (int*)graph->edgeList->src_indices->data, nnz, adj_list);
        gdf_column_view(graph->transposedAdjList->offsets, adj_list.rowOffsets,
                              nullptr, adj_list.size+1, graph->edgeList->src_indices->dtype);
        gdf_column_view(graph->transposedAdjList->indices, adj_list.colIndices,
                              nullptr, adj_list.nnz, graph->edgeList->src_indices->dtype);
      }
      if (status !=0) {
        std::cerr << "Could not generate the adj_list" << std::endl;
        return GDF_CUDA_ERROR;
      }

      graph->numberOfVertices = graph->transposedAdjList->offsets->size - 1;
    }
    return GDF_SUCCESS;
}

gdf_error gdf_add_adj_list(gdf_graph *graph) {
  if (graph->adjList != nullptr)
    return GDF_SUCCESS;

  GDF_REQUIRE( graph->edgeList != nullptr , GDF_INVALID_API_CALL);
  GDF_REQUIRE( graph->edgeList->src_indices->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE );

  if (graph->edgeList->edge_data != nullptr) {
    switch (graph->edgeList->edge_data->dtype) {
      case GDF_FLOAT32:   return gdf_add_adj_list_impl<int32_t, float>(graph);
      case GDF_FLOAT64:   return gdf_add_adj_list_impl<int32_t, double>(graph);
      default: return GDF_UNSUPPORTED_DTYPE;
    }
  }
  else {
    return gdf_add_adj_list_impl<int32_t, float>(graph);
  }
}

gdf_error gdf_add_transposed_adj_list(gdf_graph *graph) {
  if (graph->edgeList == nullptr)
    gdf_add_edge_list(graph);

  GDF_REQUIRE(graph->edgeList->src_indices->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);
  GDF_REQUIRE(graph->edgeList->dest_indices->dtype == GDF_INT32, GDF_UNSUPPORTED_DTYPE);

  if (graph->edgeList->edge_data != nullptr) {
    switch (graph->edgeList->edge_data->dtype) {
      case GDF_FLOAT32:   return gdf_add_transposed_adj_list_impl<float>(graph);
      case GDF_FLOAT64:   return gdf_add_transposed_adj_list_impl<double>(graph);
      default: return GDF_UNSUPPORTED_DTYPE;
    }
  }
  else {
    return gdf_add_transposed_adj_list_impl<float>(graph);
  }
}

gdf_error gdf_delete_adj_list(gdf_graph *graph) {
  if (graph->adjList) {
    delete graph->adjList;
  }
  graph->adjList = nullptr;
  return GDF_SUCCESS;
}

gdf_error gdf_delete_edge_list(gdf_graph *graph) {
  if (graph->edgeList) {
    delete graph->edgeList;
  }
  graph->edgeList = nullptr;
  return GDF_SUCCESS;
}

gdf_error gdf_delete_transposed_adj_list(gdf_graph *graph) {
  if (graph->transposedAdjList) {
    delete graph->transposedAdjList;
  }
  graph->transposedAdjList = nullptr;
  return GDF_SUCCESS;
}

gdf_error gdf_number_of_vertices(gdf_graph *graph) {
  if (graph->numberOfVertices != 0)
    return GDF_SUCCESS;

  //
  //  int32_t implementation for now, since that's all that
  //  is supported elsewhere.
  //
  GDF_REQUIRE( (graph->edgeList != nullptr), GDF_INVALID_API_CALL);
  GDF_REQUIRE( (graph->edgeList->src_indices->dtype == GDF_INT32), GDF_UNSUPPORTED_DTYPE );

  int32_t  h_max[2];
  int32_t *d_max;
  void    *d_temp_storage = nullptr;
  size_t   temp_storage_bytes = 0;
  
  ALLOC_TRY(&d_max, sizeof(int32_t), nullptr);
  
  //
  //  Compute size of temp storage
  //
  int32_t *tmp = static_cast<int32_t *>(graph->edgeList->src_indices->data);

  hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, tmp, d_max, graph->edgeList->src_indices->size);

  //
  //  Compute max of src indices and copy to host
  //
  ALLOC_TRY(&d_temp_storage, temp_storage_bytes, nullptr);
  hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, tmp, d_max, graph->edgeList->src_indices->size);

  CUDA_TRY(hipMemcpy(h_max, d_max, sizeof(int32_t), hipMemcpyDeviceToHost));

  //
  //  Compute max of dest indices and copy to host
  //
  tmp = static_cast<int32_t *>(graph->edgeList->dest_indices->data);
  hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, tmp, d_max, graph->edgeList->src_indices->size);
  CUDA_TRY(hipMemcpy(h_max + 1, d_max, sizeof(int32_t), hipMemcpyDeviceToHost));

  ALLOC_FREE_TRY(d_temp_storage, nullptr);
  ALLOC_FREE_TRY(d_max, nullptr);
  
  graph->numberOfVertices = 1 + std::max(h_max[0], h_max[1]);
  return GDF_SUCCESS;
}
