/*
 * Copyright (c) 2021-2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "structure/symmetrize_edgelist_impl.cuh"

namespace cugraph {

// MG instantiation

template std::tuple<rmm::device_uvector<int64_t>,
                    rmm::device_uvector<int64_t>,
                    std::optional<rmm::device_uvector<float>>,
                    std::optional<rmm::device_uvector<int64_t>>,
                    std::optional<rmm::device_uvector<int32_t>>,
                    std::optional<rmm::device_uvector<int32_t>>,
                    std::optional<rmm::device_uvector<int32_t>>>
symmetrize_edgelist<int64_t, int64_t, float, int32_t, int32_t, false, true>(
  raft::handle_t const& handle,
  rmm::device_uvector<int64_t>&& edgelist_srcs,
  rmm::device_uvector<int64_t>&& edgelist_dsts,
  std::optional<rmm::device_uvector<float>>&& edgelist_weights,
  std::optional<rmm::device_uvector<int64_t>>&& edgelist_edge_ids,
  std::optional<rmm::device_uvector<int32_t>>&& edgelist_edge_types,
  std::optional<rmm::device_uvector<int32_t>>&& edgelist_edge_start_times,
  std::optional<rmm::device_uvector<int32_t>>&& edgelist_edge_end_times,
  bool reciprocal);

template std::tuple<rmm::device_uvector<int64_t>,
                    rmm::device_uvector<int64_t>,
                    std::optional<rmm::device_uvector<float>>,
                    std::optional<rmm::device_uvector<int64_t>>,
                    std::optional<rmm::device_uvector<int32_t>>,
                    std::optional<rmm::device_uvector<int32_t>>,
                    std::optional<rmm::device_uvector<int32_t>>>
symmetrize_edgelist<int64_t, int64_t, float, int32_t, int32_t, true, true>(
  raft::handle_t const& handle,
  rmm::device_uvector<int64_t>&& edgelist_srcs,
  rmm::device_uvector<int64_t>&& edgelist_dsts,
  std::optional<rmm::device_uvector<float>>&& edgelist_weights,
  std::optional<rmm::device_uvector<int64_t>>&& edgelist_edge_ids,
  std::optional<rmm::device_uvector<int32_t>>&& edgelist_edge_types,
  std::optional<rmm::device_uvector<int32_t>>&& edgelist_edge_start_times,
  std::optional<rmm::device_uvector<int32_t>>&& edgelist_edge_end_times,
  bool reciprocal);

template std::tuple<rmm::device_uvector<int64_t>,
                    rmm::device_uvector<int64_t>,
                    std::optional<rmm::device_uvector<double>>,
                    std::optional<rmm::device_uvector<int64_t>>,
                    std::optional<rmm::device_uvector<int32_t>>,
                    std::optional<rmm::device_uvector<int32_t>>,
                    std::optional<rmm::device_uvector<int32_t>>>
symmetrize_edgelist<int64_t, int64_t, double, int32_t, int32_t, false, true>(
  raft::handle_t const& handle,
  rmm::device_uvector<int64_t>&& edgelist_srcs,
  rmm::device_uvector<int64_t>&& edgelist_dsts,
  std::optional<rmm::device_uvector<double>>&& edgelist_weights,
  std::optional<rmm::device_uvector<int64_t>>&& edgelist_edge_ids,
  std::optional<rmm::device_uvector<int32_t>>&& edgelist_edge_types,
  std::optional<rmm::device_uvector<int32_t>>&& edgelist_edge_start_times,
  std::optional<rmm::device_uvector<int32_t>>&& edgelist_edge_end_times,
  bool reciprocal);

template std::tuple<rmm::device_uvector<int64_t>,
                    rmm::device_uvector<int64_t>,
                    std::optional<rmm::device_uvector<double>>,
                    std::optional<rmm::device_uvector<int64_t>>,
                    std::optional<rmm::device_uvector<int32_t>>,
                    std::optional<rmm::device_uvector<int32_t>>,
                    std::optional<rmm::device_uvector<int32_t>>>
symmetrize_edgelist<int64_t, int64_t, double, int32_t, int32_t, true, true>(
  raft::handle_t const& handle,
  rmm::device_uvector<int64_t>&& edgelist_srcs,
  rmm::device_uvector<int64_t>&& edgelist_dsts,
  std::optional<rmm::device_uvector<double>>&& edgelist_weights,
  std::optional<rmm::device_uvector<int64_t>>&& edgelist_edge_ids,
  std::optional<rmm::device_uvector<int32_t>>&& edgelist_edge_types,
  std::optional<rmm::device_uvector<int32_t>>&& edgelist_edge_start_times,
  std::optional<rmm::device_uvector<int32_t>>&& edgelist_edge_end_times,
  bool reciprocal);

template std::tuple<rmm::device_uvector<int64_t>,
                    rmm::device_uvector<int64_t>,
                    std::optional<rmm::device_uvector<float>>,
                    std::optional<rmm::device_uvector<int64_t>>,
                    std::optional<rmm::device_uvector<int32_t>>,
                    std::optional<rmm::device_uvector<int64_t>>,
                    std::optional<rmm::device_uvector<int64_t>>>
symmetrize_edgelist<int64_t, int64_t, float, int32_t, int64_t, false, true>(
  raft::handle_t const& handle,
  rmm::device_uvector<int64_t>&& edgelist_srcs,
  rmm::device_uvector<int64_t>&& edgelist_dsts,
  std::optional<rmm::device_uvector<float>>&& edgelist_weights,
  std::optional<rmm::device_uvector<int64_t>>&& edgelist_edge_ids,
  std::optional<rmm::device_uvector<int32_t>>&& edgelist_edge_types,
  std::optional<rmm::device_uvector<int64_t>>&& edgelist_edge_start_times,
  std::optional<rmm::device_uvector<int64_t>>&& edgelist_edge_end_times,
  bool reciprocal);

template std::tuple<rmm::device_uvector<int64_t>,
                    rmm::device_uvector<int64_t>,
                    std::optional<rmm::device_uvector<float>>,
                    std::optional<rmm::device_uvector<int64_t>>,
                    std::optional<rmm::device_uvector<int32_t>>,
                    std::optional<rmm::device_uvector<int64_t>>,
                    std::optional<rmm::device_uvector<int64_t>>>
symmetrize_edgelist<int64_t, int64_t, float, int32_t, int64_t, true, true>(
  raft::handle_t const& handle,
  rmm::device_uvector<int64_t>&& edgelist_srcs,
  rmm::device_uvector<int64_t>&& edgelist_dsts,
  std::optional<rmm::device_uvector<float>>&& edgelist_weights,
  std::optional<rmm::device_uvector<int64_t>>&& edgelist_edge_ids,
  std::optional<rmm::device_uvector<int32_t>>&& edgelist_edge_types,
  std::optional<rmm::device_uvector<int64_t>>&& edgelist_edge_start_times,
  std::optional<rmm::device_uvector<int64_t>>&& edgelist_edge_end_times,
  bool reciprocal);

template std::tuple<rmm::device_uvector<int64_t>,
                    rmm::device_uvector<int64_t>,
                    std::optional<rmm::device_uvector<double>>,
                    std::optional<rmm::device_uvector<int64_t>>,
                    std::optional<rmm::device_uvector<int32_t>>,
                    std::optional<rmm::device_uvector<int64_t>>,
                    std::optional<rmm::device_uvector<int64_t>>>
symmetrize_edgelist<int64_t, int64_t, double, int32_t, int64_t, false, true>(
  raft::handle_t const& handle,
  rmm::device_uvector<int64_t>&& edgelist_srcs,
  rmm::device_uvector<int64_t>&& edgelist_dsts,
  std::optional<rmm::device_uvector<double>>&& edgelist_weights,
  std::optional<rmm::device_uvector<int64_t>>&& edgelist_edge_ids,
  std::optional<rmm::device_uvector<int32_t>>&& edgelist_edge_types,
  std::optional<rmm::device_uvector<int64_t>>&& edgelist_edge_start_times,
  std::optional<rmm::device_uvector<int64_t>>&& edgelist_edge_end_times,
  bool reciprocal);

template std::tuple<rmm::device_uvector<int64_t>,
                    rmm::device_uvector<int64_t>,
                    std::optional<rmm::device_uvector<double>>,
                    std::optional<rmm::device_uvector<int64_t>>,
                    std::optional<rmm::device_uvector<int32_t>>,
                    std::optional<rmm::device_uvector<int64_t>>,
                    std::optional<rmm::device_uvector<int64_t>>>
symmetrize_edgelist<int64_t, int64_t, double, int32_t, int64_t, true, true>(
  raft::handle_t const& handle,
  rmm::device_uvector<int64_t>&& edgelist_srcs,
  rmm::device_uvector<int64_t>&& edgelist_dsts,
  std::optional<rmm::device_uvector<double>>&& edgelist_weights,
  std::optional<rmm::device_uvector<int64_t>>&& edgelist_edge_ids,
  std::optional<rmm::device_uvector<int32_t>>&& edgelist_edge_types,
  std::optional<rmm::device_uvector<int64_t>>&& edgelist_edge_start_times,
  std::optional<rmm::device_uvector<int64_t>>&& edgelist_edge_end_times,
  bool reciprocal);

}  // namespace cugraph
