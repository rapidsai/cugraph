#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <db/db_operators.cuh>
#include <cub/device/device_select.cuh>

namespace cugraph { 
  namespace db {
    template<typename IndexType>
    struct degree_iterator {
      IndexType* offsets;
      degree_iterator(IndexType* _offsets) :
          offsets(_offsets) {
      }

      __host__ __device__
      IndexType operator[](IndexType place) {
        return offsets[place + 1] - offsets[place];
      }
    };

    template<typename It, typename IndexType>
    struct deref_functor {
      It iterator;
      deref_functor(It it) :
          iterator(it) {
      }

      __host__ __device__
      IndexType operator()(IndexType in) {
        return iterator[in];
      }
    };

    template<typename idx_t, typename flag_t>
    struct notNegativeOne {
      __host__ __device__
      flag_t operator()(idx_t in) {
        return in != -1;
      }
    };

    template<typename IndexType>
    __device__ IndexType binsearch_maxle(const IndexType *vec,
                                         const IndexType val,
                                         IndexType low,
                                         IndexType high) {
      while (true) {
        if (low == high)
          return low; //we know it exists
        if ((low + 1) == high)
          return (vec[high] <= val) ? high : low;

        IndexType mid = low + (high - low) / 2;

        if (vec[mid] > val)
          high = mid - 1;
        else
          low = mid;
      }
    }

    template<typename IndexType>
    __global__ void compute_bucket_offsets_kernel(const IndexType *frontier_degrees_exclusive_sum,
                                                  IndexType *bucket_offsets,
                                                  const IndexType frontier_size,
                                                  IndexType total_degree) {
      IndexType end = ((total_degree - 1 + FIND_MATCHES_BLOCK_SIZE) / FIND_MATCHES_BLOCK_SIZE);

      for (IndexType bid = blockIdx.x * blockDim.x + threadIdx.x;
          bid <= end;
          bid += gridDim.x * blockDim.x) {

        IndexType eid = min(bid * FIND_MATCHES_BLOCK_SIZE, total_degree - 1);

        bucket_offsets[bid] = binsearch_maxle(frontier_degrees_exclusive_sum,
                                              eid,
                                              (IndexType) 0,
                                              frontier_size - 1);

      }
    }

    template<typename idx_t>
    __global__ void findMatchesKernel(idx_t inputSize,
                                      idx_t outputSize,
                                      idx_t maxBlock,
                                      idx_t* offsets,
                                      idx_t* indirection,
                                      idx_t* blockStarts,
                                      idx_t* expandCounts,
                                      idx_t* frontier,
                                      idx_t* columnA,
                                      idx_t* columnB,
                                      idx_t* columnC,
                                      idx_t* outputA,
                                      idx_t* outputB,
                                      idx_t* outputC,
                                      idx_t* outputD,
                                      idx_t patternA,
                                      idx_t patternB,
                                      idx_t patternC) {
      __shared__ idx_t blockRange[2];
      __shared__ idx_t localExSum[FIND_MATCHES_BLOCK_SIZE * 2];
      __shared__ idx_t localFrontier[FIND_MATCHES_BLOCK_SIZE * 2];

      for (idx_t bid = blockIdx.x; bid < maxBlock; bid += gridDim.x) {
        // Copy in the block's section of the expand counts
        if (threadIdx.x == 0) {
          blockRange[0] = blockStarts[bid];
          blockRange[1] = blockStarts[bid + 1];
          if (blockRange[0] > 0) {
            blockRange[0] -= 1;
          }
        }
        __syncthreads();

        idx_t sectionSize = blockRange[1] - blockRange[0];
        for (int tid = threadIdx.x; tid <= sectionSize; tid += blockDim.x) {
          localExSum[tid] = expandCounts[blockRange[0] + tid];
          localFrontier[tid] = frontier[blockRange[0] + tid];
        }
        __syncthreads();

        // Do the work item for each thread of this virtual block:
        idx_t tid = bid * blockDim.x + threadIdx.x;
        if (tid < outputSize) {
          // Figure out which row this thread/iteration is working on
          idx_t sourceIdx = binsearch_maxle(localExSum, tid, (idx_t)0, (idx_t)sectionSize);
          idx_t source = localFrontier[sourceIdx];
          idx_t rank = tid - localExSum[sourceIdx];
          idx_t row_id = indirection[offsets[source] + rank];

          // Load in values from the row for A, B, and C columns
          idx_t valA = columnA[row_id];
          idx_t valB = columnB[row_id];
          idx_t valC = columnC[row_id];

          // Debugging output
//          printf("RowId: %d, valA: %d, valB: %d, valC: %d\n", row_id, valA, valB, valC);
//          printf("PatternA: %d, PatternB: %d, PatternC: %d\n", patternA, patternB, patternC);

          // Compare the row values with constants in the pattern
          bool matchA = outputA != nullptr ? true : patternA == valA;
          bool matchB = outputB != nullptr ? true : patternB == valB;
          bool matchC = outputC != nullptr ? true : patternC == valC;

          // If row doesn't match, set row values to -1 before writing out
          if (!(matchA && matchB && matchC)) {
            valA = -1;
            valB = -1;
            valC = -1;
            row_id = -1;
          }

          // Write out values to non-null outputs
          if (outputA != nullptr)
            outputA[tid] = valA;
          if (outputB != nullptr)
            outputB[tid] = valB;
          if (outputC != nullptr)
            outputC[tid] = valC;
          if (outputD != nullptr)
            outputD[tid] = row_id;
        }
      }
    }

    template<typename idx_t>
    db_result<idx_t> findMatches(db_pattern<idx_t>& pattern,
                        db_table<idx_t>& table,
                        gdf_column* frontier,
                        int indexPosition) {
      // Find out if the indexPosition is a variable or constant
      bool indexConstant = !pattern.getEntry(indexPosition).isVariable();

      db_column_index<idx_t>& theIndex = table.getIndex(indexPosition);

      // Check to see whether we are going to be saving out the row ids from matches
      bool saveRowIds = false;
      if (pattern.getSize() == 4)
        saveRowIds = true;

      // Check if we have a frontier to use, if we don't make one up
      bool givenInputFrontier = frontier != nullptr;
      idx_t frontierSize;
      idx_t* frontier_ptr = nullptr;
      if (givenInputFrontier) {
        frontier_ptr = (idx_t*)frontier->data;
        frontierSize = frontier->size;
      }
      else {
        if (indexConstant) {
          // Use a single value equal to the constant in the pattern
          idx_t constantValue = pattern.getEntry(indexPosition).getConstant();
          ALLOC_TRY(&frontier_ptr, sizeof(idx_t), nullptr);
          thrust::fill(rmm::exec_policy(nullptr)->on(nullptr),
                       frontier_ptr,
                       frontier_ptr + 1,
                       constantValue);
          frontierSize = 1;
        }
        else {
          // Making a sequence of values from zero to n where n is the highest ID present in the index.
          idx_t highestId = theIndex.getOffsets()->size - 2;
          ALLOC_TRY(&frontier_ptr, sizeof(idx_t) * (highestId + 1), nullptr);
          thrust::sequence(rmm::exec_policy(nullptr)->on(nullptr),
                           frontier_ptr,
                           frontier_ptr + highestId + 1);
          frontierSize = highestId + 1;
        }
      }

      // Collect all the pointers needed to run the main kernel
      idx_t* columnA = (idx_t*)table.getColumn(0)->data;
      idx_t* columnB = (idx_t*)table.getColumn(1)->data;
      idx_t* columnC = (idx_t*)table.getColumn(2)->data;
      idx_t* offsets = (idx_t*)theIndex.getOffsets()->data;
      idx_t* indirection = (idx_t*)theIndex.getIndirection()->data;

      // Load balance the input
      idx_t *exsum_degree = nullptr;
      ALLOC_TRY(&exsum_degree, sizeof(idx_t) * (frontierSize + 1), nullptr);
      degree_iterator<idx_t> deg_it(offsets);
      deref_functor<degree_iterator<idx_t>, idx_t> deref(deg_it);
      thrust::fill(rmm::exec_policy(nullptr)->on(nullptr), exsum_degree, exsum_degree + 1, 0);
      thrust::transform(rmm::exec_policy(nullptr)->on(nullptr),
                        frontier_ptr,
                        frontier_ptr + frontierSize,
                        exsum_degree + 1,
                        deref);
      thrust::inclusive_scan(rmm::exec_policy(nullptr)->on(nullptr),
                             exsum_degree + 1,
                             exsum_degree + frontierSize + 1,
                             exsum_degree + 1);
      idx_t output_size;
      hipMemcpy(&output_size, &exsum_degree[frontierSize], sizeof(idx_t), hipMemcpyDefault);

      // Debugging output
//      std::cout << "OutputSize = " << output_size << "\n";

      idx_t num_blocks = (output_size + FIND_MATCHES_BLOCK_SIZE - 1) / FIND_MATCHES_BLOCK_SIZE;
      idx_t *block_bucket_offsets = nullptr;
      ALLOC_TRY(&block_bucket_offsets, sizeof(idx_t) * (num_blocks + 1), nullptr);

      dim3 grid, block;
      block.x = 512;
      grid.x = min((idx_t) MAXBLOCKS, (num_blocks / 512) + 1);
      compute_bucket_offsets_kernel<<<grid, block, 0, nullptr>>>(exsum_degree,
                                                                 block_bucket_offsets,
                                                                 frontierSize,
                                                                 output_size);

      // Allocate space for the result
      idx_t *outputA = nullptr;
      idx_t *outputB = nullptr;
      idx_t *outputC = nullptr;
      idx_t *outputD = nullptr;
      if (pattern.getEntry(0).isVariable()) {
        ALLOC_TRY(&outputA, sizeof(idx_t) * output_size, nullptr);
      }
      if (pattern.getEntry(1).isVariable()) {
        ALLOC_TRY(&outputB, sizeof(idx_t) * output_size, nullptr);
      }
      if (pattern.getEntry(2).isVariable()) {
        ALLOC_TRY(&outputC, sizeof(idx_t) * output_size, nullptr);
      }
      if (saveRowIds) {
        ALLOC_TRY(&outputD, sizeof(idx_t) * output_size, nullptr);
      }

      // Get the constant pattern entries from the pattern to pass into the main kernel
      idx_t patternA = -1;
      idx_t patternB = -1;
      idx_t patternC = -1;
      if (!pattern.getEntry(0).isVariable()) {
        patternA = pattern.getEntry(0).getConstant();
      }
      if (!pattern.getEntry(1).isVariable()) {
        patternB = pattern.getEntry(1).getConstant();
      }
      if (!pattern.getEntry(2).isVariable()) {
        patternC = pattern.getEntry(2).getConstant();
      }

      // Call the main kernel
      block.x = FIND_MATCHES_BLOCK_SIZE;
      grid.x = min((idx_t) MAXBLOCKS,
                   (output_size + (idx_t) FIND_MATCHES_BLOCK_SIZE - 1)
                       / (idx_t) FIND_MATCHES_BLOCK_SIZE);
      findMatchesKernel<<<grid, block, 0, nullptr>>>(frontierSize,
                                                     output_size,
                                                     num_blocks,
                                                     offsets,
                                                     indirection,
                                                     block_bucket_offsets,
                                                     exsum_degree,
                                                     frontier_ptr,
                                                     columnA,
                                                     columnB,
                                                     columnC,
                                                     outputA,
                                                     outputB,
                                                     outputC,
                                                     outputD,
                                                     patternA,
                                                     patternB,
                                                     patternC);

      // Debugging output
//      if (outputA != nullptr) {
//        idx_t* outputA_h = (idx_t*)malloc(sizeof(idx_t) * output_size);
//        hipMemcpy(outputA_h, outputA, sizeof(idx_t)*output_size, hipMemcpyDefault);
//        std::cout << "OutputA: ";
//        for (int i = 0; i < output_size; i++)
//          std::cout << outputA_h[i] << " ";
//        std::cout << "\n";
//        free(outputA_h);
//      }
//      if (outputB != nullptr) {
//        idx_t* outputB_h = (idx_t*) malloc(sizeof(idx_t) * output_size);
//        hipMemcpy(outputB_h, outputB, sizeof(idx_t) * output_size, hipMemcpyDefault);
//        std::cout << "OutputB: ";
//        for (int i = 0; i < output_size; i++)
//          std::cout << outputB_h[i] << " ";
//        std::cout << "\n";
//        free(outputB_h);
//      }
//      if (outputC != nullptr) {
//        idx_t* outputC_h = (idx_t*) malloc(sizeof(idx_t) * output_size);
//        hipMemcpy(outputC_h, outputC, sizeof(idx_t) * output_size, hipMemcpyDefault);
//        std::cout << "OutputC: ";
//        for (int i = 0; i < output_size; i++)
//          std::cout << outputC_h[i] << " ";
//        std::cout << "\n";
//        free(outputC_h);
//      }
//      if (outputD != nullptr) {
//        idx_t* outputD_h = (idx_t*) malloc(sizeof(idx_t) * output_size);
//        hipMemcpy(outputD_h, outputD, sizeof(idx_t) * output_size, hipMemcpyDefault);
//        std::cout << "OutputD: ";
//        for (int i = 0; i < output_size; i++)
//          std::cout << outputD_h[i] << " ";
//        std::cout << "\n";
//        free(outputD_h);
//      }

      // Get the non-null output columns
      std::vector<idx_t*> columns;
      std::vector<std::string> names;
      if (outputA != nullptr) {
        columns.push_back(outputA);
        names.push_back(pattern.getEntry(0).getVariable());
      }
      if (outputB != nullptr) {
        columns.push_back(outputB);
        names.push_back(pattern.getEntry(1).getVariable());
      }
      if (outputC != nullptr) {
        columns.push_back(outputC);
        names.push_back(pattern.getEntry(2).getVariable());
      }
      if (outputD != nullptr) {
        columns.push_back(outputD);
        names.push_back(pattern.getEntry(3).getVariable());
      }

      // Remove non-matches from result
      int8_t* flags = nullptr;
      ALLOC_TRY(&flags, sizeof(int8_t) * output_size, nullptr);
      idx_t* col_ptr = columns[0];
      thrust::transform(rmm::exec_policy(nullptr)->on(nullptr),
                        col_ptr,
                        col_ptr + output_size,
                        flags,
                        notNegativeOne<idx_t, int8_t>());

      void* tempSpace = nullptr;
      size_t tempSpaceSize = 0;
      idx_t* compactSize_d = nullptr;
      ALLOC_TRY(&compactSize_d, sizeof(idx_t), nullptr);
      hipcub::DeviceSelect::Flagged(tempSpace,
                                 tempSpaceSize,
                                 col_ptr,
                                 flags,
                                 col_ptr,
                                 compactSize_d,
                                 output_size);
      ALLOC_TRY(&tempSpace, tempSpaceSize, nullptr);
      hipcub::DeviceSelect::Flagged(tempSpace,
                                 tempSpaceSize,
                                 col_ptr,
                                 flags,
                                 col_ptr,
                                 compactSize_d,
                                 output_size);
      idx_t compactSize_h;
      hipMemcpy(&compactSize_h, compactSize_d, sizeof(idx_t), hipMemcpyDefault);
      for (size_t i = 1; i < columns.size(); i++) {
        col_ptr = columns[i];
        hipcub::DeviceSelect::Flagged(tempSpace,
                                   tempSpaceSize,
                                   col_ptr,
                                   flags,
                                   col_ptr,
                                   compactSize_d,
                                   output_size);
      }

      // Put together the result to return
      db_result<idx_t> result;
      for (size_t i = 0; i < names.size(); i++) {
        result.addColumn(names[i]);
      }
      result.allocateColumns(compactSize_h);
      for (size_t i = 0; i < columns.size(); i++) {
        idx_t* outputPtr = result.getData(names[i]);
        idx_t* inputPtr = columns[i];
        hipMemcpy(outputPtr, inputPtr, sizeof(idx_t) * compactSize_h, hipMemcpyDefault);
      }

      // Clean up allocations
      if (!givenInputFrontier)
        ALLOC_FREE_TRY(frontier_ptr, nullptr);
      
      ALLOC_FREE_TRY(exsum_degree, nullptr);
      ALLOC_FREE_TRY(block_bucket_offsets, nullptr);
      ALLOC_FREE_TRY(tempSpace, nullptr);
      ALLOC_FREE_TRY(compactSize_d, nullptr);
      ALLOC_FREE_TRY(flags, nullptr);
      
      if (outputA != nullptr)
        ALLOC_FREE_TRY(outputA, nullptr);
      
        if (outputB != nullptr)
        ALLOC_FREE_TRY(outputB, nullptr);
      
        if (outputC != nullptr)
        ALLOC_FREE_TRY(outputC, nullptr);
      
        if (outputD != nullptr)
        ALLOC_FREE_TRY(outputD, nullptr);

      // Return the result
      return result;
    }

    template db_result<int32_t> findMatches(db_pattern<int32_t>& pattern,
                                            db_table<int32_t>& table,
                                            gdf_column* frontier,
                                            int indexPosition);
    template db_result<int64_t> findMatches(db_pattern<int64_t>& pattern,
                                            db_table<int64_t>& table,
                                            gdf_column* frontier,
                                            int indexPosition);
} } //namespace
