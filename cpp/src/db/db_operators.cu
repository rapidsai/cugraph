#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <rmm/rmm.h>
#include <utilities/error_utils.h>
#include <cub/device/device_select.cuh>
#include <db/db_operators.cuh>

namespace cugraph {
namespace db {
template <typename IndexType>
struct degree_iterator {
  IndexType* offsets;
  degree_iterator(IndexType* _offsets) : offsets(_offsets) {}

  __host__ __device__ IndexType operator[](IndexType place)
  {
    return offsets[place + 1] - offsets[place];
  }
};

template <typename It, typename IndexType>
struct deref_functor {
  It iterator;
  deref_functor(It it) : iterator(it) {}

  __host__ __device__ IndexType operator()(IndexType in) { return iterator[in]; }
};

template <typename idx_t, typename flag_t>
struct notNegativeOne {
  __host__ __device__ flag_t operator()(idx_t in) { return in != -1; }
};

template <typename IndexType>
__device__ IndexType
binsearch_maxle(const IndexType* vec, const IndexType val, IndexType low, IndexType high)
{
  while (true) {
    if (low == high) return low;  // we know it exists
    if ((low + 1) == high) return (vec[high] <= val) ? high : low;

    IndexType mid = low + (high - low) / 2;

    if (vec[mid] > val)
      high = mid - 1;
    else
      low = mid;
  }
}

template <typename IndexType>
__global__ void compute_bucket_offsets_kernel(const IndexType* frontier_degrees_exclusive_sum,
                                              IndexType* bucket_offsets,
                                              const IndexType frontier_size,
                                              IndexType total_degree)
{
  IndexType end = ((total_degree - 1 + FIND_MATCHES_BLOCK_SIZE) / FIND_MATCHES_BLOCK_SIZE);

  for (IndexType bid = blockIdx.x * blockDim.x + threadIdx.x; bid <= end;
       bid += gridDim.x * blockDim.x) {
    IndexType eid = min(bid * FIND_MATCHES_BLOCK_SIZE, total_degree - 1);

    bucket_offsets[bid] =
      binsearch_maxle(frontier_degrees_exclusive_sum, eid, (IndexType)0, frontier_size - 1);
  }
}

template <typename idx_t>
__global__ void findMatchesKernel(idx_t inputSize,
                                  idx_t outputSize,
                                  idx_t maxBlock,
                                  idx_t* offsets,
                                  idx_t* indirection,
                                  idx_t* blockStarts,
                                  idx_t* expandCounts,
                                  idx_t* frontier,
                                  idx_t* columnA,
                                  idx_t* columnB,
                                  idx_t* columnC,
                                  idx_t* outputA,
                                  idx_t* outputB,
                                  idx_t* outputC,
                                  idx_t* outputD,
                                  idx_t patternA,
                                  idx_t patternB,
                                  idx_t patternC)
{
  __shared__ idx_t blockRange[2];
  __shared__ idx_t localExSum[FIND_MATCHES_BLOCK_SIZE * 2];
  __shared__ idx_t localFrontier[FIND_MATCHES_BLOCK_SIZE * 2];

  for (idx_t bid = blockIdx.x; bid < maxBlock; bid += gridDim.x) {
    // Copy in the block's section of the expand counts
    if (threadIdx.x == 0) {
      blockRange[0] = blockStarts[bid];
      blockRange[1] = blockStarts[bid + 1];
      if (blockRange[0] > 0) { blockRange[0] -= 1; }
    }
    __syncthreads();

    idx_t sectionSize = blockRange[1] - blockRange[0];
    for (int tid = threadIdx.x; tid <= sectionSize; tid += blockDim.x) {
      localExSum[tid]    = expandCounts[blockRange[0] + tid];
      localFrontier[tid] = frontier[blockRange[0] + tid];
    }
    __syncthreads();

    // Do the work item for each thread of this virtual block:
    idx_t tid = bid * blockDim.x + threadIdx.x;
    if (tid < outputSize) {
      // Figure out which row this thread/iteration is working on
      idx_t sourceIdx = binsearch_maxle(localExSum, tid, (idx_t)0, (idx_t)sectionSize);
      idx_t source    = localFrontier[sourceIdx];
      idx_t rank      = tid - localExSum[sourceIdx];
      idx_t row_id    = indirection[offsets[source] + rank];

      // Load in values from the row for A, B, and C columns
      idx_t valA = columnA[row_id];
      idx_t valB = columnB[row_id];
      idx_t valC = columnC[row_id];

      // Compare the row values with constants in the pattern
      bool matchA = outputA != nullptr ? true : patternA == valA;
      bool matchB = outputB != nullptr ? true : patternB == valB;
      bool matchC = outputC != nullptr ? true : patternC == valC;

      // If row doesn't match, set row values to -1 before writing out
      if (!(matchA && matchB && matchC)) {
        valA   = -1;
        valB   = -1;
        valC   = -1;
        row_id = -1;
      }

      // Write out values to non-null outputs
      if (outputA != nullptr) outputA[tid] = valA;
      if (outputB != nullptr) outputB[tid] = valB;
      if (outputC != nullptr) outputC[tid] = valC;
      if (outputD != nullptr) outputD[tid] = row_id;
    }
  }
}

template <typename idx_t>
db_result<idx_t> findMatches(db_pattern<idx_t>& pattern,
                             db_table<idx_t>& table,
                             idx_t* frontier,
                             idx_t frontier_size,
                             int indexPosition)
{
  // Find out if the indexPosition is a variable or constant
  bool indexConstant = !pattern.getEntry(indexPosition).isVariable();

  db_column_index<idx_t>& theIndex = table.getIndex(indexPosition);

  // Check to see whether we are going to be saving out the row ids from matches
  bool saveRowIds = false;
  if (pattern.getSize() == 4) saveRowIds = true;

  // Check if we have a frontier to use, if we don't make one up
  bool givenInputFrontier = frontier != nullptr;
  idx_t frontierSize;
  idx_t* frontier_ptr = nullptr;
  rmm::device_buffer frontierBuffer;
  if (givenInputFrontier) {
    frontier_ptr = frontier;
    frontierSize = frontier_size;
  } else {
    if (indexConstant) {
      // Use a single value equal to the constant in the pattern
      idx_t constantValue = pattern.getEntry(indexPosition).getConstant();
      frontierBuffer.resize(sizeof(idx_t));
      thrust::fill(rmm::exec_policy(nullptr)->on(nullptr),
                   reinterpret_cast<idx_t*>(frontierBuffer.data()),
                   reinterpret_cast<idx_t*>(frontierBuffer.data()) + 1,
                   constantValue);
      frontier_ptr = reinterpret_cast<idx_t*>(frontierBuffer.data());
      frontierSize = 1;
    } else {
      // Making a sequence of values from zero to n where n is the highest ID present in the index.
      idx_t highestId = theIndex.getOffsetsSize() - 2;
      frontierBuffer.resize(sizeof(idx_t) * (highestId + 1));
      thrust::sequence(rmm::exec_policy(nullptr)->on(nullptr),
                       reinterpret_cast<idx_t*>(frontierBuffer.data()),
                       reinterpret_cast<idx_t*>(frontierBuffer.data()) + highestId + 1);
      frontier_ptr = reinterpret_cast<idx_t*>(frontierBuffer.data());
      frontierSize = highestId + 1;
    }
  }

  // Collect all the pointers needed to run the main kernel
  idx_t* columnA     = table.getColumn(0);
  idx_t* columnB     = table.getColumn(1);
  idx_t* columnC     = table.getColumn(2);
  idx_t* offsets     = theIndex.getOffsets();
  idx_t* indirection = theIndex.getIndirection();

  // Load balance the input
  rmm::device_buffer exsum_degree(sizeof(idx_t) * (frontierSize + 1));
  degree_iterator<idx_t> deg_it(offsets);
  deref_functor<degree_iterator<idx_t>, idx_t> deref(deg_it);
  thrust::fill(rmm::exec_policy(nullptr)->on(nullptr),
               reinterpret_cast<idx_t*>(exsum_degree.data()),
               reinterpret_cast<idx_t*>(exsum_degree.data()) + 1,
               0);
  thrust::transform(rmm::exec_policy(nullptr)->on(nullptr),
                    frontier_ptr,
                    frontier_ptr + frontierSize,
                    reinterpret_cast<idx_t*>(exsum_degree.data()) + 1,
                    deref);
  thrust::inclusive_scan(rmm::exec_policy(nullptr)->on(nullptr),
                         reinterpret_cast<idx_t*>(exsum_degree.data()) + 1,
                         reinterpret_cast<idx_t*>(exsum_degree.data()) + frontierSize + 1,
                         reinterpret_cast<idx_t*>(exsum_degree.data()) + 1);
  idx_t output_size;
  CUDA_TRY(hipMemcpy(&output_size,
                      reinterpret_cast<idx_t*>(exsum_degree.data()) + frontierSize,
                      sizeof(idx_t),
                      hipMemcpyDefault));

  idx_t num_blocks = (output_size + FIND_MATCHES_BLOCK_SIZE - 1) / FIND_MATCHES_BLOCK_SIZE;
  rmm::device_buffer block_bucket_offsets(sizeof(idx_t) * (num_blocks + 1));

  dim3 grid, block;
  block.x = 512;
  grid.x  = min((idx_t)MAXBLOCKS, (num_blocks / 512) + 1);
  compute_bucket_offsets_kernel<<<grid, block, 0, nullptr>>>(
    reinterpret_cast<idx_t*>(exsum_degree.data()),
    reinterpret_cast<idx_t*>(block_bucket_offsets.data()),
    frontierSize,
    output_size);

  // Allocate space for the result
  idx_t* outputA = nullptr;
  idx_t* outputB = nullptr;
  idx_t* outputC = nullptr;
  idx_t* outputD = nullptr;
  rmm::device_buffer outputABuffer;
  rmm::device_buffer outputBBuffer;
  rmm::device_buffer outputCBuffer;
  rmm::device_buffer outputDBuffer;
  if (pattern.getEntry(0).isVariable()) {
    outputABuffer.resize(sizeof(idx_t) * output_size);
    outputA = reinterpret_cast<idx_t*>(outputABuffer.data());
  }
  if (pattern.getEntry(1).isVariable()) {
    outputBBuffer.resize(sizeof(idx_t) * output_size);
    outputB = reinterpret_cast<idx_t*>(outputBBuffer.data());
  }
  if (pattern.getEntry(2).isVariable()) {
    outputCBuffer.resize(sizeof(idx_t) * output_size);
    outputC = reinterpret_cast<idx_t*>(outputCBuffer.data());
  }
  if (saveRowIds) {
    outputDBuffer.resize(sizeof(idx_t) * output_size);
    outputD = reinterpret_cast<idx_t*>(outputDBuffer.data());
  }

  // Get the constant pattern entries from the pattern to pass into the main kernel
  idx_t patternA = -1;
  idx_t patternB = -1;
  idx_t patternC = -1;
  if (!pattern.getEntry(0).isVariable()) { patternA = pattern.getEntry(0).getConstant(); }
  if (!pattern.getEntry(1).isVariable()) { patternB = pattern.getEntry(1).getConstant(); }
  if (!pattern.getEntry(2).isVariable()) { patternC = pattern.getEntry(2).getConstant(); }

  // Call the main kernel
  block.x = FIND_MATCHES_BLOCK_SIZE;
  grid.x  = min((idx_t)MAXBLOCKS,
               (output_size + (idx_t)FIND_MATCHES_BLOCK_SIZE - 1) / (idx_t)FIND_MATCHES_BLOCK_SIZE);
  findMatchesKernel<<<grid, block, 0, nullptr>>>(
    frontierSize,
    output_size,
    num_blocks,
    offsets,
    indirection,
    reinterpret_cast<idx_t*>(block_bucket_offsets.data()),
    reinterpret_cast<idx_t*>(exsum_degree.data()),
    frontier_ptr,
    columnA,
    columnB,
    columnC,
    outputA,
    outputB,
    outputC,
    outputD,
    patternA,
    patternB,
    patternC);

  // Get the non-null output columns
  std::vector<idx_t*> columns;
  std::vector<std::string> names;
  if (outputA != nullptr) {
    columns.push_back(outputA);
    names.push_back(pattern.getEntry(0).getVariable());
  }
  if (outputB != nullptr) {
    columns.push_back(outputB);
    names.push_back(pattern.getEntry(1).getVariable());
  }
  if (outputC != nullptr) {
    columns.push_back(outputC);
    names.push_back(pattern.getEntry(2).getVariable());
  }
  if (outputD != nullptr) {
    columns.push_back(outputD);
    names.push_back(pattern.getEntry(3).getVariable());
  }

  // Remove non-matches from result
  rmm::device_buffer flags(sizeof(int8_t) * output_size);

  idx_t* col_ptr = columns[0];
  thrust::transform(rmm::exec_policy(nullptr)->on(nullptr),
                    col_ptr,
                    col_ptr + output_size,
                    reinterpret_cast<int8_t*>(flags.data()),
                    notNegativeOne<idx_t, int8_t>());

  size_t tempSpaceSize = 0;
  rmm::device_buffer compactSize_d(sizeof(idx_t));
  hipcub::DeviceSelect::Flagged(nullptr,
                             tempSpaceSize,
                             col_ptr,
                             reinterpret_cast<int8_t*>(flags.data()),
                             col_ptr,
                             reinterpret_cast<idx_t*>(compactSize_d.data()),
                             output_size);
  rmm::device_buffer tempSpace(tempSpaceSize);
  hipcub::DeviceSelect::Flagged(tempSpace.data(),
                             tempSpaceSize,
                             col_ptr,
                             reinterpret_cast<int8_t*>(flags.data()),
                             col_ptr,
                             reinterpret_cast<idx_t*>(compactSize_d.data()),
                             output_size);
  idx_t compactSize_h;
  hipMemcpy(&compactSize_h, compactSize_d.data(), sizeof(idx_t), hipMemcpyDefault);

  for (size_t i = 1; i < columns.size(); i++) {
    col_ptr = columns[i];
    hipcub::DeviceSelect::Flagged(tempSpace.data(),
                               tempSpaceSize,
                               col_ptr,
                               reinterpret_cast<int8_t*>(flags.data()),
                               col_ptr,
                               reinterpret_cast<idx_t*>(compactSize_d.data()),
                               output_size);
  }

  // Put together the result to return
  db_result<idx_t> result;
  for (size_t i = 0; i < names.size(); i++) { result.addColumn(names[i]); }
  result.allocateColumns(compactSize_h);
  for (size_t i = 0; i < columns.size(); i++) {
    idx_t* outputPtr = result.getData(names[i]);
    idx_t* inputPtr  = columns[i];
    CUDA_TRY(hipMemcpy(outputPtr, inputPtr, sizeof(idx_t) * compactSize_h, hipMemcpyDefault));
  }

  // Return the result
  return result;
}

template db_result<int32_t> findMatches(db_pattern<int32_t>& pattern,
                                        db_table<int32_t>& table,
                                        int32_t* frontier,
                                        int32_t frontier_size,
                                        int indexPosition);
template db_result<int64_t> findMatches(db_pattern<int64_t>& pattern,
                                        db_table<int64_t>& table,
                                        int64_t* frontier,
                                        int64_t frontier_size,
                                        int indexPosition);
}  // namespace db
}  // namespace cugraph
