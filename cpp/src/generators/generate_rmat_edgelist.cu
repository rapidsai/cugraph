#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cugraph/detail/utility_wrappers.hpp>
#include <cugraph/graph_generators.hpp>
#include <cugraph/utilities/error.hpp>

#include <raft/handle.hpp>
#include <rmm/device_uvector.hpp>

#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>

#include <random>
#include <rmm/detail/error.hpp>
#include <tuple>

namespace cugraph {

template <typename vertex_t>
std::tuple<rmm::device_uvector<vertex_t>, rmm::device_uvector<vertex_t>> generate_rmat_edgelist(
  raft::handle_t const& handle,
  size_t scale,
  size_t num_edges,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool clip_and_flip)
{
  CUGRAPH_EXPECTS((size_t{1} << scale) <= static_cast<size_t>(std::numeric_limits<vertex_t>::max()),
                  "Invalid input argument: scale too large for vertex_t.");
  CUGRAPH_EXPECTS((a >= 0.0) && (b >= 0.0) && (c >= 0.0) && (a + b + c <= 1.0),
                  "Invalid input argument: a, b, c should be non-negative and a + b + c should not "
                  "be larger than 1.0.");

  // to limit memory footprint (1024 is a tuning parameter)
  auto max_edges_to_generate_per_iteration =
    static_cast<size_t>(handle.get_device_properties().multiProcessorCount) * 1024;
  rmm::device_uvector<float> rands(
    std::min(num_edges, max_edges_to_generate_per_iteration) * 2 * scale, handle.get_stream());

  rmm::device_uvector<vertex_t> srcs(num_edges, handle.get_stream());
  rmm::device_uvector<vertex_t> dsts(num_edges, handle.get_stream());

  size_t num_edges_generated{0};
  while (num_edges_generated < num_edges) {
    auto num_edges_to_generate =
      std::min(num_edges - num_edges_generated, max_edges_to_generate_per_iteration);
    auto pair_first = thrust::make_zip_iterator(thrust::make_tuple(srcs.begin(), dsts.begin())) +
                      num_edges_generated;

    detail::uniform_random_fill(
      handle.get_stream(), rands.data(), num_edges_to_generate * 2 * scale, 0.0f, 1.0f, seed);
    seed += num_edges_to_generate * 2 * scale;

    thrust::transform(
      handle.get_thrust_policy(),
      thrust::make_counting_iterator(size_t{0}),
      thrust::make_counting_iterator(num_edges_to_generate),
      pair_first,
      // if a + b == 0.0, a_norm is irrelevant, if (1.0 - (a+b)) == 0.0, c_norm is irrelevant
      [scale,
       clip_and_flip,
       rands    = rands.data(),
       a_plus_b = a + b,
       a_norm   = (a + b) > 0.0 ? a / (a + b) : 0.0,
       c_norm   = (1.0 - (a + b)) > 0.0 ? c / (1.0 - (a + b)) : 0.0] __device__(auto i) {
        vertex_t src{0};
        vertex_t dst{0};
        for (int bit = static_cast<int>(scale) - 1; bit >= 0; --bit) {
          auto r0          = rands[i * 2 * scale + 2 * bit];
          auto r1          = rands[i * 2 * scale + 2 * bit + 1];
          auto src_bit_set = r0 > a_plus_b;
          auto dst_bit_set = r1 > (src_bit_set ? c_norm : a_norm);
          if (clip_and_flip) {
            if (src == dst) {
              if (!src_bit_set && dst_bit_set) {
                src_bit_set = !src_bit_set;
                dst_bit_set = !dst_bit_set;
              }
            }
          }
          src += src_bit_set ? static_cast<vertex_t>(vertex_t{1} << bit) : 0;
          dst += dst_bit_set ? static_cast<vertex_t>(vertex_t{1} << bit) : 0;
        }
        return thrust::make_tuple(src, dst);
      });
    num_edges_generated += num_edges_to_generate;
  }

  return std::make_tuple(std::move(srcs), std::move(dsts));
}

template <typename vertex_t>
std::vector<std::tuple<rmm::device_uvector<vertex_t>, rmm::device_uvector<vertex_t>>>
generate_rmat_edgelists(raft::handle_t const& handle,
                        size_t n_edgelists,
                        size_t min_scale,
                        size_t max_scale,
                        size_t edge_factor,
                        generator_distribution_t component_distribution,
                        generator_distribution_t edge_distribution,
                        uint64_t seed,
                        bool clip_and_flip)
{
  CUGRAPH_EXPECTS(min_scale > 0, "minimum graph scale is 1.");
  CUGRAPH_EXPECTS(
    size_t{1} << max_scale <= static_cast<size_t>(std::numeric_limits<vertex_t>::max()),
    "Invalid input argument: scale too large for vertex_t.");

  std::vector<std::tuple<rmm::device_uvector<vertex_t>, rmm::device_uvector<vertex_t>>> output{};
  output.reserve(n_edgelists);
  std::vector<vertex_t> scale(n_edgelists);

  std::default_random_engine eng;
  eng.seed(seed);
  if (component_distribution == generator_distribution_t::UNIFORM) {
    std::uniform_int_distribution<vertex_t> dist(min_scale, max_scale);
    std::generate(scale.begin(), scale.end(), [&dist, &eng]() { return dist(eng); });
  } else {
    // May expose this as a parameter in the future
    std::exponential_distribution<float> dist(4);
    // The modulo is here to protect the range because exponential distribution is defined on
    // [0,infinity). With exponent 4 most values are between 0 and 1
    auto range = max_scale - min_scale;
    std::generate(scale.begin(), scale.end(), [&dist, &eng, &min_scale, &range]() {
      return min_scale + static_cast<vertex_t>(static_cast<float>(range) * dist(eng)) % range;
    });
  }

  // intialized to standard powerlaw values
  double a = 0.57, b = 0.19, c = 0.19;
  if (edge_distribution == generator_distribution_t::UNIFORM) {
    a = 0.25;
    b = a;
    c = a;
  }

  for (size_t i = 0; i < n_edgelists; i++) {
    output.push_back(generate_rmat_edgelist<vertex_t>(
      handle, scale[i], scale[i] * edge_factor, a, b, c, i, clip_and_flip));
  }
  return output;
}

template std::tuple<rmm::device_uvector<int32_t>, rmm::device_uvector<int32_t>>
generate_rmat_edgelist<int32_t>(raft::handle_t const& handle,
                                size_t scale,
                                size_t num_edges,
                                double a,
                                double b,
                                double c,
                                uint64_t seed,
                                bool clip_and_flip);

template std::tuple<rmm::device_uvector<int64_t>, rmm::device_uvector<int64_t>>
generate_rmat_edgelist<int64_t>(raft::handle_t const& handle,
                                size_t scale,
                                size_t num_edges,
                                double a,
                                double b,
                                double c,
                                uint64_t seed,
                                bool clip_and_flip);

template std::vector<std::tuple<rmm::device_uvector<int32_t>, rmm::device_uvector<int32_t>>>
generate_rmat_edgelists<int32_t>(raft::handle_t const& handle,
                                 size_t n_edgelists,
                                 size_t min_scale,
                                 size_t max_scale,
                                 size_t edge_factor,
                                 generator_distribution_t component_distribution,
                                 generator_distribution_t edge_distribution,
                                 uint64_t seed,
                                 bool clip_and_flip);

template std::vector<std::tuple<rmm::device_uvector<int64_t>, rmm::device_uvector<int64_t>>>
generate_rmat_edgelists<int64_t>(raft::handle_t const& handle,
                                 size_t n_edgelists,
                                 size_t min_scale,
                                 size_t max_scale,
                                 size_t edge_factor,
                                 generator_distribution_t component_distribution,
                                 generator_distribution_t edge_distribution,
                                 uint64_t seed,
                                 bool clip_and_flip);

}  // namespace cugraph
