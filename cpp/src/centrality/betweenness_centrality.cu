/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// Author: Xavier Cadet xcadet@nvidia.com
#include <cugraph.h>
#include <rmm_utils.h>

#include "betweenness_centrality.cuh"

namespace cugraph {
namespace detail {

template <typename VT, typename WT>
void BC<VT, WT>::setup() {
    // --- Set up parameters from graph adjList ---
    number_vertices  = graph-> adjList->offsets->size - 1;
    //number_vertices  = graph-> adjList->offsets->size;
    number_edges = graph->adjList->indices->size;
    offsets_ptr = (int*)graph->adjList->offsets->data;
    indices_ptr = (int*)graph->adjList->indices->data;

    edge_weights_ptr = static_cast<WT*>(graph->adjList->edge_data->data);
}

template <typename VT, typename WT>
void BC<VT, WT>::configure(WT *_betweenness) {
    // --- Working data allocation ---
    /*
    thrust::device_vector<VT> predecessors(number_vertices);
    thrust::device_vector<VT> sp_counters(sp_counters);

    thrust::device_vector<WT> sigmas(number_vertices);
    thrust::device_vector<WT> deltas(number_vertices);
    */
    /*
    ALLOC_TRY(&predecessors, sizeof(VT) * number_vertices, nullptr);
    ALLOC_TRY(&sp_counters, sizeof(VT) * number_vertices, nullptr);

    ALLOC_TRY(&sigmas, sizeof(WT) * number_vertices, nullptr);
    ALLOC_TRY(&deltas, sizeof(WT) * number_vertices, nullptr);
    */

    //VT *ptr = thrust::raw_pointer_cast(&predecessors[0]);
    //std::cout << "Predecessors is nullptr " << (ptr == nullptr) << "\n";
    // --- Bind betweenness output vector to internal ---
    betweenness = _betweenness;
    // --- Confirm that configuration went through ---
    configured = true;
}

template <typename VT, typename WT>
void BC<VT, WT>::compute() {
    CUGRAPH_EXPECTS(configured, "BC must be configured before computation");
    std::cout << "There are " << number_vertices << " nodes\n";

    thrust::device_vector<VT> d_predecessors(number_vertices, static_cast<VT>(0));
    thrust::device_vector<VT> d_sp_counters(number_vertices, static_cast<VT>(0));

    //thrust::device_vector<WT> d_betweenness(number_vertices, static_cast<WT>(0));

    thrust::host_vector<VT> h_predecessors(number_vertices, static_cast<VT>(0));
    thrust::host_vector<VT> h_sp_counters(number_vertices, static_cast<VT>(0));
    thrust::host_vector<WT> h_betweenness(number_vertices, static_cast<WT>(0));


    VT *d_predecessors_ptr = thrust::raw_pointer_cast(&d_predecessors[0]);
    VT *d_sp_counters_ptr = thrust::raw_pointer_cast(&d_sp_counters[0]);

    for (int source_vertex = 0; source_vertex < number_vertices ; ++source_vertex) {
        // Step 1) Singe-source shortest-path problem
        cugraph::sssp(graph, static_cast<WT *>(nullptr), d_predecessors_ptr, d_sp_counters_ptr, source_vertex);
        thrust::copy(d_predecessors.begin(), d_predecessors.end(), h_predecessors.begin());
        thrust::copy(d_sp_counters.begin(), d_sp_counters.end(), h_sp_counters.begin());
        hipDeviceSynchronize();
        // TODO(xcadet) Remove printing information
        std::cout << "Sigmas for source " << source_vertex << ": ";
        thrust::copy(h_sp_counters.begin(), h_sp_counters.end(), std::ostream_iterator<float>(std::cout, ", "));
        std::cout << "\n";
        // Step 2) Path reconstruction or S equivalent generation?

        //hipDeviceSynchronize();
        // sp_counters should be in floating points for the next phase
        // Step 3) Accumulation
        accumulate(h_betweenness, h_predecessors, h_sp_counters, source_vertex);
    }
    std::cout << "Betweeness: ";
    thrust::copy(h_betweenness.begin(), h_betweenness.end(), std::ostream_iterator<float>(std::cout, ", "));
    std::cout << "\n";
}


template <typename VT, typename WT>
void BC<VT, WT>::accumulate(thrust::host_vector<WT> &h_betweenness,
                            thrust::host_vector<VT> &h_predecessors,
                            thrust::host_vector<VT> &h_sp_counters,
                            VT source) {
    // TODO(xcadet) Remove the debugs messages
    std::cout << "[CUDA] Accumulating from " << source << "\n";
    std::cout << "\tPredecessors: ";
    thrust::copy(h_predecessors.begin(), h_predecessors.end(), std::ostream_iterator<float>(std::cout, ", "));
    std::cout << "\n";
    std::cout << "sp_counters: ";
    thrust::copy(h_sp_counters.begin(), h_sp_counters.end(), std::ostream_iterator<float>(std::cout, ", "));
    std::cout << "\n";

    thrust::host_vector<WT> h_deltas(number_vertices, static_cast<WT>(0));
    // TODO(xcadet) There is most likely a more efficient way to handle it
    for (VT w = 0; w < number_vertices; ++w) {
        WT factor = (static_cast<WT>(1.0) + h_deltas[w]) / static_cast<WT>(h_sp_counters[w]);
        // TODO(xcadet) The current SSSP implementation only stores 1 Node
        VT v = h_predecessors[w];
        if (v != -1) { // This node has predecessor
            h_deltas[v] = h_deltas[v] + static_cast<WT>(h_sp_counters[v]) * factor;
            std::cout << "Updated depencies for node " << v << " with " << h_deltas[v] << "\n";
        } // We should not updated our dependencies
        // The node is different than the source
        if (w != source) {
            h_betweenness[w] += h_deltas[w];
            std::cout << "Betweenness for " << w << " updated to " << h_betweenness[w] << "\n";
        }
    }

}

template <typename IndexType, typename BetweennessType>
void BC<IndexType, BetweennessType>::clean() {
    //ALLOC_FREE_TRY(predecessors, nullptr);
    //ALLOC_FREE_TRY(sp_counters, nullptr);
    //ALLOC_FREE_TRY(sigmas, nullptr);
    //ALLOC_FREE_TRY(deltas, nullptr);
    // ---  Betweenness is not ours ---
}

//  --
template <typename VT, typename ET, typename WT>
void _check_input(Graph *graph, WT *betweenness) {
}

} //namespace detail
  /**
  * ---------------------------------------------------------------------------*
  * @brief Native betweenness centrality
  *
  * @file betweenness_centrality.cu
  * --------------------------------------------------------------------------*/
  template <typename VT, typename ET, typename WT>
  void betweenness_centrality(Graph *graph, WT *betweenness) {
    CUGRAPH_EXPECTS(graph->adjList != nullptr, "Invalid API parameter: graph adjList is NULL");
    CUGRAPH_EXPECTS(betweenness != nullptr, "Invalid API parameter: output is nullptr");

    if (typeid(WT) != typeid(float) && typeid(WT) != typeid(double)) {
        CUGRAPH_FAIL("Unsupported betweenness data type, please use float or double");
    }
    // TODO fix me after gdf_column is removed from Graph
    CUGRAPH_EXPECTS(graph->adjList->offsets->dtype == GDF_INT32,
                    "Unsupported data type");
    CUGRAPH_EXPECTS(graph->adjList->indices->dtype == GDF_INT32,
                    "Unsupported data type");
    // Handle Unweighted
    if (!graph->adjList->edge_data) {

    } else { // Handle weighted graph
        CUGRAPH_EXPECTS(
            graph->adjList->edge_data->size == graph->adjList->indices->size,
            "Graph sizes mismatch");
        // TODO fix me after gdf_column is removed from Graph
        CUGRAPH_EXPECTS(graph->adjList->edge_data->dtype == GDF_FLOAT32 ||
                        graph->adjList->edge_data->dtype == GDF_FLOAT64,
                    "Invalid API parameter");
        // TODO fix me after gdf_column is removed from Graph
        // if (distances) CUGRAPH_EXPECTS(typeid(distances) == typeid(graph->adjList->edge_data), "distances and weights type mismatch");

        // BC relies on SSSP which is not defined for graphs with negative weight cycles
        // Warn user about any negative edges
        if (graph->prop && graph->prop->has_negative_edges == GDF_PROP_TRUE)
        std::cerr << "WARN: The graph has negative weight edges. BC will not "
                    "converge if the graph has negative weight cycles\n";
    }
    // Verify that WT is either float or double
    if (typeid(WT) == typeid(float) || typeid(WT) == typeid(double)) {
        cugraph::detail::BC<VT, WT> bc(graph);
        bc.configure(betweenness);
        bc.compute();
    } else { // Otherwise the datatype is invalid
        CUGRAPH_EXPECTS(graph->adjList->edge_data->dtype == GDF_FLOAT32 ||
                        graph->adjList->edge_data->dtype == GDF_FLOAT64,
                        "Invalid API parameter");
    }
  }

  // explicit instantiation
  template void betweenness_centrality<int, int, float>(Graph *graph, float *betweenness);
  template void betweenness_centrality<int, int, double>(Graph *graph, double *betweenness);
} //namespace cugraph