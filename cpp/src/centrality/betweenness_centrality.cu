/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <vector>

#include <thrust/transform.h>

#include <algorithms.hpp>
#include <graph.hpp>
#include "rmm_utils.h"

#include <utilities/error_utils.h>

#include <gunrock/gunrock.h>

namespace cugraph {

namespace gunrock {

template <typename VT, typename ET, typename WT, typename result_t>
void betweenness_centrality(experimental::GraphCSR<VT, ET, WT> const &graph,
                            result_t *result,
                            bool normalize,
                            VT const *sample_seeds    = nullptr,
                            VT number_of_sample_seeds = 0)
{
  hipStream_t stream{nullptr};

  //
  //  gunrock currently (as of 2/28/2020) only operates on a graph and results in
  //  host memory.  [That is, the first step in gunrock is to allocate device memory
  //  and copy the data into device memory, the last step is to allocate host memory
  //  and copy the results into the host memory]
  //
  //  They are working on fixing this.  In the meantime, to get the features into
  //  cuGraph we will first copy the graph back into local memory and when we are finished
  //  copy the result back into device memory.
  //
  std::vector<ET> v_offsets(graph.number_of_vertices + 1);
  std::vector<VT> v_indices(graph.number_of_edges);
  std::vector<result_t> v_result(graph.number_of_vertices);
  std::vector<float> v_sigmas(graph.number_of_vertices);
  std::vector<int> v_labels(graph.number_of_vertices);

  // fill them
  CUDA_TRY(hipMemcpy(v_offsets.data(),
                      graph.offsets,
                      sizeof(ET) * (graph.number_of_vertices + 1),
                      hipMemcpyDeviceToHost));
  CUDA_TRY(hipMemcpy(
    v_indices.data(), graph.indices, sizeof(VT) * graph.number_of_edges, hipMemcpyDeviceToHost));

  if (sample_seeds == nullptr) {
    bc(graph.number_of_vertices,
       graph.number_of_edges,
       v_offsets.data(),
       v_indices.data(),
       -1,
       v_result.data(),
       v_sigmas.data(),
       v_labels.data());
  } else {
    //
    //  Gunrock, as currently implemented
    //  doesn't support this method.
    //
    CUGRAPH_FAIL("gunrock doesn't currently support sampling seeds");
  }

  // copy to results
  CUDA_TRY(hipMemcpy(
    result, v_result.data(), sizeof(result_t) * graph.number_of_vertices, hipMemcpyHostToDevice));

  // normalize result
  if (normalize) {
    float denominator = (graph.number_of_vertices - 1) * (graph.number_of_vertices - 2);

    thrust::transform(rmm::exec_policy(stream)->on(stream),
                      result,
                      result + graph.number_of_vertices,
                      result,
                      [denominator] __device__(float f) { return (f * 2) / denominator; });
  } else {
    //
    //  gunrock answer needs to be doubled to match networkx
    //
    thrust::transform(rmm::exec_policy(stream)->on(stream),
                      result,
                      result + graph.number_of_vertices,
                      result,
                      [] __device__(float f) { return (f * 2); });
  }
}

}  // namespace gunrock

template <typename VT, typename ET, typename WT, typename result_t>
void betweenness_centrality(experimental::GraphCSR<VT, ET, WT> const &graph,
                            result_t *result,
                            bool normalize,
                            bool endpoints,
                            WT const *weight,
                            VT k,
                            VT const *vertices)
{
  //
  // NOTE:  gunrock implementation doesn't yet support the unused parameters:
  //     - endpoints
  //     - weight
  //     - k
  //     - vertices
  //
  // These parameters are present in the API to support future features.
  //
  gunrock::betweenness_centrality(graph, result, normalize);
}

template void betweenness_centrality<int, int, float, float>(
  experimental::GraphCSR<int, int, float> const &,
  float *,
  bool,
  bool,
  float const *,
  int,
  int const *);

}  // namespace cugraph
