#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <omp.h>
#include <thrust/execution_policy.h>
#include <thrust/extrema.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/tuple.h>
#include <cub/device/device_run_length_encode.cuh>
#include <sstream>
#include <string>
#include <vector>
#include "rmm_utils.h"
#include "snmg/utils.cuh"
#include "utilities/graph_utils.cuh"

namespace cugraph {
namespace snmg {

template <typename idx_t, typename val_t>
class communicator {
 public:
  idx_t* maxIds;
  idx_t* rowCounts;
  idx_t** rowPtrs;
  idx_t** colPtrs;
  unsigned long long int** reductionSpace;
  val_t** valPtrs;
  communicator(idx_t p)
  {
    maxIds    = reinterpret_cast<idx_t*>(malloc(sizeof(idx_t) * p));
    rowCounts = reinterpret_cast<idx_t*>(malloc(sizeof(idx_t) * p * p));
    rowPtrs   = reinterpret_cast<idx_t**>(malloc(sizeof(idx_t*) * p));
    colPtrs   = reinterpret_cast<idx_t**>(malloc(sizeof(idx_t*) * p));
    valPtrs   = reinterpret_cast<val_t**>(malloc(sizeof(val_t*) * p));
    reductionSpace =
      reinterpret_cast<unsigned long long int**>(malloc(sizeof(unsigned long long int*) * p));
  }
  ~communicator()
  {
    free(maxIds);
    free(rowCounts);
    free(rowPtrs);
    free(colPtrs);
    free(reductionSpace);
    free(valPtrs);
  }
};

void serializeMessage(cugraph::snmg::SNMGinfo& env, std::string message)
{
  auto i = env.get_thread_num();
  auto p = env.get_num_threads();
  for (int j = 0; j < p; j++) {
    if (i == j) std::cout << "Thread " << i << ": " << message << "\n";
#pragma omp barrier
  }
}

template <typename idx_t, typename val_t>
__global__ void __launch_bounds__(CUDA_MAX_KERNEL_THREADS)
  findStartRange(idx_t n, idx_t* result, val_t edgeCount, val_t* scanned)
{
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n; i += gridDim.x * blockDim.x)
    if (scanned[i] < edgeCount && scanned[i + 1] >= edgeCount) *result = i + 1;
}

// Define kernel for copying run length encoded values into offset slots.
template <typename T>
__global__ void offsetsKernel(T runCounts, T* unique, T* counts, T* offsets)
{
  uint64_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < runCounts) offsets[unique[tid]] = counts[tid];
}

template <typename T>
__global__ void writeSingleValue(T* ptr, T val)
{
  uint64_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid == 0) *ptr = val;
}

template <typename idx_t, typename val_t>
void snmg_coo2csr_impl(size_t* part_offsets,
                       bool free_input,
                       void** comm1,
                       gdf_column* cooRow,
                       gdf_column* cooCol,
                       gdf_column* cooVal,
                       gdf_column* csrOff,
                       gdf_column* csrInd,
                       gdf_column* csrVal)
{
  cugraph::snmg::SNMGinfo env;
  auto i = env.get_thread_num();
  auto p = env.get_num_threads();

  // First thread allocates communicator object
  if (i == 0) {
    cugraph::snmg::communicator<idx_t, val_t>* comm =
      new cugraph::snmg::communicator<idx_t, val_t>(p);
    *comm1 = reinterpret_cast<void*>(comm);
  }
#pragma omp barrier

  cugraph::snmg::communicator<idx_t, val_t>* comm =
    reinterpret_cast<cugraph::snmg::communicator<idx_t, val_t>*>(*comm1);

  // Each thread scans its cooRow and cooCol for the greatest ID
  idx_t size     = cooRow->size;
  idx_t* max_ptr = thrust::max_element(rmm::exec_policy(nullptr)->on(nullptr),
                                       reinterpret_cast<idx_t*>(cooRow->data),
                                       reinterpret_cast<idx_t*>(cooRow->data) + size);
  idx_t rowID;
  hipMemcpy(&rowID, max_ptr, sizeof(idx_t), hipMemcpyDefault);
  max_ptr = thrust::max_element(rmm::exec_policy(nullptr)->on(nullptr),
                                reinterpret_cast<idx_t*>(cooCol->data),
                                reinterpret_cast<idx_t*>(cooCol->data) + size);
  idx_t colID;
  hipMemcpy(&colID, max_ptr, sizeof(idx_t), hipMemcpyDefault);
  comm->maxIds[i] = max(rowID, colID);

#pragma omp barrier

  // First thread finds maximum global ID
  if (i == 0) {
    idx_t best_id = comm->maxIds[0];
    for (int j = 0; j < p; j++) best_id = max(best_id, comm->maxIds[j]);
    comm->maxIds[0] = best_id;
  }
#pragma omp barrier

  // Each thread allocates space for the source node counts
  idx_t maxId       = comm->maxIds[0];
  idx_t offsetsSize = maxId + 2;
  unsigned long long int* sourceCounts;
  ALLOC_TRY(&sourceCounts, sizeof(unsigned long long int) * offsetsSize, nullptr);
  hipMemset(sourceCounts, 0, sizeof(unsigned long long int) * offsetsSize);

  // Each thread computes the source node counts for its owned rows
  dim3 nthreads, nblocks;
  nthreads.x = min(size, static_cast<idx_t>(CUDA_MAX_KERNEL_THREADS));
  nthreads.y = 1;
  nthreads.z = 1;
  nblocks.x  = min(static_cast<idx_t>((size + nthreads.x - 1) / nthreads.x),
                  static_cast<idx_t>(env.get_num_sm() * 32));
  nblocks.y  = 1;
  nblocks.z  = 1;
  cugraph::detail::degree_coo<idx_t, unsigned long long int>
    <<<nblocks, nthreads>>>(size, size, reinterpret_cast<idx_t*>(cooRow->data), sourceCounts);
  hipDeviceSynchronize();
  CUDA_CHECK_LAST();

  // Threads globally reduce their local source node counts to get the global ones
  unsigned long long int* sourceCountsTemp;
  ALLOC_TRY(&sourceCountsTemp, sizeof(unsigned long long int) * offsetsSize, nullptr);
  comm->reductionSpace[i] = sourceCountsTemp;
#pragma omp barrier

  cugraph::snmg::treeReduce<unsigned long long int, thrust::plus<unsigned long long int>>(
    env, offsetsSize, sourceCounts, comm->reductionSpace);
  cugraph::snmg::treeBroadcast(env, offsetsSize, sourceCounts, comm->reductionSpace);

  // Each thread takes the exclusive scan of the global counts
  thrust::exclusive_scan(rmm::exec_policy(nullptr)->on(nullptr),
                         sourceCountsTemp,
                         sourceCountsTemp + offsetsSize,
                         sourceCountsTemp);
  ALLOC_FREE_TRY(sourceCounts, nullptr);
  hipDeviceSynchronize();
  CUDA_CHECK_LAST();

  // Each thread reads the global edgecount
  unsigned long long int globalEdgeCount;
  hipMemcpy(&globalEdgeCount,
             sourceCountsTemp + maxId + 1,
             sizeof(unsigned long long int),
             hipMemcpyDefault);
  CUDA_CHECK_LAST();

  // Each thread searches the global source node counts prefix sum to find the start of its vertex
  // ID range
  idx_t myStartVertex = 0;
  if (i != 0) {
    unsigned long long int edgeCount = (globalEdgeCount / p) * i;
    idx_t* vertexRangeStart;
    ALLOC_TRY(&vertexRangeStart, sizeof(idx_t), nullptr);
    dim3 nthreads, nblocks;
    nthreads.x = min(offsetsSize, static_cast<idx_t>(CUDA_MAX_KERNEL_THREADS));
    nthreads.y = 1;
    nthreads.z = 1;
    nblocks.x =
      min((offsetsSize + nthreads.x - 1) / nthreads.x, static_cast<idx_t>(env.get_num_sm() * 32));
    nblocks.y = 1;
    nblocks.z = 1;
    cugraph::snmg::findStartRange<<<nblocks, nthreads>>>(
      maxId, vertexRangeStart, edgeCount, sourceCountsTemp);
    hipDeviceSynchronize();
    hipMemcpy(&myStartVertex, vertexRangeStart, sizeof(idx_t), hipMemcpyDefault);
    part_offsets[i] = myStartVertex;
    ALLOC_FREE_TRY(vertexRangeStart, nullptr);
  } else {
    part_offsets[0] = 0;
    part_offsets[p] = maxId + 1;
  }
  CUDA_CHECK_LAST();
#pragma omp barrier

  // Each thread determines how many edges it will have in its partition
  idx_t myEndVertex = part_offsets[i + 1];
  unsigned long long int startEdge;
  unsigned long long int endEdge;
  hipMemcpy(&startEdge,
             sourceCountsTemp + myStartVertex,
             sizeof(unsigned long long int),
             hipMemcpyDefault);
  hipMemcpy(
    &endEdge, sourceCountsTemp + myEndVertex, sizeof(unsigned long long int), hipMemcpyDefault);
  ALLOC_FREE_TRY(sourceCountsTemp, nullptr);
  idx_t myEdgeCount = endEdge - startEdge;

  // Each thread sorts its cooRow, cooCol, and cooVal
  idx_t *cooRowTemp, *cooColTemp;
  val_t* cooValTemp;
  ALLOC_TRY(&cooRowTemp, sizeof(idx_t) * size, nullptr);
  ALLOC_TRY(&cooColTemp, sizeof(idx_t) * size, nullptr);
  hipMemcpy(cooRowTemp, cooRow->data, sizeof(idx_t) * size, hipMemcpyDefault);
  hipMemcpy(cooColTemp, cooCol->data, sizeof(idx_t) * size, hipMemcpyDefault);
  if (cooVal != nullptr) {
    ALLOC_TRY(&cooValTemp, sizeof(val_t) * size, nullptr);
    hipMemcpy(cooValTemp, cooVal->data, sizeof(val_t) * size, hipMemcpyDefault);
  } else
    cooValTemp = nullptr;
  CUDA_CHECK_LAST();

  if (cooValTemp != nullptr) {
    auto zippy = thrust::make_zip_iterator(thrust::make_tuple(cooRowTemp, cooColTemp));
    thrust::sort_by_key(rmm::exec_policy(nullptr)->on(nullptr), zippy, zippy + size, cooValTemp);
  } else {
    auto zippy = thrust::make_zip_iterator(thrust::make_tuple(cooRowTemp, cooColTemp));
    thrust::sort(rmm::exec_policy(nullptr)->on(nullptr), zippy, zippy + size);
  }
  hipDeviceSynchronize();
  CUDA_CHECK_LAST();

  // Each thread determines the count of rows it needs to transfer to each other thread
  idx_t localMinId, localMaxId;
  hipMemcpy(&localMinId, cooRowTemp, sizeof(idx_t), hipMemcpyDefault);
  hipMemcpy(&localMaxId, cooRowTemp + size - 1, sizeof(idx_t), hipMemcpyDefault);
  idx_t* endPositions;
  ALLOC_TRY(&endPositions, sizeof(idx_t) * (p - 1), nullptr);
  for (int j = 0; j < p - 1; j++) {
    idx_t endVertexId = part_offsets[j + 1];
    if (endVertexId <= localMinId) {
      // Write out zero for this position
      cugraph::snmg::writeSingleValue<<<1, 256>>>(endPositions + j, static_cast<idx_t>(0));
    } else if (endVertexId >= localMaxId) {
      // Write out size for this position
      cugraph::snmg::writeSingleValue<<<1, 256>>>(endPositions + j, size);
    } else if (endVertexId > localMinId && endVertexId < localMaxId) {
      dim3 nthreads, nblocks;
      nthreads.x = min(size, static_cast<idx_t>(CUDA_MAX_KERNEL_THREADS));
      nthreads.y = 1;
      nthreads.z = 1;
      nblocks.x =
        min((size + nthreads.x - 1) / nthreads.x, static_cast<idx_t>(env.get_num_sm() * 32));
      nblocks.y = 1;
      nblocks.z = 1;
      cugraph::snmg::findStartRange<<<nblocks, nthreads>>>(
        size, endPositions + j, endVertexId, cooRowTemp);
    }
  }
  hipDeviceSynchronize();
  CUDA_CHECK_LAST();
  std::vector<idx_t> positions(p + 1);
  hipMemcpy(&positions[1], endPositions, sizeof(idx_t) * (p - 1), hipMemcpyDefault);
  ALLOC_FREE_TRY(endPositions, nullptr);
  CUDA_CHECK_LAST();
  positions[0]       = 0;
  positions[p]       = size;
  idx_t* myRowCounts = comm->rowCounts + (i * p);
  for (int j = 0; j < p; j++) { myRowCounts[j] = positions[j + 1] - positions[j]; }

#pragma omp barrier

  int myRowCount = 0;
  for (int j = 0; j < p; j++) {
    idx_t* otherRowCounts = comm->rowCounts + (j * p);
    myRowCount += otherRowCounts[i];
  }

  // Each thread allocates space to receive their rows from others
  idx_t *cooRowNew, *cooColNew;
  val_t* cooValNew;
  ALLOC_TRY(&cooRowNew, sizeof(idx_t) * myRowCount, nullptr);
  ALLOC_TRY(&cooColNew, sizeof(idx_t) * myRowCount, nullptr);
  if (cooValTemp != nullptr) {
    ALLOC_TRY(&cooValNew, sizeof(val_t) * myRowCount, nullptr);
  } else {
    cooValNew = nullptr;
  }
  comm->rowPtrs[i] = cooRowNew;
  comm->colPtrs[i] = cooColNew;
  comm->valPtrs[i] = cooValNew;
  CUDA_CHECK_LAST();
  hipDeviceSynchronize();
#pragma omp barrier

  // Each thread copies the rows needed by other threads to them
  for (int other = 0; other < p; other++) {
    idx_t offset   = 0;
    idx_t rowCount = myRowCounts[other];
    for (int prev = 0; prev < i; prev++) {
      idx_t* prevRowCounts = comm->rowCounts + (prev * p);
      offset += prevRowCounts[other];
    }

    if (rowCount > 0) {
      hipMemcpy(comm->rowPtrs[other] + offset,
                 cooRowTemp + positions[other],
                 rowCount * sizeof(idx_t),
                 hipMemcpyDefault);
      hipMemcpy(comm->colPtrs[other] + offset,
                 cooColTemp + positions[other],
                 rowCount * sizeof(idx_t),
                 hipMemcpyDefault);
      if (cooValTemp != nullptr) {
        hipMemcpy(comm->valPtrs[other],
                   cooValTemp + positions[other],
                   rowCount * sizeof(idx_t),
                   hipMemcpyDefault);
      }
    }
  }
  CUDA_CHECK_LAST();
  cugraph::snmg::sync_all();

  // Each thread frees up the input if allowed
  ALLOC_FREE_TRY(cooRowTemp, nullptr);
  ALLOC_FREE_TRY(cooColTemp, nullptr);
  if (cooValTemp != nullptr) { ALLOC_FREE_TRY(cooValTemp, nullptr); }
  if (free_input) {
    ALLOC_FREE_TRY(cooRow->data, nullptr);
    ALLOC_FREE_TRY(cooCol->data, nullptr);
    if (cooVal != nullptr) { ALLOC_FREE_TRY(cooVal->data, nullptr); }
  }

  // Each thread applies the offset to it's row column to get locally zero-based
  idx_t myOffset = part_offsets[i];
  thrust::transform(rmm::exec_policy(nullptr)->on(nullptr),
                    cooRowNew,
                    cooRowNew + myRowCount,
                    thrust::make_constant_iterator(myOffset * -1),
                    cooRowNew,
                    thrust::plus<idx_t>());

  // Each thread does a local coo2csr on its rows
  if (cooValNew != nullptr) {
    auto zippy = thrust::make_zip_iterator(thrust::make_tuple(cooRowNew, cooColNew));
    thrust::sort_by_key(
      rmm::exec_policy(nullptr)->on(nullptr), zippy, zippy + myRowCount, cooValNew);
  } else {
    auto zippy = thrust::make_zip_iterator(thrust::make_tuple(cooRowNew, cooColNew));
    thrust::sort(rmm::exec_policy(nullptr)->on(nullptr), zippy, zippy + myEdgeCount);
  }

  CUDA_CHECK_LAST();

  localMaxId = part_offsets[i + 1] - part_offsets[i] - 1;
  idx_t* offsets;
  ALLOC_TRY(&offsets, (localMaxId + 2) * sizeof(idx_t), nullptr);
  hipMemset(offsets, 0, (localMaxId + 2) * sizeof(idx_t));
  idx_t *unique, *counts, *runcount;
  ALLOC_TRY(&unique, (localMaxId + 1) * sizeof(idx_t), nullptr);
  ALLOC_TRY(&counts, (localMaxId + 1) * sizeof(idx_t), nullptr);
  ALLOC_TRY(&runcount, sizeof(idx_t), nullptr);
  void* tmpStorage = nullptr;
  size_t tmpBytes  = 0;
  hipcub::DeviceRunLengthEncode::Encode(
    tmpStorage, tmpBytes, cooRowNew, unique, counts, runcount, myRowCount);
  ALLOC_TRY(&tmpStorage, tmpBytes, nullptr);
  hipcub::DeviceRunLengthEncode::Encode(
    tmpStorage, tmpBytes, cooRowNew, unique, counts, runcount, myRowCount);
  ALLOC_FREE_TRY(tmpStorage, nullptr);

  hipDeviceSynchronize();
  idx_t runCount_h;
  hipMemcpy(&runCount_h, runcount, sizeof(idx_t), hipMemcpyDefault);
  int threadsPerBlock = 1024;
  int numBlocks       = (runCount_h + threadsPerBlock - 1) / threadsPerBlock;

  CUDA_CHECK_LAST();

  cugraph::snmg::offsetsKernel<<<numBlocks, threadsPerBlock>>>(runCount_h, unique, counts, offsets);

  CUDA_CHECK_LAST();

  thrust::exclusive_scan(
    rmm::exec_policy(nullptr)->on(nullptr), offsets, offsets + localMaxId + 2, offsets);
  ALLOC_FREE_TRY(cooRowNew, nullptr);
  ALLOC_FREE_TRY(unique, nullptr);
  ALLOC_FREE_TRY(counts, nullptr);
  ALLOC_FREE_TRY(runcount, nullptr);

  // Each thread sets up the results into the provided gdf_columns
  cugraph::detail::gdf_col_set_defaults(csrOff);
  csrOff->dtype = cooRow->dtype;
  csrOff->size  = localMaxId + 2;
  csrOff->data  = offsets;
  cugraph::detail::gdf_col_set_defaults(csrInd);
  csrInd->dtype = cooRow->dtype;
  csrInd->size  = myRowCount;
  csrInd->data  = cooColNew;
  if (cooValNew != nullptr) {
    cugraph::detail::gdf_col_set_defaults(cooVal);
    csrVal->dtype = cooVal->dtype;
    csrVal->size  = myRowCount;
    csrVal->data  = cooValNew;
  }
#pragma omp barrier

  // First thread deletes communicator object
  if (i == 0) { delete comm; }
}

}  // namespace snmg

void snmg_coo2csr(size_t* part_offsets,
                  bool free_input,
                  void** comm1,
                  gdf_column* cooRow,
                  gdf_column* cooCol,
                  gdf_column* cooVal,
                  gdf_column* csrOff,
                  gdf_column* csrInd,
                  gdf_column* csrVal)
{
  CUGRAPH_EXPECTS(part_offsets != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(cooRow != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(cooCol != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(csrOff != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(csrInd != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(comm1 != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(cooRow->size > 0, "Invalid API parameter");
  CUGRAPH_EXPECTS(cooCol->size > 0, "Invalid API parameter");
  CUGRAPH_EXPECTS(cooCol->dtype == cooRow->dtype, "Invalid API parameter");

  if (cooVal == nullptr) {
    if (cooRow->dtype == GDF_INT32) {
      return snmg::snmg_coo2csr_impl<int32_t, float>(
        part_offsets, free_input, comm1, cooRow, cooCol, cooVal, csrOff, csrInd, csrVal);
    } else if (cooRow->dtype == GDF_INT64) {
      return snmg::snmg_coo2csr_impl<int64_t, float>(
        part_offsets, free_input, comm1, cooRow, cooCol, cooVal, csrOff, csrInd, csrVal);
    } else
      CUGRAPH_FAIL("Unsupported data type");
  } else {
    if (cooRow->dtype == GDF_INT32 && cooVal->dtype == GDF_FLOAT32) {
      return snmg::snmg_coo2csr_impl<int32_t, float>(
        part_offsets, free_input, comm1, cooRow, cooCol, cooVal, csrOff, csrInd, csrVal);
    } else if (cooRow->dtype == GDF_INT32 && cooVal->dtype == GDF_FLOAT64) {
      return snmg::snmg_coo2csr_impl<int32_t, double>(
        part_offsets, free_input, comm1, cooRow, cooCol, cooVal, csrOff, csrInd, csrVal);
    } else if (cooRow->dtype == GDF_INT64 && cooVal->dtype == GDF_FLOAT32) {
      return snmg::snmg_coo2csr_impl<int64_t, float>(
        part_offsets, free_input, comm1, cooRow, cooCol, cooVal, csrOff, csrInd, csrVal);
    } else if (cooRow->dtype == GDF_INT64 && cooVal->dtype == GDF_FLOAT64) {
      return snmg::snmg_coo2csr_impl<int64_t, double>(
        part_offsets, free_input, comm1, cooRow, cooCol, cooVal, csrOff, csrInd, csrVal);
    } else
      CUGRAPH_FAIL("Unsupported data type");
  }
}

}  // namespace cugraph