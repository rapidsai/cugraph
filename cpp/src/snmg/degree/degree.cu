#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "degree.cuh"
namespace cugraph {
/**
 * Single node multi-GPU method for degree calculation on a partitioned graph.
 * @param x Indicates whether to compute in degree, out degree, or the sum of both.
 *    0 = in + out degree
 *    1 = in-degree
 *    2 = out-degree
 * @param part_off The vertex partitioning of the global graph
 * @param off The offsets array of the local partition
 * @param ind The indices array of the local partition
 * @param degree Pointer to pointers to memory on each GPU for the result
 * @return Error code
 */
template<typename idx_t>
gdf_error snmg_degree(int x, size_t* part_off, idx_t* off, idx_t* ind, idx_t** degree) {
  sync_all();
  SNMGinfo env;
  auto i = env.get_thread_num();
  auto p = env.get_num_threads();

  // Getting the global and local vertices and edges
  size_t glob_v = part_off[p];
  size_t loc_v = part_off[i + 1] - part_off[i];
  idx_t tmp;
  CUDA_TRY(hipMemcpy(&tmp, &off[loc_v], sizeof(idx_t), hipMemcpyDeviceToHost));
  size_t loc_e = tmp;

  // Allocating the local result array, and setting all entries to zero.
  idx_t* local_result;
  ALLOC_TRY((void** )&local_result, glob_v * sizeof(idx_t), nullptr);
  thrust::fill(rmm::exec_policy(nullptr)->on(nullptr), local_result, local_result + glob_v, 0);

  // In-degree
  if (x == 1 || x == 0) {
    dim3 nthreads, nblocks;
    nthreads.x = min(static_cast<idx_t>(loc_e), static_cast<idx_t>(CUDA_MAX_KERNEL_THREADS));
    nthreads.y = 1;
    nthreads.z = 1;
    nblocks.x = min(static_cast<idx_t>((loc_e + nthreads.x - 1) / nthreads.x),
                    static_cast<idx_t>(env.get_num_sm() * 32));
    nblocks.y = 1;
    nblocks.z = 1;
    degree_coo<idx_t, idx_t> <<<nblocks, nthreads>>>(static_cast<idx_t>(loc_e),
                                                     static_cast<idx_t>(loc_e),
                                                     ind,
                                                     local_result);
    cudaCheckError();
  }

  // Out-degree
  if (x == 2 || x == 0) {
    dim3 nthreads, nblocks;
    nthreads.x = min(static_cast<idx_t>(loc_v), static_cast<idx_t>(CUDA_MAX_KERNEL_THREADS));
    nthreads.y = 1;
    nthreads.z = 1;
    nblocks.x = min(static_cast<idx_t>((loc_v + nthreads.x - 1) / nthreads.x),
                    static_cast<idx_t>(env.get_num_sm() * 32));
    nblocks.y = 1;
    nblocks.z = 1;
    degree_offsets<idx_t, idx_t> <<<nblocks, nthreads>>>(static_cast<idx_t>(loc_v),
                                                         static_cast<idx_t>(loc_e),
                                                         off,
                                                         local_result + part_off[i]);
    cudaCheckError();
  }

  // Combining the local results into global results
  sync_all();
  treeReduce<idx_t, thrust::plus<idx_t> >(env, glob_v, local_result, degree);

  // Broadcasting the global result to all GPUs
  treeBroadcast(env, glob_v, local_result, degree);

  return GDF_SUCCESS;
}

template gdf_error snmg_degree<int>(int x, size_t* part_off, int* off, int* ind, int** degree);

template<>
gdf_error snmg_degree<int64_t>(int x,
                               size_t* part_off,
                               int64_t* off,
                               int64_t* ind,
                               int64_t** degree) {
  sync_all();
  SNMGinfo env;
  auto i = env.get_thread_num();
  auto p = env.get_num_threads();

  // Getting the global and local vertices and edges
  size_t glob_v = part_off[p];
  size_t loc_v = part_off[i + 1] - part_off[i];
  int64_t tmp;
  CUDA_TRY(hipMemcpy(&tmp, &off[loc_v], sizeof(int64_t), hipMemcpyDeviceToHost));
  size_t loc_e = tmp;

  // Allocating the local result array, and setting all entries to zero.
  int64_t* local_result;
  ALLOC_TRY((void** )&local_result, glob_v * sizeof(int64_t), nullptr);
  thrust::fill(rmm::exec_policy(nullptr)->on(nullptr), local_result, local_result + glob_v, 0);

  // In-degree
  if (x == 1 || x == 0) {
    dim3 nthreads, nblocks;
    nthreads.x = min(static_cast<int64_t>(loc_e), static_cast<int64_t>(CUDA_MAX_KERNEL_THREADS));
    nthreads.y = 1;
    nthreads.z = 1;
    nblocks.x = min(static_cast<int64_t>((loc_e + nthreads.x - 1) / nthreads.x),
                    static_cast<int64_t>(env.get_num_sm() * 32));
    nblocks.y = 1;
    nblocks.z = 1;
    degree_coo<int64_t, double> <<<nblocks, nthreads>>>(static_cast<int64_t>(loc_e),
                                                        static_cast<int64_t>(loc_e),
                                                        ind,
                                                        reinterpret_cast<double*>(local_result));
    cudaCheckError();
  }

  // Out-degree
  if (x == 2 || x == 0) {
    dim3 nthreads, nblocks;
    nthreads.x = min(static_cast<int64_t>(loc_v), static_cast<int64_t>(CUDA_MAX_KERNEL_THREADS));
    nthreads.y = 1;
    nthreads.z = 1;
    nblocks.x = min(static_cast<int64_t>((loc_v + nthreads.x - 1) / nthreads.x),
                    static_cast<int64_t>(env.get_num_sm() * 32));
    nblocks.y = 1;
    nblocks.z = 1;
    degree_offsets<int64_t, double> <<<nblocks, nthreads>>>(static_cast<int64_t>(loc_v),
                                                            static_cast<int64_t>(loc_e),
                                                            off,
                                                            reinterpret_cast<double*>(local_result
                                                                + part_off[i]));
    cudaCheckError();
  }

  // Convert the values written as doubles back to int64:
  dim3 nthreads, nblocks;
  nthreads.x = min(static_cast<int64_t>(glob_v), static_cast<int64_t>(CUDA_MAX_KERNEL_THREADS));
  nthreads.y = 1;
  nthreads.z = 1;
  nblocks.x = min(static_cast<int64_t>((glob_v + nthreads.x - 1) / nthreads.x),
                  static_cast<int64_t>(env.get_num_sm() * 32));
  nblocks.y = 1;
  nblocks.z = 1;
  type_convert<double, int64_t> <<<nblocks, nthreads>>>(reinterpret_cast<double*>(local_result), glob_v);
  cudaCheckError();

  // Combining the local results into global results
  treeReduce<int64_t, thrust::plus<int64_t> >(env, glob_v, local_result, degree);

  // Broadcasting the global result to all GPUs
  treeBroadcast(env, glob_v, local_result, degree);

  return GDF_SUCCESS;
}

}//namespace


template<typename idx_t>
gdf_error gdf_snmg_degree_impl(int x,
                               size_t* part_offsets,
                               gdf_column* off,
                               gdf_column* ind,
                               gdf_column** x_cols) {
  GDF_REQUIRE(off->size > 0, GDF_INVALID_API_CALL);
  GDF_REQUIRE(ind->size > 0, GDF_INVALID_API_CALL);
  GDF_REQUIRE(off->dtype == ind->dtype, GDF_UNSUPPORTED_DTYPE);
  GDF_REQUIRE(off->null_count + ind->null_count == 0, GDF_VALIDITY_UNSUPPORTED);

  gdf_error status;
  auto p = omp_get_num_threads();

  idx_t* degree[p];
  for (auto i = 0; i < p; ++i) {
    GDF_REQUIRE(x_cols[i] != nullptr, GDF_INVALID_API_CALL);
    GDF_REQUIRE(x_cols[i]->size > 0, GDF_INVALID_API_CALL);
    degree[i] = static_cast<idx_t*>(x_cols[i]->data);
  }

  status = cugraph::snmg_degree(x,
                                part_offsets,
                                static_cast<idx_t*>(off->data),
                                static_cast<idx_t*>(ind->data),
                                degree);
  return status;
}

gdf_error gdf_snmg_degree(int x,
                          size_t* part_offsets,
                          gdf_column* off,
                          gdf_column* ind,
                          gdf_column** x_cols) {
  GDF_REQUIRE(part_offsets != nullptr, GDF_INVALID_API_CALL);
  GDF_REQUIRE(off != nullptr, GDF_INVALID_API_CALL);
  GDF_REQUIRE(ind != nullptr, GDF_INVALID_API_CALL);
  GDF_REQUIRE(x_cols != nullptr, GDF_INVALID_API_CALL);
  switch (off->dtype) {
    case GDF_INT32:
      return gdf_snmg_degree_impl<int32_t>(x, part_offsets, off, ind, x_cols);
    case GDF_INT64:
      return gdf_snmg_degree_impl<int64_t>(x, part_offsets, off, ind, x_cols);
    default:
      return GDF_INVALID_API_CALL;
  }
}
