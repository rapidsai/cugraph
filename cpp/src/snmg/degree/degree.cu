#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "degree.cuh"
namespace cugraph {
namespace snmg {
/**
 * Single node multi-GPU method for degree calculation on a partitioned graph.
 * @param x Indicates whether to compute in degree, out degree, or the sum of both.
 *    0 = in + out degree
 *    1 = in-degree
 *    2 = out-degree
 * @param part_off The vertex partitioning of the global graph
 * @param off The offsets array of the local partition
 * @param ind The indices array of the local partition
 * @param degree Pointer to pointers to memory on each GPU for the result
 * @return Error code
 */
template <typename idx_t>
void snmg_degree(int x, size_t* part_off, idx_t* off, idx_t* ind, idx_t** degree)
{
  sync_all();
  SNMGinfo env;
  auto i = env.get_thread_num();
  auto p = env.get_num_threads();

  // Getting the global and local vertices and edges
  size_t glob_v = part_off[p];
  size_t loc_v  = part_off[i + 1] - part_off[i];
  idx_t tmp;
  CUDA_TRY(hipMemcpy(&tmp, &off[loc_v], sizeof(idx_t), hipMemcpyDeviceToHost));
  size_t loc_e = tmp;

  // Allocating the local result array, and setting all entries to zero.
  idx_t* local_result;
  ALLOC_TRY((void**)&local_result, glob_v * sizeof(idx_t), nullptr);
  thrust::fill(rmm::exec_policy(nullptr)->on(nullptr), local_result, local_result + glob_v, 0);

  // In-degree
  if (x == 1 || x == 0) {
    dim3 nthreads, nblocks;
    nthreads.x = min(static_cast<idx_t>(loc_e), static_cast<idx_t>(CUDA_MAX_KERNEL_THREADS));
    nthreads.y = 1;
    nthreads.z = 1;
    nblocks.x  = min(static_cast<idx_t>((loc_e + nthreads.x - 1) / nthreads.x),
                    static_cast<idx_t>(env.get_num_sm() * 32));
    nblocks.y  = 1;
    nblocks.z  = 1;
    cugraph::detail::degree_coo<idx_t, idx_t><<<nblocks, nthreads>>>(
      static_cast<idx_t>(loc_e), static_cast<idx_t>(loc_e), ind, local_result);
    CUDA_CHECK_LAST();
  }

  // Out-degree
  if (x == 2 || x == 0) {
    dim3 nthreads, nblocks;
    nthreads.x = min(static_cast<idx_t>(loc_v), static_cast<idx_t>(CUDA_MAX_KERNEL_THREADS));
    nthreads.y = 1;
    nthreads.z = 1;
    nblocks.x  = min(static_cast<idx_t>((loc_v + nthreads.x - 1) / nthreads.x),
                    static_cast<idx_t>(env.get_num_sm() * 32));
    nblocks.y  = 1;
    nblocks.z  = 1;
    cugraph::detail::degree_offsets<idx_t, idx_t><<<nblocks, nthreads>>>(
      static_cast<idx_t>(loc_v), static_cast<idx_t>(loc_e), off, local_result + part_off[i]);
    CUDA_CHECK_LAST();
  }

  // Combining the local results into global results
  sync_all();
  treeReduce<idx_t, thrust::plus<idx_t>>(env, glob_v, local_result, degree);

  // Broadcasting the global result to all GPUs
  treeBroadcast(env, glob_v, local_result, degree);
}

template void snmg_degree<int>(int x, size_t* part_off, int* off, int* ind, int** degree);

template <>
void snmg_degree<int64_t>(int x, size_t* part_off, int64_t* off, int64_t* ind, int64_t** degree)
{
  sync_all();
  SNMGinfo env;
  auto i = env.get_thread_num();
  auto p = env.get_num_threads();

  // Getting the global and local vertices and edges
  size_t glob_v = part_off[p];
  size_t loc_v  = part_off[i + 1] - part_off[i];
  int64_t tmp;
  CUDA_TRY(hipMemcpy(&tmp, &off[loc_v], sizeof(int64_t), hipMemcpyDeviceToHost));
  size_t loc_e = tmp;

  // Allocating the local result array, and setting all entries to zero.
  int64_t* local_result;
  ALLOC_TRY((void**)&local_result, glob_v * sizeof(int64_t), nullptr);
  thrust::fill(rmm::exec_policy(nullptr)->on(nullptr), local_result, local_result + glob_v, 0);

  // In-degree
  if (x == 1 || x == 0) {
    dim3 nthreads, nblocks;
    nthreads.x = min(static_cast<int64_t>(loc_e), static_cast<int64_t>(CUDA_MAX_KERNEL_THREADS));
    nthreads.y = 1;
    nthreads.z = 1;
    nblocks.x  = min(static_cast<int64_t>((loc_e + nthreads.x - 1) / nthreads.x),
                    static_cast<int64_t>(env.get_num_sm() * 32));
    nblocks.y  = 1;
    nblocks.z  = 1;
    cugraph::detail::degree_coo<int64_t, double>
      <<<nblocks, nthreads>>>(static_cast<int64_t>(loc_e),
                              static_cast<int64_t>(loc_e),
                              ind,
                              reinterpret_cast<double*>(local_result));
    CUDA_CHECK_LAST();
  }

  // Out-degree
  if (x == 2 || x == 0) {
    dim3 nthreads, nblocks;
    nthreads.x = min(static_cast<int64_t>(loc_v), static_cast<int64_t>(CUDA_MAX_KERNEL_THREADS));
    nthreads.y = 1;
    nthreads.z = 1;
    nblocks.x  = min(static_cast<int64_t>((loc_v + nthreads.x - 1) / nthreads.x),
                    static_cast<int64_t>(env.get_num_sm() * 32));
    nblocks.y  = 1;
    nblocks.z  = 1;
    cugraph::detail::degree_offsets<int64_t, double>
      <<<nblocks, nthreads>>>(static_cast<int64_t>(loc_v),
                              static_cast<int64_t>(loc_e),
                              off,
                              reinterpret_cast<double*>(local_result + part_off[i]));
    CUDA_CHECK_LAST();
  }

  // Convert the values written as doubles back to int64:
  dim3 nthreads, nblocks;
  nthreads.x = min(static_cast<int64_t>(glob_v), static_cast<int64_t>(CUDA_MAX_KERNEL_THREADS));
  nthreads.y = 1;
  nthreads.z = 1;
  nblocks.x  = min(static_cast<int64_t>((glob_v + nthreads.x - 1) / nthreads.x),
                  static_cast<int64_t>(env.get_num_sm() * 32));
  nblocks.y  = 1;
  nblocks.z  = 1;
  cugraph::detail::type_convert<double, int64_t>
    <<<nblocks, nthreads>>>(reinterpret_cast<double*>(local_result), glob_v);
  CUDA_CHECK_LAST();

  // Combining the local results into global results
  treeReduce<int64_t, thrust::plus<int64_t>>(env, glob_v, local_result, degree);

  // Broadcasting the global result to all GPUs
  treeBroadcast(env, glob_v, local_result, degree);
}

template <typename idx_t>
void snmg_degree_impl(
  int x, size_t* part_offsets, gdf_column* off, gdf_column* ind, gdf_column** x_cols)
{
  CUGRAPH_EXPECTS(off->size > 0, "Invalid API parameter");
  CUGRAPH_EXPECTS(ind->size > 0, "Invalid API parameter");
  CUGRAPH_EXPECTS(off->dtype == ind->dtype, "Unsupported data type");
  CUGRAPH_EXPECTS(off->null_count + ind->null_count == 0, "Column must be valid");

  auto p = omp_get_num_threads();

  idx_t* degree[p];
  for (auto i = 0; i < p; ++i) {
    CUGRAPH_EXPECTS(x_cols[i] != nullptr, "Invalid API parameter");
    CUGRAPH_EXPECTS(x_cols[i]->size > 0, "Invalid API parameter");
    degree[i] = static_cast<idx_t*>(x_cols[i]->data);
  }

  snmg_degree(
    x, part_offsets, static_cast<idx_t*>(off->data), static_cast<idx_t*>(ind->data), degree);
}

}  // namespace snmg

void snmg_degree(int x, size_t* part_offsets, gdf_column* off, gdf_column* ind, gdf_column** x_cols)
{
  CUGRAPH_EXPECTS(part_offsets != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(off != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(ind != nullptr, "Invalid API parameter");
  CUGRAPH_EXPECTS(x_cols != nullptr, "Invalid API parameter");
  switch (off->dtype) {
    case GDF_INT32: return snmg::snmg_degree_impl<int32_t>(x, part_offsets, off, ind, x_cols);
    case GDF_INT64: return snmg::snmg_degree_impl<int64_t>(x, part_offsets, off, ind, x_cols);
    default: CUGRAPH_FAIL("Unsupported data type");
  }
}

}  // namespace cugraph
