#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cugraph.h>
#include <math.h>
#include <omp.h>
#include "hip/hip_runtime_api.h"
#include "gtest/gtest.h"
#include "high_res_clock.h"
#include "snmg/link_analysis/pagerank.cuh"
#include "snmg_test_utils.h"
#include "test_utils.h"

//#define SNMG_VERBOSE

typedef struct MGPagerank_Usecase_t {
  std::string matrix_file;
  std::string result_file;

  MGPagerank_Usecase_t(const std::string& a, const std::string& b)
  {
    // assume relative paths are relative to RAPIDS_DATASET_ROOT_DIR
    // if RAPIDS_DATASET_ROOT_DIR not set, default to "/datasets"
    const std::string& rapidsDatasetRootDir = get_rapids_dataset_root_dir();
    if ((a != "") && (a[0] != '/')) {
      matrix_file = rapidsDatasetRootDir + "/" + a;
    } else {
      matrix_file = a;
    }
    if ((b != "") && (b[0] != '/')) {
      result_file = rapidsDatasetRootDir + "/" + b;
    } else {
      result_file = b;
    }
  }
  MGPagerank_Usecase_t& operator=(const MGPagerank_Usecase_t& rhs)
  {
    matrix_file = rhs.matrix_file;
    result_file = rhs.result_file;
    return *this;
  }
} MGPagerank_Usecase;

template <typename val_t>
void verify_pr(gdf_column* col_pagerank, const MGPagerank_Usecase& param)
{
  // Check vs golden data
  if (param.result_file.length() > 0) {
    int m = col_pagerank->size;
    std::vector<val_t> calculated_res(m);
    CUDA_RT_CALL(hipMemcpy(
      &calculated_res[0], col_pagerank->data, sizeof(val_t) * m, hipMemcpyDeviceToHost));
    std::sort(calculated_res.begin(), calculated_res.end());
    FILE* fpin = fopen(param.result_file.c_str(), "rb");
    ASSERT_TRUE(fpin != NULL) << " Cannot read file with reference data: " << param.result_file
                              << std::endl;
    std::vector<val_t> expected_res(m);
    ASSERT_EQ(read_binary_vector(fpin, m, expected_res), 0);
    fclose(fpin);
    val_t err;
    int n_err = 0;
    for (int i = 0; i < m; i++) {
      // check for invalid values
      ASSERT_FALSE(isnan(calculated_res[i]));
      ASSERT_LE(calculated_res[i], 1.0);
      ASSERT_GE(calculated_res[i], 0.0);
      err = fabs(expected_res[i] - calculated_res[i]);
      if (err > 1e-5) {
        n_err++;  // count the number of mismatches
      }
    }
    if (n_err) {
      ASSERT_LE(n_err, 0.001 * m);  // tolerate 0.1% of values with a litte difference
    }
  }
}

class Tests_MGPagerank : public ::testing::TestWithParam<MGPagerank_Usecase> {
 public:
  Tests_MGPagerank() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}
  virtual void SetUp() {}
  virtual void TearDown() {}

  static std::vector<double> mgpr_time;

  template <typename idx_t, typename val_t>
  void run_current_test(const MGPagerank_Usecase& param)
  {
    const ::testing::TestInfo* const test_info =
      ::testing::UnitTest::GetInstance()->current_test_info();
    std::stringstream ss;
    std::string test_id = std::string(test_info->test_case_name()) + std::string(".") +
                          std::string(test_info->name()) + std::string("_") +
                          getFileName(param.matrix_file) + std::string("_") + ss.str().c_str();

    int m, k, nnz, n_gpus, max_iter = 50;
    val_t alpha = 0.85;
    MM_typecode mc;

    double t;

    FILE* fpin = fopen(param.matrix_file.c_str(), "r");
    ASSERT_NE(fpin, nullptr) << "fopen (" << param.matrix_file << ") failure.";

    ASSERT_EQ(mm_properties<int>(fpin, 1, &mc, &m, &k, &nnz), 0)
      << "could not read Matrix Market file properties"
      << "\n";
    ASSERT_TRUE(mm_is_matrix(mc));
    ASSERT_TRUE(mm_is_coordinate(mc));
    ASSERT_FALSE(mm_is_complex(mc));
    ASSERT_FALSE(mm_is_skew(mc));

    // Allocate memory on host
    std::vector<idx_t> cooRowInd(nnz), cooColInd(nnz);
    std::vector<val_t> cooVal_dummy(0);

    // Read
    ASSERT_EQ((mm_to_coo<int, val_t>(fpin, 1, nnz, &cooRowInd[0], &cooColInd[0], NULL, NULL)), 0)
      << "could not read matrix data"
      << "\n";
    ASSERT_EQ(fclose(fpin), 0);

    CUDA_RT_CALL(hipGetDeviceCount(&n_gpus));

    gdf_column* src_col_ptrs[n_gpus];
    gdf_column* dest_col_ptrs[n_gpus];
    gdf_column* pr_col = new gdf_column;

    int nthreads = n_gpus;

    // Only using the 4 fully connected GPUs on DGX1
    if (n_gpus == 8) nthreads = 4;

// Parallel load of the edge list
#pragma omp parallel num_threads(nthreads)
    {
      auto i = omp_get_thread_num();
      auto p = omp_get_num_threads();
      CUDA_RT_CALL(hipSetDevice(i));

#ifdef SNMG_VERBOSE
#pragma omp master
      {
        std::cout << "Number of GPUs : " << n_gpus << std::endl;
        std::cout << "Number of threads : " << p << std::endl;
      }
#endif

      src_col_ptrs[i]  = new gdf_column;
      dest_col_ptrs[i] = new gdf_column;

      // load a chunk of the graph on each GPU COO
      load_coo_loc(cooRowInd, cooColInd, cooVal_dummy, src_col_ptrs[i], dest_col_ptrs[i], nullptr);

#pragma omp barrier
    }

    t = omp_get_wtime();

    cugraph::snmg_pagerank(src_col_ptrs, dest_col_ptrs, pr_col, nthreads, alpha, max_iter);

    std::cout << omp_get_wtime() - t << std::endl;

    verify_pr<val_t>(pr_col, param);

// clean up
#pragma omp parallel num_threads(nthreads)
    {
      auto i = omp_get_thread_num();
      CUDA_RT_CALL(hipSetDevice(i));
      gdf_col_delete(src_col_ptrs[i]);
      gdf_col_delete(dest_col_ptrs[i]);
    }
    gdf_col_delete(pr_col);
  }
};
class Tests_MGPagerankCSR : public ::testing::TestWithParam<MGPagerank_Usecase> {
 public:
  Tests_MGPagerankCSR() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}
  virtual void SetUp() {}
  virtual void TearDown() {}

  static std::vector<double> mgpr_time;

  template <typename idx_t, typename val_t>
  void run_current_test(const MGPagerank_Usecase& param)
  {
    const ::testing::TestInfo* const test_info =
      ::testing::UnitTest::GetInstance()->current_test_info();
    std::stringstream ss;
    std::string test_id = std::string(test_info->test_case_name()) + std::string(".") +
                          std::string(test_info->name()) + std::string("_") +
                          getFileName(param.matrix_file) + std::string("_") + ss.str().c_str();

    int m, k, nnz, n_gpus, max_iter = 50;
    val_t alpha = 0.85;
    MM_typecode mc;

    double t;

    FILE* fpin = fopen(param.matrix_file.c_str(), "r");
    ASSERT_NE(fpin, nullptr) << "fopen (" << param.matrix_file << ") failure.";

    ASSERT_EQ(mm_properties<int>(fpin, 1, &mc, &m, &k, &nnz), 0)
      << "could not read Matrix Market file properties"
      << "\n";
    ASSERT_TRUE(mm_is_matrix(mc));
    ASSERT_TRUE(mm_is_coordinate(mc));
    ASSERT_FALSE(mm_is_complex(mc));
    ASSERT_FALSE(mm_is_skew(mc));

    // Allocate memory on host
    std::vector<idx_t> cooRowInd(nnz), cooColInd(nnz), csrColInd(nnz), csrRowPtr(m + 1);
    std::vector<val_t> cooVal(nnz), csrVal(nnz), pagerank_h(m, 1.0 / m);

    // Read
    ASSERT_EQ((mm_to_coo<int, val_t>(fpin, 1, nnz, &cooRowInd[0], &cooColInd[0], NULL, NULL)), 0)
      << "could not read matrix data"
      << "\n";
    ASSERT_EQ(fclose(fpin), 0);

    // WARNING transpose happening here
    coo2csr(cooColInd, cooRowInd, csrRowPtr, csrColInd);

    CUDA_RT_CALL(hipGetDeviceCount(&n_gpus));
    std::vector<size_t> v_loc(n_gpus), e_loc(n_gpus), part_offset(n_gpus + 1);
    random_vals(csrVal);
    gdf_column* col_pagerank[n_gpus];
    idx_t* degree[n_gpus];

    if (nnz < 1200000000) {
#pragma omp parallel num_threads(1)
      {
        auto i = omp_get_thread_num();
        auto p = omp_get_num_threads();
        CUDA_RT_CALL(hipSetDevice(i));

#ifdef SNMG_VERBOSE
#pragma omp master
        {
          std::cout << "Number of GPUs : " << n_gpus << std::endl;
          std::cout << "Number of threads : " << p << std::endl;
        }
#endif

        gdf_column *col_off = new gdf_column, *col_ind = new gdf_column, *col_val = new gdf_column;
        col_pagerank[i] = new gdf_column;
        create_gdf_column(pagerank_h, col_pagerank[i]);
#pragma omp barrier

        // load a chunck of the graph on each GPU
        load_csr_loc(
          csrRowPtr, csrColInd, csrVal, v_loc, e_loc, part_offset, col_off, col_ind, col_val);

        t = omp_get_wtime();
        cugraph::snmg::SNMGinfo env;
        cugraph::snmg::SNMGpagerank<idx_t, val_t> pr_solver(env,
                                                            &part_offset[0],
                                                            static_cast<idx_t*>(col_off->data),
                                                            static_cast<idx_t*>(col_ind->data));
        pr_solver.setup(alpha, degree);

        val_t* pagerank[p];
        for (auto i = 0; i < p; ++i) pagerank[i] = static_cast<val_t*>(col_pagerank[i]->data);

        pr_solver.solve(max_iter, pagerank);
#pragma omp master
        {
          std::cout << omp_get_wtime() - t << " ";
        }

        verify_pr<val_t>(col_pagerank[i], param);

        gdf_col_delete(col_off);
        gdf_col_delete(col_ind);
        gdf_col_delete(col_val);
        gdf_col_delete(col_pagerank[i]);
      }
    }

    if (n_gpus > 1) {
      // Only using the 4 fully connected GPUs on DGX1
      if (n_gpus == 8) n_gpus = 4;
#pragma omp parallel num_threads(n_gpus)
      {
        auto i = omp_get_thread_num();
        auto p = omp_get_num_threads();
        CUDA_RT_CALL(hipSetDevice(i));

#ifdef SNMG_VERBOSE
#pragma omp master
        {
          std::cout << "Number of GPUs : " << n_gpus << std::endl;
          std::cout << "Number of threads : " << p << std::endl;
        }
#endif

        gdf_column *col_off = new gdf_column, *col_ind = new gdf_column, *col_val = new gdf_column;
        col_pagerank[i] = new gdf_column;
        create_gdf_column(pagerank_h, col_pagerank[i]);
#pragma omp barrier

        // load a chunck of the graph on each GPU
        load_csr_loc(
          csrRowPtr, csrColInd, csrVal, v_loc, e_loc, part_offset, col_off, col_ind, col_val);
        t = omp_get_wtime();
        cugraph::snmg::SNMGinfo env;
        cugraph::snmg::SNMGpagerank<idx_t, val_t> pr_solver(env,
                                                            &part_offset[0],
                                                            static_cast<idx_t*>(col_off->data),
                                                            static_cast<idx_t*>(col_ind->data));
        pr_solver.setup(alpha, degree);

        val_t* pagerank[p];
        for (auto i = 0; i < p; ++i) pagerank[i] = static_cast<val_t*>(col_pagerank[i]->data);

        pr_solver.solve(max_iter, pagerank);
#pragma omp master
        {
          std::cout << omp_get_wtime() - t << " ";
        }

        verify_pr<val_t>(col_pagerank[i], param);
        gdf_col_delete(col_off);
        gdf_col_delete(col_ind);
        gdf_col_delete(col_val);
        gdf_col_delete(col_pagerank[i]);
      }
    }
    std::cout << std::endl;
  }
};

class Tests_MGPR_hibench : public ::testing::TestWithParam<MGPagerank_Usecase> {
 public:
  Tests_MGPR_hibench() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}
  virtual void SetUp() {}
  virtual void TearDown() {}

  static std::vector<double> mgspmv_time;

  template <typename idx_t, typename val_t>
  void run_current_test(const MGPagerank_Usecase& param)
  {
    const ::testing::TestInfo* const test_info =
      ::testing::UnitTest::GetInstance()->current_test_info();
    std::stringstream ss;
    std::string test_id = std::string(test_info->test_case_name()) + std::string(".") +
                          std::string(test_info->name()) + std::string("_") +
                          getFileName(param.matrix_file) + std::string("_") + ss.str().c_str();

    int m, nnz, n_gpus, max_iter = 50;
    val_t alpha = 0.85;
    std::vector<idx_t> cooRowInd, cooColInd;
    double t;

    ASSERT_EQ(read_single_file(param.matrix_file.c_str(), cooRowInd, cooColInd), 0)
      << "read_single_file(" << param.matrix_file << ", ...) failure.";
    nnz = cooRowInd.size();
    m   = 1 + std::max(*(std::max_element(cooRowInd.begin(), cooRowInd.end())),
                     *(std::max_element(cooColInd.begin(), cooColInd.end())));

    // Allocate memory on host
    std::vector<idx_t> csrColInd(nnz), csrRowPtr(m + 1);
    std::vector<val_t> cooVal(nnz), csrVal(nnz), pagerank_h(m, 1.0 / m);

    // transpose here
    coo2csr(cooColInd, cooRowInd, csrRowPtr, csrColInd);
    CUDA_RT_CALL(hipGetDeviceCount(&n_gpus));
    std::vector<size_t> v_loc(n_gpus), e_loc(n_gpus), part_offset(n_gpus + 1);
    random_vals(csrVal);
    gdf_column* col_pagerank[n_gpus];
    idx_t* degree[n_gpus];

    if (nnz < 1200000000) {
#pragma omp parallel num_threads(1)
      {
        auto i = omp_get_thread_num();
        auto p = omp_get_num_threads();
        CUDA_RT_CALL(hipSetDevice(i));

#ifdef SNMG_VERBOSE
#pragma omp master
        {
          std::cout << "Number of GPUs : " << n_gpus << std::endl;
          std::cout << "Number of threads : " << p << std::endl;
        }
#endif

        gdf_column *col_off = new gdf_column, *col_ind = new gdf_column, *col_val = new gdf_column;
        col_pagerank[i] = new gdf_column;
        create_gdf_column(pagerank_h, col_pagerank[i]);
#pragma omp barrier

        // load a chunck of the graph on each GPU
        load_csr_loc(
          csrRowPtr, csrColInd, csrVal, v_loc, e_loc, part_offset, col_off, col_ind, col_val);

        t = omp_get_wtime();
        cugraph::snmg::SNMGinfo env;
        cugraph::snmg::SNMGpagerank<idx_t, val_t> pr_solver(env,
                                                            &part_offset[0],
                                                            static_cast<idx_t*>(col_off->data),
                                                            static_cast<idx_t*>(col_ind->data));
        pr_solver.setup(alpha, degree);

        val_t* pagerank[p];
        for (auto i = 0; i < p; ++i) pagerank[i] = static_cast<val_t*>(col_pagerank[i]->data);

        pr_solver.solve(max_iter, pagerank);
#pragma omp master
        {
          std::cout << omp_get_wtime() - t << " ";
        }

        verify_pr<val_t>(col_pagerank[i], param);

        gdf_col_delete(col_off);
        gdf_col_delete(col_ind);
        gdf_col_delete(col_val);
        gdf_col_delete(col_pagerank[i]);
      }
    }
    if (n_gpus > 1) {
      // Only using the 4 fully connected GPUs on DGX1
      if (n_gpus == 8) n_gpus = 4;
#pragma omp parallel num_threads(n_gpus)
      {
        auto i = omp_get_thread_num();
        auto p = omp_get_num_threads();
        CUDA_RT_CALL(hipSetDevice(i));

#ifdef SNMG_VERBOSE
#pragma omp master
        {
          std::cout << "Number of GPUs : " << n_gpus << std::endl;
          std::cout << "Number of threads : " << p << std::endl;
        }
#endif

        gdf_column *col_off = new gdf_column, *col_ind = new gdf_column, *col_val = new gdf_column;
        col_pagerank[i] = new gdf_column;
        create_gdf_column(pagerank_h, col_pagerank[i]);
#pragma omp barrier

        // load a chunck of the graph on each GPU
        load_csr_loc(
          csrRowPtr, csrColInd, csrVal, v_loc, e_loc, part_offset, col_off, col_ind, col_val);

        t = omp_get_wtime();
        cugraph::snmg::SNMGinfo env;
        cugraph::snmg::SNMGpagerank<idx_t, val_t> pr_solver(env,
                                                            &part_offset[0],
                                                            static_cast<idx_t*>(col_off->data),
                                                            static_cast<idx_t*>(col_ind->data));
        pr_solver.setup(alpha, degree);

        val_t* pagerank[p];
        for (auto i = 0; i < p; ++i) pagerank[i] = static_cast<val_t*>(col_pagerank[i]->data);

        pr_solver.solve(max_iter, pagerank);
#pragma omp master
        {
          std::cout << omp_get_wtime() - t << " ";
        }

        verify_pr<val_t>(col_pagerank[i], param);

        gdf_col_delete(col_off);
        gdf_col_delete(col_ind);
        gdf_col_delete(col_val);
        gdf_col_delete(col_pagerank[i]);
      }
    }
    std::cout << std::endl;
  }
};

TEST_P(Tests_MGPagerankCSR, CheckFP32_mtx) { run_current_test<int, float>(GetParam()); }

TEST_P(Tests_MGPagerank, CheckFP32_mtx) { run_current_test<int, float>(GetParam()); }

TEST_P(Tests_MGPR_hibench, CheckFP32_hibench) { run_current_test<int, float>(GetParam()); }

INSTANTIATE_TEST_CASE_P(
  mtx_test,
  Tests_MGPagerankCSR,
  ::testing::Values(MGPagerank_Usecase("test/datasets/karate.mtx", ""),
                    MGPagerank_Usecase("test/datasets/wiki-Talk.mtx",
                                       "test/ref/pagerank/wiki-Talk.pagerank_val_0.85.bin"),
                    MGPagerank_Usecase("test/datasets/webbase-1M.mtx",
                                       "test/ref/pagerank/webbase-1M.pagerank_val_0.85.bin")));

INSTANTIATE_TEST_CASE_P(
  mtx_test,
  Tests_MGPagerank,
  ::testing::Values(MGPagerank_Usecase("test/datasets/netscience.mtx", ""),
                    MGPagerank_Usecase("test/datasets/web-BerkStan.mtx",
                                       "test/ref/pagerank/web-BerkStan.pagerank_val_0.85.bin"),
                    MGPagerank_Usecase("test/datasets/web-Google.mtx",
                                       "test/ref/pagerank/web-Google.pagerank_val_0.85.bin"),
                    MGPagerank_Usecase("test/datasets/cit-Patents.mtx",
                                       "test/ref/pagerank/cit-Patents.pagerank_val_0.85.bin"),
                    MGPagerank_Usecase("test/datasets/ljournal-2008.mtx",
                                       "test/ref/pagerank/ljournal-2008.pagerank_val_0.85.bin"),
                    MGPagerank_Usecase("test/datasets/wiki-Talk.mtx",
                                       "test/ref/pagerank/wiki-Talk.pagerank_val_0.85.bin"),
                    MGPagerank_Usecase("test/datasets/webbase-1M.mtx",
                                       "test/ref/pagerank/webbase-1M.pagerank_val_0.85.bin")));

INSTANTIATE_TEST_CASE_P(
  hibench_test,
  Tests_MGPR_hibench,
  ::testing::Values(MGPagerank_Usecase("benchmark/hibench/1/Input-small/edges/part-00000", ""),
                    MGPagerank_Usecase("benchmark/hibench/1/Input-large/edges/part-00000", "")));

INSTANTIATE_TEST_CASE_P(
  hibench_test_huge,
  Tests_MGPR_hibench,
  ::testing::Values(MGPagerank_Usecase("benchmark/hibench/1/Input-huge/edges/part-00000", "")));

int main(int argc, char** argv)
{
  rmmInitialize(nullptr);
  testing::InitGoogleTest(&argc, argv);
  int rc = RUN_ALL_TESTS();
  rmmFinalize();
  return rc;
}
