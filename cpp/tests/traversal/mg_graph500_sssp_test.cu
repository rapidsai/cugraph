#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "detail/graph500_forest_pruning_utils.cuh"
#include "detail/graph500_nbr_unrenumber_cache.cuh"
#include "detail/graph500_validation_utils.cuh"
#include "prims/edge_bucket.cuh"
#include "prims/transform_gather_e.cuh"
#include "prims/update_edge_src_dst_property.cuh"
#include "utilities/base_fixture.hpp"
#include "utilities/collect_comm.cuh"
#include "utilities/conversion_utilities.hpp"
#include "utilities/device_comm_wrapper.hpp"
#include "utilities/mg_utilities.hpp"
#include "utilities/property_generator_utilities.hpp"
#include "utilities/test_graphs.hpp"
#include "utilities/thrust_wrapper.hpp"

#include <cugraph/algorithms.hpp>
#include <cugraph/edge_partition_endpoint_property_device_view.cuh>
#include <cugraph/edge_property.hpp>
#include <cugraph/edge_src_dst_property.hpp>
#include <cugraph/graph.hpp>
#include <cugraph/graph_functions.hpp>
#include <cugraph/graph_view.hpp>
#include <cugraph/large_buffer_manager.hpp>
#include <cugraph/partition_manager.hpp>
#include <cugraph/shuffle_functions.hpp>
#include <cugraph/utilities/device_functors.cuh>
#include <cugraph/utilities/high_res_timer.hpp>
#include <cugraph/utilities/misc_utils.cuh>
#include <cugraph/utilities/shuffle_comm.cuh>

#include <raft/comms/mpi_comms.hpp>
#include <raft/core/comms.hpp>
#include <raft/core/handle.hpp>

#include <rmm/device_scalar.hpp>
#include <rmm/device_uvector.hpp>

#include <cuda/std/tuple>
#include <thrust/merge.h>
#include <thrust/set_operations.h>
#include <thrust/sort.h>
#include <thrust/transform.h>

#include <gtest/gtest.h>

#include <random>

struct Graph500_SSSP_Usecase {
  bool use_pruned_graph_unrenumber_cache{
    false};  // use cache to locally unrenumber (at the expense of additional memory usage)
  bool use_large_buffer{false};
  bool validate{true};
};

void init_nccl_env_variables() {}

template <typename input_usecase_t>
class Tests_GRAPH500_MGSSSP
  : public ::testing::TestWithParam<std::tuple<Graph500_SSSP_Usecase, input_usecase_t>> {
 public:
  Tests_GRAPH500_MGSSSP() {}

  static void SetUpTestCase()
  {
    init_nccl_env_variables();

    size_t pool_size =
      12;  // note that CUDA_DEVICE_MAX_CONNECTIONS (default: 8) should be set to a value larger
           // than pool_size to avoid false dependency among different streams
    handle_ = cugraph::test::initialize_mg_handle(pool_size);

    cugraph::large_buffer_manager::init(
      *handle_, cugraph::large_buffer_manager::create_memory_buffer_resource(), std::nullopt);
  }

  static void TearDownTestCase() { handle_.reset(); }

  virtual void SetUp() {}
  virtual void TearDown() {}

  template <typename vertex_t, typename edge_t>
  void run_current_test(Graph500_SSSP_Usecase const& sssp_usecase,
                        input_usecase_t const& input_usecase)
  {
    using weight_t    = float;
    using edge_type_t = int32_t;  // dummy
    using edge_time_t = int32_t;  // dummy

    bool constexpr store_transposed = false;
    bool constexpr multi_gpu        = true;
    bool constexpr renumber         = true;
    bool constexpr test_weighted    = true;
    bool constexpr shuffle = false;  // Graph 500 requirement (edges can't be pre-shuffled, edges
                                     // should be shuffled in Kernel 1)
    size_t num_warmup_starting_vertices = 1;   // to enforce all CUDA & NCCL initializations
    size_t num_timed_starting_vertices  = 64;  // Graph 500 requirement (64)

    HighResTimer hr_timer{};
    raft::random::RngState rng_state{0};

    auto& comm           = handle_->get_comms();
    auto const comm_rank = comm.get_rank();
    auto const comm_size = comm.get_size();
    auto& major_comm     = handle_->get_subcomm(cugraph::partition_manager::major_comm_name());
    auto const major_comm_rank = major_comm.get_rank();
    auto const major_comm_size = major_comm.get_size();
    auto& minor_comm = handle_->get_subcomm(cugraph::partition_manager::minor_comm_name());
    auto const minor_comm_rank = minor_comm.get_rank();
    auto const minor_comm_size = minor_comm.get_size();
    auto vertex_partition_id =
      cugraph::partition_manager::compute_vertex_partition_id_from_graph_subcomm_ranks(
        major_comm_size, minor_comm_size, major_comm_rank, minor_comm_rank);

    std::cout << "comm_size=" << comm_size << " major_comm_size=" << major_comm_size
              << " minor_comm_size=" << minor_comm_size << std::endl;

    constexpr auto invalid_distance = std::numeric_limits<weight_t>::max();
    constexpr auto invalid_vertex   = cugraph::invalid_vertex_id<vertex_t>::value;

    // 1. force NCCL P2P initialization

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      comm.barrier();
      hr_timer.start("NCCL P2P buffer initialization");
    }

    cugraph::test::enforce_p2p_initialization(comm, handle_->get_stream());
    cugraph::test::enforce_p2p_initialization(major_comm, handle_->get_stream());
    cugraph::test::enforce_p2p_initialization(minor_comm, handle_->get_stream());

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      comm.barrier();
      hr_timer.stop();
      hr_timer.display_and_clear(std::cout);
    }

    // 2. create a graph & meta data

    rmm::device_uvector<vertex_t> mg_renumber_map(0, handle_->get_stream());
    rmm::device_uvector<vertex_t> components(0, handle_->get_stream());
    rmm::device_uvector<vertex_t> parents(0, handle_->get_stream());
    rmm::device_uvector<weight_t> w_to_parents(0, handle_->get_stream());
    rmm::device_uvector<vertex_t> unrenumbered_parents(0, handle_->get_stream());
    std::vector<vertex_t> vertex_partition_range_offsets(comm_size + 1);
    vertex_t local_vertex_partition_range_first{};
    vertex_t local_vertex_partition_range_last{};

    cugraph::graph_t<vertex_t, edge_t, store_transposed, multi_gpu> mg_pruned_graph(*handle_);
    cugraph::edge_property_t<edge_t, weight_t> mg_pruned_graph_edge_weights(*handle_);
    rmm::device_uvector<vertex_t> mg_pruned_graph_renumber_map(0, handle_->get_stream());
    rmm::device_uvector<vertex_t> mg_graph_to_pruned_graph_map(
      0, handle_->get_stream());  // we may store this in host buffer to save HBM
    rmm::device_uvector<vertex_t> mg_pruned_graph_to_graph_map(0, handle_->get_stream());

    cugraph::graph_t<vertex_t, edge_t, store_transposed, multi_gpu> mg_isolated_trees(*handle_);
    cugraph::edge_property_t<edge_t, weight_t> mg_isolated_trees_edge_weights(*handle_);
    rmm::device_uvector<vertex_t> mg_isolated_trees_renumber_map(0, handle_->get_stream());
    rmm::device_uvector<vertex_t> mg_graph_to_isolated_trees_map(
      0, handle_->get_stream());  // we may store this in host buffer to save HBM
    rmm::device_uvector<vertex_t> mg_isolated_trees_to_graph_map(0, handle_->get_stream());

    std::optional<cugraph::test::nbr_unrenumber_cache_t<vertex_t>>
      mg_pruned_graph_pred_unrenumber_cache{std::nullopt};
    {
      edge_t num_input_edges{};
      edge_t num_edges{};  // after removing self-loops and multi-edges

      // 2-1. create an edge list

      if (cugraph::test::g_perf) {
        RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
        comm.barrier();
        hr_timer.start("MG Construct edge list");
      }

      std::vector<rmm::device_uvector<vertex_t>> src_chunks{};
      std::vector<rmm::device_uvector<vertex_t>> dst_chunks{};
      std::vector<rmm::device_uvector<weight_t>> weight_chunks{};
      {
        std::optional<std::vector<rmm::device_uvector<weight_t>>> tmp_weight_chunks{std::nullopt};
        std::tie(src_chunks, dst_chunks, tmp_weight_chunks, std::ignore, std::ignore) =
          input_usecase.template construct_edgelist<vertex_t, weight_t>(
            *handle_, test_weighted, store_transposed, multi_gpu, shuffle);
        weight_chunks = std::move(*tmp_weight_chunks);
      }
      ASSERT_TRUE(input_usecase.undirected());

      if (cugraph::test::g_perf) {
        RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
        comm.barrier();
        hr_timer.stop();
        hr_timer.display_and_clear(std::cout);
      }

      num_input_edges = 0;
      for (size_t i = 0; i < src_chunks.size(); ++i) {
        num_input_edges += static_cast<edge_t>(src_chunks[i].size());
      }
      num_input_edges = cugraph::host_scalar_allreduce(
        comm, num_input_edges, raft::comms::op_t::SUM, handle_->get_stream());

      // 2-2. create an MG graph

      if (cugraph::test::g_perf) {
        RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
        comm.barrier();
        hr_timer.start("MG Construct graph (Kernel 1)");
      }

      for (size_t i = 0; i < src_chunks.size(); ++i) {
        std::optional<rmm::device_uvector<weight_t>> tmp_weight_chunk{std::nullopt};
        std::tie(src_chunks[i],
                 dst_chunks[i],
                 tmp_weight_chunk,
                 std::ignore,
                 std::ignore,
                 std::ignore,
                 std::ignore) =
          cugraph::remove_self_loops<vertex_t, edge_t, weight_t, edge_type_t, edge_time_t>(
            *handle_,
            std::move(src_chunks[i]),
            std::move(dst_chunks[i]),
            std::move(weight_chunks[i]),
            std::nullopt,
            std::nullopt,
            std::nullopt,
            std::nullopt);

        std::vector<cugraph::arithmetic_device_uvector_t> edge_property_chunk{};
        edge_property_chunk.push_back(std::move(*tmp_weight_chunk));

        std::tie(src_chunks[i], dst_chunks[i], edge_property_chunk, std::ignore) =
          cugraph::shuffle_ext_edges(*handle_,
                                     std::move(src_chunks[i]),
                                     std::move(dst_chunks[i]),
                                     std::move(edge_property_chunk),
                                     store_transposed);

        weight_chunks[i] =
          std::move(std::get<rmm::device_uvector<weight_t>>(edge_property_chunk[0]));
      }

      {
        std::optional<std::vector<rmm::device_uvector<weight_t>>> tmp_weight_chunks{std::nullopt};
        std::tie(src_chunks,
                 dst_chunks,
                 tmp_weight_chunks,
                 std::ignore,
                 std::ignore,
                 std::ignore,
                 std::ignore) =
          cugraph::remove_multi_edges<vertex_t, edge_t, weight_t, edge_type_t, edge_time_t>(
            *handle_,
            std::move(src_chunks),
            std::move(dst_chunks),
            std::move(weight_chunks),
            std::nullopt,
            std::nullopt,
            std::nullopt,
            std::nullopt,
            true /* keep_min_value_edge */);
        weight_chunks = std::move(*tmp_weight_chunks);
      }

      num_edges = 0;
      for (size_t i = 0; i < src_chunks.size(); ++i) {
        num_edges += static_cast<edge_t>(src_chunks[i].size());
      }
      num_edges = cugraph::host_scalar_allreduce(
        comm, num_edges, raft::comms::op_t::SUM, handle_->get_stream());

      cugraph::graph_t<vertex_t, edge_t, store_transposed, multi_gpu> mg_graph(*handle_);
      cugraph::edge_property_t<edge_t, weight_t> mg_edge_weights(*handle_);
      {
        std::optional<cugraph::edge_property_t<edge_t, weight_t>> tmp_weights{std::nullopt};
        std::optional<rmm::device_uvector<vertex_t>> tmp_map{};
        std::tie(
          mg_graph, tmp_weights, std::ignore, std::ignore, std::ignore, std::ignore, tmp_map) =
          cugraph::create_graph_from_edgelist<vertex_t,
                                              edge_t,
                                              weight_t,
                                              edge_type_t,
                                              edge_time_t,
                                              store_transposed,
                                              multi_gpu>(
            *handle_,
            std::nullopt,
            std::move(src_chunks),
            std::move(dst_chunks),
            std::move(weight_chunks),
            std::nullopt,
            std::nullopt,
            std::nullopt,
            std::nullopt,
            cugraph::graph_properties_t{true /* symmetric */, false /* multi-graph */},
            renumber);
        mg_edge_weights = std::move(*tmp_weights);
        mg_renumber_map = std::move(*tmp_map);
      }
      {
        auto mg_graph_view = mg_graph.view();
        auto offsets       = mg_graph_view.vertex_partition_range_offsets();
        std::copy(offsets.begin(), offsets.end(), vertex_partition_range_offsets.begin());
        local_vertex_partition_range_first = mg_graph_view.local_vertex_partition_range_first();
        local_vertex_partition_range_last  = mg_graph_view.local_vertex_partition_range_last();
      }

      // 2-3. Forest pruning

      {
        auto mg_graph_view       = mg_graph.view();
        auto mg_edge_weight_view = mg_edge_weights.view();

        components.resize(mg_graph_view.local_vertex_partition_range_size(), handle_->get_stream());
        cugraph::weakly_connected_components(
          *handle_, mg_graph_view, components.data(), components.size());
        std::optional<rmm::device_uvector<weight_t>> tmp_w_to_parents{std::nullopt};
        std::tie(parents, tmp_w_to_parents) =
          find_trees_from_2cores<vertex_t, edge_t, weight_t, store_transposed, multi_gpu>(
            *handle_, mg_graph_view, mg_edge_weight_view, invalid_vertex, invalid_distance);
        w_to_parents = std::move(*tmp_w_to_parents);
      }

      std::optional<rmm::device_uvector<vertex_t>> tmp_components{std::nullopt};
      if (sssp_usecase
            .use_large_buffer) {  // temporarily store components in host buffer to free up HBM
                                  // before extracting sub-graphs (which uses a lot of HBM)
        tmp_components = cugraph::large_buffer_manager::allocate_memory_buffer<vertex_t>(
          components.size(), handle_->get_stream());
        thrust::copy(handle_->get_thrust_policy(),
                     components.begin(),
                     components.end(),
                     tmp_components->begin());
        components.resize(0, handle_->get_stream());
        components.shrink_to_fit(handle_->get_stream());
      }

      {
        std::optional<cugraph::edge_property_t<edge_t, weight_t>> tmp_mg_pruned_graph_edge_weights{
          std::nullopt};
        std::optional<cugraph::edge_property_t<edge_t, weight_t>>
          tmp_mg_isolated_trees_edge_weights{std::nullopt};
        std::tie(mg_pruned_graph,
                 tmp_mg_pruned_graph_edge_weights,
                 mg_pruned_graph_renumber_map,
                 mg_graph_to_pruned_graph_map,
                 mg_pruned_graph_to_graph_map,
                 mg_isolated_trees,
                 tmp_mg_isolated_trees_edge_weights,
                 mg_isolated_trees_renumber_map,
                 mg_graph_to_isolated_trees_map,
                 mg_isolated_trees_to_graph_map) =
          extract_forest_pruned_graph_and_isolated_trees<vertex_t,
                                                         edge_t,
                                                         weight_t,
                                                         edge_type_t,
                                                         edge_time_t,
                                                         store_transposed,
                                                         multi_gpu>(
            *handle_,
            std::move(mg_graph),
            std::move(mg_edge_weights),
            raft::device_span<vertex_t const>(mg_renumber_map.data(), mg_renumber_map.size()),
            raft::device_span<vertex_t const>(parents.data(), parents.size()),
            invalid_vertex,
            sssp_usecase.use_large_buffer ? std::make_optional(cugraph::large_buffer_type_t::MEMORY)
                                          : std::nullopt);
        mg_pruned_graph_edge_weights   = std::move(*tmp_mg_pruned_graph_edge_weights);
        mg_isolated_trees_edge_weights = std::move(*tmp_mg_isolated_trees_edge_weights);
      }

      if (sssp_usecase.use_large_buffer) {
        components.resize(tmp_components->size(), handle_->get_stream());
        thrust::copy(handle_->get_thrust_policy(),
                     tmp_components->begin(),
                     tmp_components->end(),
                     components.begin());
        tmp_components = std::nullopt;
      }

      unrenumbered_parents.resize(parents.size(), handle_->get_stream());
      thrust::copy(
        handle_->get_thrust_policy(), parents.begin(), parents.end(), unrenumbered_parents.begin());
      cugraph::unrenumber_int_vertices<vertex_t, multi_gpu>(
        *handle_,
        unrenumbered_parents.data(),
        unrenumbered_parents.size(),
        mg_renumber_map.data(),
        raft::host_span<vertex_t const>(vertex_partition_range_offsets.data() + 1,
                                        vertex_partition_range_offsets.size() - 1));

      if (sssp_usecase.use_pruned_graph_unrenumber_cache) {
        mg_pruned_graph_pred_unrenumber_cache = cugraph::test::build_nbr_unrenumber_cache(
          *handle_,
          mg_pruned_graph.view(),
          raft::device_span<vertex_t const>(mg_pruned_graph_renumber_map.data(),
                                            mg_pruned_graph_renumber_map.size()),
          invalid_vertex,
          sssp_usecase.use_large_buffer ? std::make_optional(cugraph::large_buffer_type_t::MEMORY)
                                        : std::nullopt);
      }

      if (cugraph::test::g_perf) {
        RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
        comm.barrier();
        hr_timer.stop();
        hr_timer.display_and_clear(std::cout);
      }

      std::cout << "num_input_edges=" << num_input_edges
                << " V=" << vertex_partition_range_offsets.back() << " E=" << num_edges
                << " undirected E=" << (num_edges / 2) << std::endl;
    }
    auto mg_pruned_graph_view               = mg_pruned_graph.view();
    auto mg_pruned_graph_edge_weight_view   = mg_pruned_graph_edge_weights.view();
    auto mg_isolated_trees_view             = mg_isolated_trees.view();
    auto mg_isolated_trees_edge_weight_view = mg_isolated_trees_edge_weights.view();
    std::cout << "mg_pruned_graph V=" << mg_pruned_graph_view.number_of_vertices()
              << " E=" << mg_pruned_graph_view.compute_number_of_edges(*handle_)
              << " mg_isolated_trees_view V=" << mg_isolated_trees_view.number_of_vertices()
              << " E=" << mg_isolated_trees_view.compute_number_of_edges(*handle_) << std::endl;

    // 3. randomly select starting vertices

    std::vector<vertex_t> starting_vertices{};
    {
      ASSERT_TRUE(vertex_partition_range_offsets.back() > 0)
        << "Invalid input graph, the input graph should have at least one vertex";
      rmm::device_uvector<vertex_t> d_starting_vertices(
        num_warmup_starting_vertices + num_timed_starting_vertices, handle_->get_stream());
      if (comm_rank == 0) {
        cugraph::detail::uniform_random_fill(handle_->get_stream(),
                                             d_starting_vertices.data(),
                                             d_starting_vertices.size(),
                                             vertex_partition_range_offsets[0],
                                             vertex_partition_range_offsets.back(),
                                             rng_state);
        raft::print_device_vector(
          "d_starting_vertices", d_starting_vertices.data(), d_starting_vertices.size(), std::cout);
      }
      cugraph::device_bcast(comm,
                            d_starting_vertices.data(),
                            d_starting_vertices.data(),
                            d_starting_vertices.size(),
                            int{0},
                            handle_->get_stream());
      starting_vertices = cugraph::test::to_host(*handle_, d_starting_vertices);
    }

    // 4. run MG SSSP

    rmm::device_uvector<weight_t> d_mg_distances(mg_renumber_map.size(), handle_->get_stream());
    rmm::device_uvector<vertex_t> d_mg_unrenumbered_predecessors(mg_renumber_map.size(),
                                                                 handle_->get_stream());
    rmm::device_uvector<weight_t> d_mg_w_to_predecessors(mg_renumber_map.size(),
                                                         handle_->get_stream());  // for validation

    double total_elapsed{0.0};
    double tteps_sum{0.0};
    double one_over_tteps_sum{0.0};  // to compute harmonic mean
    for (size_t i = 0; i < (num_warmup_starting_vertices + num_timed_starting_vertices); ++i) {
      double elapsed{0.0};

      thrust::fill(handle_->get_thrust_policy(),
                   d_mg_w_to_predecessors.begin(),
                   d_mg_w_to_predecessors.end(),
                   invalid_distance);

      if (cugraph::test::g_perf) {
        RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
        comm.barrier();
        hr_timer.start("MG SSSP (Kernel 2)");
      }

      thrust::fill(handle_->get_thrust_policy(),
                   d_mg_distances.begin(),
                   d_mg_distances.end(),
                   invalid_distance);

      auto starting_vertex = starting_vertices[i];
      auto starting_vertex_vertex_partition_id =
        static_cast<int>(std::distance(vertex_partition_range_offsets.begin() + 1,
                                       std::upper_bound(vertex_partition_range_offsets.begin() + 1,
                                                        vertex_partition_range_offsets.end(),
                                                        starting_vertex)));

      vertex_t unrenumbered_starting_vertex{};
      vertex_t starting_vertex_parent{starting_vertex};
      weight_t w_to_starting_vertex_parent{};
      vertex_t starting_vertex_component{};
      if (starting_vertex_vertex_partition_id == vertex_partition_id) {
        unrenumbered_starting_vertex = mg_renumber_map.element(
          starting_vertex - local_vertex_partition_range_first, handle_->get_stream());
        starting_vertex_parent = parents.element(
          starting_vertex - local_vertex_partition_range_first, handle_->get_stream());
        w_to_starting_vertex_parent = w_to_parents.element(
          starting_vertex - local_vertex_partition_range_first, handle_->get_stream());
        starting_vertex_component = components.element(
          starting_vertex - local_vertex_partition_range_first, handle_->get_stream());
      }
      thrust::tie(unrenumbered_starting_vertex,
                  starting_vertex_parent,
                  w_to_starting_vertex_parent,
                  starting_vertex_component) =
        cugraph::host_scalar_bcast(
          comm,
          cuda::std::make_tuple(unrenumbered_starting_vertex,
                                starting_vertex_parent,
                                w_to_starting_vertex_parent,
                                starting_vertex_component),
          cugraph::partition_manager::compute_global_comm_rank_from_vertex_partition_id(
            major_comm_size, minor_comm_size, starting_vertex_vertex_partition_id),
          handle_->get_stream());
      bool reachable_from_2cores{starting_vertex_parent != invalid_vertex};
      bool in_2cores{starting_vertex == starting_vertex_parent};

      if (reachable_from_2cores) {
        thrust::transform(handle_->get_thrust_policy(),
                          unrenumbered_parents.begin(),
                          unrenumbered_parents.end(),
                          components.begin(),
                          d_mg_unrenumbered_predecessors.begin(),
                          cuda::proclaim_return_type<vertex_t>(
                            [starting_vertex_component, invalid_vertex] __device__(auto p, auto c) {
                              return (c == starting_vertex_component)
                                       ? p /* for the vertices in 2-cores (or the vertices in the
                                              path from the starting vertex to the first reachable
                                              2-core vertex), this will be over-written */
                                       : invalid_vertex;
                            }));
      } else {
        thrust::fill(handle_->get_thrust_policy(),
                     d_mg_unrenumbered_predecessors.begin(),
                     d_mg_unrenumbered_predecessors.end(),
                     invalid_vertex);
      }

      vertex_t subgraph_starting_vertex{starting_vertex};
      int subgraph_starting_vertex_vertex_partition_id{starting_vertex_vertex_partition_id};
      weight_t subgraph_starting_vertex_distance{0.0};
      vertex_t unrenumbered_subgraph_starting_vertex_parent{};
      weight_t w_to_subgraph_starting_vertex_parent{0.0};
      if (reachable_from_2cores && !in_2cores) {  // find the path from starting_vertex to a 2-core
        std::optional<weight_t> tmp_w_to_subgraph_starting_vertex_parent{};
        std::tie(subgraph_starting_vertex,
                 subgraph_starting_vertex_vertex_partition_id,
                 subgraph_starting_vertex_distance,
                 unrenumbered_subgraph_starting_vertex_parent,
                 tmp_w_to_subgraph_starting_vertex_parent) =
          traverse_to_pruned_graph<vertex_t, weight_t>(
            *handle_,
            raft::device_span<vertex_t const>(parents.data(), parents.size()),
            raft::device_span<weight_t const>(w_to_parents.data(), w_to_parents.size()),
            raft::device_span<vertex_t const>(mg_renumber_map.data(), mg_renumber_map.size()),
            raft::host_span<vertex_t const>(vertex_partition_range_offsets.data(),
                                            vertex_partition_range_offsets.size()),
            raft::device_span<vertex_t>(d_mg_unrenumbered_predecessors.data(),
                                        d_mg_unrenumbered_predecessors.size()),
            raft::device_span<weight_t>(d_mg_w_to_predecessors.data(),
                                        d_mg_w_to_predecessors.size()),
            raft::device_span<weight_t>(d_mg_distances.data(), d_mg_distances.size()),
            starting_vertex,
            unrenumbered_starting_vertex,
            starting_vertex_vertex_partition_id,
            starting_vertex_parent,
            w_to_starting_vertex_parent,
            local_vertex_partition_range_first,
            vertex_partition_id);
        w_to_subgraph_starting_vertex_parent = *tmp_w_to_subgraph_starting_vertex_parent;
      }
      std::cout << "reachable_from_2cores=" << reachable_from_2cores << " in_2cores=" << in_2cores
                << " starting_vertex=" << starting_vertex
                << " subgraph_starting_vertex=" << subgraph_starting_vertex << std::endl;

      vertex_t sssp_starting_vertex{};
      if (subgraph_starting_vertex_vertex_partition_id == vertex_partition_id) {
        sssp_starting_vertex =
          reachable_from_2cores ? mg_pruned_graph_view.local_vertex_partition_range_first() +
                                    mg_graph_to_pruned_graph_map.element(
                                      subgraph_starting_vertex - local_vertex_partition_range_first,
                                      handle_->get_stream())
                                : mg_isolated_trees_view.local_vertex_partition_range_first() +
                                    mg_graph_to_isolated_trees_map.element(
                                      subgraph_starting_vertex - local_vertex_partition_range_first,
                                      handle_->get_stream());
      }
      sssp_starting_vertex = cugraph::host_scalar_bcast(
        comm,
        sssp_starting_vertex,
        cugraph::partition_manager::compute_global_comm_rank_from_vertex_partition_id(
          major_comm_size, minor_comm_size, subgraph_starting_vertex_vertex_partition_id),
        handle_->get_stream());

      rmm::device_uvector<vertex_t> d_mg_sssp_predecessors(
        reachable_from_2cores ? mg_pruned_graph_view.local_vertex_partition_range_size()
                              : mg_isolated_trees_view.local_vertex_partition_range_size(),
        handle_->get_stream());
      rmm::device_uvector<weight_t> d_mg_sssp_distances(d_mg_sssp_predecessors.size(),
                                                        handle_->get_stream());

      cugraph::sssp(*handle_,
                    reachable_from_2cores ? mg_pruned_graph_view : mg_isolated_trees_view,
                    reachable_from_2cores ? mg_pruned_graph_edge_weight_view
                                          : mg_isolated_trees_edge_weight_view,
                    d_mg_sssp_distances.data(),
                    d_mg_sssp_predecessors.data(),
                    sssp_starting_vertex,
                    std::numeric_limits<weight_t>::max());

      if (reachable_from_2cores && mg_pruned_graph_pred_unrenumber_cache) {
        mg_pruned_graph_pred_unrenumber_cache->unrenumber(
          *handle_,
          raft::device_span<vertex_t>(d_mg_sssp_predecessors.data(),
                                      d_mg_sssp_predecessors.size()));
      } else {
        cugraph::unrenumber_int_vertices<vertex_t, multi_gpu>(
          *handle_,
          d_mg_sssp_predecessors.data(),
          d_mg_sssp_predecessors.size(),
          reachable_from_2cores ? mg_pruned_graph_renumber_map.data()
                                : mg_isolated_trees_renumber_map.data(),
          reachable_from_2cores ? mg_pruned_graph_view.vertex_partition_range_lasts()
                                : mg_isolated_trees_view.vertex_partition_range_lasts());
      }

      thrust::scatter(handle_->get_thrust_policy(),
                      d_mg_sssp_predecessors.begin(),
                      d_mg_sssp_predecessors.end(),
                      reachable_from_2cores ? mg_pruned_graph_to_graph_map.begin()
                                            : mg_isolated_trees_to_graph_map.begin(),
                      d_mg_unrenumbered_predecessors.begin());

      {  // update the starting vertex's parent
        if (subgraph_starting_vertex_vertex_partition_id ==
            vertex_partition_id) {  // cugraph::sssp sets the predecessor of the starting vertex to
                                    // invalid_vertex
          if (subgraph_starting_vertex_distance > weight_t{0.0}) {
            d_mg_unrenumbered_predecessors.set_element_async(
              subgraph_starting_vertex - local_vertex_partition_range_first,
              unrenumbered_subgraph_starting_vertex_parent,
              handle_->get_stream());
          } else {
            assert(starting_vertex == subgraph_starting_vertex);
            d_mg_unrenumbered_predecessors.set_element_async(
              starting_vertex - local_vertex_partition_range_first,
              unrenumbered_starting_vertex,
              handle_->get_stream());
          }
          handle_->sync_stream();
        }
      }

      // update d_mg_distances

      if (reachable_from_2cores) {
        if (subgraph_starting_vertex_distance > weight_t{0.0}) {
          thrust::transform(handle_->get_thrust_policy(),
                            d_mg_sssp_distances.begin(),
                            d_mg_sssp_distances.end(),
                            d_mg_sssp_distances.begin(),
                            cuda::proclaim_return_type<weight_t>(
                              [delta = subgraph_starting_vertex_distance] __device__(auto d) {
                                if (d != invalid_distance) {
                                  return d + delta;
                                } else {
                                  return invalid_distance;
                                }
                              }));
        }
        thrust::scatter(handle_->get_thrust_policy(),
                        d_mg_sssp_distances.begin(),
                        d_mg_sssp_distances.end(),
                        mg_pruned_graph_to_graph_map.begin(),
                        d_mg_distances.begin());
        update_unvisited_vertex_distances<vertex_t, weight_t>(
          *handle_,
          raft::device_span<vertex_t const>(parents.data(), parents.size()),
          raft::device_span<weight_t const>(w_to_parents.data(), w_to_parents.size()),
          raft::device_span<vertex_t const>(components.data(), components.size()),
          raft::host_span<vertex_t const>(vertex_partition_range_offsets.data(),
                                          vertex_partition_range_offsets.size()),
          raft::device_span<weight_t>(d_mg_distances.data(), d_mg_distances.size()),
          starting_vertex_component,
          local_vertex_partition_range_first,
          local_vertex_partition_range_last,
          invalid_distance);
      } else {
        assert(subgraph_starting_vertex_distance == vertex_t{0});
        thrust::scatter(handle_->get_thrust_policy(),
                        d_mg_sssp_distances.begin(),
                        d_mg_sssp_distances.end(),
                        mg_isolated_trees_to_graph_map.begin(),
                        d_mg_distances.begin());
      }

      if (cugraph::test::g_perf) {
        RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
        comm.barrier();
        elapsed = hr_timer.stop();
        if (i >= num_warmup_starting_vertices) { total_elapsed += elapsed; }
        hr_timer.display_and_clear(std::cout);
      }

      /* compute the number of visisted edges */

      {
        edge_t visited_edge_count = compute_number_of_visited_undirected_edges(
          *handle_,
          raft::device_span<weight_t const>(d_mg_distances.data(), d_mg_distances.size()),
          mg_pruned_graph_view,
          reachable_from_2cores ? std::make_optional(raft::device_span<weight_t const>(
                                    d_mg_sssp_distances.data(), d_mg_sssp_distances.size()))
                                : std::nullopt,
          raft::device_span<vertex_t const>(mg_graph_to_pruned_graph_map.data(),
                                            mg_graph_to_pruned_graph_map.size()),
          invalid_vertex,
          invalid_distance);
        auto tteps = (static_cast<double>(visited_edge_count) / 1e12) / elapsed;
        if (i >= num_warmup_starting_vertices) {
          tteps_sum += tteps;
          one_over_tteps_sum +=
            (tteps > 0.0) ? 1.0 / tteps : std::numeric_limits<double>::infinity();
        }
        std::cout << "# visited undirected edges=" << visited_edge_count
                  << " TTEPS=" << (static_cast<double>(visited_edge_count) / 1e12) / elapsed
                  << std::endl;
      }

      if (sssp_usecase.validate) {
        /* renumber for validation */

        rmm::device_uvector<vertex_t> d_mg_predecessors(d_mg_unrenumbered_predecessors.size(),
                                                        handle_->get_stream());
        thrust::copy(handle_->get_thrust_policy(),
                     d_mg_unrenumbered_predecessors.begin(),
                     d_mg_unrenumbered_predecessors.end(),
                     d_mg_predecessors.begin());
        cugraph::renumber_ext_vertices<vertex_t, multi_gpu>(*handle_,
                                                            d_mg_predecessors.data(),
                                                            d_mg_predecessors.size(),
                                                            mg_renumber_map.data(),
                                                            local_vertex_partition_range_first,
                                                            local_vertex_partition_range_last);

        /* check starting vertex's predecessor */

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.start("validate (starting vertex's predecessor)");
        }

        {
          size_t num_invalids{0};
          if (starting_vertex_vertex_partition_id == vertex_partition_id) {
            auto starting_vertex_predecessor = d_mg_predecessors.element(
              starting_vertex - local_vertex_partition_range_first, handle_->get_stream());
            if (starting_vertex_predecessor != starting_vertex) { ++num_invalids; }
          }
          num_invalids = cugraph::host_scalar_allreduce(
            comm, num_invalids, raft::comms::op_t::SUM, handle_->get_stream());
          ASSERT_EQ(num_invalids, 0) << "predecessor of a starting vertex should be itself";
        }

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.stop();
          hr_timer.display_and_clear(std::cout);
        }

        /* check for cycles (update predecessor to predecessor's predecessor till reaching the
         * starting vertex, if there exists a cycle, this won't finish) */

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.start("validate (cycle)");
        }

        {
          bool test_passed = is_valid_predecessor_tree(
            *handle_,
            raft::device_span<vertex_t const>(d_mg_predecessors.data(), d_mg_predecessors.size()),
            raft::host_span<vertex_t const>(vertex_partition_range_offsets.data(),
                                            vertex_partition_range_offsets.size()),
            starting_vertex,
            local_vertex_partition_range_first,
            invalid_vertex);
          ASSERT_TRUE(test_passed) << "SSSP predecessor tree is invalid (failed to backtrace to "
                                      "the starting vertex) or has a cycle.";
        }

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.stop();
          hr_timer.display_and_clear(std::cout);
        }

        /* check that distance(v) = distance(predecssor(v)) + w */

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.start("validate (predecessor tree distances)");
        }

        {
          if (reachable_from_2cores) {
            auto pair_first = thrust::make_zip_iterator(components.begin(), w_to_parents.begin());
            thrust::transform_if(
              handle_->get_thrust_policy(),
              pair_first,
              pair_first + components.size(),
              d_mg_w_to_predecessors.begin(),
              d_mg_w_to_predecessors.begin(),
              cuda::proclaim_return_type<weight_t>(
                [starting_vertex_component] __device__(auto pair) {
                  auto c           = cuda::std::get<0>(pair);
                  auto w_to_parent = cuda::std::get<1>(pair);
                  return (c == starting_vertex_component)
                           ? w_to_parent /* for the vertices in 2-cores, this will be overwritten */
                           : invalid_distance;
                }),
              cuda::proclaim_return_type<bool>([invalid_distance] __device__(auto w_to_pred) {
                return w_to_pred == invalid_distance;  // otherwise, already updated (the vertidces
                                                       // in the path from the starting vertex to
                                                       // the first reachable 2-core vertex)
              }));
          }

          if (subgraph_starting_vertex_vertex_partition_id == vertex_partition_id) {
            d_mg_w_to_predecessors.set_element_async(
              subgraph_starting_vertex - local_vertex_partition_range_first,
              w_to_subgraph_starting_vertex_parent,
              handle_->get_stream());
          }

          {
            auto mg_subgraph_view =
              reachable_from_2cores ? mg_pruned_graph_view : mg_isolated_trees_view;
            auto mg_subgraph_edge_weight_view = reachable_from_2cores
                                                  ? mg_pruned_graph_edge_weight_view
                                                  : mg_isolated_trees_edge_weight_view;
            auto mg_subgraph_renumber_map_ptr = reachable_from_2cores
                                                  ? mg_pruned_graph_renumber_map.data()
                                                  : mg_isolated_trees_renumber_map.data();

            rmm::device_uvector<vertex_t> tree_srcs(
              d_mg_sssp_predecessors.size() /* unrenumbered */, handle_->get_stream());
            rmm::device_uvector<vertex_t> tree_dsts(tree_srcs.size(), handle_->get_stream());
            auto input_pair_first =
              thrust::make_zip_iterator(d_mg_sssp_predecessors.begin(),
                                        thrust::make_counting_iterator(
                                          mg_subgraph_view.local_vertex_partition_range_first()));
            auto output_pair_first =
              thrust::make_zip_iterator(tree_srcs.begin(), tree_dsts.begin());
            tree_srcs.resize(
              cuda::std::distance(
                output_pair_first,
                thrust::copy_if(handle_->get_thrust_policy(),
                                input_pair_first,
                                input_pair_first + d_mg_sssp_predecessors.size(),
                                output_pair_first,
                                cuda::proclaim_return_type<bool>(
                                  [sssp_starting_vertex, invalid_vertex] __device__(auto pair) {
                                    auto pred = cuda::std::get<0>(pair);
                                    auto v    = cuda::std::get<1>(pair);
                                    return (pred != invalid_vertex) && (v != sssp_starting_vertex);
                                  }))),
              handle_->get_stream());
            tree_dsts.resize(tree_srcs.size(), handle_->get_stream());
            cugraph::renumber_ext_vertices<vertex_t, multi_gpu>(
              *handle_,
              tree_srcs.data(),
              tree_srcs.size(),
              mg_subgraph_renumber_map_ptr,
              mg_subgraph_view.local_vertex_partition_range_first(),
              mg_subgraph_view.local_vertex_partition_range_last());
            std::tie(tree_srcs, tree_dsts, std::ignore, std::ignore) =
              cugraph::shuffle_int_edges(*handle_,
                                         std::move(tree_srcs),
                                         std::move(tree_dsts),
                                         std::vector<cugraph::arithmetic_device_uvector_t>{},
                                         store_transposed,
                                         mg_subgraph_view.vertex_partition_range_lasts());
            cugraph::edge_bucket_t<vertex_t, edge_t, !store_transposed, multi_gpu, true> edge_list(
              *handle_, false);
            auto edge_pair_first = thrust::make_zip_iterator(tree_srcs.begin(), tree_dsts.begin());
            thrust::sort(
              handle_->get_thrust_policy(), edge_pair_first, edge_pair_first + tree_srcs.size());
            edge_list.insert(tree_srcs.begin(),
                             tree_srcs.end(),
                             tree_dsts.begin(),
                             std::optional<edge_t const*>{std::nullopt});
            tree_srcs.resize(0, handle_->get_stream());
            tree_srcs.shrink_to_fit(handle_->get_stream());
            rmm::device_uvector<weight_t> tree_weights(edge_list.size(), handle_->get_stream());
            cugraph::transform_gather_e(
              *handle_,
              mg_subgraph_view,
              edge_list,
              cugraph::edge_src_dummy_property_t{}.view(),
              cugraph::edge_dst_dummy_property_t{}.view(),
              mg_subgraph_edge_weight_view,
              cuda::proclaim_return_type<weight_t>(
                [] __device__(auto, auto, auto, auto, auto w) { return w; }),
              tree_weights.begin());
            std::vector<cugraph::arithmetic_device_uvector_t> edge_dst_properties{};
            edge_dst_properties.push_back(std::move(tree_weights));
            std::tie(tree_dsts, edge_dst_properties) =
              cugraph::shuffle_local_edge_dsts(*handle_,
                                               std::move(tree_dsts),
                                               std::move(edge_dst_properties),
                                               mg_subgraph_view.vertex_partition_range_lasts(),
                                               store_transposed);
            tree_weights =
              std::move(std::get<rmm::device_uvector<weight_t>>(edge_dst_properties[0]));
            thrust::scatter(
              handle_->get_thrust_policy(),
              tree_weights.begin(),
              tree_weights.end(),
              thrust::make_transform_iterator(
                tree_dsts.begin(),
                cuda::proclaim_return_type<vertex_t>(
                  [map_first = reachable_from_2cores ? mg_pruned_graph_to_graph_map.begin()
                                                     : mg_isolated_trees_to_graph_map.begin(),
                   v_first =
                     mg_subgraph_view.local_vertex_partition_range_first()] __device__(auto v) {
                    return *(map_first + (v - v_first));
                  })),
              d_mg_w_to_predecessors.begin());
          }

          handle_->sync_stream();  // make sure the above d_mg_w_to_predecessors.set_element_async()
                                   // finishes before w_to_subgraph_starting_vertex_parent becomes
                                   // out-of-scope
          bool test_passed = check_distance_from_parents<vertex_t, weight_t>(
            *handle_,
            raft::device_span<vertex_t const>(d_mg_predecessors.data(), d_mg_predecessors.size()),
            raft::device_span<weight_t const>(d_mg_distances.data(), d_mg_distances.size()),
            raft::device_span<weight_t const>(d_mg_w_to_predecessors.data(),
                                              d_mg_w_to_predecessors.size()),
            raft::host_span<vertex_t const>(vertex_partition_range_offsets.data(),
                                            vertex_partition_range_offsets.size()),
            starting_vertex,
            local_vertex_partition_range_first,
            invalid_vertex);
          ASSERT_TRUE(test_passed) << " source and destination vertices in the BFS predecessor "
                                      "tree are not the edge weight away";
        }

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.stop();
          hr_timer.display_and_clear(std::cout);
        }

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.start("validate (graph distances)");
        }

        /* for every edge e = (u, v), abs(dist(u) - dist(v)) <= w or dist(u) == dist(v) ==
         * invalid_distance */

        {
          bool test_passed = check_edge_endpoint_distances<vertex_t, edge_t, weight_t>(
            *handle_,
            raft::device_span<vertex_t const>(parents.data(), parents.size()),
            raft::device_span<weight_t const>(w_to_parents.data(), w_to_parents.size()),
            raft::device_span<weight_t const>(d_mg_distances.data(), d_mg_distances.size()),
            mg_pruned_graph_view,
            mg_pruned_graph_edge_weight_view,
            raft::device_span<vertex_t const>(mg_graph_to_pruned_graph_map.data(),
                                              mg_graph_to_pruned_graph_map.size()),
            mg_isolated_trees_view,
            mg_isolated_trees_edge_weight_view,
            raft::device_span<vertex_t const>(mg_graph_to_isolated_trees_map.data(),
                                              mg_graph_to_isolated_trees_map.size()),
            raft::host_span<vertex_t const>(vertex_partition_range_offsets.data(),
                                            vertex_partition_range_offsets.size()),
            local_vertex_partition_range_first,
            invalid_vertex,
            invalid_distance,
            reachable_from_2cores);
          ASSERT_TRUE(test_passed)
            << " only one of the two connected vertices are reachable from the starting vertex or "
               "the distance from the starting vertex differ by more than the edge weight.";
        }

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.stop();
          hr_timer.display_and_clear(std::cout);
        }

        /* all the reachable vertices are in the same connected component, all the unreachable
         * vertices in different connected components */

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.start("validate (connected components)");
        }

        {
          bool test_passed = check_connected_components(
            *handle_,
            raft::device_span<vertex_t const>(components.data(), components.size()),
            raft::device_span<vertex_t const>(d_mg_predecessors.data(), d_mg_predecessors.size()),
            starting_vertex_component,
            invalid_vertex);
          ASSERT_TRUE(test_passed)
            << "the SSSP tree does not span the entire connected component of the starting vertex.";
        }

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.stop();
          hr_timer.display_and_clear(std::cout);
        }

        /* check that predecessor->v edges exist in the input graph */

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.start("validate (predecessor->v edge existence)");
        }

        {
          bool test_passed = check_has_edge_from_parents(
            *handle_,
            raft::device_span<vertex_t const>(parents.data(), parents.size()),
            raft::device_span<vertex_t const>(d_mg_predecessors.data(), d_mg_predecessors.size()),
            mg_pruned_graph_view,
            raft::device_span<vertex_t const>(mg_graph_to_pruned_graph_map.data(),
                                              mg_graph_to_pruned_graph_map.size()),
            mg_isolated_trees_view,
            raft::device_span<vertex_t const>(mg_graph_to_isolated_trees_map.data(),
                                              mg_graph_to_isolated_trees_map.size()),
            raft::host_span<vertex_t const>(vertex_partition_range_offsets.data(),
                                            vertex_partition_range_offsets.size()),
            starting_vertex,
            local_vertex_partition_range_first,
            invalid_vertex,
            reachable_from_2cores,
            in_2cores);
          ASSERT_TRUE(test_passed) << "predecessor->v missing in the input graph.";
        }

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.stop();
          hr_timer.display_and_clear(std::cout);
        }
      }
    }

    std::cout << "average MG SSSP (Kernel 2) time: "
              << (total_elapsed / num_timed_starting_vertices)
              << " TTEPS (arithmetic mean)=" << tteps_sum / num_timed_starting_vertices
              << " TTEPS (harmonic_mean)="
              << (one_over_tteps_sum > 0.0
                    ? static_cast<double>(num_timed_starting_vertices) / one_over_tteps_sum
                    : std::numeric_limits<double>::infinity())
              << std::endl;
  }

 private:
  static std::unique_ptr<raft::handle_t> handle_;
};

template <typename input_usecase_t>
std::unique_ptr<raft::handle_t> Tests_GRAPH500_MGSSSP<input_usecase_t>::handle_ = nullptr;

using Tests_GRAPH500_MGSSSP_Rmat = Tests_GRAPH500_MGSSSP<cugraph::test::Rmat_Usecase>;

TEST_P(Tests_GRAPH500_MGSSSP_Rmat, CheckInt64Int64)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t>(
    std::get<0>(param), override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

INSTANTIATE_TEST_SUITE_P(
  rmat_small_test,
  Tests_GRAPH500_MGSSSP_Rmat,
  ::testing::Values(
    // enable correctness checks
    std::make_tuple(Graph500_SSSP_Usecase{true, true, true},
                    cugraph::test::Rmat_Usecase(10,
                                                16,
                                                0.57,
                                                0.19,
                                                0.19,
                                                0 /* base RNG seed */,
                                                true /* undirected */,
                                                true /* scramble vertex ID */))));

INSTANTIATE_TEST_SUITE_P(
  rmat_benchmark_test, /* note that scale & edge factor can be overridden in benchmarking (with
                          --gtest_filter to select only the rmat_benchmark_test with a specific
                          vertex & edge type combination) by command line arguments and do not
                          include more than one Rmat_Usecase that differ only in scale or edge
                          factor (to avoid running same benchmarks more than once) */
  Tests_GRAPH500_MGSSSP_Rmat,
  ::testing::Values(
    // disable correctness checks for large graphs
    std::make_tuple(Graph500_SSSP_Usecase{true, true, false},
                    cugraph::test::Rmat_Usecase(20,
                                                16,
                                                0.57,
                                                0.19,
                                                0.19,
                                                0 /* base RNG seed */,
                                                true /* undirected */,
                                                true /* scramble vertex IDs */))));

CUGRAPH_MG_TEST_PROGRAM_MAIN()
