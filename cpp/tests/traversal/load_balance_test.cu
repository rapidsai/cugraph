#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <utilities/base_fixture.hpp>
#include <utilities/error.hpp>
#include <utilities/test_utilities.hpp>

#include <traversal/mg/load_balance.cuh>

#include <raft/handle.hpp>

// ============================================================================
// Test Suite
// ============================================================================
typedef struct LB_Usecase_t {
  std::string config_;     // Path to graph file
  std::string file_path_;  // Complete path to graph using dataset_root_dir
  LB_Usecase_t(const std::string &config) : config_(config)
  {
    const std::string &rapidsDatasetRootDir = cugraph::test::get_rapids_dataset_root_dir();
    if ((config_ != "") && (config_[0] != '/')) {
      file_path_ = rapidsDatasetRootDir + "/" + config_;
    } else {
      file_path_ = config_;
    }
  };
} LB_Usecase;

template <typename VT, typename ET>
struct in_degree {
  ET *in_degree_counter_;
  in_degree(ET *in_degree_counter) : in_degree_counter_(in_degree_counter) {}
  __device__ void operator()(VT src, VT dst) { atomicAdd(in_degree_counter_ + dst, 1); }
};

template <typename VT, typename ET>
struct in_degree_simple {
  ET *in_degree_counter_;
  in_degree_simple(ET *in_degree_counter) : in_degree_counter_(in_degree_counter) {}
  __device__ void operator()(VT dst) { atomicAdd(in_degree_counter_ + dst, 1); }
};

class Tests_LB : public ::testing::TestWithParam<LB_Usecase> {
 public:
  Tests_LB() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}

  // VT                 vertex identifier data type
  // ET                 edge identifier data type
  // WT                 edge weight data type
  template <typename VT, typename ET, typename WT>
  void run_current_test(const LB_Usecase &configuration)
  {
    // Step 1: Construction of the graph based on configuration
    // VT number_of_vertices;
    // ET number_of_edges;
    bool directed = false;
    auto csr =
      cugraph::test::generate_graph_csr_from_mm<VT, ET, WT>(directed, configuration.file_path_);
    hipDeviceSynchronize();
    cugraph::GraphCSRView<VT, ET, WT> G = csr->view();
    G.prop.directed                     = directed;

    rmm::device_vector<ET> in_degree_lb(G.number_of_vertices, 0);

    raft::handle_t handle;
    cugraph::mg::detail::LoadBalanceExecution<VT, ET, WT> lb(handle, G);
    CUDA_TRY(hipGetLastError());
    in_degree<VT, ET> in_degree_op(in_degree_lb.data().get());
    lb.run(in_degree_op);
    CUDA_TRY(hipGetLastError());

    hipStream_t stream = 0;

    // Calculate the in degree of destinations
    rmm::device_vector<ET> gold_indegree(G.number_of_vertices, 0);
    rmm::device_vector<VT> destinations(G.number_of_edges);
    CUDA_TRY(hipMemcpy(destinations.data().get(),
                        G.indices,
                        sizeof(VT) * destinations.size(),
                        hipMemcpyDeviceToDevice));
    thrust::for_each(rmm::exec_policy(stream)->on(stream),
                     destinations.begin(),
                     destinations.end(),
                     in_degree_simple<VT, ET>(gold_indegree.data().get()));

    bool is_result_equal = thrust::equal(rmm::exec_policy(stream)->on(stream),
                                         in_degree_lb.begin(),
                                         in_degree_lb.end(),
                                         gold_indegree.begin());

    EXPECT_TRUE(is_result_equal);
  }
};

TEST_P(Tests_LB, CheckFP32_SP_COUNTER) { run_current_test<int, int, float>(GetParam()); }

INSTANTIATE_TEST_CASE_P(simple_test,
                        Tests_LB,
                        ::testing::Values(LB_Usecase("test/datasets/karate.mtx"),
                                          LB_Usecase("test/datasets/polbooks.mtx"),
                                          LB_Usecase("test/datasets/netscience.mtx"),
                                          LB_Usecase("test/datasets/netscience.mtx"),
                                          LB_Usecase("test/datasets/wiki2003.mtx"),
                                          LB_Usecase("test/datasets/wiki-Talk.mtx")));

CUGRAPH_TEST_PROGRAM_MAIN()
