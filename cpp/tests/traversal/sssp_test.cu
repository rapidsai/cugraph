#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited
 *
 */

#include <gtest/gtest.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <algorithm>
#include <queue>
#include <unordered_map>
#include <utility>
#include "high_res_clock.h"

#include "utilities/test_utilities.hpp"

#include <converters/COOtoCSR.cuh>

#include <rmm/mr/device/cuda_memory_resource.hpp>
#include "algorithms.hpp"
#include "graph.hpp"

typedef enum graph_type { RMAT, MTX } GraphType;

template <typename MaxEType, typename MaxVType, typename DistType>
void ref_bfs(const std::vector<MaxEType>& rowPtr,
             const std::vector<MaxVType>& colInd,
             const MaxVType source_vertex,
             std::vector<DistType>& distances,
             std::vector<MaxVType>& predecessors)
{
  typename std::vector<MaxEType>::size_type n   = rowPtr.size() - 1;
  typename std::vector<MaxVType>::size_type nnz = colInd.size();

  ASSERT_LE(n, static_cast<decltype(n)>(std::numeric_limits<MaxVType>::max()) - 1);
  ASSERT_LE(nnz, static_cast<decltype(nnz)>(std::numeric_limits<MaxEType>::max()));
  ASSERT_EQ(distances.size(), rowPtr.size() - 1);

  std::fill(distances.begin(), distances.end(), std::numeric_limits<DistType>::max());
  std::fill(predecessors.begin(), predecessors.end(), -1);

  std::queue<MaxVType> q;
  q.push(source_vertex);
  distances[source_vertex] = 0;

  while (!q.empty()) {
    MaxVType u = q.front();
    q.pop();

    for (auto iCol = rowPtr[u]; iCol != rowPtr[u + 1]; ++iCol) {
      MaxVType v = colInd[iCol];
      // undiscovered
      if (distances[v] == std::numeric_limits<DistType>::max()) {
        distances[v]    = distances[u] + 1;
        predecessors[v] = u;
        q.push(v);
      }
    }
  }
}

template <typename MaxEType, typename MaxVType, typename DistType>
void ref_sssp(const std::vector<MaxEType>& rowPtr,
              const std::vector<MaxVType>& colInd,
              const std::vector<DistType>& weights,
              const MaxVType source_vertex,
              std::vector<DistType>& distances,
              std::vector<MaxVType>& predecessors)
{
  typename std::vector<MaxEType>::size_type n   = rowPtr.size() - 1;
  typename std::vector<MaxVType>::size_type nnz = colInd.size();

  ASSERT_LE(n, static_cast<decltype(n)>(std::numeric_limits<MaxVType>::max()) - 1);
  ASSERT_LE(nnz, static_cast<decltype(nnz)>(std::numeric_limits<MaxEType>::max()));
  ASSERT_EQ(nnz, weights.size());
  ASSERT_EQ(distances.size(), rowPtr.size() - 1);

  std::fill(distances.begin(), distances.end(), std::numeric_limits<DistType>::max());
  std::fill(predecessors.begin(), predecessors.end(), -1);

  std::set<MaxVType> curr_frontier;
  curr_frontier.insert(source_vertex);
  distances[source_vertex] = 0;
  MaxVType nf              = 1;

  while (nf > 0) {
    std::set<MaxVType> next_frontier;
    for (auto it = curr_frontier.begin(); it != curr_frontier.end(); ++it) {
      MaxVType u = *it;

      for (auto iCol = rowPtr[u]; iCol != rowPtr[u + 1]; ++iCol) {
        MaxVType v = colInd[iCol];
        // relax
        if (distances[u] + weights[iCol] < distances[v]) {
          distances[v] = distances[u] + weights[iCol];
          next_frontier.insert(v);
          predecessors[v] = u;
        }
      }
    }

    curr_frontier = next_frontier;
    nf            = curr_frontier.size();
  }
}

// do the perf measurements
// enabled by command line parameter s'--perf'
static int PERF = 0;

// iterations for perf tests
// enabled by command line parameter '--perf-iters"
static int PERF_MULTIPLIER = 5;

typedef struct SSSP_Usecase_t {
  GraphType type_;
  std::string config_;
  std::string file_path_;
  uint64_t src_;
  SSSP_Usecase_t(const GraphType& type, const std::string& config, const int src)
    : type_(type), config_(config), src_(src)
  {
    // assume relative paths are relative to RAPIDS_DATASET_ROOT_DIR
    // FIXME: Use platform independent stuff from c++14/17 on compiler update
    if (type_ == MTX) {
      const std::string& rapidsDatasetRootDir = cugraph::test::get_rapids_dataset_root_dir();
      if ((config_ != "") && (config_[0] != '/')) {
        file_path_ = rapidsDatasetRootDir + "/" + config_;
      } else {
        file_path_ = config_;
      }
    }
  };
} SSSP_Usecase;

class Tests_SSSP : public ::testing::TestWithParam<SSSP_Usecase> {
 public:
  Tests_SSSP() {}
  static void SetupTestCase() {}
  static void TearDownTestCase()
  {
    if (PERF) {
      for (size_t i = 0; i < SSSP_time.size(); ++i) {
        std::cout << SSSP_time[i] / PERF_MULTIPLIER << std::endl;
      }
    }
  }
  virtual void SetUp() {}
  virtual void TearDown() {}

  static std::vector<double> SSSP_time;

  // MaxVType:        Data type of vertices id (signed int-32)
  // MaxEType:        Data type of edges id (signed int-32)
  // DistType:        Data type of weights (float / double)
  // DoRandomWeights: SSSP Implementation requires weights to operate,
  //                    if true: generate random weights before calling
  //                    else:    relies on self inner code using fake 1.0 weights
  // DoDist:          User can provide weights or not, simulate this behavior
  // DoPreds:         User can provide predecessors or not, simulate this behavior
  template <typename MaxVType,
            typename MaxEType,
            typename DistType,
            bool DoRandomWeights,
            bool DoDist,
            bool DoPreds>
  void run_current_test(const SSSP_Usecase& param)
  {
    // Allocate memory on host (We will resize later on)
    std::vector<MaxVType> cooRowInd;
    std::vector<MaxVType> cooColInd;
    std::vector<DistType> cooVal;

    DistType* distances = nullptr;
    MaxVType* preds     = nullptr;

    MaxVType num_vertices;
    MaxEType num_edges;
    const MaxVType src = param.src_;

    ASSERT_LE(param.src_, static_cast<uint64_t>(std::numeric_limits<MaxVType>::max()));
    // src = static_cast<MaxVType>(param.src_);

    // Input
    ASSERT_TRUE(typeid(MaxVType) == typeid(int));  // We don't have support for other types yet
    ASSERT_TRUE(typeid(MaxEType) == typeid(int));  // We don't have support for other types yet
    ASSERT_TRUE((typeid(DistType) == typeid(float)) || (typeid(DistType) == typeid(double)));
    if (param.type_ == RMAT) {
      // This is size_t due to grmat_gen which should be fixed there
      // FIXME: rmat is disabled
      return;
    } else if (param.type_ == MTX) {
      MaxVType m, k;
      MaxEType nnz;
      MM_typecode mc;

      FILE* fpin = fopen(param.file_path_.c_str(), "r");
      ASSERT_NE(fpin, static_cast<FILE*>(nullptr)) << "fopen (" << param.file_path_ << ") failure.";

      // mm_properties has only one template param which should be fixed there
      ASSERT_EQ(cugraph::test::mm_properties<MaxVType>(fpin, 1, &mc, &m, &k, &nnz), 0)
        << "could not read Matrix Market file properties"
        << "\n";
      ASSERT_TRUE(mm_is_matrix(mc));
      ASSERT_TRUE(mm_is_coordinate(mc));
      ASSERT_FALSE(mm_is_complex(mc));
      ASSERT_FALSE(mm_is_skew(mc));

      // Allocate memory on host
      cooRowInd.resize(nnz);
      cooColInd.resize(nnz);

      // Read weights if given
      if (!mm_is_pattern(mc)) {
        cooVal.resize(nnz);
        ASSERT_EQ((cugraph::test::mm_to_coo(fpin,
                                            1,
                                            nnz,
                                            &cooRowInd[0],
                                            &cooColInd[0],
                                            &cooVal[0],
                                            static_cast<DistType*>(nullptr))),
                  0)
          << "could not read matrix data"
          << "\n";
      } else {
        ASSERT_EQ((cugraph::test::mm_to_coo(fpin,
                                            1,
                                            nnz,
                                            &cooRowInd[0],
                                            &cooColInd[0],
                                            static_cast<DistType*>(nullptr),
                                            static_cast<DistType*>(nullptr))),
                  0)
          << "could not read matrix data"
          << "\n";
        // Set random weights
        if (std::is_same<DistType, float>::value || std::is_same<DistType, double>::value) {
          cooVal.resize(nnz);
          for (auto i = 0; i < nnz; i++) {
            cooVal[i] = static_cast<DistType>(rand()) / static_cast<DistType>(RAND_MAX);
          }
        }
      }

      ASSERT_EQ(fclose(fpin), 0);

      num_vertices = m;
      num_edges    = nnz;
    } else {
      ASSERT_TRUE(0);
    }

    cugraph::experimental::GraphCOOView<MaxVType, MaxEType, DistType> G_coo(
      &cooRowInd[0],
      &cooColInd[0],
      (DoRandomWeights ? &cooVal[0] : nullptr),
      num_vertices,
      num_edges);
    auto G_unique = cugraph::coo_to_csr(G_coo);
    cugraph::experimental::GraphCSRView<MaxVType, MaxEType, DistType> G = G_unique->view();
    hipDeviceSynchronize();

    std::vector<DistType> dist_vec;
    std::vector<MaxVType> pred_vec;
    rmm::device_vector<DistType> ddist_vec;
    rmm::device_vector<MaxVType> dpred_vec;

    if (DoDist) {
      dist_vec = std::vector<DistType>(num_vertices, std::numeric_limits<DistType>::max());
      // device alloc
      ddist_vec.resize(num_vertices);
      thrust::fill(ddist_vec.begin(), ddist_vec.end(), std::numeric_limits<DistType>::max());
      distances = thrust::raw_pointer_cast(ddist_vec.data());
    }

    if (DoPreds) {
      pred_vec = std::vector<MaxVType>(num_vertices, -1);
      dpred_vec.resize(num_vertices);
      preds = thrust::raw_pointer_cast(dpred_vec.data());
    }

    HighResClock hr_clock;
    double time_tmp;

    hipDeviceSynchronize();
    if (PERF) {
      hr_clock.start();
      for (auto i = 0; i < PERF_MULTIPLIER; ++i) {
        cugraph::sssp(G, distances, preds, src);
        hipDeviceSynchronize();
      }
      hr_clock.stop(&time_tmp);
      SSSP_time.push_back(time_tmp);
    } else {
      cugraph::sssp(G, distances, preds, src);
      hipDeviceSynchronize();
    }

    // MTX may have zero-degree vertices. So reset num_vertices after
    // conversion to CSR
    num_vertices = G.number_of_vertices;

    if (DoDist)
      hipMemcpy(
        (void*)&dist_vec[0], distances, sizeof(DistType) * num_vertices, hipMemcpyDeviceToHost);

    if (DoPreds)
      hipMemcpy(
        (void*)&pred_vec[0], preds, sizeof(MaxVType) * num_vertices, hipMemcpyDeviceToHost);

    // Create ref host structures
    std::vector<MaxEType> vlist(num_vertices + 1);
    std::vector<MaxVType> elist(num_edges);
    std::vector<DistType> ref_distances(num_vertices), weights(num_edges);
    std::vector<MaxVType> ref_predecessors(num_vertices);

    hipMemcpy(
      (void*)&vlist[0], G.offsets, sizeof(MaxEType) * (num_vertices + 1), hipMemcpyDeviceToHost);
    hipMemcpy((void*)&elist[0], G.indices, sizeof(MaxVType) * (num_edges), hipMemcpyDeviceToHost);
    if (G.edge_data != nullptr) {
      hipMemcpy(
        (void*)&weights[0], G.edge_data, sizeof(DistType) * (num_edges), hipMemcpyDeviceToHost);
    } else {  // If SSSP is given no weights it uses unit weights by default
      std::fill(weights.begin(), weights.end(), static_cast<DistType>(1));
    }

    std::map<std::pair<MaxVType, MaxVType>, DistType> min_edge_map;

    if (DoPreds) {
      for (auto i = 0; i < num_vertices; ++i) {
        for (auto offset = vlist[i]; offset < vlist[i + 1]; ++offset) {
          DistType weight = weights[offset];
          auto key        = std::make_pair(i, elist[offset]);
          if (min_edge_map.find(key) != min_edge_map.end()) {
            min_edge_map[key] = std::min(weight, min_edge_map[key]);
          } else {
            min_edge_map[key] = weight;
          }
        }
      }
    }

    ref_sssp(vlist, elist, weights, src, ref_distances, ref_predecessors);

    for (auto i = 0; i < num_vertices; ++i) {
      if (DoDist)
        ASSERT_EQ(dist_vec[i], ref_distances[i])
          << "vid: " << i << "ref dist " << ref_distances[i] << " actual dist " << dist_vec[i];

      if (DoPreds) {
        if (pred_vec[i] != -1) {
          auto key                 = std::make_pair(pred_vec[i], i);
          DistType min_edge_weight = min_edge_map.at(key);

          ASSERT_EQ(ref_distances[pred_vec[i]] + min_edge_weight, ref_distances[i])
            << "vid: " << i << "pred " << pred_vec[i] << " ref dist " << ref_distances[i]
            << " observed " << ref_distances[pred_vec[i]] << " + " << min_edge_weight << " = "
            << ref_distances[pred_vec[i]] + min_edge_weight << "\n";
        } else {
          ASSERT_EQ(pred_vec[i], ref_predecessors[i])
            << "vid: " << i << "ref pred " << ref_predecessors[i] << " actual " << pred_vec[i];
        }
      }
    }
  }
};

std::vector<double> Tests_SSSP::SSSP_time;

TEST_P(Tests_SSSP, CheckFP32_NO_RANDOM_DIST_NO_PREDS)
{
  run_current_test<int, int, float, false, true, false>(GetParam());
}
TEST_P(Tests_SSSP, CheckFP32_NO_RANDOM_NO_DIST_PREDS)
{
  run_current_test<int, int, float, false, false, true>(GetParam());
}
TEST_P(Tests_SSSP, CheckFP32_NO_RANDOM_DIST_PREDS)
{
  run_current_test<int, int, float, false, true, true>(GetParam());
}
TEST_P(Tests_SSSP, CheckFP64_NO_RANDOM_DIST_NO_PREDS)
{
  run_current_test<int, int, double, false, true, false>(GetParam());
}
TEST_P(Tests_SSSP, CheckFP64_NO_RANDOM_NO_DIST_PREDS)
{
  run_current_test<int, int, double, false, false, true>(GetParam());
}
TEST_P(Tests_SSSP, CheckFP64_NO_RANDOM_DIST_PREDS)
{
  run_current_test<int, int, double, false, true, true>(GetParam());
}

// FIXME: There might be some tests that are done twice (MTX that are not patterns)
TEST_P(Tests_SSSP, CheckFP32_RANDOM_DIST_NO_PREDS)
{
  run_current_test<int, int, float, true, true, false>(GetParam());
}
TEST_P(Tests_SSSP, CheckFP32_RANDOM_NO_DIST_PREDS)
{
  run_current_test<int, int, float, true, false, true>(GetParam());
}
TEST_P(Tests_SSSP, CheckFP32_RANDOM_DIST_PREDS)
{
  run_current_test<int, int, float, true, true, true>(GetParam());
}
TEST_P(Tests_SSSP, CheckFP64_RANDOM_DIST_NO_PREDS)
{
  run_current_test<int, int, double, true, true, false>(GetParam());
}
TEST_P(Tests_SSSP, CheckFP64_RANDOM_NO_DIST_PREDS)
{
  run_current_test<int, int, double, true, false, true>(GetParam());
}
TEST_P(Tests_SSSP, CheckFP64_RANDOM_DIST_PREDS)
{
  run_current_test<int, int, double, true, true, true>(GetParam());
}

// --gtest_filter=*simple_test*

INSTANTIATE_TEST_CASE_P(simple_test,
                        Tests_SSSP,
                        ::testing::Values(SSSP_Usecase(MTX, "test/datasets/dblp.mtx", 100),
                                          SSSP_Usecase(MTX, "test/datasets/wiki2003.mtx", 100000),
                                          SSSP_Usecase(MTX, "test/datasets/karate.mtx", 1)));

int main(int argc, char** argv)
{
  testing::InitGoogleTest(&argc, argv);
  auto resource = std::make_unique<rmm::mr::cuda_memory_resource>();
  rmm::mr::set_default_resource(resource.get());
  int rc = RUN_ALL_TESTS();
  return rc;
}
