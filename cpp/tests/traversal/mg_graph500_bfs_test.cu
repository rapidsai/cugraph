#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "detail/graph500_forest_pruning_utils.cuh"
#include "detail/graph_partition_utils.cuh"
#include "detail/shuffle_wrappers.hpp"
#include "nbr_unrenumber_cache.cuh"
#include "prims/count_if_e.cuh"
#include "prims/extract_transform_if_e.cuh"
#include "prims/fill_edge_src_dst_property.cuh"
#include "prims/kv_store.cuh"
#include "prims/transform_e.cuh"
#include "prims/update_edge_src_dst_property.cuh"
#include "utilities/base_fixture.hpp"
#include "utilities/collect_comm.cuh"
#include "utilities/conversion_utilities.hpp"
#include "utilities/device_comm_wrapper.hpp"
#include "utilities/mg_utilities.hpp"
#include "utilities/property_generator_utilities.hpp"
#include "utilities/test_graphs.hpp"
#include "utilities/thrust_wrapper.hpp"

#include <cugraph/algorithms.hpp>
#include <cugraph/edge_partition_endpoint_property_device_view.cuh>
#include <cugraph/edge_property.hpp>
#include <cugraph/edge_src_dst_property.hpp>
#include <cugraph/graph.hpp>
#include <cugraph/graph_functions.hpp>
#include <cugraph/graph_view.hpp>
#include <cugraph/large_buffer_manager.hpp>
#include <cugraph/partition_manager.hpp>
#include <cugraph/shuffle_functions.hpp>
#include <cugraph/utilities/device_functors.cuh>
#include <cugraph/utilities/high_res_timer.hpp>
#include <cugraph/utilities/misc_utils.cuh>
#include <cugraph/utilities/shuffle_comm.cuh>

#include <raft/comms/mpi_comms.hpp>
#include <raft/core/comms.hpp>
#include <raft/core/handle.hpp>

#include <rmm/device_scalar.hpp>
#include <rmm/device_uvector.hpp>

#include <thrust/merge.h>
#include <thrust/set_operations.h>
#include <thrust/sort.h>
#include <thrust/tabulate.h>
#include <thrust/transform.h>
#include <thrust/unique.h>

#include <gtest/gtest.h>

#include <random>

struct Graph500_BFS_Usecase {
  bool use_pruned_graph_unrenumber_cache{
    false};  // use cache to locally unrenumber (at the expense of additional memory usage)
  bool use_large_buffer{false};
  bool validate{true};
};

void init_nccl_env_variables() {}

template <typename input_usecase_t>
class Tests_GRAPH500_MGBFS
  : public ::testing::TestWithParam<std::tuple<Graph500_BFS_Usecase, input_usecase_t>> {
 public:
  Tests_GRAPH500_MGBFS() {}

  static void SetUpTestCase()
  {
    init_nccl_env_variables();

    size_t pool_size =
      12;  // note that CUDA_DEVICE_MAX_CONNECTIONS (default: 8) should be set to a value larger
           // than pool_size to avoid false dependency among different streams
    handle_ = cugraph::test::initialize_mg_handle(pool_size);

    cugraph::large_buffer_manager::init(
      *handle_, cugraph::large_buffer_manager::create_memory_buffer_resource(), std::nullopt);
  }

  static void TearDownTestCase() { handle_.reset(); }

  virtual void SetUp() {}
  virtual void TearDown() {}

  template <typename vertex_t, typename edge_t>
  void run_current_test(Graph500_BFS_Usecase const& bfs_usecase,
                        input_usecase_t const& input_usecase)
  {
    using weight_t    = float;    // dummy
    using edge_type_t = int32_t;  // dummy
    using edge_time_t = int32_t;  // dummy

    bool constexpr store_transposed = false;
    bool constexpr multi_gpu        = true;
    bool constexpr renumber         = true;
    bool constexpr test_weighted    = false;
    bool constexpr shuffle = false;  // Graph 500 requirement (edges can't be pre-shuffled, edges
                                     // should be shuffled in Kernel 1)
    size_t num_warmup_starting_vertices = 1;   // to enforce all CUDA & NCCL initializations
    size_t num_timed_starting_vertices  = 64;  // Graph 500 requirement (64)

    HighResTimer hr_timer{};
    raft::random::RngState rng_state{0};

    auto& comm           = handle_->get_comms();
    auto const comm_rank = comm.get_rank();
    auto const comm_size = comm.get_size();
    auto& major_comm     = handle_->get_subcomm(cugraph::partition_manager::major_comm_name());
    auto const major_comm_rank = major_comm.get_rank();
    auto const major_comm_size = major_comm.get_size();
    auto& minor_comm = handle_->get_subcomm(cugraph::partition_manager::minor_comm_name());
    auto const minor_comm_rank = minor_comm.get_rank();
    auto const minor_comm_size = minor_comm.get_size();
    auto vertex_partition_id =
      cugraph::partition_manager::compute_vertex_partition_id_from_graph_subcomm_ranks(
        major_comm_size, minor_comm_size, major_comm_rank, minor_comm_rank);

    std::cout << "comm_size=" << comm_size << " major_comm_size=" << major_comm_size
              << " minor_comm_size=" << minor_comm_size << std::endl;

    constexpr auto invalid_distance = std::numeric_limits<vertex_t>::max();
    constexpr auto invalid_vertex   = cugraph::invalid_vertex_id<vertex_t>::value;

    // 1. force NCCL P2P initialization

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      comm.barrier();
      hr_timer.start("NCCL P2P buffer initialization");
    }

    cugraph::test::enforce_p2p_initialization(comm, handle_->get_stream());
    cugraph::test::enforce_p2p_initialization(major_comm, handle_->get_stream());
    cugraph::test::enforce_p2p_initialization(minor_comm, handle_->get_stream());

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      comm.barrier();
      hr_timer.stop();
      hr_timer.display_and_clear(std::cout);
    }

    // 2. create a graph & meta data

    rmm::device_uvector<vertex_t> mg_renumber_map(0, handle_->get_stream());
    rmm::device_uvector<vertex_t> components(0, handle_->get_stream());
    rmm::device_uvector<vertex_t> parents(0, handle_->get_stream());
    rmm::device_uvector<vertex_t> unrenumbered_parents(0, handle_->get_stream());
    std::vector<vertex_t> vertex_partition_range_offsets(comm_size + 1);
    vertex_t local_vertex_partition_range_first{};
    vertex_t local_vertex_partition_range_last{};

    cugraph::graph_t<vertex_t, edge_t, store_transposed, multi_gpu> mg_pruned_graph(*handle_);
    rmm::device_uvector<vertex_t> mg_pruned_graph_renumber_map(0, handle_->get_stream());
    rmm::device_uvector<vertex_t> mg_graph_to_pruned_graph_map(
      0, handle_->get_stream());  // we may store this in host buffer to save HBM
    rmm::device_uvector<vertex_t> mg_pruned_graph_to_graph_map(0, handle_->get_stream());

    cugraph::graph_t<vertex_t, edge_t, store_transposed, multi_gpu> mg_isolated_trees(*handle_);
    rmm::device_uvector<vertex_t> mg_isolated_trees_renumber_map(0, handle_->get_stream());
    rmm::device_uvector<vertex_t> mg_graph_to_isolated_trees_map(
      0, handle_->get_stream());  // we may store this in host buffer to save HBM
    rmm::device_uvector<vertex_t> mg_isolated_trees_to_graph_map(0, handle_->get_stream());

    std::optional<cugraph::test::nbr_unrenumber_cache_t<vertex_t>>
      mg_pruned_graph_pred_unrenumber_cache{std::nullopt};
    {
      edge_t num_input_edges{};
      edge_t num_edges{};  // after removing self-loops and multi-edges

      // 2-1. create an edge list

      if (cugraph::test::g_perf) {
        RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
        comm.barrier();
        hr_timer.start("MG Construct edge list");
      }

      std::vector<rmm::device_uvector<vertex_t>> src_chunks{};
      std::vector<rmm::device_uvector<vertex_t>> dst_chunks{};
      std::tie(src_chunks, dst_chunks, std::ignore, std::ignore, std::ignore) =
        input_usecase.template construct_edgelist<vertex_t, weight_t>(
          *handle_, test_weighted, store_transposed, multi_gpu, shuffle);
      ASSERT_TRUE(input_usecase.undirected());

      if (cugraph::test::g_perf) {
        RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
        comm.barrier();
        hr_timer.stop();
        hr_timer.display_and_clear(std::cout);
      }

      num_input_edges = 0;
      for (size_t i = 0; i < src_chunks.size(); ++i) {
        num_input_edges += static_cast<edge_t>(src_chunks[i].size());
      }
      num_input_edges = cugraph::host_scalar_allreduce(
        comm, num_input_edges, raft::comms::op_t::SUM, handle_->get_stream());

      // 2-2. create an MG graph

      if (cugraph::test::g_perf) {
        RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
        comm.barrier();
        hr_timer.start("MG Construct graph (Kernel 1)");
      }

      for (size_t i = 0; i < src_chunks.size(); ++i) {
        std::tie(src_chunks[i],
                 dst_chunks[i],
                 std::ignore,
                 std::ignore,
                 std::ignore,
                 std::ignore,
                 std::ignore) =
          cugraph::remove_self_loops<vertex_t, edge_t, weight_t, edge_type_t, edge_time_t>(
            *handle_,
            std::move(src_chunks[i]),
            std::move(dst_chunks[i]),
            std::nullopt,
            std::nullopt,
            std::nullopt,
            std::nullopt,
            std::nullopt);

        std::vector<cugraph::arithmetic_device_uvector_t> dummy_edge_property_chunk{};

        std::tie(src_chunks[i], dst_chunks[i], dummy_edge_property_chunk, std::ignore) =
          cugraph::shuffle_ext_edges(*handle_,
                                     std::move(src_chunks[i]),
                                     std::move(dst_chunks[i]),
                                     std::move(dummy_edge_property_chunk),
                                     store_transposed);
      }

      std::tie(
        src_chunks, dst_chunks, std::ignore, std::ignore, std::ignore, std::ignore, std::ignore) =
        cugraph::remove_multi_edges<vertex_t, edge_t, weight_t, edge_type_t, edge_time_t>(
          *handle_,
          std::move(src_chunks),
          std::move(dst_chunks),
          std::nullopt,
          std::nullopt,
          std::nullopt,
          std::nullopt,
          std::nullopt,
          true /* keep_min_value_edge */);

      num_edges = 0;
      for (size_t i = 0; i < src_chunks.size(); ++i) {
        num_edges += static_cast<edge_t>(src_chunks[i].size());
      }
      num_edges = cugraph::host_scalar_allreduce(
        comm, num_edges, raft::comms::op_t::SUM, handle_->get_stream());

      cugraph::graph_t<vertex_t, edge_t, store_transposed, multi_gpu> mg_graph(*handle_);
      std::optional<rmm::device_uvector<vertex_t>> tmp_map{};
      std::tie(mg_graph, std::ignore, std::ignore, std::ignore, std::ignore, std::ignore, tmp_map) =
        cugraph::create_graph_from_edgelist<vertex_t,
                                            edge_t,
                                            weight_t,
                                            edge_type_t,
                                            edge_time_t,
                                            store_transposed,
                                            multi_gpu>(
          *handle_,
          std::nullopt,
          std::move(src_chunks),
          std::move(dst_chunks),
          std::nullopt,
          std::nullopt,
          std::nullopt,
          std::nullopt,
          std::nullopt,
          cugraph::graph_properties_t{true /* symmetric */, false /* multi-graph */},
          renumber);
      mg_renumber_map = std::move(*tmp_map);
      {
        auto mg_graph_view = mg_graph.view();
        auto offsets       = mg_graph_view.vertex_partition_range_offsets();
        std::copy(offsets.begin(), offsets.end(), vertex_partition_range_offsets.begin());
        local_vertex_partition_range_first = mg_graph_view.local_vertex_partition_range_first();
        local_vertex_partition_range_last  = mg_graph_view.local_vertex_partition_range_last();
      }

      // 2-3. Forest pruning

      {
        auto mg_graph_view = mg_graph.view();

        components.resize(mg_graph_view.local_vertex_partition_range_size(), handle_->get_stream());
        cugraph::weakly_connected_components(
          *handle_, mg_graph_view, components.data(), components.size());
        parents = find_trees_from_2cores(*handle_, mg_graph_view, invalid_vertex);
      }

      std::optional<rmm::device_uvector<vertex_t>> tmp_components{std::nullopt};
      if (bfs_usecase
            .use_large_buffer) {  // temporarily store components in host buffer to free up HBM
                                  // before extracting sub-graphs (which uses a lot of HBM)
        tmp_components = cugraph::large_buffer_manager::allocate_memory_buffer<vertex_t>(
          components.size(), handle_->get_stream());
        thrust::copy(handle_->get_thrust_policy(),
                     components.begin(),
                     components.end(),
                     tmp_components->begin());
        components.resize(0, handle_->get_stream());
        components.shrink_to_fit(handle_->get_stream());
      }

      std::tie(mg_pruned_graph,
               std::ignore,
               mg_pruned_graph_renumber_map,
               mg_graph_to_pruned_graph_map,
               mg_pruned_graph_to_graph_map,
               mg_isolated_trees,
               std::ignore,
               mg_isolated_trees_renumber_map,
               mg_graph_to_isolated_trees_map,
               mg_isolated_trees_to_graph_map) =
        extract_forest_pruned_graph_and_isolated_trees<vertex_t,
                                                       edge_t,
                                                       weight_t,
                                                       edge_type_t,
                                                       edge_time_t,
                                                       store_transposed,
                                                       multi_gpu>(
          *handle_,
          std::move(mg_graph),
          std::nullopt,
          raft::device_span<vertex_t const>(mg_renumber_map.data(), mg_renumber_map.size()),
          raft::device_span<vertex_t const>(parents.data(), parents.size()),
          invalid_vertex,
          bfs_usecase.use_large_buffer ? std::make_optional(cugraph::large_buffer_type_t::MEMORY)
                                       : std::nullopt);

      if (bfs_usecase.use_large_buffer) {
        components.resize(tmp_components->size(), handle_->get_stream());
        thrust::copy(handle_->get_thrust_policy(),
                     tmp_components->begin(),
                     tmp_components->end(),
                     components.begin());
        tmp_components = std::nullopt;
      }

      unrenumbered_parents.resize(parents.size(), handle_->get_stream());
      thrust::copy(
        handle_->get_thrust_policy(), parents.begin(), parents.end(), unrenumbered_parents.begin());
      cugraph::unrenumber_int_vertices<vertex_t, multi_gpu>(
        *handle_,
        unrenumbered_parents.data(),
        unrenumbered_parents.size(),
        mg_renumber_map.data(),
        raft::host_span<vertex_t const>(vertex_partition_range_offsets.data() + 1,
                                        vertex_partition_range_offsets.size() - 1));

      if (bfs_usecase.use_pruned_graph_unrenumber_cache) {
        mg_pruned_graph_pred_unrenumber_cache = cugraph::test::build_nbr_unrenumber_cache(
          *handle_,
          mg_pruned_graph.view(),
          raft::device_span<vertex_t const>(mg_pruned_graph_renumber_map.data(),
                                            mg_pruned_graph_renumber_map.size()),
          invalid_vertex,
          bfs_usecase.use_large_buffer ? std::make_optional(cugraph::large_buffer_type_t::MEMORY)
                                       : std::nullopt);
      }

      if (cugraph::test::g_perf) {
        RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
        comm.barrier();
        hr_timer.stop();
        hr_timer.display_and_clear(std::cout);
      }

      std::cout << "num_input_edges=" << num_input_edges
                << " V=" << vertex_partition_range_offsets.back() << " E=" << num_edges
                << " undirected E=" << (num_edges / 2) << std::endl;
    }
    auto mg_pruned_graph_view   = mg_pruned_graph.view();
    auto mg_isolated_trees_view = mg_isolated_trees.view();
    std::cout << "mg_pruned_graph V=" << mg_pruned_graph_view.number_of_vertices()
              << " E=" << mg_pruned_graph_view.compute_number_of_edges(*handle_)
              << " mg_isolated_trees_view V=" << mg_isolated_trees_view.number_of_vertices()
              << " E=" << mg_isolated_trees_view.compute_number_of_edges(*handle_) << std::endl;

    // 3. randomly select starting vertices

    std::vector<vertex_t> starting_vertices{};
    {
      ASSERT_TRUE(vertex_partition_range_offsets.back() > 0)
        << "Invalid input graph, the input graph should have at least one vertex";
      rmm::device_uvector<vertex_t> d_starting_vertices(
        num_warmup_starting_vertices + num_timed_starting_vertices, handle_->get_stream());
      if (comm_rank == 0) {
        cugraph::detail::uniform_random_fill(handle_->get_stream(),
                                             d_starting_vertices.data(),
                                             d_starting_vertices.size(),
                                             vertex_partition_range_offsets[0],
                                             vertex_partition_range_offsets.back(),
                                             rng_state);
        raft::print_device_vector(
          "d_starting_vertices", d_starting_vertices.data(), d_starting_vertices.size(), std::cout);
      }
      cugraph::device_bcast(comm,
                            d_starting_vertices.data(),
                            d_starting_vertices.data(),
                            d_starting_vertices.size(),
                            int{0},
                            handle_->get_stream());
      starting_vertices = cugraph::test::to_host(*handle_, d_starting_vertices);
    }

    // 4. run MG BFS

    rmm::device_uvector<vertex_t> d_mg_distances(
      mg_renumber_map.size(),
      handle_->get_stream());  // Graph500 doesn't require computing distances (so we can update
                               // this outside the timed region)
    rmm::device_uvector<vertex_t> d_mg_unrenumbered_predecessors(mg_renumber_map.size(),
                                                                 handle_->get_stream());

    double total_elapsed{0.0};
    double tteps_sum{0.0};
    double one_over_tteps_sum{0.0};  // to compute harmonic mean
    for (size_t i = 0; i < (num_warmup_starting_vertices + num_timed_starting_vertices); ++i) {
      double elapsed{0.0};

      thrust::fill(handle_->get_thrust_policy(),
                   d_mg_distances.begin(),
                   d_mg_distances.end(),
                   invalid_distance);

      if (cugraph::test::g_perf) {
        RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
        comm.barrier();
        hr_timer.start("MG BFS (Kernel 2)");
      }

      auto starting_vertex = starting_vertices[i];
      auto starting_vertex_vertex_partition_id =
        static_cast<int>(std::distance(vertex_partition_range_offsets.begin() + 1,
                                       std::upper_bound(vertex_partition_range_offsets.begin() + 1,
                                                        vertex_partition_range_offsets.end(),
                                                        starting_vertex)));

      vertex_t unrenumbered_starting_vertex{};
      vertex_t starting_vertex_parent{starting_vertex};
      vertex_t starting_vertex_component{};
      if (starting_vertex_vertex_partition_id == vertex_partition_id) {
        unrenumbered_starting_vertex = mg_renumber_map.element(
          starting_vertex - local_vertex_partition_range_first, handle_->get_stream());
        starting_vertex_parent = parents.element(
          starting_vertex - local_vertex_partition_range_first, handle_->get_stream());
        starting_vertex_component = components.element(
          starting_vertex - local_vertex_partition_range_first, handle_->get_stream());
      }
      thrust::tie(unrenumbered_starting_vertex, starting_vertex_parent, starting_vertex_component) =
        cugraph::host_scalar_bcast(
          comm,
          thrust::make_tuple(
            unrenumbered_starting_vertex, starting_vertex_parent, starting_vertex_component),
          cugraph::partition_manager::compute_global_comm_rank_from_vertex_partition_id(
            major_comm_size, minor_comm_size, starting_vertex_vertex_partition_id),
          handle_->get_stream());
      bool reachable_from_2cores{starting_vertex_parent != invalid_vertex};
      bool in_2cores{starting_vertex == starting_vertex_parent};

      if (reachable_from_2cores) {
        thrust::transform(handle_->get_thrust_policy(),
                          unrenumbered_parents.begin(),
                          unrenumbered_parents.end(),
                          components.begin(),
                          d_mg_unrenumbered_predecessors.begin(),
                          [starting_vertex_component, invalid_vertex] __device__(auto p, auto c) {
                            return (c == starting_vertex_component)
                                     ? p /* for the vertices in 2-cores (or the vertices in the
                                            path from the starting vertex to the first reachable
                                            2-core vertex), this will be over-written */
                                     : invalid_vertex;
                          });
      } else {
        thrust::fill(handle_->get_thrust_policy(),
                     d_mg_unrenumbered_predecessors.begin(),
                     d_mg_unrenumbered_predecessors.end(),
                     invalid_vertex);
      }

      vertex_t subgraph_starting_vertex{starting_vertex};
      int subgraph_starting_vertex_vertex_partition_id{starting_vertex_vertex_partition_id};
      vertex_t subgraph_starting_vertex_distance{0};
      vertex_t unrenumbered_subgraph_starting_vertex_parent{};
      if (reachable_from_2cores && !in_2cores) {  // find the path from starting_vertex to a 2-core
        if (starting_vertex_vertex_partition_id == vertex_partition_id) {
          d_mg_unrenumbered_predecessors.set_element_async(
            starting_vertex - local_vertex_partition_range_first,
            unrenumbered_starting_vertex,  // Graph 500 requires the predecessor of a starting
                                           // vertex to be itself
            handle_->get_stream());
          d_mg_distances.set_element_to_zero_async(
            starting_vertex - local_vertex_partition_range_first, handle_->get_stream());
          handle_->sync_stream();
        }

        auto unrenumbered_v = unrenumbered_starting_vertex;
        auto n = starting_vertex_parent;  // reverse the parent child relationship till we reach
                                          // a 2-core
        while (true) {
          assert(v != n);  // in this case, v is already a 2-core vertex
          ++subgraph_starting_vertex_distance;
          auto n_vertex_partition_id = static_cast<int>(
            std::distance(vertex_partition_range_offsets.begin() + 1,
                          std::upper_bound(vertex_partition_range_offsets.begin() + 1,
                                           vertex_partition_range_offsets.end(),
                                           n)));
          vertex_t unrenumbered_n{};
          vertex_t nn{};
          if (n_vertex_partition_id == vertex_partition_id) {
            unrenumbered_n = mg_renumber_map.element(n - local_vertex_partition_range_first,
                                                     handle_->get_stream());
            nn = parents.element(n - local_vertex_partition_range_first, handle_->get_stream());
          }
          thrust::tie(unrenumbered_n, nn) = cugraph::host_scalar_bcast(
            comm,
            thrust::make_tuple(unrenumbered_n, nn),
            cugraph::partition_manager::compute_global_comm_rank_from_vertex_partition_id(
              major_comm_size, minor_comm_size, n_vertex_partition_id),
            handle_->get_stream());

          if (n == nn) {  // reached a 2-core
            subgraph_starting_vertex                     = n;
            subgraph_starting_vertex_vertex_partition_id = n_vertex_partition_id;
            unrenumbered_subgraph_starting_vertex_parent = unrenumbered_v;
            break;
          }

          if (n_vertex_partition_id == vertex_partition_id) {
            d_mg_unrenumbered_predecessors.set_element_async(
              n - local_vertex_partition_range_first, unrenumbered_v, handle_->get_stream());
            d_mg_distances.set_element_async(n - local_vertex_partition_range_first,
                                             subgraph_starting_vertex_distance,
                                             handle_->get_stream());
            handle_->sync_stream();
          }

          unrenumbered_v = unrenumbered_n;
          n              = nn;
        }
      }

      std::optional<rmm::device_scalar<vertex_t>> d_bfs_starting_vertex{std::nullopt};
      if (subgraph_starting_vertex_vertex_partition_id == vertex_partition_id) {
        auto bfs_starting_vertex =
          reachable_from_2cores ? mg_pruned_graph_view.local_vertex_partition_range_first() +
                                    mg_graph_to_pruned_graph_map.element(
                                      subgraph_starting_vertex - local_vertex_partition_range_first,
                                      handle_->get_stream())
                                : mg_isolated_trees_view.local_vertex_partition_range_first() +
                                    mg_graph_to_isolated_trees_map.element(
                                      subgraph_starting_vertex - local_vertex_partition_range_first,
                                      handle_->get_stream());
        d_bfs_starting_vertex =
          rmm::device_scalar<vertex_t>(bfs_starting_vertex, handle_->get_stream());
      }

      rmm::device_uvector<vertex_t> d_mg_bfs_predecessors(
        reachable_from_2cores ? mg_pruned_graph_view.local_vertex_partition_range_size()
                              : mg_isolated_trees_view.local_vertex_partition_range_size(),
        handle_->get_stream());
      rmm::device_uvector<vertex_t> d_mg_bfs_distances(d_mg_bfs_predecessors.size(),
                                                       handle_->get_stream());

      cugraph::bfs(*handle_,
                   reachable_from_2cores ? mg_pruned_graph_view : mg_isolated_trees_view,
                   d_mg_bfs_distances.data(),
                   d_mg_bfs_predecessors.data(),
                   d_bfs_starting_vertex ? d_bfs_starting_vertex->data()
                                         : static_cast<vertex_t const*>(nullptr),
                   d_bfs_starting_vertex ? size_t{1} : size_t{0},
                   true /* direction_optimizing */,
                   std::numeric_limits<vertex_t>::max() /* depth limit */);

      if (reachable_from_2cores && mg_pruned_graph_pred_unrenumber_cache) {
        mg_pruned_graph_pred_unrenumber_cache->unrenumber(
          *handle_,
          raft::device_span<vertex_t>(d_mg_bfs_predecessors.data(), d_mg_bfs_predecessors.size()));
      } else {
        cugraph::unrenumber_int_vertices<vertex_t, multi_gpu>(
          *handle_,
          d_mg_bfs_predecessors.data(),
          d_mg_bfs_predecessors.size(),
          reachable_from_2cores ? mg_pruned_graph_renumber_map.data()
                                : mg_isolated_trees_renumber_map.data(),
          reachable_from_2cores ? mg_pruned_graph_view.vertex_partition_range_lasts()
                                : mg_isolated_trees_view.vertex_partition_range_lasts());
      }

      thrust::scatter(handle_->get_thrust_policy(),
                      d_mg_bfs_predecessors.begin(),
                      d_mg_bfs_predecessors.end(),
                      reachable_from_2cores ? mg_pruned_graph_to_graph_map.begin()
                                            : mg_isolated_trees_to_graph_map.begin(),
                      d_mg_unrenumbered_predecessors.begin());

      {  // update the starting vertex's parent
        if (subgraph_starting_vertex_vertex_partition_id ==
            vertex_partition_id) {  // cugraph::bfs sets the predecessor of the starting vertex to
                                    // invalid_vertex
          if (subgraph_starting_vertex_distance > vertex_t{0}) {
            d_mg_unrenumbered_predecessors.set_element_async(
              subgraph_starting_vertex - local_vertex_partition_range_first,
              unrenumbered_subgraph_starting_vertex_parent,
              handle_->get_stream());
          } else {
            assert(starting_vertex == subgraph_starting_vertex);
            d_mg_unrenumbered_predecessors.set_element_async(
              starting_vertex - local_vertex_partition_range_first,
              unrenumbered_starting_vertex,
              handle_->get_stream());
          }
          handle_->sync_stream();
        }
      }

      if (cugraph::test::g_perf) {
        RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
        comm.barrier();
        elapsed = hr_timer.stop();
        if (i >= num_warmup_starting_vertices) { total_elapsed += elapsed; }
        hr_timer.display_and_clear(std::cout);
      }

      // update d_mg_distances (for validation, Graph 500 doesn't require computing distances)

      if (reachable_from_2cores) {
        if (subgraph_starting_vertex_distance > vertex_t{0}) {
          thrust::transform(handle_->get_thrust_policy(),
                            d_mg_bfs_distances.begin(),
                            d_mg_bfs_distances.end(),
                            d_mg_bfs_distances.begin(),
                            [delta = subgraph_starting_vertex_distance] __device__(auto d) {
                              if (d != invalid_distance) {
                                return d + delta;
                              } else {
                                return invalid_distance;
                              }
                            });
        }
        thrust::scatter(handle_->get_thrust_policy(),
                        d_mg_bfs_distances.begin(),
                        d_mg_bfs_distances.end(),
                        mg_pruned_graph_to_graph_map.begin(),
                        d_mg_distances.begin());
        rmm::device_uvector<vertex_t> remaining_vertices(
          mg_renumber_map.size(),
          handle_->get_stream());  // unvisited vertices in the forest (but in the same connected
                                   // component with the starting vertex)
        remaining_vertices.resize(
          thrust::distance(
            remaining_vertices.begin(),
            thrust::copy_if(handle_->get_thrust_policy(),
                            thrust::make_counting_iterator(local_vertex_partition_range_first),
                            thrust::make_counting_iterator(local_vertex_partition_range_last),
                            thrust::make_zip_iterator(components.begin(), d_mg_distances.begin()),
                            remaining_vertices.begin(),
                            [starting_vertex_component, invalid_distance] __device__(auto pair) {
                              return thrust::get<0>(pair) == starting_vertex_component &&
                                     thrust::get<1>(pair) == invalid_distance;
                            })),
          handle_->get_stream());
        while (true) {
          auto tot_remaining_vertex_count = cugraph::host_scalar_allreduce(
            comm, remaining_vertices.size(), raft::comms::op_t::SUM, handle_->get_stream());
          if (tot_remaining_vertex_count == 0) { break; }
          rmm::device_uvector<vertex_t> preds(remaining_vertices.size(), handle_->get_stream());
          auto gather_offset_first = thrust::make_transform_iterator(
            remaining_vertices.begin(),
            cugraph::detail::shift_left_t<vertex_t>{local_vertex_partition_range_first});
          thrust::gather(handle_->get_thrust_policy(),
                         gather_offset_first,
                         gather_offset_first + remaining_vertices.size(),
                         parents.begin(),
                         preds.begin());
          auto dists = cugraph::collect_values_for_int_vertices(
            *handle_,
            preds.begin(),
            preds.end(),
            d_mg_distances.begin(),
            raft::host_span<vertex_t const>(vertex_partition_range_offsets.data() + 1,
                                            vertex_partition_range_offsets.size() - 1),
            local_vertex_partition_range_first);
          auto pair_first = thrust::make_zip_iterator(remaining_vertices.begin(), dists.begin());
          auto remaining_last = thrust::partition(
            handle_->get_thrust_policy(),
            pair_first,
            pair_first + remaining_vertices.size(),
            [] __device__(auto pair) { return thrust::get<1>(pair) == invalid_distance; });
          auto new_size = thrust::distance(pair_first, remaining_last);
          auto dist_first =
            thrust::make_transform_iterator(
              dists.begin(),
              cuda::proclaim_return_type<vertex_t>([invalid_distance] __device__(auto d) {
                return (d != invalid_distance) ? static_cast<vertex_t>(d + 1) : invalid_distance;
              })) +
            new_size;
          auto scatter_offset_first =
            thrust::make_transform_iterator(
              remaining_vertices.begin(),
              cugraph::detail::shift_left_t<vertex_t>{local_vertex_partition_range_first}) +
            new_size;
          thrust::scatter(handle_->get_thrust_policy(),
                          dist_first,
                          dist_first + (remaining_vertices.size() - new_size),
                          scatter_offset_first,
                          d_mg_distances.begin());
          remaining_vertices.resize(new_size, handle_->get_stream());
        }
      } else {
        assert(subgraph_starting_vertex_distance == vertex_t{0});
        thrust::scatter(handle_->get_thrust_policy(),
                        d_mg_bfs_distances.begin(),
                        d_mg_bfs_distances.end(),
                        mg_isolated_trees_to_graph_map.begin(),
                        d_mg_distances.begin());
      }

      /* compute the number of visisted edges */

      {
        edge_t tot_edge_count{};
        if (reachable_from_2cores) {
          rmm::device_uvector<bool> visited(
            mg_pruned_graph_view.local_vertex_partition_range_size(), handle_->get_stream());
          thrust::transform(handle_->get_thrust_policy(),
                            d_mg_bfs_distances.begin(),
                            d_mg_bfs_distances.end(),
                            visited.begin(),
                            cuda::proclaim_return_type<bool>([invalid_distance] __device__(auto d) {
                              return d != invalid_distance;
                            }));
          cugraph::edge_src_property_t<vertex_t, bool> edge_src_visited(*handle_,
                                                                        mg_pruned_graph_view);
          cugraph::update_edge_src_property(
            *handle_, mg_pruned_graph_view, visited.begin(), edge_src_visited.mutable_view());
          tot_edge_count =
            cugraph::count_if_e(
              *handle_,
              mg_pruned_graph_view,
              edge_src_visited.view(),
              cugraph::edge_dst_dummy_property_t{}.view(),
              cugraph::edge_dummy_property_t{}.view(),
              [] __device__(auto, auto, auto src_visited, auto, auto) { return src_visited; }) /
            edge_t{2};
          auto forest_edge_count = thrust::count_if(
            handle_->get_thrust_policy(),
            thrust::make_zip_iterator(d_mg_distances.begin(), mg_graph_to_pruned_graph_map.begin()),
            thrust::make_zip_iterator(d_mg_distances.end(), mg_graph_to_pruned_graph_map.end()),
            [invalid_distance, invalid_vertex] __device__(auto pair) {
              return (thrust::get<0>(pair) != invalid_distance /* reachable */) &&
                     (thrust::get<1>(pair) == invalid_vertex /* not in the pruned graph */);
            });  // # vertices reachable from 2-cores but not in 2-cores
          forest_edge_count = cugraph::host_scalar_allreduce(
            comm, forest_edge_count, raft::comms::op_t::SUM, handle_->get_stream());
          tot_edge_count += forest_edge_count;
        } else {
          auto num_visited = thrust::count_if(
            handle_->get_thrust_policy(),
            d_mg_distances.begin(),
            d_mg_distances.end(),
            [invalid_distance] __device__(auto d) { return d != invalid_distance; });
          auto tot_num_visited = cugraph::host_scalar_allreduce(
            comm, num_visited, raft::comms::op_t::SUM, handle_->get_stream());
          tot_edge_count = tot_num_visited - 1;  // # edges in a tree is # vertices - 1
        }
        auto tteps = (static_cast<double>(tot_edge_count) / 1e12) / elapsed;
        if (i >= num_warmup_starting_vertices) {
          tteps_sum += tteps;
          one_over_tteps_sum +=
            (tteps > 0.0) ? 1.0 / tteps : std::numeric_limits<double>::infinity();
        }
        std::cout << "# visited undirected edges=" << tot_edge_count
                  << " TTEPS=" << (static_cast<double>(tot_edge_count) / 1e12) / elapsed
                  << std::endl;
      }

      if (bfs_usecase.validate) {
        /* renumber for validation */

        rmm::device_uvector<vertex_t> d_mg_predecessors(d_mg_unrenumbered_predecessors.size(),
                                                        handle_->get_stream());
        thrust::copy(handle_->get_thrust_policy(),
                     d_mg_unrenumbered_predecessors.begin(),
                     d_mg_unrenumbered_predecessors.end(),
                     d_mg_predecessors.begin());
        cugraph::renumber_ext_vertices<vertex_t, multi_gpu>(*handle_,
                                                            d_mg_predecessors.data(),
                                                            d_mg_predecessors.size(),
                                                            mg_renumber_map.data(),
                                                            local_vertex_partition_range_first,
                                                            local_vertex_partition_range_last);

        /* check starting vertex's predecessor */

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.start("validate (starting vertex's predecessor)");
        }

        {
          size_t num_invalids{0};
          if (starting_vertex_vertex_partition_id == vertex_partition_id) {
            auto starting_vertex_predecessor = d_mg_predecessors.element(
              starting_vertex - local_vertex_partition_range_first, handle_->get_stream());
            if (starting_vertex_predecessor != starting_vertex) { ++num_invalids; }
          }
          num_invalids = cugraph::host_scalar_allreduce(
            comm, num_invalids, raft::comms::op_t::SUM, handle_->get_stream());
          ASSERT_EQ(num_invalids, 0) << "predecessor of a starting vertex should be itself";
        }

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.stop();
          hr_timer.display_and_clear(std::cout);
        }

        /* check for cycles (update predecessor to predecessor's predecessor till reaching the
         * starting vertex, if there exists a cycle, this won't finish) */

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.start("validate (cycle)");
        }

        {
          cugraph::kv_store_t<vertex_t, vertex_t, true /* use_binary_search */> kv_store(
            thrust::make_counting_iterator(local_vertex_partition_range_first),
            thrust::make_counting_iterator(local_vertex_partition_range_last),
            d_mg_predecessors.begin(),
            invalid_vertex,
            true /* key_sorted */,
            handle_->get_stream());
          auto kv_store_view = kv_store.view();
          rmm::device_uvector<vertex_t> d_vertex_partition_range_offsets(
            vertex_partition_range_offsets.size(), handle_->get_stream());
          raft::update_device(d_vertex_partition_range_offsets.data(),
                              vertex_partition_range_offsets.data(),
                              vertex_partition_range_offsets.size(),
                              handle_->get_stream());

          rmm::device_uvector<vertex_t> ancestors(d_mg_predecessors.size(), handle_->get_stream());
          ancestors.resize(
            thrust::distance(
              ancestors.begin(),
              thrust::copy_if(handle_->get_thrust_policy(),
                              d_mg_predecessors.begin(),
                              d_mg_predecessors.end(),
                              ancestors.begin(),
                              cuda::proclaim_return_type<bool>(
                                [starting_vertex, invalid_vertex] __device__(auto pred) {
                                  return (pred != starting_vertex) && (pred != invalid_vertex);
                                }))),
            handle_->get_stream());

          size_t level{0};
          auto aggregate_size = cugraph::host_scalar_allreduce(
            comm, ancestors.size(), raft::comms::op_t::SUM, handle_->get_stream());
          while (aggregate_size > size_t{0}) {
            ASSERT_TRUE(level < vertex_partition_range_offsets.back() - 1)
              << "BFS predecessor tree has a cycle.";
            auto num_invalids = thrust::count(
              handle_->get_thrust_policy(), ancestors.begin(), ancestors.end(), invalid_vertex);
            num_invalids = cugraph::host_scalar_allreduce(
              comm, num_invalids, raft::comms::op_t::SUM, handle_->get_stream());
            ASSERT_EQ(num_invalids, 0) << "Invalid BFS predecessor tree, failed to backtrace from "
                                          "a reachable vertex to the starting vertex";
            ancestors = cugraph::collect_values_for_keys(
              comm,
              kv_store_view,
              ancestors.begin(),
              ancestors.end(),
              cugraph::detail::compute_gpu_id_from_int_vertex_t<vertex_t>{
                raft::device_span<vertex_t const>(d_vertex_partition_range_offsets.data() + 1,
                                                  d_vertex_partition_range_offsets.size() - 1),
                major_comm_size,
                minor_comm_size},
              handle_->get_stream());
            ancestors.resize(
              thrust::distance(
                ancestors.begin(),
                thrust::remove_if(handle_->get_thrust_policy(),
                                  ancestors.begin(),
                                  ancestors.end(),
                                  cugraph::detail::is_equal_t<vertex_t>{starting_vertex})),
              handle_->get_stream());
            aggregate_size = cugraph::host_scalar_allreduce(
              comm, ancestors.size(), raft::comms::op_t::SUM, handle_->get_stream());
            ++level;
          }
        }

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.stop();
          hr_timer.display_and_clear(std::cout);
        }

        /* check that distance(v) = distance(predecssor(v)) + 1 */

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.start("validate (predecessor tree distances)");
        }

        {
          rmm::device_uvector<vertex_t> tree_srcs(mg_renumber_map.size(), handle_->get_stream());
          rmm::device_uvector<vertex_t> tree_dsts(tree_srcs.size(), handle_->get_stream());
          auto input_pair_first = thrust::make_zip_iterator(
            d_mg_predecessors.begin(),
            thrust::make_counting_iterator(local_vertex_partition_range_first));
          auto output_pair_first = thrust::make_zip_iterator(tree_srcs.begin(), tree_dsts.begin());
          tree_srcs.resize(
            cuda::std::distance(
              output_pair_first,
              thrust::copy_if(handle_->get_thrust_policy(),
                              input_pair_first,
                              input_pair_first + mg_renumber_map.size(),
                              output_pair_first,
                              cuda::proclaim_return_type<bool>(
                                [starting_vertex, invalid_vertex] __device__(auto pair) {
                                  auto pred = thrust::get<0>(pair);
                                  auto v    = thrust::get<1>(pair);
                                  return (pred != invalid_vertex) && (v != starting_vertex);
                                }))),
            handle_->get_stream());
          tree_dsts.resize(tree_srcs.size(), handle_->get_stream());

          auto tree_src_dists = cugraph::collect_values_for_int_vertices(
            *handle_,
            tree_srcs.begin(),
            tree_srcs.end(),
            d_mg_distances.begin(),
            raft::host_span<vertex_t const>(vertex_partition_range_offsets.data() + 1,
                                            vertex_partition_range_offsets.size() - 1),
            local_vertex_partition_range_first);

          rmm::device_uvector<vertex_t> tree_dst_dists(tree_dsts.size(), handle_->get_stream());
          thrust::transform(handle_->get_thrust_policy(),
                            tree_dsts.begin(),
                            tree_dsts.end(),
                            tree_dst_dists.begin(),
                            cuda::proclaim_return_type<vertex_t>(
                              [mg_distances = raft::device_span<vertex_t const>(
                                 d_mg_distances.data(), d_mg_distances.size()),
                               v_first = local_vertex_partition_range_first] __device__(auto v) {
                                return mg_distances[v - v_first];
                              }));

          ASSERT_EQ(tree_src_dists.size(), tree_dst_dists.size());
          auto dist_pair_first =
            thrust::make_zip_iterator(tree_src_dists.begin(), tree_dst_dists.begin());
          auto num_invalids =
            thrust::count_if(handle_->get_thrust_policy(),
                             dist_pair_first,
                             dist_pair_first + tree_src_dists.size(),
                             cuda::proclaim_return_type<bool>([] __device__(auto pair) {
                               auto src_dist = thrust::get<0>(pair);
                               auto dst_dist = thrust::get<1>(pair);
                               return (src_dist + 1) != dst_dist;
                             }));
          num_invalids = cugraph::host_scalar_allreduce(
            comm, num_invalids, raft::comms::op_t::SUM, handle_->get_stream());

          ASSERT_EQ(num_invalids, 0) << " source and destination vertices in the BFS predecessor "
                                        "tree are not one hop away.";
        }

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.stop();
          hr_timer.display_and_clear(std::cout);
        }

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.start("validate (graph distances)");
        }

        /* for every edge e = (u, v), abs(dist(u) - dist(v)) <= 1 or dist(u) == dist(v) ==
         * invalid_distance */

        {
          auto const& mg_subgraph_view =
            reachable_from_2cores ? mg_pruned_graph_view : mg_isolated_trees_view;
          cugraph::edge_src_property_t<vertex_t, bool> edge_src_flags(*handle_, mg_subgraph_view);
          cugraph::edge_dst_property_t<vertex_t, bool> edge_dst_flags(*handle_, mg_subgraph_view);

          // first, validate the edges in the subgraph

          auto max_distance = thrust::transform_reduce(
            handle_->get_thrust_policy(),
            d_mg_distances.begin(),
            d_mg_distances.end(),
            cuda::proclaim_return_type<vertex_t>([invalid_distance] __device__(auto d) {
              return d == invalid_distance ? vertex_t{0} : d;
            }),
            vertex_t{0},
            thrust::maximum<vertex_t>{});
          max_distance = cugraph::host_scalar_allreduce(
            comm, max_distance, raft::comms::op_t::MAX, handle_->get_stream());

          auto pair_first = thrust::make_zip_iterator(reachable_from_2cores
                                                        ? mg_graph_to_pruned_graph_map.begin()
                                                        : mg_graph_to_isolated_trees_map.begin(),
                                                      d_mg_distances.begin());
          for (vertex_t level = 0; level <= max_distance;
               ++level) {  // validate in multple round to cut peak memory usage (to store
                           // source|destination vertex properties using 1 bit per vertex)
            rmm::device_uvector<vertex_t> subgraph_level_v_offsets(
              mg_subgraph_view.local_vertex_partition_range_size(),
              handle_->get_stream());  // vertices with d_mg_distances[] == level
            rmm::device_uvector<vertex_t> subgraph_adjacent_level_v_offsets(
              mg_subgraph_view.local_vertex_partition_range_size(),
              handle_->get_stream());  // abs(vertices with d_mg_distances[] - level) <= 1
            subgraph_level_v_offsets.resize(
              thrust::distance(
                subgraph_level_v_offsets.begin(),
                thrust::copy_if(handle_->get_thrust_policy(),
                                reachable_from_2cores ? mg_graph_to_pruned_graph_map.begin()
                                                      : mg_graph_to_isolated_trees_map.begin(),
                                reachable_from_2cores ? mg_graph_to_pruned_graph_map.end()
                                                      : mg_graph_to_isolated_trees_map.end(),
                                pair_first,
                                subgraph_level_v_offsets.begin(),
                                [level, invalid_distance] __device__(auto pair) {
                                  auto d = thrust::get<1>(pair);
                                  return (thrust::get<0>(pair) !=
                                          invalid_vertex /* in the subgraph */) &&
                                         (d == level);
                                })),
              handle_->get_stream());
            subgraph_level_v_offsets.shrink_to_fit(handle_->get_stream());
            subgraph_adjacent_level_v_offsets.resize(
              thrust::distance(
                subgraph_adjacent_level_v_offsets.begin(),
                thrust::copy_if(
                  handle_->get_thrust_policy(),
                  reachable_from_2cores ? mg_graph_to_pruned_graph_map.begin()
                                        : mg_graph_to_isolated_trees_map.begin(),
                  reachable_from_2cores ? mg_graph_to_pruned_graph_map.end()
                                        : mg_graph_to_isolated_trees_map.end(),
                  pair_first,
                  subgraph_adjacent_level_v_offsets.begin(),
                  cuda::proclaim_return_type<bool>([level, invalid_distance] __device__(auto pair) {
                    auto d = thrust::get<1>(pair);
                    return (thrust::get<0>(pair) != invalid_vertex /* in the subgraph */) &&
                           (((d >= level) ? (d - level) : (level - d)) <= 1);
                  }))),
              handle_->get_stream());
            subgraph_adjacent_level_v_offsets.shrink_to_fit(handle_->get_stream());

            auto subgraph_level_vs = std::move(subgraph_level_v_offsets);
            thrust::transform(
              handle_->get_thrust_policy(),
              subgraph_level_vs.begin(),
              subgraph_level_vs.end(),
              subgraph_level_vs.begin(),
              cuda::proclaim_return_type<vertex_t>(
                [v_first = mg_subgraph_view.local_vertex_partition_range_first()] __device__(
                  auto v_offset) { return v_first + v_offset; }));
            thrust::sort(
              handle_->get_thrust_policy(), subgraph_level_vs.begin(), subgraph_level_vs.end());
            cugraph::fill_edge_src_property(
              *handle_, mg_subgraph_view, edge_src_flags.mutable_view(), false);
            cugraph::fill_edge_src_property(*handle_,
                                            mg_subgraph_view,
                                            subgraph_level_vs.begin(),
                                            subgraph_level_vs.end(),
                                            edge_src_flags.mutable_view(),
                                            true);  // true if the distance is level
            auto subgraph_adjacent_level_vs = std::move(subgraph_adjacent_level_v_offsets);
            thrust::transform(
              handle_->get_thrust_policy(),
              subgraph_adjacent_level_vs.begin(),
              subgraph_adjacent_level_vs.end(),
              subgraph_adjacent_level_vs.begin(),
              cuda::proclaim_return_type<vertex_t>(
                [v_first = mg_subgraph_view.local_vertex_partition_range_first()] __device__(
                  auto v_offset) { return v_first + v_offset; }));
            thrust::sort(handle_->get_thrust_policy(),
                         subgraph_adjacent_level_vs.begin(),
                         subgraph_adjacent_level_vs.end());
            cugraph::fill_edge_dst_property(
              *handle_, mg_subgraph_view, edge_dst_flags.mutable_view(), false);
            cugraph::fill_edge_dst_property(*handle_,
                                            mg_subgraph_view,
                                            subgraph_adjacent_level_vs.begin(),
                                            subgraph_adjacent_level_vs.end(),
                                            edge_dst_flags.mutable_view(),
                                            true);  // true if the abs(distance - level) <= 1
            auto num_invalids = cugraph::count_if_e(
              *handle_,
              mg_subgraph_view,
              edge_src_flags.view(),
              edge_dst_flags.view(),
              cugraph::edge_dummy_property_t{}.view(),
              cuda::proclaim_return_type<bool>(
                [level, invalid_distance] __device__(
                  auto src, auto dst, bool level_src, bool adjacent_level_dst, auto) {
                  return level_src && !adjacent_level_dst;
                }));
            ASSERT_EQ(num_invalids, 0)
              << "only one of the two connected vertices is reachable from the starting vertex or "
                 "the distances from the starting vertex differ by more than one.";
          }

          {
            rmm::device_uvector<vertex_t> unreachable_v_offsets(
              mg_subgraph_view.local_vertex_partition_range_size(), handle_->get_stream());
            unreachable_v_offsets.resize(
              thrust::distance(
                unreachable_v_offsets.begin(),
                thrust::copy_if(
                  handle_->get_thrust_policy(),
                  reachable_from_2cores ? mg_graph_to_pruned_graph_map.begin()
                                        : mg_graph_to_isolated_trees_map.begin(),
                  reachable_from_2cores ? mg_graph_to_pruned_graph_map.end()
                                        : mg_graph_to_isolated_trees_map.end(),
                  pair_first,
                  unreachable_v_offsets.begin(),
                  cuda::proclaim_return_type<bool>(
                    [invalid_vertex, invalid_distance] __device__(auto pair) {
                      return (thrust::get<0>(pair) != invalid_vertex /* in the subgraph */) &&
                             (thrust::get<1>(pair) == invalid_distance /* unreachable */);
                    }))),
              handle_->get_stream());
            auto unreachable_vs = std::move(unreachable_v_offsets);
            thrust::transform(
              handle_->get_thrust_policy(),
              unreachable_vs.begin(),
              unreachable_vs.end(),
              unreachable_vs.begin(),
              cuda::proclaim_return_type<vertex_t>(
                [v_first = mg_subgraph_view.local_vertex_partition_range_first()] __device__(
                  auto v_offset) { return v_first + v_offset; }));
            cugraph::fill_edge_src_property(
              *handle_, mg_subgraph_view, edge_src_flags.mutable_view(), false);
            cugraph::fill_edge_src_property(*handle_,
                                            mg_subgraph_view,
                                            unreachable_vs.begin(),
                                            unreachable_vs.end(),
                                            edge_src_flags.mutable_view(),
                                            true);  // true if the distance is invalid_distance
            cugraph::fill_edge_dst_property(
              *handle_, mg_subgraph_view, edge_dst_flags.mutable_view(), false);
            cugraph::fill_edge_dst_property(*handle_,
                                            mg_subgraph_view,
                                            unreachable_vs.begin(),
                                            unreachable_vs.end(),
                                            edge_dst_flags.mutable_view(),
                                            true);  // true if the distance is invalid_distance
            auto num_invalids = cugraph::count_if_e(
              *handle_,
              mg_subgraph_view,
              edge_src_flags.view(),
              edge_dst_flags.view(),
              cugraph::edge_dummy_property_t{}.view(),
              cuda::proclaim_return_type<bool>(
                [] __device__(
                  auto src, auto dst, bool src_unreachable, bool dst_unreachable, auto) {
                  return src_unreachable != dst_unreachable;
                }));
            ASSERT_EQ(num_invalids, 0)
              << "only one of the two connected vertices is reachable from the starting vertex.";
          }

          // second, validate the edges in the pruned forest (if reachble_from_2cores is true)

          if (reachable_from_2cores) {
            rmm::device_uvector<vertex_t> forest_edge_parents(mg_renumber_map.size(),
                                                              handle_->get_stream());
            rmm::device_uvector<vertex_t> forest_edge_vertices(forest_edge_parents.size(),
                                                               handle_->get_stream());
            auto input_first = thrust::make_zip_iterator(
              parents.begin(), thrust::make_counting_iterator(local_vertex_partition_range_first));
            auto output_first =
              thrust::make_zip_iterator(forest_edge_parents.begin(), forest_edge_vertices.begin());
            forest_edge_parents.resize(
              thrust::distance(
                output_first,
                thrust::copy_if(handle_->get_thrust_policy(),
                                input_first,
                                input_first + forest_edge_parents.size(),
                                output_first,
                                cuda::proclaim_return_type<bool>([] __device__(auto pair) {
                                  auto p = thrust::get<0>(pair);
                                  auto v = thrust::get<1>(pair);
                                  return (p != invalid_vertex /* reachable from 2-cores */) &&
                                         (p != v /* not in a 2-core */);
                                }))),
              handle_->get_stream());
            forest_edge_vertices.resize(forest_edge_parents.size(), handle_->get_stream());
            auto forest_edge_src_dists = cugraph::collect_values_for_int_vertices(
              *handle_,
              forest_edge_parents.begin(),
              forest_edge_parents.end(),
              d_mg_distances.begin(),
              raft::host_span<vertex_t const>(vertex_partition_range_offsets.data() + 1,
                                              vertex_partition_range_offsets.size() - 1),
              local_vertex_partition_range_first);
            auto forest_edge_dst_dists = cugraph::collect_values_for_int_vertices(
              *handle_,
              forest_edge_vertices.begin(),
              forest_edge_vertices.end(),
              d_mg_distances.begin(),
              raft::host_span<vertex_t const>(vertex_partition_range_offsets.data() + 1,
                                              vertex_partition_range_offsets.size() - 1),
              local_vertex_partition_range_first);
            auto dist_pair_first = thrust::make_zip_iterator(forest_edge_src_dists.begin(),
                                                             forest_edge_dst_dists.begin());
            auto num_invalids =
              thrust::count_if(handle_->get_thrust_policy(),
                               dist_pair_first,
                               dist_pair_first + forest_edge_src_dists.size(),
                               cuda::proclaim_return_type<bool>([] __device__(auto pair) {
                                 auto src_dist = thrust::get<0>(pair);
                                 auto dst_dist = thrust::get<1>(pair);
                                 if (src_dist == invalid_distance) {
                                   return dst_dist != invalid_distance;
                                 } else {
                                   return (dst_dist == invalid_distance) ||
                                          (((src_dist >= dst_dist) ? (src_dist - dst_dist)
                                                                   : (dst_dist - src_dist)) > 1);
                                 }
                               }));
            num_invalids = cugraph::host_scalar_allreduce(
              comm, num_invalids, raft::comms::op_t::SUM, handle_->get_stream());
            ASSERT_EQ(num_invalids, 0)
              << "the distances from the starting vertex differ by more than one.";
          }
        }

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.stop();
          hr_timer.display_and_clear(std::cout);
        }

        /* all the reachable vertices are in the same connected component, all the unreachable
         * vertices in different connected components */

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.start("validate (connected components)");
        }

        {
          auto pair_first =
            thrust::make_zip_iterator(components.begin(), d_mg_predecessors.begin());
          auto num_invalids =
            thrust::count_if(handle_->get_thrust_policy(),
                             pair_first,
                             pair_first + components.size(),
                             cuda::proclaim_return_type<bool>(
                               [starting_vertex_component, invalid_vertex] __device__(auto pair) {
                                 auto c    = thrust::get<0>(pair);
                                 auto pred = thrust::get<1>(pair);
                                 if (c == starting_vertex_component) {
                                   return pred == invalid_vertex;
                                 } else {
                                   return pred != invalid_vertex;
                                 }
                               }));
          num_invalids = cugraph::host_scalar_allreduce(
            comm, num_invalids, raft::comms::op_t::SUM, handle_->get_stream());
          ASSERT_EQ(num_invalids, 0) << "the BFS tree does not span the entire connected "
                                        "component of the starting vertex.";
        }

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.stop();
          hr_timer.display_and_clear(std::cout);
        }

        /* check that predecessor->v edges exist in the input graph */

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.start("validate (predecessor->v edge existence)");
        }

        {
          rmm::device_uvector<vertex_t> query_preds(d_mg_predecessors.size(),
                                                    handle_->get_stream());
          rmm::device_uvector<vertex_t> query_vertices(query_preds.size(), handle_->get_stream());
          auto input_edge_first = thrust::make_zip_iterator(
            d_mg_predecessors.begin(),
            thrust::make_counting_iterator(local_vertex_partition_range_first));
          auto output_edge_first =
            thrust::make_zip_iterator(query_preds.begin(), query_vertices.begin());
          query_preds.resize(
            thrust::distance(
              output_edge_first,
              thrust::copy_if(handle_->get_thrust_policy(),
                              input_edge_first,
                              input_edge_first + d_mg_predecessors.size(),
                              output_edge_first,
                              cuda::proclaim_return_type<bool>(
                                [invalid_vertex, starting_vertex] __device__(auto pair) {
                                  auto pred = thrust::get<0>(pair);
                                  auto v    = thrust::get<1>(pair);
                                  return (pred != invalid_vertex /* reachable */) &&
                                         (v != starting_vertex);
                                }))),
            handle_->get_stream());
          query_vertices.resize(query_preds.size(), handle_->get_stream());
          if (reachable_from_2cores) {  // exclude the edges in the forest (parents[v] -> v)
            auto query_edge_first =
              thrust::make_zip_iterator(query_preds.begin(), query_vertices.begin());
            query_preds.resize(
              thrust::distance(
                query_edge_first,
                thrust::remove_if(
                  handle_->get_thrust_policy(),
                  query_edge_first,
                  query_edge_first + query_preds.size(),
                  cuda::proclaim_return_type<bool>(
                    [parents = raft::device_span<vertex_t const>(parents.data(), parents.size()),
                     v_first = local_vertex_partition_range_first] __device__(auto pair) {
                      auto pred   = thrust::get<0>(pair);
                      auto v      = thrust::get<1>(pair);
                      auto parent = parents[v - v_first];
                      return parent == pred;  // the query edge exists in the forest
                    }))),
              handle_->get_stream());
            query_vertices.resize(query_preds.size(), handle_->get_stream());
            if (!in_2cores) {  // found BFS predecessor tree may contain edges from v ->
                               // parents[v] (instead of parents[v] -> v)
              rmm::device_uvector<vertex_t> forest_edge_vertices(parents.size(),
                                                                 handle_->get_stream());
              rmm::device_uvector<vertex_t> forest_edge_parents(forest_edge_vertices.size(),
                                                                handle_->get_stream());
              auto input_first = thrust::make_zip_iterator(
                thrust::make_counting_iterator(local_vertex_partition_range_first),
                parents.begin());
              auto output_first = thrust::make_zip_iterator(forest_edge_vertices.begin(),
                                                            forest_edge_parents.begin());
              forest_edge_vertices.resize(
                thrust::distance(
                  output_first,
                  thrust::copy_if(handle_->get_thrust_policy(),
                                  input_first,
                                  input_first + mg_renumber_map.size(),
                                  output_first,
                                  cuda::proclaim_return_type<bool>([] __device__(auto pair) {
                                    auto v      = thrust::get<0>(pair);
                                    auto parent = thrust::get<1>(pair);
                                    return (parent != invalid_vertex /* reachable */) &&
                                           (parent != v /* v is not in 2-cores */);
                                  }))),
                handle_->get_stream());
              forest_edge_parents.resize(forest_edge_vertices.size(), handle_->get_stream());
              std::tie(forest_edge_parents, forest_edge_vertices) =
                cugraph::detail::shuffle_int_vertex_value_pairs_to_local_gpu_by_vertex_partitioning<
                  vertex_t,
                  vertex_t>(
                  *handle_,
                  std::move(forest_edge_parents) /* vertex in (vertex, value) pair */,
                  std::move(forest_edge_vertices) /* value in (vertex, value) pair */,
                  raft::host_span<vertex_t const>(vertex_partition_range_offsets.data() + 1,
                                                  vertex_partition_range_offsets.size() - 1));
              auto forest_edge_first = thrust::make_zip_iterator(forest_edge_vertices.begin(),
                                                                 forest_edge_parents.begin());
              thrust::sort(handle_->get_thrust_policy(),
                           forest_edge_first,
                           forest_edge_first + forest_edge_vertices.size());
              query_edge_first =
                thrust::make_zip_iterator(query_preds.begin(), query_vertices.begin());
              query_preds.resize(
                thrust::distance(
                  query_edge_first,
                  thrust::remove_if(
                    handle_->get_thrust_policy(),
                    query_edge_first,
                    query_edge_first + query_preds.size(),
                    cuda::proclaim_return_type<bool>(
                      [forest_edge_first,
                       forest_edge_last =
                         forest_edge_first + forest_edge_vertices.size()] __device__(auto pair) {
                        auto pred = thrust::get<0>(pair);
                        auto v    = thrust::get<1>(pair);
                        auto key  = thrust::make_tuple(pred, v);
                        auto it   = thrust::lower_bound(
                          thrust::seq, forest_edge_first, forest_edge_last, key);
                        return (it != forest_edge_last) && (*it == key);
                      }))),
                handle_->get_stream());
              query_vertices.resize(query_preds.size(), handle_->get_stream());
            }
          }

          auto mg_graph_to_subgraph_map = raft::device_span<vertex_t const>(
            reachable_from_2cores ? mg_graph_to_pruned_graph_map.data()
                                  : mg_graph_to_isolated_trees_map.data(),
            reachable_from_2cores ? mg_graph_to_pruned_graph_map.size()
                                  : mg_graph_to_isolated_trees_map.size());
          auto mg_subgraph_view =
            reachable_from_2cores ? mg_pruned_graph_view : mg_isolated_trees_view;

          thrust::transform(
            handle_->get_thrust_policy(),
            query_vertices.begin(),
            query_vertices.end(),
            query_vertices.begin(),
            [mg_graph_to_subgraph_map,
             subgraph_v_first = mg_subgraph_view.local_vertex_partition_range_first(),
             v_first          = local_vertex_partition_range_first,
             invalid_vertex] __device__(auto v) {
              auto v_offset = mg_graph_to_subgraph_map[v - v_first];
              return (v_offset != invalid_vertex) ? (subgraph_v_first + v_offset) : invalid_vertex;
            });
          std::tie(query_preds, query_vertices) = cugraph::detail::
            shuffle_int_vertex_value_pairs_to_local_gpu_by_vertex_partitioning<vertex_t, vertex_t>(
              *handle_,
              std::move(query_preds) /* vertex in (vertex, value) pair */,
              std::move(query_vertices) /* value in (vertex, value) pair */,
              raft::host_span<vertex_t const>(vertex_partition_range_offsets.data() + 1,
                                              vertex_partition_range_offsets.size() - 1));
          thrust::transform(
            handle_->get_thrust_policy(),
            query_preds.begin(),
            query_preds.end(),
            query_preds.begin(),
            [mg_graph_to_subgraph_map,
             subgraph_v_first = mg_subgraph_view.local_vertex_partition_range_first(),
             v_first          = local_vertex_partition_range_first,
             invalid_vertex] __device__(auto v) {
              auto v_offset = mg_graph_to_subgraph_map[v - v_first];
              return (v_offset != invalid_vertex) ? (subgraph_v_first + v_offset) : invalid_vertex;
            });
          auto num_invalids = thrust::count_if(
            handle_->get_thrust_policy(),
            query_preds.begin(),
            query_preds.end(),
            [invalid_vertex] __device__(auto pred) { return pred == invalid_vertex; });
          num_invalids +=
            thrust::count_if(handle_->get_thrust_policy(),
                             query_vertices.begin(),
                             query_vertices.end(),
                             [invalid_vertex] __device__(auto v) { return v == invalid_vertex; });
          num_invalids = cugraph::host_scalar_allreduce(
            comm, num_invalids, raft::comms::op_t::SUM, handle_->get_stream());
          ASSERT_EQ(num_invalids, 0) << "predecessor->v missing in the input graph.";

          std::vector<cugraph::arithmetic_device_uvector_t> edge_properties{};

          std::tie(query_preds, query_vertices, std::ignore, std::ignore) =
            cugraph::shuffle_int_edges(*handle_,
                                       std::move(query_preds),
                                       std::move(query_vertices),
                                       std::move(edge_properties),
                                       store_transposed,
                                       mg_subgraph_view.vertex_partition_range_lasts());

          auto flags = mg_subgraph_view.has_edge(
            *handle_,
            raft::device_span<vertex_t const>(query_preds.data(), query_preds.size()),
            raft::device_span<vertex_t const>(query_vertices.data(), query_vertices.size()));
          num_invalids =
            thrust::count(handle_->get_thrust_policy(), flags.begin(), flags.end(), false);
          num_invalids = cugraph::host_scalar_allreduce(
            comm, num_invalids, raft::comms::op_t::SUM, handle_->get_stream());
          ASSERT_EQ(num_invalids, 0) << "predecessor->v missing in the input graph.";
        }

        if (cugraph::test::g_perf) {
          RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
          comm.barrier();
          hr_timer.stop();
          hr_timer.display_and_clear(std::cout);
        }
      }
    }

    std::cout << "average MG BFS (Kernel 2) time: " << (total_elapsed / num_timed_starting_vertices)
              << " TTEPS (arithmetic mean)=" << tteps_sum / num_timed_starting_vertices
              << " TTEPS (harmonic_mean)="
              << (one_over_tteps_sum > 0.0
                    ? static_cast<double>(num_timed_starting_vertices) / one_over_tteps_sum
                    : std::numeric_limits<double>::infinity())
              << std::endl;
  }

 private:
  static std::unique_ptr<raft::handle_t> handle_;
};

template <typename input_usecase_t>
std::unique_ptr<raft::handle_t> Tests_GRAPH500_MGBFS<input_usecase_t>::handle_ = nullptr;

using Tests_GRAPH500_MGBFS_Rmat = Tests_GRAPH500_MGBFS<cugraph::test::Rmat_Usecase>;

TEST_P(Tests_GRAPH500_MGBFS_Rmat, CheckInt64Int64)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t>(
    std::get<0>(param), override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

INSTANTIATE_TEST_SUITE_P(
  rmat_small_test,
  Tests_GRAPH500_MGBFS_Rmat,
  ::testing::Values(
    // enable correctness checks
    std::make_tuple(Graph500_BFS_Usecase{true, true, true},
                    cugraph::test::Rmat_Usecase(10,
                                                16,
                                                0.57,
                                                0.19,
                                                0.19,
                                                0 /* base RNG seed */,
                                                true /* undirected */,
                                                true /* scramble vertex ID */))));

INSTANTIATE_TEST_SUITE_P(
  rmat_benchmark_test, /* note that scale & edge factor can be overridden in benchmarking (with
                          --gtest_filter to select only the rmat_benchmark_test with a specific
                          vertex & edge type combination) by command line arguments and do not
                          include more than one Rmat_Usecase that differ only in scale or edge
                          factor (to avoid running same benchmarks more than once) */
  Tests_GRAPH500_MGBFS_Rmat,
  ::testing::Values(
    // disable correctness checks for large graphs
    std::make_tuple(Graph500_BFS_Usecase{true, true, false},
                    cugraph::test::Rmat_Usecase(20,
                                                16,
                                                0.57,
                                                0.19,
                                                0.19,
                                                0 /* base RNG seed */,
                                                true /* undirected */,
                                                true /* scramble vertex IDs */))));

CUGRAPH_MG_TEST_PROGRAM_MAIN()
