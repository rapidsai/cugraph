/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <raft/error.hpp>

#include <nccl.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>

#include <gtest/gtest.h>
#include <mpi.h>

#include <cstring>

TEST(allgather, success)
{
  int p = 1, r = 0, dev = 0, dev_count = 0;
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &p));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &r));
  CUDA_TRY(hipGetDeviceCount(&dev_count));

  // shortcut for device ID here
  // may need something smarter later
  dev = r % dev_count;
  // hipSetDevice must happen before ncclCommInitRank
  CUDA_TRY(hipSetDevice(dev));

  // print info
  printf("#   Rank %2d - Pid %6d - device %2d\n", r, getpid(), dev);

  // NCCL init
  ncclUniqueId id;
  ncclComm_t comm;
  if (r == 0) NCCL_TRY(ncclGetUniqueId(&id));
  MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));
  NCCL_TRY(ncclCommInitRank(&comm, p, id, r));
  MPICHECK(MPI_Barrier(MPI_COMM_WORLD));

  // allocate device buffers
  int size = 3;
  float *sendbuff, *recvbuff;
  CUDA_TRY(hipMalloc(&sendbuff, size * sizeof(float)));
  CUDA_TRY(hipMalloc(&recvbuff, size * p * sizeof(float)));

  // init values
  thrust::fill(
    thrust::device_pointer_cast(sendbuff), thrust::device_pointer_cast(sendbuff + size), (float)r);
  thrust::fill(
    thrust::device_pointer_cast(recvbuff), thrust::device_pointer_cast(recvbuff + size * p), -1.0f);

  // ncclAllGather
  NCCL_TRY(ncclAllGather(
    (const void *)sendbuff, (void *)recvbuff, size, ncclFloat, comm, hipStreamDefault));

  // expect each rankid printed size times in ascending order
  if (r == 0) {
    thrust::device_ptr<float> dev_ptr(recvbuff);
    std::cout.precision(15);
    thrust::copy(dev_ptr, dev_ptr + size * p, std::ostream_iterator<float>(std::cout, " "));
    std::cout << std::endl;
  }

  // free device buffers
  CUDA_TRY(hipFree(sendbuff));
  CUDA_TRY(hipFree(recvbuff));

  // finalizing NCCL
  NCCL_TRY(ncclCommDestroy(comm));
}

int main(int argc, char **argv)
{
  testing::InitGoogleTest(&argc, argv);
  MPI_Init(&argc, &argv);
  {
    auto resource = std::make_unique<rmm::mr::cuda_memory_resource>();
    rmm::mr::set_default_resource(resource.get());
    int rc = RUN_ALL_TESTS();
  }
  MPI_Finalize();
  return rc;
}
