#include <mpi.h>
#include <nccl.h>
#include <string.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include "gtest/gtest.h"

#include "utilities/test_utilities.hpp"

TEST(allgather, success)
{
  int p = 1, r = 0, dev = 0, dev_count = 0;
  MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &p));
  MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &r));
  CUDA_RT_CALL(hipGetDeviceCount(&dev_count));

  // shortcut for device ID here
  // may need something smarter later
  dev = r % dev_count;
  // hipSetDevice must happen before ncclCommInitRank
  CUDA_RT_CALL(hipSetDevice(dev));

  // print info
  printf("#   Rank %2d - Pid %6d - device %2d\n", r, getpid(), dev);

  // NCCL init
  ncclUniqueId id;
  ncclComm_t comm;
  if (r == 0) NCCLCHECK(ncclGetUniqueId(&id));
  MPICHECK(MPI_Bcast((void *)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));
  NCCLCHECK(ncclCommInitRank(&comm, p, id, r));
  MPICHECK(MPI_Barrier(MPI_COMM_WORLD));

  // allocate device buffers
  int size = 3;
  float *sendbuff, *recvbuff;
  CUDA_RT_CALL(hipMalloc(&sendbuff, size * sizeof(float)));
  CUDA_RT_CALL(hipMalloc(&recvbuff, size * p * sizeof(float)));

  // init values
  thrust::fill(
    thrust::device_pointer_cast(sendbuff), thrust::device_pointer_cast(sendbuff + size), (float)r);
  thrust::fill(
    thrust::device_pointer_cast(recvbuff), thrust::device_pointer_cast(recvbuff + size * p), -1.0f);

  // ncclAllGather
  NCCLCHECK(ncclAllGather(
    (const void *)sendbuff, (void *)recvbuff, size, ncclFloat, comm, hipStreamDefault));

  // expect each rankid printed size times in ascending order
  if (r == 0) {
    thrust::device_ptr<float> dev_ptr(recvbuff);
    std::cout.precision(15);
    thrust::copy(dev_ptr, dev_ptr + size * p, std::ostream_iterator<float>(std::cout, " "));
    std::cout << std::endl;
  }

  // free device buffers
  CUDA_RT_CALL(hipFree(sendbuff));
  CUDA_RT_CALL(hipFree(recvbuff));

  // finalizing NCCL
  NCCLCHECK(ncclCommDestroy(comm));
}

int main(int argc, char **argv)
{
  testing::InitGoogleTest(&argc, argv);
  MPI_Init(&argc, &argv);
  {
    auto resource = std::make_unique<rmm::mr::cuda_memory_resource>();
    rmm::mr::set_default_resource(resource.get());
    int rc = RUN_ALL_TESTS();
  }
  MPI_Finalize();
  return rc;
}
