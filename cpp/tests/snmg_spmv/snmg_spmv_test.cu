#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cugraph.h>
#include <omp.h>
#include "hip/hip_runtime_api.h"
#include "gtest/gtest.h"
#include "high_res_clock.h"
#include "snmg_test_utils.h"
#include "test_utils.h"

//#define SNMG_VERBOSE

// ref SPMV on the host
template <typename idx_t, typename val_t>
void csrmv_h(std::vector<idx_t>& off_h,
             std::vector<idx_t>& ind_h,
             std::vector<val_t>& val_h,
             std::vector<val_t>& x,
             std::vector<val_t>& y)
{
#pragma omp parallel for
  for (auto i = size_t{0}; i < y.size(); ++i) {
    // std::cout<< omp_get_num_threads()<<std::endl;
    for (auto j = off_h[i]; j < off_h[i + 1]; ++j) y[i] += val_h[j] * x[ind_h[j]];
  }
}

typedef struct MGSpmv_Usecase_t {
  std::string matrix_file;
  MGSpmv_Usecase_t(const std::string& a)
  {
    // assume relative paths are relative to RAPIDS_DATASET_ROOT_DIR
    // if RAPIDS_DATASET_ROOT_DIR not set, default to "/datasets"
    const std::string& rapidsDatasetRootDir = get_rapids_dataset_root_dir();
    if ((a != "") && (a[0] != '/')) {
      matrix_file = rapidsDatasetRootDir + "/" + a;
    } else {
      matrix_file = a;
    }
  }
  MGSpmv_Usecase_t& operator=(const MGSpmv_Usecase_t& rhs)
  {
    matrix_file = rhs.matrix_file;
    return *this;
  }
} MGSpmv_Usecase;

class Tests_MGSpmv : public ::testing::TestWithParam<MGSpmv_Usecase> {
 public:
  Tests_MGSpmv() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}
  virtual void SetUp() {}
  virtual void TearDown() {}

  static std::vector<double> mgspmv_time;

  template <typename idx_t, typename val_t>
  void run_current_test(const MGSpmv_Usecase& param)
  {
    const ::testing::TestInfo* const test_info =
      ::testing::UnitTest::GetInstance()->current_test_info();
    std::stringstream ss;
    std::string test_id = std::string(test_info->test_case_name()) + std::string(".") +
                          std::string(test_info->name()) + std::string("_") +
                          getFileName(param.matrix_file) + std::string("_") + ss.str().c_str();

    int m, k, nnz, n_gpus;
    MM_typecode mc;

    double t;

    FILE* fpin = fopen(param.matrix_file.c_str(), "r");
    ASSERT_NE(fpin, nullptr) << "fopen (" << param.matrix_file << ") failure.";

    ASSERT_EQ(mm_properties<int>(fpin, 1, &mc, &m, &k, &nnz), 0)
      << "could not read Matrix Market file properties"
      << "\n";
    ASSERT_TRUE(mm_is_matrix(mc));
    ASSERT_TRUE(mm_is_coordinate(mc));
    ASSERT_FALSE(mm_is_complex(mc));
    ASSERT_FALSE(mm_is_skew(mc));

    // Allocate memory on host
    std::vector<idx_t> cooRowInd(nnz), cooColInd(nnz), csrColInd(nnz), csrRowPtr(m + 1);
    std::vector<val_t> cooVal(nnz), csrVal(nnz), x_h(m, 1.0), y_h(m, 0.0), y_ref(m, 0.0);

    // Read
    ASSERT_EQ((mm_to_coo<int, val_t>(fpin, 1, nnz, &cooRowInd[0], &cooColInd[0], NULL, NULL)), 0)
      << "could not read matrix data"
      << "\n";
    ASSERT_EQ(fclose(fpin), 0);
    coo2csr(cooRowInd, cooColInd, csrRowPtr, csrColInd);

    CUDA_RT_CALL(hipGetDeviceCount(&n_gpus));
    std::vector<size_t> v_loc(n_gpus), e_loc(n_gpus), part_offset(n_gpus + 1);
    random_vals(csrVal);
    random_vals(x_h);
    gdf_column* col_x[n_gpus];
    // reference result
    t = omp_get_wtime();
    csrmv_h<idx_t, val_t>(csrRowPtr, csrColInd, csrVal, x_h, y_ref);
    std::cout << omp_get_wtime() - t << " ";
    if (nnz < 1200000000) {
#pragma omp parallel num_threads(1)
      {
        auto i = omp_get_thread_num();
        auto p = omp_get_num_threads();
        CUDA_RT_CALL(hipSetDevice(i));

#ifdef SNMG_VERBOSE
#pragma omp master
        {
          std::cout << "Number of GPUs : " << n_gpus << std::endl;
          std::cout << "Number of threads : " << p << std::endl;
        }
#endif

        gdf_column *col_off = new gdf_column, *col_ind = new gdf_column, *col_val = new gdf_column;
        col_x[i] = new gdf_column;
        create_gdf_column(x_h, col_x[i]);
#pragma omp barrier

        // load a chunck of the graph on each GPU
        load_csr_loc(
          csrRowPtr, csrColInd, csrVal, v_loc, e_loc, part_offset, col_off, col_ind, col_val);
        t = omp_get_wtime();
        cugraph::snmg_csrmv(&part_offset[0], col_off, col_ind, col_val, col_x);

#pragma omp master
        {
          std::cout << omp_get_wtime() - t << " ";
        }

#pragma omp master
        {
          CUDA_RT_CALL(
            hipMemcpy(&y_h[0], col_x[0]->data, sizeof(val_t) * m, hipMemcpyDeviceToHost));

          for (auto j = size_t{0}; j < y_h.size(); ++j) EXPECT_LE(fabs(y_ref[j] - y_h[j]), 0.0001);
        }

        gdf_col_delete(col_off);
        gdf_col_delete(col_ind);
        gdf_col_delete(col_val);
        gdf_col_delete(col_x[i]);
      }
    }
    if (n_gpus > 1) {
      // Only using the 4 fully connected GPUs on DGX1
      if (n_gpus == 8) n_gpus = 4;

#pragma omp parallel num_threads(n_gpus)
      {
        auto i = omp_get_thread_num();
        auto p = omp_get_num_threads();
        CUDA_RT_CALL(hipSetDevice(i));

#ifdef SNMG_VERBOSE
#pragma omp master
        {
          std::cout << "Number of GPUs : " << n_gpus << std::endl;
          std::cout << "Number of threads : " << p << std::endl;
        }
#endif

        gdf_column *col_off = new gdf_column, *col_ind = new gdf_column, *col_val = new gdf_column;
        col_x[i] = new gdf_column;
        create_gdf_column(x_h, col_x[i]);
#pragma omp barrier

        load_csr_loc(
          csrRowPtr, csrColInd, csrVal, v_loc, e_loc, part_offset, col_off, col_ind, col_val);
        t = omp_get_wtime();
        cugraph::snmg_csrmv(&part_offset[0], col_off, col_ind, col_val, col_x);

#pragma omp master
        {
          std::cout << omp_get_wtime() - t << " ";
        }

#pragma omp master
        {
          CUDA_RT_CALL(
            hipMemcpy(&y_h[0], col_x[0]->data, sizeof(val_t) * m, hipMemcpyDeviceToHost));

          for (auto j = size_t{0}; j < y_h.size(); ++j) EXPECT_LE(fabs(y_ref[j] - y_h[j]), 0.0001);
        }

        gdf_col_delete(col_off);
        gdf_col_delete(col_ind);
        gdf_col_delete(col_val);
        gdf_col_delete(col_x[i]);
      }
    }
    std::cout << std::endl;
  }
};

TEST_P(Tests_MGSpmv, CheckFP32_mtx) { run_current_test<int, float>(GetParam()); }
TEST_P(Tests_MGSpmv, CheckFP64) { run_current_test<int, double>(GetParam()); }

class Tests_MGSpmv_hibench : public ::testing::TestWithParam<MGSpmv_Usecase> {
 public:
  Tests_MGSpmv_hibench() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}
  virtual void SetUp() {}
  virtual void TearDown() {}

  static std::vector<double> mgspmv_time;

  template <typename idx_t, typename val_t>
  void run_current_test(const MGSpmv_Usecase& param)
  {
    const ::testing::TestInfo* const test_info =
      ::testing::UnitTest::GetInstance()->current_test_info();
    std::stringstream ss;
    std::string test_id = std::string(test_info->test_case_name()) + std::string(".") +
                          std::string(test_info->name()) + std::string("_") +
                          getFileName(param.matrix_file) + std::string("_") + ss.str().c_str();

    int m, nnz, n_gpus;

    std::vector<idx_t> cooRowInd, cooColInd;
    double t;

    ASSERT_EQ(read_single_file(param.matrix_file.c_str(), cooRowInd, cooColInd), 0)
      << "read_single_file(" << param.matrix_file << ", ...) failure.";
    nnz = cooRowInd.size();
    m   = 1 + std::max(*(std::max_element(cooRowInd.begin(), cooRowInd.end())),
                     *(std::max_element(cooColInd.begin(), cooColInd.end())));

    // Allocate memory on host
    std::vector<idx_t> csrColInd(nnz), csrRowPtr(m + 1);
    std::vector<val_t> cooVal(nnz), csrVal(nnz), x_h(m, 1.0), y_h(m, 0.0), y_ref(m, 0.0);
    coo2csr(cooRowInd, cooColInd, csrRowPtr, csrColInd);
    CUDA_RT_CALL(hipGetDeviceCount(&n_gpus));
    std::vector<size_t> v_loc(n_gpus), e_loc(n_gpus), part_offset(n_gpus + 1);
    random_vals(csrVal);
    random_vals(x_h);
    gdf_column* col_x[n_gpus];
    // reference result
    t = omp_get_wtime();
    csrmv_h(csrRowPtr, csrColInd, csrVal, x_h, y_ref);
    std::cout << omp_get_wtime() - t << " ";

    if (nnz < 1200000000) {
#pragma omp parallel num_threads(1)
      {
        auto i = omp_get_thread_num();
        auto p = omp_get_num_threads();
        CUDA_RT_CALL(hipSetDevice(i));

#ifdef SNMG_VERBOSE
#pragma omp master
        {
          std::cout << "Number of GPUs : " << n_gpus << std::endl;
          std::cout << "Number of threads : " << p << std::endl;
        }
#endif

        gdf_column *col_off = new gdf_column, *col_ind = new gdf_column, *col_val = new gdf_column;
        col_x[i] = new gdf_column;
        create_gdf_column(x_h, col_x[i]);
#pragma omp barrier

        // load a chunck of the graph on each GPU
        load_csr_loc(
          csrRowPtr, csrColInd, csrVal, v_loc, e_loc, part_offset, col_off, col_ind, col_val);
        t = omp_get_wtime();
        cugraph::snmg_csrmv(&part_offset[0], col_off, col_ind, col_val, col_x);

#pragma omp master
        {
          std::cout << omp_get_wtime() - t << " ";
        }

#pragma omp master
        {
          CUDA_RT_CALL(
            hipMemcpy(&y_h[0], col_x[0]->data, sizeof(val_t) * m, hipMemcpyDeviceToHost));

          for (auto j = size_t{0}; j < y_h.size(); ++j) EXPECT_LE(fabs(y_ref[j] - y_h[j]), 0.0001);
        }

        gdf_col_delete(col_off);
        gdf_col_delete(col_ind);
        gdf_col_delete(col_val);
        gdf_col_delete(col_x[i]);
      }
    }
    if (n_gpus > 1) {
      // Only using the 4 fully connected GPUs on DGX1
      if (n_gpus == 8) n_gpus = 4;

#pragma omp parallel num_threads(n_gpus)
      {
        auto i = omp_get_thread_num();
        auto p = omp_get_num_threads();
        CUDA_RT_CALL(hipSetDevice(i));

#ifdef SNMG_VERBOSE
#pragma omp master
        {
          std::cout << "Number of GPUs : " << n_gpus << std::endl;
          std::cout << "Number of threads : " << p << std::endl;
        }
#endif

        gdf_column *col_off = new gdf_column, *col_ind = new gdf_column, *col_val = new gdf_column;
        col_x[i] = new gdf_column;
        create_gdf_column(x_h, col_x[i]);
#pragma omp barrier

        // load a chunck of the graph on each GPU
        load_csr_loc(
          csrRowPtr, csrColInd, csrVal, v_loc, e_loc, part_offset, col_off, col_ind, col_val);
        t = omp_get_wtime();
        cugraph::snmg_csrmv(&part_offset[0], col_off, col_ind, col_val, col_x);

#pragma omp master
        {
          std::cout << omp_get_wtime() - t << " ";
        }

#pragma omp master
        {
          CUDA_RT_CALL(
            hipMemcpy(&y_h[0], col_x[0]->data, sizeof(val_t) * m, hipMemcpyDeviceToHost));

          for (auto j = size_t{0}; j < y_h.size(); ++j) EXPECT_LE(fabs(y_ref[j] - y_h[j]), 0.0001);
        }

        gdf_col_delete(col_off);
        gdf_col_delete(col_ind);
        gdf_col_delete(col_val);
        gdf_col_delete(col_x[i]);
      }
    }
    std::cout << std::endl;
  }
};

TEST_P(Tests_MGSpmv_hibench, CheckFP32_hibench) { run_current_test<int, float>(GetParam()); }

class Tests_MGSpmv_unsorted : public ::testing::TestWithParam<MGSpmv_Usecase> {
 public:
  Tests_MGSpmv_unsorted() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}
  virtual void SetUp() {}
  virtual void TearDown() {}

  static std::vector<double> mgspmv_time;

  template <typename idx_t, typename val_t>
  void run_current_test(const MGSpmv_Usecase& param)
  {
    const ::testing::TestInfo* const test_info =
      ::testing::UnitTest::GetInstance()->current_test_info();
    std::stringstream ss;
    std::string test_id = std::string(test_info->test_case_name()) + std::string(".") +
                          std::string(test_info->name()) + std::string("_") +
                          getFileName(param.matrix_file) + std::string("_") + ss.str().c_str();

    int m, k, nnz, n_gpus;
    MM_typecode mc;

    double t;

    FILE* fpin = fopen(param.matrix_file.c_str(), "r");
    ASSERT_NE(fpin, nullptr) << "fopen (" << param.matrix_file << ") failure.";

    ASSERT_EQ(mm_properties<int>(fpin, 1, &mc, &m, &k, &nnz), 0)
      << "could not read Matrix Market file properties"
      << "\n";
    ASSERT_TRUE(mm_is_matrix(mc));
    ASSERT_TRUE(mm_is_coordinate(mc));
    ASSERT_FALSE(mm_is_complex(mc));
    ASSERT_FALSE(mm_is_skew(mc));

    // Allocate memory on host
    std::vector<idx_t> cooRowInd(nnz), cooColInd(nnz), csrColInd(nnz), csrRowPtr(m + 1);
    std::vector<val_t> cooVal(nnz), csrVal(nnz), x_h(m, 1.0), y_h(m, 0.0), y_ref(m, 0.0);

    // Read
    ASSERT_EQ((mm_to_coo<int, val_t>(fpin, 1, nnz, &cooRowInd[0], &cooColInd[0], NULL, NULL)), 0)
      << "could not read matrix data"
      << "\n";
    ASSERT_EQ(fclose(fpin), 0);
    coo2csr(cooRowInd, cooColInd, csrRowPtr, csrColInd);

    // unsorted random indices
    for (size_t i = 0; i < csrColInd.size(); i++)
      csrColInd[i] = static_cast<idx_t>(std::rand() % m);

    CUDA_RT_CALL(hipGetDeviceCount(&n_gpus));
    std::vector<size_t> v_loc(n_gpus), e_loc(n_gpus), part_offset(n_gpus + 1);
    random_vals(csrVal);
    random_vals(x_h);
    gdf_column* col_x[n_gpus];
    // reference result
    t = omp_get_wtime();
    csrmv_h(csrRowPtr, csrColInd, csrVal, x_h, y_ref);
    std::cout << omp_get_wtime() - t << " ";
    if (nnz < 1200000000) {
#pragma omp parallel num_threads(1)
      {
        auto i = omp_get_thread_num();
        auto p = omp_get_num_threads();
        CUDA_RT_CALL(hipSetDevice(i));

#ifdef SNMG_VERBOSE
#pragma omp master
        {
          std::cout << "Number of GPUs : " << n_gpus << std::endl;
          std::cout << "Number of threads : " << p << std::endl;
        }
#endif

        gdf_column *col_off = new gdf_column, *col_ind = new gdf_column, *col_val = new gdf_column;
        col_x[i] = new gdf_column;
        create_gdf_column(x_h, col_x[i]);
#pragma omp barrier

        // load a chunck of the graph on each GPU
        load_csr_loc(
          csrRowPtr, csrColInd, csrVal, v_loc, e_loc, part_offset, col_off, col_ind, col_val);
        t = omp_get_wtime();
        cugraph::snmg_csrmv(&part_offset[0], col_off, col_ind, col_val, col_x);

#pragma omp master
        {
          std::cout << omp_get_wtime() - t << " ";
        }

#pragma omp master
        {
          CUDA_RT_CALL(
            hipMemcpy(&y_h[0], col_x[0]->data, sizeof(val_t) * m, hipMemcpyDeviceToHost));

          for (size_t j = 0; j < y_h.size(); ++j) EXPECT_LE(fabs(y_ref[j] - y_h[j]), 0.0001);
        }

        gdf_col_delete(col_off);
        gdf_col_delete(col_ind);
        gdf_col_delete(col_val);
        gdf_col_delete(col_x[i]);
      }
    }
    if (n_gpus > 1) {
      // Only using the 4 fully connected GPUs on DGX1
      if (n_gpus == 8) n_gpus = 4;

#pragma omp parallel num_threads(n_gpus)
      {
        auto i = omp_get_thread_num();
        auto p = omp_get_num_threads();
        CUDA_RT_CALL(hipSetDevice(i));

#ifdef SNMG_VERBOSE
#pragma omp master
        {
          std::cout << "Number of GPUs : " << n_gpus << std::endl;
          std::cout << "Number of threads : " << p << std::endl;
        }
#endif

        gdf_column *col_off = new gdf_column, *col_ind = new gdf_column, *col_val = new gdf_column;
        col_x[i] = new gdf_column;
        create_gdf_column(x_h, col_x[i]);
#pragma omp barrier

        // load a chunck of the graph on each GPU
        load_csr_loc(
          csrRowPtr, csrColInd, csrVal, v_loc, e_loc, part_offset, col_off, col_ind, col_val);
        t = omp_get_wtime();
        cugraph::snmg_csrmv(&part_offset[0], col_off, col_ind, col_val, col_x);

#pragma omp master
        {
          std::cout << omp_get_wtime() - t << " ";
        }

#pragma omp master
        {
          CUDA_RT_CALL(
            hipMemcpy(&y_h[0], col_x[0]->data, sizeof(val_t) * m, hipMemcpyDeviceToHost));

          for (size_t j = 0; j < y_h.size(); ++j) EXPECT_LE(fabs(y_ref[j] - y_h[j]), 0.0001);
        }

        gdf_col_delete(col_off);
        gdf_col_delete(col_ind);
        gdf_col_delete(col_val);
        gdf_col_delete(col_x[i]);
      }
    }
    std::cout << std::endl;
  }
};

TEST_P(Tests_MGSpmv_unsorted, CheckFP32_mtx) { run_current_test<int, float>(GetParam()); }
TEST_P(Tests_MGSpmv_unsorted, CheckFP64) { run_current_test<int, double>(GetParam()); }

INSTANTIATE_TEST_CASE_P(mtx_test,
                        Tests_MGSpmv,
                        ::testing::Values(MGSpmv_Usecase("test/datasets/karate.mtx"),
                                          MGSpmv_Usecase("test/datasets/netscience.mtx"),
                                          MGSpmv_Usecase("test/datasets/cit-Patents.mtx"),
                                          MGSpmv_Usecase("test/datasets/webbase-1M.mtx"),
                                          MGSpmv_Usecase("test/datasets/web-Google.mtx"),
                                          MGSpmv_Usecase("test/datasets/wiki-Talk.mtx")));

INSTANTIATE_TEST_CASE_P(mtx_test,
                        Tests_MGSpmv_unsorted,
                        ::testing::Values(MGSpmv_Usecase("test/datasets/karate.mtx"),
                                          MGSpmv_Usecase("test/datasets/netscience.mtx"),
                                          MGSpmv_Usecase("test/datasets/cit-Patents.mtx"),
                                          MGSpmv_Usecase("test/datasets/webbase-1M.mtx"),
                                          MGSpmv_Usecase("test/datasets/web-Google.mtx"),
                                          MGSpmv_Usecase("test/datasets/wiki-Talk.mtx")));
INSTANTIATE_TEST_CASE_P(
  hibench_test,
  Tests_MGSpmv_hibench,
  ::testing::Values(MGSpmv_Usecase("benchmark/hibench/1/Input-small/edges/part-00000"),
                    MGSpmv_Usecase("benchmark/hibench/1/Input-large/edges/part-00000")));

INSTANTIATE_TEST_CASE_P(
  hibench_test_huge,
  Tests_MGSpmv_hibench,
  ::testing::Values(MGSpmv_Usecase("benchmark/hibench/1/Input-huge/edges/part-00000")));

int main(int argc, char** argv)
{
  rmmInitialize(nullptr);
  testing::InitGoogleTest(&argc, argv);
  int rc = RUN_ALL_TESTS();
  rmmFinalize();
  return rc;
}
