#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <utilities/base_fixture.hpp>
#include <utilities/high_res_clock.h>
#include <utilities/test_utilities.hpp>

#include <converters/COOtoCSR.cuh>

#include <rmm/device_vector.hpp>

#include <cugraph/algorithms.hpp>
#include <cugraph/legacy/graph.hpp>
#include <rmm/device_vector.hpp>

#include <thrust/device_ptr.h>

#include <fstream>

std::vector<int> getGoldenTopKIds(std::ifstream& fs_result, int k = 10)
{
  std::vector<int> vec;
  int val;
  int count = 0;
  while (fs_result >> val && ((count++) < k)) {
    vec.push_back(val);
  }
  vec.resize(k);
  return vec;
}

std::vector<int> getTopKIds(double* p_katz, int count, int k = 10)
{
  rmm::device_vector<int> id(count);
  thrust::sequence(rmm::exec_policy(rmm::cuda_stream_default), id.begin(), id.end());
  thrust::sort_by_key(rmm::exec_policy(rmm::cuda_stream_default),
                      p_katz,
                      p_katz + count,
                      id.begin(),
                      thrust::greater<double>());
  std::vector<int> topK(k);
  thrust::copy(id.begin(), id.begin() + k, topK.begin());
  return topK;
}

template <typename VT, typename ET, typename WT>
int getMaxDegree(cugraph::legacy::GraphCSRView<VT, ET, WT> const& g)
{
  rmm::device_vector<ET> degree_vector(g.number_of_vertices);
  ET* p_degree = degree_vector.data().get();
  g.degree(p_degree, cugraph::legacy::DegreeDirection::OUT);
  ET max_out_degree = thrust::reduce(rmm::exec_policy(rmm::cuda_stream_default),
                                     p_degree,
                                     p_degree + g.number_of_vertices,
                                     static_cast<ET>(-1),
                                     thrust::maximum<ET>());
  return max_out_degree;
}

typedef struct Katz_Usecase_t {
  std::string matrix_file;
  std::string result_file;
  Katz_Usecase_t(const std::string& a, const std::string& b)
  {
    // assume relative paths are relative to RAPIDS_DATASET_ROOT_DIR
    const std::string& rapidsDatasetRootDir = cugraph::test::get_rapids_dataset_root_dir();
    if ((a != "") && (a[0] != '/')) {
      matrix_file = rapidsDatasetRootDir + "/" + a;
    } else {
      matrix_file = a;
    }
    if ((b != "") && (b[0] != '/')) {
      result_file = rapidsDatasetRootDir + "/" + b;
    } else {
      result_file = b;
    }
  }
  Katz_Usecase_t& operator=(const Katz_Usecase_t& rhs)
  {
    matrix_file = rhs.matrix_file;
    result_file = rhs.result_file;
    return *this;
  }
} Katz_Usecase;

class Tests_Katz : public ::testing::TestWithParam<Katz_Usecase> {
 public:
  Tests_Katz() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}
  virtual void SetUp() {}
  virtual void TearDown() {}

  void run_current_test(const Katz_Usecase& param)
  {
    FILE* fpin = fopen(param.matrix_file.c_str(), "r");
    ASSERT_NE(fpin, nullptr) << "fopen (" << param.matrix_file << ") failure.";

    std::ifstream fs_result(param.result_file);
    ASSERT_EQ(fs_result.is_open(), true) << "file open (" << param.result_file << ") failure.";

    int m, k;
    int nnz;
    MM_typecode mc;
    ASSERT_EQ(cugraph::test::mm_properties<int>(fpin, 1, &mc, &m, &k, &nnz), 0)
      << "could not read Matrix Market file properties"
      << "\n";
    ASSERT_TRUE(mm_is_matrix(mc));
    ASSERT_TRUE(mm_is_coordinate(mc));
    ASSERT_FALSE(mm_is_complex(mc));
    ASSERT_FALSE(mm_is_skew(mc));

    // Allocate memory on host
    std::vector<int> cooRowInd(nnz), cooColInd(nnz);
    std::vector<int> cooVal(nnz);
    std::vector<double> katz_centrality(m);

    // Read
    ASSERT_EQ((cugraph::test::mm_to_coo<int, int>(
                fpin, 1, nnz, &cooRowInd[0], &cooColInd[0], &cooVal[0], NULL)),
              0)
      << "could not read matrix data"
      << "\n";
    ASSERT_EQ(fclose(fpin), 0);

    cugraph::legacy::GraphCOOView<int, int, float> cooview(
      &cooColInd[0], &cooRowInd[0], nullptr, m, nnz);
    auto csr                                         = cugraph::coo_to_csr(cooview);
    cugraph::legacy::GraphCSRView<int, int, float> G = csr->view();

    rmm::device_vector<double> katz_vector(m);
    double* d_katz = thrust::raw_pointer_cast(katz_vector.data());

    int max_out_degree = getMaxDegree(G);
    double alpha       = 1 / (static_cast<double>(max_out_degree) + 1);

    cugraph::katz_centrality(G, d_katz, alpha, 100, 1e-6, false, true);

    auto threshold_ratio     = 1e-3;
    auto threshold_magnitude = (1.0 / static_cast<double>(m)) * threshold_ratio;

    std::vector<int> top10CUGraph = getTopKIds(d_katz, m);
    std::vector<int> top10Golden  = getGoldenTopKIds(fs_result);

    auto nearly_equal = [threshold_ratio, threshold_magnitude](auto lhs, auto rhs) {
      return std::abs(lhs - rhs) <
             std::max(std::max(lhs, rhs) * threshold_ratio, threshold_magnitude);
    };

    ASSERT_TRUE(
      std::equal(top10CUGraph.begin(), top10CUGraph.end(), top10Golden.begin(), nearly_equal))
      << "Katz centrality values do not match with the reference values.";
  }
};

INSTANTIATE_TEST_SUITE_P(
  simple_test,
  Tests_Katz,
  ::testing::Values(Katz_Usecase("test/datasets/karate.mtx", "ref/katz/karate.csv"),
                    // Katz_Usecase("test/datasets/netscience.mtx", "ref/katz/netscience.csv"),
                    Katz_Usecase("test/datasets/polbooks.mtx", "ref/katz/polbooks.csv"),
                    Katz_Usecase("test/datasets/dolphins.mtx", "ref/katz/dolphins.csv")));

TEST_P(Tests_Katz, Check) { run_current_test(GetParam()); }

CUGRAPH_TEST_PROGRAM_MAIN()
