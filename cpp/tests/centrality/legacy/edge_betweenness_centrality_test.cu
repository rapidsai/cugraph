/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <traversal/legacy/bfs_ref.h>
#include <utilities/base_fixture.hpp>
#include <utilities/test_utilities.hpp>

#include <raft/error.hpp>
#include <raft/handle.hpp>
#include <rmm/device_vector.hpp>

#include <rmm/device_vector.hpp>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <cugraph/algorithms.hpp>
#include <cugraph/legacy/graph.hpp>

#include <fstream>
#include <queue>
#include <stack>
#include <utility>

#ifndef TEST_EPSILON
#define TEST_EPSILON 0.0001
#endif

// NOTE: Defines under which values the difference should  be discarded when
// considering values are close to zero
//  i.e: Do we consider that the difference between 1.3e-9 and 8.e-12 is
// significant
#ifndef TEST_ZERO_THRESHOLD
#define TEST_ZERO_THRESHOLD 1e-10
#endif

// ============================================================================
// C++ Reference Implementation
// ============================================================================

template <typename vertex_t, typename edge_t, typename weight_t, typename result_t>
edge_t get_edge_index_from_source_and_destination(vertex_t source_vertex,
                                                  vertex_t destination_vertex,
                                                  vertex_t const* indices,
                                                  edge_t const* offsets)
{
  edge_t index          = -1;
  edge_t first_edge_idx = offsets[source_vertex];
  edge_t last_edge_idx  = offsets[source_vertex + 1];
  auto index_it = std::find(indices + first_edge_idx, indices + last_edge_idx, destination_vertex);
  if (index_it != (indices + last_edge_idx)) { index = std::distance(indices, index_it); }
  return index;
}

template <typename vertex_t, typename edge_t, typename weight_t, typename result_t>
void ref_accumulation(result_t* result,
                      vertex_t const* indices,
                      edge_t const* offsets,
                      vertex_t const number_of_vertices,
                      std::stack<vertex_t>& S,
                      std::vector<std::vector<vertex_t>>& pred,
                      std::vector<double>& sigmas,
                      std::vector<double>& deltas,
                      vertex_t source)
{
  for (vertex_t v = 0; v < number_of_vertices; ++v) {
    deltas[v] = 0;
  }
  while (!S.empty()) {
    vertex_t w = S.top();
    S.pop();
    for (vertex_t v : pred[w]) {
      edge_t edge_idx =
        get_edge_index_from_source_and_destination<vertex_t, edge_t, weight_t, result_t>(
          v, w, indices, offsets);
      double coefficient = (sigmas[v] / sigmas[w]) * (1.0 + deltas[w]);

      deltas[v] += coefficient;
      result[edge_idx] += coefficient;
    }
  }
}

// Algorithm 1: Shortest-path vertex betweenness, (Brandes, 2001)
template <typename vertex_t, typename edge_t, typename weight_t, typename result_t>
void reference_edge_betweenness_centrality_impl(vertex_t* indices,
                                                edge_t* offsets,
                                                vertex_t const number_of_vertices,
                                                result_t* result,
                                                vertex_t const* sources,
                                                vertex_t const number_of_sources)
{
  std::queue<vertex_t> Q;
  std::stack<vertex_t> S;
  // NOTE: dist is of type vertex_t not weight_t
  std::vector<vertex_t> dist(number_of_vertices);
  std::vector<std::vector<vertex_t>> pred(number_of_vertices);
  std::vector<double> sigmas(number_of_vertices);
  std::vector<double> deltas(number_of_vertices);

  std::vector<vertex_t> neighbors;

  if (sources) {
    for (vertex_t source_idx = 0; source_idx < number_of_sources; ++source_idx) {
      vertex_t s = sources[source_idx];
      // Step 1: Single-source shortest-paths problem
      //   a. Initialization
      ref_bfs<vertex_t, edge_t>(indices, offsets, number_of_vertices, Q, S, dist, pred, sigmas, s);
      //  Step 2: Accumulation
      //          Back propagation of dependencies
      ref_accumulation<vertex_t, edge_t, weight_t, result_t>(
        result, indices, offsets, number_of_vertices, S, pred, sigmas, deltas, s);
    }
  } else {
    for (vertex_t s = 0; s < number_of_vertices; ++s) {
      // Step 1: Single-source shortest-paths problem
      //   a. Initialization
      ref_bfs<vertex_t, edge_t>(indices, offsets, number_of_vertices, Q, S, dist, pred, sigmas, s);
      //  Step 2: Accumulation
      //          Back propagation of dependencies
      ref_accumulation<vertex_t, edge_t, weight_t, result_t>(
        result, indices, offsets, number_of_vertices, S, pred, sigmas, deltas, s);
    }
  }
}

template <typename vertex_t, typename edge_t, typename weight_t, typename result_t>
void reference_rescale(result_t* result,
                       bool directed,
                       bool normalize,
                       vertex_t const number_of_vertices,
                       edge_t const number_of_edges)
{
  result_t rescale_factor            = static_cast<result_t>(1);
  result_t casted_number_of_vertices = static_cast<result_t>(number_of_vertices);
  if (normalize) {
    if (number_of_vertices > 1) {
      rescale_factor /= ((casted_number_of_vertices) * (casted_number_of_vertices - 1));
    }
  } else {
    if (!directed) { rescale_factor /= static_cast<result_t>(2); }
  }
  for (auto idx = 0; idx < number_of_edges; ++idx) {
    result[idx] *= rescale_factor;
  }
}

template <typename vertex_t, typename edge_t, typename weight_t, typename result_t>
void reference_edge_betweenness_centrality(
  cugraph::legacy::GraphCSRView<vertex_t, edge_t, weight_t> const& graph,
  result_t* result,
  bool normalize,
  vertex_t const number_of_sources,
  vertex_t const* sources)
{
  vertex_t number_of_vertices = graph.number_of_vertices;
  edge_t number_of_edges      = graph.number_of_edges;
  thrust::host_vector<vertex_t> h_indices(number_of_edges);
  thrust::host_vector<edge_t> h_offsets(number_of_vertices + 1);

  thrust::device_ptr<vertex_t> d_indices((vertex_t*)&graph.indices[0]);
  thrust::device_ptr<edge_t> d_offsets((edge_t*)&graph.offsets[0]);

  thrust::copy(d_indices, d_indices + number_of_edges, h_indices.begin());
  thrust::copy(d_offsets, d_offsets + (number_of_vertices + 1), h_offsets.begin());

  hipDeviceSynchronize();

  reference_edge_betweenness_centrality_impl<vertex_t, edge_t, weight_t, result_t>(
    &h_indices[0], &h_offsets[0], number_of_vertices, result, sources, number_of_sources);
  reference_rescale<vertex_t, edge_t, weight_t, result_t>(
    result, graph.prop.directed, normalize, number_of_vertices, number_of_edges);
}

// =============================================================================
// Utility functions
// =============================================================================
// Compare while allowing relatie error of epsilon
// zero_threshold indicates when  we should drop comparison for small numbers
template <typename T, typename precision_t>
bool compare_close(const T& a, const T& b, const precision_t epsilon, precision_t zero_threshold)
{
  return ((zero_threshold > a && zero_threshold > b)) ||
         (a >= b * (1.0 - epsilon)) && (a <= b * (1.0 + epsilon));
}

// =============================================================================
// Test Suite
// =============================================================================
// Defines Betweenness Centrality UseCase
// SSSP's test suite code uses type of Graph parameter that could be used
// (MTX / RMAT)
typedef struct EdgeBC_Usecase_t {
  std::string config_;     // Path to graph file
  std::string file_path_;  // Complete path to graph using dataset_root_dir
  int number_of_sources_;  // Starting point from the traversal
  EdgeBC_Usecase_t(const std::string& config, int number_of_sources)
    : config_(config), number_of_sources_(number_of_sources)
  {
    // assume relative paths are relative to RAPIDS_DATASET_ROOT_DIR
    // FIXME: Use platform independent stuff from c++14/17 on compiler update
    const std::string& rapidsDatasetRootDir = cugraph::test::get_rapids_dataset_root_dir();
    if ((config_ != "") && (config_[0] != '/')) {
      file_path_ = rapidsDatasetRootDir + "/" + config_;
    } else {
      file_path_ = config_;
    }
  };
} EdgeBC_Usecase;

class Tests_EdgeBC : public ::testing::TestWithParam<EdgeBC_Usecase> {
  raft::handle_t handle;

 public:
  Tests_EdgeBC() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}
  // FIXME: Should normalize be part of the configuration instead?
  // vertex_t         vertex identifier data type
  // edge_t         edge identifier data type
  // weight_t         edge weight data type
  // result_t   result data type
  // normalize  should the result be normalized
  template <typename vertex_t,
            typename edge_t,
            typename weight_t,
            typename result_t,
            bool normalize>
  void run_current_test(const EdgeBC_Usecase& configuration)
  {
    // Step 1: Construction of the graph based on configuration
    bool is_directed = false;
    auto csr         = cugraph::test::generate_graph_csr_from_mm<vertex_t, edge_t, weight_t>(
      is_directed, configuration.file_path_);
    hipDeviceSynchronize();
    cugraph::legacy::GraphCSRView<vertex_t, edge_t, weight_t> G = csr->view();
    G.prop.directed                                             = is_directed;
    RAFT_CUDA_TRY(hipGetLastError());
    std::vector<result_t> result(G.number_of_edges, 0);
    std::vector<result_t> expected(G.number_of_edges, 0);

    // Step 2: Generation of sources based on configuration
    //         if number_of_sources_ is 0 then sources must be nullptr
    //         Otherwise we only  use the first k values
    ASSERT_TRUE(configuration.number_of_sources_ >= 0 &&
                configuration.number_of_sources_ <= G.number_of_vertices)
      << "Number number of sources should be >= 0 and"
      << " less than the number of vertices in the graph";
    std::vector<vertex_t> sources(configuration.number_of_sources_);
    thrust::sequence(thrust::host, sources.begin(), sources.end(), 0);

    vertex_t* sources_ptr = nullptr;
    if (configuration.number_of_sources_ > 0) { sources_ptr = sources.data(); }

    reference_edge_betweenness_centrality(
      G, expected.data(), normalize, configuration.number_of_sources_, sources_ptr);

    sources_ptr = nullptr;
    if (configuration.number_of_sources_ > 0) { sources_ptr = sources.data(); }

    rmm::device_vector<result_t> d_result(G.number_of_edges);
    cugraph::edge_betweenness_centrality(handle,
                                         G,
                                         d_result.data().get(),
                                         normalize,
                                         static_cast<weight_t*>(nullptr),
                                         configuration.number_of_sources_,
                                         sources_ptr);
    RAFT_CUDA_TRY(hipMemcpy(result.data(),
                             d_result.data().get(),
                             sizeof(result_t) * G.number_of_edges,
                             hipMemcpyDeviceToHost));
    for (int i = 0; i < G.number_of_edges; ++i)
      EXPECT_TRUE(compare_close(result[i], expected[i], TEST_EPSILON, TEST_ZERO_THRESHOLD))
        << "[MISMATCH] vaid = " << i << ", cugraph = " << result[i]
        << " expected = " << expected[i];
  }
};

// ============================================================================
// Tests
// ============================================================================
// Verifiy Un-Normalized results
TEST_P(Tests_EdgeBC, CheckFP32_NO_NORMALIZE)
{
  run_current_test<int, int, float, float, false>(GetParam());
}

#if 0
// Temporarily disable some of the test combinations
//  Full solution will be explored for issue #1555
TEST_P(Tests_EdgeBC, CheckFP64_NO_NORMALIZE)
{
  run_current_test<int, int, double, double, false>(GetParam());
}

// Verifiy Normalized results
TEST_P(Tests_EdgeBC, CheckFP32_NORMALIZE)
{
  run_current_test<int, int, float, float, true>(GetParam());
}
#endif

TEST_P(Tests_EdgeBC, CheckFP64_NORMALIZE)
{
  run_current_test<int, int, double, double, true>(GetParam());
}

#if 0
// Temporarily disable some of the test combinations
//  Full solution will be explored for issue #1555
INSTANTIATE_TEST_SUITE_P(simple_test,
                         Tests_EdgeBC,
                         ::testing::Values(EdgeBC_Usecase("test/datasets/karate.mtx", 0),
                                           EdgeBC_Usecase("test/datasets/netscience.mtx", 0),
                                           EdgeBC_Usecase("test/datasets/netscience.mtx", 4),
                                           EdgeBC_Usecase("test/datasets/wiki2003.mtx", 4),
                                           EdgeBC_Usecase("test/datasets/wiki-Talk.mtx", 4)));
#else
INSTANTIATE_TEST_SUITE_P(simple_test,
                         Tests_EdgeBC,
                         ::testing::Values(EdgeBC_Usecase("test/datasets/karate.mtx", 0),
                                           EdgeBC_Usecase("test/datasets/netscience.mtx", 0),
                                           EdgeBC_Usecase("test/datasets/netscience.mtx", 4)));
#endif

CUGRAPH_TEST_PROGRAM_MAIN()
