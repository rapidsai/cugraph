/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <traversal/legacy/bfs_ref.h>
#include <utilities/base_fixture.hpp>
#include <utilities/test_utilities.hpp>

#include <cugraph/algorithms.hpp>
#include <cugraph/legacy/graph.hpp>

#include <raft/error.hpp>
#include <raft/handle.hpp>
#include <rmm/device_vector.hpp>

#include <rmm/device_vector.hpp>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <fstream>
#include <queue>
#include <stack>
#include <utility>

#ifndef TEST_EPSILON
#define TEST_EPSILON 0.0001
#endif

// NOTE: Defines under which values the difference should  be discarded when
// considering values are close to zero
//  i.e: Do we consider that the difference between 1.3e-9 and 8.e-12 is
// significant
#ifndef TEST_ZERO_THRESHOLD
#define TEST_ZERO_THRESHOLD 1e-10
#endif

// ============================================================================
// C++ Reference Implementation
// ============================================================================
template <typename vertex_t, typename edge_t, typename weight_t, typename result_t>
void ref_accumulation(result_t* result,
                      vertex_t const number_of_vertices,
                      std::stack<vertex_t>& S,
                      std::vector<std::vector<vertex_t>>& pred,
                      std::vector<double>& sigmas,
                      std::vector<double>& deltas,
                      vertex_t source)
{
  for (vertex_t v = 0; v < number_of_vertices; ++v) {
    deltas[v] = 0;
  }
  while (!S.empty()) {
    vertex_t w = S.top();
    S.pop();
    for (vertex_t v : pred[w]) {
      deltas[v] += (sigmas[v] / sigmas[w]) * (1.0 + deltas[w]);
    }
    if (w != source) { result[w] += deltas[w]; }
  }
}

template <typename vertex_t, typename edge_t, typename weight_t, typename result_t>
void ref_endpoints_accumulation(result_t* result,
                                vertex_t const number_of_vertices,
                                std::stack<vertex_t>& S,
                                std::vector<std::vector<vertex_t>>& pred,
                                std::vector<double>& sigmas,
                                std::vector<double>& deltas,
                                vertex_t source)
{
  result[source] += S.size() - 1;
  for (vertex_t v = 0; v < number_of_vertices; ++v) {
    deltas[v] = 0;
  }
  while (!S.empty()) {
    vertex_t w = S.top();
    S.pop();
    for (vertex_t v : pred[w]) {
      deltas[v] += (sigmas[v] / sigmas[w]) * (1.0 + deltas[w]);
    }
    if (w != source) { result[w] += deltas[w] + 1; }
  }
}

template <typename vertex_t, typename edge_t, typename weight_t, typename result_t>
void ref_edge_accumulation(result_t* result,
                           vertex_t const number_of_vertices,
                           std::stack<vertex_t>& S,
                           std::vector<std::vector<vertex_t>>& pred,
                           std::vector<double>& sigmas,
                           std::vector<double>& deltas,
                           vertex_t source)
{
  for (vertex_t v = 0; v < number_of_vertices; ++v) {
    deltas[v] = 0;
  }
  while (!S.empty()) {
    vertex_t w = S.top();
    S.pop();
    for (vertex_t v : pred[w]) {
      deltas[v] += (sigmas[v] / sigmas[w]) * (1.0 + deltas[w]);
    }
    if (w != source) { result[w] += deltas[w]; }
  }
}

// Algorithm 1: Shortest-path vertex betweenness, (Brandes, 2001)
template <typename vertex_t, typename edge_t, typename weight_t, typename result_t>
void reference_betweenness_centrality_impl(vertex_t* indices,
                                           edge_t* offsets,
                                           vertex_t const number_of_vertices,
                                           result_t* result,
                                           bool endpoints,
                                           vertex_t const* sources,
                                           vertex_t const number_of_sources)
{
  std::queue<vertex_t> Q;
  std::stack<vertex_t> S;
  // NOTE: dist is of type vertex_t not weight_t
  std::vector<vertex_t> dist(number_of_vertices);
  std::vector<std::vector<vertex_t>> pred(number_of_vertices);
  std::vector<double> sigmas(number_of_vertices);
  std::vector<double> deltas(number_of_vertices);

  std::vector<vertex_t> neighbors;

  if (sources) {
    for (vertex_t source_idx = 0; source_idx < number_of_sources; ++source_idx) {
      vertex_t s = sources[source_idx];
      // Step 1: Single-source shortest-paths problem
      //   a. Initialization
      ref_bfs<vertex_t, edge_t>(indices, offsets, number_of_vertices, Q, S, dist, pred, sigmas, s);
      //  Step 2: Accumulation
      //          Back propagation of dependencies
      if (endpoints) {
        ref_endpoints_accumulation<vertex_t, edge_t, weight_t, result_t>(
          result, number_of_vertices, S, pred, sigmas, deltas, s);
      } else {
        ref_accumulation<vertex_t, edge_t, weight_t, result_t>(
          result, number_of_vertices, S, pred, sigmas, deltas, s);
      }
    }
  } else {
    for (vertex_t s = 0; s < number_of_vertices; ++s) {
      // Step 1: Single-source shortest-paths problem
      //   a. Initialization
      ref_bfs<vertex_t, edge_t>(indices, offsets, number_of_vertices, Q, S, dist, pred, sigmas, s);
      //  Step 2: Accumulation
      //          Back propagation of dependencies
      if (endpoints) {
        ref_endpoints_accumulation<vertex_t, edge_t, weight_t, result_t>(
          result, number_of_vertices, S, pred, sigmas, deltas, s);
      } else {
        ref_accumulation<vertex_t, edge_t, weight_t, result_t>(
          result, number_of_vertices, S, pred, sigmas, deltas, s);
      }
    }
  }
}

template <typename vertex_t, typename edge_t, typename weight_t, typename result_t>
void reference_rescale(result_t* result,
                       bool directed,
                       bool normalize,
                       bool endpoints,
                       vertex_t const number_of_vertices,
                       vertex_t const number_of_sources)
{
  bool modified                      = false;
  result_t rescale_factor            = static_cast<result_t>(1);
  result_t casted_number_of_sources  = static_cast<result_t>(number_of_sources);
  result_t casted_number_of_vertices = static_cast<result_t>(number_of_vertices);
  if (normalize) {
    if (number_of_vertices > 2) {
      if (endpoints) {
        rescale_factor /= (casted_number_of_vertices * (casted_number_of_vertices - 1));
      } else {
        rescale_factor /= ((casted_number_of_vertices - 1) * (casted_number_of_vertices - 2));
      }
      modified = true;
    }
  } else {
    if (!directed) {
      rescale_factor /= static_cast<result_t>(2);
      modified = true;
    }
  }
  if (modified) {
    if (number_of_sources > 0) {
      rescale_factor *= (casted_number_of_vertices / casted_number_of_sources);
    }
  }
  for (auto idx = 0; idx < number_of_vertices; ++idx) {
    result[idx] *= rescale_factor;
  }
}

template <typename vertex_t, typename edge_t, typename weight_t, typename result_t>
void reference_betweenness_centrality(
  cugraph::legacy::GraphCSRView<vertex_t, edge_t, weight_t> const& graph,
  result_t* result,
  bool normalize,
  bool endpoints,  // This is not yet implemented
  vertex_t const number_of_sources,
  vertex_t const* sources)
{
  vertex_t number_of_vertices = graph.number_of_vertices;
  edge_t number_of_edges      = graph.number_of_edges;
  thrust::host_vector<vertex_t> h_indices(number_of_edges);
  thrust::host_vector<edge_t> h_offsets(number_of_vertices + 1);

  thrust::device_ptr<vertex_t> d_indices((vertex_t*)&graph.indices[0]);
  thrust::device_ptr<edge_t> d_offsets((edge_t*)&graph.offsets[0]);

  thrust::copy(d_indices, d_indices + number_of_edges, h_indices.begin());
  thrust::copy(d_offsets, d_offsets + (number_of_vertices + 1), h_offsets.begin());

  hipDeviceSynchronize();

  reference_betweenness_centrality_impl<vertex_t, edge_t, weight_t, result_t>(&h_indices[0],
                                                                              &h_offsets[0],
                                                                              number_of_vertices,
                                                                              result,
                                                                              endpoints,
                                                                              sources,
                                                                              number_of_sources);
  reference_rescale<vertex_t, edge_t, weight_t, result_t>(
    result, graph.prop.directed, normalize, endpoints, number_of_vertices, number_of_sources);
}
// Explicit instantiation
/*    FIXME!!!
template void reference_betweenness_centrality<int, int, float, float>(
  cugraph::legacy::GraphCSRView<int, int, float> const &,
  float *,
  bool,
  bool,
  const int,
  int const *);
template void reference_betweenness_centrality<int, int, double, double>(
  cugraph::legacy::GraphCSRView<int, int, double> const &,
  double *,
  bool,
  bool,
  const int,
  int const *);
*/

// =============================================================================
// Utility functions
// =============================================================================
// Compare while allowing relatie error of epsilon
// zero_threshold indicates when  we should drop comparison for small numbers
template <typename T, typename precision_t>
bool compare_close(const T& a, const T& b, const precision_t epsilon, precision_t zero_threshold)
{
  return ((zero_threshold > a && zero_threshold > b)) ||
         (a >= b * (1.0 - epsilon)) && (a <= b * (1.0 + epsilon));
}

// =============================================================================
// Test Suite
// =============================================================================
// Defines Betweenness Centrality UseCase
// SSSP's test suite code uses type of Graph parameter that could be used
// (MTX / RMAT)
typedef struct BC_Usecase_t {
  std::string config_;     // Path to graph file
  std::string file_path_;  // Complete path to graph using dataset_root_dir
  int number_of_sources_;  // Starting point from the traversal
  BC_Usecase_t(const std::string& config, int number_of_sources)
    : config_(config), number_of_sources_(number_of_sources)
  {
    // assume relative paths are relative to RAPIDS_DATASET_ROOT_DIR
    // FIXME: Use platform independent stuff from c++14/17 on compiler update
    const std::string& rapidsDatasetRootDir = cugraph::test::get_rapids_dataset_root_dir();
    if ((config_ != "") && (config_[0] != '/')) {
      file_path_ = rapidsDatasetRootDir + "/" + config_;
    } else {
      file_path_ = config_;
    }
  };
} BC_Usecase;

class Tests_BC : public ::testing::TestWithParam<BC_Usecase> {
  raft::handle_t handle;

 public:
  Tests_BC() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}
  // vertex_t         vertex identifier data type
  // edge_t         edge identifier data type
  // weight_t         edge weight data type
  // result_t   result data type
  // normalize  should the result be normalized
  // endpoints  should the endpoints be included
  template <typename vertex_t,
            typename edge_t,
            typename weight_t,
            typename result_t,
            bool normalize,
            bool endpoints>
  void run_current_test(const BC_Usecase& configuration)
  {
    // Step 1: Construction of the graph based on configuration
    bool is_directed = false;
    auto csr         = cugraph::test::generate_graph_csr_from_mm<vertex_t, edge_t, weight_t>(
      is_directed, configuration.file_path_);
    hipDeviceSynchronize();
    cugraph::legacy::GraphCSRView<vertex_t, edge_t, weight_t> G = csr->view();
    G.prop.directed                                             = is_directed;
    RAFT_CUDA_TRY(hipGetLastError());
    std::vector<result_t> result(G.number_of_vertices, 0);
    std::vector<result_t> expected(G.number_of_vertices, 0);

    // Step 2: Generation of sources based on configuration
    //         if number_of_sources_ is 0 then sources must be nullptr
    //         Otherwise we only  use the first k values
    ASSERT_TRUE(configuration.number_of_sources_ >= 0 &&
                configuration.number_of_sources_ <= G.number_of_vertices)
      << "Number number of sources should be >= 0 and"
      << " less than the number of vertices in the graph";
    std::vector<vertex_t> sources(configuration.number_of_sources_);
    thrust::sequence(thrust::host, sources.begin(), sources.end(), 0);

    vertex_t* sources_ptr = nullptr;
    if (configuration.number_of_sources_ > 0) { sources_ptr = sources.data(); }

    reference_betweenness_centrality(
      G, expected.data(), normalize, endpoints, configuration.number_of_sources_, sources_ptr);

    sources_ptr = nullptr;
    if (configuration.number_of_sources_ > 0) { sources_ptr = sources.data(); }

    rmm::device_vector<result_t> d_result(G.number_of_vertices);
    cugraph::betweenness_centrality(handle,
                                    G,
                                    d_result.data().get(),
                                    normalize,
                                    endpoints,
                                    static_cast<weight_t*>(nullptr),
                                    configuration.number_of_sources_,
                                    sources_ptr);
    hipDeviceSynchronize();
    RAFT_CUDA_TRY(hipMemcpy(result.data(),
                             d_result.data().get(),
                             sizeof(result_t) * G.number_of_vertices,
                             hipMemcpyDeviceToHost));
    hipDeviceSynchronize();
    for (int i = 0; i < G.number_of_vertices; ++i)
      EXPECT_TRUE(compare_close(result[i], expected[i], TEST_EPSILON, TEST_ZERO_THRESHOLD))
        << "[MISMATCH] vaid = " << i << ", cugraph = " << result[i]
        << " expected = " << expected[i];
  }
};

// ============================================================================
// Tests
// ============================================================================
// Verifiy Un-Normalized results
TEST_P(Tests_BC, CheckFP32_NO_NORMALIZE_NO_ENDPOINTS)
{
  run_current_test<int, int, float, float, false, false>(GetParam());
}

#if 0
// Temporarily disable some of the test combinations
//  Full solution will be explored for issue #1555
TEST_P(Tests_BC, CheckFP64_NO_NORMALIZE_NO_ENDPOINTS)
{
  run_current_test<int, int, double, double, false, false>(GetParam());
}

TEST_P(Tests_BC, CheckFP32_NO_NORMALIZE_ENDPOINTS)
{
  run_current_test<int, int, float, float, false, true>(GetParam());
}
#endif

TEST_P(Tests_BC, CheckFP64_NO_NORMALIZE_ENDPOINTS)
{
  run_current_test<int, int, double, double, false, true>(GetParam());
}

// Verifiy Normalized results
TEST_P(Tests_BC, CheckFP32_NORMALIZE_NO_ENDPOINTS)
{
  run_current_test<int, int, float, float, true, false>(GetParam());
}

#if 0
// Temporarily disable some of the test combinations
//  Full solution will be explored for issue #1555
TEST_P(Tests_BC, CheckFP64_NORMALIZE_NO_ENDPOINTS)
{
  run_current_test<int, int, double, double, true, false>(GetParam());
}

TEST_P(Tests_BC, CheckFP32_NORMALIZE_ENDPOINTS)
{
  run_current_test<int, int, float, float, true, true>(GetParam());
}
#endif

TEST_P(Tests_BC, CheckFP64_NORMALIZE_ENDPOINTS)
{
  run_current_test<int, int, double, double, true, true>(GetParam());
}

#if 0
// Temporarily disable some of the test combinations
//  Full solution will be explored for issue #1555
INSTANTIATE_TEST_SUITE_P(simple_test,
                         Tests_BC,
                         ::testing::Values(BC_Usecase("test/datasets/karate.mtx", 0),
                                           BC_Usecase("test/datasets/netscience.mtx", 0),
                                           BC_Usecase("test/datasets/netscience.mtx", 4),
                                           BC_Usecase("test/datasets/wiki2003.mtx", 4),
                                           BC_Usecase("test/datasets/wiki-Talk.mtx", 4)));
#else
INSTANTIATE_TEST_SUITE_P(simple_test,
                         Tests_BC,
                         ::testing::Values(BC_Usecase("test/datasets/karate.mtx", 0),
                                           BC_Usecase("test/datasets/netscience.mtx", 0),
                                           BC_Usecase("test/datasets/netscience.mtx", 4)));
#endif

CUGRAPH_TEST_PROGRAM_MAIN()
