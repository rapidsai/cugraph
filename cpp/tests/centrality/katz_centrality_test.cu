#include <thrust/device_ptr.h>
#include <algorithms.hpp>
#include <converters/COOtoCSR.cuh>
#include <fstream>
#include <graph.hpp>
#include <rmm/mr/device/cuda_memory_resource.hpp>
#include "hip/hip_runtime_api.h"
#include "gmock/gmock-generated-matchers.h"
#include "gmock/gmock.h"
#include "gtest/gtest.h"
#include "utilities/high_res_clock.h"

#include "utilities/test_utilities.hpp"

std::vector<int> getGoldenTopKIds(std::ifstream& fs_result, int k = 10)
{
  std::vector<int> vec;
  int val;
  int count = 0;
  while (fs_result >> val && ((count++) < k)) { vec.push_back(val); }
  vec.resize(k);
  return vec;
}

std::vector<int> getTopKIds(double* p_katz, int count, int k = 10)
{
  hipStream_t stream = nullptr;
  rmm::device_vector<int> id(count);
  thrust::sequence(rmm::exec_policy(stream)->on(stream), id.begin(), id.end());
  thrust::sort_by_key(rmm::exec_policy(stream)->on(stream),
                      p_katz,
                      p_katz + count,
                      id.begin(),
                      thrust::greater<double>());
  std::vector<int> topK(k);
  thrust::copy(id.begin(), id.begin() + k, topK.begin());
  return topK;
}

template <typename VT, typename ET, typename WT>
int getMaxDegree(cugraph::experimental::GraphCSRView<VT, ET, WT> const& g)
{
  hipStream_t stream{nullptr};

  rmm::device_vector<ET> degree_vector(g.number_of_vertices);
  ET* p_degree = degree_vector.data().get();
  g.degree(p_degree, cugraph::experimental::DegreeDirection::OUT);
  ET max_out_degree = thrust::reduce(rmm::exec_policy(stream)->on(stream),
                                     p_degree,
                                     p_degree + g.number_of_vertices,
                                     static_cast<ET>(-1),
                                     thrust::maximum<ET>());
  return max_out_degree;
}

typedef struct Katz_Usecase_t {
  std::string matrix_file;
  std::string result_file;
  Katz_Usecase_t(const std::string& a, const std::string& b)
  {
    // assume relative paths are relative to RAPIDS_DATASET_ROOT_DIR
    const std::string& rapidsDatasetRootDir = cugraph::test::get_rapids_dataset_root_dir();
    if ((a != "") && (a[0] != '/')) {
      matrix_file = rapidsDatasetRootDir + "/" + a;
    } else {
      matrix_file = a;
    }
    if ((b != "") && (b[0] != '/')) {
      result_file = rapidsDatasetRootDir + "/" + b;
    } else {
      result_file = b;
    }
  }
  Katz_Usecase_t& operator=(const Katz_Usecase_t& rhs)
  {
    matrix_file = rhs.matrix_file;
    result_file = rhs.result_file;
    return *this;
  }
} Katz_Usecase;

class Tests_Katz : public ::testing::TestWithParam<Katz_Usecase> {
 public:
  Tests_Katz() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}
  virtual void SetUp() {}
  virtual void TearDown() {}

  void run_current_test(const Katz_Usecase& param)
  {
    FILE* fpin = fopen(param.matrix_file.c_str(), "r");
    ASSERT_NE(fpin, nullptr) << "fopen (" << param.matrix_file << ") failure.";

    std::ifstream fs_result(param.result_file);
    ASSERT_EQ(fs_result.is_open(), true) << "file open (" << param.result_file << ") failure.";

    int m, k;
    int nnz;
    MM_typecode mc;
    ASSERT_EQ(cugraph::test::mm_properties<int>(fpin, 1, &mc, &m, &k, &nnz), 0)
      << "could not read Matrix Market file properties"
      << "\n";
    ASSERT_TRUE(mm_is_matrix(mc));
    ASSERT_TRUE(mm_is_coordinate(mc));
    ASSERT_FALSE(mm_is_complex(mc));
    ASSERT_FALSE(mm_is_skew(mc));

    // Allocate memory on host
    std::vector<int> cooRowInd(nnz), cooColInd(nnz);
    std::vector<int> cooVal(nnz);
    std::vector<double> katz_centrality(m);

    // Read
    ASSERT_EQ((cugraph::test::mm_to_coo<int, int>(
                fpin, 1, nnz, &cooRowInd[0], &cooColInd[0], &cooVal[0], NULL)),
              0)
      << "could not read matrix data"
      << "\n";
    ASSERT_EQ(fclose(fpin), 0);

    cugraph::experimental::GraphCOOView<int, int, float> cooview(
      &cooColInd[0], &cooRowInd[0], nullptr, m, nnz);
    auto csr                                               = cugraph::coo_to_csr(cooview);
    cugraph::experimental::GraphCSRView<int, int, float> G = csr->view();

    rmm::device_vector<double> katz_vector(m);
    double* d_katz = thrust::raw_pointer_cast(katz_vector.data());

    int max_out_degree = getMaxDegree(G);
    double alpha       = 1 / (static_cast<double>(max_out_degree) + 1);

    cugraph::katz_centrality(G, d_katz, alpha, 100, 1e-6, false, true);

    std::vector<int> top10CUGraph = getTopKIds(d_katz, m);
    std::vector<int> top10Golden  = getGoldenTopKIds(fs_result);

    EXPECT_THAT(top10CUGraph, ::testing::ContainerEq(top10Golden));
  }
};

// --gtest_filter=*simple_test*
INSTANTIATE_TEST_CASE_P(
  simple_test,
  Tests_Katz,
  ::testing::Values(Katz_Usecase("test/datasets/karate.mtx", "ref/katz/karate.csv"),
                    Katz_Usecase("test/datasets/netscience.mtx", "ref/katz/netscience.csv"),
                    Katz_Usecase("test/datasets/polbooks.mtx", "ref/katz/polbooks.csv"),
                    Katz_Usecase("test/datasets/dolphins.mtx", "ref/katz/dolphins.csv")));

TEST_P(Tests_Katz, Check) { run_current_test(GetParam()); }

int main(int argc, char** argv)
{
  testing::InitGoogleTest(&argc, argv);
  auto resource = std::make_unique<rmm::mr::cuda_memory_resource>();
  rmm::mr::set_default_resource(resource.get());
  int rc = RUN_ALL_TESTS();
  return rc;
}
