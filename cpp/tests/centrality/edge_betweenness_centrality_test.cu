/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "gmock/gmock.h"
#include "gtest/gtest.h"

#include <thrust/device_vector.h>
#include <utility>
#include "utilities/test_utilities.hpp"

#include <algorithms.hpp>
#include <graph.hpp>

#include <queue>
#include <stack>

#include <fstream>

#include <rmm/mr/device/cuda_memory_resource.hpp>
#include "traversal/bfs_ref.h"

#ifndef TEST_EPSILON
#define TEST_EPSILON 0.0001
#endif

// NOTE: Defines under which values the difference should  be discarded when
// considering values are close to zero
//  i.e: Do we consider that the difference between 1.3e-9 and 8.e-12 is
// significant
#ifndef TEST_ZERO_THRESHOLD
#define TEST_ZERO_THRESHOLD 1e-10
#endif

// ============================================================================
// C++ Reference Implementation
// ============================================================================

template <typename VT, typename ET, typename WT, typename result_t>
ET get_edge_index_from_source_and_destination(VT source_vertex,
                                              VT destination_vertex,
                                              VT const *indices,
                                              ET const *offsets)
{
  ET index          = -1;
  ET first_edge_idx = offsets[source_vertex];
  ET last_edge_idx  = offsets[source_vertex + 1];
  auto index_it = std::find(indices + first_edge_idx, indices + last_edge_idx, destination_vertex);
  if (index_it != (indices + last_edge_idx)) { index = std::distance(indices, index_it); }
  return index;
}

template <typename VT, typename ET, typename WT, typename result_t>
void ref_accumulation(result_t *result,
                      VT const *indices,
                      ET const *offsets,
                      VT const number_of_vertices,
                      std::stack<VT> &S,
                      std::vector<std::vector<VT>> &pred,
                      std::vector<double> &sigmas,
                      std::vector<double> &deltas,
                      VT source)
{
  for (VT v = 0; v < number_of_vertices; ++v) { deltas[v] = 0; }
  while (!S.empty()) {
    VT w = S.top();
    S.pop();
    for (VT v : pred[w]) {
      ET edge_idx =
        get_edge_index_from_source_and_destination<VT, ET, WT, result_t>(v, w, indices, offsets);
      double coefficient = (sigmas[v] / sigmas[w]) * (1.0 + deltas[w]);

      deltas[v] += coefficient;
      result[edge_idx] += coefficient;
    }
  }
}

// Algorithm 1: Shortest-path vertex betweenness, (Brandes, 2001)
template <typename VT, typename ET, typename WT, typename result_t>
void reference_edge_betweenness_centrality_impl(VT *indices,
                                                ET *offsets,
                                                VT const number_of_vertices,
                                                result_t *result,
                                                VT const *sources,
                                                VT const number_of_sources)
{
  std::queue<VT> Q;
  std::stack<VT> S;
  // NOTE: dist is of type VT not WT
  std::vector<VT> dist(number_of_vertices);
  std::vector<std::vector<VT>> pred(number_of_vertices);
  std::vector<double> sigmas(number_of_vertices);
  std::vector<double> deltas(number_of_vertices);

  std::vector<VT> neighbors;

  if (sources) {
    for (VT source_idx = 0; source_idx < number_of_sources; ++source_idx) {
      VT s = sources[source_idx];
      // Step 1: Single-source shortest-paths problem
      //   a. Initialization
      ref_bfs<VT, ET>(indices, offsets, number_of_vertices, Q, S, dist, pred, sigmas, s);
      //  Step 2: Accumulation
      //          Back propagation of dependencies
      ref_accumulation<VT, ET, WT, result_t>(
        result, indices, offsets, number_of_vertices, S, pred, sigmas, deltas, s);
    }
  } else {
    for (VT s = 0; s < number_of_vertices; ++s) {
      // Step 1: Single-source shortest-paths problem
      //   a. Initialization
      ref_bfs<VT, ET>(indices, offsets, number_of_vertices, Q, S, dist, pred, sigmas, s);
      //  Step 2: Accumulation
      //          Back propagation of dependencies
      ref_accumulation<VT, ET, WT, result_t>(
        result, indices, offsets, number_of_vertices, S, pred, sigmas, deltas, s);
    }
  }
}

template <typename VT, typename ET, typename WT, typename result_t>
void reference_rescale(result_t *result,
                       bool directed,
                       bool normalize,
                       VT const number_of_vertices,
                       ET const number_of_edges)
{
  result_t rescale_factor            = static_cast<result_t>(1);
  result_t casted_number_of_vertices = static_cast<result_t>(number_of_vertices);
  if (normalize) {
    if (number_of_vertices > 1) {
      rescale_factor /= ((casted_number_of_vertices) * (casted_number_of_vertices - 1));
    }
  } else {
    if (!directed) { rescale_factor /= static_cast<result_t>(2); }
  }
  for (auto idx = 0; idx < number_of_edges; ++idx) { result[idx] *= rescale_factor; }
}

template <typename VT, typename ET, typename WT, typename result_t>
void reference_edge_betweenness_centrality(
  cugraph::experimental::GraphCSRView<VT, ET, WT> const &graph,
  result_t *result,
  bool normalize,
  VT const number_of_sources,
  VT const *sources)
{
  VT number_of_vertices = graph.number_of_vertices;
  ET number_of_edges    = graph.number_of_edges;
  thrust::host_vector<VT> h_indices(number_of_edges);
  thrust::host_vector<ET> h_offsets(number_of_vertices + 1);

  thrust::device_ptr<VT> d_indices((VT *)&graph.indices[0]);
  thrust::device_ptr<ET> d_offsets((ET *)&graph.offsets[0]);

  thrust::copy(d_indices, d_indices + number_of_edges, h_indices.begin());
  thrust::copy(d_offsets, d_offsets + (number_of_vertices + 1), h_offsets.begin());

  hipDeviceSynchronize();

  reference_edge_betweenness_centrality_impl<VT, ET, WT, result_t>(
    &h_indices[0], &h_offsets[0], number_of_vertices, result, sources, number_of_sources);
  reference_rescale<VT, ET, WT, result_t>(
    result, graph.prop.directed, normalize, number_of_vertices, number_of_edges);
}
// Explicit instantiation
template void reference_edge_betweenness_centrality<int, int, float, float>(
  cugraph::experimental::GraphCSRView<int, int, float> const &,
  float *,
  bool,
  const int,
  int const *);
template void reference_edge_betweenness_centrality<int, int, double, double>(
  cugraph::experimental::GraphCSRView<int, int, double> const &,
  double *,
  bool,
  const int,
  int const *);

// =============================================================================
// Utility functions
// =============================================================================
// Compare while allowing relatie error of epsilon
// zero_threshold indicates when  we should drop comparison for small numbers
template <typename T, typename precision_t>
bool compare_close(const T &a, const T &b, const precision_t epsilon, precision_t zero_threshold)
{
  return ((zero_threshold > a && zero_threshold > b)) ||
         (a >= b * (1.0 - epsilon)) && (a <= b * (1.0 + epsilon));
}

// =============================================================================
// Test Suite
// =============================================================================
// Defines Betweenness Centrality UseCase
// SSSP's test suite code uses type of Graph parameter that could be used
// (MTX / RMAT)
typedef struct EdgeBC_Usecase_t {
  std::string config_;     // Path to graph file
  std::string file_path_;  // Complete path to graph using dataset_root_dir
  int number_of_sources_;  // Starting point from the traversal
  EdgeBC_Usecase_t(const std::string &config, int number_of_sources)
    : config_(config), number_of_sources_(number_of_sources)
  {
    // assume relative paths are relative to RAPIDS_DATASET_ROOT_DIR
    // FIXME: Use platform independent stuff from c++14/17 on compiler update
    const std::string &rapidsDatasetRootDir = cugraph::test::get_rapids_dataset_root_dir();
    if ((config_ != "") && (config_[0] != '/')) {
      file_path_ = rapidsDatasetRootDir + "/" + config_;
    } else {
      file_path_ = config_;
    }
  };
} EdgeBC_Usecase;

class Tests_EdgeBC : public ::testing::TestWithParam<EdgeBC_Usecase> {
 public:
  Tests_EdgeBC() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}
  // FIXME: Should normalize be part of the configuration instead?
  // VT         vertex identifier data type
  // ET         edge identifier data type
  // WT         edge weight data type
  // result_t   result data type
  // normalize  should the result be normalized
  template <typename VT, typename ET, typename WT, typename result_t, bool normalize>
  void run_current_test(const EdgeBC_Usecase &configuration)
  {
    // Step 1: Construction of the graph based on configuration
    bool is_directed = false;
    auto csr =
      cugraph::test::generate_graph_csr_from_mm<VT, ET, WT>(is_directed, configuration.file_path_);
    hipDeviceSynchronize();
    cugraph::experimental::GraphCSRView<VT, ET, WT> G = csr->view();
    G.prop.directed                                   = is_directed;
    CUDA_RT_CALL(hipGetLastError());
    std::vector<result_t> result(G.number_of_edges, 0);
    std::vector<result_t> expected(G.number_of_edges, 0);

    // Step 2: Generation of sources based on configuration
    //         if number_of_sources_ is 0 then sources must be nullptr
    //         Otherwise we only  use the first k values
    ASSERT_TRUE(configuration.number_of_sources_ >= 0 &&
                configuration.number_of_sources_ <= G.number_of_vertices)
      << "Number number of sources should be >= 0 and"
      << " less than the number of vertices in the graph";
    std::vector<VT> sources(configuration.number_of_sources_);
    thrust::sequence(thrust::host, sources.begin(), sources.end(), 0);

    VT *sources_ptr = nullptr;
    if (configuration.number_of_sources_ > 0) { sources_ptr = sources.data(); }

    reference_edge_betweenness_centrality(
      G, expected.data(), normalize, configuration.number_of_sources_, sources_ptr);

    sources_ptr = nullptr;
    if (configuration.number_of_sources_ > 0) { sources_ptr = sources.data(); }

    thrust::device_vector<result_t> d_result(G.number_of_edges);
    cugraph::edge_betweenness_centrality(G,
                                         d_result.data().get(),
                                         normalize,
                                         static_cast<WT *>(nullptr),
                                         configuration.number_of_sources_,
                                         sources_ptr);
    CUDA_RT_CALL(hipMemcpy(result.data(),
                            d_result.data().get(),
                            sizeof(result_t) * G.number_of_edges,
                            hipMemcpyDeviceToHost));
    for (int i = 0; i < G.number_of_edges; ++i)
      EXPECT_TRUE(compare_close(result[i], expected[i], TEST_EPSILON, TEST_ZERO_THRESHOLD))
        << "[MISMATCH] vaid = " << i << ", cugraph = " << result[i]
        << " expected = " << expected[i];
  }
};

// ============================================================================
// Tests
// ============================================================================
// Verifiy Un-Normalized results
TEST_P(Tests_EdgeBC, CheckFP32_NO_NORMALIZE)
{
  run_current_test<int, int, float, float, false>(GetParam());
}

TEST_P(Tests_EdgeBC, CheckFP64_NO_NORMALIZE)
{
  run_current_test<int, int, double, double, false>(GetParam());
}

// Verifiy Normalized results
TEST_P(Tests_EdgeBC, CheckFP32_NORMALIZE)
{
  run_current_test<int, int, float, float, true>(GetParam());
}

TEST_P(Tests_EdgeBC, CheckFP64_NORMALIZE)
{
  run_current_test<int, int, double, double, true>(GetParam());
}

// FIXME: There is an InvalidValue on a Memcopy only on tests/datasets/dblp.mtx
INSTANTIATE_TEST_CASE_P(simple_test,
                        Tests_EdgeBC,
                        ::testing::Values(EdgeBC_Usecase("test/datasets/karate.mtx", 0),
                                          EdgeBC_Usecase("test/datasets/netscience.mtx", 0),
                                          EdgeBC_Usecase("test/datasets/netscience.mtx", 4),
                                          EdgeBC_Usecase("test/datasets/wiki2003.mtx", 4),
                                          EdgeBC_Usecase("test/datasets/wiki-Talk.mtx", 4)));

int main(int argc, char **argv)
{
  testing::InitGoogleTest(&argc, argv);
  auto resource = std::make_unique<rmm::mr::cuda_memory_resource>();
  rmm::mr::set_default_resource(resource.get());
  int rc = RUN_ALL_TESTS();
  return rc;
}
