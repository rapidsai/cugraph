/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "hip/hip_runtime_api.h"
#include "gtest/gtest.h"

#include <utilities/base_fixture.hpp>
#include <utilities/test_utilities.hpp>

#include <rmm/exec_policy.hpp>
#include <thrust/random.h>

#include <cugraph/algorithms.hpp>
#include <sampling/random_walks.cuh>

#include <raft/handle.hpp>
#include <raft/random/rng.cuh>

#include "random_walks_utils.cuh"

#include <algorithm>
#include <iterator>
#include <limits>
#include <numeric>
#include <tuple>
#include <utilities/high_res_timer.hpp>
#include <vector>

namespace {  // anonym.
template <typename vertex_t, typename index_t>
void fill_start(raft::handle_t const& handle,
                rmm::device_uvector<vertex_t>& d_start,
                index_t num_vertices)
{
  index_t num_paths = d_start.size();

  thrust::transform(handle.get_thrust_policy(),
                    thrust::make_counting_iterator<index_t>(0),
                    thrust::make_counting_iterator<index_t>(num_paths),

                    d_start.begin(),
                    [num_vertices] __device__(auto indx) { return indx % num_vertices; });
}
}  // namespace

namespace impl_details = cugraph::detail;

enum class traversal_id_t : int { HORIZONTAL = 0, VERTICAL };

struct RandomWalks_Usecase {
  std::string graph_file_full_path{};
  bool test_weighted{false};

  RandomWalks_Usecase(std::string const& graph_file_path, bool test_weighted)
    : test_weighted(test_weighted)
  {
    if ((graph_file_path.length() > 0) && (graph_file_path[0] != '/')) {
      graph_file_full_path = cugraph::test::get_rapids_dataset_root_dir() + "/" + graph_file_path;
    } else {
      graph_file_full_path = graph_file_path;
    }
  };
};

class Tests_RandomWalks
  : public ::testing::TestWithParam<std::tuple<traversal_id_t, int, RandomWalks_Usecase>> {
 public:
  Tests_RandomWalks() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}

  template <typename vertex_t, typename edge_t, typename weight_t>
  void run_current_test(std::tuple<traversal_id_t, int, RandomWalks_Usecase> const& configuration)
  {
    raft::handle_t handle{};

    // debuf info:
    //
    // std::cout << "read graph file: " << configuration.graph_file_full_path << std::endl;

    traversal_id_t trv_id = std::get<0>(configuration);
    int sampling_id       = std::get<1>(configuration);
    auto const& target    = std::get<2>(configuration);

    cugraph::graph_t<vertex_t, edge_t, weight_t, false, false> graph(handle);
    std::tie(graph, std::ignore) =
      cugraph::test::read_graph_from_matrix_market_file<vertex_t, edge_t, weight_t, false, false>(
        handle, target.graph_file_full_path, target.test_weighted, false);

    auto graph_view = graph.view();

    // call random_walks:
    start_random_walks(handle, graph_view, trv_id, sampling_id);
  }

  template <typename graph_vt>
  void start_random_walks(raft::handle_t const& handle,
                          graph_vt const& graph_view,
                          traversal_id_t trv_id,
                          int sampling_id)
  {
    using vertex_t = typename graph_vt::vertex_type;
    using edge_t   = typename graph_vt::edge_type;
    using weight_t = typename graph_vt::weight_type;
    using real_t   = float;

    edge_t num_paths = 10;
    rmm::device_uvector<vertex_t> d_start(num_paths, handle.get_stream());

    vertex_t num_vertices = graph_view.number_of_vertices();
    fill_start(handle, d_start, num_vertices);

    // 0-copy const device view:
    //
    impl_details::device_const_vector_view<vertex_t, edge_t> d_start_view{d_start.data(),
                                                                          num_paths};

    edge_t max_depth{10};

    weight_t p{4};
    weight_t q{8};

    if (trv_id == traversal_id_t::HORIZONTAL) {
      // `node2vec` without alpha buffer:
      //
      if (sampling_id == 2) {
        auto ret_tuple = cugraph::random_walks(
          handle,
          graph_view,
          d_start_view.begin(),
          num_paths,
          max_depth,
          false,
          std::make_unique<cugraph::sampling_params_t>(sampling_id, p, q, false));

        // check results:
        //
        bool test_all_paths = cugraph::test::host_check_rw_paths(handle,
                                                                 graph_view,
                                                                 std::get<0>(ret_tuple),
                                                                 std::get<1>(ret_tuple),
                                                                 std::get<2>(ret_tuple));

        ASSERT_TRUE(test_all_paths);
      }

      // the alpha buffer case should also be tested for `node2vec`
      // and for the others is irrelevant, so this block is necessary
      // for any sampling method:
      //
      {
        auto ret_tuple = cugraph::random_walks(
          handle,
          graph_view,
          d_start_view.begin(),
          num_paths,
          max_depth,
          false,
          std::make_unique<cugraph::sampling_params_t>(sampling_id, p, q, true));

        // check results:
        //
        bool test_all_paths = cugraph::test::host_check_rw_paths(handle,
                                                                 graph_view,
                                                                 std::get<0>(ret_tuple),
                                                                 std::get<1>(ret_tuple),
                                                                 std::get<2>(ret_tuple));

        ASSERT_TRUE(test_all_paths);
      }
    } else {  // VERTICAL: needs to be force-called via detail
      if (sampling_id == 0) {
        impl_details::uniform_selector_t<graph_vt, real_t> selector{handle, graph_view, real_t{0}};

        auto ret_tuple = impl_details::random_walks_impl<graph_vt,
                                                         decltype(selector),
                                                         impl_details::vertical_traversal_t>(
          handle,  // required to prevent clang-format to separate functin name from its namespace
          graph_view,
          d_start_view,
          max_depth,
          selector);

        // check results:
        //
        bool test_all_paths = cugraph::test::host_check_rw_paths(handle,
                                                                 graph_view,
                                                                 std::get<0>(ret_tuple),
                                                                 std::get<1>(ret_tuple),
                                                                 std::get<2>(ret_tuple));

        if (!test_all_paths)
          std::cout << "starting seed on failure: " << std::get<3>(ret_tuple) << '\n';

        ASSERT_TRUE(test_all_paths);
      } else if (sampling_id == 1) {
        impl_details::biased_selector_t<graph_vt, real_t> selector{handle, graph_view, real_t{0}};

        auto ret_tuple = impl_details::random_walks_impl<graph_vt,
                                                         decltype(selector),
                                                         impl_details::vertical_traversal_t>(
          handle,  // required to prevent clang-format to separate functin name from its namespace
          graph_view,
          d_start_view,
          max_depth,
          selector);

        // check results:
        //
        bool test_all_paths = cugraph::test::host_check_rw_paths(handle,
                                                                 graph_view,
                                                                 std::get<0>(ret_tuple),
                                                                 std::get<1>(ret_tuple),
                                                                 std::get<2>(ret_tuple));

        if (!test_all_paths)
          std::cout << "starting seed on failure: " << std::get<3>(ret_tuple) << '\n';

        ASSERT_TRUE(test_all_paths);
      } else {
        impl_details::node2vec_selector_t<graph_vt, real_t> selector{
          handle, graph_view, real_t{0}, p, q, num_paths};

        auto ret_tuple = impl_details::random_walks_impl<graph_vt,
                                                         decltype(selector),
                                                         impl_details::vertical_traversal_t>(
          handle,  // required to prevent clang-format to separate functin name from its namespace
          graph_view,
          d_start_view,
          max_depth,
          selector);

        // check results:
        //
        bool test_all_paths = cugraph::test::host_check_rw_paths(handle,
                                                                 graph_view,
                                                                 std::get<0>(ret_tuple),
                                                                 std::get<1>(ret_tuple),
                                                                 std::get<2>(ret_tuple));

        if (!test_all_paths)
          std::cout << "starting seed on failure: " << std::get<3>(ret_tuple) << '\n';

        ASSERT_TRUE(test_all_paths);
      }
    }
  }
};

TEST_P(Tests_RandomWalks, Initialize_i32_i32_f)
{
  run_current_test<int32_t, int32_t, float>(GetParam());
}

INSTANTIATE_TEST_SUITE_P(
  simple_test,
  Tests_RandomWalks,
  ::testing::Combine(::testing::Values(traversal_id_t::HORIZONTAL, traversal_id_t::VERTICAL),
                     ::testing::Values(int{0}, int{1}, int{2}),
                     ::testing::Values(RandomWalks_Usecase("test/datasets/karate.mtx", true),
                                       RandomWalks_Usecase("test/datasets/web-Google.mtx", true),
                                       RandomWalks_Usecase("test/datasets/ljournal-2008.mtx", true),
                                       RandomWalks_Usecase("test/datasets/webbase-1M.mtx", true))));

CUGRAPH_TEST_PROGRAM_MAIN()
