#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "nbr_sampling_utils.cuh"
#include <gtest/gtest.h>

struct Prims_Usecase {
  bool check_correctness{true};
};

template <typename input_usecase_t>
class Tests_MG_GatherEdges
  : public ::testing::TestWithParam<std::tuple<Prims_Usecase, input_usecase_t>> {
 public:
  Tests_MG_GatherEdges() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}

  template <typename vertex_t, typename edge_t, typename weight_t>
  void run_current_test(Prims_Usecase const& prims_usecase, input_usecase_t const& input_usecase)
  {
    using namespace cugraph::test;
    // 1. initialize handle

    raft::handle_t handle{};
    HighResClock hr_clock{};

    raft::comms::initialize_mpi_comms(&handle, MPI_COMM_WORLD);
    auto& comm           = handle.get_comms();
    auto const comm_size = comm.get_size();
    auto const comm_rank = comm.get_rank();

    auto row_comm_size = static_cast<int>(sqrt(static_cast<double>(comm_size)));
    while (comm_size % row_comm_size != 0) {
      --row_comm_size;
    }
    cugraph::partition_2d::subcomm_factory_t<cugraph::partition_2d::key_naming_t, vertex_t>
      subcomm_factory(handle, row_comm_size);

    // 2. create MG graph

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      hr_clock.start();
    }

    constexpr bool sort_adjacency_list = true;

    auto [mg_graph, mg_renumber_map_labels] =
      cugraph::test::construct_graph<vertex_t, edge_t, weight_t, false, true>(
        handle, input_usecase, true, true, false, sort_adjacency_list);

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      double elapsed_time{0.0};
      hr_clock.stop(&elapsed_time);
      std::cout << "MG construct_graph took " << elapsed_time * 1e-6 << " s.\n";
    }

    auto mg_graph_view                        = mg_graph.view();
    constexpr edge_t indices_per_source       = 2;
    constexpr vertex_t repetitions_per_vertex = 5;
    constexpr vertex_t source_sample_count    = 3;

    // 3. Gather mnmg call
    // Generate random vertex ids in the range of current gpu

    auto [global_degree_offsets, global_out_degrees] =
      cugraph::detail::get_global_degree_information(handle, mg_graph_view);
    auto global_adjacency_list_offsets = cugraph::detail::get_global_adjacency_offset(
      handle, mg_graph_view, global_degree_offsets, global_out_degrees);

    // Generate random sources to gather on
    auto random_sources = random_vertex_ids(handle,
                                            mg_graph_view.local_vertex_partition_range_first(),
                                            mg_graph_view.local_vertex_partition_range_last(),
                                            source_sample_count,
                                            repetitions_per_vertex);
    rmm::device_uvector<int> random_source_gpu_ids(random_sources.size(), handle.get_stream());
    thrust::fill(handle.get_thrust_policy(),
                 random_source_gpu_ids.begin(),
                 random_source_gpu_ids.end(),
                 comm_rank);

    auto [active_sources, active_source_gpu_ids] =
      cugraph::detail::gather_active_majors(handle,
                                            mg_graph_view,
                                            random_sources.cbegin(),
                                            random_sources.cend(),
                                            random_source_gpu_ids.cbegin());

    // get source global out degrees to generate indices
    auto active_source_degrees = cugraph::detail::get_active_major_global_degrees(
      handle, mg_graph_view, active_sources, global_out_degrees);

    auto random_destination_indices =
      generate_random_destination_indices(handle,
                                          active_source_degrees,
                                          mg_graph_view.number_of_vertices(),
                                          mg_graph_view.number_of_edges(),
                                          indices_per_source);
    rmm::device_uvector<edge_t> input_destination_indices(random_destination_indices.size(),
                                                          handle.get_stream());
    raft::update_device(input_destination_indices.data(),
                        random_destination_indices.data(),
                        random_destination_indices.size(),
                        handle.get_stream());

    auto [src, dst, gpu_ids, dst_map] =
      cugraph::detail::gather_local_edges(handle,
                                          mg_graph_view,
                                          active_sources,
                                          active_source_gpu_ids,
                                          std::move(input_destination_indices),
                                          indices_per_source,
                                          global_degree_offsets,
                                          global_adjacency_list_offsets);

    if (prims_usecase.check_correctness) {
      // Gather outputs
      auto mg_out_srcs = cugraph::test::device_gatherv(handle, src.data(), src.size());
      auto mg_out_dsts = cugraph::test::device_gatherv(handle, dst.data(), dst.size());

      // Gather inputs
      auto& col_comm      = handle.get_subcomm(cugraph::partition_2d::key_naming_t().col_name());
      auto const col_rank = col_comm.get_rank();
      auto sg_random_srcs = cugraph::test::device_gatherv(
        handle, active_sources.data(), col_rank == 0 ? active_sources.size() : 0);
      auto sg_random_dst_indices =
        cugraph::test::device_gatherv(handle,
                                      random_destination_indices.data(),
                                      col_rank == 0 ? random_destination_indices.size() : 0);

      // Gather input graph edgelist
      rmm::device_uvector<vertex_t> sg_src(0, handle.get_stream());
      rmm::device_uvector<vertex_t> sg_dst(0, handle.get_stream());
      std::tie(sg_src, sg_dst, std::ignore) =
        mg_graph_view.decompress_to_edgelist(handle, std::nullopt);

      auto aggregated_sg_src = cugraph::test::device_gatherv(handle, sg_src.begin(), sg_src.size());
      auto aggregated_sg_dst = cugraph::test::device_gatherv(handle, sg_dst.begin(), sg_dst.size());

      sort_coo(handle, mg_out_srcs, mg_out_dsts);

      if (handle.get_comms().get_rank() == int{0}) {
        cugraph::graph_t<vertex_t, edge_t, weight_t, false, false> sg_graph(handle);
        auto aggregated_edge_iter = thrust::make_zip_iterator(
          thrust::make_tuple(aggregated_sg_src.begin(), aggregated_sg_dst.begin()));
        thrust::sort(handle.get_thrust_policy(),
                     aggregated_edge_iter,
                     aggregated_edge_iter + aggregated_sg_src.size());
        auto sg_graph_properties =
          cugraph::graph_properties_t{mg_graph_view.is_symmetric(), mg_graph_view.is_multigraph()};

        std::tie(sg_graph, std::ignore) =
          cugraph::create_graph_from_edgelist<vertex_t, edge_t, weight_t, false, false>(
            handle,
            std::nullopt,
            std::move(aggregated_sg_src),
            std::move(aggregated_sg_dst),
            std::nullopt,
            sg_graph_properties,
            false);
        auto sg_graph_view = sg_graph.view();
        // Call single gpu gather
        auto [sg_out_srcs, sg_out_dsts] = sg_gather_edges(handle,
                                                          sg_graph_view,
                                                          sg_random_srcs.begin(),
                                                          sg_random_srcs.end(),
                                                          sg_random_dst_indices.begin(),
                                                          sg_graph_view.number_of_vertices(),
                                                          indices_per_source);
        sort_coo(handle, sg_out_srcs, sg_out_dsts);

        auto passed = thrust::equal(
          handle.get_thrust_policy(), sg_out_srcs.begin(), sg_out_srcs.end(), mg_out_srcs.begin());
        passed &= thrust::equal(
          handle.get_thrust_policy(), sg_out_dsts.begin(), sg_out_dsts.end(), mg_out_dsts.begin());
        ASSERT_TRUE(passed);
      }
    }
  }
};

using Tests_MG_GatherEdges_File = Tests_MG_GatherEdges<cugraph::test::File_Usecase>;

using Tests_MG_GatherEdges_Rmat = Tests_MG_GatherEdges<cugraph::test::Rmat_Usecase>;

TEST_P(Tests_MG_GatherEdges_File, CheckInt32Int32Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MG_GatherEdges_File, CheckInt32Int64Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MG_GatherEdges_File, CheckInt64Int64Float)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MG_GatherEdges_Rmat, CheckInt32Int32Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MG_GatherEdges_Rmat, CheckInt32Int64Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MG_GatherEdges_Rmat, CheckInt64Int64Float)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

INSTANTIATE_TEST_SUITE_P(
  file_test,
  Tests_MG_GatherEdges_File,
  ::testing::Combine(
    ::testing::Values(Prims_Usecase{true}),
    ::testing::Values(cugraph::test::File_Usecase("test/datasets/karate.mtx"),
                      cugraph::test::File_Usecase("test/datasets/web-Google.mtx"),
                      cugraph::test::File_Usecase("test/datasets/ljournal-2008.mtx"),
                      cugraph::test::File_Usecase("test/datasets/webbase-1M.mtx"))));

INSTANTIATE_TEST_SUITE_P(
  rmat_small_test,
  Tests_MG_GatherEdges_Rmat,
  ::testing::Combine(::testing::Values(Prims_Usecase{false}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       10, 16, 0.57, 0.19, 0.19, 0, false, false, 0, true))));

INSTANTIATE_TEST_SUITE_P(
  rmat_benchmark_test, /* note that scale & edge factor can be overridden in benchmarking (with
                          --gtest_filter to select only the rmat_benchmark_test with a specific
                          vertex & edge type combination) by command line arguments and do not
                          include more than one Rmat_Usecase that differ only in scale or edge
                          factor (to avoid running same benchmarks more than once) */
  Tests_MG_GatherEdges_Rmat,
  ::testing::Combine(::testing::Values(Prims_Usecase{false}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       20, 32, 0.57, 0.19, 0.19, 0, false, false, 0, true))));

CUGRAPH_MG_TEST_PROGRAM_MAIN()
