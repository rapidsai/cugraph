#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "nbr_sampling_utils.cuh"
#include <sampling/nbr_sampling_impl.cuh>

#include <gtest/gtest.h>

struct Prims_Usecase {
  bool check_correctness{true};
  bool flag_replacement{true};
};

template <typename input_usecase_t>
class Tests_MG_Nbr_Sampling
  : public ::testing::TestWithParam<std::tuple<Prims_Usecase, input_usecase_t>> {
 public:
  Tests_MG_Nbr_Sampling() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}

  template <typename vertex_t, typename edge_t, typename weight_t>
  void run_current_test(Prims_Usecase const& prims_usecase, input_usecase_t const& input_usecase)
  {
    using namespace cugraph::test;
    using gpu_t = int;

    // 1. initialize handle

    raft::handle_t handle{};
    HighResClock hr_clock{};

    raft::comms::initialize_mpi_comms(&handle, MPI_COMM_WORLD);
    auto& comm           = handle.get_comms();
    auto const comm_size = comm.get_size();
    auto const comm_rank = comm.get_rank();

    auto row_comm_size = static_cast<int>(sqrt(static_cast<double>(comm_size)));
    while (comm_size % row_comm_size != 0) {
      --row_comm_size;
    }
    cugraph::partition_2d::subcomm_factory_t<cugraph::partition_2d::key_naming_t, vertex_t>
      subcomm_factory(handle, row_comm_size);

    // 2. create MG graph

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      hr_clock.start();
    }

    constexpr bool sort_adjacency_list = true;

    auto [mg_graph, mg_renumber_map_labels] =
      cugraph::test::construct_graph<vertex_t, edge_t, weight_t, false, true>(
        handle, input_usecase, true, true, false, sort_adjacency_list);

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      double elapsed_time{0.0};
      hr_clock.stop(&elapsed_time);
      std::cout << "MG construct_graph took " << elapsed_time * 1e-6 << " s.\n";
    }

    auto mg_graph_view                        = mg_graph.view();
    constexpr edge_t indices_per_source       = 2;
    constexpr vertex_t repetitions_per_vertex = 5;
    constexpr vertex_t source_sample_count    = 3;

    // Generate random vertex ids in the range of current gpu

    // Generate random sources to gather on
    auto random_sources = random_vertex_ids(handle,
                                            mg_graph_view.local_vertex_partition_range_first(),
                                            mg_graph_view.local_vertex_partition_range_last(),
                                            source_sample_count,
                                            repetitions_per_vertex);
    rmm::device_uvector<gpu_t> random_source_gpu_ids(random_sources.size(), handle.get_stream());
    thrust::fill(handle.get_thrust_policy(),
                 random_source_gpu_ids.begin(),
                 random_source_gpu_ids.end(),
                 comm_rank);

    std::vector<int> h_fan_out{indices_per_source};  // depth = 1

    auto begin_in_pairs = thrust::make_zip_iterator(
      thrust::make_tuple(random_sources.begin(), random_source_gpu_ids.begin()));
    auto end_in_pairs = thrust::make_zip_iterator(
      thrust::make_tuple(random_sources.end(), random_source_gpu_ids.end()));

    // gather input:
    //
    auto&& [tuple_vertex_ranks, counts] = cugraph::detail::shuffle_to_gpus(
      handle, mg_graph_view, begin_in_pairs, end_in_pairs, gpu_t{});

    auto&& [tuple_quad, v_sizes] = cugraph::uniform_nbr_sample(handle,
                                                               mg_graph_view,
                                                               random_sources.begin(),
                                                               random_source_gpu_ids.begin(),
                                                               random_sources.size(),
                                                               h_fan_out,
                                                               prims_usecase.flag_replacement);

    auto&& d_src_out = std::get<0>(tuple_quad);
    auto&& d_dst_out = std::get<1>(tuple_quad);
    auto&& d_gpu_ids = std::get<2>(tuple_quad);

    if (prims_usecase.check_correctness) {
      auto self_rank = handle.get_comms().get_rank();

      // bring inputs and outputs on one rank
      // and check if test passed:
      //
      if (self_rank == gpu_t{0}) {
        auto num_ranks = v_sizes.size();
        ASSERT_TRUE(counts.size() == num_ranks);  // == #ranks

        // CAVEAT: in size << out_size;
        //
        auto total_in_sizes  = std::accumulate(counts.begin(), counts.end(), 0);
        auto total_out_sizes = std::accumulate(v_sizes.begin(), v_sizes.end(), 0);

        // merge inputs / outputs to be checked on host:
        //
        std::vector<vertex_t> h_start_in{};
        h_start_in.reserve(total_in_sizes);

        std::vector<gpu_t> h_ranks_in{};
        h_ranks_in.reserve(total_in_sizes);

        std::vector<vertex_t> h_src_out{};
        h_src_out.reserve(total_out_sizes);

        std::vector<vertex_t> h_dst_out{};
        h_dst_out.reserve(total_out_sizes);

        std::vector<gpu_t> h_ranks_out{};
        h_ranks_out.reserve(total_out_sizes);

        auto filler = [&handle](auto const& coalesced_in,
                                auto& accumulator,
                                auto& v_per_rank,
                                auto count,
                                auto offset) {
          auto start_offset_in = coalesced_in.cbegin() + offset;

          raft::update_host(
            v_per_rank.data(), start_offset_in, static_cast<size_t>(count), handle.get_stream());

          accumulator.insert(accumulator.begin() + offset, v_per_rank.begin(), v_per_rank.end());
        };

        size_t in_offset  = 0;
        size_t out_offset = 0;
        for (size_t index_rank = 0; index_rank < num_ranks; ++index_rank) {
          auto in_sz = counts[index_rank];
          std::vector<vertex_t> per_rank_start_in(in_sz);
          std::vector<gpu_t> per_rank_in(in_sz);

          filler(std::get<0>(tuple_vertex_ranks), h_start_in, per_rank_start_in, in_sz, in_offset);

          filler(std::get<1>(tuple_vertex_ranks), h_ranks_in, per_rank_in, in_sz, in_offset);

          auto out_sz = v_sizes[index_rank];
          std::vector<vertex_t> per_rank_src_out(out_sz);
          std::vector<vertex_t> per_rank_dst_out(out_sz);
          std::vector<gpu_t> per_rank_out(out_sz);

          filler(d_src_out, h_src_out, per_rank_src_out, out_sz, out_offset);
          filler(d_dst_out, h_dst_out, per_rank_dst_out, out_sz, out_offset);
          filler(d_gpu_ids, h_ranks_out, per_rank_out, out_sz, out_offset);

          in_offset += in_sz;
          out_offset += out_sz;
        }

        bool passed = cugraph::test::check_forest_trees_by_rank(
          h_start_in, h_ranks_in, h_src_out, h_dst_out, h_ranks_out);

        ASSERT_TRUE(passed);
      }
    }
  }
};

using Tests_MG_Nbr_Sampling_File = Tests_MG_Nbr_Sampling<cugraph::test::File_Usecase>;

using Tests_MG_Nbr_Sampling_Rmat = Tests_MG_Nbr_Sampling<cugraph::test::Rmat_Usecase>;

TEST_P(Tests_MG_Nbr_Sampling_File, CheckInt32Int32Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MG_Nbr_Sampling_File, CheckInt32Int64Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MG_Nbr_Sampling_File, CheckInt64Int64Float)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MG_Nbr_Sampling_Rmat, CheckInt32Int32Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MG_Nbr_Sampling_Rmat, CheckInt32Int64Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MG_Nbr_Sampling_Rmat, CheckInt64Int64Float)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

INSTANTIATE_TEST_SUITE_P(
  file_test,
  Tests_MG_Nbr_Sampling_File,
  ::testing::Combine(
    ::testing::Values(Prims_Usecase{true, true}, Prims_Usecase{true, false}),
    ::testing::Values(cugraph::test::File_Usecase("test/datasets/karate.mtx"),
                      cugraph::test::File_Usecase("test/datasets/web-Google.mtx"),
                      cugraph::test::File_Usecase("test/datasets/ljournal-2008.mtx"),
                      cugraph::test::File_Usecase("test/datasets/webbase-1M.mtx"))));

INSTANTIATE_TEST_SUITE_P(
  rmat_small_test,
  Tests_MG_Nbr_Sampling_Rmat,
  ::testing::Combine(::testing::Values(Prims_Usecase{false, true}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       10, 16, 0.57, 0.19, 0.19, 0, false, false, 0, true))));

INSTANTIATE_TEST_SUITE_P(
  rmat_benchmark_test, /* note that scale & edge factor can be overridden in benchmarking (with
                          --gtest_filter to select only the rmat_benchmark_test with a specific
                          vertex & edge type combination) by command line arguments and do not
                          include more than one Rmat_Usecase that differ only in scale or edge
                          factor (to avoid running same benchmarks more than once) */
  Tests_MG_Nbr_Sampling_Rmat,
  ::testing::Combine(::testing::Values(Prims_Usecase{false, true}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       20, 32, 0.57, 0.19, 0.19, 0, false, false, 0, true))));

CUGRAPH_MG_TEST_PROGRAM_MAIN()
