/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "detail/nbr_sampling_utils.cuh"

#include <utilities/mg_utilities.hpp>

#include <thrust/distance.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

#include <gtest/gtest.h>

struct Uniform_Neighbor_Sampling_Usecase {
  std::vector<int32_t> fanout{{-1}};
  int32_t batch_size{10};
  bool check_correctness{true};
  bool flag_replacement{true};
};

template <typename input_usecase_t>
class Tests_MGUniform_Neighbor_Sampling
  : public ::testing::TestWithParam<
      std::tuple<Uniform_Neighbor_Sampling_Usecase, input_usecase_t>> {
 public:
  Tests_MGUniform_Neighbor_Sampling() {}

  static void SetUpTestCase() { handle_ = cugraph::test::initialize_mg_handle(); }

  static void TearDownTestCase() { handle_.reset(); }

  virtual void SetUp() {}
  virtual void TearDown() {}

  template <typename vertex_t, typename edge_t, typename weight_t>
  void run_current_test(Uniform_Neighbor_Sampling_Usecase const& uniform_neighbor_sampling_usecase,
                        input_usecase_t const& input_usecase)
  {
    HighResTimer hr_timer{};

    // 1. create MG graph

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle_->get_comms().barrier();
      hr_timer.start("MG construct graph");
    }

    constexpr bool sort_adjacency_list = true;

    auto [mg_graph, mg_edge_weights, mg_renumber_map_labels] =
      cugraph::test::construct_graph<vertex_t, edge_t, weight_t, false, true>(
        *handle_, input_usecase, true, true, false, sort_adjacency_list);

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle_->get_comms().barrier();
      hr_timer.stop();
      hr_timer.display_and_clear(std::cout);
    }

    auto mg_graph_view = mg_graph.view();
    auto mg_edge_weight_view =
      mg_edge_weights ? std::make_optional((*mg_edge_weights).view()) : std::nullopt;

    //
    // Test is designed like GNN sampling.  We'll select 90% of vertices
    // to be included in sampling batches
    //
    constexpr float select_probability{0.9};

    // FIXME:  Update the tests to initialize RngState and use it instead
    //         of seed...
    uint64_t seed{static_cast<uint64_t>(handle_->get_comms().get_rank())};

    raft::random::RngState rng_state(seed);

    rmm::device_uvector<float> random_numbers(mg_graph_view.local_vertex_partition_range_size(),
                                              handle_->get_stream());
    rmm::device_uvector<vertex_t> random_sources(mg_graph_view.local_vertex_partition_range_size(),
                                                 handle_->get_stream());

    cugraph::detail::uniform_random_fill(handle_->get_stream(),
                                         random_numbers.data(),
                                         random_numbers.size(),
                                         float{0},
                                         float{1},
                                         seed);

    auto random_sources_end = thrust::copy_if(
      handle_->get_thrust_policy(),
      thrust::make_counting_iterator(vertex_t{0}),
      thrust::make_counting_iterator(mg_graph_view.local_vertex_partition_range_size()),
      random_sources.begin(),
      [d_random_number = random_numbers.data(), select_probability] __device__(vertex_t offset) {
        return d_random_number[offset] < select_probability;
      });

    random_sources.resize(thrust::distance(random_sources.begin(), random_sources_end),
                          handle_->get_stream());
    random_sources.shrink_to_fit(handle_->get_stream());

    random_numbers.resize(random_sources.size(), handle_->get_stream());
    random_numbers.shrink_to_fit(handle_->get_stream());

    if (mg_graph_view.local_vertex_partition_range_first() > 0)
      thrust::transform(
        handle_->get_thrust_policy(),
        random_sources.begin(),
        random_sources.end(),
        random_sources.begin(),
        [base_offset = mg_graph_view.local_vertex_partition_range_first()] __device__(vertex_t v) {
          return v + base_offset;
        });

    //
    //  Now we'll assign the vertices to batches
    //
    cugraph::detail::uniform_random_fill(handle_->get_stream(),
                                         random_numbers.data(),
                                         random_numbers.size(),
                                         float{0},
                                         float{1},
                                         seed);

    thrust::sort_by_key(handle_->get_thrust_policy(),
                        random_numbers.begin(),
                        random_numbers.end(),
                        random_sources.begin());

    random_numbers.resize(0, handle_->get_stream());
    random_numbers.shrink_to_fit(handle_->get_stream());

    rmm::device_uvector<int32_t> batch_number(random_sources.size(), handle_->get_stream());

    thrust::tabulate(handle_->get_thrust_policy(),
                     batch_number.begin(),
                     batch_number.end(),
                     [batch_size = uniform_neighbor_sampling_usecase.batch_size] __device__(
                       int32_t index) { return index / batch_size; });

    rmm::device_uvector<vertex_t> random_sources_copy(random_sources.size(), handle_->get_stream());

    raft::copy(random_sources_copy.data(),
               random_sources.data(),
               random_sources.size(),
               handle_->get_stream());

#ifdef NO_CUGRAPH_OPS
    EXPECT_THROW(cugraph::uniform_neighbor_sample(
                   *handle_,
                   handle,
                   mg_graph_view,
                   mg_edge_weight_view,
                   std::nullopt,
                   std::move(random_sources_copy),
                   std::move(batch_number),
                   raft::host_span<int32_t const>(uniform_neighbor_sampling_usecase.fanout.data(),
                                                  uniform_neighbor_sampling_usecase.fanout.size()),
                   rng_state,
                   uniform_neighbor_sampling_usecase.flag_replacement),
                 std::exception);
#else
    auto&& [src_out, dst_out, wgt_out, edge_id, edge_type, hop, labels] =
      cugraph::uniform_neighbor_sample(
        *handle_,
        mg_graph_view,
        mg_edge_weight_view,
        std::optional<cugraph::edge_property_view_t<
          edge_t,
          thrust::zip_iterator<thrust::tuple<edge_t const*, int32_t const*>>>>{std::nullopt},
        std::move(random_sources_copy),
        std::move(batch_number),
        raft::host_span<int32_t const>(uniform_neighbor_sampling_usecase.fanout.data(),
                                       uniform_neighbor_sampling_usecase.fanout.size()),
        rng_state,
        uniform_neighbor_sampling_usecase.flag_replacement);

    if (uniform_neighbor_sampling_usecase.check_correctness) {
      // Consolidate results on GPU 0
      auto mg_start_src = cugraph::test::device_gatherv(
        *handle_, raft::device_span<vertex_t const>{random_sources.data(), random_sources.size()});
      auto mg_aggregate_src = cugraph::test::device_gatherv(
        *handle_, raft::device_span<vertex_t const>{src_out.data(), src_out.size()});
      auto mg_aggregate_dst = cugraph::test::device_gatherv(
        *handle_, raft::device_span<vertex_t const>{dst_out.data(), dst_out.size()});
      auto mg_aggregate_wgt =
        wgt_out ? std::make_optional(cugraph::test::device_gatherv(
                    *handle_, raft::device_span<weight_t const>{wgt_out->data(), wgt_out->size()}))
                : std::nullopt;

      //  First validate that the extracted edges are actually a subset of the
      //  edges in the input graph
      rmm::device_uvector<vertex_t> vertices(2 * mg_aggregate_src.size(), handle_->get_stream());
      raft::copy(
        vertices.data(), mg_aggregate_src.data(), mg_aggregate_src.size(), handle_->get_stream());
      raft::copy(vertices.data() + mg_aggregate_src.size(),
                 mg_aggregate_dst.data(),
                 mg_aggregate_dst.size(),
                 handle_->get_stream());
      thrust::sort(handle_->get_thrust_policy(), vertices.begin(), vertices.end());
      auto vertices_end =
        thrust::unique(handle_->get_thrust_policy(), vertices.begin(), vertices.end());
      vertices.resize(thrust::distance(vertices.begin(), vertices_end), handle_->get_stream());

      vertices = cugraph::detail::shuffle_int_vertices_to_local_gpu_by_vertex_partitioning(
        *handle_, std::move(vertices), mg_graph_view.vertex_partition_range_lasts());

      thrust::sort(handle_->get_thrust_policy(), vertices.begin(), vertices.end());

      rmm::device_uvector<size_t> d_subgraph_offsets(2, handle_->get_stream());
      std::vector<size_t> h_subgraph_offsets({0, vertices.size()});

      raft::update_device(d_subgraph_offsets.data(),
                          h_subgraph_offsets.data(),
                          h_subgraph_offsets.size(),
                          handle_->get_stream());

      rmm::device_uvector<vertex_t> src_compare(0, handle_->get_stream());
      rmm::device_uvector<vertex_t> dst_compare(0, handle_->get_stream());
      std::optional<rmm::device_uvector<weight_t>> wgt_compare{std::nullopt};
      std::tie(src_compare, dst_compare, wgt_compare, std::ignore) = extract_induced_subgraphs(
        *handle_,
        mg_graph_view,
        mg_edge_weight_view,
        raft::device_span<size_t const>(d_subgraph_offsets.data(), 2),
        raft::device_span<vertex_t const>(vertices.data(), vertices.size()),
        true);

      auto mg_aggregate_src_compare = cugraph::test::device_gatherv(
        *handle_, raft::device_span<vertex_t const>{src_compare.data(), src_compare.size()});
      auto mg_aggregate_dst_compare = cugraph::test::device_gatherv(
        *handle_, raft::device_span<vertex_t const>{dst_compare.data(), dst_compare.size()});
      auto mg_aggregate_wgt_compare =
        wgt_compare
          ? std::make_optional(cugraph::test::device_gatherv(
              *handle_,
              raft::device_span<weight_t const>{wgt_compare->data(), wgt_compare->size()}))
          : std::nullopt;

      if (handle_->get_comms().get_rank() == 0) {
        cugraph::test::validate_extracted_graph_is_subgraph(*handle_,
                                                            mg_aggregate_src_compare,
                                                            mg_aggregate_dst_compare,
                                                            mg_aggregate_wgt_compare,
                                                            mg_aggregate_src,
                                                            mg_aggregate_dst,
                                                            mg_aggregate_wgt);

        if (random_sources.size() < 100) {
          // This validation is too expensive for large number of vertices
          if (mg_aggregate_src.size() > 0) {
            cugraph::test::validate_sampling_depth(*handle_,
                                                   std::move(mg_aggregate_src),
                                                   std::move(mg_aggregate_dst),
                                                   std::move(mg_aggregate_wgt),
                                                   std::move(mg_start_src),
                                                   uniform_neighbor_sampling_usecase.fanout.size());
          }
        }
      }
    }
#endif
  }

 private:
  static std::unique_ptr<raft::handle_t> handle_;
};

template <typename input_usecase_t>
std::unique_ptr<raft::handle_t> Tests_MGUniform_Neighbor_Sampling<input_usecase_t>::handle_ =
  nullptr;

using Tests_MGUniform_Neighbor_Sampling_File =
  Tests_MGUniform_Neighbor_Sampling<cugraph::test::File_Usecase>;

using Tests_MGUniform_Neighbor_Sampling_Rmat =
  Tests_MGUniform_Neighbor_Sampling<cugraph::test::Rmat_Usecase>;

TEST_P(Tests_MGUniform_Neighbor_Sampling_File, CheckInt32Int32Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MGUniform_Neighbor_Sampling_File, CheckInt32Int64Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MGUniform_Neighbor_Sampling_File, CheckInt64Int64Float)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MGUniform_Neighbor_Sampling_Rmat, CheckInt32Int32Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MGUniform_Neighbor_Sampling_Rmat, CheckInt32Int64Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MGUniform_Neighbor_Sampling_Rmat, CheckInt64Int64Float)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

INSTANTIATE_TEST_SUITE_P(
  file_test,
  Tests_MGUniform_Neighbor_Sampling_File,
  ::testing::Combine(
    ::testing::Values(Uniform_Neighbor_Sampling_Usecase{{2}, 100, true, true},
                      Uniform_Neighbor_Sampling_Usecase{{2}, 100, true, false}),
    ::testing::Values(cugraph::test::File_Usecase("test/datasets/karate.mtx"),
                      cugraph::test::File_Usecase("test/datasets/web-Google.mtx"),
                      cugraph::test::File_Usecase("test/datasets/ljournal-2008.mtx"),
                      cugraph::test::File_Usecase("test/datasets/webbase-1M.mtx"))));

INSTANTIATE_TEST_SUITE_P(
  rmat_small_test,
  Tests_MGUniform_Neighbor_Sampling_Rmat,
  ::testing::Combine(::testing::Values(Uniform_Neighbor_Sampling_Usecase{{2}, 10, false, true}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       10, 16, 0.57, 0.19, 0.19, 0, false, false, 0, true))));

INSTANTIATE_TEST_SUITE_P(
  rmat_benchmark_test, /* note that scale & edge factor can be overridden in benchmarking (with
                          --gtest_filter to select only the rmat_benchmark_test with a specific
                          vertex & edge type combination) by command line arguments and do not
                          include more than one Rmat_Usecase that differ only in scale or edge
                          factor (to avoid running same benchmarks more than once) */
  Tests_MGUniform_Neighbor_Sampling_Rmat,
  ::testing::Combine(::testing::Values(Uniform_Neighbor_Sampling_Usecase{{2}, 500, false, true}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       20, 32, 0.57, 0.19, 0.19, 0, false, false, 0, true))));

CUGRAPH_MG_TEST_PROGRAM_MAIN()
