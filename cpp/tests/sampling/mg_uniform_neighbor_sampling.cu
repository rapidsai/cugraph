#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "detail/nbr_sampling_utils.cuh"

#include <utilities/mg_utilities.hpp>

#include <thrust/distance.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

#include <gtest/gtest.h>

struct Prims_Usecase {
  bool check_correctness{true};
  bool flag_replacement{true};
};

template <typename input_usecase_t>
class Tests_MGNbrSampling
  : public ::testing::TestWithParam<std::tuple<Prims_Usecase, input_usecase_t>> {
 public:
  Tests_MGNbrSampling() {}

  static void SetUpTestCase() { handle_ = cugraph::test::initialize_mg_handle(); }

  static void TearDownTestCase() { handle_.reset(); }

  virtual void SetUp() {}
  virtual void TearDown() {}

  template <typename vertex_t, typename edge_t, typename weight_t>
  void run_current_test(Prims_Usecase const& prims_usecase, input_usecase_t const& input_usecase)
  {
    HighResClock hr_clock{};

    // 1. create MG graph

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle_->get_comms().barrier();
      hr_clock.start();
    }

    constexpr bool sort_adjacency_list = true;

    auto [mg_graph, mg_edge_weights, mg_renumber_map_labels] =
      cugraph::test::construct_graph<vertex_t, edge_t, weight_t, false, true>(
        *handle_, input_usecase, true, true, false, sort_adjacency_list);

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle_->get_comms().barrier();
      double elapsed_time{0.0};
      hr_clock.stop(&elapsed_time);
      std::cout << "MG construct_graph took " << elapsed_time * 1e-6 << " s.\n";
    }

    auto mg_graph_view = mg_graph.view();
    auto mg_edge_weight_view =
      mg_edge_weights ? std::make_optional((*mg_edge_weights).view()) : std::nullopt;

    constexpr edge_t indices_per_source       = 2;
    constexpr vertex_t repetitions_per_vertex = 5;
    constexpr vertex_t source_sample_count    = 3;

    // Generate random vertex ids in the range of current gpu
    auto random_sources =
      cugraph::test::random_vertex_ids(*handle_,
                                       mg_graph_view.local_vertex_partition_range_first(),
                                       mg_graph_view.local_vertex_partition_range_last(),
                                       std::min(mg_graph_view.local_vertex_partition_range_size() *
                                                  (repetitions_per_vertex + vertex_t{1}),
                                                source_sample_count),
                                       repetitions_per_vertex,
                                       handle_->get_comms().get_rank());

    std::vector<int> h_fan_out{indices_per_source};  // depth = 1

#ifdef NO_CUGRAPH_OPS
    EXPECT_THROW(cugraph::uniform_nbr_sample(
                   *handle_,
                   mg_graph_view,
                   mg_edge_weight_view,
                   raft::device_span<vertex_t>(random_sources.data(), random_sources.size()),
                   raft::host_span<const int>(h_fan_out.data(), h_fan_out.size()),
                   prims_usecase.flag_replacement),
                 std::exception);
#else
    auto&& [d_src_out, d_dst_out, d_indices, d_counts] = cugraph::uniform_nbr_sample(
      *handle_,
      mg_graph_view,
      mg_edge_weight_view,
      raft::device_span<vertex_t>(random_sources.data(), random_sources.size()),
      raft::host_span<const int>(h_fan_out.data(), h_fan_out.size()),
      prims_usecase.flag_replacement);

    if (prims_usecase.check_correctness) {
      // Consolidate results on GPU 0
      auto d_mg_start_src = cugraph::test::device_gatherv(
        *handle_, raft::device_span<vertex_t const>{random_sources.data(), random_sources.size()});
      auto d_mg_aggregate_src = cugraph::test::device_gatherv(
        *handle_, raft::device_span<vertex_t const>{d_src_out.data(), d_src_out.size()});
      auto d_mg_aggregate_dst = cugraph::test::device_gatherv(
        *handle_, raft::device_span<vertex_t const>{d_dst_out.data(), d_dst_out.size()});
      auto d_mg_aggregate_indices = cugraph::test::device_gatherv(
        *handle_, raft::device_span<weight_t const>{d_indices.data(), d_indices.size()});

#if 0
      // FIXME:  extract_induced_subgraphs not currently support MG, so we'll skip this validation
      //         step

      //  First validate that the extracted edges are actually a subset of the
      //  edges in the input graph
      rmm::device_uvector<vertex_t> d_vertices(2 * d_mg_aggregate_src.size(), handle_->get_stream());
      raft::copy(d_vertices.data(), d_mg_aggregate_src.data(), d_mg_aggregate_src.size(), handle_->get_stream());
      raft::copy(d_vertices.data() + d_mg_aggregate_src.size(),
                 d_mg_aggregate_dst.data(),
                 d_mg_aggregate_dst.size(),
                 handle_->get_stream());
      thrust::sort(handle_->get_thrust_policy(), d_vertices.begin(), d_vertices.end());
      auto vertices_end =
        thrust::unique(handle_->get_thrust_policy(), d_vertices.begin(), d_vertices.end());
      d_vertices.resize(thrust::distance(d_vertices.begin(), vertices_end), handle_->get_stream());

      d_vertices = cugraph::detail::shuffle_int_vertices_by_gpu_id(*handle_, std::move(d_vertices), mg_graph_view.vertex_partition_range_lasts());

      thrust::sort(handle_->get_thrust_policy(), d_vertices.begin(), d_vertices.end());

      rmm::device_uvector<size_t> d_subgraph_offsets(2, handle_->get_stream());
      std::vector<size_t> h_subgraph_offsets({0, d_vertices.size()});

      raft::update_device(d_subgraph_offsets.data(),
                          h_subgraph_offsets.data(),
                          h_subgraph_offsets.size(),
                          handle_->get_stream());

      auto [d_src_in, d_dst_in, d_indices_in, d_ignore] = extract_induced_subgraphs(
        *handle_, mg_graph_view, d_subgraph_offsets.data(), d_vertices.data(), 1, true);

      cugraph::test::validate_extracted_graph_is_subgraph(
        *handle_, d_src_in, d_dst_in, *d_indices_in, d_src_out, d_dst_out, d_indices);
#endif

      if (d_mg_aggregate_src.size() > 0) {
        cugraph::test::validate_sampling_depth(*handle_,
                                               std::move(d_mg_aggregate_src),
                                               std::move(d_mg_aggregate_dst),
                                               std::move(d_mg_aggregate_indices),
                                               std::move(d_mg_start_src),
                                               h_fan_out.size());
      }
    }
#endif
  }

 private:
  static std::unique_ptr<raft::handle_t> handle_;
};

template <typename input_usecase_t>
std::unique_ptr<raft::handle_t> Tests_MGNbrSampling<input_usecase_t>::handle_ = nullptr;

using Tests_MGNbrSampling_File = Tests_MGNbrSampling<cugraph::test::File_Usecase>;

using Tests_MGNbrSampling_Rmat = Tests_MGNbrSampling<cugraph::test::Rmat_Usecase>;

TEST_P(Tests_MGNbrSampling_File, CheckInt32Int32Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MGNbrSampling_File, CheckInt32Int64Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MGNbrSampling_File, CheckInt64Int64Float)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MGNbrSampling_Rmat, CheckInt32Int32Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MGNbrSampling_Rmat, CheckInt32Int64Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MGNbrSampling_Rmat, CheckInt64Int64Float)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

INSTANTIATE_TEST_SUITE_P(
  file_test,
  Tests_MGNbrSampling_File,
  ::testing::Combine(
    ::testing::Values(Prims_Usecase{true, true}, Prims_Usecase{true, false}),
    ::testing::Values(cugraph::test::File_Usecase("test/datasets/karate.mtx"),
                      cugraph::test::File_Usecase("test/datasets/web-Google.mtx"),
                      cugraph::test::File_Usecase("test/datasets/ljournal-2008.mtx"),
                      cugraph::test::File_Usecase("test/datasets/webbase-1M.mtx"))));

INSTANTIATE_TEST_SUITE_P(
  rmat_small_test,
  Tests_MGNbrSampling_Rmat,
  ::testing::Combine(::testing::Values(Prims_Usecase{false, true}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       10, 16, 0.57, 0.19, 0.19, 0, false, false, 0, true))));

INSTANTIATE_TEST_SUITE_P(
  rmat_benchmark_test, /* note that scale & edge factor can be overridden in benchmarking (with
                          --gtest_filter to select only the rmat_benchmark_test with a specific
                          vertex & edge type combination) by command line arguments and do not
                          include more than one Rmat_Usecase that differ only in scale or edge
                          factor (to avoid running same benchmarks more than once) */
  Tests_MGNbrSampling_Rmat,
  ::testing::Combine(::testing::Values(Prims_Usecase{false, true}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       20, 32, 0.57, 0.19, 0.19, 0, false, false, 0, true))));

CUGRAPH_MG_TEST_PROGRAM_MAIN()
