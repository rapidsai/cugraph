#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "detail/nbr_sampling_utils.cuh"

#include <gtest/gtest.h>

#include <thrust/distance.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

struct Prims_Usecase {
  bool check_correctness{true};
  bool flag_replacement{true};
};

template <typename input_usecase_t>
class Tests_Uniform_Neighbor_Sampling
  : public ::testing::TestWithParam<std::tuple<Prims_Usecase, input_usecase_t>> {
 public:
  Tests_Uniform_Neighbor_Sampling() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}

  template <typename vertex_t, typename edge_t, typename weight_t>
  void run_current_test(Prims_Usecase const& prims_usecase, input_usecase_t const& input_usecase)
  {
    raft::handle_t handle{};
    HighResClock hr_clock{};

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      hr_clock.start();
    }

    auto [graph, renumber_map_labels] =
      cugraph::test::construct_graph<vertex_t, edge_t, weight_t, false, false>(
        handle, input_usecase, true, true);

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      double elapsed_time{0.0};
      hr_clock.stop(&elapsed_time);
      std::cout << "construct_graph took " << elapsed_time * 1e-6 << " s.\n";
    }

    auto graph_view                           = graph.view();
    constexpr edge_t indices_per_source       = 2;
    constexpr vertex_t repetitions_per_vertex = 5;
    constexpr vertex_t source_sample_count    = 3;

    // Generate random vertex ids in the range of current gpu

    // Generate random sources to gather on
    auto random_sources =
      cugraph::test::random_vertex_ids(handle,
                                       graph_view.local_vertex_partition_range_first(),
                                       graph_view.local_vertex_partition_range_last(),
                                       std::min(graph_view.local_vertex_partition_range_size() *
                                                  (repetitions_per_vertex + vertex_t{1}),
                                                source_sample_count),
                                       repetitions_per_vertex,
                                       uint64_t{0});

    std::vector<int> h_fan_out{indices_per_source};  // depth = 1

#ifdef NO_CUGRAPH_OPS
    EXPECT_THROW(cugraph::uniform_nbr_sample(
                   handle,
                   graph_view,
                   raft::device_span<vertex_t>(random_sources.data(), random_sources.size()),
                   raft::host_span<const int>(h_fan_out.data(), h_fan_out.size()),
                   prims_usecase.flag_replacement),
                 std::exception);
#else
    auto&& [d_src_out, d_dst_out, d_indices, d_counts] = cugraph::uniform_nbr_sample(
      handle,
      graph_view,
      raft::device_span<vertex_t>(random_sources.data(), random_sources.size()),
      raft::host_span<const int>(h_fan_out.data(), h_fan_out.size()),
      prims_usecase.flag_replacement);

    if (prims_usecase.check_correctness) {
      //  First validate that the extracted edges are actually a subset of the
      //  edges in the input graph
      rmm::device_uvector<vertex_t> d_vertices(2 * d_src_out.size(), handle.get_stream());
      raft::copy(d_vertices.data(), d_src_out.data(), d_src_out.size(), handle.get_stream());
      raft::copy(d_vertices.data() + d_src_out.size(),
                 d_dst_out.data(),
                 d_dst_out.size(),
                 handle.get_stream());
      thrust::sort(handle.get_thrust_policy(), d_vertices.begin(), d_vertices.end());
      auto vertices_end =
        thrust::unique(handle.get_thrust_policy(), d_vertices.begin(), d_vertices.end());
      d_vertices.resize(thrust::distance(d_vertices.begin(), vertices_end), handle.get_stream());

      rmm::device_uvector<size_t> d_subgraph_offsets(2, handle.get_stream());
      std::vector<size_t> h_subgraph_offsets({0, d_vertices.size()});

      raft::update_device(d_subgraph_offsets.data(),
                          h_subgraph_offsets.data(),
                          h_subgraph_offsets.size(),
                          handle.get_stream());

      auto [d_src_in, d_dst_in, d_indices_in, d_ignore] = extract_induced_subgraphs(
        handle,
        graph_view,
        raft::device_span<size_t const>(d_subgraph_offsets.data(), d_subgraph_offsets.size()),
        raft::device_span<vertex_t const>(d_vertices.data(), d_vertices.size()),
        1,
        true);

      cugraph::test::validate_extracted_graph_is_subgraph(
        handle, d_src_in, d_dst_in, *d_indices_in, d_src_out, d_dst_out, d_indices);

      cugraph::test::validate_sampling_depth(handle,
                                             std::move(d_src_out),
                                             std::move(d_dst_out),
                                             std::move(d_indices),
                                             std::move(random_sources),
                                             h_fan_out.size());
    }
#endif
  }
};

using Tests_Uniform_Neighbor_Sampling_File =
  Tests_Uniform_Neighbor_Sampling<cugraph::test::File_Usecase>;

using Tests_Uniform_Neighbor_Sampling_Rmat =
  Tests_Uniform_Neighbor_Sampling<cugraph::test::Rmat_Usecase>;

TEST_P(Tests_Uniform_Neighbor_Sampling_File, CheckInt32Int32Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_Uniform_Neighbor_Sampling_File, CheckInt32Int64Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_Uniform_Neighbor_Sampling_File, CheckInt64Int64Float)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_Uniform_Neighbor_Sampling_Rmat, CheckInt32Int32Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_Uniform_Neighbor_Sampling_Rmat, CheckInt32Int64Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_Uniform_Neighbor_Sampling_Rmat, CheckInt64Int64Float)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

INSTANTIATE_TEST_SUITE_P(
  file_test,
  Tests_Uniform_Neighbor_Sampling_File,
  ::testing::Combine(
    ::testing::Values(Prims_Usecase{true, true}, Prims_Usecase{true, false}),
    ::testing::Values(cugraph::test::File_Usecase("test/datasets/karate.mtx"),
                      cugraph::test::File_Usecase("test/datasets/web-Google.mtx"),
                      cugraph::test::File_Usecase("test/datasets/ljournal-2008.mtx"),
                      cugraph::test::File_Usecase("test/datasets/webbase-1M.mtx"))));

INSTANTIATE_TEST_SUITE_P(
  rmat_small_test,
  Tests_Uniform_Neighbor_Sampling_Rmat,
  ::testing::Combine(::testing::Values(Prims_Usecase{false, true}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       10, 16, 0.57, 0.19, 0.19, 0, false, false, 0, false))));

INSTANTIATE_TEST_SUITE_P(
  rmat_benchmark_test, /* note that scale & edge factor can be overridden in benchmarking (with
                          --gtest_filter to select only the rmat_benchmark_test with a specific
                          vertex & edge type combination) by command line arguments and do not
                          include more than one Rmat_Usecase that differ only in scale or edge
                          factor (to avoid running same benchmarks more than once) */
  Tests_Uniform_Neighbor_Sampling_Rmat,
  ::testing::Combine(::testing::Values(Prims_Usecase{false, true}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       20, 32, 0.57, 0.19, 0.19, 0, false, false, 0, false))));

CUGRAPH_TEST_PROGRAM_MAIN()
