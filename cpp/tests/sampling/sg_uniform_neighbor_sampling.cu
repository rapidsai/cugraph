/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "detail/nbr_sampling_utils.cuh"

#include <gtest/gtest.h>

#include <thrust/distance.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

struct Uniform_Neighbor_Sampling_Usecase {
  std::vector<int32_t> fanout{{-1}};
  int32_t batch_size{10};
  bool check_correctness{true};
  bool flag_replacement{true};
};

template <typename input_usecase_t>
class Tests_Uniform_Neighbor_Sampling
  : public ::testing::TestWithParam<
      std::tuple<Uniform_Neighbor_Sampling_Usecase, input_usecase_t>> {
 public:
  Tests_Uniform_Neighbor_Sampling() {}

  static void SetUpTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}

  template <typename vertex_t, typename edge_t, typename weight_t>
  void run_current_test(Uniform_Neighbor_Sampling_Usecase const& uniform_neighbor_sampling_usecase,
                        input_usecase_t const& input_usecase)
  {
    raft::handle_t handle{};
    HighResTimer hr_timer{};

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      hr_timer.start("Construct graph");
    }

    auto [graph, edge_weights, renumber_map_labels] =
      cugraph::test::construct_graph<vertex_t, edge_t, weight_t, false, false>(
        handle, input_usecase, true, true);

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      hr_timer.stop();
      hr_timer.display_and_clear(std::cout);
    }

    auto graph_view = graph.view();
    auto edge_weight_view =
      edge_weights ? std::make_optional((*edge_weights).view()) : std::nullopt;

    //
    // Test is designed like GNN sampling.  We'll select 90% of vertices
    // to be included in sampling batches
    //
    constexpr float select_probability{0.9};

    // FIXME:  Update the tests to initialize RngState and use it instead
    //         of seed...
    constexpr uint64_t seed{0};

    raft::random::RngState rng_state(seed);
    rmm::device_uvector<float> random_numbers(graph_view.local_vertex_partition_range_size(),
                                              handle.get_stream());
    rmm::device_uvector<vertex_t> random_sources(graph_view.local_vertex_partition_range_size(),
                                                 handle.get_stream());

    cugraph::detail::uniform_random_fill(
      handle.get_stream(), random_numbers.data(), random_numbers.size(), float{0}, float{1}, seed);

    auto random_sources_end = thrust::copy_if(
      handle.get_thrust_policy(),
      thrust::make_counting_iterator(vertex_t{0}),
      thrust::make_counting_iterator(graph_view.local_vertex_partition_range_size()),
      random_sources.begin(),
      [d_random_number = random_numbers.data(), select_probability] __device__(vertex_t offset) {
        return d_random_number[offset] < select_probability;
      });

    random_sources.resize(thrust::distance(random_sources.begin(), random_sources_end),
                          handle.get_stream());
    random_sources.shrink_to_fit(handle.get_stream());

    random_numbers.resize(random_sources.size(), handle.get_stream());
    random_numbers.shrink_to_fit(handle.get_stream());

    //
    //  Now we'll assign the vertices to batches
    //
    cugraph::detail::uniform_random_fill(
      handle.get_stream(), random_numbers.data(), random_numbers.size(), float{0}, float{1}, seed);

    thrust::sort_by_key(handle.get_thrust_policy(),
                        random_numbers.begin(),
                        random_numbers.end(),
                        random_sources.begin());

    random_numbers.resize(0, handle.get_stream());
    random_numbers.shrink_to_fit(handle.get_stream());

    rmm::device_uvector<int32_t> batch_number(random_sources.size(), handle.get_stream());

    thrust::tabulate(handle.get_thrust_policy(),
                     batch_number.begin(),
                     batch_number.end(),
                     [batch_size = uniform_neighbor_sampling_usecase.batch_size] __device__(
                       int32_t index) { return index / batch_size; });

    rmm::device_uvector<vertex_t> random_sources_copy(random_sources.size(), handle.get_stream());

    raft::copy(random_sources_copy.data(),
               random_sources.data(),
               random_sources.size(),
               handle.get_stream());

#ifdef NO_CUGRAPH_OPS
    EXPECT_THROW(cugraph::uniform_neighbor_sample(
                   handle,
                   graph_view,
                   edge_weight_view,
                   std::nullopt,
                   std::move(random_sources_copy),
                   std::move(batch_number),
                   raft::host_span<int32_t const>(uniform_neighbor_sampling_usecase.fanout.data(),
                                                  uniform_neighbor_sampling_usecase.fanout.size()),
                   rng_state,
                   uniform_neighbor_sampling_usecase.flag_replacement),
                 std::exception);
#else
    auto&& [src_out, dst_out, wgt_out, edge_id, edge_type, hop, labels] =
      cugraph::uniform_neighbor_sample(
        handle,
        graph_view,
        edge_weight_view,
        std::optional<cugraph::edge_property_view_t<
          edge_t,
          thrust::zip_iterator<thrust::tuple<edge_t const*, int32_t const*>>>>{std::nullopt},
        std::move(random_sources_copy),
        std::move(batch_number),
        raft::host_span<int32_t const>(uniform_neighbor_sampling_usecase.fanout.data(),
                                       uniform_neighbor_sampling_usecase.fanout.size()),
        rng_state,
        uniform_neighbor_sampling_usecase.flag_replacement);

    if (uniform_neighbor_sampling_usecase.check_correctness) {
      //  First validate that the extracted edges are actually a subset of the
      //  edges in the input graph
      rmm::device_uvector<vertex_t> vertices(2 * src_out.size(), handle.get_stream());
      raft::copy(vertices.data(), src_out.data(), src_out.size(), handle.get_stream());
      raft::copy(
        vertices.data() + src_out.size(), dst_out.data(), dst_out.size(), handle.get_stream());
      thrust::sort(handle.get_thrust_policy(), vertices.begin(), vertices.end());
      auto vertices_end =
        thrust::unique(handle.get_thrust_policy(), vertices.begin(), vertices.end());
      vertices.resize(thrust::distance(vertices.begin(), vertices_end), handle.get_stream());

      rmm::device_uvector<size_t> d_subgraph_offsets(2, handle.get_stream());
      std::vector<size_t> h_subgraph_offsets({0, vertices.size()});

      raft::update_device(d_subgraph_offsets.data(),
                          h_subgraph_offsets.data(),
                          h_subgraph_offsets.size(),
                          handle.get_stream());

      rmm::device_uvector<vertex_t> src_compare(0, handle.get_stream());
      rmm::device_uvector<vertex_t> dst_compare(0, handle.get_stream());
      std::optional<rmm::device_uvector<weight_t>> wgt_compare{std::nullopt};

      std::tie(src_compare, dst_compare, wgt_compare, std::ignore) = extract_induced_subgraphs(
        handle,
        graph_view,
        edge_weight_view,
        raft::device_span<size_t const>(d_subgraph_offsets.data(), 2),
        raft::device_span<vertex_t const>(vertices.data(), vertices.size()),
        true);

      cugraph::test::validate_extracted_graph_is_subgraph(
        handle, src_compare, dst_compare, wgt_compare, src_out, dst_out, wgt_out);

      if (random_sources.size() < 100) {
        // This validation is too expensive for large number of vertices
        cugraph::test::validate_sampling_depth(handle,
                                               std::move(src_out),
                                               std::move(dst_out),
                                               std::move(wgt_out),
                                               std::move(random_sources),
                                               uniform_neighbor_sampling_usecase.fanout.size());
      }
    }
#endif
  }
};

using Tests_Uniform_Neighbor_Sampling_File =
  Tests_Uniform_Neighbor_Sampling<cugraph::test::File_Usecase>;

using Tests_Uniform_Neighbor_Sampling_Rmat =
  Tests_Uniform_Neighbor_Sampling<cugraph::test::Rmat_Usecase>;

TEST_P(Tests_Uniform_Neighbor_Sampling_File, CheckInt32Int32Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_Uniform_Neighbor_Sampling_File, CheckInt32Int64Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_Uniform_Neighbor_Sampling_File, CheckInt64Int64Float)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_Uniform_Neighbor_Sampling_Rmat, CheckInt32Int32Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_Uniform_Neighbor_Sampling_Rmat, CheckInt32Int64Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_Uniform_Neighbor_Sampling_Rmat, CheckInt64Int64Float)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t, float>(std::get<0>(param), std::get<1>(param));
}

INSTANTIATE_TEST_SUITE_P(
  file_test,
  Tests_Uniform_Neighbor_Sampling_File,
  ::testing::Combine(
    ::testing::Values(Uniform_Neighbor_Sampling_Usecase{{2}, 100, true, true},
                      Uniform_Neighbor_Sampling_Usecase{{2}, 100, true, false}),
    ::testing::Values(cugraph::test::File_Usecase("test/datasets/karate.mtx"),
                      cugraph::test::File_Usecase("test/datasets/web-Google.mtx"),
                      cugraph::test::File_Usecase("test/datasets/ljournal-2008.mtx"),
                      cugraph::test::File_Usecase("test/datasets/webbase-1M.mtx"))));

INSTANTIATE_TEST_SUITE_P(
  rmat_small_test,
  Tests_Uniform_Neighbor_Sampling_Rmat,
  ::testing::Combine(::testing::Values(Uniform_Neighbor_Sampling_Usecase{{2}, 10, false, true}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       10, 16, 0.57, 0.19, 0.19, 0, false, false, 0, false))));

INSTANTIATE_TEST_SUITE_P(
  rmat_benchmark_test, /* note that scale & edge factor can be overridden in benchmarking (with
                          --gtest_filter to select only the rmat_benchmark_test with a specific
                          vertex & edge type combination) by command line arguments and do not
                          include more than one Rmat_Usecase that differ only in scale or edge
                          factor (to avoid running same benchmarks more than once) */
  Tests_Uniform_Neighbor_Sampling_Rmat,
  ::testing::Combine(::testing::Values(Uniform_Neighbor_Sampling_Usecase{{2}, 500, false, true}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       20, 32, 0.57, 0.19, 0.19, 0, false, false, 0, false))));

CUGRAPH_TEST_PROGRAM_MAIN()
