/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governin_from_mtxg permissions and
 * limitations under the License.
 */

#include <utilities/base_fixture.hpp>
#include <utilities/test_utilities.hpp>

#include <raft/cudart_utils.h>
#include <raft/handle.hpp>
#include <rmm/mr/device/cuda_memory_resource.hpp>

#include <experimental/graph.hpp>
#include <experimental/louvain.cuh>

#include <algorithms.hpp>

#include <raft/cudart_utils.h>
#include <raft/handle.hpp>
#include <rmm/mr/device/cuda_memory_resource.hpp>

#include <gtest/gtest.h>

#include <algorithm>
#include <iterator>
#include <limits>
#include <numeric>
#include <vector>

typedef struct Louvain_Usecase_t {
  std::string graph_file_full_path{};
  bool test_weighted{false};

  Louvain_Usecase_t(std::string const& graph_file_path, bool test_weighted)
    : test_weighted(test_weighted)
  {
    if ((graph_file_path.length() > 0) && (graph_file_path[0] != '/')) {
      graph_file_full_path = cugraph::test::get_rapids_dataset_root_dir() + "/" + graph_file_path;
    } else {
      graph_file_full_path = graph_file_path;
    }
  };
} Louvain_Usecase;

class Tests_Louvain : public ::testing::TestWithParam<Louvain_Usecase> {
 public:
  Tests_Louvain() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}

  template <typename vertex_t, typename edge_t, typename weight_t, typename result_t>
  void run_current_test(Louvain_Usecase const& configuration)
  {
    raft::handle_t handle{};

    std::cout << "read graph file: " << configuration.graph_file_full_path << std::endl;

    cugraph::experimental::graph_t<vertex_t, edge_t, weight_t, false, false> graph(handle);
    std::tie(graph, std::ignore) =
      cugraph::test::read_graph_from_matrix_market_file<vertex_t, edge_t, weight_t, false, false>(
        handle, configuration.graph_file_full_path, configuration.test_weighted, false);

    auto graph_view = graph.view();

    // "FIXME": remove this check once we drop support for Pascal
    //
    // Calling louvain on Pascal will throw an exception, we'll check that
    // this is the behavior while we still support Pascal (device_prop.major < 7)
    //
    hipDeviceProp_t device_prop;
    CUDA_CHECK(hipGetDeviceProperties(&device_prop, 0));

    if (device_prop.major < 7) {
      EXPECT_THROW(louvain(graph_view), cugraph::logic_error);
    } else {
      louvain(graph_view);
    }
  }

  template <typename graph_t>
  void louvain(graph_t const& graph_view)
  {
    using vertex_t = typename graph_t::vertex_type;
    using weight_t = typename graph_t::weight_type;

    raft::handle_t handle{};

    rmm::device_vector<vertex_t> clustering_v(graph_view.get_number_of_local_vertices());
    size_t level;
    weight_t modularity;

    std::tie(level, modularity) =
      cugraph::louvain(handle, graph_view, clustering_v.data().get(), size_t{100}, weight_t{1});

    CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement

    std::cout << "level = " << level << std::endl;
    std::cout << "modularity = " << modularity << std::endl;
  }
};

// FIXME: add tests for type combinations
TEST_P(Tests_Louvain, CheckInt32Int32FloatFloat)
{
  run_current_test<int32_t, int32_t, float, float>(GetParam());
}

INSTANTIATE_TEST_CASE_P(simple_test,
                        Tests_Louvain,
                        ::testing::Values(Louvain_Usecase("test/datasets/karate.mtx", true)
#if 0
			,
                                          Louvain_Usecase("test/datasets/web-Google.mtx", true),
                                          Louvain_Usecase("test/datasets/ljournal-2008.mtx", true),
                                          Louvain_Usecase("test/datasets/webbase-1M.mtx", true)
#endif
                                            ));

CUGRAPH_TEST_PROGRAM_MAIN()
