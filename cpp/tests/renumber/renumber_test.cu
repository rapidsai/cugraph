#include "hip/hip_runtime.h"
// -*-c++-*-

/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "gmock/gmock.h"
#include "gtest/gtest.h"

#include "hip/hip_runtime_api.h"

#include <rmm/rmm.h>
#include <rmm/thrust_rmm_allocator.h>
#include <rmm/mr/device/cuda_memory_resource.hpp>
#include "converters/renumber.cuh"

#include <chrono>

#include <hiprand/hiprand_kernel.h>

struct RenumberingTest : public ::testing::Test {
};

__global__ void display_list(const char *label, uint32_t *verts, size_t length)
{
  printf("%s\n", label);

  for (size_t i = 0; i < length; ++i) { printf("  %u\n", verts[i]); }
}

__global__ void setup_generator(hiprandState *state)
{
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(43, id, 0, &state[id]);
}

__global__ void generate_sources(hiprandState *state, int n, uint32_t *verts)
{
  int first  = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  hiprandState local_state = state[first];
  for (int id = first; id < n; id += stride) { verts[id] = hiprand(&local_state); }

  state[first] = local_state;
}

__global__ void generate_destinations(hiprandState *state,
                                      int n,
                                      const uint32_t *sources,
                                      uint32_t *destinations)
{
  int first  = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  hiprandState local_state = state[first];
  for (int id = first; id < n; id += stride) {
    destinations[id] = sources[hiprand(&local_state) % n];
  }

  state[first] = local_state;
}

TEST_F(RenumberingTest, SmallFixedVertexList)
{
  uint32_t src_data[] = {4U, 6U, 8U, 20U, 1U};
  uint32_t dst_data[] = {1U, 29U, 35U, 0U, 77U};

  uint32_t src_expected[] = {2U, 3U, 4U, 5U, 1U};
  uint32_t dst_expected[] = {1U, 6U, 7U, 0U, 8U};

  size_t length = sizeof(src_data) / sizeof(src_data[0]);

  uint32_t *src_d;
  uint32_t *dst_d;

  uint32_t tmp_results[length];
  uint32_t tmp_map[2 * length];

  rmm::device_vector<uint32_t> src(length);
  rmm::device_vector<uint32_t> dst(length);
  src_d = src.data().get();
  dst_d = dst.data().get();

  EXPECT_EQ(hipMemcpy(src_d, src_data, sizeof(uint32_t) * length, hipMemcpyHostToDevice),
            hipSuccess);
  EXPECT_EQ(hipMemcpy(dst_d, dst_data, sizeof(uint32_t) * length, hipMemcpyHostToDevice),
            hipSuccess);

  size_t unique_verts = 0;

  auto number_map = cugraph::detail::renumber_vertices(length,
                                                       src_d,
                                                       dst_d,
                                                       src_d,
                                                       dst_d,
                                                       &unique_verts,
                                                       cugraph::detail::HashFunctionObjectInt(511),
                                                       thrust::less<uint32_t>(),
                                                       rmm::mr::get_default_resource());

  EXPECT_EQ(hipMemcpy(
              tmp_map, number_map->data(), sizeof(uint32_t) * unique_verts, hipMemcpyDeviceToHost),
            hipSuccess);
  EXPECT_EQ(hipMemcpy(tmp_results, src_d, sizeof(uint32_t) * length, hipMemcpyDeviceToHost),
            hipSuccess);

  for (size_t i = 0; i < length; ++i) {
    EXPECT_EQ(tmp_results[i], src_expected[i]);
    EXPECT_EQ(tmp_map[tmp_results[i]], src_data[i]);
  }

  EXPECT_EQ(hipMemcpy(tmp_results, dst_d, sizeof(uint32_t) * length, hipMemcpyDeviceToHost),
            hipSuccess);
  for (size_t i = 0; i < length; ++i) {
    EXPECT_EQ(tmp_results[i], dst_expected[i]);
    EXPECT_EQ(tmp_map[tmp_results[i]], dst_data[i]);
  }
}

TEST_F(RenumberingTest, SmallFixedVertexListNegative)
{
  int64_t src_data[] = {4, 6, 8, -20, 1};
  int64_t dst_data[] = {1, 29, 35, 0, 77};

  int64_t src_expected[] = {2, 3, 4, 8, 1};
  int64_t dst_expected[] = {1, 5, 6, 0, 7};

  size_t length = sizeof(src_data) / sizeof(src_data[0]);

  int64_t *src_d;
  int64_t *dst_d;

  int64_t tmp_results[length];
  int64_t tmp_map[2 * length];

  rmm::device_vector<int64_t> src(length);
  rmm::device_vector<int64_t> dst(length);
  src_d = src.data().get();
  dst_d = dst.data().get();

  EXPECT_EQ(hipMemcpy(src_d, src_data, sizeof(int64_t) * length, hipMemcpyHostToDevice),
            hipSuccess);
  EXPECT_EQ(hipMemcpy(dst_d, dst_data, sizeof(int64_t) * length, hipMemcpyHostToDevice),
            hipSuccess);

  size_t unique_verts = 0;

  auto number_map = cugraph::detail::renumber_vertices(length,
                                                       src_d,
                                                       dst_d,
                                                       src_d,
                                                       dst_d,
                                                       &unique_verts,
                                                       cugraph::detail::HashFunctionObjectInt(511),
                                                       thrust::less<int64_t>(),
                                                       rmm::mr::get_default_resource());

  EXPECT_EQ(
    hipMemcpy(tmp_map, number_map->data(), sizeof(int64_t) * unique_verts, hipMemcpyDeviceToHost),
    hipSuccess);
  EXPECT_EQ(hipMemcpy(tmp_results, src_d, sizeof(int64_t) * length, hipMemcpyDeviceToHost),
            hipSuccess);

  for (size_t i = 0; i < length; ++i) {
    EXPECT_EQ(tmp_results[i], src_expected[i]);
    EXPECT_EQ(tmp_map[tmp_results[i]], src_data[i]);
  }

  EXPECT_EQ(hipMemcpy(tmp_results, dst_d, sizeof(int64_t) * length, hipMemcpyDeviceToHost),
            hipSuccess);
  for (size_t i = 0; i < length; ++i) {
    EXPECT_EQ(tmp_results[i], dst_expected[i]);
    EXPECT_EQ(tmp_map[tmp_results[i]], dst_data[i]);
  }
}

TEST_F(RenumberingTest, SmallFixedVertexList64Bit)
{
  uint64_t src_data[] = {4U, 6U, 8U, 20U, 1U};
  uint64_t dst_data[] = {1U, 29U, 35U, 0U, 77U};

  uint64_t src_expected[] = {2U, 3U, 4U, 5U, 1U};
  uint64_t dst_expected[] = {1U, 6U, 7U, 0U, 8U};

  size_t length = sizeof(src_data) / sizeof(src_data[0]);

  uint64_t *src_d;
  uint64_t *dst_d;

  uint64_t tmp_results[length];
  uint64_t tmp_map[2 * length];

  rmm::device_vector<uint64_t> src(length);
  rmm::device_vector<uint64_t> dst(length);
  src_d = src.data().get();
  dst_d = dst.data().get();

  EXPECT_EQ(hipMemcpy(src_d, src_data, sizeof(uint64_t) * length, hipMemcpyHostToDevice),
            hipSuccess);
  EXPECT_EQ(hipMemcpy(dst_d, dst_data, sizeof(uint64_t) * length, hipMemcpyHostToDevice),
            hipSuccess);

  size_t unique_verts = 0;

  auto number_map = cugraph::detail::renumber_vertices(length,
                                                       src_d,
                                                       dst_d,
                                                       src_d,
                                                       dst_d,
                                                       &unique_verts,
                                                       cugraph::detail::HashFunctionObjectInt(511),
                                                       thrust::less<uint64_t>(),
                                                       rmm::mr::get_default_resource());

  EXPECT_EQ(hipMemcpy(
              tmp_map, number_map->data(), sizeof(uint64_t) * unique_verts, hipMemcpyDeviceToHost),
            hipSuccess);
  EXPECT_EQ(hipMemcpy(tmp_results, src_d, sizeof(uint64_t) * length, hipMemcpyDeviceToHost),
            hipSuccess);

  for (size_t i = 0; i < length; ++i) {
    EXPECT_EQ(tmp_results[i], src_expected[i]);
    EXPECT_EQ(tmp_map[tmp_results[i]], src_data[i]);
  }

  EXPECT_EQ(hipMemcpy(tmp_results, dst_d, sizeof(uint64_t) * length, hipMemcpyDeviceToHost),
            hipSuccess);
  for (size_t i = 0; i < length; ++i) {
    EXPECT_EQ(tmp_results[i], dst_expected[i]);
    EXPECT_EQ(tmp_map[tmp_results[i]], dst_data[i]);
  }
}

TEST_F(RenumberingTest, SmallFixedVertexList64BitTo32Bit)
{
  uint64_t src_data[] = {4U, 6U, 8U, 20U, 1U};
  uint64_t dst_data[] = {1U, 29U, 35U, 0U, 77U};

  uint32_t src_expected[] = {2U, 3U, 4U, 5U, 1U};
  uint32_t dst_expected[] = {1U, 6U, 7U, 0U, 8U};

  size_t length = sizeof(src_data) / sizeof(src_data[0]);

  uint64_t *src_d;
  uint64_t *dst_d;
  uint32_t *src_renumbered_d;
  uint32_t *dst_renumbered_d;

  uint32_t tmp_results[length];
  uint64_t tmp_map[2 * length];

  rmm::device_vector<uint64_t> src(length);
  rmm::device_vector<uint64_t> dst(length);
  src_d = src.data().get();
  dst_d = dst.data().get();
  rmm::device_vector<uint32_t> src_renumbered(length);
  rmm::device_vector<uint32_t> dst_renumbered(length);
  src_renumbered_d = src_renumbered.data().get();
  dst_renumbered_d = dst_renumbered.data().get();

  EXPECT_EQ(hipMemcpy(src_d, src_data, sizeof(uint64_t) * length, hipMemcpyHostToDevice),
            hipSuccess);
  EXPECT_EQ(hipMemcpy(dst_d, dst_data, sizeof(uint64_t) * length, hipMemcpyHostToDevice),
            hipSuccess);

  size_t unique_verts = 0;

  auto number_map = cugraph::detail::renumber_vertices(length,
                                                       src_d,
                                                       dst_d,
                                                       src_renumbered_d,
                                                       dst_renumbered_d,
                                                       &unique_verts,
                                                       cugraph::detail::HashFunctionObjectInt(511),
                                                       thrust::less<uint64_t>(),
                                                       rmm::mr::get_default_resource());

  EXPECT_EQ(hipMemcpy(
              tmp_map, number_map->data(), sizeof(uint64_t) * unique_verts, hipMemcpyDeviceToHost),
            hipSuccess);
  EXPECT_EQ(
    hipMemcpy(tmp_results, src_renumbered_d, sizeof(uint32_t) * length, hipMemcpyDeviceToHost),
    hipSuccess);

  for (size_t i = 0; i < length; ++i) {
    EXPECT_EQ(tmp_results[i], src_expected[i]);
    EXPECT_EQ(tmp_map[tmp_results[i]], src_data[i]);
  }

  EXPECT_EQ(
    hipMemcpy(tmp_results, dst_renumbered_d, sizeof(uint32_t) * length, hipMemcpyDeviceToHost),
    hipSuccess);
  for (size_t i = 0; i < length; ++i) {
    EXPECT_EQ(tmp_results[i], dst_expected[i]);
    EXPECT_EQ(tmp_map[tmp_results[i]], dst_data[i]);
  }
}

TEST_F(RenumberingTest, Random100KVertexSet)
{
  const int num_verts = 100000;

  uint64_t *src_d;
  uint64_t *dst_d;

  std::vector<uint64_t> src_data_vec(num_verts);
  std::vector<uint64_t> dst_data_vec(num_verts);
  std::vector<uint64_t> tmp_results_vec(num_verts);
  std::vector<uint64_t> tmp_map_vec(2 * num_verts);

  uint64_t *src_data    = src_data_vec.data();
  uint64_t *dst_data    = dst_data_vec.data();
  uint64_t *tmp_results = tmp_results_vec.data();
  uint64_t *tmp_map     = tmp_map_vec.data();
  rmm::device_vector<uint64_t> src(num_verts);
  rmm::device_vector<uint64_t> dst(num_verts);
  src_d = src.data().get();
  dst_d = dst.data().get();

  //
  //  Generate random source and vertex values
  //
  srand(43);

  for (int i = 0; i < num_verts; ++i) { src_data[i] = (uint64_t)rand(); }

  for (int i = 0; i < num_verts; ++i) { dst_data[i] = (uint64_t)rand(); }

  EXPECT_EQ(hipMemcpy(src_d, src_data, sizeof(uint64_t) * num_verts, hipMemcpyHostToDevice),
            hipSuccess);
  EXPECT_EQ(hipMemcpy(dst_d, dst_data, sizeof(uint64_t) * num_verts, hipMemcpyHostToDevice),
            hipSuccess);

  //
  //  Renumber everything
  //
  size_t unique_verts = 0;
  size_t n_verts{num_verts};

  auto start = std::chrono::system_clock::now();

  auto number_map = cugraph::detail::renumber_vertices(n_verts,
                                                       src_d,
                                                       dst_d,
                                                       src_d,
                                                       dst_d,
                                                       &unique_verts,
                                                       cugraph::detail::HashFunctionObjectInt(511),
                                                       thrust::less<uint64_t>(),
                                                       rmm::mr::get_default_resource());

  auto end                                      = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end - start;

  std::cout << "Renumber kernel elapsed time (ms): " << elapsed_seconds.count() * 1000 << std::endl;

  EXPECT_EQ(hipMemcpy(
              tmp_map, number_map->data(), sizeof(uint64_t) * unique_verts, hipMemcpyDeviceToHost),
            hipSuccess);
  EXPECT_EQ(hipMemcpy(tmp_results, src_d, sizeof(uint64_t) * num_verts, hipMemcpyDeviceToHost),
            hipSuccess);

  size_t min_id = unique_verts;
  size_t max_id = 0;

  size_t cnt = 0;
  for (size_t i = 0; i < num_verts; ++i) {
    min_id = min(min_id, tmp_results[i]);
    max_id = max(max_id, tmp_results[i]);
    if (tmp_map[tmp_results[i]] != src_data[i]) ++cnt;

    if (cnt < 20) EXPECT_EQ(tmp_map[tmp_results[i]], src_data[i]);
  }

  if (cnt > 0) printf("  src error count = %ld out of %d\n", cnt, num_verts);

  EXPECT_EQ(hipMemcpy(tmp_results, dst_d, sizeof(uint64_t) * num_verts, hipMemcpyDeviceToHost),
            hipSuccess);
  for (size_t i = 0; i < num_verts; ++i) {
    min_id = min(min_id, tmp_results[i]);
    max_id = max(max_id, tmp_results[i]);
    if (tmp_map[tmp_results[i]] != dst_data[i]) ++cnt;

    if (cnt < 20) EXPECT_EQ(tmp_map[tmp_results[i]], dst_data[i]);
  }

  if (cnt > 0) printf("  src error count = %ld out of %d\n", cnt, num_verts);

  EXPECT_EQ(min_id, 0);
  EXPECT_EQ(max_id, (unique_verts - 1));
}

TEST_F(RenumberingTest, Random10MVertexSet)
{
  const int num_verts = 10000000;

  //  A sampling of performance on single Quadro GV100
  // const int hash_size =  32767;       // 238 ms
  // const int hash_size =  8191;      // 224 ms
  const int hash_size = 511;  // 224 ms

  uint32_t *src_d;
  uint32_t *dst_d;

  rmm::device_vector<uint32_t> src(num_verts);
  rmm::device_vector<uint32_t> dst(num_verts);
  src_d = src.data().get();
  dst_d = dst.data().get();

  //
  //  Init the random number generate
  //
  const int num_threads{64};
  hiprandState *state;

  rmm::device_vector<hiprandState> state_vals(num_threads);
  state = state_vals.data().get();
  setup_generator<<<num_threads, 1>>>(state);
  generate_sources<<<num_threads, 1>>>(state, num_verts, src_d);
  generate_destinations<<<num_threads, 1>>>(state, num_verts, src_d, dst_d);

  std::cout << "done with initialization" << std::endl;

  //
  //  Renumber everything
  //
  size_t unique_verts = 0;
  size_t n_verts{num_verts};

  auto start = std::chrono::system_clock::now();
  auto number_map =
    cugraph::detail::renumber_vertices(n_verts,
                                       src_d,
                                       dst_d,
                                       src_d,
                                       dst_d,
                                       &unique_verts,
                                       cugraph::detail::HashFunctionObjectInt(hash_size),
                                       thrust::less<uint64_t>(),
                                       rmm::mr::get_default_resource());
  auto end                                      = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end - start;

  std::cout << "Renumber kernel elapsed time (ms): " << elapsed_seconds.count() * 1000 << std::endl;
  std::cout << "  unique verts = " << unique_verts << std::endl;
  std::cout << "  hash size = " << hash_size << std::endl;
}

TEST_F(RenumberingTest, Random100MVertexSet)
{
  const int num_verts = 100000000;

  //  A sampling of performance on single Quadro GV100
  // const int hash_size =  8192;        // 1811 ms
  // const int hash_size =  16384;       // 1746 ms
  // const int hash_size =  32768;       // 1662 ms
  // const int hash_size =  65536;       // 1569 ms
  // const int hash_size =  16777216;      // 1328 ms
  const int hash_size = 511;

  uint32_t *src_d;
  uint32_t *dst_d;

  rmm::device_vector<uint32_t> src(num_verts);
  rmm::device_vector<uint32_t> dst(num_verts);
  src_d = src.data().get();
  dst_d = dst.data().get();

  //
  //  Init the random number generate
  //
  const int num_threads{64};
  hiprandState *state;

  rmm::device_vector<hiprandState> state_vals(num_threads);
  state = state_vals.data().get();
  setup_generator<<<num_threads, 1>>>(state);
  generate_sources<<<num_threads, 1>>>(state, num_verts, src_d);
  generate_destinations<<<num_threads, 1>>>(state, num_verts, src_d, dst_d);

  std::cout << "done with initialization" << std::endl;

  //
  //  Renumber everything
  //
  size_t unique_verts = 0;
  size_t n_verts{num_verts};

  auto start = std::chrono::system_clock::now();
  auto number_map =
    cugraph::detail::renumber_vertices(n_verts,
                                       src_d,
                                       dst_d,
                                       src_d,
                                       dst_d,
                                       &unique_verts,
                                       cugraph::detail::HashFunctionObjectInt(hash_size),
                                       thrust::less<uint64_t>(),
                                       rmm::mr::get_default_resource());
  auto end                                      = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end - start;

  std::cout << "Renumber kernel elapsed time (ms): " << elapsed_seconds.count() * 1000 << std::endl;
  std::cout << "  unique verts = " << unique_verts << std::endl;
  std::cout << "  hash size = " << hash_size << std::endl;
}

TEST_F(RenumberingTest, Random500MVertexSet)
{
  const int num_verts = 500000000;

  //  A sampling of performance on single Quadro GV100
  // const int hash_size =  8192;      // 9918 ms
  // const int hash_size =  16384;      // 9550 ms
  // const int hash_size =  32768;      // 9146 ms
  // const int hash_size =  131072;      // 8537 ms
  const int hash_size = 1048576;  // 7335 ms
  // const int hash_size =  511;      // 7335 ms

  uint32_t *src_d;
  uint32_t *dst_d;

  rmm::device_vector<uint32_t> src(num_verts);
  rmm::device_vector<uint32_t> dst(num_verts);
  src_d = src.data().get();
  dst_d = dst.data().get();

  //
  //  Init the random number generate
  //
  const int num_threads{64};
  hiprandState *state;

  rmm::device_vector<hiprandState> state_vals(num_threads);
  state = state_vals.data().get();
  setup_generator<<<num_threads, 1>>>(state);
  generate_sources<<<num_threads, 1>>>(state, num_verts, src_d);
  generate_destinations<<<num_threads, 1>>>(state, num_verts, src_d, dst_d);

  std::cout << "done with initialization" << std::endl;

  //
  //  Renumber everything
  //
  size_t unique_verts = 0;
  size_t n_verts{num_verts};

  auto start = std::chrono::system_clock::now();
  auto number_map =
    cugraph::detail::renumber_vertices(n_verts,
                                       src_d,
                                       dst_d,
                                       src_d,
                                       dst_d,
                                       &unique_verts,
                                       cugraph::detail::HashFunctionObjectInt(hash_size),
                                       thrust::less<uint64_t>(),
                                       rmm::mr::get_default_resource());
  auto end                                      = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end - start;

  std::cout << "Renumber kernel elapsed time (ms): " << elapsed_seconds.count() * 1000 << std::endl;
  std::cout << "  unique verts = " << unique_verts << std::endl;
  std::cout << "  hash size = " << hash_size << std::endl;
}

int main(int argc, char **argv)
{
  testing::InitGoogleTest(&argc, argv);
  auto resource = std::make_unique<rmm::mr::cuda_memory_resource>();
  rmm::mr::set_default_resource(resource.get());
  int rc = RUN_ALL_TESTS();
  return rc;
}
