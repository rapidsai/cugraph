#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <utilities/high_res_clock.h>
#include <utilities/base_fixture.hpp>
#include <utilities/device_comm_wrapper.hpp>
#include <utilities/test_graphs.hpp>
#include <utilities/test_utilities.hpp>
#include <utilities/thrust_wrapper.hpp>

#include <cugraph/algorithms.hpp>
#include <cugraph/partition_manager.hpp>

#include <cuco/detail/hash_functions.cuh>
#include <cugraph/graph_view.hpp>
#include <cugraph/matrix_partition_view.hpp>
#include <cugraph/prims/copy_to_adj_matrix_row_col.cuh>
#include <cugraph/prims/transform_reduce_e.cuh>

#include <thrust/count.h>
#include <raft/comms/comms.hpp>
#include <raft/comms/mpi_comms.hpp>
#include <raft/handle.hpp>
#include <rmm/device_scalar.hpp>
#include <rmm/device_uvector.hpp>

#include <gtest/gtest.h>

#include <random>

template <typename vertex_t, typename... Args>
struct property_transform : public thrust::unary_function<vertex_t, thrust::tuple<Args...>> {
  int mod{};
  property_transform(int mod_count) : mod(mod_count) {}
  constexpr __device__ auto operator()(const vertex_t& val)
  {
    cuco::detail::MurmurHash3_32<vertex_t> hash_func{};
    auto value = hash_func(val) % mod;
    return thrust::make_tuple(static_cast<Args>(value)...);
  }
};
template <typename vertex_t, template <typename...> typename Tuple, typename... Args>
struct property_transform<vertex_t, Tuple<Args...>> : public property_transform<vertex_t, Args...> {
};

template <typename Tuple, std::size_t... I>
auto make_iterator_tuple(Tuple& data, std::index_sequence<I...>)
{
  return thrust::make_tuple((std::get<I>(data).begin())...);
}

template <typename... Args>
auto get_zip_iterator(std::tuple<Args...>& data)
{
  return thrust::make_zip_iterator(make_iterator_tuple(
    data, std::make_index_sequence<std::tuple_size<std::tuple<Args...>>::value>()));
}

template <typename T>
auto get_property_iterator(std::tuple<T>& data)
{
  return (std::get<0>(data)).begin();
}

template <typename T0, typename... Args>
auto get_property_iterator(std::tuple<T0, Args...>& data)
{
  return get_zip_iterator(data);
}

template <typename... Args>
struct generate_impl {
  static thrust::tuple<Args...> initial_value(int init)
  {
    return thrust::make_tuple(static_cast<Args>(init)...);
  }
  template <typename label_t>
  static std::tuple<rmm::device_uvector<Args>...> vertex_property(rmm::device_uvector<label_t>& labels,
                                                           int hash_bin_count,
                                                           raft::handle_t const& handle)
  {
    auto data = std::make_tuple(rmm::device_uvector<Args>(labels.size(), handle.get_stream())...);
    auto zip  = get_zip_iterator(data);
    thrust::transform(handle.get_thrust_policy(),
                      labels.begin(),
                      labels.end(),
                      zip,
                      property_transform<label_t, Args...>(hash_bin_count));
    return data;
  }
  template <typename label_t>
  static std::tuple<rmm::device_uvector<Args>...> vertex_property(thrust::counting_iterator<label_t> begin,
                                                           thrust::counting_iterator<label_t> end,
                                                           int hash_bin_count,
                                                           raft::handle_t const& handle)
  {
    auto length = thrust::distance(begin, end);
    auto data   = std::make_tuple(rmm::device_uvector<Args>(length, handle.get_stream())...);
    auto zip    = get_zip_iterator(data);
    thrust::transform(handle.get_thrust_policy(),
                      begin,
                      end,
                      zip,
                      property_transform<label_t, Args...>(hash_bin_count));
    return data;
  }
  template <typename Op, typename T1, typename T2, std::size_t... I>
    static constexpr void copy_property_impl(Op&& op, T1&& t1, T2&& t2, std::index_sequence<I...>)
    {
      (op(std::get<I>(t1), std::get<I>(t2)), ...);
    }

  template <typename Op, typename Tuple>
    static void
    copy_property(Tuple const& property,
                  Tuple& output_property,
                  Op op)
    {
      copy_property_impl(op, property, output_property, std::make_index_sequence<std::tuple_size<Tuple>::value>());
    }

  template <typename GraphViewType>
    static std::tuple<rmm::device_uvector<Args>...>
    column_property(raft::handle_t const& handle,
                    GraphViewType const& graph_view,
                    std::tuple<rmm::device_uvector<Args>...>& property)
    {
      if (true) {
        std::cerr<<"ERR DEBUG MESSAGE "<<graph_view.get_number_of_local_adj_matrix_partition_cols()<<" "<<
          graph_view.get_number_of_local_vertices()<<"\n";
        hipDeviceSynchronize();
      }
      auto output_property = std::make_tuple(rmm::device_uvector<Args>(graph_view.get_number_of_local_adj_matrix_partition_cols(), handle.get_stream())...);
      copy_property(property, output_property,
                    [&handle, &graph_view] (const auto& in, auto& out) {
                      copy_to_adj_matrix_col(handle, graph_view, in.begin(), out.begin());
                    }
                   );
      return output_property;
    }

  template <typename GraphViewType>
    static std::tuple<rmm::device_uvector<Args>...>
    row_property(raft::handle_t const& handle,
                    GraphViewType const& graph_view,
                    std::tuple<rmm::device_uvector<Args>...>& property)
    {
      if (true) {
        std::cerr<<"ERR DEBUG MESSAGE "<<graph_view.get_number_of_local_adj_matrix_partition_rows()<<" "<<
          graph_view.get_number_of_local_vertices()<<"\n";
        hipDeviceSynchronize();
      }
      auto output_property = std::make_tuple(rmm::device_uvector<Args>(graph_view.get_number_of_local_adj_matrix_partition_rows(), handle.get_stream())...);
      copy_property(property, output_property,
                    [&handle, &graph_view] (const auto& in, auto& out) {
                      copy_to_adj_matrix_row(handle, graph_view, in.begin(), out.begin());
                    }
                   );
      return output_property;
    }

};

template <typename T>
struct result_compare {
  static constexpr double threshold_ratio{1e-3};
  constexpr auto operator()(const T& t1, const T& t2)
  {
    if constexpr (std::is_floating_point_v<T>) {
      return std::abs(t1 - t2) < (std::max(t1, t2) * threshold_ratio);
    }
    return t1 == t2;
  }
};

template <typename... Args>
struct result_compare<thrust::tuple<Args...>> {
  static constexpr double threshold_ratio{1e-3};

  using Type = thrust::tuple<Args...>;
  constexpr auto operator()(const Type& t1, const Type& t2)
  {
    return equality_impl(t1, t2, std::make_index_sequence<thrust::tuple_size<Type>::value>());
  }

 private:
  template <typename T>
  constexpr bool equal(T t1, T t2)
  {
    if constexpr (std::is_floating_point_v<T>) {
      return std::abs(t1 - t2) < (std::max(t1, t2) * threshold_ratio);
    }
    return t1 == t2;
  }
  template <typename T, std::size_t... I>
  constexpr auto equality_impl(T& t1, T& t2, std::index_sequence<I...>)
  {
    return (... && (equal(thrust::get<I>(t1), thrust::get<I>(t2))));
  }
};

template <typename T>
struct generate : public generate_impl<T> {
  static T initial_value(int init) { return static_cast<T>(init); }
};
template <typename... Args>
struct generate<std::tuple<Args...>> : public generate_impl<Args...> {
};

struct Prims_Usecase {
  bool check_correctness{true};
  bool test_weighted{false};
};

template <typename input_usecase_t>
class Tests_MG_TransformReduceE
  : public ::testing::TestWithParam<std::tuple<Prims_Usecase, input_usecase_t>> {
 public:
  Tests_MG_TransformReduceE() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}

  // Compare the results of reduce_if_v primitive and thrust reduce on a single GPU
  template <typename vertex_t,
            typename edge_t,
            typename weight_t,
            typename result_t,
            bool store_transposed>
  void run_current_test(Prims_Usecase const& prims_usecase, input_usecase_t const& input_usecase)
  {
    // 1. initialize handle

    raft::handle_t handle{};
    HighResClock hr_clock{};

    raft::comms::initialize_mpi_comms(&handle, MPI_COMM_WORLD);
    auto& comm           = handle.get_comms();
    auto const comm_size = comm.get_size();
    auto const comm_rank = comm.get_rank();

    auto row_comm_size = static_cast<int>(sqrt(static_cast<double>(comm_size)));
    while (comm_size % row_comm_size != 0) {
      --row_comm_size;
    }
    cugraph::partition_2d::subcomm_factory_t<cugraph::partition_2d::key_naming_t, vertex_t>
      subcomm_factory(handle, row_comm_size);

    // 2. create MG graph

    if (cugraph::test::g_perf) {
      CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      hr_clock.start();
    }
    auto [mg_graph, d_mg_renumber_map_labels] =
      input_usecase.template construct_graph<vertex_t, edge_t, weight_t, store_transposed, true>(
        handle, prims_usecase.test_weighted, true);

    if (cugraph::test::g_perf) {
      CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      double elapsed_time{0.0};
      hr_clock.stop(&elapsed_time);
      std::cout << "MG construct_graph took " << elapsed_time * 1e-6 << " s.\n";
    }

    auto mg_graph_view = mg_graph.view();

    // 3. run MG transform reduce

    const int hash_bin_count = 5;
    const int initial_value  = 10;

    auto property_initial_value = generate<result_t>::initial_value(initial_value);
    using property_t   = decltype(property_initial_value);
    auto vertex_property_data =
      generate<result_t>::vertex_property((*d_mg_renumber_map_labels), hash_bin_count, handle);
    auto col_prop = generate<result_t>::column_property(handle, mg_graph_view, vertex_property_data);
    auto row_prop = generate<result_t>::row_property(handle, mg_graph_view, vertex_property_data);
    auto col_property_iter = get_property_iterator(col_prop);
    auto row_property_iter = get_property_iterator(row_prop);

    if (cugraph::test::g_perf) {
      CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      hr_clock.start();
    }

    auto result = transform_reduce_e(
      handle, mg_graph_view,
      row_property_iter,
      col_property_iter,
      [] __device__(auto row, auto col, weight_t wt, auto row_property, auto col_property) {
      if (row_property < col_property) {
        return row_property;
      } else {
        return col_property;
      }
      },
      property_initial_value);

    if (cugraph::test::g_perf) {
      CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      double elapsed_time{0.0};
      hr_clock.stop(&elapsed_time);
      std::cout << "MG transform reduce took " << elapsed_time * 1e-6 << " s.\n";
    }

    //// 4. compare SG & MG results

    if (prims_usecase.check_correctness) {
//      auto [sg_graph, d_sg_renumber_map_labels] =
//        input_usecase.template construct_graph<vertex_t, edge_t, weight_t, store_transposed, false>(
//          handle, true, false);
//      auto sg_graph_view = sg_graph.view();
//
//      auto sg_vertex_property_data =
//        generate<result_t>::vertex_property((*d_sg_renumber_map_labels), hash_bin_count, handle);
//      auto sg_col_prop = generate<result_t>::column_property(handle, sg_graph_view, sg_vertex_property_data);
//      auto sg_row_prop = generate<result_t>::row_property(handle, sg_graph_view, sg_vertex_property_data);
//      auto sg_col_property_iter = get_property_iterator(sg_col_prop);
//      auto sg_row_property_iter = get_property_iterator(sg_row_prop);
//
//      auto expected_result = transform_reduce_e(
//          handle, sg_graph_view,
//      sg_row_property_iter,
//      sg_col_property_iter,
//      [] __device__(auto row, auto col, weight_t wt, auto row_property, auto col_property) {
//      if (row_property < col_property) {
//        return row_property;
//      } else {
//        return col_property;
//      }
//      },
//      property_initial_value);
//      result_compare<property_t> compare{};
//      ASSERT_TRUE(compare(expected_result, result));
//
    }
  }
};

using Tests_MG_TransformReduceE_File = Tests_MG_TransformReduceE<cugraph::test::File_Usecase>;
using Tests_MG_TransformReduceE_Rmat = Tests_MG_TransformReduceE<cugraph::test::Rmat_Usecase>;

TEST_P(Tests_MG_TransformReduceE_File, CheckInt32Int32FloatTupleIntFloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, std::tuple<int, float>, false>(std::get<0>(param),
                                                                           std::get<1>(param));
}

TEST_P(Tests_MG_TransformReduceE_Rmat, CheckInt32Int32FloatTupleIntFloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, std::tuple<int, float>, false>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MG_TransformReduceE_File, CheckInt32Int32FloatTupleIntFloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, std::tuple<int, float>, true>(std::get<0>(param),
                                                                          std::get<1>(param));
}

TEST_P(Tests_MG_TransformReduceE_Rmat, CheckInt32Int32FloatTupleIntFloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, std::tuple<int, float>, true>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MG_TransformReduceE_File, CheckInt32Int32FloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, int, false>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MG_TransformReduceE_Rmat, CheckInt32Int32FloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, int, false>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MG_TransformReduceE_File, CheckInt32Int32FloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, int, true>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MG_TransformReduceE_Rmat, CheckInt32Int32FloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, int, true>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

INSTANTIATE_TEST_SUITE_P(
  file_test,
  Tests_MG_TransformReduceE_File,
  ::testing::Combine(
    ::testing::Values(Prims_Usecase{true}),
    ::testing::Values(cugraph::test::File_Usecase("test/datasets/karate.mtx"),
                      cugraph::test::File_Usecase("test/datasets/web-Google.mtx"),
                      cugraph::test::File_Usecase("test/datasets/ljournal-2008.mtx"),
                      cugraph::test::File_Usecase("test/datasets/webbase-1M.mtx"))));

INSTANTIATE_TEST_SUITE_P(
  rmat_small_test,
  Tests_MG_TransformReduceE_Rmat,
  ::testing::Combine(::testing::Values(Prims_Usecase{true}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       10, 16, 0.57, 0.19, 0.19, 0, false, false, 0, true))));

INSTANTIATE_TEST_SUITE_P(
  rmat_large_test,
  Tests_MG_TransformReduceE_Rmat,
  ::testing::Combine(::testing::Values(Prims_Usecase{false}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       20, 32, 0.57, 0.19, 0.19, 0, false, false, 0, true))));

CUGRAPH_MG_TEST_PROGRAM_MAIN()
