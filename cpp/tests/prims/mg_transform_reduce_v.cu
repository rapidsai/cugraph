#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "property_generator.cuh"

#include <utilities/base_fixture.hpp>
#include <utilities/device_comm_wrapper.hpp>
#include <utilities/mg_utilities.hpp>
#include <utilities/test_graphs.hpp>
#include <utilities/test_utilities.hpp>
#include <utilities/thrust_wrapper.hpp>

#include <prims/transform_reduce_v.cuh>

#include <cugraph/algorithms.hpp>
#include <cugraph/graph_view.hpp>
#include <cugraph/partition_manager.hpp>
#include <cugraph/utilities/high_res_timer.hpp>

#include <cuco/detail/hash_functions.cuh>

#include <raft/comms/mpi_comms.hpp>
#include <raft/core/comms.hpp>
#include <raft/core/handle.hpp>
#include <rmm/device_scalar.hpp>
#include <rmm/device_uvector.hpp>
#include <thrust/count.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/tuple.h>

#include <gtest/gtest.h>

#include <random>

template <typename vertex_t, typename property_t>
struct v_op_t {
  int32_t mod{};

  __device__ auto operator()(vertex_t, vertex_t val) const
  {
    cuco::detail::MurmurHash3_32<vertex_t> hash_func{};
    return cugraph::test::detail::make_property_value<property_t>(hash_func(val) % mod);
  }
};

template <typename T>
struct result_compare {
  static constexpr double threshold_ratio{1e-3};
  constexpr auto operator()(const T& t1, const T& t2)
  {
    if constexpr (std::is_floating_point_v<T>) {
      bool passed = (t1 == t2)  // when t1 == t2 == 0
                    ||
                    (std::abs(t1 - t2) < (std::max(std::abs(t1), std::abs(t2)) * threshold_ratio));
      return passed;
    }
    return t1 == t2;
  }
};

template <typename... Args>
struct result_compare<thrust::tuple<Args...>> {
  static constexpr double threshold_ratio{1e-3};

  using Type = thrust::tuple<Args...>;
  constexpr auto operator()(const Type& t1, const Type& t2)
  {
    return equality_impl(t1, t2, std::make_index_sequence<thrust::tuple_size<Type>::value>());
  }

 private:
  template <typename T>
  constexpr bool equal(T t1, T t2)
  {
    if constexpr (std::is_floating_point_v<T>) {
      bool passed = (t1 == t2)  // when t1 == t2 == 0
                    ||
                    (std::abs(t1 - t2) < (std::max(std::abs(t1), std::abs(t2)) * threshold_ratio));
      return passed;
    }
    return t1 == t2;
  }
  template <typename T, std::size_t... I>
  constexpr auto equality_impl(T& t1, T& t2, std::index_sequence<I...>)
  {
    return (... && (equal(thrust::get<I>(t1), thrust::get<I>(t2))));
  }
};

struct Prims_Usecase {
  bool check_correctness{true};
};

template <typename input_usecase_t>
class Tests_MGTransformReduceV
  : public ::testing::TestWithParam<std::tuple<Prims_Usecase, input_usecase_t>> {
 public:
  Tests_MGTransformReduceV() {}

  static void SetUpTestCase() { handle_ = cugraph::test::initialize_mg_handle(); }

  static void TearDownTestCase() { handle_.reset(); }

  virtual void SetUp() {}
  virtual void TearDown() {}

  // Compare the results of reduce_if_v primitive and thrust reduce on a single GPU
  template <typename vertex_t,
            typename edge_t,
            typename weight_t,
            typename result_t,
            bool store_transposed>
  void run_current_test(Prims_Usecase const& prims_usecase, input_usecase_t const& input_usecase)
  {
    HighResTimer hr_timer{};

    // 1. create MG graph

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle_->get_comms().barrier();
      hr_timer.start("MG Construct graph");
    }

    cugraph::graph_t<vertex_t, edge_t, store_transposed, true> mg_graph(*handle_);
    std::optional<rmm::device_uvector<vertex_t>> d_mg_renumber_map_labels{std::nullopt};
    std::tie(mg_graph, std::ignore, d_mg_renumber_map_labels) =
      cugraph::test::construct_graph<vertex_t, edge_t, weight_t, store_transposed, true>(
        *handle_, input_usecase, true, true);

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle_->get_comms().barrier();
      hr_timer.stop();
      hr_timer.display_and_clear(std::cout);
    }

    auto mg_graph_view = mg_graph.view();

    // 2. run MG transform reduce

    const int hash_bin_count = 5;
    const int initial_value  = 10;

    v_op_t<vertex_t, result_t> v_op{hash_bin_count};
    auto property_initial_value =
      cugraph::test::generate<vertex_t, result_t>::initial_value(initial_value);
    enum class reduction_type_t { PLUS, MINIMUM, MAXIMUM };
    std::array<reduction_type_t, 3> reduction_types = {
      reduction_type_t::PLUS, reduction_type_t::MINIMUM, reduction_type_t::MAXIMUM};

    std::unordered_map<reduction_type_t, result_t> results;

    for (auto reduction_type : reduction_types) {
      if (cugraph::test::g_perf) {
        RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
        handle_->get_comms().barrier();
        hr_timer.start("MG transform_reduce_v");
      }

      switch (reduction_type) {
        case reduction_type_t::PLUS:
          results[reduction_type] = transform_reduce_v(*handle_,
                                                       mg_graph_view,
                                                       (*d_mg_renumber_map_labels).begin(),
                                                       v_op,
                                                       property_initial_value,
                                                       cugraph::reduce_op::plus<result_t>{});
          break;
        case reduction_type_t::MINIMUM:
          results[reduction_type] = transform_reduce_v(*handle_,
                                                       mg_graph_view,
                                                       (*d_mg_renumber_map_labels).begin(),
                                                       v_op,
                                                       property_initial_value,
                                                       cugraph::reduce_op::minimum<result_t>{});
          break;
        case reduction_type_t::MAXIMUM:
          results[reduction_type] = transform_reduce_v(*handle_,
                                                       mg_graph_view,
                                                       (*d_mg_renumber_map_labels).begin(),
                                                       v_op,
                                                       property_initial_value,
                                                       cugraph::reduce_op::maximum<result_t>{});
          break;
        default: FAIL() << "should not be reached.";
      }

      if (cugraph::test::g_perf) {
        RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
        handle_->get_comms().barrier();
        hr_timer.stop();
        hr_timer.display_and_clear(std::cout);
      }
    }

    // 3. compare SG & MG results

    if (prims_usecase.check_correctness) {
      cugraph::graph_t<vertex_t, edge_t, store_transposed, false> sg_graph(*handle_);
      std::tie(sg_graph, std::ignore, std::ignore) =
        cugraph::test::construct_graph<vertex_t, edge_t, weight_t, store_transposed, false>(
          *handle_, input_usecase, true, false);
      auto sg_graph_view = sg_graph.view();

      for (auto reduction_type : reduction_types) {
        result_t expected_result{};
        switch (reduction_type) {
          case reduction_type_t::PLUS:
            expected_result = transform_reduce_v(
              *handle_,
              sg_graph_view,
              thrust::make_counting_iterator(sg_graph_view.local_vertex_partition_range_first()),
              v_op,
              property_initial_value,
              cugraph::reduce_op::plus<result_t>{});
            break;
          case reduction_type_t::MINIMUM:
            expected_result = transform_reduce_v(
              *handle_,
              sg_graph_view,
              thrust::make_counting_iterator(sg_graph_view.local_vertex_partition_range_first()),
              v_op,
              property_initial_value,
              cugraph::reduce_op::minimum<result_t>{});
            break;
          case reduction_type_t::MAXIMUM:
            expected_result = transform_reduce_v(
              *handle_,
              sg_graph_view,
              thrust::make_counting_iterator(sg_graph_view.local_vertex_partition_range_first()),
              v_op,
              property_initial_value,
              cugraph::reduce_op::maximum<result_t>{});
            break;
          default: FAIL() << "should not be reached.";
        }
        result_compare<result_t> compare{};
        ASSERT_TRUE(compare(expected_result, results[reduction_type]));
      }
    }
  }

 private:
  static std::unique_ptr<raft::handle_t> handle_;
};

template <typename input_usecase_t>
std::unique_ptr<raft::handle_t> Tests_MGTransformReduceV<input_usecase_t>::handle_ = nullptr;

using Tests_MGTransformReduceV_File = Tests_MGTransformReduceV<cugraph::test::File_Usecase>;
using Tests_MGTransformReduceV_Rmat = Tests_MGTransformReduceV<cugraph::test::Rmat_Usecase>;

TEST_P(Tests_MGTransformReduceV_File, CheckInt32Int32FloatTupleIntFloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, thrust::tuple<int, float>, false>(std::get<0>(param),
                                                                              std::get<1>(param));
}

TEST_P(Tests_MGTransformReduceV_Rmat, CheckInt32Int32FloatTupleIntFloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, thrust::tuple<int, float>, false>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGTransformReduceV_Rmat, CheckInt32Int64FloatTupleIntFloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int64_t, float, thrust::tuple<int, float>, false>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGTransformReduceV_Rmat, CheckInt64Int64FloatTupleIntFloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t, float, thrust::tuple<int, float>, false>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGTransformReduceV_File, CheckInt32Int32FloatTupleIntFloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, thrust::tuple<int, float>, true>(std::get<0>(param),
                                                                             std::get<1>(param));
}

TEST_P(Tests_MGTransformReduceV_Rmat, CheckInt32Int32FloatTupleIntFloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, thrust::tuple<int, float>, true>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGTransformReduceV_Rmat, CheckInt32Int64FloatTupleIntFloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int64_t, float, thrust::tuple<int, float>, true>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGTransformReduceV_Rmat, CheckInt64Int64FloatTupleIntFloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t, float, thrust::tuple<int, float>, true>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGTransformReduceV_File, CheckInt32Int32FloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, int, false>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MGTransformReduceV_Rmat, CheckInt32Int32FloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, int, false>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGTransformReduceV_Rmat, CheckInt32Int64FloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int64_t, float, int, false>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGTransformReduceV_Rmat, CheckInt64Int64FloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t, float, int, false>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGTransformReduceV_File, CheckInt32Int32FloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, int, true>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MGTransformReduceV_Rmat, CheckInt32Int32FloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, int, true>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGTransformReduceV_Rmat, CheckInt32Int64FloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int64_t, float, int, true>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGTransformReduceV_Rmat, CheckInt64Int64FloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t, float, int, true>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

INSTANTIATE_TEST_SUITE_P(
  file_test,
  Tests_MGTransformReduceV_File,
  ::testing::Combine(
    ::testing::Values(Prims_Usecase{true}),
    ::testing::Values(cugraph::test::File_Usecase("test/datasets/karate.mtx"),
                      cugraph::test::File_Usecase("test/datasets/web-Google.mtx"),
                      cugraph::test::File_Usecase("test/datasets/ljournal-2008.mtx"),
                      cugraph::test::File_Usecase("test/datasets/webbase-1M.mtx"))));

INSTANTIATE_TEST_SUITE_P(
  rmat_small_test,
  Tests_MGTransformReduceV_Rmat,
  ::testing::Combine(::testing::Values(Prims_Usecase{true}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       10, 16, 0.57, 0.19, 0.19, 0, false, false, 0, true))));

INSTANTIATE_TEST_SUITE_P(
  rmat_benchmark_test, /* note that scale & edge factor can be overridden in benchmarking (with
                          --gtest_filter to select only the rmat_benchmark_test with a specific
                          vertex & edge type combination) by command line arguments and do not
                          include more than one Rmat_Usecase that differ only in scale or edge
                          factor (to avoid running same benchmarks more than once) */
  Tests_MGTransformReduceV_Rmat,
  ::testing::Combine(::testing::Values(Prims_Usecase{false}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       20, 32, 0.57, 0.19, 0.19, 0, false, false, 0, true))));

CUGRAPH_MG_TEST_PROGRAM_MAIN()
