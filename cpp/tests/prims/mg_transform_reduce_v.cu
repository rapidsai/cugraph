#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <utilities/base_fixture.hpp>
#include <utilities/device_comm_wrapper.hpp>
#include <utilities/high_res_clock.h>
#include <utilities/test_graphs.hpp>
#include <utilities/test_utilities.hpp>
#include <utilities/thrust_wrapper.hpp>

#include <cugraph/algorithms.hpp>
#include <cugraph/partition_manager.hpp>

#include <cuco/detail/hash_functions.cuh>
#include <cugraph/graph_view.hpp>
#include <cugraph/prims/transform_reduce_v.cuh>

#include <raft/comms/comms.hpp>
#include <raft/comms/mpi_comms.hpp>
#include <raft/handle.hpp>
#include <rmm/device_scalar.hpp>
#include <rmm/device_uvector.hpp>
#include <thrust/count.h>

#include <gtest/gtest.h>

#include <random>

template <typename vertex_t, typename T>
struct property_transform : public thrust::unary_function<vertex_t, T> {
  int mod{};
  property_transform(int mod_count) : mod(mod_count) {}
  constexpr __device__ auto operator()(const vertex_t& val)
  {
    cuco::detail::MurmurHash3_32<vertex_t> hash_func{};
    auto value = hash_func(val) % mod;
    return static_cast<T>(value);
  }
};

template <typename vertex_t, typename... Args>
struct property_transform<vertex_t, std::tuple<Args...>>
  : public thrust::unary_function<vertex_t, thrust::tuple<Args...>> {
  int mod{};
  property_transform(int mod_count) : mod(mod_count) {}
  constexpr __device__ auto operator()(const vertex_t& val)
  {
    cuco::detail::MurmurHash3_32<vertex_t> hash_func{};
    auto value = hash_func(val) % mod;
    return thrust::make_tuple(static_cast<Args>(value)...);
  }
};

template <typename T>
struct result_compare {
  static constexpr double threshold_ratio{1e-3};
  constexpr auto operator()(const T& t1, const T& t2)
  {
    if constexpr (std::is_floating_point_v<T>) {
      bool passed = (t1 == t2)  // when t1 == t2 == 0
                    ||
                    (std::abs(t1 - t2) < (std::max(std::abs(t1), std::abs(t2)) * threshold_ratio));
      return passed;
    }
    return t1 == t2;
  }
};

template <typename... Args>
struct result_compare<thrust::tuple<Args...>> {
  static constexpr double threshold_ratio{1e-3};

  using Type = thrust::tuple<Args...>;
  constexpr auto operator()(const Type& t1, const Type& t2)
  {
    return equality_impl(t1, t2, std::make_index_sequence<thrust::tuple_size<Type>::value>());
  }

 private:
  template <typename T>
  constexpr bool equal(T t1, T t2)
  {
    if constexpr (std::is_floating_point_v<T>) {
      bool passed = (t1 == t2)  // when t1 == t2 == 0
                    ||
                    (std::abs(t1 - t2) < (std::max(std::abs(t1), std::abs(t2)) * threshold_ratio));
      return passed;
    }
    return t1 == t2;
  }
  template <typename T, std::size_t... I>
  constexpr auto equality_impl(T& t1, T& t2, std::index_sequence<I...>)
  {
    return (... && (equal(thrust::get<I>(t1), thrust::get<I>(t2))));
  }
};

template <typename T>
struct generate {
  static T initial_value(int init) { return static_cast<T>(init); }
};
template <typename... Args>
struct generate<std::tuple<Args...>> {
  static thrust::tuple<Args...> initial_value(int init)
  {
    return thrust::make_tuple(static_cast<Args>(init)...);
  }
};

struct Prims_Usecase {
  bool check_correctness{true};
};

template <typename input_usecase_t>
class Tests_MG_TransformReduceV
  : public ::testing::TestWithParam<std::tuple<Prims_Usecase, input_usecase_t>> {
 public:
  Tests_MG_TransformReduceV() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}

  // Compare the results of reduce_if_v primitive and thrust reduce on a single GPU
  template <typename vertex_t,
            typename edge_t,
            typename weight_t,
            typename result_t,
            bool store_transposed>
  void run_current_test(Prims_Usecase const& prims_usecase, input_usecase_t const& input_usecase)
  {
    // 1. initialize handle

    raft::handle_t handle{};
    HighResClock hr_clock{};

    raft::comms::initialize_mpi_comms(&handle, MPI_COMM_WORLD);
    auto& comm           = handle.get_comms();
    auto const comm_size = comm.get_size();
    auto const comm_rank = comm.get_rank();

    auto row_comm_size = static_cast<int>(sqrt(static_cast<double>(comm_size)));
    while (comm_size % row_comm_size != 0) {
      --row_comm_size;
    }
    cugraph::partition_2d::subcomm_factory_t<cugraph::partition_2d::key_naming_t, vertex_t>
      subcomm_factory(handle, row_comm_size);

    // 2. create MG graph

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      hr_clock.start();
    }
    auto [mg_graph, d_mg_renumber_map_labels] =
      cugraph::test::construct_graph<vertex_t, edge_t, weight_t, store_transposed, true>(
        handle, input_usecase, true, true);

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle.get_comms().barrier();
      double elapsed_time{0.0};
      hr_clock.stop(&elapsed_time);
      std::cout << "MG construct_graph took " << elapsed_time * 1e-6 << " s.\n";
    }

    auto mg_graph_view = mg_graph.view();

    // 3. run MG transform reduce

    const int hash_bin_count = 5;
    const int initial_value  = 10;

    property_transform<vertex_t, result_t> prop(hash_bin_count);
    auto property_initial_value = generate<result_t>::initial_value(initial_value);
    using property_t            = decltype(property_initial_value);
    raft::comms::op_t ops[]     = {
      raft::comms::op_t::SUM, raft::comms::op_t::MIN, raft::comms::op_t::MAX};

    std::unordered_map<raft::comms::op_t, property_t> results;

    for (auto op : ops) {
      if (cugraph::test::g_perf) {
        RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
        handle.get_comms().barrier();
        hr_clock.start();
      }

      results[op] = transform_reduce_v(
        handle, mg_graph_view, d_mg_renumber_map_labels->begin(), prop, property_initial_value, op);

      if (cugraph::test::g_perf) {
        RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
        handle.get_comms().barrier();
        double elapsed_time{0.0};
        hr_clock.stop(&elapsed_time);
        std::cout << "MG transform reduce took " << elapsed_time * 1e-6 << " s.\n";
      }
    }

    //// 4. compare SG & MG results

    if (prims_usecase.check_correctness) {
      cugraph::graph_t<vertex_t, edge_t, weight_t, store_transposed, false> sg_graph(handle);
      std::tie(sg_graph, std::ignore) =
        cugraph::test::construct_graph<vertex_t, edge_t, weight_t, store_transposed, false>(
          handle, input_usecase, true, false);
      auto sg_graph_view = sg_graph.view();

      for (auto op : ops) {
        auto expected_result = cugraph::op_dispatch<property_t>(
          op, [&handle, &sg_graph_view, prop, property_initial_value](auto op) {
            return thrust::transform_reduce(
              handle.get_thrust_policy(),
              thrust::make_counting_iterator(sg_graph_view.local_vertex_partition_range_first()),
              thrust::make_counting_iterator(sg_graph_view.local_vertex_partition_range_last()),
              prop,
              property_initial_value,
              op);
          });
        result_compare<property_t> compare{};
        ASSERT_TRUE(compare(expected_result, results[op]));
      }
    }
  }
};

using Tests_MG_TransformReduceV_File = Tests_MG_TransformReduceV<cugraph::test::File_Usecase>;
using Tests_MG_TransformReduceV_Rmat = Tests_MG_TransformReduceV<cugraph::test::Rmat_Usecase>;

TEST_P(Tests_MG_TransformReduceV_File, CheckInt32Int32FloatTupleIntFloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, std::tuple<int, float>, false>(std::get<0>(param),
                                                                           std::get<1>(param));
}

TEST_P(Tests_MG_TransformReduceV_Rmat, CheckInt32Int32FloatTupleIntFloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, std::tuple<int, float>, false>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MG_TransformReduceV_File, CheckInt32Int32FloatTupleIntFloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, std::tuple<int, float>, true>(std::get<0>(param),
                                                                          std::get<1>(param));
}

TEST_P(Tests_MG_TransformReduceV_Rmat, CheckInt32Int32FloatTupleIntFloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, std::tuple<int, float>, true>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MG_TransformReduceV_File, CheckInt32Int32FloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, int, false>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MG_TransformReduceV_Rmat, CheckInt32Int32FloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, int, false>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MG_TransformReduceV_File, CheckInt32Int32FloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, int, true>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MG_TransformReduceV_Rmat, CheckInt32Int32FloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, int, true>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

INSTANTIATE_TEST_SUITE_P(
  file_test,
  Tests_MG_TransformReduceV_File,
  ::testing::Combine(
    ::testing::Values(Prims_Usecase{true}),
    ::testing::Values(cugraph::test::File_Usecase("test/datasets/karate.mtx"),
                      cugraph::test::File_Usecase("test/datasets/web-Google.mtx"),
                      cugraph::test::File_Usecase("test/datasets/ljournal-2008.mtx"),
                      cugraph::test::File_Usecase("test/datasets/webbase-1M.mtx"))));

INSTANTIATE_TEST_SUITE_P(
  rmat_small_test,
  Tests_MG_TransformReduceV_Rmat,
  ::testing::Combine(::testing::Values(Prims_Usecase{true}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       10, 16, 0.57, 0.19, 0.19, 0, false, false, 0, true))));

INSTANTIATE_TEST_SUITE_P(
  rmat_large_test,
  Tests_MG_TransformReduceV_Rmat,
  ::testing::Combine(::testing::Values(Prims_Usecase{false}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       20, 32, 0.57, 0.19, 0.19, 0, false, false, 0, true))));

CUGRAPH_MG_TEST_PROGRAM_MAIN()
