/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "prims/transform_e.cuh"
#include "prims/transform_reduce_dst_nbr_intersection_of_e_endpoints_by_v.cuh"
#include "prims/update_edge_src_dst_property.cuh"
#include "utilities/base_fixture.hpp"
#include "utilities/conversion_utilities.hpp"
#include "utilities/device_comm_wrapper.hpp"
#include "utilities/mg_utilities.hpp"
#include "utilities/property_generator_utilities.hpp"
#include "utilities/test_graphs.hpp"

#include <cugraph/detail/shuffle_wrappers.hpp>
#include <cugraph/edge_property.hpp>
#include <cugraph/edge_src_dst_property.hpp>
#include <cugraph/graph_functions.hpp>
#include <cugraph/graph_view.hpp>
#include <cugraph/utilities/dataframe_buffer.hpp>
#include <cugraph/utilities/high_res_timer.hpp>
#include <cugraph/utilities/host_scalar_comm.hpp>
#include <cugraph/utilities/thrust_tuple_utils.hpp>

#include <raft/comms/mpi_comms.hpp>
#include <raft/core/comms.hpp>
#include <raft/core/device_span.hpp>
#include <raft/core/handle.hpp>

#include <rmm/device_uvector.hpp>

#include <thrust/iterator/counting_iterator.h>
#include <thrust/tuple.h>

#include <gtest/gtest.h>

#include <random>

template <typename vertex_t, typename edge_t>
struct intersection_op_t {
  __device__ thrust::tuple<edge_t, edge_t, edge_t> operator()(
    vertex_t v0,
    vertex_t v1,
    edge_t v0_prop,
    edge_t v1_prop,
    raft::device_span<vertex_t const> intersection) const
  {
    return thrust::make_tuple(
      v0_prop + v1_prop, v0_prop + v1_prop, static_cast<edge_t>(intersection.size()));
  }
};

struct Prims_Usecase {
  bool edge_masking{false};
  bool check_correctness{true};
};

template <typename input_usecase_t>
class Tests_MGTransformReduceDstNbrIntersectionOfEEndpointsByV
  : public ::testing::TestWithParam<std::tuple<Prims_Usecase, input_usecase_t>> {
 public:
  Tests_MGTransformReduceDstNbrIntersectionOfEEndpointsByV() {}

  static void SetUpTestCase() { handle_ = cugraph::test::initialize_mg_handle(); }

  static void TearDownTestCase() { handle_.reset(); }

  virtual void SetUp() {}
  virtual void TearDown() {}

  // Verify the results of transform_reduce_dst_nbr_intersection_of_e_endpoints_by_v primitive
  template <typename vertex_t, typename edge_t, typename weight_t>
  void run_current_test(Prims_Usecase const& prims_usecase, input_usecase_t const& input_usecase)
  {
    HighResTimer hr_timer{};

    auto const comm_rank = handle_->get_comms().get_rank();
    auto const comm_size = handle_->get_comms().get_size();

    // 1. create MG graph

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle_->get_comms().barrier();
      hr_timer.start("MG Construct graph");
    }

    cugraph::graph_t<vertex_t, edge_t, false, true> mg_graph(*handle_);
    std::optional<rmm::device_uvector<vertex_t>> mg_renumber_map{std::nullopt};
    std::tie(mg_graph, std::ignore, mg_renumber_map) =
      cugraph::test::construct_graph<vertex_t, edge_t, weight_t, false, true>(
        *handle_, input_usecase, false, true);

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle_->get_comms().barrier();
      hr_timer.stop();
      hr_timer.display_and_clear(std::cout);
    }

    auto mg_graph_view = mg_graph.view();

    std::optional<cugraph::edge_property_t<decltype(mg_graph_view), bool>> edge_mask{std::nullopt};
    if (prims_usecase.edge_masking) {
      edge_mask = cugraph::test::generate<decltype(mg_graph_view), bool>::edge_property(
        *handle_, mg_graph_view, 2);
      mg_graph_view.attach_edge_mask((*edge_mask).view());
    }

    // 2. run MG transform_reduce_dst_nbr_intersection_of_e_endpoints_by_v primitive

    const int hash_bin_count = 5;
    const int initial_value  = 4;

    auto property_initial_value =
      cugraph::test::generate<decltype(mg_graph_view), edge_t>::initial_value(initial_value);

    auto mg_vertex_prop = cugraph::test::generate<decltype(mg_graph_view), edge_t>::vertex_property(
      *handle_, *mg_renumber_map, hash_bin_count);
    auto mg_src_prop = cugraph::test::generate<decltype(mg_graph_view), edge_t>::src_property(
      *handle_, mg_graph_view, mg_vertex_prop);
    auto mg_dst_prop = cugraph::test::generate<decltype(mg_graph_view), edge_t>::dst_property(
      *handle_, mg_graph_view, mg_vertex_prop);

    auto mg_result_buffer = rmm::device_uvector<edge_t>(
      mg_graph_view.local_vertex_partition_range_size(), handle_->get_stream());

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle_->get_comms().barrier();
      hr_timer.start("MG transform_reduce_dst_nbr_intersection_of_e_endpoints_by_v");
    }

    cugraph::transform_reduce_dst_nbr_intersection_of_e_endpoints_by_v(
      *handle_,
      mg_graph_view,
      mg_src_prop.view(),
      mg_dst_prop.view(),
      intersection_op_t<vertex_t, edge_t>{},
      property_initial_value,
      mg_result_buffer.begin());

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle_->get_comms().barrier();
      hr_timer.stop();
      hr_timer.display_and_clear(std::cout);
    }

    // 3. validate MG results

    if (prims_usecase.check_correctness) {
      rmm::device_uvector<edge_t> mg_aggregate_result_buffer(0, handle_->get_stream());
      std::tie(std::ignore, mg_aggregate_result_buffer) =
        cugraph::test::mg_vertex_property_values_to_sg_vertex_property_values(
          *handle_,
          std::make_optional<raft::device_span<vertex_t const>>((*mg_renumber_map).data(),
                                                                (*mg_renumber_map).size()),
          mg_graph_view.local_vertex_partition_range(),
          std::optional<raft::device_span<vertex_t const>>{std::nullopt},
          std::optional<raft::device_span<vertex_t const>>{std::nullopt},
          raft::device_span<edge_t const>(mg_result_buffer.data(), mg_result_buffer.size()));

      cugraph::graph_t<vertex_t, edge_t, false, false> sg_graph(*handle_);
      std::tie(sg_graph, std::ignore, std::ignore) = cugraph::test::mg_graph_to_sg_graph(
        *handle_,
        mg_graph_view,
        std::optional<cugraph::edge_property_view_t<edge_t, weight_t const*>>{std::nullopt},
        std::make_optional<raft::device_span<vertex_t const>>((*mg_renumber_map).data(),
                                                              (*mg_renumber_map).size()),
        false);

      if (handle_->get_comms().get_rank() == 0) {
        auto sg_graph_view = sg_graph.view();

        auto sg_vertex_prop =
          cugraph::test::generate<decltype(sg_graph_view), edge_t>::vertex_property(
            *handle_,
            thrust::make_counting_iterator(sg_graph_view.local_vertex_partition_range_first()),
            thrust::make_counting_iterator(sg_graph_view.local_vertex_partition_range_last()),
            hash_bin_count);
        auto sg_src_prop = cugraph::test::generate<decltype(sg_graph_view), edge_t>::src_property(
          *handle_, sg_graph_view, sg_vertex_prop);
        auto sg_dst_prop = cugraph::test::generate<decltype(sg_graph_view), edge_t>::dst_property(
          *handle_, sg_graph_view, sg_vertex_prop);

        auto sg_result_buffer = cugraph::allocate_dataframe_buffer<edge_t>(
          sg_graph_view.number_of_vertices(), handle_->get_stream());

        cugraph::transform_reduce_dst_nbr_intersection_of_e_endpoints_by_v(
          *handle_,
          sg_graph_view,
          sg_src_prop.view(),
          sg_dst_prop.view(),
          intersection_op_t<vertex_t, edge_t>{},
          property_initial_value,
          sg_result_buffer.begin());

        bool valid = thrust::equal(handle_->get_thrust_policy(),
                                   mg_aggregate_result_buffer.begin(),
                                   mg_aggregate_result_buffer.end(),
                                   sg_result_buffer.begin());

        ASSERT_TRUE(valid);
      }
    }
  }

 private:
  static std::unique_ptr<raft::handle_t> handle_;
};

template <typename input_usecase_t>
std::unique_ptr<raft::handle_t>
  Tests_MGTransformReduceDstNbrIntersectionOfEEndpointsByV<input_usecase_t>::handle_ = nullptr;

using Tests_MGTransformReduceDstNbrIntersectionOfEEndpointsByV_File =
  Tests_MGTransformReduceDstNbrIntersectionOfEEndpointsByV<cugraph::test::File_Usecase>;
using Tests_MGTransformReduceDstNbrIntersectionOfEEndpointsByV_Rmat =
  Tests_MGTransformReduceDstNbrIntersectionOfEEndpointsByV<cugraph::test::Rmat_Usecase>;

TEST_P(Tests_MGTransformReduceDstNbrIntersectionOfEEndpointsByV_File, CheckInt32Int32Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MGTransformReduceDstNbrIntersectionOfEEndpointsByV_Rmat, CheckInt32Int32Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGTransformReduceDstNbrIntersectionOfEEndpointsByV_Rmat, CheckInt32Int64Float)
{
  auto param = GetParam();
  run_current_test<int32_t, int64_t, float>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGTransformReduceDstNbrIntersectionOfEEndpointsByV_Rmat, CheckInt64Int64Float)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t, float>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

INSTANTIATE_TEST_SUITE_P(
  file_test,
  Tests_MGTransformReduceDstNbrIntersectionOfEEndpointsByV_File,
  ::testing::Combine(
    ::testing::Values(Prims_Usecase{false, true}, Prims_Usecase{true, true}),
    ::testing::Values(cugraph::test::File_Usecase("test/datasets/karate.mtx"),
                      cugraph::test::File_Usecase("test/datasets/netscience.mtx"))));

INSTANTIATE_TEST_SUITE_P(rmat_small_test,
                         Tests_MGTransformReduceDstNbrIntersectionOfEEndpointsByV_Rmat,
                         ::testing::Combine(::testing::Values(Prims_Usecase{false, true},
                                                              Prims_Usecase{true, true}),
                                            ::testing::Values(cugraph::test::Rmat_Usecase(
                                              10, 16, 0.57, 0.19, 0.19, 0, false, false))));

INSTANTIATE_TEST_SUITE_P(
  rmat_benchmark_test, /* note that scale & edge factor can be overridden in benchmarking (with
                          --gtest_filter to select only the rmat_benchmark_test with a specific
                          vertex & edge type combination) by command line arguments and do not
                          include more than one Rmat_Usecase that differ only in scale or edge
                          factor (to avoid running same benchmarks more than once) */
  Tests_MGTransformReduceDstNbrIntersectionOfEEndpointsByV_Rmat,
  ::testing::Combine(
    ::testing::Values(Prims_Usecase{false, false}, Prims_Usecase{true, false}),
    ::testing::Values(cugraph::test::Rmat_Usecase(20, 32, 0.57, 0.19, 0.19, 0, false, false))));

CUGRAPH_MG_TEST_PROGRAM_MAIN()
