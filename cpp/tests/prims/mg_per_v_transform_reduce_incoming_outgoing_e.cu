#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "property_generator.cuh"

#include <utilities/base_fixture.hpp>
#include <utilities/device_comm_wrapper.hpp>
#include <utilities/mg_utilities.hpp>
#include <utilities/test_graphs.hpp>
#include <utilities/test_utilities.hpp>
#include <utilities/thrust_wrapper.hpp>

#include <prims/per_v_transform_reduce_incoming_outgoing_e.cuh>
#include <prims/reduce_op.cuh>
#include <prims/update_edge_src_dst_property.cuh>

#include <cugraph/algorithms.hpp>
#include <cugraph/edge_partition_view.hpp>
#include <cugraph/edge_src_dst_property.hpp>
#include <cugraph/graph_view.hpp>
#include <cugraph/partition_manager.hpp>
#include <cugraph/utilities/dataframe_buffer.hpp>
#include <cugraph/utilities/high_res_timer.hpp>

#include <cuco/detail/hash_functions.cuh>

#include <raft/comms/mpi_comms.hpp>
#include <raft/core/comms.hpp>
#include <raft/core/handle.hpp>
#include <rmm/device_scalar.hpp>
#include <rmm/device_uvector.hpp>
#include <sstream>
#include <thrust/count.h>
#include <thrust/distance.h>
#include <thrust/equal.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>

#include <gtest/gtest.h>

#include <random>

template <typename vertex_t, typename result_t>
struct e_op_t {
  __device__ result_t operator()(vertex_t src,
                                 vertex_t dst,
                                 result_t src_property,
                                 result_t dst_property,
                                 thrust::nullopt_t) const
  {
    if (src_property < dst_property) {
      return src_property;
    } else {
      return dst_property;
    }
  }
};

template <typename T>
struct comparator {
  static constexpr double threshold_ratio{1e-2};
  __host__ __device__ bool operator()(T t1, T t2) const
  {
    if constexpr (std::is_floating_point_v<T>) {
      bool passed = (t1 == t2)  // when t1 == t2 == 0
                    ||
                    (std::abs(t1 - t2) < (std::max(std::abs(t1), std::abs(t2)) * threshold_ratio));
      return passed;
    }
    return t1 == t2;
  }
};

struct result_compare {
  const raft::handle_t& handle_;
  result_compare(raft::handle_t const& handle) : handle_(handle) {}

  template <typename... Args>
  auto operator()(const std::tuple<rmm::device_uvector<Args>...>& t1,
                  const std::tuple<rmm::device_uvector<Args>...>& t2)
  {
    using type = thrust::tuple<Args...>;
    return equality_impl(t1, t2, std::make_index_sequence<thrust::tuple_size<type>::value>());
  }

  template <typename T>
  auto operator()(const rmm::device_uvector<T>& t1, const rmm::device_uvector<T>& t2)
  {
    return thrust::equal(
      handle_.get_thrust_policy(), t1.begin(), t1.end(), t2.begin(), comparator<T>());
  }

 private:
  template <typename T, std::size_t... I>
  auto equality_impl(T& t1, T& t2, std::index_sequence<I...>)
  {
    return (... && (result_compare::operator()(std::get<I>(t1), std::get<I>(t2))));
  }
};

template <typename buffer_type>
buffer_type aggregate(const raft::handle_t& handle, const buffer_type& result)
{
  auto aggregated_result =
    cugraph::allocate_dataframe_buffer<cugraph::dataframe_element_t<buffer_type>>(
      0, handle.get_stream());
  cugraph::transform(result, aggregated_result, [&handle](auto& input, auto& output) {
    output = cugraph::test::device_gatherv(handle, input.data(), input.size());
  });
  return aggregated_result;
}

struct Prims_Usecase {
  bool check_correctness{true};
  bool test_weighted{false};
};

template <typename input_usecase_t>
class Tests_MGPerVTransformReduceIncomingOutgoingE
  : public ::testing::TestWithParam<std::tuple<Prims_Usecase, input_usecase_t>> {
 public:
  Tests_MGPerVTransformReduceIncomingOutgoingE() {}

  static void SetUpTestCase() { handle_ = cugraph::test::initialize_mg_handle(); }

  static void TearDownTestCase() { handle_.reset(); }

  virtual void SetUp() {}
  virtual void TearDown() {}

  // Compare the results of per_v_transform_reduce_incoming|outgoing_e primitive
  template <typename vertex_t,
            typename edge_t,
            typename weight_t,
            typename result_t,
            bool store_transposed>
  void run_current_test(Prims_Usecase const& prims_usecase, input_usecase_t const& input_usecase)
  {
    HighResTimer hr_timer{};

    // 1. create MG graph

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle_->get_comms().barrier();
      hr_timer.start("MG Construct graph");
    }

    cugraph::graph_t<vertex_t, edge_t, store_transposed, true> mg_graph(*handle_);
    std::optional<rmm::device_uvector<vertex_t>> d_mg_renumber_map_labels{std::nullopt};
    std::tie(mg_graph, std::ignore, d_mg_renumber_map_labels) =
      cugraph::test::construct_graph<vertex_t, edge_t, weight_t, store_transposed, true>(
        *handle_, input_usecase, prims_usecase.test_weighted, true);

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
      handle_->get_comms().barrier();
      hr_timer.stop();
      hr_timer.display_and_clear(std::cout);
    }

    auto mg_graph_view = mg_graph.view();

    // 2. run MG transform reduce

    const int hash_bin_count = 5;
    const int initial_value  = 4;

    auto property_initial_value =
      cugraph::test::generate<vertex_t, result_t>::initial_value(initial_value);

    auto mg_vertex_prop = cugraph::test::generate<vertex_t, result_t>::vertex_property(
      *handle_, *d_mg_renumber_map_labels, hash_bin_count);
    auto mg_src_prop = cugraph::test::generate<vertex_t, result_t>::src_property(
      *handle_, mg_graph_view, mg_vertex_prop);
    auto mg_dst_prop = cugraph::test::generate<vertex_t, result_t>::dst_property(
      *handle_, mg_graph_view, mg_vertex_prop);

    enum class reduction_type_t { PLUS, MINIMUM, MAXIMUM };
    std::array<reduction_type_t, 3> reduction_types = {
      reduction_type_t::PLUS, reduction_type_t::MINIMUM, reduction_type_t::MAXIMUM};

    std::vector<decltype(cugraph::allocate_dataframe_buffer<result_t>(0, rmm::cuda_stream_view{}))>
      out_results{};
    std::vector<decltype(cugraph::allocate_dataframe_buffer<result_t>(0, rmm::cuda_stream_view{}))>
      in_results{};
    out_results.reserve(reduction_types.size());
    in_results.reserve(reduction_types.size());

    for (size_t i = 0; i < reduction_types.size(); ++i) {
      in_results.push_back(cugraph::allocate_dataframe_buffer<result_t>(
        mg_graph_view.local_vertex_partition_range_size(), handle_->get_stream()));

      if (cugraph::test::g_perf) {
        RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
        handle_->get_comms().barrier();
        hr_timer.start("MG per_v_transform_reduce_incoming_e");
      }

      switch (reduction_types[i]) {
        case reduction_type_t::PLUS:
          per_v_transform_reduce_incoming_e(*handle_,
                                            mg_graph_view,
                                            mg_src_prop.view(),
                                            mg_dst_prop.view(),
                                            cugraph::edge_dummy_property_t{}.view(),
                                            e_op_t<vertex_t, result_t>{},
                                            property_initial_value,
                                            cugraph::reduce_op::plus<result_t>{},
                                            cugraph::get_dataframe_buffer_begin(in_results[i]));
          break;
        case reduction_type_t::MINIMUM:
          per_v_transform_reduce_incoming_e(*handle_,
                                            mg_graph_view,
                                            mg_src_prop.view(),
                                            mg_dst_prop.view(),
                                            cugraph::edge_dummy_property_t{}.view(),
                                            e_op_t<vertex_t, result_t>{},
                                            property_initial_value,
                                            cugraph::reduce_op::minimum<result_t>{},
                                            cugraph::get_dataframe_buffer_begin(in_results[i]));
          break;
        case reduction_type_t::MAXIMUM:
          per_v_transform_reduce_incoming_e(*handle_,
                                            mg_graph_view,
                                            mg_src_prop.view(),
                                            mg_dst_prop.view(),
                                            cugraph::edge_dummy_property_t{}.view(),
                                            e_op_t<vertex_t, result_t>{},
                                            property_initial_value,
                                            cugraph::reduce_op::maximum<result_t>{},
                                            cugraph::get_dataframe_buffer_begin(in_results[i]));
          break;
        default: FAIL() << "should not be reached.";
      }

      if (cugraph::test::g_perf) {
        RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
        handle_->get_comms().barrier();
        hr_timer.stop();
        hr_timer.display_and_clear(std::cout);
      }

      out_results.push_back(cugraph::allocate_dataframe_buffer<result_t>(
        mg_graph_view.local_vertex_partition_range_size(), handle_->get_stream()));

      if (cugraph::test::g_perf) {
        RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
        handle_->get_comms().barrier();
        hr_timer.start("MG per_v_transform_reduce_outgoing_e");
      }

      switch (reduction_types[i]) {
        case reduction_type_t::PLUS:
          per_v_transform_reduce_outgoing_e(*handle_,
                                            mg_graph_view,
                                            mg_src_prop.view(),
                                            mg_dst_prop.view(),
                                            cugraph::edge_dummy_property_t{}.view(),
                                            e_op_t<vertex_t, result_t>{},
                                            property_initial_value,
                                            cugraph::reduce_op::plus<result_t>{},
                                            cugraph::get_dataframe_buffer_begin(out_results[i]));
          break;
        case reduction_type_t::MINIMUM:
          per_v_transform_reduce_outgoing_e(*handle_,
                                            mg_graph_view,
                                            mg_src_prop.view(),
                                            mg_dst_prop.view(),
                                            cugraph::edge_dummy_property_t{}.view(),
                                            e_op_t<vertex_t, result_t>{},
                                            property_initial_value,
                                            cugraph::reduce_op::minimum<result_t>{},
                                            cugraph::get_dataframe_buffer_begin(out_results[i]));
          break;
        case reduction_type_t::MAXIMUM:
          per_v_transform_reduce_outgoing_e(*handle_,
                                            mg_graph_view,
                                            mg_src_prop.view(),
                                            mg_dst_prop.view(),
                                            cugraph::edge_dummy_property_t{}.view(),
                                            e_op_t<vertex_t, result_t>{},
                                            property_initial_value,
                                            cugraph::reduce_op::maximum<result_t>{},
                                            cugraph::get_dataframe_buffer_begin(out_results[i]));
          break;
        default: FAIL() << "should not be reached.";
      }

      if (cugraph::test::g_perf) {
        RAFT_CUDA_TRY(hipDeviceSynchronize());  // for consistent performance measurement
        handle_->get_comms().barrier();
        hr_timer.stop();
        hr_timer.display_and_clear(std::cout);
      }
    }

    // 3. compare SG & MG results

    if (prims_usecase.check_correctness) {
      cugraph::graph_t<vertex_t, edge_t, store_transposed, false> sg_graph(*handle_);
      std::tie(sg_graph, std::ignore, std::ignore) =
        cugraph::test::construct_graph<vertex_t, edge_t, weight_t, store_transposed, false>(
          *handle_, input_usecase, true, false);

      auto sg_graph_view = sg_graph.view();

      auto sg_vertex_prop = cugraph::test::generate<vertex_t, result_t>::vertex_property(
        *handle_,
        thrust::make_counting_iterator(sg_graph_view.local_vertex_partition_range_first()),
        thrust::make_counting_iterator(sg_graph_view.local_vertex_partition_range_last()),
        hash_bin_count);
      auto sg_dst_prop = cugraph::test::generate<vertex_t, result_t>::dst_property(
        *handle_, sg_graph_view, sg_vertex_prop);
      auto sg_src_prop = cugraph::test::generate<vertex_t, result_t>::src_property(
        *handle_, sg_graph_view, sg_vertex_prop);
      result_compare comp{*handle_};

      for (size_t i = 0; i < reduction_types.size(); ++i) {
        auto global_out_result = cugraph::allocate_dataframe_buffer<result_t>(
          sg_graph_view.local_vertex_partition_range_size(), handle_->get_stream());

        switch (reduction_types[i]) {
          case reduction_type_t::PLUS:
            per_v_transform_reduce_outgoing_e(
              *handle_,
              sg_graph_view,
              sg_src_prop.view(),
              sg_dst_prop.view(),
              cugraph::edge_dummy_property_t{}.view(),
              e_op_t<vertex_t, result_t>{},
              property_initial_value,
              cugraph::reduce_op::plus<result_t>{},
              cugraph::get_dataframe_buffer_begin(global_out_result));
            break;
          case reduction_type_t::MINIMUM:
            per_v_transform_reduce_outgoing_e(
              *handle_,
              sg_graph_view,
              sg_src_prop.view(),
              sg_dst_prop.view(),
              cugraph::edge_dummy_property_t{}.view(),
              e_op_t<vertex_t, result_t>{},
              property_initial_value,
              cugraph::reduce_op::minimum<result_t>{},
              cugraph::get_dataframe_buffer_begin(global_out_result));
            break;
          case reduction_type_t::MAXIMUM:
            per_v_transform_reduce_outgoing_e(
              *handle_,
              sg_graph_view,
              sg_src_prop.view(),
              sg_dst_prop.view(),
              cugraph::edge_dummy_property_t{}.view(),
              e_op_t<vertex_t, result_t>{},
              property_initial_value,
              cugraph::reduce_op::maximum<result_t>{},
              cugraph::get_dataframe_buffer_begin(global_out_result));
            break;
          default: FAIL() << "should not be reached.";
        }

        auto global_in_result = cugraph::allocate_dataframe_buffer<result_t>(
          sg_graph_view.local_vertex_partition_range_size(), handle_->get_stream());

        switch (reduction_types[i]) {
          case reduction_type_t::PLUS:
            per_v_transform_reduce_incoming_e(
              *handle_,
              sg_graph_view,
              sg_src_prop.view(),
              sg_dst_prop.view(),
              cugraph::edge_dummy_property_t{}.view(),
              e_op_t<vertex_t, result_t>{},
              property_initial_value,
              cugraph::reduce_op::plus<result_t>{},
              cugraph::get_dataframe_buffer_begin(global_in_result));
            break;
          case reduction_type_t::MINIMUM:
            per_v_transform_reduce_incoming_e(
              *handle_,
              sg_graph_view,
              sg_src_prop.view(),
              sg_dst_prop.view(),
              cugraph::edge_dummy_property_t{}.view(),
              e_op_t<vertex_t, result_t>{},
              property_initial_value,
              cugraph::reduce_op::minimum<result_t>{},
              cugraph::get_dataframe_buffer_begin(global_in_result));
            break;
          case reduction_type_t::MAXIMUM:
            per_v_transform_reduce_incoming_e(
              *handle_,
              sg_graph_view,
              sg_src_prop.view(),
              sg_dst_prop.view(),
              cugraph::edge_dummy_property_t{}.view(),
              e_op_t<vertex_t, result_t>{},
              property_initial_value,
              cugraph::reduce_op::maximum<result_t>{},
              cugraph::get_dataframe_buffer_begin(global_in_result));
            break;
          default: FAIL() << "should not be reached.";
        }

        auto aggregate_labels      = aggregate(*handle_, *d_mg_renumber_map_labels);
        auto aggregate_out_results = aggregate(*handle_, out_results[i]);
        auto aggregate_in_results  = aggregate(*handle_, in_results[i]);
        if (handle_->get_comms().get_rank() == int{0}) {
          std::tie(std::ignore, aggregate_out_results) =
            cugraph::test::sort_by_key(*handle_, aggregate_labels, aggregate_out_results);
          std::tie(std::ignore, aggregate_in_results) =
            cugraph::test::sort_by_key(*handle_, aggregate_labels, aggregate_in_results);
          ASSERT_TRUE(comp(aggregate_out_results, global_out_result));
          ASSERT_TRUE(comp(aggregate_in_results, global_in_result));
        }
      }
    }
  }

 private:
  static std::unique_ptr<raft::handle_t> handle_;
};

template <typename input_usecase_t>
std::unique_ptr<raft::handle_t>
  Tests_MGPerVTransformReduceIncomingOutgoingE<input_usecase_t>::handle_ = nullptr;

using Tests_MGPerVTransformReduceIncomingOutgoingE_File =
  Tests_MGPerVTransformReduceIncomingOutgoingE<cugraph::test::File_Usecase>;
using Tests_MGPerVTransformReduceIncomingOutgoingE_Rmat =
  Tests_MGPerVTransformReduceIncomingOutgoingE<cugraph::test::Rmat_Usecase>;

TEST_P(Tests_MGPerVTransformReduceIncomingOutgoingE_File,
       CheckInt32Int32FloatTupleIntFloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, thrust::tuple<int, float>, false>(std::get<0>(param),
                                                                              std::get<1>(param));
}

TEST_P(Tests_MGPerVTransformReduceIncomingOutgoingE_Rmat,
       CheckInt32Int32FloatTupleIntFloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, thrust::tuple<int, float>, false>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGPerVTransformReduceIncomingOutgoingE_Rmat,
       CheckInt32Int64FloatTupleIntFloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int64_t, float, thrust::tuple<int, float>, false>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGPerVTransformReduceIncomingOutgoingE_Rmat,
       CheckInt64Int64FloatTupleIntFloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t, float, thrust::tuple<int, float>, false>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGPerVTransformReduceIncomingOutgoingE_File,
       CheckInt32Int32FloatTupleIntFloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, thrust::tuple<int, float>, true>(std::get<0>(param),
                                                                             std::get<1>(param));
}

TEST_P(Tests_MGPerVTransformReduceIncomingOutgoingE_Rmat,
       CheckInt32Int32FloatTupleIntFloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, thrust::tuple<int, float>, true>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGPerVTransformReduceIncomingOutgoingE_Rmat,
       CheckInt32Int64FloatTupleIntFloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int64_t, float, thrust::tuple<int, float>, true>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGPerVTransformReduceIncomingOutgoingE_Rmat,
       CheckInt64Int64FloatTupleIntFloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t, float, thrust::tuple<int, float>, true>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGPerVTransformReduceIncomingOutgoingE_File, CheckInt32Int32FloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, int, false>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MGPerVTransformReduceIncomingOutgoingE_Rmat, CheckInt32Int32FloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, int, false>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGPerVTransformReduceIncomingOutgoingE_Rmat, CheckInt32Int64FloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int32_t, int64_t, float, int, false>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGPerVTransformReduceIncomingOutgoingE_Rmat, CheckInt64Int64FloatTransposeFalse)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t, float, int, false>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGPerVTransformReduceIncomingOutgoingE_File, CheckInt32Int32FloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, int, true>(std::get<0>(param), std::get<1>(param));
}

TEST_P(Tests_MGPerVTransformReduceIncomingOutgoingE_Rmat, CheckInt32Int32FloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int32_t, float, int, true>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGPerVTransformReduceIncomingOutgoingE_Rmat, CheckInt32Int64FloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int32_t, int64_t, float, int, true>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

TEST_P(Tests_MGPerVTransformReduceIncomingOutgoingE_Rmat, CheckInt64Int64FloatTransposeTrue)
{
  auto param = GetParam();
  run_current_test<int64_t, int64_t, float, int, true>(
    std::get<0>(param),
    cugraph::test::override_Rmat_Usecase_with_cmd_line_arguments(std::get<1>(param)));
}

INSTANTIATE_TEST_SUITE_P(
  file_test,
  Tests_MGPerVTransformReduceIncomingOutgoingE_File,
  ::testing::Combine(
    ::testing::Values(Prims_Usecase{true}),
    ::testing::Values(cugraph::test::File_Usecase("test/datasets/karate.mtx"),
                      cugraph::test::File_Usecase("test/datasets/web-Google.mtx"),
                      cugraph::test::File_Usecase("test/datasets/ljournal-2008.mtx"),
                      cugraph::test::File_Usecase("test/datasets/webbase-1M.mtx"))));

INSTANTIATE_TEST_SUITE_P(
  rmat_small_test,
  Tests_MGPerVTransformReduceIncomingOutgoingE_Rmat,
  ::testing::Combine(::testing::Values(Prims_Usecase{true}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       10, 16, 0.57, 0.19, 0.19, 0, false, false, 0, true))));

INSTANTIATE_TEST_SUITE_P(
  rmat_benchmark_test, /* note that scale & edge factor can be overridden in benchmarking (with
                          --gtest_filter to select only the rmat_benchmark_test with a specific
                          vertex & edge type combination) by command line arguments and do not
                          include more than one Rmat_Usecase that differ only in scale or edge
                          factor (to avoid running same benchmarks more than once) */
  Tests_MGPerVTransformReduceIncomingOutgoingE_Rmat,
  ::testing::Combine(::testing::Values(Prims_Usecase{false}),
                     ::testing::Values(cugraph::test::Rmat_Usecase(
                       20, 32, 0.57, 0.19, 0.19, 0, false, false, 0, true))));

CUGRAPH_MG_TEST_PROGRAM_MAIN()
