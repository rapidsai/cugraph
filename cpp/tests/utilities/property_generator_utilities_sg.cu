/*
 * SPDX-FileCopyrightText: Copyright (c) 2021-2025, NVIDIA CORPORATION.
 * SPDX-License-Identifier: Apache-2.0
 */
#include "utilities/property_generator_utilities_impl.cuh"

namespace cugraph {
namespace test {

template struct generate<cugraph::graph_view_t<int32_t, int32_t, false, false>, bool>;
template struct generate<cugraph::graph_view_t<int32_t, int32_t, false, false>, int32_t>;
template struct generate<cugraph::graph_view_t<int32_t, int32_t, false, false>, int64_t>;
template struct generate<cugraph::graph_view_t<int32_t, int32_t, false, false>,
                         cuda::std::tuple<int, float>>;

template struct generate<cugraph::graph_view_t<int32_t, int32_t, true, false>, bool>;
template struct generate<cugraph::graph_view_t<int32_t, int32_t, true, false>, int32_t>;
template struct generate<cugraph::graph_view_t<int32_t, int32_t, true, false>, int64_t>;
template struct generate<cugraph::graph_view_t<int32_t, int32_t, true, false>,
                         cuda::std::tuple<int, float>>;

template struct generate<cugraph::graph_view_t<int64_t, int64_t, false, false>, bool>;
template struct generate<cugraph::graph_view_t<int64_t, int64_t, false, false>, int32_t>;
template struct generate<cugraph::graph_view_t<int64_t, int64_t, false, false>, int64_t>;
template struct generate<cugraph::graph_view_t<int64_t, int64_t, false, false>,
                         cuda::std::tuple<int, float>>;

template struct generate<cugraph::graph_view_t<int64_t, int64_t, true, false>, bool>;
template struct generate<cugraph::graph_view_t<int64_t, int64_t, true, false>, int32_t>;
template struct generate<cugraph::graph_view_t<int64_t, int64_t, true, false>, int64_t>;
template struct generate<cugraph::graph_view_t<int64_t, int64_t, true, false>,
                         cuda::std::tuple<int, float>>;

}  // namespace test
}  // namespace cugraph
