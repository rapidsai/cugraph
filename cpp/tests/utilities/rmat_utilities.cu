#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <utilities/test_utilities.hpp>

#include <cugraph/experimental/detail/graph_utils.cuh>
#include <cugraph/experimental/graph_functions.hpp>
#include <cugraph/experimental/graph_generator.hpp>
#include <cugraph/partition_manager.hpp>
#include <cugraph/utilities/error.hpp>
#include <cugraph/utilities/shuffle_comm.cuh>

#include <rmm/thrust_rmm_allocator.h>
#include <raft/random/rng.cuh>

#include <thrust/sequence.h>

#include <cstdint>

namespace cugraph {
namespace test {

template <typename vertex_t,
          typename edge_t,
          typename weight_t,
          bool store_transposed,
          bool multi_gpu>
std::tuple<cugraph::experimental::graph_t<vertex_t, edge_t, weight_t, store_transposed, multi_gpu>,
           rmm::device_uvector<vertex_t>>
generate_graph_from_rmat_params(raft::handle_t const& handle,
                                size_t scale,
                                size_t edge_factor,
                                double a,
                                double b,
                                double c,
                                uint64_t base_seed,
                                bool undirected,
                                bool scramble_vertex_ids,
                                bool test_weighted,
                                bool renumber,
                                std::vector<size_t> const& partition_ids,
                                size_t num_partitions)
{
  CUGRAPH_EXPECTS(!multi_gpu || renumber, "renumber should be true if multi_gpu is true.");
  CUGRAPH_EXPECTS(size_t{1} << scale <= static_cast<size_t>(std::numeric_limits<vertex_t>::max()),
                  "vertex_t overflow.");
  CUGRAPH_EXPECTS(
    (size_t{1} << scale) * edge_factor <= static_cast<size_t>(std::numeric_limits<edge_t>::max()),
    " edge_t overflow.");

  vertex_t number_of_vertices = static_cast<vertex_t>(size_t{1} << scale);
  edge_t number_of_edges =
    static_cast<edge_t>(static_cast<size_t>(number_of_vertices) * edge_factor);

  std::vector<edge_t> partition_edge_counts(partition_ids.size());
  std::vector<vertex_t> partition_vertex_firsts(partition_ids.size());
  std::vector<vertex_t> partition_vertex_lasts(partition_ids.size());
  for (size_t i = 0; i < partition_ids.size(); ++i) {
    auto id = partition_ids[i];

    partition_edge_counts[i] = number_of_edges / num_partitions +
                               (id < number_of_edges % num_partitions ? edge_t{1} : edge_t{0});

    partition_vertex_firsts[i] = (number_of_vertices / num_partitions) * id;
    partition_vertex_lasts[i]  = (number_of_vertices / num_partitions) * (id + 1);
    if (id < number_of_vertices % num_partitions) {
      partition_vertex_firsts[i] += id;
      partition_vertex_lasts[i] += id + 1;
    } else {
      partition_vertex_firsts[i] += number_of_vertices % num_partitions;
      partition_vertex_lasts[i] += number_of_vertices % num_partitions;
    }
  }

  rmm::device_uvector<vertex_t> d_edgelist_rows(0, handle.get_stream());
  rmm::device_uvector<vertex_t> d_edgelist_cols(0, handle.get_stream());
  rmm::device_uvector<weight_t> d_edgelist_weights(0, handle.get_stream());
  for (size_t i = 0; i < partition_ids.size(); ++i) {
    auto id = partition_ids[i];

    rmm::device_uvector<vertex_t> d_tmp_rows(0, handle.get_stream());
    rmm::device_uvector<vertex_t> d_tmp_cols(0, handle.get_stream());
    std::tie(i == 0 ? d_edgelist_rows : d_tmp_rows, i == 0 ? d_edgelist_cols : d_tmp_cols) =
      cugraph::experimental::generate_rmat_edgelist<vertex_t>(handle,
                                                              scale,
                                                              partition_edge_counts[i],
                                                              a,
                                                              b,
                                                              c,
                                                              base_seed + id,
                                                              undirected ? true : false,
                                                              scramble_vertex_ids);

    rmm::device_uvector<weight_t> d_tmp_weights(0, handle.get_stream());
    if (test_weighted) {
      if (i == 0) {
        d_edgelist_weights.resize(d_edgelist_rows.size(), handle.get_stream());
      } else {
        d_tmp_weights.resize(d_tmp_rows.size(), handle.get_stream());
      }

      raft::random::Rng rng(base_seed + num_partitions + id);
      rng.uniform<weight_t, size_t>(i == 0 ? d_edgelist_weights.data() : d_tmp_weights.data(),
                                    i == 0 ? d_edgelist_weights.size() : d_tmp_weights.size(),
                                    weight_t{0.0},
                                    weight_t{1.0},
                                    handle.get_stream());
    }

    if (i > 0) {
      auto start_offset = d_edgelist_rows.size();
      d_edgelist_rows.resize(start_offset + d_tmp_rows.size(), handle.get_stream());
      d_edgelist_cols.resize(d_edgelist_rows.size(), handle.get_stream());
      thrust::copy(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                   d_tmp_rows.begin(),
                   d_tmp_rows.end(),
                   d_edgelist_rows.begin() + start_offset);
      thrust::copy(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                   d_tmp_cols.begin(),
                   d_tmp_cols.end(),
                   d_edgelist_cols.begin() + start_offset);
      if (test_weighted) {
        d_edgelist_weights.resize(d_edgelist_rows.size(), handle.get_stream());
        thrust::copy(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                     d_tmp_weights.begin(),
                     d_tmp_weights.end(),
                     d_edgelist_weights.begin() + start_offset);
      }
    }
  }

  if (undirected) {
// FIXME: may need to undo this and handle symmetrization elsewhere once the new test graph
// generation API gets integrated
#if 1
    auto offset = d_edgelist_rows.size();
    d_edgelist_rows.resize(offset * 2, handle.get_stream());
    d_edgelist_cols.resize(d_edgelist_rows.size(), handle.get_stream());
    d_edgelist_weights.resize(test_weighted ? d_edgelist_rows.size() : size_t{0},
                              handle.get_stream());
    thrust::copy(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                 d_edgelist_cols.begin(),
                 d_edgelist_cols.begin() + offset,
                 d_edgelist_rows.begin() + offset);
    thrust::copy(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                 d_edgelist_rows.begin(),
                 d_edgelist_rows.begin() + offset,
                 d_edgelist_cols.begin() + offset);
    if (test_weighted) {
      thrust::copy(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                   d_edgelist_weights.begin(),
                   d_edgelist_weights.begin() + offset,
                   d_edgelist_weights.begin() + offset);
    }
#endif
  }

  if (multi_gpu) {
    auto& comm               = handle.get_comms();
    auto const comm_size     = comm.get_size();
    auto& row_comm           = handle.get_subcomm(cugraph::partition_2d::key_naming_t().row_name());
    auto const row_comm_size = row_comm.get_size();
    auto& col_comm           = handle.get_subcomm(cugraph::partition_2d::key_naming_t().col_name());
    auto const col_comm_size = col_comm.get_size();

    rmm::device_uvector<vertex_t> d_rx_edgelist_rows(0, handle.get_stream());
    rmm::device_uvector<vertex_t> d_rx_edgelist_cols(0, handle.get_stream());
    rmm::device_uvector<weight_t> d_rx_edgelist_weights(0, handle.get_stream());
    if (test_weighted) {
      auto edge_first = thrust::make_zip_iterator(
        thrust::make_tuple(store_transposed ? d_edgelist_cols.begin() : d_edgelist_rows.begin(),
                           store_transposed ? d_edgelist_rows.begin() : d_edgelist_cols.begin(),
                           d_edgelist_weights.begin()));

      std::forward_as_tuple(std::tie(store_transposed ? d_rx_edgelist_cols : d_rx_edgelist_rows,
                                     store_transposed ? d_rx_edgelist_rows : d_rx_edgelist_cols,
                                     d_rx_edgelist_weights),
                            std::ignore) =
        cugraph::experimental::groupby_gpuid_and_shuffle_values(
          comm,  // handle.get_comms(),
          edge_first,
          edge_first + d_edgelist_rows.size(),
          [key_func =
             cugraph::experimental::detail::compute_gpu_id_from_edge_t<vertex_t>{
               comm_size, row_comm_size, col_comm_size}] __device__(auto val) {
            return key_func(thrust::get<0>(val), thrust::get<1>(val));
          },
          handle.get_stream());
    } else {
      auto edge_first = thrust::make_zip_iterator(
        thrust::make_tuple(store_transposed ? d_edgelist_cols.begin() : d_edgelist_rows.begin(),
                           store_transposed ? d_edgelist_rows.begin() : d_edgelist_cols.begin()));

      std::forward_as_tuple(std::tie(store_transposed ? d_rx_edgelist_cols : d_rx_edgelist_rows,
                                     store_transposed ? d_rx_edgelist_rows : d_rx_edgelist_cols),
                            std::ignore) =
        cugraph::experimental::groupby_gpuid_and_shuffle_values(
          comm,  // handle.get_comms(),
          edge_first,
          edge_first + d_edgelist_rows.size(),
          [key_func =
             cugraph::experimental::detail::compute_gpu_id_from_edge_t<vertex_t>{
               comm_size, row_comm_size, col_comm_size}] __device__(auto val) {
            return key_func(thrust::get<0>(val), thrust::get<1>(val));
          },
          handle.get_stream());
    }

    d_edgelist_rows    = std::move(d_rx_edgelist_rows);
    d_edgelist_cols    = std::move(d_rx_edgelist_cols);
    d_edgelist_weights = std::move(d_rx_edgelist_weights);
  }

  rmm::device_uvector<vertex_t> d_vertices(0, handle.get_stream());
  for (size_t i = 0; i < partition_ids.size(); ++i) {
    auto id = partition_ids[i];

    auto start_offset = d_vertices.size();
    d_vertices.resize(start_offset + (partition_vertex_lasts[i] - partition_vertex_firsts[i]),
                      handle.get_stream());
    thrust::sequence(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                     d_vertices.begin() + start_offset,
                     d_vertices.end(),
                     partition_vertex_firsts[i]);
  }

  if (multi_gpu) {
    auto& comm           = handle.get_comms();
    auto const comm_size = comm.get_size();

    rmm::device_uvector<vertex_t> d_rx_vertices(0, handle.get_stream());
    std::tie(d_rx_vertices, std::ignore) = cugraph::experimental::groupby_gpuid_and_shuffle_values(
      comm,  // handle.get_comms(),
      d_vertices.begin(),
      d_vertices.end(),
      [key_func =
         cugraph::experimental::detail::compute_gpu_id_from_vertex_t<vertex_t>{
           comm_size}] __device__(auto val) { return key_func(val); },
      handle.get_stream());
    d_vertices = std::move(d_rx_vertices);
  }

  return cugraph::experimental::
    create_graph_from_edgelist<vertex_t, edge_t, weight_t, store_transposed, multi_gpu>(
      handle,
      std::optional<std::tuple<vertex_t const*, vertex_t>>{
        std::make_tuple(d_vertices.data(), static_cast<vertex_t>(d_vertices.size()))},
      std::move(d_edgelist_rows),
      std::move(d_edgelist_cols),
      std::move(d_edgelist_weights),
      cugraph::experimental::graph_properties_t{undirected, true, test_weighted},
      renumber);
}

// explicit instantiations

template std::tuple<cugraph::experimental::graph_t<int32_t, int32_t, float, false, false>,
                    rmm::device_uvector<int32_t>>
generate_graph_from_rmat_params<int32_t, int32_t, float, false, false>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int32_t, int32_t, float, false, true>,
                    rmm::device_uvector<int32_t>>
generate_graph_from_rmat_params<int32_t, int32_t, float, false, true>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int32_t, int32_t, float, true, false>,
                    rmm::device_uvector<int32_t>>
generate_graph_from_rmat_params<int32_t, int32_t, float, true, false>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int32_t, int32_t, float, true, true>,
                    rmm::device_uvector<int32_t>>
generate_graph_from_rmat_params<int32_t, int32_t, float, true, true>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int32_t, int32_t, double, false, false>,
                    rmm::device_uvector<int32_t>>
generate_graph_from_rmat_params<int32_t, int32_t, double, false, false>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int32_t, int32_t, double, false, true>,
                    rmm::device_uvector<int32_t>>
generate_graph_from_rmat_params<int32_t, int32_t, double, false, true>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int32_t, int32_t, double, true, false>,
                    rmm::device_uvector<int32_t>>
generate_graph_from_rmat_params<int32_t, int32_t, double, true, false>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int32_t, int32_t, double, true, true>,
                    rmm::device_uvector<int32_t>>
generate_graph_from_rmat_params<int32_t, int32_t, double, true, true>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int32_t, int64_t, float, false, false>,
                    rmm::device_uvector<int32_t>>
generate_graph_from_rmat_params<int32_t, int64_t, float, false, false>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int32_t, int64_t, float, false, true>,
                    rmm::device_uvector<int32_t>>
generate_graph_from_rmat_params<int32_t, int64_t, float, false, true>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int32_t, int64_t, float, true, false>,
                    rmm::device_uvector<int32_t>>
generate_graph_from_rmat_params<int32_t, int64_t, float, true, false>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int32_t, int64_t, float, true, true>,
                    rmm::device_uvector<int32_t>>
generate_graph_from_rmat_params<int32_t, int64_t, float, true, true>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int32_t, int64_t, double, false, false>,
                    rmm::device_uvector<int32_t>>
generate_graph_from_rmat_params<int32_t, int64_t, double, false, false>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int32_t, int64_t, double, false, true>,
                    rmm::device_uvector<int32_t>>
generate_graph_from_rmat_params<int32_t, int64_t, double, false, true>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int32_t, int64_t, double, true, false>,
                    rmm::device_uvector<int32_t>>
generate_graph_from_rmat_params<int32_t, int64_t, double, true, false>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int32_t, int64_t, double, true, true>,
                    rmm::device_uvector<int32_t>>
generate_graph_from_rmat_params<int32_t, int64_t, double, true, true>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int64_t, int64_t, float, false, false>,
                    rmm::device_uvector<int64_t>>
generate_graph_from_rmat_params<int64_t, int64_t, float, false, false>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int64_t, int64_t, float, false, true>,
                    rmm::device_uvector<int64_t>>
generate_graph_from_rmat_params<int64_t, int64_t, float, false, true>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int64_t, int64_t, float, true, false>,
                    rmm::device_uvector<int64_t>>
generate_graph_from_rmat_params<int64_t, int64_t, float, true, false>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int64_t, int64_t, float, true, true>,
                    rmm::device_uvector<int64_t>>
generate_graph_from_rmat_params<int64_t, int64_t, float, true, true>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int64_t, int64_t, double, false, false>,
                    rmm::device_uvector<int64_t>>
generate_graph_from_rmat_params<int64_t, int64_t, double, false, false>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int64_t, int64_t, double, false, true>,
                    rmm::device_uvector<int64_t>>
generate_graph_from_rmat_params<int64_t, int64_t, double, false, true>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int64_t, int64_t, double, true, false>,
                    rmm::device_uvector<int64_t>>
generate_graph_from_rmat_params<int64_t, int64_t, double, true, false>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

template std::tuple<cugraph::experimental::graph_t<int64_t, int64_t, double, true, true>,
                    rmm::device_uvector<int64_t>>
generate_graph_from_rmat_params<int64_t, int64_t, double, true, true>(
  raft::handle_t const& handle,
  size_t scale,
  size_t edge_factor,
  double a,
  double b,
  double c,
  uint64_t seed,
  bool undirected,
  bool scramble_vertex_ids,
  bool test_weighted,
  bool renumber,
  std::vector<size_t> const& partition_ids,
  size_t num_partitions);

}  // namespace test
}  // namespace cugraph
