/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <utilities/thrust_wrapper.hpp>

#include <raft/handle.hpp>
#include <rmm/exec_policy.hpp>

#include <thrust/copy.h>
#include <thrust/sort.h>

namespace cugraph {
namespace test {

template <typename vertex_t, typename value_t>
std::tuple<rmm::device_uvector<vertex_t>, rmm::device_uvector<value_t>> sort_by_key(
  raft::handle_t const& handle, vertex_t const* keys, value_t const* values, size_t num_pairs)
{
  rmm::device_uvector<vertex_t> sorted_keys(num_pairs, handle.get_stream_view());
  rmm::device_uvector<value_t> sorted_values(num_pairs, handle.get_stream_view());

  auto execution_policy = handle.get_thrust_policy();
  thrust::copy(execution_policy, keys, keys + num_pairs, sorted_keys.begin());
  thrust::copy(execution_policy, values, values + num_pairs, sorted_values.begin());

  thrust::sort_by_key(
    execution_policy, sorted_keys.begin(), sorted_keys.end(), sorted_values.begin());

  return std::make_tuple(std::move(sorted_keys), std::move(sorted_values));
}

template std::tuple<rmm::device_uvector<int32_t>, rmm::device_uvector<float>>
sort_by_key<int32_t, float>(raft::handle_t const& handle,
                            int32_t const* keys,
                            float const* values,
                            size_t num_pairs);

template std::tuple<rmm::device_uvector<int32_t>, rmm::device_uvector<double>>
sort_by_key<int32_t, double>(raft::handle_t const& handle,
                             int32_t const* keys,
                             double const* values,
                             size_t num_pairs);

template std::tuple<rmm::device_uvector<int32_t>, rmm::device_uvector<int32_t>>
sort_by_key<int32_t, int32_t>(raft::handle_t const& handle,
                              int32_t const* keys,
                              int32_t const* values,
                              size_t num_pairs);

template std::tuple<rmm::device_uvector<int64_t>, rmm::device_uvector<float>>
sort_by_key<int64_t, float>(raft::handle_t const& handle,
                            int64_t const* keys,
                            float const* values,
                            size_t num_pairs);

template std::tuple<rmm::device_uvector<int64_t>, rmm::device_uvector<double>>
sort_by_key<int64_t, double>(raft::handle_t const& handle,
                             int64_t const* keys,
                             double const* values,
                             size_t num_pairs);

template std::tuple<rmm::device_uvector<int64_t>, rmm::device_uvector<int64_t>>
sort_by_key<int64_t, int64_t>(raft::handle_t const& handle,
                              int64_t const* keys,
                              int64_t const* values,
                              size_t num_pairs);

template <typename vertex_t>
void translate_vertex_ids(raft::handle_t const& handle,
                          rmm::device_uvector<vertex_t>& d_src_v,
                          rmm::device_uvector<vertex_t>& d_dst_v,
                          vertex_t vertex_id_offset)
{
  auto execution_policy = handle.get_thrust_policy();
  thrust::transform(execution_policy,
                    d_src_v.begin(),
                    d_src_v.end(),
                    d_src_v.begin(),
                    [offset = vertex_id_offset] __device__(vertex_t v) { return offset + v; });

  thrust::transform(execution_policy,
                    d_dst_v.begin(),
                    d_dst_v.end(),
                    d_dst_v.begin(),
                    [offset = vertex_id_offset] __device__(vertex_t v) { return offset + v; });
}

template <typename vertex_t>
void populate_vertex_ids(raft::handle_t const& handle,
                         rmm::device_uvector<vertex_t>& d_vertices_v,
                         vertex_t vertex_id_offset)
{
  thrust::sequence(
    handle.get_thrust_policy(), d_vertices_v.begin(), d_vertices_v.end(), vertex_id_offset);
}

template void translate_vertex_ids(raft::handle_t const& handle,
                                   rmm::device_uvector<int32_t>& d_src_v,
                                   rmm::device_uvector<int32_t>& d_dst_v,
                                   int32_t vertex_id_offset);

template void translate_vertex_ids(raft::handle_t const& handle,
                                   rmm::device_uvector<int64_t>& d_src_v,
                                   rmm::device_uvector<int64_t>& d_dst_v,
                                   int64_t vertex_id_offset);

template void populate_vertex_ids(raft::handle_t const& handle,
                                  rmm::device_uvector<int32_t>& d_vertices_v,
                                  int32_t vertex_id_offset);

template void populate_vertex_ids(raft::handle_t const& handle,
                                  rmm::device_uvector<int64_t>& d_vertices_v,
                                  int64_t vertex_id_offset);

}  // namespace test
}  // namespace cugraph
