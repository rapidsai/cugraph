/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// Mst solver tests
// Author: Alex Fender afender@nvidia.com

#include <utilities/base_fixture.hpp>
#include <utilities/high_res_clock.h>
#include <utilities/test_utilities.hpp>

#include <cugraph/algorithms.hpp>
#include <cugraph/legacy/graph.hpp>

#include <raft/error.hpp>
#include <raft/handle.hpp>

#include <hip/hip_runtime_api.h>

#include <cmath>

#include "../src/converters/COOtoCSR.cuh"
#include <thrust/iterator/constant_iterator.h>
#include <thrust/reduce.h>

typedef struct Mst_Usecase_t {
  std::string matrix_file;
  Mst_Usecase_t(const std::string& a)
  {
    // assume relative paths are relative to RAPIDS_DATASET_ROOT_DIR
    const std::string& rapidsDatasetRootDir = cugraph::test::get_rapids_dataset_root_dir();
    if ((a != "") && (a[0] != '/')) {
      matrix_file = rapidsDatasetRootDir + "/" + a;
    } else {
      matrix_file = a;
    }
  }
  Mst_Usecase_t& operator=(const Mst_Usecase_t& rhs)
  {
    matrix_file = rhs.matrix_file;
    return *this;
  }
} Mst_Usecase;

class Tests_Mst : public ::testing::TestWithParam<Mst_Usecase> {
 public:
  Tests_Mst() {}
  static void SetupTestCase() {}
  static void TearDownTestCase() {}
  virtual void SetUp() {}
  virtual void TearDown() {}

  template <typename T>
  void run_current_test(const Mst_Usecase& param)
  {
    const ::testing::TestInfo* const test_info =
      ::testing::UnitTest::GetInstance()->current_test_info();
    std::stringstream ss;
    std::string test_id = std::string(test_info->test_case_name()) + std::string(".") +
                          std::string(test_info->name()) + std::string("_") +
                          cugraph::test::getFileName(param.matrix_file) + std::string("_") +
                          ss.str().c_str();

    int m, k, nnz;
    MM_typecode mc;

    HighResClock hr_clock;
    double time_tmp;

    FILE* fpin = fopen(param.matrix_file.c_str(), "r");
    ASSERT_NE(fpin, nullptr) << "fopen (" << param.matrix_file << ") failure.";

    ASSERT_EQ(cugraph::test::mm_properties<int>(fpin, 1, &mc, &m, &k, &nnz), 0)
      << "could not read Matrix Market file properties"
      << "\n";
    ASSERT_TRUE(mm_is_matrix(mc));
    ASSERT_TRUE(mm_is_coordinate(mc));
    ASSERT_FALSE(mm_is_complex(mc));
    ASSERT_FALSE(mm_is_skew(mc));

    // Allocate memory on host
    std::vector<int> cooRowInd(nnz), cooColInd(nnz);
    std::vector<T> cooVal(nnz), mst(m);

    // Read
    ASSERT_EQ((cugraph::test::mm_to_coo<int, T>(
                fpin, 1, nnz, &cooRowInd[0], &cooColInd[0], &cooVal[0], NULL)),
              0)
      << "could not read matrix data"
      << "\n";
    ASSERT_EQ(fclose(fpin), 0);

    raft::handle_t handle;

    std::cout << std::endl;
    cugraph::legacy::GraphCOOView<int, int, T> G_coo(
      &cooRowInd[0], &cooColInd[0], &cooVal[0], m, nnz);
    auto G_unique = cugraph::coo_to_csr(G_coo);
    cugraph::legacy::GraphCSRView<int, int, T> G(G_unique->view().offsets,
                                                 G_unique->view().indices,
                                                 G_unique->view().edge_data,
                                                 G_unique->view().number_of_vertices,
                                                 G_unique->view().number_of_edges);

    hipDeviceSynchronize();

    hr_clock.start();
    hipProfilerStart();
    auto mst_edges = cugraph::minimum_spanning_tree<int, int, T>(handle, G);
    hipProfilerStop();

    hipDeviceSynchronize();
    hr_clock.stop(&time_tmp);
    std::cout << "mst_time: " << time_tmp << " us" << std::endl;

    auto expected_mst_weight = thrust::reduce(
      thrust::device_pointer_cast(G_unique->view().edge_data),
      thrust::device_pointer_cast(G_unique->view().edge_data) + G_unique->view().number_of_edges);

    auto calculated_mst_weight = thrust::reduce(
      thrust::device_pointer_cast(mst_edges->view().edge_data),
      thrust::device_pointer_cast(mst_edges->view().edge_data) + mst_edges->view().number_of_edges);

    std::cout << "calculated_mst_weight: " << calculated_mst_weight << std::endl;
    std::cout << "number_of_MST_edges: " << mst_edges->view().number_of_edges << std::endl;

    EXPECT_LE(calculated_mst_weight, expected_mst_weight);
    EXPECT_LE(mst_edges->view().number_of_edges, 2 * m - 2);
  }
};

TEST_P(Tests_Mst, CheckFP32_T) { run_current_test<float>(GetParam()); }

TEST_P(Tests_Mst, CheckFP64_T) { run_current_test<double>(GetParam()); }

INSTANTIATE_TEST_SUITE_P(simple_test,
                         Tests_Mst,
                         ::testing::Values(Mst_Usecase("test/datasets/netscience.mtx")));

CUGRAPH_TEST_PROGRAM_MAIN()
