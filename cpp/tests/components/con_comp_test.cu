/*
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 */

// connected components tests
// Author: Andrei Schaffer aschaffer@nvidia.com

#include "hip/hip_runtime_api.h"
#include "gtest/gtest.h"
#include "high_res_clock.h"

#include <algorithm>
#include <algorithms.hpp>
#include <converters/COOtoCSR.cuh>
#include <graph.hpp>
#include <iterator>
#include "test_utils.h"

// do the perf measurements
// enabled by command line parameter s'--perf'
//
static int PERF = 0;

namespace {  // un-nammed
struct Usecase {
  explicit Usecase(const std::string& a)
  {
    // assume relative paths are relative to RAPIDS_DATASET_ROOT_DIR
    const std::string& rapidsDatasetRootDir = get_rapids_dataset_root_dir();
    if ((a != "") && (a[0] != '/')) {
      matrix_file = rapidsDatasetRootDir + "/" + a;
    } else {
      matrix_file = a;
    }
  }

  const std::string& get_matrix_file(void) const { return matrix_file; }

 private:
  std::string matrix_file;
};

}  // namespace

struct Tests_Weakly_CC : ::testing::TestWithParam<Usecase> {
  Tests_Weakly_CC() {}
  static void SetupTestCase() {}
  static void TearDownTestCase()
  {
    if (PERF) {
      for (unsigned int i = 0; i < weakly_cc_time.size(); ++i) {
        std::cout << weakly_cc_time[i] << std::endl;
      }
    }
  }
  virtual void SetUp() {}
  virtual void TearDown() {}

  static std::vector<double> weakly_cc_time;

  void run_current_test(const Usecase& param)
  {
    const ::testing::TestInfo* const test_info =
      ::testing::UnitTest::GetInstance()->current_test_info();
    std::stringstream ss;
    std::string test_id =
      std::string(test_info->test_case_name()) + std::string(".") + std::string(test_info->name()) +
      std::string("_") + getFileName(param.get_matrix_file()) + std::string("_") + ss.str().c_str();

    int m, k, nnz;  //
    MM_typecode mc;

    HighResClock hr_clock;
    double time_tmp;

    FILE* fpin = fopen(param.get_matrix_file().c_str(), "r");
    ASSERT_NE(fpin, nullptr) << "fopen (" << param.get_matrix_file() << ") failure.";

    ASSERT_EQ(mm_properties<int>(fpin, 1, &mc, &m, &k, &nnz), 0)
      << "could not read Matrix Market file properties"
      << "\n";
    ASSERT_TRUE(mm_is_matrix(mc));
    ASSERT_TRUE(mm_is_coordinate(mc));
    ASSERT_TRUE(mm_is_symmetric(mc));  // weakly cc only works w/ undirected graphs, for now;

    // rmmInitialize(nullptr);

#ifdef _DEBUG_WEAK_CC
    std::cout << "matrix nrows: " << m << "\n";
    std::cout << "matrix nnz: " << nnz << "\n";
#endif

    // Allocate memory on host
    std::vector<int> cooRowInd(nnz);
    std::vector<int> cooColInd(nnz);
    std::vector<int> labels(m);  // for G(V, E), m := |V|
    std::vector<int> verts(m);

    // Read: COO Format
    //
    ASSERT_EQ((mm_to_coo<int, int>(fpin, 1, nnz, &cooRowInd[0], &cooColInd[0], nullptr, nullptr)),
              0)
      << "could not read matrix data"
      << "\n";
    ASSERT_EQ(fclose(fpin), 0);

    CSR_Result<int> result;
    ConvertCOOtoCSR(&cooColInd[0], &cooRowInd[0], nnz, result);

    cugraph::experimental::GraphCSR<int, int, float> G(
      result.rowOffsets, result.colIndices, nullptr, m, nnz);

    rmm::device_vector<int> d_labels(m);

    if (PERF) {
      hr_clock.start();
      cugraph::connected_components<int, int, float>(
        G, cugraph::cugraph_cc_t::CUGRAPH_WEAK, d_labels.data().get());
      hipDeviceSynchronize();
      hr_clock.stop(&time_tmp);
      weakly_cc_time.push_back(time_tmp);
    } else {
      hipProfilerStart();
      cugraph::connected_components<int, int, float>(
        G, cugraph::cugraph_cc_t::CUGRAPH_WEAK, d_labels.data().get());
      hipProfilerStop();
      hipDeviceSynchronize();
    }
  }
};

std::vector<double> Tests_Weakly_CC::weakly_cc_time;

TEST_P(Tests_Weakly_CC, Weakly_CC) { run_current_test(GetParam()); }

// --gtest_filter=*simple_test*
INSTANTIATE_TEST_CASE_P(simple_test,
                        Tests_Weakly_CC,
                        ::testing::Values(Usecase("test/datasets/dolphins.mtx"),
                                          Usecase("test/datasets/coPapersDBLP.mtx"),
                                          Usecase("test/datasets/coPapersCiteseer.mtx"),
                                          Usecase("test/datasets/hollywood.mtx")));

int main(int argc, char** argv)
{
  rmmInitialize(nullptr);
  testing::InitGoogleTest(&argc, argv);
  int rc = RUN_ALL_TESTS();
  rmmFinalize();
  return rc;
}
