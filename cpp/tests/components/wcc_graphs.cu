/*
 * Copyright (c) 2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 */

#include <components/wcc_graphs.hpp>
#include <utilities/test_utilities.hpp>

#include <cugraph/detail/utility_wrappers.hpp>
#include <cugraph/graph_functions.hpp>

#include <rmm/exec_policy.hpp>

#include <thrust/sort.h>

namespace cugraph {
namespace test {

template <typename vertex_t,
          typename edge_t,
          typename weight_t,
          bool store_transposed,
          bool multi_gpu>
std::tuple<cugraph::graph_t<vertex_t, edge_t, weight_t, store_transposed, multi_gpu>,
           std::optional<rmm::device_uvector<vertex_t>>>
LineGraph_Usecase::construct_graph(raft::handle_t const& handle,
                                   bool test_weighted,
                                   bool renumber) const
{
  uint64_t seed{0};

  edge_t num_edges = 2 * (num_vertices_ - 1);

  rmm::device_uvector<vertex_t> vertices_v(num_vertices_, handle.get_stream());
  rmm::device_uvector<vertex_t> src_v(num_edges, handle.get_stream());
  rmm::device_uvector<vertex_t> dst_v(num_edges, handle.get_stream());
  rmm::device_uvector<double> order_v(num_vertices_, handle.get_stream());

  auto execution_policy = handle.get_thrust_policy();
  thrust::sequence(execution_policy, vertices_v.begin(), vertices_v.end(), vertex_t{0});

  cugraph::detail::uniform_random_fill(
    handle.get_stream_view(), order_v.data(), num_vertices_, double{0.0}, double{1.0}, seed);

  thrust::sort_by_key(execution_policy, order_v.begin(), order_v.end(), vertices_v.begin());

  raft::copy(src_v.begin(), vertices_v.begin(), (num_vertices_ - 1), handle.get_stream());
  raft::copy(dst_v.begin(), vertices_v.begin() + 1, (num_vertices_ - 1), handle.get_stream());

  raft::copy(src_v.begin() + (num_vertices_ - 1),
             vertices_v.begin() + 1,
             (num_vertices_ - 1),
             handle.get_stream());
  raft::copy(dst_v.begin() + (num_vertices_ - 1),
             vertices_v.begin(),
             (num_vertices_ - 1),
             handle.get_stream());

  thrust::sequence(execution_policy, vertices_v.begin(), vertices_v.end(), vertex_t{0});

  handle.get_stream_view().synchronize();

  return cugraph::
    create_graph_from_edgelist<vertex_t, edge_t, weight_t, store_transposed, multi_gpu>(
      handle,
      std::move(vertices_v),
      std::move(src_v),
      std::move(dst_v),
      std::nullopt,
      cugraph::graph_properties_t{true, false},
      false);
}

template std::tuple<cugraph::graph_t<int32_t, int32_t, float, false, false>,
                    std::optional<rmm::device_uvector<int32_t>>>
LineGraph_Usecase::construct_graph(raft::handle_t const&, bool, bool) const;

}  // namespace test
}  // namespace cugraph
