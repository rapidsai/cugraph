/*
 * Copyright (c) 2024, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governin_from_mtxg permissions and
 * limitations under the License.
 */

#include "prims/per_v_transform_reduce_incoming_outgoing_e.cuh"
#include "prims/reduce_op.cuh"
#include "utilities/base_fixture.hpp"
#include "utilities/test_graphs.hpp"

#include <cugraph/algorithms.hpp>
#include <cugraph/edge_partition_view.hpp>
#include <cugraph/edge_property.hpp>
#include <cugraph/edge_src_dst_property.hpp>
#include <cugraph/graph_functions.hpp>
#include <cugraph/graph_view.hpp>
#include <cugraph/utilities/high_res_timer.hpp>

#include <raft/random/rng_state.hpp>

#include <gtest/gtest.h>

#include <chrono>
#include <iostream>
#include <random>

struct MaximalIndependentSet_Usecase {
  bool check_correctness{true};
};

template <typename input_usecase_t>
class Tests_SGMaximalIndependentSet
  : public ::testing::TestWithParam<std::tuple<MaximalIndependentSet_Usecase, input_usecase_t>> {
 public:
  Tests_SGMaximalIndependentSet() {}

  static void SetUpTestCase() {}
  static void TearDownTestCase() {}

  virtual void SetUp() {}
  virtual void TearDown() {}

  template <typename vertex_t, typename edge_t, typename weight_t, typename result_t>
  void run_current_test(std::tuple<MaximalIndependentSet_Usecase, input_usecase_t> const& param)
  {
    auto [mis_usecase, input_usecase] = param;

    raft::handle_t handle{};
    HighResTimer hr_timer{};

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());
      hr_timer.start("Construct graph");
    }

    constexpr bool multi_gpu = false;

    auto [sg_graph, sg_edge_weights, sg_renumber_map] =
      cugraph::test::construct_graph<vertex_t, edge_t, weight_t, false, multi_gpu>(
        handle, input_usecase, false, true);

    if (cugraph::test::g_perf) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());

      hr_timer.stop();
      hr_timer.display_and_clear(std::cout);
    }

    auto sg_graph_view = sg_graph.view();
    auto sg_edge_weight_view =
      sg_edge_weights ? std::make_optional((*sg_edge_weights).view()) : std::nullopt;

    raft::random::RngState rng_state(0);
    auto d_mis = cugraph::maximal_independent_set<vertex_t, edge_t, multi_gpu>(
      handle, sg_graph_view, rng_state);

    // Test MIS
    if (mis_usecase.check_correctness) {
      RAFT_CUDA_TRY(hipDeviceSynchronize());
      std::vector<vertex_t> h_mis(d_mis.size());
      raft::update_host(h_mis.data(), d_mis.data(), d_mis.size(), handle.get_stream());

      RAFT_CUDA_TRY(hipDeviceSynchronize());

      auto vertex_first = sg_graph_view.local_vertex_partition_range_first();
      auto vertex_last  = sg_graph_view.local_vertex_partition_range_last();

      std::for_each(h_mis.begin(), h_mis.end(), [vertex_first, vertex_last](vertex_t v) {
        ASSERT_TRUE((v >= vertex_first) && (v < vertex_last));
      });

      // If a vertex is included in MIS, then none of its neighbor should be

      vertex_t local_vtx_partitoin_size = sg_graph_view.local_vertex_partition_range_size();
      rmm::device_uvector<vertex_t> d_total_outgoing_nbrs_included_mis(local_vtx_partitoin_size,
                                                                       handle.get_stream());

      rmm::device_uvector<vertex_t> inclusiong_flags(local_vtx_partitoin_size, handle.get_stream());

      thrust::uninitialized_fill(
        handle.get_thrust_policy(), inclusiong_flags.begin(), inclusiong_flags.end(), vertex_t{0});

      thrust::for_each(
        handle.get_thrust_policy(),
        d_mis.begin(),
        d_mis.end(),
        [inclusiong_flags =
           raft::device_span<vertex_t>(inclusiong_flags.data(), inclusiong_flags.size()),
         v_first = sg_graph_view.local_vertex_partition_range_first()] __device__(auto v) {
          auto v_offset              = v - v_first;
          inclusiong_flags[v_offset] = vertex_t{1};
        });

      RAFT_CUDA_TRY(hipDeviceSynchronize());

      per_v_transform_reduce_outgoing_e(
        handle,
        sg_graph_view,
        cugraph::detail::edge_major_property_view_t<vertex_t, vertex_t const*>(
          inclusiong_flags.data()),
        cugraph::detail::edge_minor_property_view_t<vertex_t, vertex_t const*>(
          inclusiong_flags.data(), vertex_t{0}),
        cugraph::edge_dummy_property_t{}.view(),
        [] __device__(auto src, auto dst, auto src_included, auto dst_included, auto wt) {
          return (src == dst) ? 0 : dst_included;
        },
        vertex_t{0},
        cugraph::reduce_op::plus<vertex_t>{},
        d_total_outgoing_nbrs_included_mis.begin());

      RAFT_CUDA_TRY(hipDeviceSynchronize());

      std::vector<vertex_t> h_total_outgoing_nbrs_included_mis(
        d_total_outgoing_nbrs_included_mis.size());
      raft::update_host(h_total_outgoing_nbrs_included_mis.data(),
                        d_total_outgoing_nbrs_included_mis.data(),
                        d_total_outgoing_nbrs_included_mis.size(),
                        handle.get_stream());

      RAFT_CUDA_TRY(hipDeviceSynchronize());

      {
        auto vertex_first = sg_graph_view.local_vertex_partition_range_first();
        auto vertex_last  = sg_graph_view.local_vertex_partition_range_last();

        std::for_each(h_mis.begin(),
                      h_mis.end(),
                      [vertex_first, vertex_last, &h_total_outgoing_nbrs_included_mis](vertex_t v) {
                        ASSERT_TRUE((v >= vertex_first) && (v < vertex_last))
                          << v << " is not within vertex parition range" << std::endl;

                        ASSERT_TRUE(h_total_outgoing_nbrs_included_mis[v - vertex_first] == 0)
                          << v << "'s neighbor is included in MIS" << std::endl;
                      });
      }
    }
  }
};

using Tests_SGMaximalIndependentSet_File =
  Tests_SGMaximalIndependentSet<cugraph::test::File_Usecase>;
using Tests_SGMaximalIndependentSet_Rmat =
  Tests_SGMaximalIndependentSet<cugraph::test::Rmat_Usecase>;

TEST_P(Tests_SGMaximalIndependentSet_File, CheckInt32Int32FloatFloat)
{
  run_current_test<int32_t, int32_t, float, int>(
    override_File_Usecase_with_cmd_line_arguments(GetParam()));
}

TEST_P(Tests_SGMaximalIndependentSet_File, CheckInt64Int64FloatFloat)
{
  run_current_test<int64_t, int64_t, float, int>(
    override_File_Usecase_with_cmd_line_arguments(GetParam()));
}

TEST_P(Tests_SGMaximalIndependentSet_Rmat, CheckInt32Int32FloatFloat)
{
  run_current_test<int32_t, int32_t, float, int>(
    override_Rmat_Usecase_with_cmd_line_arguments(GetParam()));
}

TEST_P(Tests_SGMaximalIndependentSet_Rmat, CheckInt64Int64FloatFloat)
{
  run_current_test<int64_t, int64_t, float, int>(
    override_Rmat_Usecase_with_cmd_line_arguments(GetParam()));
}

bool constexpr check_correctness = false;
INSTANTIATE_TEST_SUITE_P(
  file_test,
  Tests_SGMaximalIndependentSet_File,
  ::testing::Combine(::testing::Values(MaximalIndependentSet_Usecase{check_correctness},
                                       MaximalIndependentSet_Usecase{check_correctness}),
                     ::testing::Values(cugraph::test::File_Usecase("test/datasets/karate.mtx"))));

INSTANTIATE_TEST_SUITE_P(
  rmat_small_test,
  Tests_SGMaximalIndependentSet_Rmat,
  ::testing::Combine(
    ::testing::Values(MaximalIndependentSet_Usecase{check_correctness}),
    ::testing::Values(cugraph::test::Rmat_Usecase(3, 4, 0.57, 0.19, 0.19, 0, true, false))));

INSTANTIATE_TEST_SUITE_P(
  rmat_benchmark_test, /* note that scale & edge factor can be overridden in benchmarking (with
                          --gtest_filter to select only the rmat_benchmark_test with a specific
                          vertex & edge type combination) by command line arguments and do not
                          include more than one Rmat_Usecase that differ only in scale or edge
                          factor (to avoid running same benchmarks more than once) */
  Tests_SGMaximalIndependentSet_Rmat,
  ::testing::Combine(
    ::testing::Values(MaximalIndependentSet_Usecase{check_correctness},
                      MaximalIndependentSet_Usecase{check_correctness}),
    ::testing::Values(cugraph::test::Rmat_Usecase(20, 32, 0.57, 0.19, 0.19, 0, false, false))));

CUGRAPH_TEST_PROGRAM_MAIN()
