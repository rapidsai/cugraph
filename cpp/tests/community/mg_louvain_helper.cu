/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "mg_louvain_helper.hpp"

#include <cugraph/experimental/graph.hpp>

#include <cugraph/utilities/device_comm.cuh>
#include <cugraph/utilities/error.hpp>
#include <cugraph/utilities/host_scalar_comm.cuh>

#include <rmm/thrust_rmm_allocator.h>

#include <thrust/for_each.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>

namespace cugraph {
namespace test {

template <typename T>
void single_gpu_renumber_edgelist_given_number_map(raft::handle_t const &handle,
                                                   rmm::device_uvector<T> &edgelist_rows_v,
                                                   rmm::device_uvector<T> &edgelist_cols_v,
                                                   rmm::device_uvector<T> &renumber_map_gathered_v)
{
  rmm::device_uvector<T> index_v(renumber_map_gathered_v.size(), handle.get_stream());

  thrust::for_each(
    rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
    thrust::make_counting_iterator<size_t>(0),
    thrust::make_counting_iterator<size_t>(renumber_map_gathered_v.size()),
    [d_renumber_map_gathered = renumber_map_gathered_v.data(), d_index = index_v.data()] __device__(
      auto idx) { d_index[d_renumber_map_gathered[idx]] = idx; });

  thrust::transform(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                    edgelist_rows_v.begin(),
                    edgelist_rows_v.end(),
                    edgelist_rows_v.begin(),
                    [d_index = index_v.data()] __device__(auto v) { return d_index[v]; });

  thrust::transform(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                    edgelist_cols_v.begin(),
                    edgelist_cols_v.end(),
                    edgelist_cols_v.begin(),
                    [d_index = index_v.data()] __device__(auto v) { return d_index[v]; });
}

template <typename vertex_t, typename edge_t, typename weight_t>
std::tuple<rmm::device_uvector<vertex_t>,
           rmm::device_uvector<vertex_t>,
           std::optional<rmm::device_uvector<weight_t>>>
compressed_sparse_to_edgelist(edge_t const *compressed_sparse_offsets,
                              vertex_t const *compressed_sparse_indices,
                              std::optional<weight_t const *> compressed_sparse_weights,
                              vertex_t major_first,
                              vertex_t major_last,
                              hipStream_t stream)
{
  edge_t number_of_edges{0};
  raft::update_host(
    &number_of_edges, compressed_sparse_offsets + (major_last - major_first), 1, stream);
  CUDA_TRY(hipStreamSynchronize(stream));
  rmm::device_uvector<vertex_t> edgelist_major_vertices(number_of_edges, stream);
  rmm::device_uvector<vertex_t> edgelist_minor_vertices(number_of_edges, stream);
  auto edgelist_weights =
    compressed_sparse_weights
      ? std::make_optional<rmm::device_uvector<weight_t>>(number_of_edges, stream)
      : std::nullopt;

  // FIXME: this is highly inefficient for very high-degree vertices, for better performance, we can
  // fill high-degree vertices using one CUDA block per vertex, mid-degree vertices using one CUDA
  // warp per vertex, and low-degree vertices using one CUDA thread per block
  thrust::for_each(rmm::exec_policy(stream)->on(stream),
                   thrust::make_counting_iterator(major_first),
                   thrust::make_counting_iterator(major_last),
                   [compressed_sparse_offsets,
                    major_first,
                    p_majors = edgelist_major_vertices.begin()] __device__(auto v) {
                     auto first = compressed_sparse_offsets[v - major_first];
                     auto last  = compressed_sparse_offsets[v - major_first + 1];
                     thrust::fill(thrust::seq, p_majors + first, p_majors + last, v);
                   });
  thrust::copy(rmm::exec_policy(stream)->on(stream),
               compressed_sparse_indices,
               compressed_sparse_indices + number_of_edges,
               edgelist_minor_vertices.begin());
  if (compressed_sparse_weights) {
    thrust::copy(rmm::exec_policy(stream)->on(stream),
                 (*compressed_sparse_weights),
                 (*compressed_sparse_weights) + number_of_edges,
                 (*edgelist_weights).data());
  }

  return std::make_tuple(std::move(edgelist_major_vertices),
                         std::move(edgelist_minor_vertices),
                         std::move(edgelist_weights));
}

template <typename vertex_t, typename weight_t>
void sort_and_coarsen_edgelist(
  rmm::device_uvector<vertex_t> &edgelist_major_vertices /* [INOUT] */,
  rmm::device_uvector<vertex_t> &edgelist_minor_vertices /* [INOUT] */,
  std::optional<rmm::device_uvector<weight_t>> &edgelist_weights /* [INOUT] */,
  hipStream_t stream)
{
  auto pair_first = thrust::make_zip_iterator(
    thrust::make_tuple(edgelist_major_vertices.begin(), edgelist_minor_vertices.begin()));

  size_t number_of_edges{0};
  if (edgelist_weights) {
    thrust::sort_by_key(rmm::exec_policy(stream)->on(stream),
                        pair_first,
                        pair_first + edgelist_major_vertices.size(),
                        (*edgelist_weights).begin());

    rmm::device_uvector<vertex_t> tmp_edgelist_major_vertices(edgelist_major_vertices.size(),
                                                              stream);
    rmm::device_uvector<vertex_t> tmp_edgelist_minor_vertices(tmp_edgelist_major_vertices.size(),
                                                              stream);
    rmm::device_uvector<weight_t> tmp_edgelist_weights(tmp_edgelist_major_vertices.size(), stream);
    auto it = thrust::reduce_by_key(
      rmm::exec_policy(stream)->on(stream),
      pair_first,
      pair_first + edgelist_major_vertices.size(),
      (*edgelist_weights).begin(),
      thrust::make_zip_iterator(thrust::make_tuple(tmp_edgelist_major_vertices.begin(),
                                                   tmp_edgelist_minor_vertices.begin())),
      tmp_edgelist_weights.begin());
    number_of_edges = thrust::distance(tmp_edgelist_weights.begin(), thrust::get<1>(it));

    edgelist_major_vertices = std::move(tmp_edgelist_major_vertices);
    edgelist_minor_vertices = std::move(tmp_edgelist_minor_vertices);
    (*edgelist_weights)     = std::move(tmp_edgelist_weights);
  } else {
    thrust::sort(rmm::exec_policy(stream)->on(stream),
                 pair_first,
                 pair_first + edgelist_major_vertices.size());
    auto it         = thrust::unique(rmm::exec_policy(stream)->on(stream),
                             pair_first,
                             pair_first + edgelist_major_vertices.size());
    number_of_edges = thrust::distance(pair_first, it);
  }

  edgelist_major_vertices.resize(number_of_edges, stream);
  edgelist_minor_vertices.resize(number_of_edges, stream);
  edgelist_major_vertices.shrink_to_fit(stream);
  edgelist_minor_vertices.shrink_to_fit(stream);
  if (edgelist_weights) {
    (*edgelist_weights).resize(number_of_edges, stream);
    (*edgelist_weights).shrink_to_fit(stream);
  }
}

template <typename vertex_t, typename edge_t, typename weight_t>
std::tuple<rmm::device_uvector<vertex_t>,
           rmm::device_uvector<vertex_t>,
           std::optional<rmm::device_uvector<weight_t>>>
compressed_sparse_to_relabeled_and_sorted_and_coarsened_edgelist(
  edge_t const *compressed_sparse_offsets,
  vertex_t const *compressed_sparse_indices,
  std::optional<weight_t const *> compressed_sparse_weights,
  vertex_t const *p_major_labels,
  vertex_t const *p_minor_labels,
  vertex_t major_first,
  vertex_t major_last,
  vertex_t minor_first,
  vertex_t minor_last,
  hipStream_t stream)
{
  // FIXME: it might be possible to directly create relabled & coarsened edgelist from the
  // compressed sparse format to save memory

  auto [edgelist_major_vertices, edgelist_minor_vertices, edgelist_weights] =
    compressed_sparse_to_edgelist(compressed_sparse_offsets,
                                  compressed_sparse_indices,
                                  compressed_sparse_weights,
                                  major_first,
                                  major_last,
                                  stream);

  auto pair_first = thrust::make_zip_iterator(
    thrust::make_tuple(edgelist_major_vertices.begin(), edgelist_minor_vertices.begin()));
  thrust::transform(
    rmm::exec_policy(stream)->on(stream),
    pair_first,
    pair_first + edgelist_major_vertices.size(),
    pair_first,
    [p_major_labels, p_minor_labels, major_first, minor_first] __device__(auto val) {
      return thrust::make_tuple(p_major_labels[thrust::get<0>(val) - major_first],
                                p_minor_labels[thrust::get<1>(val) - minor_first]);
    });

  sort_and_coarsen_edgelist(
    edgelist_major_vertices, edgelist_minor_vertices, edgelist_weights, stream);

  return std::make_tuple(std::move(edgelist_major_vertices),
                         std::move(edgelist_minor_vertices),
                         std::move(edgelist_weights));
}

// FIXME: better add "bool renumber" (which must be false in MG) to the coarsen_grpah function
// instead of replicating the code here. single-GPU version
template <typename vertex_t, typename edge_t, typename weight_t, bool store_transposed>
std::unique_ptr<cugraph::experimental::graph_t<vertex_t, edge_t, weight_t, store_transposed, false>>
coarsen_graph(
  raft::handle_t const &handle,
  cugraph::experimental::graph_view_t<vertex_t, edge_t, weight_t, store_transposed, false> const
    &graph_view,
  vertex_t const *labels)
{
  auto [coarsened_edgelist_major_vertices,
        coarsened_edgelist_minor_vertices,
        coarsened_edgelist_weights] =
    compressed_sparse_to_relabeled_and_sorted_and_coarsened_edgelist(
      graph_view.get_matrix_partition_view().get_offsets(),
      graph_view.get_matrix_partition_view().get_indices(),
      graph_view.get_matrix_partition_view().get_weights(),
      labels,
      labels,
      vertex_t{0},
      graph_view.get_number_of_vertices(),
      vertex_t{0},
      graph_view.get_number_of_vertices(),
      handle.get_stream());

  cugraph::experimental::edgelist_t<vertex_t, edge_t, weight_t> edgelist{};
  edgelist.p_src_vertices = store_transposed ? coarsened_edgelist_minor_vertices.data()
                                             : coarsened_edgelist_major_vertices.data();
  edgelist.p_dst_vertices = store_transposed ? coarsened_edgelist_major_vertices.data()
                                             : coarsened_edgelist_minor_vertices.data();
  edgelist.p_edge_weights =
    coarsened_edgelist_weights
      ? std::optional<weight_t const *>{(*coarsened_edgelist_weights).data()}
      : std::nullopt;
  edgelist.number_of_edges = static_cast<edge_t>(coarsened_edgelist_major_vertices.size());

  vertex_t new_number_of_vertices =
    1 + thrust::reduce(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                       labels,
                       labels + graph_view.get_number_of_vertices(),
                       vertex_t{0},
                       thrust::maximum<vertex_t>());

  return std::make_unique<
    cugraph::experimental::graph_t<vertex_t, edge_t, weight_t, store_transposed, false>>(
    handle,
    edgelist,
    new_number_of_vertices,
    cugraph::experimental::graph_properties_t{graph_view.is_symmetric(), false},
    std::nullopt);
}

// explicit instantiation

template void single_gpu_renumber_edgelist_given_number_map(
  raft::handle_t const &handle,
  rmm::device_uvector<int> &d_edgelist_rows,
  rmm::device_uvector<int> &d_edgelist_cols,
  rmm::device_uvector<int> &d_renumber_map_gathered_v);

template std::unique_ptr<cugraph::experimental::graph_t<int32_t, int32_t, float, false, false>>
coarsen_graph(
  raft::handle_t const &handle,
  cugraph::experimental::graph_view_t<int32_t, int32_t, float, false, false> const &graph_view,
  int32_t const *labels);

}  // namespace test
}  // namespace cugraph
