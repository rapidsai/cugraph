#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "mg_louvain_helper.hpp"

#include <experimental/graph.hpp>

#include <utilities/device_comm.cuh>
#include <utilities/error.hpp>
#include <utilities/host_scalar_comm.cuh>

#include <rmm/thrust_rmm_allocator.h>

#include <thrust/for_each.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>

namespace cugraph {
namespace test {

template <typename T>
rmm::device_uvector<T> gather_distributed_vector(raft::handle_t const &handle,
                                                 T const *d_input,
                                                 size_t size)
{
  auto rx_sizes =
    cugraph::experimental::host_scalar_gather(handle.get_comms(), size, 0, handle.get_stream());
  std::vector<size_t> rx_displs(static_cast<size_t>(handle.get_comms().get_rank()) == 0
                                  ? handle.get_comms().get_size()
                                  : int{0},
                                size_t{0});
  if (static_cast<size_t>(handle.get_comms().get_rank()) == 0) {
    std::partial_sum(rx_sizes.begin(), rx_sizes.end() - 1, rx_displs.begin() + 1);
  }

  auto total_size = thrust::reduce(thrust::host, rx_sizes.begin(), rx_sizes.end());
  rmm::device_uvector<T> gathered_v(total_size, handle.get_stream());

  cugraph::experimental::device_gatherv(handle.get_comms(),
                                        d_input,
                                        gathered_v.data(),
                                        size,
                                        rx_sizes,
                                        rx_displs,
                                        0,
                                        handle.get_stream());

  return gathered_v;
}

template <typename vertex_t>
bool compare_renumbered_vectors(raft::handle_t const &handle,
                                rmm::device_uvector<vertex_t> const &v1,
                                rmm::device_uvector<vertex_t> const &v2)
{
  vertex_t max = 1 + thrust::reduce(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                                    v1.begin(),
                                    v1.end(),
                                    vertex_t{0});

  rmm::device_uvector<size_t> map(max, size_t{0});

  auto iter = thrust::make_zip_iterator(thrust::make_tuple(v1.begin(), v2.begin()));

  thrust::for_each(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                   iter,
                   iter + v1.size(),
                   [d_map = map.data()] __device__(auto pair) {
                     vertex_t e1 = thrust::get<0>(pair);
                     vertex_t e2 = thrust::get<1>(pair);

                     d_map[e1] = e2;
                   });

  auto error_count =
    thrust::count_if(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                     iter,
                     iter + v1.size(),
                     [d_map = map.data()] __device__(auto pair) {
                       vertex_t e1 = thrust::get<0>(pair);
                       vertex_t e2 = thrust::get<1>(pair);

                       return (d_map[e1] != e2);
                     });

  return (error_count == 0);
}

template <typename T>
void single_gpu_renumber_edgelist_given_number_map(raft::handle_t const &handle,
                                                   rmm::device_uvector<T> &edgelist_rows_v,
                                                   rmm::device_uvector<T> &edgelist_cols_v,
                                                   rmm::device_uvector<T> &renumber_map_gathered_v)
{
  rmm::device_uvector<T> index_v(renumber_map_gathered_v.size(), handle.get_stream());

  thrust::for_each(
    rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
    thrust::make_counting_iterator<size_t>(0),
    thrust::make_counting_iterator<size_t>(renumber_map_gathered_v.size()),
    [d_renumber_map_gathered = renumber_map_gathered_v.data(), d_index = index_v.data()] __device__(
      auto idx) { d_index[d_renumber_map_gathered[idx]] = idx; });

  thrust::transform(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                    edgelist_rows_v.begin(),
                    edgelist_rows_v.end(),
                    edgelist_rows_v.begin(),
                    [d_index = index_v.data()] __device__(auto v) { return d_index[v]; });

  thrust::transform(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                    edgelist_cols_v.begin(),
                    edgelist_cols_v.end(),
                    edgelist_cols_v.begin(),
                    [d_index = index_v.data()] __device__(auto v) { return d_index[v]; });
}

template <typename vertex_t, typename edge_t, typename weight_t>
std::
  tuple<rmm::device_uvector<vertex_t>, rmm::device_uvector<vertex_t>, rmm::device_uvector<weight_t>>
  compressed_sparse_to_edgelist(edge_t const *compressed_sparse_offsets,
                                vertex_t const *compressed_sparse_indices,
                                weight_t const *compressed_sparse_weights,
                                vertex_t major_first,
                                vertex_t major_last,
                                hipStream_t stream)
{
  edge_t number_of_edges{0};
  raft::update_host(
    &number_of_edges, compressed_sparse_offsets + (major_last - major_first), 1, stream);
  CUDA_TRY(hipStreamSynchronize(stream));
  rmm::device_uvector<vertex_t> edgelist_major_vertices(number_of_edges, stream);
  rmm::device_uvector<vertex_t> edgelist_minor_vertices(number_of_edges, stream);
  rmm::device_uvector<weight_t> edgelist_weights(
    compressed_sparse_weights != nullptr ? number_of_edges : 0, stream);

  // FIXME: this is highly inefficient for very high-degree vertices, for better performance, we can
  // fill high-degree vertices using one CUDA block per vertex, mid-degree vertices using one CUDA
  // warp per vertex, and low-degree vertices using one CUDA thread per block
  thrust::for_each(rmm::exec_policy(stream)->on(stream),
                   thrust::make_counting_iterator(major_first),
                   thrust::make_counting_iterator(major_last),
                   [compressed_sparse_offsets,
                    major_first,
                    p_majors = edgelist_major_vertices.begin()] __device__(auto v) {
                     auto first = compressed_sparse_offsets[v - major_first];
                     auto last  = compressed_sparse_offsets[v - major_first + 1];
                     thrust::fill(thrust::seq, p_majors + first, p_majors + last, v);
                   });
  thrust::copy(rmm::exec_policy(stream)->on(stream),
               compressed_sparse_indices,
               compressed_sparse_indices + number_of_edges,
               edgelist_minor_vertices.begin());
  if (compressed_sparse_weights != nullptr) {
    thrust::copy(rmm::exec_policy(stream)->on(stream),
                 compressed_sparse_weights,
                 compressed_sparse_weights + number_of_edges,
                 edgelist_weights.data());
  }

  return std::make_tuple(std::move(edgelist_major_vertices),
                         std::move(edgelist_minor_vertices),
                         std::move(edgelist_weights));
}

template <typename vertex_t, typename weight_t>
void sort_and_coarsen_edgelist(rmm::device_uvector<vertex_t> &edgelist_major_vertices /* [INOUT] */,
                               rmm::device_uvector<vertex_t> &edgelist_minor_vertices /* [INOUT] */,
                               rmm::device_uvector<weight_t> &edgelist_weights /* [INOUT] */,
                               hipStream_t stream)
{
  auto pair_first = thrust::make_zip_iterator(
    thrust::make_tuple(edgelist_major_vertices.begin(), edgelist_minor_vertices.begin()));

  size_t number_of_edges{0};
  if (edgelist_weights.size() > 0) {
    thrust::sort_by_key(rmm::exec_policy(stream)->on(stream),
                        pair_first,
                        pair_first + edgelist_major_vertices.size(),
                        edgelist_weights.begin());

    rmm::device_uvector<vertex_t> tmp_edgelist_major_vertices(edgelist_major_vertices.size(),
                                                              stream);
    rmm::device_uvector<vertex_t> tmp_edgelist_minor_vertices(tmp_edgelist_major_vertices.size(),
                                                              stream);
    rmm::device_uvector<weight_t> tmp_edgelist_weights(tmp_edgelist_major_vertices.size(), stream);
    auto it = thrust::reduce_by_key(
      rmm::exec_policy(stream)->on(stream),
      pair_first,
      pair_first + edgelist_major_vertices.size(),
      edgelist_weights.begin(),
      thrust::make_zip_iterator(thrust::make_tuple(tmp_edgelist_major_vertices.begin(),
                                                   tmp_edgelist_minor_vertices.begin())),
      tmp_edgelist_weights.begin());
    number_of_edges = thrust::distance(tmp_edgelist_weights.begin(), thrust::get<1>(it));

    edgelist_major_vertices = std::move(tmp_edgelist_major_vertices);
    edgelist_minor_vertices = std::move(tmp_edgelist_minor_vertices);
    edgelist_weights        = std::move(tmp_edgelist_weights);
  } else {
    thrust::sort(rmm::exec_policy(stream)->on(stream),
                 pair_first,
                 pair_first + edgelist_major_vertices.size());
    auto it         = thrust::unique(rmm::exec_policy(stream)->on(stream),
                             pair_first,
                             pair_first + edgelist_major_vertices.size());
    number_of_edges = thrust::distance(pair_first, it);
  }

  edgelist_major_vertices.resize(number_of_edges, stream);
  edgelist_minor_vertices.resize(number_of_edges, stream);
  edgelist_weights.resize(number_of_edges, stream);
  edgelist_major_vertices.shrink_to_fit(stream);
  edgelist_minor_vertices.shrink_to_fit(stream);
  edgelist_weights.shrink_to_fit(stream);
}

template <typename vertex_t, typename edge_t, typename weight_t>
std::
  tuple<rmm::device_uvector<vertex_t>, rmm::device_uvector<vertex_t>, rmm::device_uvector<weight_t>>
  compressed_sparse_to_relabeled_and_sorted_and_coarsened_edgelist(
    edge_t const *compressed_sparse_offsets,
    vertex_t const *compressed_sparse_indices,
    weight_t const *compressed_sparse_weights,
    vertex_t const *p_major_labels,
    vertex_t const *p_minor_labels,
    vertex_t major_first,
    vertex_t major_last,
    vertex_t minor_first,
    vertex_t minor_last,
    hipStream_t stream)
{
  // FIXME: it might be possible to directly create relabled & coarsened edgelist from the
  // compressed sparse format to save memory

  rmm::device_uvector<vertex_t> edgelist_major_vertices(0, stream);
  rmm::device_uvector<vertex_t> edgelist_minor_vertices(0, stream);
  rmm::device_uvector<weight_t> edgelist_weights(0, stream);
  std::tie(edgelist_major_vertices, edgelist_minor_vertices, edgelist_weights) =
    compressed_sparse_to_edgelist(compressed_sparse_offsets,
                                  compressed_sparse_indices,
                                  compressed_sparse_weights,
                                  major_first,
                                  major_last,
                                  stream);

  auto pair_first = thrust::make_zip_iterator(
    thrust::make_tuple(edgelist_major_vertices.begin(), edgelist_minor_vertices.begin()));
  thrust::transform(
    rmm::exec_policy(stream)->on(stream),
    pair_first,
    pair_first + edgelist_major_vertices.size(),
    pair_first,
    [p_major_labels, p_minor_labels, major_first, minor_first] __device__(auto val) {
      return thrust::make_tuple(p_major_labels[thrust::get<0>(val) - major_first],
                                p_minor_labels[thrust::get<1>(val) - minor_first]);
    });

  sort_and_coarsen_edgelist(
    edgelist_major_vertices, edgelist_minor_vertices, edgelist_weights, stream);

  return std::make_tuple(std::move(edgelist_major_vertices),
                         std::move(edgelist_minor_vertices),
                         std::move(edgelist_weights));
}

// single-GPU version
template <typename vertex_t, typename edge_t, typename weight_t, bool store_transposed>
std::unique_ptr<cugraph::experimental::graph_t<vertex_t, edge_t, weight_t, store_transposed, false>>
coarsen_graph(
  raft::handle_t const &handle,
  cugraph::experimental::graph_view_t<vertex_t, edge_t, weight_t, store_transposed, false> const
    &graph_view,
  vertex_t const *labels)
{
  rmm::device_uvector<vertex_t> coarsened_edgelist_major_vertices(0, handle.get_stream());
  rmm::device_uvector<vertex_t> coarsened_edgelist_minor_vertices(0, handle.get_stream());
  rmm::device_uvector<weight_t> coarsened_edgelist_weights(0, handle.get_stream());
  std::tie(coarsened_edgelist_major_vertices,
           coarsened_edgelist_minor_vertices,
           coarsened_edgelist_weights) =
    compressed_sparse_to_relabeled_and_sorted_and_coarsened_edgelist(
      graph_view.offsets(),
      graph_view.indices(),
      graph_view.weights(),
      labels,
      labels,
      vertex_t{0},
      graph_view.get_number_of_vertices(),
      vertex_t{0},
      graph_view.get_number_of_vertices(),
      handle.get_stream());

  cugraph::experimental::edgelist_t<vertex_t, edge_t, weight_t> edgelist{};
  edgelist.p_src_vertices = store_transposed ? coarsened_edgelist_minor_vertices.data()
                                             : coarsened_edgelist_major_vertices.data();
  edgelist.p_dst_vertices = store_transposed ? coarsened_edgelist_major_vertices.data()
                                             : coarsened_edgelist_minor_vertices.data();
  edgelist.p_edge_weights  = coarsened_edgelist_weights.data();
  edgelist.number_of_edges = static_cast<edge_t>(coarsened_edgelist_major_vertices.size());

  vertex_t new_number_of_vertices =
    1 + thrust::reduce(rmm::exec_policy(handle.get_stream())->on(handle.get_stream()),
                       labels,
                       labels + graph_view.get_number_of_vertices(),
                       vertex_t{0},
                       thrust::maximum<vertex_t>());

  return std::make_unique<
    cugraph::experimental::graph_t<vertex_t, edge_t, weight_t, store_transposed, false>>(
    handle,
    edgelist,
    new_number_of_vertices,
    cugraph::experimental::graph_properties_t{
      graph_view.is_symmetric(), false, graph_view.is_weighted()},
    true);
}

// explicit instantiation

template void single_gpu_renumber_edgelist_given_number_map(
  raft::handle_t const &handle,
  rmm::device_uvector<int> &d_edgelist_rows,
  rmm::device_uvector<int> &d_edgelist_cols,
  rmm::device_uvector<int> &d_renumber_map_gathered_v);

template rmm::device_uvector<int> gather_distributed_vector(raft::handle_t const &handle,
                                                            int const *d_input,
                                                            size_t size);

template bool compare_renumbered_vectors(raft::handle_t const &handle,
                                         rmm::device_uvector<int> const &v1,
                                         rmm::device_uvector<int> const &v2);

template std::unique_ptr<cugraph::experimental::graph_t<int32_t, int32_t, float, false, false>>
coarsen_graph(
  raft::handle_t const &handle,
  cugraph::experimental::graph_view_t<int32_t, int32_t, float, false, false> const &graph_view,
  int32_t const *labels);

}  // namespace test
}  // namespace cugraph
