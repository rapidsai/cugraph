/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <graph_contracting_visitor.hxx>

namespace nvgraph
{
  //------------------------- Graph Contraction: ----------------------
  //
  CsrGraph<int>* contract_graph_csr_min(CsrGraph<int>& graph,
                                int* pV, size_t n,
                                hipStream_t stream,
                                const int& VCombine,
                                const int& VReduce,
                                const int& ECombine,
                                const int& EReduce)
  {
    return contract_from_aggregates_t<int, double, SemiRingFctrSelector<Min, double>::FctrType >(graph, pV, n, stream,
                                                                                                       static_cast<SemiRingFunctorTypes>(VCombine),
                                                                                                       static_cast<SemiRingFunctorTypes>(VReduce),
                                                                                                       static_cast<SemiRingFunctorTypes>(ECombine),
                                                                                                       static_cast<SemiRingFunctorTypes>(EReduce));
  }
 
}
