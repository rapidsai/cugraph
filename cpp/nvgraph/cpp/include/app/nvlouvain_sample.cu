/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
// Turn on to see stats for each level
//#define ENABLE_LOG true
#include "nvlouvain.cuh"



/* Louvain Clustering Sample

Social network example: Zachary Karate Club 
W. Zachary, “An information flow model for conflict and fission in small groups,” Journal of Anthropological Research, vol. 33, pp. 452–473, 1977
https://en.wikipedia.org/wiki/Zachary's_karate_club
--------------------------------------------------------------------
V = 34
E = 78 bidirectional, 156 directed edges

Bidirectional edges list:
[2 1] [3 1] [3 2] [4 1] [4 2] [4 3] [5 1] [6 1] [7 1] [7 5] [7 6] [8 1] [8 2] [8 3] [8 4] [9 1] [9 3] [10 3] [11 1] [11 5] [11 6] [12 1] [13 1] [13 4] [14 1] [14 2] [14 3] [14 4] [17 6] [17 7] 
[18 1] [18 2] [20 1] [20 2] [22 1] [22 2] [26 24] [26 25] [28 3] [28 24] [28 25] [29 3] [30 24] [30 27] [31 2] [31 9] [32 1] [32 25] [32 26] [32 29] [33 3] [33 9] [33 15] [33 16] 
[33 19] [33 21] [33 23] [33 24] [33 30] [33 31] [33 32] [34 9] [34 10] [34 14] [34 15] [34 16] [34 19] [34 20] [34 21] [34 23] [34 24] [34 27] [34 28] [34 29] [34 30] [34 31] 
[34 32] [34 33]

CSR representation (directed):
csrRowPtrA_h {0, 16, 25, 35, 41, 44, 48, 52, 56, 61, 63, 66, 67, 69, 74, 76, 78, 80, 82, 84, 87, 89, 91, 93, 98, 101, 104, 106, 110, 113, 117, 121, 127, 139, 156}
csrColIndA_h {1, 2, 3, 4, 5, 6, 7, 8, 10, 11, 12, 13, 17, 19, 21, 31, 0, 2, 3, 7, 13, 17, 19, 21, 30, 0, 1, 3, 7, 8, 9, 13, 27, 28, 32, 0, 1, 2, 7, 12, 13, 0, 6, 10, 0, 6, 10, 16, 0, 
4, 5, 16, 0, 1, 2, 3, 0, 2, 30, 32, 33, 2, 33, 0, 4, 5, 0, 0, 3, 0, 1, 2, 3, 33, 32, 33, 32, 33, 5, 6, 0, 1, 32, 33, 0, 1, 33, 32, 33, 0, 1, 32, 33, 25, 27, 29, 32, 33, 25, 27, 31, 23, 
24, 31, 29, 33, 2, 23, 24, 33, 2, 31, 33, 23, 26, 32, 33, 1, 8, 32, 33, 0, 24, 25, 28, 32, 33, 2, 8, 14, 15, 18, 20, 22, 23, 29, 30, 31, 33, 8, 9, 13, 14, 15, 18, 19, 20, 22, 23, 
26, 27, 28, 29, 30, 31, 32}
csrValA_h {1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0}

--------------------------------------------------------------------

Operation: Louvain Clustering  default parameters in modularity maximization

--------------------------------------------------------------------

Expected output: 
This sample prints the modlarity score and compare against the python reference (https://python-louvain.readthedocs.io/en/latest/api.html)


*/

using namespace nvlouvain;

void check_status(nvlouvainStatus_t status)
{
    if ((int)status != 0)
    {
        printf("ERROR : %s\n",nvlouvainStatusGetString(status));
        exit(0);
    }
}

int main(int argc, char **argv)
{
    // Hard-coded Zachary Karate Club network input
    int csrRowPtrA_input [] = {0, 16, 25, 35, 41, 44, 48, 52, 56, 61, 63, 66, 67, 69, 74, 76, 78, 80, 82, 84, 87, 89, 91, 93, 98, 101, 104, 106, 110, 113, 117, 121, 127, 
        139, 156};
    int csrColIndA_input [] = {1, 2, 3, 4, 5, 6, 7, 8, 10, 11, 12, 13, 17, 19, 21, 31, 0, 2, 3, 7, 13, 17, 19, 21, 30, 0, 1, 3, 7, 8, 9, 13, 27, 28, 32, 0, 1, 2, 7, 12, 13, 0, 6, 10, 0, 
        6, 10, 16, 0, 4, 5, 16, 0, 1, 2, 3, 0, 2, 30, 32, 33, 2, 33, 0, 4, 5, 0, 0, 3, 0, 1, 2, 3, 33, 32, 33, 32, 33, 5, 6, 0, 1, 32, 33, 0, 1, 33, 32, 33, 0, 1, 32, 33, 25, 27, 29, 32, 33, 
        25, 27, 31, 23, 24, 31, 29, 33, 2, 23, 24, 33, 2, 31, 33, 23, 26, 32, 33, 1, 8, 32, 33, 0, 24, 25, 28, 32, 33, 2, 8, 14, 15, 18, 20, 22, 23, 29, 30, 31, 33, 8, 9, 13, 14, 15, 
        18, 19, 20, 22, 23, 26, 27, 28, 29, 30, 31, 32};
    float csrValA_input [] = {1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
        1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
        1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
        1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 
        1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0};
    int ref_clustering [] = {0, 0, 0, 0, 1, 1, 1, 0, 2, 0, 1, 0, 0, 0, 2, 2, 1, 0, 2, 0, 2, 0, 2, 3, 3, 3, 2, 3, 3, 2, 2, 3, 2, 2};
    int *csrRowPtrA_h = &csrRowPtrA_input[0];
    int *csrColIndA_h = &csrColIndA_input[0];
    float *csrValA_h = &csrValA_input[0];
    
    // Variables
    const size_t  n = 34, nnz = 156;
    bool weighted = false;
    bool has_init_cluster = false;    
    int *clustering_h, *init_cluster_ptr = nullptr;;
    int num_levels = 0, hits =0;
    float final_modulartiy = 0; 
    // Allocate host data for nvgraphSpectralClustering output
    clustering_h = (int*)malloc(n*sizeof(int));
     
    //Solve clustering with modularity maximization algorithm
    check_status(louvain<int,float>(csrRowPtrA_h, csrColIndA_h, csrValA_h, n, nnz, weighted, has_init_cluster, init_cluster_ptr, final_modulartiy, clustering_h, num_levels));

    //Print quality (modualrity)
    printf("Modularity_score: %f\n", final_modulartiy);
    printf("num levels: %d\n", num_levels);
    for (int i = 0; i < (int)n; i++)
        if (clustering_h[i] == ref_clustering[i])
            hits++;
    printf("Hit rate : %f%% (%d hits)\n", (hits*100.0)/n, hits);
    // Print the clustering vector in csv format
    //for (int i = 0; i < (int)(n-1); i++)
    //    printf("%d,",clustering_h[i]);
    //printf("%d,\n",clustering_h[n-1]);
    free(clustering_h);
    printf("Done!\n");

    return EXIT_SUCCESS;
}

