#include "hip/hip_runtime.h"
// -*-c++-*-

/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "gtest/gtest.h"
#include "gmock/gmock.h"

#include "hip/hip_runtime_api.h"

#include "renumber.cuh"

#include <chrono>

#include <hiprand/hiprand_kernel.h>


struct RenumberingTest : public ::testing::Test
{
};

__global__ void display_list(const char *label, uint32_t *verts, size_t length) {

  printf("%s\n", label);

  for (size_t i = 0 ; i < length ; ++i) {
    printf("  %lu\n", verts[i]);
  }
}

__global__ void setup_generator(hiprandState *state) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(43, id, 0, &state[id]);
}

__global__ void generate_sources(hiprandState *state, int n, uint32_t *verts) {
  int first = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  hiprandState local_state = state[first];
  for (int id = first ; id < n ; id += stride) {
    verts[id] = hiprand(&local_state);
  }

  state[first] = local_state;
}
  
__global__ void generate_destinations(hiprandState *state, int n, const uint32_t *sources, uint32_t *destinations) {
  int first = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  hiprandState local_state = state[first];
  for (int id = first ; id < n ; id += stride) {
    destinations[id] = sources[hiprand(&local_state) % n];
  }

  state[first] = local_state;
}

TEST_F(RenumberingTest, SmallFixedVertexList)
{
  uint32_t src_data[] = { 4U,  6U,  8U, 20U,  1U };
  uint32_t dst_data[] = { 1U, 29U, 35U,  0U, 77U };

  uint32_t src_expected[] = { 2U, 3U, 4U, 5U, 1U };
  uint32_t dst_expected[] = { 1U, 6U, 7U, 0U, 8U };

  size_t length = sizeof(src_data) / sizeof(src_data[0]);

  uint32_t *src_d;
  uint32_t *dst_d;
  uint32_t *number_map_d;

  uint32_t tmp_results[length];
  uint32_t tmp_map[2 * length];

  EXPECT_EQ(hipMalloc(&src_d, sizeof(uint32_t) * length), hipSuccess);
  EXPECT_EQ(hipMalloc(&dst_d, sizeof(uint32_t) * length), hipSuccess);

  EXPECT_EQ(hipMemcpy(src_d, src_data, sizeof(uint32_t) * length, hipMemcpyHostToDevice), hipSuccess);
  EXPECT_EQ(hipMemcpy(dst_d, dst_data, sizeof(uint32_t) * length, hipMemcpyHostToDevice), hipSuccess);

  size_t unique_verts = 0;
  EXPECT_EQ(cugraph::renumber_vertices(length, src_d, dst_d, src_d, dst_d, &unique_verts, &number_map_d), GDF_SUCCESS);

  EXPECT_EQ(hipMemcpy(tmp_map, number_map_d, sizeof(uint32_t) * unique_verts, hipMemcpyDeviceToHost), hipSuccess);
  EXPECT_EQ(hipMemcpy(tmp_results, src_d, sizeof(uint32_t) * length, hipMemcpyDeviceToHost), hipSuccess);

  for (size_t i = 0 ; i < length ; ++i) {
    EXPECT_EQ(tmp_results[i], src_expected[i]);
    EXPECT_EQ(tmp_map[tmp_results[i]], src_data[i]);
  }

  EXPECT_EQ(hipMemcpy(tmp_results, dst_d, sizeof(uint32_t) * length, hipMemcpyDeviceToHost), hipSuccess);
  for (size_t i = 0 ; i < length ; ++i) {
    EXPECT_EQ(tmp_results[i], dst_expected[i]);
    EXPECT_EQ(tmp_map[tmp_results[i]], dst_data[i]);
  }

  EXPECT_EQ(hipFree(src_d), hipSuccess);
  EXPECT_EQ(hipFree(dst_d), hipSuccess);
  EXPECT_EQ(hipFree(number_map_d), hipSuccess);
}

TEST_F(RenumberingTest, SmallFixedVertexList64Bit)
{
  uint64_t src_data[] = { 4U,  6U,  8U, 20U,  1U };
  uint64_t dst_data[] = { 1U, 29U, 35U,  0U, 77U };

  uint64_t src_expected[] = { 2U, 3U, 4U, 5U, 1U };
  uint64_t dst_expected[] = { 1U, 6U, 7U, 0U, 8U };

  size_t length = sizeof(src_data) / sizeof(src_data[0]);

  uint64_t *src_d;
  uint64_t *dst_d;
  uint64_t *number_map_d;

  uint64_t tmp_results[length];
  uint64_t tmp_map[2 * length];

  EXPECT_EQ(hipMalloc(&src_d, sizeof(uint64_t) * length), hipSuccess);
  EXPECT_EQ(hipMalloc(&dst_d, sizeof(uint64_t) * length), hipSuccess);

  EXPECT_EQ(hipMemcpy(src_d, src_data, sizeof(uint64_t) * length, hipMemcpyHostToDevice), hipSuccess);
  EXPECT_EQ(hipMemcpy(dst_d, dst_data, sizeof(uint64_t) * length, hipMemcpyHostToDevice), hipSuccess);

  size_t unique_verts = 0;
  EXPECT_EQ(cugraph::renumber_vertices(length, src_d, dst_d, src_d, dst_d, &unique_verts, &number_map_d), GDF_SUCCESS);

  EXPECT_EQ(hipMemcpy(tmp_map, number_map_d, sizeof(uint64_t) * unique_verts, hipMemcpyDeviceToHost), hipSuccess);
  EXPECT_EQ(hipMemcpy(tmp_results, src_d, sizeof(uint64_t) * length, hipMemcpyDeviceToHost), hipSuccess);

  for (size_t i = 0 ; i < length ; ++i) {
    EXPECT_EQ(tmp_results[i], src_expected[i]);
    EXPECT_EQ(tmp_map[tmp_results[i]], src_data[i]);
  }

  EXPECT_EQ(hipMemcpy(tmp_results, dst_d, sizeof(uint64_t) * length, hipMemcpyDeviceToHost), hipSuccess);
  for (size_t i = 0 ; i < length ; ++i) {
    EXPECT_EQ(tmp_results[i], dst_expected[i]);
    EXPECT_EQ(tmp_map[tmp_results[i]], dst_data[i]);
  }

  EXPECT_EQ(hipFree(src_d), hipSuccess);
  EXPECT_EQ(hipFree(dst_d), hipSuccess);
  EXPECT_EQ(hipFree(number_map_d), hipSuccess);
}

TEST_F(RenumberingTest, SmallFixedVertexList64BitTo32Bit)
{
  uint64_t src_data[] = { 4U,  6U,  8U, 20U,  1U };
  uint64_t dst_data[] = { 1U, 29U, 35U,  0U, 77U };

  uint32_t src_expected[] = { 2U, 3U, 4U, 5U, 1U };
  uint32_t dst_expected[] = { 1U, 6U, 7U, 0U, 8U };

  size_t length = sizeof(src_data) / sizeof(src_data[0]);

  uint64_t *src_d;
  uint64_t *dst_d;
  uint32_t *src_renumbered_d;
  uint32_t *dst_renumbered_d;
  uint64_t *number_map_d;

  uint32_t tmp_results[length];
  uint64_t tmp_map[2 * length];

  EXPECT_EQ(hipMalloc(&src_d, sizeof(uint64_t) * length), hipSuccess);
  EXPECT_EQ(hipMalloc(&dst_d, sizeof(uint64_t) * length), hipSuccess);
  EXPECT_EQ(hipMalloc(&src_renumbered_d, sizeof(uint32_t) * length), hipSuccess);
  EXPECT_EQ(hipMalloc(&dst_renumbered_d, sizeof(uint32_t) * length), hipSuccess);

  EXPECT_EQ(hipMemcpy(src_d, src_data, sizeof(uint64_t) * length, hipMemcpyHostToDevice), hipSuccess);
  EXPECT_EQ(hipMemcpy(dst_d, dst_data, sizeof(uint64_t) * length, hipMemcpyHostToDevice), hipSuccess);

  size_t unique_verts = 0;
  EXPECT_EQ(cugraph::renumber_vertices(length, src_d, dst_d, src_renumbered_d, dst_renumbered_d, &unique_verts, &number_map_d), GDF_SUCCESS);

  EXPECT_EQ(hipMemcpy(tmp_map, number_map_d, sizeof(uint64_t) * unique_verts, hipMemcpyDeviceToHost), hipSuccess);
  EXPECT_EQ(hipMemcpy(tmp_results, src_renumbered_d, sizeof(uint32_t) * length, hipMemcpyDeviceToHost), hipSuccess);

  for (size_t i = 0 ; i < length ; ++i) {
    EXPECT_EQ(tmp_results[i], src_expected[i]);
    EXPECT_EQ(tmp_map[tmp_results[i]], src_data[i]);
  }

  EXPECT_EQ(hipMemcpy(tmp_results, dst_renumbered_d, sizeof(uint32_t) * length, hipMemcpyDeviceToHost), hipSuccess);
  for (size_t i = 0 ; i < length ; ++i) {
    EXPECT_EQ(tmp_results[i], dst_expected[i]);
    EXPECT_EQ(tmp_map[tmp_results[i]], dst_data[i]);
  }

  EXPECT_EQ(hipFree(src_d), hipSuccess);
  EXPECT_EQ(hipFree(dst_d), hipSuccess);
  EXPECT_EQ(hipFree(number_map_d), hipSuccess);
}

TEST_F(RenumberingTest, Random100KVertexSet)
{
  const int num_verts = 100000;

  uint64_t *src_d;
  uint64_t *dst_d;
  uint64_t *number_map_d;

  uint64_t *src_data    = (uint64_t *) malloc(num_verts * sizeof(uint64_t));
  uint64_t *dst_data    = (uint64_t *) malloc(num_verts * sizeof(uint64_t));
  uint64_t *tmp_results = (uint64_t *) malloc(num_verts * sizeof(uint64_t));
  uint64_t *tmp_map     = (uint64_t *) malloc(2 * num_verts * sizeof(uint64_t));

  EXPECT_EQ(hipMalloc(&src_d, sizeof(uint64_t) * num_verts), hipSuccess);
  EXPECT_EQ(hipMalloc(&dst_d, sizeof(uint64_t) * num_verts), hipSuccess);

  //
  //  Generate random source and vertex values
  //
  srand(43);

  for (int i = 0 ; i < num_verts ; ++i) {
    src_data[i] = (uint64_t) rand();
  }

  for (int i = 0 ; i < num_verts ; ++i) {
    dst_data[i] = (uint64_t) rand();
  }

  EXPECT_EQ(hipMemcpy(src_d, src_data, sizeof(uint64_t) * num_verts, hipMemcpyHostToDevice), hipSuccess);
  EXPECT_EQ(hipMemcpy(dst_d, dst_data, sizeof(uint64_t) * num_verts, hipMemcpyHostToDevice), hipSuccess);

  //
  //  Renumber everything
  //
  size_t unique_verts = 0;

  auto start = std::chrono::system_clock::now();
  EXPECT_EQ(cugraph::renumber_vertices(num_verts, src_d, dst_d, src_d, dst_d, &unique_verts, &number_map_d), GDF_SUCCESS);
  auto end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end-start;

  std::cout << "Renumber kernel elapsed time (ms): " << elapsed_seconds.count()*1000 << std::endl;


  EXPECT_EQ(hipMemcpy(tmp_map, number_map_d, sizeof(uint64_t) * unique_verts, hipMemcpyDeviceToHost), hipSuccess);
  EXPECT_EQ(hipMemcpy(tmp_results, src_d, sizeof(uint64_t) * num_verts, hipMemcpyDeviceToHost), hipSuccess);

  size_t min_id = unique_verts;
  size_t max_id = 0;

  for (size_t i = 0 ; i < num_verts ; ++i) {
    min_id = min(min_id, tmp_results[i]);
    max_id = max(max_id, tmp_results[i]);
    EXPECT_EQ(tmp_map[tmp_results[i]], src_data[i]);
  }

  EXPECT_EQ(hipMemcpy(tmp_results, dst_d, sizeof(uint64_t) * num_verts, hipMemcpyDeviceToHost), hipSuccess);
  for (size_t i = 0 ; i < num_verts ; ++i) {
    min_id = min(min_id, tmp_results[i]);
    max_id = max(max_id, tmp_results[i]);
    EXPECT_EQ(tmp_map[tmp_results[i]], dst_data[i]);
  }

  EXPECT_EQ(min_id, 0);
  EXPECT_EQ(max_id, (unique_verts - 1));
  EXPECT_EQ(hipFree(src_d), hipSuccess);
  EXPECT_EQ(hipFree(dst_d), hipSuccess);
  EXPECT_EQ(hipFree(number_map_d), hipSuccess);
  free(src_data);
  free(dst_data);
  free(tmp_results);
  free(tmp_map);
}

TEST_F(RenumberingTest, Random10MVertexSet)
{
  const int num_verts = 10000000;

  //  A sampling of performance on aschaffer-DGX-Station
  //const int hash_size =  33554467;  // 907 ms
  //const int hash_size =  3355453;   // 743 ms
  //const int hash_size =  335557;    // 719 ms
  const int hash_size =  32767;       // 515 ms
  //const int hash_size =  8191;      // 633 ms

  uint32_t *src_d;
  uint32_t *dst_d;
  uint32_t *number_map_d;

  EXPECT_EQ(hipMalloc(&src_d, sizeof(uint32_t) * num_verts), hipSuccess);
  EXPECT_EQ(hipMalloc(&dst_d, sizeof(uint32_t) * num_verts), hipSuccess);

  //
  //  Init the random number generate
  //
  const int num_threads{64};
  hiprandState *state;

  EXPECT_EQ(hipMalloc(&state, sizeof(hiprandState) * num_threads), hipSuccess);
  setup_generator<<<num_threads,1>>>(state);
  generate_sources<<<num_threads,1>>>(state, num_verts, src_d);
  generate_destinations<<<num_threads,1>>>(state, num_verts, src_d, dst_d);

  std::cout << "done with initialization" << std::endl;

  //
  //  Renumber everything
  //
  size_t unique_verts = 0;
  auto start = std::chrono::system_clock::now();
  EXPECT_EQ(cugraph::renumber_vertices(num_verts, src_d, dst_d, src_d, dst_d, &unique_verts, &number_map_d, 64, 64, hash_size), GDF_SUCCESS);
  auto end = std::chrono::system_clock::now();
  std::chrono::duration<double> elapsed_seconds = end-start;

  std::cout << "Renumber kernel elapsed time (ms): " << elapsed_seconds.count()*1000 << std::endl;
  std::cout << "  unique verts = " << unique_verts << std::endl;
  std::cout << "  hash size = " << hash_size << std::endl;

  EXPECT_EQ(hipFree(src_d), hipSuccess);
  EXPECT_EQ(hipFree(dst_d), hipSuccess);
  EXPECT_EQ(hipFree(number_map_d), hipSuccess);
}
