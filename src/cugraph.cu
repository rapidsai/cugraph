 /*
 * Copyright (c) 2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 */

// Graph analytics features 
// Author: Alex Fender afender@nvidia.com

#include <cugraph.h>
#include "graph_utils.cuh"
#include "pagerank.cuh"
#include "COOtoCSR.cuh"
#include "utilities/error_utils.h"

#include <rmm_utils.h>

void gdf_col_delete(gdf_column* col) {
  if (col)
  {
    col->size = 0; 
    if(col->data)
        {
        ALLOC_FREE_TRY(col->data, nullptr);
        }
    delete col;
    col->data = nullptr;
    col = nullptr;  
  }                                                       
}

void gdf_col_release(gdf_column* col) {
  delete col;                                                          
}

void cpy_column_view(const gdf_column *in, gdf_column *out) {
  if (in != nullptr && out !=nullptr) {
    gdf_column_view(out, in->data, in->valid, in->size, in->dtype);
  }
}

gdf_error gdf_adj_list_view(gdf_graph *graph, const gdf_column *offsets, 
                                 const gdf_column *indices, const gdf_column *edge_data) {
  GDF_REQUIRE( offsets->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );                    
  GDF_REQUIRE( indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( (offsets->dtype == indices->dtype), GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( ((offsets->dtype == GDF_INT32) || (offsets->dtype == GDF_INT64)), GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( (offsets->size > 0), GDF_DATASET_EMPTY ); 
  GDF_REQUIRE( (graph->adjList == nullptr) , GDF_INVALID_API_CALL);

  graph->adjList = new gdf_adj_list;
  graph->adjList->offsets = new gdf_column;
  graph->adjList->indices = new gdf_column;
  graph->adjList->ownership = 0;

  cpy_column_view(offsets, graph->adjList->offsets);
  cpy_column_view(indices, graph->adjList->indices);
  if (edge_data) {
      GDF_REQUIRE( indices->size == edge_data->size, GDF_COLUMN_SIZE_MISMATCH );
      graph->adjList->edge_data = new gdf_column;
      cpy_column_view(edge_data, graph->adjList->edge_data);
  }
  else {
    graph->adjList->edge_data = nullptr;
  }
  return GDF_SUCCESS;
}

gdf_error gdf_adj_list::get_vertex_identifiers(gdf_column *identifiers) {
  GDF_REQUIRE( offsets != nullptr , GDF_INVALID_API_CALL);
  GDF_REQUIRE( offsets->data != nullptr , GDF_INVALID_API_CALL);
  cugraph::sequence<int>((int)offsets->size-1, (int*)identifiers->data);
  return GDF_SUCCESS;
}

gdf_error gdf_adj_list::get_source_indices (gdf_column *src_indices) {
  GDF_REQUIRE( offsets != nullptr , GDF_INVALID_API_CALL);
  GDF_REQUIRE( offsets->data != nullptr , GDF_INVALID_API_CALL);
  GDF_REQUIRE( src_indices->size == indices->size, GDF_COLUMN_SIZE_MISMATCH );
  GDF_REQUIRE( src_indices->dtype == indices->dtype, GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( src_indices->size > 0, GDF_DATASET_EMPTY ); 
  cugraph::offsets_to_indices<int>((int*)offsets->data, offsets->size-1, (int*)src_indices->data);

  return GDF_SUCCESS;
}

gdf_error gdf_edge_list_view(gdf_graph *graph, const gdf_column *src_indices, 
                                 const gdf_column *dest_indices, const gdf_column *edge_data) {
  GDF_REQUIRE( src_indices->size == dest_indices->size, GDF_COLUMN_SIZE_MISMATCH );
  GDF_REQUIRE( src_indices->dtype == dest_indices->dtype, GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( ((src_indices->dtype == GDF_INT32) || (src_indices->dtype == GDF_INT64)), GDF_UNSUPPORTED_DTYPE );
  GDF_REQUIRE( src_indices->size > 0, GDF_DATASET_EMPTY ); 
  GDF_REQUIRE( src_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );                    
  GDF_REQUIRE( dest_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( graph->edgeList == nullptr , GDF_INVALID_API_CALL);

  graph->edgeList = new gdf_edge_list;
  graph->edgeList->src_indices = new gdf_column;
  graph->edgeList->dest_indices = new gdf_column;
  graph->edgeList->ownership = 0;

  cpy_column_view(src_indices, graph->edgeList->src_indices);
  cpy_column_view(dest_indices, graph->edgeList->dest_indices);
  if (edge_data) {
      GDF_REQUIRE( src_indices->size == edge_data->size, GDF_COLUMN_SIZE_MISMATCH );
      graph->edgeList->edge_data = new gdf_column;
      cpy_column_view(edge_data, graph->edgeList->edge_data);
  }
  else {
    graph->edgeList->edge_data = nullptr;
  }
  return GDF_SUCCESS;
}

template <typename WT>
gdf_error gdf_add_adj_list_impl (gdf_graph *graph) {
    if (graph->adjList == nullptr) {
      GDF_REQUIRE( graph->edgeList != nullptr , GDF_INVALID_API_CALL);
      int nnz = graph->edgeList->src_indices->size, status = 0;
      graph->adjList = new gdf_adj_list;
      graph->adjList->offsets = new gdf_column;
      graph->adjList->indices = new gdf_column;
      graph->adjList->ownership = 1;

    if (graph->edgeList->edge_data!= nullptr) {
      graph->adjList->edge_data = new gdf_column;

      CSR_Result_Weighted<int,WT> adj_list;
      status = ConvertCOOtoCSR_weighted((int*)graph->edgeList->src_indices->data, (int*)graph->edgeList->dest_indices->data, (WT*)graph->edgeList->edge_data->data, nnz, adj_list);
      
      gdf_column_view(graph->adjList->offsets, adj_list.rowOffsets, 
                            nullptr, adj_list.size+1, graph->edgeList->src_indices->dtype);
      gdf_column_view(graph->adjList->indices, adj_list.colIndices, 
                            nullptr, adj_list.nnz, graph->edgeList->src_indices->dtype);
      gdf_column_view(graph->adjList->edge_data, adj_list.edgeWeights, 
                          nullptr, adj_list.nnz, graph->edgeList->edge_data->dtype);
    }
    else {
      CSR_Result<int> adj_list;
      status = ConvertCOOtoCSR((int*)graph->edgeList->src_indices->data,(int*)graph->edgeList->dest_indices->data, nnz, adj_list);      
      gdf_column_view(graph->adjList->offsets, adj_list.rowOffsets, 
                            nullptr, adj_list.size+1, graph->edgeList->src_indices->dtype);
      gdf_column_view(graph->adjList->indices, adj_list.colIndices, 
                            nullptr, adj_list.nnz, graph->edgeList->src_indices->dtype);
    }
    if (status !=0) {
      std::cerr << "Could not generate the adj_list" << std::endl;
      return GDF_CUDA_ERROR;
    }
  }
  return GDF_SUCCESS;
}

gdf_error gdf_add_edge_list (gdf_graph *graph) {
    if (graph->edgeList == nullptr) {
      GDF_REQUIRE( graph->adjList != nullptr , GDF_INVALID_API_CALL);
      int *d_src;
      graph->edgeList = new gdf_edge_list;
      graph->edgeList->src_indices = new gdf_column;
      graph->edgeList->dest_indices = new gdf_column;
      graph->edgeList->ownership = 2;


      CUDA_TRY(hipMallocManaged ((void**)&d_src, sizeof(int) * graph->adjList->indices->size));

      cugraph::offsets_to_indices<int>((int*)graph->adjList->offsets->data, 
                                  graph->adjList->offsets->size-1, 
                                  (int*)d_src);

      gdf_column_view(graph->edgeList->src_indices, d_src, 
                      nullptr, graph->adjList->indices->size, graph->adjList->indices->dtype);
      cpy_column_view(graph->adjList->indices, graph->edgeList->dest_indices);
      
      if (graph->adjList->edge_data != nullptr) {
        graph->edgeList->edge_data = new gdf_column;
        cpy_column_view(graph->adjList->edge_data, graph->edgeList->edge_data);
      }
  }
  return GDF_SUCCESS;
}


template <typename WT>
gdf_error gdf_add_transpose_impl (gdf_graph *graph) {
    if (graph->transposedAdjList == nullptr ) {
      GDF_REQUIRE( graph->edgeList != nullptr , GDF_INVALID_API_CALL);
      int nnz = graph->edgeList->src_indices->size, status = 0;
      graph->transposedAdjList = new gdf_adj_list;
      graph->transposedAdjList->offsets = new gdf_column;
      graph->transposedAdjList->indices = new gdf_column;
      graph->transposedAdjList->ownership = 1;
    
      if (graph->edgeList->edge_data) {
        graph->transposedAdjList->edge_data = new gdf_column;
        CSR_Result_Weighted<int,WT> adj_list;
        status = ConvertCOOtoCSR_weighted( (int*)graph->edgeList->dest_indices->data, (int*)graph->edgeList->src_indices->data, (WT*)graph->edgeList->edge_data->data, nnz, adj_list);
        gdf_column_view(graph->transposedAdjList->offsets, adj_list.rowOffsets, 
                              nullptr, adj_list.size+1, graph->edgeList->src_indices->dtype);
        gdf_column_view(graph->transposedAdjList->indices, adj_list.colIndices, 
                              nullptr, adj_list.nnz, graph->edgeList->src_indices->dtype);
        gdf_column_view(graph->transposedAdjList->edge_data, adj_list.edgeWeights, 
                            nullptr, adj_list.nnz, graph->edgeList->edge_data->dtype);
      }
      else {

        CSR_Result<int> adj_list;
        status = ConvertCOOtoCSR((int*)graph->edgeList->dest_indices->data, (int*)graph->edgeList->src_indices->data, nnz, adj_list);      
        gdf_column_view(graph->transposedAdjList->offsets, adj_list.rowOffsets, 
                              nullptr, adj_list.size+1, graph->edgeList->src_indices->dtype);
        gdf_column_view(graph->transposedAdjList->indices, adj_list.colIndices, 
                              nullptr, adj_list.nnz, graph->edgeList->src_indices->dtype);
      }
      if (status !=0) {
        std::cerr << "Could not generate the adj_list" << std::endl;
        return GDF_CUDA_ERROR;
      }
    }
    return GDF_SUCCESS;
}

template <typename WT>
gdf_error gdf_pagerank_impl (gdf_graph *graph,
                      gdf_column *pagerank, float alpha = 0.85,
                      float tolerance = 1e-4, int max_iter = 200,
                      bool has_guess = false) {

  
  GDF_REQUIRE( graph->edgeList != nullptr, GDF_VALIDITY_UNSUPPORTED );
  GDF_REQUIRE( graph->edgeList->src_indices->size == graph->edgeList->dest_indices->size, GDF_COLUMN_SIZE_MISMATCH ); 
  GDF_REQUIRE( graph->edgeList->src_indices->dtype == graph->edgeList->dest_indices->dtype, GDF_UNSUPPORTED_DTYPE );  
  GDF_REQUIRE( graph->edgeList->src_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );                 
  GDF_REQUIRE( graph->edgeList->dest_indices->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );  
  GDF_REQUIRE( pagerank != nullptr , GDF_INVALID_API_CALL ); 
  GDF_REQUIRE( pagerank->data != nullptr , GDF_INVALID_API_CALL ); 
  GDF_REQUIRE( pagerank->null_count == 0 , GDF_VALIDITY_UNSUPPORTED );          
  GDF_REQUIRE( pagerank->size > 0 , GDF_INVALID_API_CALL );         

  int m=pagerank->size, nnz = graph->edgeList->src_indices->size, status = 0;
  WT *d_pr, *d_val = nullptr, *d_leaf_vector = nullptr; 
  WT res = 1.0;
  WT *residual = &res;

  if (graph->transposedAdjList == nullptr) {
    gdf_add_transpose(graph);
  }
  hipStream_t stream{nullptr};
  ALLOC_MANAGED_TRY((void**)&d_leaf_vector, sizeof(WT) * m, stream);
  ALLOC_MANAGED_TRY((void**)&d_val, sizeof(WT) * nnz , stream);
  ALLOC_MANAGED_TRY((void**)&d_pr,    sizeof(WT) * m, stream);

  cugraph::HT_matrix_csc_coo(m, nnz, (int*)graph->transposedAdjList->offsets->data, (int*)graph->transposedAdjList->indices->data, d_val, d_leaf_vector);

  if (has_guess)
  {
    GDF_REQUIRE( pagerank->data != nullptr, GDF_VALIDITY_UNSUPPORTED );
    cugraph::copy<WT>(m, (WT*)pagerank->data, d_pr);
  }

  status = cugraph::pagerank<int,WT>( m,nnz, (int*)graph->transposedAdjList->offsets->data, (int*)graph->transposedAdjList->indices->data, 
    d_val, alpha, d_leaf_vector, false, tolerance, max_iter, d_pr, residual);
 
  if (status !=0)
    switch ( status ) { 
      case -1: std::cerr<< "Error : bad parameters in Pagerank"<<std::endl; return GDF_CUDA_ERROR; 
      case 1: std::cerr<< "Warning : Pagerank did not reached the desired tolerance"<<std::endl;  return GDF_CUDA_ERROR; 
      default:  std::cerr<< "Pagerank failed"<<std::endl;  return GDF_CUDA_ERROR; 
    }   
 
  cugraph::copy<WT>(m, d_pr, (WT*)pagerank->data);

  ALLOC_FREE_TRY(d_val, stream);
  ALLOC_FREE_TRY(d_pr, stream);
  ALLOC_FREE_TRY(d_leaf_vector, stream);

  return GDF_SUCCESS;
}


gdf_error gdf_add_adj_list(gdf_graph *graph)
{ 
  GDF_REQUIRE( graph->edgeList != nullptr , GDF_INVALID_API_CALL);
  GDF_REQUIRE( graph->adjList == nullptr , GDF_INVALID_API_CALL);

  if (graph->edgeList->edge_data != nullptr) {
    switch (graph->edgeList->edge_data->dtype) {
      case GDF_FLOAT32:   return gdf_add_adj_list_impl<float>(graph);
      case GDF_FLOAT64:   return gdf_add_adj_list_impl<double>(graph);
      default: return GDF_UNSUPPORTED_DTYPE;
    }
  }
  else {
    return gdf_add_adj_list_impl<float>(graph);
  }
}

gdf_error gdf_add_transpose(gdf_graph *graph)
{
  if (graph->edgeList->edge_data != nullptr) {
    switch (graph->edgeList->edge_data->dtype) {
      case GDF_FLOAT32:   return gdf_add_transpose_impl<float>(graph);
      case GDF_FLOAT64:   return gdf_add_transpose_impl<double>(graph);
      default: return GDF_UNSUPPORTED_DTYPE;
    }
  }
  else {
    return gdf_add_transpose_impl<float>(graph);
  }
}

gdf_error gdf_delete_adj_list(gdf_graph *graph) {
  if (graph->adjList) {
    graph->adjList->ownership = 1;
    delete graph->adjList;
  }
  graph->adjList = nullptr;
  return GDF_SUCCESS;
}
gdf_error gdf_delete_edge_list(gdf_graph *graph) {
  if (graph->edgeList) {
    graph->edgeList->ownership = 1;
    delete graph->edgeList;
  }
  graph->edgeList = nullptr;
  return GDF_SUCCESS;
}
gdf_error gdf_delete_transpose(gdf_graph *graph) {
  if (graph->transposedAdjList) {
    graph->transposedAdjList->ownership = 1;
    delete graph->transposedAdjList;
  }
  graph->transposedAdjList = nullptr;
  return GDF_SUCCESS;
}

gdf_error gdf_pagerank(gdf_graph *graph, gdf_column *pagerank, float alpha, float tolerance, int max_iter, bool has_guess)
{ 
  switch (pagerank->dtype) {
    case GDF_FLOAT32:   return gdf_pagerank_impl<float>(graph, pagerank, alpha, tolerance, max_iter, has_guess);
    case GDF_FLOAT64:   return gdf_pagerank_impl<double>(graph, pagerank, alpha, tolerance, max_iter, has_guess);
    default: return GDF_UNSUPPORTED_DTYPE;
  }
}
